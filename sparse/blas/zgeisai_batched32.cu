#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/
#include "magmasparse_internal.h"
#include <hip/hip_runtime_api.h>

#define PRECISION_z
#define COMPLEX
#define BLOCKSIZE 32
#define WARP_SIZE 32
#define WRP 32
#define WRQ 4

#include <hip/hip_runtime.h>  // for CUDA_VERSION

#if (CUDA_VERSION >= 7000) // only for cuda>6000


const int MaxBlockSize = 32;


template <int block_size>
__device__ void
magma_zlowerisai_regs_kernel(
magma_int_t num_rows,
const magma_index_t * __restrict__ Arow,
const magma_index_t * __restrict__ Acol,
const magmaDoubleComplex * __restrict__ Aval,
magma_index_t *Mrow,
magma_index_t *Mcol,
magmaDoubleComplex *Mval )
{
#if (defined( REAL ) && ( __CUDA_ARCH__ >= 300 ))
    int tid = threadIdx.x;
    int row = gridDim.x*blockIdx.y*blockDim.y + blockIdx.x*blockDim.y + threadIdx.y;

    if( tid >= block_size )
        return;

    if( row >= num_rows )
        return;

    // only if within the size
    int mstart = Mrow[ row ];
    int mlim = Mrow[ row+1 ];

    magmaDoubleComplex rB;      // registers for trsv
    magmaDoubleComplex dA[ block_size ];  // registers for trisystem
    magmaDoubleComplex rA;

    // set dA to 0
    #pragma unroll
     for( int j = 0; j < block_size; j++ ){
         dA[ j ] = MAGMA_Z_ZERO;
     }

    // generate the triangular systems
    int t = Mcol[ mstart + tid ];
    int k = Arow[ t ];
    int alim = Arow[ t+1 ];
    int l = mstart;
    int idx = 0;
    while( k < alim && l < mlim ){ // stop once this column is done
        int mcol =  Mcol[ l ];
        int acol = Acol[k];
        if( mcol == acol ){ //match
            dA[ idx ] = Aval[ k ];
            k++;
            l++;
            idx++;
        } else if( acol < mcol ){// need to check next element
            k++;
        } else { // element does not exist, i.e. l < LC.col[k]
            l++; // check next elment in the sparsity pattern
            idx++; // leave this element equal zero
        }
    }

    // second: solve the triangular systems - in registers
    // we know how RHS looks like
    rB = ( tid == 0 ) ? MAGMA_Z_ONE : MAGMA_Z_ZERO;


        // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < block_size; k++)
    {
        rA = dA[ k ];
         if (k % block_size == tid)
             rB /= rA;
         magmaDoubleComplex top = __shfl(rB, k % block_size);
         if ( tid > k)
             rB -= (top*rA);
    }

    // Drop B to dev memory - in ISAI preconditioner M
    Mval[ mstart + tid ] = rB;

#endif

}


template <int block_size>
__device__ __forceinline__ void
magma_zlowerisai_regs_select(
int N,
magma_int_t num_rows,
const magma_index_t * __restrict__ Arow,
const magma_index_t * __restrict__ Acol,
const magmaDoubleComplex * __restrict__ Aval,
magma_index_t *Mrow,
magma_index_t *Mcol,
magmaDoubleComplex *Mval )
{
    if (N == block_size) {
        magma_zlowerisai_regs_kernel<block_size>(
                num_rows, Arow, Acol, Aval, Mrow, Mcol, Mval);
    } else {
        magma_zlowerisai_regs_select<block_size-1>(
                N, num_rows, Arow, Acol, Aval, Mrow, Mcol, Mval);
    }
}


/*
template <int block_size, template <int> class func>
class Switcher {
public:
    static __device__ void
    switch_func(
            int N,
            magma_int_t num_rows,
            const magma_index_t * __restrict__ Arow,
            const magma_index_t * __restrict__ Acol,
            const magmaDoubleComplex * __restrict__ Aval,
            magma_index_t *Mrow,
            magma_index_t *Mcol,
            magmaDoubleComplex *Mval )
    {
        if (N == block_size) {
            func<block_size>(num_rows, Arow, Acol, Aval, Mrow, Mcol, Mval);
        } else {
            Switcher<block_size-1,func>::switch_func(
                    N, num_rows, Arow, Acol, Aval, Mrow, Mcol, Mval);
        }
    }

};

template<template <int> class func>
class Switcher<0, func> {
public:
    static __device__ void
    switch_func(
            int N,
            magma_int_t num_rows,
            const magma_index_t * __restrict__ Arow,
            const magma_index_t * __restrict__ Acol,
            const magmaDoubleComplex * __restrict__ Aval,
            magma_index_t *Mrow,
            magma_index_t *Mcol,
            magmaDoubleComplex *Mval )
    {
        // TODO(Hartwig): Are you soure we want to have printfs called from the
        //                device?
        printf("%% error: size out of range: %d\n", N);
    }
};
*/

template <>
__device__ __forceinline__ void
magma_zlowerisai_regs_select<0>(
int N,
magma_int_t num_rows,
const magma_index_t * __restrict__ Arow,
const magma_index_t * __restrict__ Acol,
const magmaDoubleComplex * __restrict__ Aval,
magma_index_t *Mrow,
magma_index_t *Mcol,
magmaDoubleComplex *Mval )
{
    // TODO(Hartwig): Are you soure we want to have printfs called from the
    //                device?
    printf("%% error: size out of range: %d\n", N);
}


__global__ void
magma_zlowerisai_regs_switch(
magma_int_t num_rows,
const magma_index_t * __restrict__ Arow,
const magma_index_t * __restrict__ Acol,
const magmaDoubleComplex * __restrict__ Aval,
magma_index_t *Mrow,
magma_index_t *Mcol,
magmaDoubleComplex *Mval )
{
    int row = gridDim.x*blockIdx.y*blockDim.y + blockIdx.x*blockDim.y + threadIdx.y;
    if( row < num_rows ){
        int N = Mrow[ row+1 ] - Mrow[ row ];
        //Switcher<MaxBlockSize, magma_zlowerisai_regs_kernel>::switch_func(
        magma_zlowerisai_regs_select<MaxBlockSize>(
                N, num_rows, Arow, Acol, Aval, Mrow, Mcol, Mval);
    }
}


template <int block_size>
__device__ void
magma_zupperisai_regs_kernel(
magma_int_t num_rows,
const magma_index_t * __restrict__ Arow,
const magma_index_t * __restrict__ Acol,
const magmaDoubleComplex * __restrict__ Aval,
magma_index_t *Mrow,
magma_index_t *Mcol,
magmaDoubleComplex *Mval )
{
#if (defined( REAL ) && ( __CUDA_ARCH__ >= 300 ))
    int tid = threadIdx.x;
    int row = gridDim.x*blockIdx.y*blockDim.y + blockIdx.x*blockDim.y + threadIdx.y;

    if( tid >= block_size )
        return;

    if( row >= num_rows )
        return;

    // only if within the size
    int mstart = Mrow[ row ];
    int mlim = Mrow[ row+1 ];

    magmaDoubleComplex rB;      // registers for trsv
    magmaDoubleComplex dA[ block_size ];  // registers for trisystem
    magmaDoubleComplex rA;

    // set dA to 0
    #pragma unroll
     for( int j = 0; j < block_size; j++ ){
         dA[ j ] = MAGMA_Z_ZERO;
     }

    // generate the triangular systems
    int t = Mcol[ mstart + tid ];
    int k = Arow[ t ];
    int alim = Arow[ t+1 ];
    int l = mstart;
    int idx = 0;
    while( k < alim && l < mlim ){ // stop once this column is done
        int mcol =  Mcol[ l ];
        int acol = Acol[k];
        if( mcol == acol ){ //match
            dA[ idx ] = Aval[ k ];
            k++;
            l++;
            idx++;
        } else if( acol < mcol ){// need to check next element
            k++;
        } else { // element does not exist, i.e. l < LC.col[k]
            l++; // check next elment in the sparsity pattern
            idx++; // leave this element equal zero
        }
    }

    // second: solve the triangular systems - in registers
    // we know how RHS looks like
    rB = ( tid == block_size-1 ) ? MAGMA_Z_ONE : MAGMA_Z_ZERO;


        // Triangular solve in regs.
    #pragma unroll
    for (int k = block_size-1; k >-1; k--)
    {
        rA = dA[ k ];
         if (k%block_size == tid)
             rB /= rA;
         magmaDoubleComplex bottom = __shfl(rB, k%block_size);
         if ( tid < k)
             rB -= (bottom*rA);
    }

    // Drop B to dev memory - in ISAI preconditioner M
    Mval[ mstart + tid ] = rB;

#endif

}


template <int block_size>
__device__ __forceinline__ void
magma_zupperisai_regs_select(
int N,
magma_int_t num_rows,
const magma_index_t * __restrict__ Arow,
const magma_index_t * __restrict__ Acol,
const magmaDoubleComplex * __restrict__ Aval,
magma_index_t *Mrow,
magma_index_t *Mcol,
magmaDoubleComplex *Mval )
{
    if (N == block_size) {
        magma_zupperisai_regs_kernel<block_size>(
                num_rows, Arow, Acol, Aval, Mrow, Mcol, Mval);
    } else {
        magma_zupperisai_regs_select<block_size-1>(
                N, num_rows, Arow, Acol, Aval, Mrow, Mcol, Mval);
    }
}


template <>
__device__ __forceinline__ void
magma_zupperisai_regs_select<0>(
int N,
magma_int_t num_rows,
const magma_index_t * __restrict__ Arow,
const magma_index_t * __restrict__ Acol,
const magmaDoubleComplex * __restrict__ Aval,
magma_index_t *Mrow,
magma_index_t *Mcol,
magmaDoubleComplex *Mval )
{
    // TODO(Hartwig): Are you soure we want to have printfs called from the
    //                device?
    printf("%% error: size out of range: %d\n", N);
}


__global__ void
magma_zupperisai_regs_switch(
magma_int_t num_rows,
const magma_index_t * __restrict__ Arow,
const magma_index_t * __restrict__ Acol,
const magmaDoubleComplex * __restrict__ Aval,
magma_index_t *Mrow,
magma_index_t *Mcol,
magmaDoubleComplex *Mval )
{
    int row = gridDim.x*blockIdx.y*blockDim.y + blockIdx.x*blockDim.y + threadIdx.y;
    if( row < num_rows ){
        int N = Mrow[ row+1 ] - Mrow[ row ];
        magma_zupperisai_regs_select<MaxBlockSize>(
                N, num_rows, Arow, Acol, Aval, Mrow, Mcol, Mval);
    }
}


template <int block_size>
__device__ void
magma_zlowerisai_regs_inv_kernel(
magma_int_t num_rows,
const magma_index_t * __restrict__ Arow,
const magma_index_t * __restrict__ Acol,
const magmaDoubleComplex * __restrict__ Aval,
magma_index_t *Mrow,
magma_index_t *Mcol,
magmaDoubleComplex *Mval )
{
#if (defined( REAL ) && ( __CUDA_ARCH__ >= 300 ))
    int tid = threadIdx.x;
    int row = gridDim.x*blockIdx.y*blockDim.y + blockIdx.x*blockDim.y + threadIdx.y;

    if( tid >= block_size )
        return;

    if( row >= num_rows )
        return;

    // only if within the size
    int mstart = Mrow[ row ];
    int mlim = Mrow[ row ]-1;

    magmaDoubleComplex rB;      // registers for trsv
    magmaDoubleComplex dA[ block_size ];  // registers for trisystem
    magmaDoubleComplex rA;

    // set dA to 0
    #pragma unroll
     for( int j = 0; j < block_size; j++ ){
         dA[ j ] = MAGMA_Z_ZERO;
     }

    // generate the triangular systems
    int t = Mcol[ mstart + tid ];
    int k = Arow[ t+1 ] - 1;
    int alim = Arow[ t ]-1;
    int l = Mrow[ row+1 ]-1;
    int idx = block_size-1;
    while( k > alim && l > mlim  ){ // stop once this column is done
        int mcol =  Mcol[ l ];
        int acol = Acol[k];
        if( mcol == acol ){ //match
            dA[ idx ] = Aval[ k ];
            k--;
            l--;
            idx--;
        } else if( acol > mcol ){// need to check next element
            k--;
        } else { // element does not exist, i.e. l < LC.col[k]
            l--; // check next elment in the sparsity pattern
            idx--; // leave this element equal zero
        }
    }

    // second: solve the triangular systems - in registers
    // we know how RHS looks like
    rB = ( tid == 0 ) ? MAGMA_Z_ONE : MAGMA_Z_ZERO;

        // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < block_size; k++)
    {
        rA = dA[ k ];
         if (k%block_size == tid)
             rB /= rA;
         magmaDoubleComplex top = __shfl(rB, k%block_size);
         if ( tid > k)
             rB -= (top*rA);
    }

    // Drop B to dev memory - in ISAI preconditioner M
    Mval[ mstart + tid ] = rB;

#endif

}


template <int block_size>
__device__ __forceinline__ void
magma_zlowerisai_regs_inv_select(
int N,
magma_int_t num_rows,
const magma_index_t * __restrict__ Arow,
const magma_index_t * __restrict__ Acol,
const magmaDoubleComplex * __restrict__ Aval,
magma_index_t *Mrow,
magma_index_t *Mcol,
magmaDoubleComplex *Mval )
{
    if (N == block_size) {
        magma_zlowerisai_regs_inv_kernel<block_size>(
                num_rows, Arow, Acol, Aval, Mrow, Mcol, Mval);
    } else {
        magma_zlowerisai_regs_inv_select<block_size-1>(
                N, num_rows, Arow, Acol, Aval, Mrow, Mcol, Mval);
    }
}


template <>
__device__ __forceinline__ void
magma_zlowerisai_regs_inv_select<0>(
int N,
magma_int_t num_rows,
const magma_index_t * __restrict__ Arow,
const magma_index_t * __restrict__ Acol,
const magmaDoubleComplex * __restrict__ Aval,
magma_index_t *Mrow,
magma_index_t *Mcol,
magmaDoubleComplex *Mval )
{
    // TODO(Hartwig): Are you soure we want to have printfs called from the
    //                device?
    printf("%% error: size out of range: %d\n", N);
}


__global__ void
magma_zlowerisai_regs_inv_switch(
magma_int_t num_rows,
const magma_index_t * __restrict__ Arow,
const magma_index_t * __restrict__ Acol,
const magmaDoubleComplex * __restrict__ Aval,
magma_index_t *Mrow,
magma_index_t *Mcol,
magmaDoubleComplex *Mval )
{
    int row = gridDim.x*blockIdx.y*blockDim.y + blockIdx.x*blockDim.y + threadIdx.y;
    if( row < num_rows ){
        int N = Mrow[ row+1 ] - Mrow[ row ];
        magma_zlowerisai_regs_inv_select<MaxBlockSize>(
                N, num_rows, Arow, Acol, Aval, Mrow, Mcol, Mval);
    }
}


template <int block_size>
__device__ void
magma_zupperisai_regs_inv_kernel(
magma_int_t num_rows,
const magma_index_t * __restrict__ Arow,
const magma_index_t * __restrict__ Acol,
const magmaDoubleComplex * __restrict__ Aval,
magma_index_t *Mrow,
magma_index_t *Mcol,
magmaDoubleComplex *Mval )
{
#if (defined( REAL ) && ( __CUDA_ARCH__ >= 300 ))
    int tid = threadIdx.x;
    int row = gridDim.x*blockIdx.y*blockDim.y + blockIdx.x*blockDim.y + threadIdx.y;

    if( tid >= block_size )
        return;

    if( row >= num_rows )
        return;

    // only if within the size
    int mstart = Mrow[ row ];
    int mlim = Mrow[ row ]-1;

    magmaDoubleComplex rB;      // registers for trsv
    magmaDoubleComplex dA[ block_size ];  // registers for trisystem
    magmaDoubleComplex rA;

    // set dA to 0
    #pragma unroll
     for( int j = 0; j < block_size; j++ ){
         dA[ j ] = MAGMA_Z_ZERO;
     }

    // generate the triangular systems
    int t = Mcol[ mstart + tid ];
    int k = Arow[ t+1 ] - 1;
    int alim = Arow[ t ]-1;
    int l = Mrow[ row+1 ]-1;
    int idx = block_size-1;
    while( k > alim && l > mlim  ){ // stop once this column is done
        int mcol =  Mcol[ l ];
        int acol = Acol[k];
        if( mcol == acol ){ //match
            dA[ idx ] = Aval[ k ];
            k--;
            l--;
            idx--;
        } else if( acol > mcol ){// need to check next element
            k--;
        } else { // element does not exist, i.e. l < LC.col[k]
            l--; // check next elment in the sparsity pattern
            idx--; // leave this element equal zero
        }
    }

    // second: solve the triangular systems - in registers
    // we know how RHS looks like
    rB = ( tid == block_size-1 ) ? MAGMA_Z_ONE : MAGMA_Z_ZERO;

        // Triangular solve in regs.
    #pragma unroll
    for (int k = block_size-1; k >-1; k--)
    {
        rA = dA[ k ];
         if (k%block_size == tid)
             rB /= rA;
         magmaDoubleComplex bottom = __shfl(rB, k%block_size);
         if ( tid < k)
             rB -= (bottom*rA);
    }

    // Drop B to dev memory - in ISAI preconditioner M
    Mval[ mstart + tid ] = rB;

#endif
}


template <int block_size>
__device__ __forceinline__ void
magma_zupperisai_regs_inv_select(
int N,
magma_int_t num_rows,
const magma_index_t * __restrict__ Arow,
const magma_index_t * __restrict__ Acol,
const magmaDoubleComplex * __restrict__ Aval,
magma_index_t *Mrow,
magma_index_t *Mcol,
magmaDoubleComplex *Mval )
{
    if (N == block_size) {
        magma_zupperisai_regs_inv_kernel<block_size>(
                num_rows, Arow, Acol, Aval, Mrow, Mcol, Mval);
    } else {
        magma_zupperisai_regs_inv_select<block_size-1>(
                N, num_rows, Arow, Acol, Aval, Mrow, Mcol, Mval);
    }
}


template <>
__device__ __forceinline__ void
magma_zupperisai_regs_inv_select<0>(
int N,
magma_int_t num_rows,
const magma_index_t * __restrict__ Arow,
const magma_index_t * __restrict__ Acol,
const magmaDoubleComplex * __restrict__ Aval,
magma_index_t *Mrow,
magma_index_t *Mcol,
magmaDoubleComplex *Mval )
{
    // TODO(Hartwig): Are you soure we want to have printfs called from the
    //                device?
    printf("%% error: size out of range: %d\n", N);
}


__global__ void
magma_zupperisai_regs_inv_switch(
magma_int_t num_rows,
const magma_index_t * __restrict__ Arow,
const magma_index_t * __restrict__ Acol,
const magmaDoubleComplex * __restrict__ Aval,
magma_index_t *Mrow,
magma_index_t *Mcol,
magmaDoubleComplex *Mval )
{
    int row = gridDim.x*blockIdx.y*blockDim.y + blockIdx.x*blockDim.y + threadIdx.y;
    if( row < num_rows ){
        int N = Mrow[ row+1 ] - Mrow[ row ];
        magma_zupperisai_regs_inv_select<MaxBlockSize>(
                N, num_rows, Arow, Acol, Aval, Mrow, Mcol, Mval);
    }
}

#endif


/**
    Purpose
    -------
    This routine is designet to combine all kernels into one.

    Arguments
    ---------


    @param[in]
    uplotype    magma_uplo_t
                lower or upper triangular

    @param[in]
    transtype   magma_trans_t
                possibility for transposed matrix

    @param[in]
    diagtype    magma_diag_t
                unit diagonal or not

    @param[in]
    L           magma_z_matrix
                triangular factor for which the ISAI matrix is computed.
                Col-Major CSR storage.

    @param[in,out]
    M           magma_z_matrix*
                SPAI preconditioner CSR col-major

    @param[out]
    sizes       magma_int_t*
                Number of Elements that are replaced.

    @param[out]
    locations   magma_int_t*
                Array indicating the locations.

    @param[out]
    trisystems  magmaDoubleComplex*
                trisystems

    @param[out]
    rhs         magmaDoubleComplex*
                right-hand sides

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zaux
    ********************************************************************/

extern "C" magma_int_t
magma_zisai_generator_regs(
    magma_uplo_t uplotype,
    magma_trans_t transtype,
    magma_diag_t diagtype,
    magma_z_matrix L,
    magma_z_matrix *M,
    magma_index_t *sizes,
    magma_index_t *locations,
    magmaDoubleComplex *trisystems,
    magmaDoubleComplex *rhs,
    magma_queue_t queue )
{
    magma_int_t info = 0;

    magma_int_t arch = magma_getdevice_arch();

    hipDeviceSetCacheConfig( hipFuncCachePreferL1 );


    // routine 1
    // int r1bs1 = 32;
    // int r1bs2 = 1;
    // int r1dg1 = min( int( sqrt( double( M->num_rows ))), 65535 );
    // int r1dg2 = min(magma_ceildiv( M->num_rows, r1dg1 ), 65535);
    // int r1dg3 = magma_ceildiv( M->num_rows, r1dg1*r1dg2 );
    // //printf(" grid: %d x %d x %d\n", r1dg1, r1dg2, r1dg3 );
    // dim3 r1block( r1bs1, r1bs2, 1 );
    // dim3 r1grid( r1dg1, r1dg2, r1dg3 );

    int r2bs1 = 32;
    int r2bs2 = 4;
    int necessary_blocks = L.num_rows/r2bs2;
    int r2dg1 = min( int( sqrt( double( necessary_blocks ))), 65535 );
    int r2dg2 = min(magma_ceildiv( necessary_blocks, r2dg1 ), 65535);
    int r2dg3 = magma_ceildiv( necessary_blocks, r2dg1*r2dg2 );
    dim3 r2block( r2bs1, r2bs2, 1 );
    dim3 r2grid( r2dg1, r2dg2, r2dg3 );

 //   int r2dg1 = min( int( sqrt( double( magma_ceildiv( M->num_rows, r2bs2 )))), 65535);
 //   int r2dg2 = min(magma_ceildiv( M->num_rows, r2dg1 ), 65535);
 //   int r2dg3 = magma_ceildiv( M->num_rows, r2dg1*r2dg2 );
 //   dim3 r2block( r2bs1, r2bs2, 1 );
 //   dim3 r2grid( r2dg1, r2dg2, r2dg3 );

#if (CUDA_VERSION >= 7000)
    if (arch >= 300) {
        if (uplotype == MagmaLower) { //printf("in here lower new kernel\n");
            magma_zlowerisai_regs_inv_switch<<< r2grid, r2block, 0, queue->hip_stream() >>>(
                L.num_rows,
                L.row,
                L.col,
                L.val,
                M->row,
                M->col,
                M->val );
        }
        else { // printf("in here upper new kernel\n");
            magma_zupperisai_regs_inv_switch<<< r2grid, r2block, 0, queue->hip_stream() >>>(
                L.num_rows,
                L.row,
                L.col,
                L.val,
                M->row,
                M->col,
                M->val );
        }
    }
    else {
       printf( "%% error: ISAI preconditioner requires CUDA ARCHITECTURE >= 300.\n" );
       info = MAGMA_ERR_NOT_SUPPORTED;
    }
#else
   printf( "%% error: ISAI preconditioner requires CUDA >= 7.0.\n" );
   info = MAGMA_ERR_NOT_SUPPORTED;
#endif

    return info;
}

