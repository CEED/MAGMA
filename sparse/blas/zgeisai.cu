#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/
#include "magmasparse_internal.h"

#define PRECISION_z
#define COMPLEX
#define BLOCKSIZE 32
#define WARP_SIZE 32
#define WRP 32
#define WRQ 4


// initialize arrays with zero
__global__ void
magma_zgpumemzero_z(  
    magmaDoubleComplex * d, 
    int n, 
    int dim_x,
    int dim_y )
{
    int i = blockIdx.y * gridDim.x + blockIdx.x;
    int idx = threadIdx.x;
    
    if( i >= n ){
       return;
    }
    if( idx >= dim_x ){
       return;
    }
    
    for( int j=0; j<dim_y; j++)
        d[ i*dim_x*dim_y + j*dim_y + idx ] = MAGMA_Z_MAKE( 0.0, 0.0 );
}

__global__ void 
magma_zlocations_lower_kernel(    
    magma_int_t n, 
    magma_index_t *row,
    magma_index_t *col,
    magmaDoubleComplex *val,
    magma_index_t *sizes,
    magma_index_t *locations,
    magmaDoubleComplex *trisystems,
    magmaDoubleComplex *rhs )
{
    int i = threadIdx.x;
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    
    if( j >= n ){
        return;
    }
    int start = row[j];
    int end = row[j+1];
    int count = end-start;
    if( i == 0 ){
        sizes[j] = count;
        rhs[ j*WARP_SIZE ] = MAGMA_Z_ONE;
    }
    
    if ( i<count ){
        locations[ j*WARP_SIZE + i ] = col[ row[j]+i ];
    }
}// kernel 


__global__ void 
magma_zlocations_trunc_lower_kernel(    
    magma_int_t n, 
    magma_index_t *row,
    magma_index_t *col,
    magmaDoubleComplex *val,
    magma_index_t *sizes,
    magma_index_t *locations,
    magmaDoubleComplex *trisystems,
    magmaDoubleComplex *rhs )
{
    int i = threadIdx.x;
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    
    if( j >= n ){
        return;
    }
    int start = row[j];
    int end = row[j+1];
    int count = end-start;
    
    // normal case
    if( count <= BLOCKSIZE ){ // normal case
        if( i == 0 ){
            sizes[j] = count;
            rhs[ j*WARP_SIZE ] = MAGMA_Z_ONE;
        }
        if ( i<count ){
            locations[ j*WARP_SIZE + i ] = col[ row[j]+i ];
        }
    } else { // truncate in this row to the blocksize, 
             // take only the 32 elements close to the main diagonal into account
          count = BLOCKSIZE;
       if( i == 0 ){
            sizes[j] = count;
            rhs[ j*WARP_SIZE ] = MAGMA_Z_ONE;
        }
        
        locations[ j*WARP_SIZE + i ] = col[ row[j+1]-BLOCKSIZE+i ];
        
    }
}// kernel 



__global__ void 
magma_zlocations_upper_kernel(    
    magma_int_t n, 
    magma_index_t *row,
    magma_index_t *col,
    magmaDoubleComplex *val,
    magma_index_t *sizes,
    magma_index_t *locations,
    magmaDoubleComplex *trisystems,
    magmaDoubleComplex *rhs )
{
    int i = threadIdx.x;
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    
    if( j >= n ){
        return;
    }
    int start = row[j];
    int end = row[j+1];
    int count = end-start;
    if( i == 0 ){
        sizes[j] = count;
        rhs[ j*WARP_SIZE+count-1 ] = MAGMA_Z_ONE;
    }
    
    if ( i<count ){
        locations[ j*WARP_SIZE + i ] = col[ row[j]+i ];
    }
}// kernel 

__global__ void 
magma_zlocations_trunc_upper_kernel(    
    magma_int_t n, 
    magma_index_t *row,
    magma_index_t *col,
    magmaDoubleComplex *val,
    magma_index_t *sizes,
    magma_index_t *locations,
    magmaDoubleComplex *trisystems,
    magmaDoubleComplex *rhs )
{
    int i = threadIdx.x;
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    
    if( j >= n ){
        return;
    }
    int start = row[j];
    int end = row[j+1];
    int count = end-start;
    
    // normal case
    if( count <= BLOCKSIZE ){ // normal case
        if( i == 0 ){
            sizes[j] = count;
            rhs[ j*WARP_SIZE+count-1 ] = MAGMA_Z_ONE;
        }
        if ( i<count ){
            locations[ j*WARP_SIZE + i ] = col[ row[j]+i ];
        }
    } else { // truncate in this row to the blocksize, 
             // take only the 32 elements close to the main diagonal into account
          count = BLOCKSIZE;
       if( i == 0 ){
            sizes[j] = count;
            rhs[ j*WARP_SIZE+count-1 ] = MAGMA_Z_ONE;
        }
        
        locations[ j*WARP_SIZE + i ] = col[ row[j]+i ];
        
    }
}// kernel 

__global__ void 
magma_zfilltrisystems_kernel(    
    magma_int_t n, 
    magma_index_t *row,
    magma_index_t *col,
    magmaDoubleComplex *val,
    magma_index_t *sizes,
    magma_index_t *locations,
    magmaDoubleComplex *trisystems,
    magmaDoubleComplex *rhs )
{
    int i = (blockDim.x * blockIdx.x + threadIdx.x);

    if ( i>=n ){
        return;
    }
    for( int j=0; j<sizes[ i ]; j++ ){// no need for first
        int k = row[ locations[ j+i*WARP_SIZE ] ];
        int l = i*WARP_SIZE;
        int idx = 0;
        while( k < row[ locations[ j+i*WARP_SIZE ]+1 ] && l < (i+1)*WARP_SIZE ){ // stop once this column is done
            if( locations[ l ] == col[k] ){ //match
                // int loc = i*WARP_SIZE*WARP_SIZE + j*WARP_SIZE + idx;
                trisystems[ i*WARP_SIZE*WARP_SIZE + j*WARP_SIZE + idx ] 
                                                        = val[ k ];
                k++;
                l++;
                idx++;
            } else if( col[k] < locations[ l ] ){// need to check next element
                k++;
            } else { // element does not exist, i.e. l < LC.col[k]
                // printf("increment l\n");
                l++; // check next elment in the sparsity pattern
                idx++; // leave this element equal zero
            }
        } 
        
    }
}// kernel 











/**
    Purpose
    -------
    
    This routine prepares the batch of small triangular systems that 
    need to be solved for computing the ISAI preconditioner.
    

    Arguments
    ---------

    @param[in]
    uplotype    magma_uplo_t
                input matrix
                
    @param[in]
    transtype   magma_trans_t
                input matrix
                
    @param[in]
    diagtype    magma_diag_t
                input matrix
                
    @param[in]
    L           magma_z_matrix
                triangular factor for which the ISAI matrix is computed.
                Col-Major CSR storage.
                
    @param[in]
    LC          magma_z_matrix
                sparsity pattern of the ISAI matrix.
                Col-Major CSR storage.
                
    @param[in,out]
    sizes       magma_index_t*
                array containing the sizes of the small triangular systems 
                
    @param[in,out]
    locations   magma_index_t*
                array containing the locations in the respective column of L
                
    @param[in,out]
    trisystems  magmaDoubleComplex*
                batch of generated small triangular systems. All systems are
                embedded in uniform memory blocks of size BLOCKSIZE x BLOCKSIZE
                
    @param[in,out]
    rhs         magmaDoubleComplex*
                RHS of the small triangular systems
                
    @param[in]
    queue       magma_queue_t
                Queue to execute in.
                
    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_zmprepare_batched_gpu(
    magma_uplo_t uplotype,
    magma_trans_t transtype,
    magma_diag_t diagtype,
    magma_z_matrix L,
    magma_z_matrix LC,
    magma_index_t *sizes,
    magma_index_t *locations,
    magmaDoubleComplex *trisystems,
    magmaDoubleComplex *rhs,    
    magma_queue_t queue )
{
    
    int blocksize1 = WARP_SIZE;
    int blocksize2 = 1;
    int dimgrid1 = min( int( sqrt( double( LC.num_rows ))), 65535 );
    int dimgrid2 = min(magma_ceildiv( LC.num_rows, dimgrid1 ), 65535);
    int dimgrid3 = magma_ceildiv( LC.num_rows, dimgrid1*dimgrid2 );
    hipDeviceSetCacheConfig( hipFuncCachePreferL1 );
    dim3 grid( dimgrid1, dimgrid2, dimgrid3 );
    dim3 block( blocksize1, blocksize2, 1 );
    

    int blocksize21 = BLOCKSIZE;
    int blocksize22 = 1;

    int dimgrid21 = magma_ceildiv( LC.num_rows, blocksize21 );
    int dimgrid22 = 1;
    int dimgrid23 = 1;
    dim3 grid2( dimgrid21, dimgrid22, dimgrid23 );
    dim3 block2( blocksize21, blocksize22, 1 );
    
    magma_zgpumemzero_z<<< grid, block, 0, queue->hip_stream() >>>(  
        trisystems, LC.num_rows, WARP_SIZE, WARP_SIZE );
    
    magma_zgpumemzero_z<<< grid, block, 0, queue->hip_stream() >>>(  
        rhs, LC.num_rows, WARP_SIZE, 1);

    
   // magma_zprint_gpu( 32, 32, L.dval, 32, queue );
    
    // Runtime API
    // hipFuncCachePreferShared: shared memory is 48 KB
    // hipFuncCachePreferEqual: shared memory is 32 KB
    // hipFuncCachePreferL1: shared memory is 16 KB
    // hipFuncCachePreferNone: no preference
    //hipFuncSetCacheConfig(hipFuncCachePreferShared);

    
    if( uplotype == MagmaLower ){
        magma_zlocations_lower_kernel<<< grid, block, 0, queue->hip_stream() >>>(    
                        LC.num_rows, 
                        LC.drow,
                        LC.dcol,
                        LC.dval,
                        sizes,
                        locations,
                        trisystems,
                        rhs );
    } else {
        magma_zlocations_upper_kernel<<< grid, block, 0, queue->hip_stream() >>>(    
                        LC.num_rows, 
                        LC.drow,
                        LC.dcol,
                        LC.dval,
                        sizes,
                        locations,
                        trisystems,
                        rhs );
    }
    
    // magma_zprint_gpu( 32, 32, L.dval, 32, queue );

    
    magma_zfilltrisystems_kernel<<< grid2, block2, 0, queue->hip_stream() >>>(    
                        L.num_rows, 
                        L.drow,
                        L.dcol,
                        L.dval,
                        sizes,
                        locations,
                        trisystems,
                        rhs );
    //magma_zprint_gpu( 32, 32, L.dval, 32, queue );

    return MAGMA_SUCCESS;
}









__global__ void 
magma_zbackinsert_kernel(    
    magma_int_t n, 
    magma_index_t *row,
    magma_index_t *col,
    magmaDoubleComplex *val,
    magma_index_t *sizes,
    magmaDoubleComplex *rhs )
{
    int i = threadIdx.x;
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int end = sizes[j];
    if( j >= n ){
        return;
    }
    
    if ( i>=end ){
        return;
    }
    
    val[row[j]+i] = rhs[j*WARP_SIZE+i];
}// kernel 



/**
    Purpose
    -------
    Inserts the values into the preconditioner matrix

    Arguments
    ---------
    

    @param[in]
    uplotype    magma_uplo_t
                lower or upper triangular
                
    @param[in]
    transtype   magma_trans_t
                possibility for transposed matrix
                
    @param[in]
    diagtype    magma_diag_t
                unit diagonal or not
                
    @param[in,out]
    M           magma_z_matrix*
                SPAI preconditioner CSR col-major
                
    @param[out]
    sizes       magma_int_t*
                Number of Elements that are replaced.
                
    @param[out]
    locations   magma_int_t*
                Array indicating the locations.
                
    @param[out]
    trisystems  magmaDoubleComplex*
                trisystems
                
    @param[out]
    rhs         magmaDoubleComplex*
                right-hand sides

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zaux
    ********************************************************************/

extern "C" magma_int_t
magma_zmbackinsert_batched_gpu(
    magma_uplo_t uplotype,
    magma_trans_t transtype,
    magma_diag_t diagtype,
    magma_z_matrix *M,
    magma_index_t *sizes,
    magma_index_t *locations,
    magmaDoubleComplex *trisystems,
    magmaDoubleComplex *rhs,    
    magma_queue_t queue )
{
    magma_int_t info = 0;
    
    int blocksize1 = WARP_SIZE;
    int blocksize2 = 1;
    int dimgrid1 = min( int( sqrt( double( M->num_rows ))), 65535 );
    int dimgrid2 = min(magma_ceildiv( M->num_rows, dimgrid1 ), 65535);
    int dimgrid3 = magma_ceildiv( M->num_rows, dimgrid1*dimgrid2 );
    
    dim3 grid( dimgrid1, dimgrid2, dimgrid3 );
    dim3 block( blocksize1, blocksize2, 1 );

    magma_zbackinsert_kernel<<< grid, block, 0, queue->hip_stream() >>>(    
            M->num_rows, 
            M->drow,
            M->dcol,
            M->dval,
            sizes,
            rhs );
    
    return info;
}





