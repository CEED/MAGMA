#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/
#include "magmasparse_internal.h"

#define PRECISION_z


__global__ void 
magma_ziterilu_csr_kernel(   
    magma_int_t num_rows, 
    magma_int_t nnz,  
    magma_index_t *rowidxA, 
    magma_index_t *colidxA,
    const magmaDoubleComplex * __restrict__ A, 
    magma_index_t *rowptrL, 
    magma_index_t *colidxL, 
    magmaDoubleComplex *valL, 
    magma_index_t *rowptrU, 
    magma_index_t *rowidxU, 
    magmaDoubleComplex *valU )
{
    int i, j;
    int k = blockDim.x * blockIdx.x + threadIdx.x;


    magmaDoubleComplex zero = MAGMA_Z_MAKE(0.0, 0.0);
    magmaDoubleComplex s, sp;
    int il, iu, jl, ju;
    

    if (k < nnz)
    {
        i = rowidxA[k];
        j = colidxA[k];

#if (__CUDA_ARCH__ >= 350) && (defined(PRECISION_d) || defined(PRECISION_s))
        s =  __ldg( A+k );
#else
        s =  A[k];
#endif

        il = rowptrL[i];
        iu = rowptrU[j];

        while (il < rowptrL[i+1] && iu < rowptrU[j+1])
        {
            sp = zero;
            jl = colidxL[il];
            ju = rowidxU[iu];

            // avoid branching
            sp = ( jl == ju ) ? valL[il] * valU[iu] : sp;
            s = ( jl == ju ) ? s-sp : s;
            il = ( jl <= ju ) ? il+1 : il;
            iu = ( jl >= ju ) ? iu+1 : iu;
        }
        // undo the last operation (it must be the last)
        s += sp;
        __syncthreads();
        
        if ( i > j )      // modify l entry
            valL[il-1] =  s / valU[rowptrU[j+1]-1];
        else {            // modify u entry
            valU[iu-1] = s;
        }
    }
}// kernel 





/**
    Purpose
    -------
    
    This routine iteratively computes an incomplete LU factorization.
    The idea is according to Edmond Chow's presentation at SIAM 2014.
    This routine was used in the ISC 2015 paper:
    E. Chow et al.: 'Study of an Asynchronous Iterative Algorithm
                     for Computing Incomplete Factorizations on GPUs'
 
    The input format of the matrix is Magma_CSRCOO for the upper and lower 
    triangular parts. Note however, that we flip col and rowidx for the 
    U-part.
    Every component of L and U is handled by one thread. 

    Arguments
    ---------

    @param[in]
    A           magma_z_matrix
                input matrix A determing initial guess & processing order

    @param[in,out]
    L           magma_z_matrix
                input/output matrix L containing the ILU approximation

    @param[in,out]
    U           magma_z_matrix
                input/output matrix U containing the ILU approximation
                              
    @param[in]
    queue       magma_queue_t
                Queue to execute in.
                
    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_ziterilu_csr( 
    magma_z_matrix A,
    magma_z_matrix L,
    magma_z_matrix U,
    magma_queue_t queue )
{
    int blocksize1 = 128;
    int blocksize2 = 1;

    int dimgrid1 = magma_ceildiv( A.nnz, blocksize1 );
    int dimgrid2 = 1;
    int dimgrid3 = 1;

    // Runtime API
    // hipFuncCachePreferShared: shared memory is 48 KB
    // hipFuncCachePreferEqual: shared memory is 32 KB
    // hipFuncCachePreferL1: shared memory is 16 KB
    // hipFuncCachePreferNone: no preference
    //hipFuncSetCacheConfig(hipFuncCachePreferShared);

    hipDeviceSetCacheConfig( hipFuncCachePreferL1 );

    dim3 grid( dimgrid1, dimgrid2, dimgrid3 );
    dim3 block( blocksize1, blocksize2, 1 );
    magma_ziterilu_csr_kernel<<< grid, block, 0, queue->hip_stream() >>>
        ( A.num_rows, A.nnz, 
          A.rowidx, A.col, A.val, 
          L.row, L.col, L.val, 
          U.row, U.col, U.val );


    return MAGMA_SUCCESS;
}
