#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/
#include "magmasparse_internal.h"
//#include <hip/hip_runtime_api.h>

#define PRECISION_z
#define COMPLEX
#define BLOCKSIZE 256
#define WARP_SIZE 32
#define WRP 32
#define WRQ 1



#include <hip/hip_runtime.h>  // for CUDA_VERSION

#if (CUDA_VERSION > 6000)
#if (CUDA_ARCH >= 300)

__device__
void ztrsv_lower_kernel_general(magmaDoubleComplex *dA, magmaDoubleComplex *dB, int *sizes)
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;

    magmaDoubleComplex rB[ 2 ];
    magmaDoubleComplex rA[ 2 ];

    int n;
    int k;
    int N = sizes[j];

    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;


    // Read B to regs.
    #pragma unroll
    for (n = 0; n < 2; n++)
        rB[n] = dB[n*WARP_SIZE+idn];


    // Triangular solve in regs.
    #pragma unroll
    for (k = 0; k < N; k++)
    {
        #pragma unroll
        for (n = 0; n < 2; n++)
            rA[n] = dA[k*WARP_SIZE+n*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB[k/WARP_SIZE] /= rA[k/WARP_SIZE];

        magmaDoubleComplex top = __shfl(rB[k/WARP_SIZE], k%WARP_SIZE);

        #pragma unroll
        for (n = 0; n < 2; n++)
          if (n*WARP_SIZE+idn > k)
            rB[n] -= (top*rA[n]);
    }
    // Drop B to dev mem.
    #pragma unroll
    for (n = 0; n < 2; n++)
        if (n*WARP_SIZE+idn < N)
            dB[n*WARP_SIZE+idn] = rB[n];

#endif
}


__device__
void ztrsv_upper_kernel_general(magmaDoubleComplex *dA, magmaDoubleComplex *dB, int *sizes)
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;

    magmaDoubleComplex rB[ 2 ];
    magmaDoubleComplex rA[ 2 ];

    int n;
    int k;
    int N = sizes[j];

    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;


    // Read B to regs.
    #pragma unroll
    for (n = 0; n < 2; n++)
        rB[n] = dB[n*WARP_SIZE+idn];


    // Triangular solve in regs.
    #pragma unroll
    for (int k = N-1; k > -1; k--)
    {
        #pragma unroll
        for (n = 0; n < 2; n++)
            rA[n] = dA[k*WARP_SIZE+n*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB[k/WARP_SIZE] /= rA[k/WARP_SIZE];

        magmaDoubleComplex top = __shfl(rB[k/WARP_SIZE], k%WARP_SIZE);

        #pragma unroll
        for (n = 0; n < 2; n++)
          if (n*WARP_SIZE+idn < k)
            rB[n] -= (top*rA[n]);
    }
    // Drop B to dev mem.
    #pragma unroll
    for (n = 0; n < 2; n++)
        if (n*WARP_SIZE+idn < N)
            dB[n*WARP_SIZE+idn] = rB[n];

#endif
}



__device__
void ztrsv_lower_kernel_1(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 1; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_2(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 2; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_3(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 3; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_4(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 4; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_5(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 5; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_6(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 6; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_7(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 7; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_8(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 8; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_9(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 9; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_10(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 10; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_11(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 11; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_12(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 12; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_13(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 13; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_14(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 14; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_15(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 15; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_16(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 16; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_17(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 17; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_18(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 18; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_19(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 19; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_20(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 20; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_21(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 21; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_22(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 22; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_23(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 23; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_24(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 24; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_25(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 25; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_26(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 26; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_27(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 27; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_28(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 28; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_29(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 29; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_30(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 30; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_31(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 31; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_lower_kernel_32(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 0; k < 32; k++)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);
        if ( idn > k)
            rB -= (top*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}








__global__
void ztrsv_lower_kernel_switch(magmaDoubleComplex *dA, magmaDoubleComplex *dB, int *sizes, int num_rows )
{


    int j = blockIdx.y * gridDim.x + blockIdx.x;
    if( j < num_rows ){
    int N = sizes[j];
    switch( N ) {
        case  1:
            ztrsv_lower_kernel_1( dA, dB ); break;
        case  2:
            ztrsv_lower_kernel_2( dA, dB ); break;
        case  3:
            ztrsv_lower_kernel_3( dA, dB ); break;
        case  4:
            ztrsv_lower_kernel_4( dA, dB ); break;
        case  5:
            ztrsv_lower_kernel_5( dA, dB ); break;
        case  6:
            ztrsv_lower_kernel_6( dA, dB ); break;
        case  7:
            ztrsv_lower_kernel_7( dA, dB ); break;
        case  8:
            ztrsv_lower_kernel_8( dA, dB ); break;
        case  9:
            ztrsv_lower_kernel_9( dA, dB ); break;
        case  10:
            ztrsv_lower_kernel_10( dA, dB ); break;
        case  11:
            ztrsv_lower_kernel_11( dA, dB ); break;
        case  12:
            ztrsv_lower_kernel_12( dA, dB ); break;
        case  13:
            ztrsv_lower_kernel_13( dA, dB ); break;
        case  14:
            ztrsv_lower_kernel_14( dA, dB ); break;
        case  15:
            ztrsv_lower_kernel_15( dA, dB ); break;
        case  16:
            ztrsv_lower_kernel_16( dA, dB ); break;
        case  17:
            ztrsv_lower_kernel_17( dA, dB ); break;
        case  18:
            ztrsv_lower_kernel_18( dA, dB ); break;
        case  19:
            ztrsv_lower_kernel_19( dA, dB ); break;
        case  20:
            ztrsv_lower_kernel_20( dA, dB ); break;
        case  21:
            ztrsv_lower_kernel_21( dA, dB ); break;
        case  22:
            ztrsv_lower_kernel_22( dA, dB ); break;
        case  23:
            ztrsv_lower_kernel_23( dA, dB ); break;
        case  24:
            ztrsv_lower_kernel_24( dA, dB ); break;
        case  25:
            ztrsv_lower_kernel_25( dA, dB ); break;
        case  26:
            ztrsv_lower_kernel_26( dA, dB ); break;
        case  27:
            ztrsv_lower_kernel_27( dA, dB ); break;
        case  28:
            ztrsv_lower_kernel_28( dA, dB ); break;
        case  29:
            ztrsv_lower_kernel_29( dA, dB ); break;
        case  30:
            ztrsv_lower_kernel_30( dA, dB ); break;
        case  31:
            ztrsv_lower_kernel_31( dA, dB ); break;
        case  32:
            ztrsv_lower_kernel_32( dA, dB ); break;
        default:
            ztrsv_lower_kernel_general( dA, dB, sizes );break;
    }
    }
}
__device__
void ztrsv_upper_kernel_1(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 1-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_2(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 2-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_3(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 3-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_4(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 4-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_5(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 5-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_6(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 6-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_7(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 7-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_8(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 8-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_9(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 9-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_10(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 10-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_11(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 11-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_12(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 12-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_13(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 13-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_14(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 14-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_15(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 15-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_16(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 16-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_17(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 17-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_18(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 18-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_19(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 19-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_20(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 20-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_21(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 21-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_22(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 22-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_23(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 23-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_24(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 24-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_25(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 25-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_26(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 26-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_27(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 27-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_28(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 28-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_29(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 29-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_30(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 30-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_31(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 31-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}



__device__
void ztrsv_upper_kernel_32(magmaDoubleComplex *dA, magmaDoubleComplex *dB )
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    magmaDoubleComplex rB;
    magmaDoubleComplex rA;
    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;

    // Read B to regs.
    rB = dB[idn];
    // Triangular solve in regs.
    #pragma unroll
    for (int k = 32-1; k >-1; k--)
    {
        rA = dA[k*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB /= rA;
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);
        if ( idn < k)
            rB -= (bottom*rA);
    }
    // Drop B to dev mem.
    dB[idn] = rB;
#endif
}








__global__
void ztrsv_upper_kernel_switch(magmaDoubleComplex *dA, magmaDoubleComplex *dB, int *sizes, int num_rows )
{


    int j = blockIdx.y * gridDim.x + blockIdx.x;
    if( j < num_rows ){
    int N = sizes[j];
    switch( N ) {
        case  1:
            ztrsv_upper_kernel_1( dA, dB ); break;
        case  2:
            ztrsv_upper_kernel_2( dA, dB ); break;
        case  3:
            ztrsv_upper_kernel_3( dA, dB ); break;
        case  4:
            ztrsv_upper_kernel_4( dA, dB ); break;
        case  5:
            ztrsv_upper_kernel_5( dA, dB ); break;
        case  6:
            ztrsv_upper_kernel_6( dA, dB ); break;
        case  7:
            ztrsv_upper_kernel_7( dA, dB ); break;
        case  8:
            ztrsv_upper_kernel_8( dA, dB ); break;
        case  9:
            ztrsv_upper_kernel_9( dA, dB ); break;
        case  10:
            ztrsv_upper_kernel_10( dA, dB ); break;
        case  11:
            ztrsv_upper_kernel_11( dA, dB ); break;
        case  12:
            ztrsv_upper_kernel_12( dA, dB ); break;
        case  13:
            ztrsv_upper_kernel_13( dA, dB ); break;
        case  14:
            ztrsv_upper_kernel_14( dA, dB ); break;
        case  15:
            ztrsv_upper_kernel_15( dA, dB ); break;
        case  16:
            ztrsv_upper_kernel_16( dA, dB ); break;
        case  17:
            ztrsv_upper_kernel_17( dA, dB ); break;
        case  18:
            ztrsv_upper_kernel_18( dA, dB ); break;
        case  19:
            ztrsv_upper_kernel_19( dA, dB ); break;
        case  20:
            ztrsv_upper_kernel_20( dA, dB ); break;
        case  21:
            ztrsv_upper_kernel_21( dA, dB ); break;
        case  22:
            ztrsv_upper_kernel_22( dA, dB ); break;
        case  23:
            ztrsv_upper_kernel_23( dA, dB ); break;
        case  24:
            ztrsv_upper_kernel_24( dA, dB ); break;
        case  25:
            ztrsv_upper_kernel_25( dA, dB ); break;
        case  26:
            ztrsv_upper_kernel_26( dA, dB ); break;
        case  27:
            ztrsv_upper_kernel_27( dA, dB ); break;
        case  28:
            ztrsv_upper_kernel_28( dA, dB ); break;
        case  29:
            ztrsv_upper_kernel_29( dA, dB ); break;
        case  30:
            ztrsv_upper_kernel_30( dA, dB ); break;
        case  31:
            ztrsv_upper_kernel_31( dA, dB ); break;
        case  32:
            ztrsv_upper_kernel_32( dA, dB ); break;
        default:
            ztrsv_upper_kernel_general( dA, dB, sizes );break;
    }
    }
}
#endif
#endif
/**
    Purpose
    -------
    Does all triangular solves

    Arguments
    ---------


    @param[in]
    uplotype    magma_uplo_t
                lower or upper triangular

    @param[in]
    transtype   magma_trans_t
                possibility for transposed matrix

    @param[in]
    diagtype    magma_diag_t
                unit diagonal or not

    @param[in]
    L           magma_z_matrix
                Matrix in CSR format

    @param[in]
    LC          magma_z_matrix
                same matrix, also CSR, but col-major

    @param[out]
    sizes       magma_int_t*
                Number of Elements that are replaced.

    @param[out]
    locations   magma_int_t*
                Array indicating the locations.

    @param[out]
    trisystems  magmaDoubleComplex*
                trisystems

    @param[out]
    rhs         magmaDoubleComplex*
                right-hand sides

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zaux
    ********************************************************************/

extern "C" magma_int_t
magma_zmtrisolve_batched_gpu(
    magma_uplo_t uplotype,
    magma_trans_t transtype,
    magma_diag_t diagtype,
    magma_z_matrix L,
    magma_z_matrix LC,
    magma_index_t *sizes,
    magma_index_t *locations,
    magmaDoubleComplex *trisystems,
    magmaDoubleComplex *rhs,
    magma_queue_t queue )
{
    magma_int_t info = 0;

    int blocksize1 = WARP_SIZE;
    int blocksize2 = 1;
    int dimgrid1 = min( int( sqrt( double( LC.num_rows ))), 65535 );
    int dimgrid2 = min(magma_ceildiv( LC.num_rows, dimgrid1 ), 65535);
    int dimgrid3 = magma_ceildiv( LC.num_rows, dimgrid1*dimgrid2 );

    dim3 grid( dimgrid1, dimgrid2, dimgrid3 );
    dim3 block( blocksize1, blocksize2, 1 );

#if (CUDA_VERSION > 6000)
#if (CUDA_ARCH >= 300)
    if( uplotype == MagmaLower ){
        //hipProfilerStart();
        ztrsv_lower_kernel_switch<<< grid, block, 0, queue->hip_stream() >>>(
                trisystems,
                rhs,
                sizes,
                LC.num_rows );
        //hipProfilerStop();
    } else {
        ztrsv_upper_kernel_switch<<< grid, block, 0, queue->hip_stream() >>>(
                trisystems,
                rhs,
                sizes,
                LC.num_rows );
    }
#endif
#endif

    return info;


}
