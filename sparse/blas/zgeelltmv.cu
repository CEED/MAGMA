#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/
#include "magmasparse_internal.h"

#define BLOCK_SIZE 512


// ELL SpMV kernel
//Michael Garland
template<bool betazero>
__global__ void 
zgeelltmv_kernel( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    magmaDoubleComplex alpha, 
    magmaDoubleComplex * dval, 
    magma_index_t * dcolind,
    magmaDoubleComplex * dx,
    magmaDoubleComplex beta, 
    magmaDoubleComplex * dy)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    if (row < num_rows ) {
        magmaDoubleComplex dot = MAGMA_Z_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row; n++ ) {
            int col = dcolind [ num_rows * n + row ];
            magmaDoubleComplex val = dval [ num_rows * n + row ];
            //if ( val != MAGMA_Z_ZERO )
                dot += val * dx[col ];
        }
        if (betazero) {
            dy[ row ] = dot * alpha;
        } else {
            dy[ row ] = dot * alpha + beta * dy [ row ];
        }
    }
}

// shifted ELL SpMV kernel
//Michael Garland
__global__ void 
zgeelltmv_kernel_shift( 
    int num_rows, 
    int num_cols,
    int num_cols_per_row,
    magmaDoubleComplex alpha, 
    magmaDoubleComplex lambda, 
    magmaDoubleComplex * dval, 
    magma_index_t * dcolind,
    magmaDoubleComplex * dx,
    magmaDoubleComplex beta, 
    int offset,
    int blocksize,
    magma_index_t * addrows,
    magmaDoubleComplex * dy)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    if (row < num_rows ) {
        magmaDoubleComplex dot = MAGMA_Z_MAKE(0.0, 0.0);
        for ( int n = 0; n < num_cols_per_row; n++ ) {
            int col = dcolind [ num_rows * n + row ];
            magmaDoubleComplex val = dval [ num_rows * n + row ];
            if ( val != 0)
                dot += val * dx[col ];
        }
        if ( row < blocksize )
            dy[ row ] = dot * alpha - lambda 
                    * dx[ offset+row ] + beta * dy [ row ];
        else
            dy[ row ] = dot * alpha - lambda 
                    * dx[ addrows[row-blocksize] ] + beta * dy [ row ];            
    }
}




/**
    Purpose
    -------
    
    This routine computes y = alpha *  A^t *  x + beta * y on the GPU.
    Input format is ELL.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_trans_t
                transposition parameter for A
                
    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 
                
    @param[in]
    nnz_per_row magma_int_t
                number of elements in the longest row 

    @param[in]
    alpha       magmaDoubleComplex
                scalar multiplier

    @param[in]
    dval        magmaDoubleComplex_ptr
                array containing values of A in ELL

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in ELL

    @param[in]
    dx          magmaDoubleComplex_ptr
                input vector x

    @param[in]
    beta        magmaDoubleComplex
                scalar multiplier

    @param[out]
    dy          magmaDoubleComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_z
    ********************************************************************/

extern "C" magma_int_t
magma_zgeelltmv(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    magma_int_t nnz_per_row,
    magmaDoubleComplex alpha,
    magmaDoubleComplex_ptr dval,
    magmaIndex_ptr dcolind,
    magmaDoubleComplex_ptr dx,
    magmaDoubleComplex beta,
    magmaDoubleComplex_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( m, BLOCK_SIZE ) );
    magma_int_t threads = BLOCK_SIZE;
    if (beta == MAGMA_Z_ZERO) {
        zgeelltmv_kernel<true><<< grid, threads, 0, queue->hip_stream() >>>
                  ( m, n, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );
    } else {
        zgeelltmv_kernel<false><<< grid, threads, 0, queue->hip_stream() >>>
                  ( m, n, nnz_per_row, alpha, dval, dcolind, dx, beta, dy );
    }


    return MAGMA_SUCCESS;
}


/**
    Purpose
    -------
    
    This routine computes y = alpha *( A - lambda I ) * x + beta * y on the GPU.
    Input format is ELL.
    
    Arguments
    ---------

    @param[in]
    transA      magma_trans_t
                transposition parameter for A    

    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 
                
    @param[in]
    nnz_per_row magma_int_t
                number of elements in the longest row 

    @param[in]
    alpha       magmaDoubleComplex
                scalar multiplier

    @param[in]
    lambda      magmaDoubleComplex
                scalar multiplier

    @param[in]
    dval        magmaDoubleComplex_ptr
                array containing values of A in ELL

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in ELL

    @param[in]
    dx          magmaDoubleComplex_ptr
                input vector x

    @param[in]
    beta        magmaDoubleComplex
                scalar multiplier
                
    @param[in]
    offset      magma_int_t 
                in case not the main diagonal is scaled
                
    @param[in]
    blocksize   magma_int_t 
                in case of processing multiple vectors  
                
    @param[in]
    addrows     magmaIndex_ptr
                in case the matrixpowerskernel is used

    @param[out]
    dy          magmaDoubleComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zblas
    ********************************************************************/

extern "C" magma_int_t
magma_zgeelltmv_shift(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    magma_int_t nnz_per_row,
    magmaDoubleComplex alpha,
    magmaDoubleComplex lambda,
    magmaDoubleComplex_ptr dval,
    magmaIndex_ptr dcolind,
    magmaDoubleComplex_ptr dx,
    magmaDoubleComplex beta,
    magma_int_t offset,
    magma_int_t blocksize,
    magmaIndex_ptr addrows,
    magmaDoubleComplex_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( m, BLOCK_SIZE ) );
    magma_int_t threads = BLOCK_SIZE;
    magmaDoubleComplex tmp_shift;
    //magma_zsetvector(1,&lambda,1,&tmp_shift,1); 
    tmp_shift = lambda;
    zgeelltmv_kernel_shift<<< grid, threads, 0, queue->hip_stream() >>>
                  ( m, n, nnz_per_row, alpha, tmp_shift, dval, dcolind, dx, 
                            beta, offset, blocksize, addrows, dy );


    return MAGMA_SUCCESS;
}
