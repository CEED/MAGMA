#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions mixed zc -> ds
       @author Mark Gates
*/
#include "common_magma.h"

#define PRECISION_z

#define BLK_X 64
#define BLK_Y 32

// TODO get rid of global variable!
static __device__ int flag = 0;


/*
    Divides matrix into ceil( n/BLK_X ) x ceil( n/BLK_Y ) blocks.
    Each block has BLK_X threads.
    Each thread loops across one row, updating BLK_Y entries.
    Updates only the diagonal and below.
    Blocks that are fully above the diagonal exit immediately.
    
    Code similar to zlag2c and zlaset.
*/
__global__
void zlat2c_lower(
    int n,
    const magmaDoubleComplex *A, int lda,
    magmaFloatComplex *SA,       int ldsa,
    double rmax )
{
    magmaDoubleComplex tmp;
    double neg_rmax = - rmax;
    
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    /* check if full block-column && (below diag) */
    bool full = (iby + BLK_Y <= n && (ind >= iby + BLK_Y));
    /* do only rows inside matrix, and blocks not above diag */
    if ( ind < n && ind + BLK_X > iby ) {
        A  += ind + iby*lda;
        SA += ind + iby*ldsa;
        if ( full ) {
            // full block-column, off-diagonal block
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                tmp = A[j*lda];
                if (   (hipCreal(tmp) < neg_rmax) || (hipCreal(tmp) > rmax)
#if defined(PRECISION_z) || defined(PRECISION_c)
                    || (hipCimag(tmp) < neg_rmax) || (hipCimag(tmp) > rmax)
#endif
                    )
                {
                    flag = 1;
                }
                SA[j*ldsa] = hipComplexDoubleToFloat( tmp );
            }
        }
        else {
            // either partial block-column or diagonal block
            for( int j=0; j < BLK_Y && iby+j < n && ind >= iby+j; ++j ) {
                tmp = A[j*lda];
                if (   (hipCreal(tmp) < neg_rmax) || (hipCreal(tmp) > rmax)
#if defined(PRECISION_z) || defined(PRECISION_c)
                    || (hipCimag(tmp) < neg_rmax) || (hipCimag(tmp) > rmax)
#endif
                    )
                {
                    flag = 1;
                }
                SA[j*ldsa] = hipComplexDoubleToFloat( tmp );
            }
        }
    }
}


/*
    Similar to zlat2c_full, but updates only the diagonal and above.
    Blocks that are fully below the diagonal exit immediately.
    
    Code similar to zlag2c and zlaset.
*/
__global__
void zlat2c_upper(
    int n,
    const magmaDoubleComplex *A, int lda,
    magmaFloatComplex *SA,       int ldsa,
    double rmax )
{
    magmaDoubleComplex tmp;
    double neg_rmax = - rmax;
    
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    /* check if full block-column && (above diag) */
    bool full = (iby + BLK_Y <= n && (ind + BLK_X <= iby));
    /* do only rows inside matrix, and blocks not below diag */
    if ( ind < n && ind < iby + BLK_Y ) {
        A  += ind + iby*lda;
        SA += ind + iby*ldsa;
        if ( full ) {
            // full block-column, off-diagonal block
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                tmp = A[j*lda];
                if (   (hipCreal(tmp) < neg_rmax) || (hipCreal(tmp) > rmax)
#if defined(PRECISION_z) || defined(PRECISION_c)
                    || (hipCimag(tmp) < neg_rmax) || (hipCimag(tmp) > rmax)
#endif
                    )
                {
                    flag = 1;
                }
                SA[j*ldsa] = hipComplexDoubleToFloat( tmp );
            }
        }
        else {
            // either partial block-column or diagonal block
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
                if ( ind <= iby+j ) {
                    tmp = A[j*lda];
                    if (   (hipCreal(tmp) < neg_rmax) || (hipCreal(tmp) > rmax)
#if defined(PRECISION_z) || defined(PRECISION_c)
                         || (hipCimag(tmp) < neg_rmax) || (hipCimag(tmp) > rmax)
#endif
                        )
                    {
                        flag = 1;
                    }
                    SA[j*ldsa] = hipComplexDoubleToFloat( tmp );
                }
            }
        }
    }
}


/**
    Purpose
    -------
    ZLAT2C converts a double-complex matrix, A,
                 to a single-complex matrix, SA.
    
    RMAX is the overflow for the single-complex arithmetic.
    ZLAT2C checks that all the entries of A are between -RMAX and
    RMAX. If not, the conversion is aborted and a flag is raised.
        
    Arguments
    ---------
    @param[in]
    uplo    magma_uplo_t
            Specifies the part of the matrix A to be converted.
      -     = MagmaUpper:      Upper triangular part
      -     = MagmaLower:      Lower triangular part
    
    @param[in]
    n       INTEGER
            The number of columns of the matrix A.  n >= 0.
    
    @param[in]
    A       COMPLEX_16 array, dimension (LDA,n)
            On entry, the n-by-n coefficient matrix A.
    
    @param[in]
    lda     INTEGER
            The leading dimension of the array A.  LDA >= max(1,n).
    
    @param[out]
    SA      COMPLEX array, dimension (LDSA,n)
            On exit, if INFO=0, the n-by-n coefficient matrix SA;
            if INFO > 0, the content of SA is unspecified.
    
    @param[in]
    ldsa    INTEGER
            The leading dimension of the array SA.  LDSA >= max(1,n).
    
    @param[out]
    info    INTEGER
      -     = 0:  successful exit.
      -     < 0:  if INFO = -i, the i-th argument had an illegal value
      -     = 1:  an entry of the matrix A is greater than the COMPLEX
                  overflow threshold, in this case, the content
                  of SA on exit is unspecified.
    
    @param[in]
    stream  magma_queue_t
            Stream to execute in.

    @ingroup magma_zaux2
    ********************************************************************/
extern "C" void
magmablas_zlat2c_stream(
    magma_uplo_t uplo, magma_int_t n,
    const magmaDoubleComplex *A, magma_int_t lda,
    magmaFloatComplex *SA,       magma_int_t ldsa,
    magma_queue_t stream,
    magma_int_t *info )
{
    *info = 0;
    if ( uplo != MagmaLower && uplo != MagmaUpper )
        *info = -1;
    else if ( n < 0 )
        *info = -2;
    else if ( lda < max(1,n) )
        *info = -4;
    else if ( ldsa < max(1,n) )
        *info = -6;
    
    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return; //*info;
    }

    /* quick return */
    if ( n == 0 ) {
        return;
    }
    
    double rmax = (double)lapackf77_slamch("O");

    dim3 threads( BLK_X );
    dim3 grid( (n+BLK_X-1)/BLK_X, (n+BLK_Y-1)/BLK_Y );
    hipMemcpyToSymbol(HIP_SYMBOL( flag), info, sizeof(flag) );    // flag = 0
    
    if (uplo == MagmaLower)
        zlat2c_lower<<< grid, threads, 0, stream >>> (n, A, lda, SA, ldsa, rmax);
    else if (uplo == MagmaUpper)
        zlat2c_upper<<< grid, threads, 0, stream >>> (n, A, lda, SA, ldsa, rmax);
    
    hipMemcpyFromSymbol( info, HIP_SYMBOL(flag), sizeof(flag) );  // info = flag
}


/**
    @see magmablas_zlat2c_stream
    @ingroup magma_zaux2
    ********************************************************************/
extern "C" void
magmablas_zlat2c(
    magma_uplo_t uplo, magma_int_t n,
    const magmaDoubleComplex *A, magma_int_t lda,
    magmaFloatComplex *SA,       magma_int_t ldsa,
    magma_int_t *info )
{
    magmablas_zlat2c_stream( uplo, n, A, lda, SA, ldsa, magma_stream, info );
}
