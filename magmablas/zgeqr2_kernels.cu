#include "hip/hip_runtime.h"
/*
   -- MAGMA (version 1.5) --
   Univ. of Tennessee, Knoxville
   Univ. of California, Berkeley
   Univ. of Colorado, Denver
   @date

   @author Azzam Haidar
   @author Tingxing Dong

   @precisions normal z -> s d c
*/

#include "common_magma.h"
#include "batched_kernel_param.h"



__global__ void
zgeqrf_copy_upper_kernel_batched(                
                  int n, int nb,
                  magmaDoubleComplex **dV_array,    int ldv,
                  magmaDoubleComplex **dR_array,    int ldr)
{
    magmaDoubleComplex *dV = dV_array[blockIdx.x];
    magmaDoubleComplex *dR = dR_array[blockIdx.x];

    int tid = threadIdx.x;

    int column = (tid / nb + 1) * nb; 
    
    if ( tid < n && column < n) 
    {
        for (int i=column; i < n; i++)
        {
            dR[tid + i * ldr]  =  dV[tid + i * ldv];  
        }
    }
}
/*

    Purpose
    -------
    These are internal routines that might have many assumption.
    They are used in zgeqrf_batched.cpp   

    Copy part of the data in dV to dR
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The order of the matrix .  N >= 0.

    @param[in]
    nb      INTEGER
            Tile size in matrix.  nb <= N.

    @param[in]
    dV_array    Array of pointers, dimension (batchCount).
            Each is a COMPLEX_16 array on the GPU, dimension (LDDA,N).

    @param[in]
    lddv    INTEGER
            The leading dimension of each array V.  LDDV >= max(1,N).


    @param[in,out]
    dR_array    Array of pointers, dimension (batchCount).
            Each is a COMPLEX_16 array on the GPU, dimension (LDDR,N).

    @param[in]
    lddr    INTEGER
            The leading dimension of each array R.  LDDR >= max(1,N).

    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_zgeqrf_comp

*/

void zgeqrf_copy_upper_batched(                
                  magma_int_t n, magma_int_t nb,
                  magmaDoubleComplex **dV_array,    magma_int_t ldv,
                  magmaDoubleComplex **dR_array,    magma_int_t ldr,
          magma_int_t batchCount, magma_queue_t queue)
{
    /* 
        copy some data in dV to dR
    */
    if ( nb >= n) return;
    
    zgeqrf_copy_upper_kernel_batched<<<batchCount, n, 0, queue>>>(n, nb, dV_array, ldv, dR_array, ldr);
}
