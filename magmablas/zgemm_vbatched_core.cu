/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z

       @author Jakub Kurzak
       @author Stan Tomov
       @author Mark Gates
       @author Azzam Haidar
       @author Ahmad Abdelfattah
       
*/

#include "magma_internal.h"
#define PRECISION_z
#include "gemm_template_kernel_vbatched.cuh"

#include "gemm_config/zgemm_param_nn.h"
#include "gemm_config/zgemm_param_nt.h"
#include "gemm_config/zgemm_param_tn.h"
#include "gemm_config/zgemm_param_tt.h"

#define version(s,v) s ## _V_ ## v

#ifdef __cplusplus
extern "C" 
#endif
void 
magmablas_zgemm_vbatched_core(
    magma_trans_t transA, magma_trans_t transB, 
    magma_int_t* m, magma_int_t* n, magma_int_t* k,
    magmaDoubleComplex alpha,
    magmaDoubleComplex const * const * dA_array, magma_int_t* ldda,
    magmaDoubleComplex const * const * dB_array, magma_int_t* lddb,
    magmaDoubleComplex beta,
    magmaDoubleComplex **dC_array, magma_int_t* lddc, 
    magma_int_t max_m, magma_int_t max_n, magma_int_t max_k, 
    magma_int_t roffA, magma_int_t coffA, magma_int_t roffB, magma_int_t coffB, magma_int_t roffC, magma_int_t coffC, 
    magma_int_t spec_m, magma_int_t spec_n, magma_int_t spec_k, 
    magma_int_t batchCount, magma_queue_t queue )
{
    if(max_m <=0 || max_n <= 0 || max_k <= 0) return;
    
    magma_int_t shape = 0;
    if      (transA == MagmaNoTrans   && transB == MagmaNoTrans)   {shape = 0;} // nn
    else if (transA == MagmaNoTrans   && transB == MagmaTrans)     {shape = 1;} // nt
    else if (transA == MagmaNoTrans   && transB == MagmaConjTrans) {shape = 2;} // nc
    else if (transA == MagmaTrans     && transB == MagmaNoTrans)   {shape = 3;} // tn
    else if (transA == MagmaTrans     && transB == MagmaTrans)     {shape = 4;} // tt
    else if (transA == MagmaTrans     && transB == MagmaConjTrans) {shape = 5;} // tc
    else if (transA == MagmaConjTrans && transB == MagmaNoTrans)   {shape = 6;} // cn
    else if (transA == MagmaConjTrans && transB == MagmaTrans)     {shape = 7;} // ct
    else if (transA == MagmaConjTrans && transB == MagmaConjTrans) {shape = 8;} // cc
    
    switch(shape)
    {
        case 0: // nn
            {
                gemm_template_vbatched_nn<magmaDoubleComplex, version(NN,18), 0, 0>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
            }
            break;
        case 1: // nt
            {
                if(max_k <= 8)
                {
                    // version 58
                    gemm_template_vbatched_nt<magmaDoubleComplex, version(NT,58), 0, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                }
                else
                {
                    // version 29
                    gemm_template_vbatched_nt<magmaDoubleComplex, version(NT,29), 0, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                }
            }
            break;
        case 2: // nc
            {
                if(max_k <= 8)
                {
                    // version 58
                    gemm_template_vbatched_nt<magmaDoubleComplex, version(NT,58), 0, 1>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                }
                else
                {
                    // version 29
                    gemm_template_vbatched_nt<magmaDoubleComplex, version(NT,29), 0, 1>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                }
            }
            break;
        case 3: // tn
            {
                // version 72
                gemm_template_vbatched_tn<magmaDoubleComplex, version(TN,72), 0, 0>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
            }
            break;
        case 6: // cn
            {
                // version 72
                gemm_template_vbatched_tn<magmaDoubleComplex, version(TN,72), 1, 0>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
            }
            break;
        case 4: // tt
            {
                // version 13
                gemm_template_vbatched_tt<magmaDoubleComplex, version(TT,13), 0, 0>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
            }
            break;
        case 5: // tc
            {
                // version 13
                gemm_template_vbatched_tt<magmaDoubleComplex, version(TT,13), 0, 1>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
            }
            break;
        case 7: // ct
            {
                // version 13
                gemm_template_vbatched_tt<magmaDoubleComplex, version(TT,13), 1, 0>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
            }
            break;
        case 8: // cc
            {
                // version 13
                gemm_template_vbatched_tt<magmaDoubleComplex, version(TT,13), 1, 1>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
            }
            break;
        default:; // propose something
    }
}
///////////////////////////////////////////////////////////////////////////////////////////////////
