#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010
*/
#include "common_magma.h"
#include "commonblas_s.h"

static __device__ void saxpy(float a,float *b, float *c) {
        c[0] += a * b[0];
        c[1] += a * b[1];
        c[2] += a * b[2];
        c[3] += a * b[3];
        c[4] += a * b[4];
        c[5] += a * b[5];
        c[6] += a * b[6];
        c[7] += a * b[7];
        c[8] += a * b[8];
        c[9] += a * b[9];
        c[10] += a * b[10];
        c[11] += a * b[11];
        c[12] += a * b[12];
        c[13] += a * b[13];
        c[14] += a * b[14];
        c[15] += a * b[15];
}

extern "C" __global__ void 
sgemm_kernel_N_N_64_16_16_16_4_special(float *C, const float *A, const float *B, 
                                       int m, int n, int k, 
                                       int lda, int ldb, int ldc, 
                                       float alpha, float beta)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose:
    ========
    This routine computes
       C = alpha* A*B  + beta * C

    B is put into shared memory
    Parameters Used:
        blk_M=64 blk_N=16 blk_K=16 nthd_x=16 nthd_y=4

    This kernel is for matrices devisible by the corresponding
    blocking sizes.
    ===============================================================  */

        const int tx = threadIdx.x;
        const int ty = threadIdx.y;

        const int ibx = blockIdx.x * 64;
        const int iby = blockIdx.y *16;

        const int idt = ty * 16 + tx;

        B+=tx+__mul24(iby+ty,ldb);
        A += ibx + idt;
        C += ibx +idt +__mul24( iby,ldc);

        const float *Bend = B + k;


        float Cb[16] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
        m = 2*lda ; 
        n = 3*lda ;

        do {
                //float Ab[4] = {A[0], A[lda], A[2*lda], A[3*lda]};
                float Ab[4] = {A[0], A[lda], A[m], A[n]};
                __shared__ float Bb[16][17];
                Bb[tx][ty+0] = B[0];
                Bb[tx][ty+4] = B[4*ldb];
                Bb[tx][ty+8] = B[8*ldb];
                Bb[tx][ty+12] = B[12*ldb];

                __syncthreads();

                A += 4 * lda;
                saxpy(Ab[0], &Bb[0][0], Cb); Ab[0] = A[0];
                saxpy(Ab[1], &Bb[1][0], Cb); Ab[1] = A[lda];
                saxpy(Ab[2], &Bb[2][0], Cb); Ab[2] = A[m];
                saxpy(Ab[3], &Bb[3][0], Cb); Ab[3] = A[n];

                A += 4 * lda;
                saxpy(Ab[0], &Bb[4][0], Cb); Ab[0] = A[0];
                saxpy(Ab[1], &Bb[5][0], Cb); Ab[1] = A[lda];
                saxpy(Ab[2], &Bb[6][0], Cb); Ab[2] = A[m];
                saxpy(Ab[3], &Bb[7][0], Cb); Ab[3] = A[n];

                A += 4 * lda;
                saxpy(Ab[0], &Bb[8][0], Cb); Ab[0] = A[0];
                saxpy(Ab[1], &Bb[9][0], Cb); Ab[1] = A[lda];
                saxpy(Ab[2], &Bb[10][0], Cb); Ab[2] = A[m];
                saxpy(Ab[3], &Bb[11][0], Cb); Ab[3] = A[n];

                A += 4 * lda;
                saxpy(Ab[0], &Bb[12][0], Cb);
                saxpy(Ab[1], &Bb[13][0], Cb);
                saxpy(Ab[2], &Bb[14][0], Cb);
                saxpy(Ab[3], &Bb[15][0], Cb);

                B += 16;

                __syncthreads();
        } while (B < Bend);

        #pragma unroll 16
        for (int i = 0; i < 16; i++, C += ldc) {
                C[0] =alpha*Cb[i] + beta * C[0];
        }
}

extern "C" void
magmablas_sgemm_kernel_N_N_64_16_16_16_4_special(float *C, 
                                                 const float *A, 
                                                 const float *B,
                                                 int m, int n, int k,
                                                 int lda, int ldb, int ldc, 
                                                 float alpha, float beta)
{
        dim3 threads( 16, 4 );
        dim3 grid(m/64,n/16);
        sgemm_kernel_N_N_64_16_16_16_4_special<<< grid, threads, 0, magma_stream >>>(C, A, B, 
                                                                    m, n, k,
                                                                    lda, ldb, ldc, 
                                                                    alpha, beta);
}

