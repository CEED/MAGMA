#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011
       
       @precisions normal s -> s

       @author Peng Du
*/
#include "common_magma.h"

#define qmod(a,b) ((a)-(__mul24((b),(a)/(b))))
//#define hipblasSgemm magmablas_sgemm_fermi64

extern "C"
void diag_strtri (magma_int_t M, char uplo, char diag, const float *A, float *d_dinvA, magma_int_t lda);

__global__ void
b_copy_kernel (int M, int N, float *b, int ldb, float *d_x, int ldx);
#define b_copy();        dim3 dimBlock((M>=MAX_THREAD_PER_BLOCK)?MAX_THREAD_PER_BLOCK:(WARP_SIZE*((M/WARP_SIZE)+(M%WARP_SIZE!=0))), 1);\
                                        dim3 dimGrid(M/dimBlock.x+(M%dimBlock.x!=0), N);\
                                        b_copy_kernel<<< dimGrid, dimBlock, 0, magma_stream >>>(M, N, b, ldb, d_x, M);

#define MAX_THREAD_PER_BLOCK 512
#define WARP_SIZE 32

#define BLOCK_SIZE 16 // inner blocking size, <=32
#define NB 128// outer blocking size, >BLOCK_SIZE 


/*
 * magmablas_strsm
 */
extern "C"
void magmablas_strsm_work( char side, char uplo, char tran, char diag, magma_int_t M, magma_int_t N, 
                           float alpha, const float* A, magma_int_t lda, float* b, magma_int_t ldb,
                           float *d_dinvA, float *d_x)
{
        /*  -- MAGMA (version 1.1) --
                Univ. of Tennessee, Knoxville
                Univ. of California, Berkeley
                Univ. of Colorado, Denver
                November 2011

                Purpose
                =======

                strsm  solves one of the matrix equations on gpu

                op( a )*x = alpha*b,   or   x*op( a ) = alpha*b,

                where alpha is a scalar, x and b are m by n matrices, a is a unit, or
                non-unit,  upper or lower triangular matrix  and  op( a )  is one  of

                op( A ) = A   or   op( A ) = A'.

                The matrix X is overwritten on B.

                When M or N is not a multiple of blocking size, which is 32 for now, hipblasStrsm will
                be called instead. There soon will not be this limitation both for arbitrary problem 
                size and blocking size.
           
                This is an asynchronous version of magmablas_strsm with "workspace" as an argument.

           Arguments
           ==========
           
           side   - CHARACTER*1.
                    On entry, side specifies whether op( A ) appears on the left
                    or right of X as follows:
           
                       side = 'L' or 'l'   op( A )*X = alpha*B.
           
                       side = 'R' or 'r'   X*op( A ) = alpha*B.
           
                    Unchanged on exit.
           
           uplo   - CHARACTER*1.
                    On entry, uplo specifies whether the matrix A is an upper or
                    lower triangular matrix as follows:
           
                       uplo = 'U' or 'u'   A is an upper triangular matrix.
           
                       uplo = 'L' or 'l'   A is a lower triangular matrix.
           
                    Unchanged on exit.
           
           tran - CHARACTER*1.
                    On entry, tran specifies the form of op( A ) to be used in
                    the matrix multiplication as follows:
           
                       tran = 'N' or 'n'   op( A ) = A.
           
                       tran = 'T' or 't'   op( A ) = A'.
           
                       tran = 'C' or 'c'   op( A ) = A'.
           
                    Unchanged on exit.
           
           diag   - CHARACTER*1.
                    On entry, diag specifies whether or not A is unit triangular
                    as follows:
           
                       diag = 'U' or 'u'   A is assumed to be unit triangular.
           
                       diag = 'N' or 'n'   A is not assumed to be unit
                                           triangular.
           
                    Unchanged on exit.
           
           m      - INTEGER.
                    On entry, m specifies the number of rows of B. m must be at
                    least zero.
                    Unchanged on exit.
           
                                n      - INTEGER.
                     On entry, n specifies the number of columns of B.  n must be
                     at least zero.
                     Unchanged on exit.
           
            alpha  - REAL.
                     On entry,  alpha specifies the scalar  alpha. When  alpha is
                     zero then  A is not referenced and  B need not be set before
                     entry.
                     Unchanged on exit.
           
            A      - REAL             array of DIMENSION ( lda, k ), where k is m
                     when  side = 'L' or 'l'  and is  n  when  side = 'R' or 'r'.
                     Before entry  with  uplo = 'U' or 'u',  the  leading  k by k
                     upper triangular part of the array  A must contain the upper
                     triangular matrix  and the strictly lower triangular part of
                     A is not referenced.
                     Before entry  with  uplo = 'L' or 'l',  the  leading  k by k
                     lower triangular part of the array  A must contain the lower
                     triangular matrix  and the strictly upper triangular part of
                     A is not referenced.
                     Note that when  diag = 'U' or 'u',  the diagonal elements of
                     A  are not referenced either,  but are assumed to be  unity.
                     Unchanged on exit.
           
            lda    - INTEGER.
                     On entry, lda specifies the first dimension of A as declared
                     in the calling (sub) program.  When  side = 'L' or 'l'  then
                     lda  must be at least  max( 1, m ),  when  side = 'R' or 'r'
                     then lda must be at least max( 1, n ).
                     Unchanged on exit.
           
            b      - REAL             array of DIMENSION ( ldb, n ).
                     Before entry,  the leading  m by n part of the array  B must
                     contain  the  right-hand  side  matrix  B,  and  on exit  is
                     overwritten by the solution matrix  X.
           
            ldb    - INTEGER.
                     On entry, ldb specifies the first dimension of B as declared
                     in  the  calling  (sub)  program.   ldb  must  be  at  least
                     max( 1, m ).
                     Unchanged on exit.
           
           
            Level 3 Blas routine.
                *
    ===================================================================== */

        int i;
        //float *d_dinvA, *d_x;

        /* quick return on wrong size */
        if (M<=0 || N<=0)
                return;

        if (side == 'l' || side == 'L')
        {
                /* invert the diagonals
                 * Allocate device memory for the inverted diagonal blocks, size=m*NB
                 */
                //hipMalloc((void**)&d_dinvA, NB*((M/NB)+(M%NB!=0))*NB*sizeof(float));
                //hipMalloc((void**)&d_x, N*M*sizeof(float));
                //hipMemset (d_x, 0, N*M*sizeof(float));
                //hipMemset (d_dinvA, 0, NB*((M/NB)+(M%NB!=0))*NB*sizeof(float));
                diag_strtri (M, uplo, diag, A, d_dinvA, lda);

                if (tran == 'N' || tran == 'n')
                /* the non-transpose case */
                {
                        if (uplo == 'L' || uplo == 'l')
                        {
                        /* the lower case */
                                
                                /* handle the first block seperately with alpha */

                                int MM = min (NB, M); 
                                hipblasSgemm ('N', 'N', MM, N, MM, alpha, d_dinvA, NB, b, ldb, 0, d_x, M);  

                                if (NB>=M)
                                {
                                        b_copy();
                                        //hipFree(d_dinvA);
                                        //hipFree(d_x);
                                        return;
                                }

                                hipblasSgemm ('N', 'N', M-NB, N, NB, -1.0, A+NB, lda, d_x, M, alpha, b+NB, ldb);

                                /* the rest blocks */
                                for (i=NB; i<M; i+=NB)
                                {
                                        MM = min (M-i, NB);
                                        hipblasSgemm ('N', 'N', MM, N, MM, 1.0, d_dinvA+i*NB, NB, b+i, ldb, 0, d_x+i, M);  
                                        
                                        if (i+NB>=M)
                                                break;

                                        hipblasSgemm ('N', 'N', M-i-NB, N, NB, -1.0, A+i*lda+i+NB, lda, d_x+i, M, 1.0, b+i+NB, ldb);
                                }
                        }
                        else
                        {
                        /* the upper case */

                                /* handle the first block seperately with alpha */
                                int MM = (M%NB==0)?NB:(M%NB); 
                                i = M-MM;
                                hipblasSgemm ('N', 'N', MM, N, MM, alpha, d_dinvA+i*NB, NB, b+i, ldb, 0.0, d_x+i, M); 
                                        
                                if (i-NB<0)
                                {
                                        b_copy();
                                        //hipFree(d_dinvA);
                                        //hipFree(d_x);
                                        return;
                                }

                                hipblasSgemm ('N', 'N', i, N, MM, -1.0, A+i*lda, lda, d_x+i, M, alpha, b, ldb);

                                /* the rest blocks */
                                for (i=M-MM-NB; i>=0; i-=NB)
                                {
                                        hipblasSgemm ('N', 'N', NB, N, NB, 1.0, d_dinvA+i*NB, NB, b+i, ldb, 0.0, d_x+i, M);

                                        if (i-NB<0)
                                                break;

                                        hipblasSgemm ('N', 'N', i, N, NB, -1.0, A+i*lda, lda, d_x+i, M, 1.0, b, ldb);
                                }

                        }
                }
                else
                /* the transpose case */
                {
                        if (uplo == 'L' || uplo == 'l')
                        {
                        /* the lower case */
                                
                                /* handle the first block seperately with alpha */
                                int MM = (M%NB==0)?NB:(M%NB); 
                                i=M-MM; 
                                hipblasSgemm ('T', 'N', MM, N, MM, alpha, d_dinvA+i*NB, NB, b+i, ldb, 0, d_x+i, M);  

                                if (i-NB<0)
                                {
                                        b_copy();
                                        //hipFree(d_dinvA);
                                        //hipFree(d_x);
                                        return;
                                }

                                hipblasSgemm ('T', 'N', i, N, MM, -1.0, A+i, lda, d_x+i, M, alpha, b, ldb);

                                /* the rest blocks */
                                for (i=M-MM-NB; i>=0; i-=NB)
                                {
                                        hipblasSgemm ('T', 'N', NB, N, NB, 1.0, d_dinvA+i*NB, NB, b+i, ldb, 0, d_x+i, M);  

                                        if (i-NB<0)
                                                break;

                                        hipblasSgemm ('T', 'N', i, N, NB, -1.0, A+i, lda, d_x+i, M, 1.0, b, ldb);
                                }
                        }
                        else
                        {
                        /* the upper case */
                                        
                                /* handle the first block seperately with alpha */
                                int MM = min (NB, M); 
                                hipblasSgemm ('T', 'N', MM, N, MM, alpha, d_dinvA, NB, b, ldb, 0, d_x, M);  

                                if (NB>=M)
                                {
                                        b_copy();
                                        //hipFree(d_dinvA);
                                        //hipFree(d_x);
                                        return;
                                }

                                hipblasSgemm ('T', 'N', M-NB, N, NB, -1.0, A+(NB)*lda, lda, d_x, M, alpha, b+NB, ldb);

                                /* the rest blocks */
                                for (i=NB; i<M; i+=NB)
                                {
                                        MM = min (M-i, NB);
                                        hipblasSgemm ('T', 'N', MM, N, MM, 1.0, d_dinvA+i*NB, NB, b+i, ldb, 0, d_x+i, M);  
                                        
                                        if (i+NB>=M)
                                                break;

                                        hipblasSgemm ('T', 'N', M-i-NB, N, NB, -1.0, A+(i+NB)*lda+i, lda, d_x+i, M, 1.0, b+i+NB, ldb);
                                }
                        }
                }
        }
        else
        {        // side=R

                /* invert the diagonals
                 * Allocate device memory for the inverted diagonal blocks, size=N*BLOCK_SIZE 
                 */
                //hipMalloc((void**)&d_dinvA, NB*((N/NB)+(N%NB!=0))*NB*sizeof(float));
                //hipMalloc((void**)&d_x, N*M*sizeof(float));
                //hipMemset (d_x, 0, N*M*sizeof(float));
                //hipMemset (d_dinvA, 0, NB*((N/NB)+(N%NB!=0))*NB*sizeof(float));
                diag_strtri (N, uplo, diag, A, d_dinvA, lda);

                if (tran == 'N' || tran == 'n')
                /* the non-transpose case */
                {
                        if (uplo == 'L' || uplo == 'l')
                        {
                        /* the lower case */
                                
                                /* handle the first block seperately with alpha */
                                int NN = (N%NB==0)?NB:(N%NB);
                                i=N-NN;
                                hipblasSgemm ('N', 'N', M, NN, NN, alpha, b+ldb*i, ldb, d_dinvA+i*NB, NB, 0.0, d_x+i*M, M); 

                                if (i-NB<0)
                                {
                                        b_copy();
                                        //hipFree(d_x);
                                        //hipFree(d_dinvA);
                                        return;
                                }

                                hipblasSgemm ('N', 'N', M, i, NN, -1.0, d_x+i*M, M, A+i, lda, alpha, b, ldb);

                                /* the rest blocks */
                                for (i=N-NN-NB; i>=0; i-=NB)
                                {
                                        hipblasSgemm ('N', 'N', M, NB, NB, 1.0, b+ldb*i, ldb, d_dinvA+i*NB, NB, 0.0, d_x+i*M, M); 
                                        
                                        if (i-NB<0)
                                                break;

                                        hipblasSgemm ('N', 'N', M, i, NB, -1.0, d_x+i*M, M, A+i, lda, 1.0, b, ldb);
                                }
                        }
                        else
                        {
                        /* the upper case */
                                
                                /* handle the first block seperately with alpha */
                                int NN = min(NB, N); 
                                hipblasSgemm ('N', 'N', M, NN, NN, alpha, b, ldb, d_dinvA, NB, 0, d_x, M);  

                                if (NB>=N)
                                {
                                        b_copy();
                                        //hipFree(d_x);
                                        //hipFree(d_dinvA);
                                        return;
                                }

                                hipblasSgemm ('N', 'N', M, N-NB, NB, -1.0, d_x, M, A+NB*lda, lda, alpha, b+NB*ldb, ldb);
                                
                                /* the rest blocks */
                                for (i=NB; i<N; i+=NB)
                                {
                                        NN = min(NB, N-i); 
                                        hipblasSgemm ('N', 'N', M, NN, NN, 1.0, b+ldb*i, ldb, d_dinvA+i*NB, NB, 0, d_x+i*M, M);  

                                        if (i+NB>=N)
                                                break;

                                        hipblasSgemm ('N', 'N', M, N-i-NB, NB, -1.0, d_x+i*M, M,   A+(i+NB)*lda+i, lda, 1.0, b+(i+NB)*ldb, ldb);
                                }
                        }
                }
                else
                /* the transpose case */
                {
                        if (uplo == 'L' || uplo == 'l')
                        {
                        /* the lower case */
                                
                                /* handle the first block seperately with alpha */
                                int NN = min(NB, N); 
                                hipblasSgemm ('N', 'T', M, NN, NN, alpha, b, ldb, d_dinvA, NB, 0, d_x, M);  

                                if (NB>=N)
                                {
                                        b_copy();
                                        //hipFree(d_x);
                                        //hipFree(d_dinvA);
                                        return;
                                }

                                hipblasSgemm ('N', 'T', M, N-NB, NB, -1.0, d_x, M, A+NB, lda, alpha, b+NB*ldb, ldb);

                                /* the rest blocks */
                                for (i=NB; i<N; i+=NB)
                                {
                                        NN = min(NB, N-i); 
                                        hipblasSgemm ('N', 'T', M, NN, NN, 1.0, b+ldb*i, ldb, d_dinvA+i*NB, NB, 0, d_x+i*M, M);  

                                        if (i+NB>=N)
                                                break;

                                        hipblasSgemm ('N', 'T', M, N-i-NB, NB, -1.0, d_x+i*M, M,   A+i*lda+NB+i, lda, 1.0, b+(i+NB)*ldb, ldb);
                                }
                        }
                        else
                        {
                        /* the upper case */
                                
                                /* handle the first block seperately with alpha */
                                int NN = (N%NB==0)?NB:(N%NB);
                                i=N-NN;
                                hipblasSgemm ('N', 'T', M, NN, NN, alpha, b+ldb*i, ldb, d_dinvA+i*NB, NB, 0.0, d_x+i*M, M); 

                                if (i-NB<0)
                                {
                                        b_copy();
                                        //hipFree(d_x);
                                        //hipFree(d_dinvA);
                                        return;
                                }

                                hipblasSgemm ('N', 'T', M, i, NN, -1.0, d_x+i*M, M, A+i*lda, lda, alpha, b, ldb);
                                
                                /* the rest blocks */
                                for (i=N-NN-NB; i>=0; i-=NB)
                                {
                                        hipblasSgemm ('N', 'T', M, NB, NB, 1.0, b+ldb*i, ldb, d_dinvA+i*NB, NB, 0.0, d_x+i*M, M); 

                                        if (i-NB<0)
                                                break;

                                        hipblasSgemm ('N', 'T', M, i, NB, -1.0, d_x+i*M, M, A+i*lda, lda, 1.0, b, ldb);
                                }
                        }
                }
        }
                
        b_copy();
        //hipFree(d_dinvA);
        //hipFree(d_x);
}

