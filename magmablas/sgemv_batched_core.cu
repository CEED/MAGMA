#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @author Tingxing Dong
       @author Azzam Haidar

*/
#include "magma_internal.h"
#include "magma_templates.h"

#define PRECISION_s

#include "gemv_template_kernel_batched.cuh"
#include "gemv_config/gemvn_param.h"
#include "gemv_config/gemvt_param.h"

#define version(s,v) s ## _V_ ## v


/***************************************************************************//**
    Purpose
    -------
    SGEMV performs one of the matrix-vector operations
    
        y := alpha*A*x    + beta*y,   or
        y := alpha*A**T*x + beta*y,   or
        y := alpha*A**H*x + beta*y,
    
    where alpha and beta are scalars, x and y are vectors and A is an
    m by n matrix.

    Arguments
    ----------
    @param[in]
    trans   magma_trans_t
            On entry, TRANS specifies the operation to be performed as
            follows:
      -     = MagmaNoTrans:    y := alpha*A  *x + beta*y
      -     = MagmaTrans:      y := alpha*A^T*x + beta*y
      -     = MagmaConjTrans:  y := alpha*A^H*x + beta*y

    @param[in]
    m       INTEGER
            On entry, m specifies the number of rows of the matrix A.

    @param[in]
    n       INTEGER
            On entry, n specifies the number of columns of the matrix A
 
    @param[in]
    alpha   REAL
            On entry, ALPHA specifies the scalar alpha.


    @param[in]
    dA_array     Array of pointers, dimension (batchCount).
             Each is a REAL array A of DIMENSION ( ldda, n ) on the GPU
   
    @param[in]
    ldda    INTEGER
            LDDA specifies the leading dimension of A.

    @param[in]
    dx_array     Array of pointers, dimension (batchCount).
            Each is a REAL array of dimension
            n if trans == MagmaNoTrans
            m if trans == MagmaTrans or MagmaConjTrans
     
    @param[in]
    incx    Specifies the increment for the elements of X.
            INCX must not be zero.
  
    @param[in]
    beta    REAL
            On entry, ALPHA specifies the scalar beta. When BETA is
            supplied as zero then Y need not be set on input.

    @param[out]
    dy_array     Array of pointers, dimension (batchCount).
            Each is a REAL array of dimension
            m if trans == MagmaNoTrans
            n if trans == MagmaTrans or MagmaConjTrans

    @param[in]
    incy    Specifies the increment for the elements of Y.
            INCY must not be zero.

    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_gemv_batched
*******************************************************************************/
extern "C" void
magmablas_sgemv_batched(
    magma_trans_t trans, magma_int_t m, magma_int_t n, 
    float alpha,
    magmaFloat_ptr dA_array[], magma_int_t ldda, 
    magmaFloat_ptr dx_array[], magma_int_t incx,
    float beta,
    magmaFloat_ptr dy_array[], magma_int_t incy, 
    magma_int_t batchCount, magma_queue_t queue)
{
    magma_int_t info = 0;
    if ( trans != MagmaNoTrans && trans != MagmaTrans && trans != MagmaConjTrans )
        info = -1;
    else if ( m < 0 )
        info = -2;
    else if ( n < 0 )
        info = -3;
    else if ( ldda < m )
        info = -6;
    else if ( incx == 0 )
        info = -8;
    else if ( incy == 0 )
        info = -11;
    
    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }    
                                                                                   
    if ( trans == MagmaNoTrans ) {                                                   
        if (max(m, n) <= 96) { // small size                         
            if (m < n) { // Fat matrix
                if ( m <= 8) 
                {
                    gemvn_template_batched<float, version(N, 32)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else if ( m <= 16)            
                {
                    gemvn_template_batched<float, version(N, 72)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else if ( m <= 32)            
                {
                    gemvn_template_batched<float, version(N, 97)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else if ( m <= 64)            
                {
                    gemvn_template_batched<float, version(N, 120)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else
                {
                    gemvn_template_batched<float, version(N, 130)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }  
            }
            else {   // Tall matrix
                if ( n <= 16) 
                {    
                    gemvn_template_batched<float, version(N, 118)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else 
                {
                    gemvn_template_batched<float, version(N, 120)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }  
            }
        }
        else { // big size
            if (m < n) { // Fat matrix
                if (m <= 16)
                {
                    gemvn_template_batched<float, version(N, 79)>              
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else if (m <= 32)
                {
                    gemvn_template_batched<float, version(N, 103)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else if (m <= 64)
                {
                    gemvn_template_batched<float, version(N, 126)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else
                {
                    gemvn_template_batched<float, version(N, 135)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
            }
            else { // Tall or square matrix
                if (m <= 256)
                {
                    gemvn_template_batched<float, version(N, 137)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else
                {
                    gemvn_template_batched<float, version(N, 140)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
            }
        }// big size        
    }
    else {
        if (max(m, n) <= 96) // small size
        {                              
            gemvc_template_batched<float, version(T, 46)>             
                ( trans, m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );    
        }
        else // big size
        {
            if (m <= n) //  Fat or square matrix
            {    
                if (m <= 64)
                {
                    gemvc_template_batched<float, version(T, 47)>             
                        ( trans, m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else
                {
                    gemvc_template_batched<float, version(T, 133)>             
                        ( trans, m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
            }                           
            else// (m > n) Tall matrix
            {
                if (n <= 8)
                {
                    gemvc_template_batched<float, version(T, 130)>             
                        ( trans, m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else
                {
                    gemvc_template_batched<float, version(T, 131)>             
                        ( trans, m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
            }
        }        
    }                   
}
