#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal d

*/
#include "common_magma.h"

#define magmablas_dgemv_fermi magmablas_dgemv

#define num_threads 64
#define dgemv_bs 64
#define threadSize 128



__global__ void 
dgemvn_kernel_fermi(magma_int_t n, magma_int_t m, magma_int_t n1, double alpha,  double* A, magma_int_t lda, double *x, double *y)
{
  magma_int_t ind = blockIdx.x*num_threads + threadIdx.x;

  A += ind;
  x += threadIdx.x;


  double res = 0.0;

  __shared__ double buff[dgemv_bs];
  for(magma_int_t i=0; i<n1; i += dgemv_bs ){
    __syncthreads();
    buff[threadIdx.x]  = x[i];

    __syncthreads();
    #pragma unroll
    for(magma_int_t j=0; j < dgemv_bs ; j++){
       res+=A[0]*buff[j];
       A+=lda;
    }
  }
  __syncthreads();

  if (m>n1){
     buff[threadIdx.x]  = x[n1];

     __syncthreads();
     for(magma_int_t j=0; j<(m-n1); j++){
         res += A[0]*buff[j];
         A+=lda;
     }
  }

  if (ind<n)
     y[ind] = alpha * res;
}


extern "C" void
magmablas_dgemvn_fermi(magma_int_t n, magma_int_t m, double alpha, double *A, magma_int_t lda, double *x, double *y)
{
/*  -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

    Purpose
    =======

    This routine computes y = alpha A x on the GPU.

    N      - (input) INTEGER.
             On entry, N specifies the number of rows of the matrix A.

    M      - (input) INTEGER.
             On entry, M specifies the number of columns of the matrix A

    A      - (input) DOUBLE PRECISION array of dimension ( LDA, m ) on the GPU.
   
    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) DOUBLE PRECISION array of dimension m.
     
    Y      - (output) DOUBLE PRECISION array of        dimension m. 
             On exit Y = alpha A X.

    ===================================================================== */

    magma_int_t blocks;
    if (n % num_threads==0)
        blocks = n/num_threads;
    else
        blocks = n/num_threads + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(num_threads, 1, 1);
 
    dgemvn_kernel_fermi<<< grid, threads, 0, magma_stream >>>(n, m, (m / dgemv_bs)*dgemv_bs, 
                                    alpha, A, lda, x, y);
}


__global__ void
dgemvt_kernel_fermi(magma_int_t m, magma_int_t n, double alpha, magma_int_t n1, double* A, magma_int_t lda,
              double *x, double *y)
{
        magma_int_t tx = threadIdx.x;

        __shared__ double sdata[threadSize];


        double res;
        res = 0.0;

        for(magma_int_t i=0; i<n1; i+= threadSize)
        {
                res += A[tx + i + lda * blockIdx.y] * x[tx + i];
        }

        if(m > n1)
        {
                if( tx + n1 <  m )
                {
                        res  += A[tx + n1 + lda *blockIdx.y] * x[tx + n1];
                }
                else
                {
                        res  = res;
                }
    }

        sdata[tx] = res;
         __syncthreads();

        for(int s=blockDim.x/2; s>32;s>>=1)
        {
                        if(tx<s)
                        {
                                        sdata[tx] += sdata[tx+s];
                        } 
                        __syncthreads();
        }

        if(tx<32)
        {
                sdata[tx] += sdata[tx+32];
        }

    if(tx == 0)
        {
                for(int i=1;i<32;i++)
                {
                        sdata[tx] += sdata[tx + i];
                }
        }

        if( tx == 0 )
        {
                y[blockIdx.y] = sdata[0];
                
                if (blockIdx.y < n)
                {
                        y[blockIdx.y] = y[blockIdx.y] * alpha;
                }
        }
}




extern "C" void
magmablas_dgemvt_fermi(magma_int_t m, magma_int_t n, double alpha, double *A, magma_int_t lda,
                 double *x, double *y)
{
/*  -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

    Purpose
    =======

    This routine computes y = alpha A^t x on the GPU.

    M      - (input) INTEGER.
             On entry, m specifies the number of rows of the matrix A.

    N      - (input) INTEGER.
             On entry, n specifies the number of columns of the matrix A

    A      - (input) DOUBLE PRECISION array of dimension ( LDA, n ) on the GPU.

    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) DOUBLE PRECISION array of dimension m.

    Y      - (output) DOUBLE PRECISION array of dimension n.
             On exit y = alpha A^t X.

    ===================================================================== */

        dim3 grid    ( 1,  n,  1);
        dim3 threads ( threadSize,   1,  1);

        dgemvt_kernel_fermi<<< grid, threads, 0, magma_stream >>>( m, n, alpha, ( m / threadSize) * threadSize,
                                                                       A, lda, x, y);
}



extern "C" 
void magmablas_dgemv_fermi(char trans,
                           magma_int_t m, magma_int_t n,
                           double alpha, 
                           double *A, magma_int_t lda, 
                           double *x, magma_int_t incx,
                           double beta,
                           double *z, magma_int_t incz)
{
/*  -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

    Purpose
    =======
    This routine computes:
    1) z =       A   x    if trans == 'N' or 'n', alpha == 1, beta == 0, 
                          and incx == incz == 1 (using magmablas code)
    2) z = alpha A^t x    if trans == 'T' or 't', beta == 0,
                          and incx == incz == 1 (using magmablas code)
    3) z = alpha A^trans x + beta z
                          otherwise, using CUBLAS.

   Arguments
   ==========
    TRANS  - CHARACTER*1
             On entry, TRANS specifies the operation to be performed as
             follows:
               TRANS = 'N' or 'n'   z := alpha*A *x + beta*z
               TRANS = 'T' or 't'   z := alpha*A'*x + beta*z

    M      - (input) INTEGER
             On entry, m specifies the number of rows of the matrix A.

    N      - (input) INTEGER
             On entry, n specifies the number of columns of the matrix A
 
    ALPHA  - DOUBLE REAL
             On entry, ALPHA specifies the scalar alpha.
             Unchanged on exit.

    A      - (input) DOUBLE PRECISION array of dimension ( LDA, n ) on the GPU.
   
    LDA    - (input) INTEGER
             LDA specifies the leading dimension of A.

    X      - (input) DOUBLE PRECISION array of dimension 
             n if trans == 'n'
             m if trans == 't'
     
    INCX   - (input) Specifies the increment for the elements of X.
             INCX must not be zero. Unchanged on exit.
  
    BETA   - DOUBLE REAL
             On entry, BETA specifies the scalar beta. When BETA is
             supplied as zero then Y need not be set on input.
             Unchanged on exit.

    Z      - (output) DOUBLE PRECISION array of        dimension 
             m if trans == 'n'
             n if trans == 't' 

    INCZ  - (input) Specifies the increment for the elements of Z.
            INCZ must not be zero. Unchanged on exit.
    ===================================================================== */

    if (incx == 1 && incz == 1 && beta == 0.) {
       if (trans == 'n' || trans == 'N')
           {
               if ( m >= 7000 && m <= 8000 )
                hipblasDgemv(trans, m, n, alpha, A, lda, x, incx, beta, z, incz);
                   else 
                                magmablas_dgemvn_fermi(m,  n, alpha, A, lda, x, z);
           }
       else if (trans == 't' || trans == 'T')
          magmablas_dgemvt_fermi(m,  n, alpha, A, lda, x, z);
       else
          printf("trans = %c in sgemv_fermi is not available\n", trans);               
    }
    else
       hipblasDgemv(trans, m, n, alpha, A, lda, x, incx, beta, z, incz);
}

#undef num_threads
#undef dgemv_bs
#undef threadSize
