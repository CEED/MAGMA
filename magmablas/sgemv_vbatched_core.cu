#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @author Tingxing Dong
       @author Azzam Haidar
       @author Ahmad Abdelfattah

*/
#include "magma_internal.h"
#include "magma_templates.h"

#define PRECISION_s

#include "gemv_template_kernel_vbatched.cuh"
#include "gemv_config/gemvn_param.h"
#include "gemv_config/gemvt_param.h"

#define version(s,v) s ## _V_ ## v

/******************************************************************************/
extern "C" void
magmablas_sgemv_vbatched_max_nocheck(
    magma_trans_t trans, magma_int_t* m, magma_int_t* n, 
    float alpha,
    magmaFloat_ptr dA_array[], magma_int_t* ldda, 
    magmaFloat_ptr dx_array[], magma_int_t* incx,
    float beta,
    magmaFloat_ptr dy_array[], magma_int_t* incy, 
    magma_int_t batchCount, 
    magma_int_t max_m, magma_int_t max_n, magma_queue_t queue)
{
    if ( trans == MagmaNoTrans ) {                                                   
        if (max(max_m, max_n) <= 96) { // small size                         
            if (max_m < max_n) { // Fat matrix
                if ( max_m <= 8) 
                {
                    gemvn_template_vbatched<float, version(N, 32)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else if ( max_m <= 16)            
                {
                    gemvn_template_vbatched<float, version(N, 72)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else if ( max_m <= 32)            
                {
                    gemvn_template_vbatched<float, version(N, 97)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else if ( max_m <= 64)            
                {
                    gemvn_template_vbatched<float, version(N, 120)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else
                {
                    gemvn_template_vbatched<float, version(N, 130)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }  
            }
            else {   // Tall matrix
                if ( max_n <= 16) 
                {    
                    gemvn_template_vbatched<float, version(N, 118)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else 
                {
                    gemvn_template_vbatched<float, version(N, 120)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }  
            }
        }
        else { // big size
            if (max_m < max_n) { // Fat matrix
                if (max_m <= 16)
                {
                    gemvn_template_vbatched<float, version(N, 79)>              
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else if (max_m <= 32)
                {
                    gemvn_template_vbatched<float, version(N, 103)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else if (max_m <= 64)
                {
                    gemvn_template_vbatched<float, version(N, 126)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else
                {
                    gemvn_template_vbatched<float, version(N, 135)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
            }
            else { // Tall or square matrix
                if (max_m <= 256)
                {
                    gemvn_template_vbatched<float, version(N, 137)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else
                {
                    gemvn_template_vbatched<float, version(N, 140)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
            }
        }// big size        
    }
    else {
        if (max(max_m, max_n) <= 96) // small size
        {                              
            gemvc_template_vbatched<float, version(T, 46)>             
                ( trans, m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );    
        }
        else // big size
        {
            if (max_m <= max_n) //  Fat or square matrix
            {    
                if (max_m <= 64)
                {
                    gemvc_template_vbatched<float, version(T, 47)>             
                        ( trans, m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else
                {
                    gemvc_template_vbatched<float, version(T, 133)>             
                        ( trans, m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
            }                           
            else// (m > n) Tall matrix
            {
                if (max_n <= 8)
                {
                    gemvc_template_vbatched<float, version(T, 130)>             
                        ( trans, m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else
                {
                    gemvc_template_vbatched<float, version(T, 131)>             
                        ( trans, m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
            }
        }        
    }                   
}
