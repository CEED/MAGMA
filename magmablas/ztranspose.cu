#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

       @precisions normal z -> s d c

*/
#include "common_magma.h"
#define PRECISION_z
#include "commonblas.h"

__global__ void ztranspose_32( hipDoubleComplex *B, int ldb, hipDoubleComplex *A, int lda )
{        
        __shared__ hipDoubleComplex a[32][ZSIZE_1SHARED+1];
        
        int inx = threadIdx.x;
        int iny = threadIdx.y;
        int ibx = blockIdx.x*32;
        int iby = blockIdx.y*32;
        
        A += ibx + inx + __mul24( iby + iny, lda );
        B += iby + inx + __mul24( ibx + iny, ldb );
        
        a[iny+0][inx] = A[0*lda];
        a[iny+8][inx] = A[8*lda];
        a[iny+16][inx] = A[16*lda];
        a[iny+24][inx] = A[24*lda];
        
        __syncthreads();
        
#if defined(PRECISION_s) || defined(PRECISION_d) || defined(PRECISION_c)
        B[0*ldb] = a[inx][iny+0];
        B[8*ldb] = a[inx][iny+8];
        B[16*ldb] = a[inx][iny+16];
        B[24*ldb] = a[inx][iny+24];
#else /* defined(PRECISION_z) */
        B[0*ldb]    = a[inx][iny+0];
        B[8*ldb]    = a[inx][iny+8];
        B[0*ldb+16] = a[inx+16][iny+0];
        B[8*ldb+16] = a[inx+16][iny+8];

        __syncthreads();
        A += ZSIZE_1SHARED;
        B += __mul24( 16, ldb);

        a[iny+0][inx] = A[0*lda];
        a[iny+8][inx] = A[8*lda];
        a[iny+16][inx] = A[16*lda];
        a[iny+24][inx] = A[24*lda];

        __syncthreads();

        B[0*ldb] = a[inx][iny+0];
        B[8*ldb] = a[inx][iny+8];
        B[0*ldb+16] = a[inx+16][iny+0];
        B[8*ldb+16] = a[inx+16][iny+8];
#endif
} 

//
//        m, n - dimensions in the source matrix
//             This version works when m and n are divisible by 32.
//
extern "C" void 
magmablas_ztranspose(hipDoubleComplex *odata, int ldo, 
                     hipDoubleComplex *idata, int ldi, 
                     int m, int n )
{
        //assert( (m%32) == 0 && (n%32) == 0, "misaligned transpose" );
        dim3 threads( ZSIZE_1SHARED, 8, 1 );
        dim3 grid( m/32, n/32, 1 );
        ztranspose_32<<< grid, threads, 0, magma_stream >>>( odata, ldo, idata, ldi );
}
