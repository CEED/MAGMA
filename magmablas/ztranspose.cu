#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

       @precisions normal z -> s d c

*/
#define PRECISION_z
#include "commonblas.h"

__global__ void ztranspose_32( double2 *B, int ldb, double2 *A, int lda )
{	
	__shared__ double2 a[32][ZTRANSPOSE_SIZE];
	
	int inx = threadIdx.x;
	int iny = threadIdx.y;
	int ibx = blockIdx.x*32;
	int iby = blockIdx.y*32;
	
	A += ibx + inx + __mul24( iby + iny, lda );
	B += iby + inx + __mul24( ibx + iny, ldb );
	
	a[iny+0][inx] = A[0*lda];
	a[iny+8][inx] = A[8*lda];
	a[iny+16][inx] = A[16*lda];
	a[iny+24][inx] = A[24*lda];
	
	__syncthreads();
	
#if !defined(PRECISION_z)
	B[0*ldb] = a[inx][iny+0];
	B[8*ldb] = a[inx][iny+8];
	B[16*ldb] = a[inx][iny+16];
	B[24*ldb] = a[inx][iny+24];
#else
	B[0*ldb]    = a[inx][iny+0];
	B[8*ldb]    = a[inx][iny+8];
	B[0*ldb+16] = a[inx+16][iny+0];
	B[8*ldb+16] = a[inx+16][iny+8];

	__syncthreads();
	A += ZTRANSPOSE_SIZE;
	B += __mul24( 16, ldb);

        a[iny+0][inx] = A[0*lda];
        a[iny+8][inx] = A[8*lda];
        a[iny+16][inx] = A[16*lda];
        a[iny+24][inx] = A[24*lda];

        __syncthreads();

	B[0*ldb] = a[inx][iny+0];
	B[8*ldb] = a[inx][iny+8];
	B[0*ldb+16] = a[inx+16][iny+0];
	B[8*ldb+16] = a[inx+16][iny+8];
#endif
} 

//
//	m, n - dimensions in the source matrix
//             This version works when m and n are divisible by 32.
//
extern "C" void 
magmablas_ztranspose(double2 *odata, int ldo, 
                     double2 *idata, int ldi, 
                     int m, int n )
{
	//assert( (m%32) == 0 && (n%32) == 0, "misaligned transpose" );
	dim3 threads( ZTRANSPOSE_SIZE, 8, 1 );
	dim3 grid( m/32, n/32, 1 );
	ztranspose_32<<< grid, threads >>>( odata, ldo, idata, ldi );
}
