#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010
*/

#include <hipblas.h>
#include "magmablas.h"
#include "commonblas_d.h"

static __device__ void daxpy(double a,double *b, double *c) {
	c[0] += a * b[0];
	c[1] += a * b[1];
	c[2] += a * b[2];
	c[3] += a * b[3];
	c[4] += a * b[4];
	c[5] += a * b[5];
	c[6] += a * b[6];
	c[7] += a * b[7];
	c[8] += a * b[8];
	c[9] += a * b[9];
	c[10] += a * b[10];
	c[11] += a * b[11];
	c[12] += a * b[12];
	c[13] += a * b[13];
	c[14] += a * b[14];
	c[15] += a * b[15];
}

extern "C" __global__ void 
dgemm_kernel_T_T_64_16_16_16_4_v2(double *C, const double *A, const double *B, 
                                  int m, int n, int k,
                                  int lda, int ldb, int ldc, 
                                  double alpha, double beta)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose:
    ========
    This routine computes
       C = alpha* A^T*B^T  + beta * C

    B is put into shared memory
    Parameters Used:
        blk_M=64 blk_N=16 blk_K=16 nthd_x=16 nthd_y=4

    This code should run for any matrix size.
    This kernel outperforms cuda-2.2 when m,n,k >=512
    ===============================================================  */

	__shared__ double Bb[16][17];
	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	int iby = ((blockIdx.y + blockIdx.x ) % (n/16))*16;
	const int idt = ty * 16 + tx;
	int ibx = blockIdx.x *64+idt;
	//int iby = blockIdx.y *16;

	A += ibx ;
	B+=tx+__mul24(iby+ty,ldb);
	C += __mul24(ibx ,ldc) + iby;

	const double *Bend = B + k;


	double Cb[16] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

	do {
		double Ab[4] = {A[0], A[lda], A[2*lda], A[3*lda]};
		Bb[tx][ty+0] = B[0*ldb];
		Bb[tx][ty+4] = B[4*ldb];
		Bb[tx][ty+8] = B[8*ldb];
		Bb[tx][ty+12] = B[12*ldb];

		__syncthreads();

		A += 4 * lda;
		daxpy(Ab[0], &Bb[0][0], Cb); Ab[0] = A[0*lda];
		daxpy(Ab[1], &Bb[1][0], Cb); Ab[1] = A[1*lda];
		daxpy(Ab[2], &Bb[2][0], Cb); Ab[2] = A[2*lda];
		daxpy(Ab[3], &Bb[3][0], Cb); Ab[3] = A[3*lda];

		A += 4 * lda;
		daxpy(Ab[0], &Bb[4][0], Cb); Ab[0] = A[0*lda];
		daxpy(Ab[1], &Bb[5][0], Cb); Ab[1] = A[1*lda];
		daxpy(Ab[2], &Bb[6][0], Cb); Ab[2] = A[2*lda];
		daxpy(Ab[3], &Bb[7][0], Cb); Ab[3] = A[3*lda];

		A += 4 * lda;
		daxpy(Ab[0], &Bb[8][0], Cb); Ab[0] = A[0*lda];
		daxpy(Ab[1], &Bb[9][0], Cb); Ab[1] = A[1*lda];
		daxpy(Ab[2], &Bb[10][0], Cb); Ab[2] = A[2*lda];
		daxpy(Ab[3], &Bb[11][0], Cb); Ab[3] = A[3*lda];

		A += 4 * lda;
		daxpy(Ab[0], &Bb[12][0], Cb);
		daxpy(Ab[1], &Bb[13][0], Cb);
		daxpy(Ab[2], &Bb[14][0], Cb);
		daxpy(Ab[3], &Bb[15][0], Cb);

		B += 16;

		__syncthreads();
	} while (B < Bend);

	#pragma unroll 16
	for (int i = 0; i < 16; i++) {
		C[i] =alpha*Cb[i] + beta * C[i];
	}
}

extern "C" void
magmablas_dgemm_kernel_T_T_64_16_16_16_4_v2(double *C,
                                            const double *A,
                                            const double *B, 
                                            int m, int n, int k, 
                                            int lda, int ldb, int ldc,
                                            double alpha, double beta)
{
        dim3 threads( 16, 4 );
        dim3 grid(m/64,n/16);
        dgemm_kernel_T_T_64_16_16_16_4_v2<<< grid, threads >>>(C, A, B, 
                                                               m, n, k,
                                                               lda, ldb, ldc, 
                                                               alpha, beta);
}

