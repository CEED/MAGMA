#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

       @author Peng Du
       @author Tingxing Dong
       @author Mark Gates
       @author Azzam Haidar
       
       File named ztrtri_diag.cu to avoid name conflict with src/ztrtri.o
       in the library. The actual kernels are in ztrtri_lower.cu and ztrtri_upper.cu
*/

#include "magma_internal.h"

#define    TRTRI_NONBATCHED
#include "ztrtri.cuh"


/**
    Inverts the NB x NB diagonal blocks of a triangular matrix.
    This routine is used in ztrsm.
    
    Same as ztrtri_diag, but adds queue argument.
    
    @ingroup magma_zblas3
    ********************************************************************/
/**
    Purpose
    -------
    ztrtri_diag inverts the NB x NB diagonal blocks of A.

    Arguments
    ----------
    @param[in]
    uplo    magma_uplo_t.
            On entry, uplo specifies whether the matrix A is an upper or
            lower triangular matrix as follows:
      -     = MagmaUpper:  A is an upper triangular matrix.
      -     = MagmaLower:  A is a  lower triangular matrix.

    @param[in]
    diag    magma_diag_t.
            On entry, diag specifies whether or not A is unit triangular
            as follows:
      -     = MagmaUnit:     A is assumed to be unit triangular.
      -     = MagmaNonUnit:  A is not assumed to be unit triangular.

    @param[in]
    n       INTEGER.
            On entry, n specifies the order of the matrix A. N >= 0.

    @param[in]
    dA      COMPLEX_16 array of dimension ( ldda, n )
            The triangular matrix A.
    \n
            If UPLO = MagmaUpper, the leading N-by-N upper triangular part of A
            contains the upper triangular matrix, and the strictly lower
            triangular part of A is not referenced.
    \n
            If UPLO = MagmaLower, the leading N-by-N lower triangular part of A
            contains the lower triangular matrix, and the strictly upper
            triangular part of A is not referenced.
    \n
            If DIAG = MagmaUnit, the diagonal elements of A are also not referenced
            and are assumed to be 1.

    @param[in]
    ldda    INTEGER.
            The leading dimension of the array A.  LDDA >= max(1,N).

    @param[out]
    d_dinvA COMPLEX_16 array of dimension (NB, ceil(n/NB)*NB),
            where NB = 128.
            On exit, contains inverses of the NB-by-NB diagonal blocks of A.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_zblas3
    ********************************************************************/
extern "C" void
magmablas_ztrtri_diag_q(
    magma_uplo_t uplo, magma_diag_t diag, magma_int_t n,
    magmaDoubleComplex_const_ptr dA, magma_int_t ldda,
    magmaDoubleComplex_ptr d_dinvA,
    magma_queue_t queue)
{
    magma_int_t info = 0;
    if (uplo != MagmaLower && uplo != MagmaUpper)
        info = -1;
    else if (diag != MagmaNonUnit && diag != MagmaUnit)
        info = -2;
    else if (n < 0)
        info = -3;
    else if (ldda < n)
        info = -5;

    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info
    }
    
    int nblocks = magma_ceildiv( n, IB );

    hipMemset( d_dinvA, 0, magma_roundup( n, NB )*NB * sizeof(magmaDoubleComplex) );
    
    if ( uplo == MagmaLower ) {
        // invert diagonal IB x IB inner blocks
        ztrtri_diag_lower_kernel
            <<< nblocks, IB, 0, queue->hip_stream() >>>
            ( diag, n, dA, ldda, d_dinvA );

        // build up NB x NB blocks (assuming IB=16 here):
        // use   16 x 16  blocks to build  32 x 32  blocks,  1 x (1 x npages) grid,  4 x 4 threads;
        // then  32 x 32  blocks to build  64 x 64  blocks,  1 x (2 x npages) grid,  8 x 4 threads;
        // then  64 x 64  blocks to build 128 x 128 blocks,  1 x (4 x npages) grid, 16 x 4 threads;
        // then 128 x 128 blocks to build 256 x 256 blocks,  2 x (8 x npages) grid, 16 x 4 threads.
        for( int jb=IB; jb < NB; jb *= 2 ) {
            int kb = jb*2;
            int npages = magma_ceildiv( n, kb );
            dim3 threads( (jb <= 32 ? jb/4 : 16), 4 );
            dim3 grid( jb/(threads.x*threads.y), npages*(jb/16) );  // emulate 3D grid: NX * (NY*npages), for CUDA ARCH 1.x
            
            //printf( "n %d, jb %d, grid %d x %d (%d x %d)\n", n, jb, grid.x, grid.y, grid.y / npages, npages );
            switch (jb) {
                case 16:
                    triple_zgemm16_part1_lower_kernel<<< grid, threads, 0, queue->hip_stream() >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_zgemm16_part2_lower_kernel<<< grid, threads, 0, queue->hip_stream() >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
                case 32:
                    triple_zgemm32_part1_lower_kernel<<< grid, threads, 0, queue->hip_stream() >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_zgemm32_part2_lower_kernel<<< grid, threads, 0, queue->hip_stream() >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
                case 64:
                    triple_zgemm64_part1_lower_kernel<<< grid, threads, 0, queue->hip_stream() >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_zgemm64_part2_lower_kernel<<< grid, threads, 0, queue->hip_stream() >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
                default:
                    triple_zgemm_above64_part1_lower_kernel<<< grid, threads, 0, queue->hip_stream() >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_zgemm_above64_part2_lower_kernel<<< grid, threads, 0, queue->hip_stream() >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_zgemm_above64_part3_lower_kernel<<< grid, threads, 0, queue->hip_stream() >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
            }
            if ( kb >= n ) break;
        }
    }
    else {
        ztrtri_diag_upper_kernel
            <<< nblocks, IB, 0, queue->hip_stream() >>>
            ( diag, n, dA, ldda, d_dinvA );

        // update the inverse up to the size of IB
        for( int jb=IB; jb < NB; jb *= 2 ) {
            int kb = jb*2;
            int npages = magma_ceildiv( n, kb );
            dim3 threads( (jb <= 32 ? jb/4 : 16), 4 );
            dim3 grid( jb/(threads.x*threads.y), npages*(jb/16) );  // emulate 3D grid: NX * (NY*npages), for CUDA ARCH 1.x
            
            switch (jb) {
                case 16:
                    triple_zgemm16_part1_upper_kernel<<< grid, threads, 0, queue->hip_stream() >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_zgemm16_part2_upper_kernel<<< grid, threads, 0, queue->hip_stream() >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
                case 32:
                    triple_zgemm32_part1_upper_kernel<<< grid, threads, 0, queue->hip_stream() >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_zgemm32_part2_upper_kernel<<< grid, threads, 0, queue->hip_stream() >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
                case 64:
                    triple_zgemm64_part1_upper_kernel<<< grid, threads, 0, queue->hip_stream() >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_zgemm64_part2_upper_kernel<<< grid, threads, 0, queue->hip_stream() >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
                default:
                    triple_zgemm_above64_part1_upper_kernel<<< grid, threads, 0, queue->hip_stream() >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_zgemm_above64_part2_upper_kernel<<< grid, threads, 0, queue->hip_stream() >>>( n, dA, ldda, d_dinvA, jb, npages );
                    triple_zgemm_above64_part3_upper_kernel<<< grid, threads, 0, queue->hip_stream() >>>( n, dA, ldda, d_dinvA, jb, npages );
                    break;
            }
            if ( kb >= n ) break;
        }
    }
}
