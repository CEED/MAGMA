#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal z -> s d c
       @author Mark Gates
*/
#include "common_magma.h"
#include <assert.h>

#define NB 64

/*
    Matrix is m x m, and is divided into block rows, each NB x m.
    Each block has NB threads.
    Each thread copies one row, iterating across all columns below diagonal.
    The bottom block of rows may be partially outside the matrix;
    if so, rows outside the matrix (i >= m) are disabled.
*/
__global__ void
zsymmetrize_lower( int m, hipDoubleComplex *dA, int ldda )
{
    // dA iterates across row i and dAT iterates down column i.
    int i = blockIdx.x*NB + threadIdx.x;
    hipDoubleComplex *dAT = dA;
    if ( i < m ) {
        dA  += i;
        dAT += i*ldda;
        hipDoubleComplex *dAend = dA + i*ldda;
        while( dA < dAend ) {
            *dAT = hipConj(*dA);  // upper := lower
            dA  += ldda;
            dAT += 1;
        }
    }
}


// only difference with _lower version is direction dA=dAT instead of dAT=dA.
__global__ void
zsymmetrize_upper( int m, hipDoubleComplex *dA, int ldda )
{
    // dA iterates across row i and dAT iterates down column i.
    int i = blockIdx.x*NB + threadIdx.x;
    hipDoubleComplex *dAT = dA;
    if ( i < m ) {
        dA  += i;
        dAT += i*ldda;
        hipDoubleComplex *dAend = dA + i*ldda;
        while( dA < dAend ) {
            *dA = hipConj(*dAT);  // lower := upper
            dA  += ldda;
            dAT += 1;
        }
    }
}


extern "C" void
magmablas_zsymmetrize( char uplo, magma_int_t m, hipDoubleComplex *dA, magma_int_t ldda )
{
/*
    Purpose
    =======
    
    ZSYMMETRIZE copies lower triangle to upper triangle, or vice-versa,
    to make dA a general representation of a symmetric matrix.
    
    Arguments
    =========
    
    UPLO    (input) CHARACTER*1
            Specifies the part of the matrix dA that is valid on input.
            = 'U':      Upper triangular part
            = 'L':      Lower triangular part
    
    M       (input) INTEGER
            The number of rows of the matrix dA.  M >= 0.
    
    dA      (input/output) COMPLEX DOUBLE PRECISION array, dimension (LDDA,N)
            The m by m matrix dA.
    
    LDDA    (input) INTEGER
            The leading dimension of the array dA.  LDDA >= max(1,M).
    
    =====================================================================   */

    //printf( "m %d, grid %d, threads %d\n", m, grid.x, threads.x );
    if ( m == 0 )
        return;
    
    assert( m >= 0 );
    assert( ldda >= m );
    
    dim3 threads( NB );
    dim3 grid( (m + NB - 1)/NB );
    
    if ( (uplo == 'U') || (uplo == 'u') ) {
        zsymmetrize_upper<<< grid, threads, 0, magma_stream >>>( m, dA, ldda );
    }
    else if ( (uplo == 'L') || (uplo == 'l') ) {
        zsymmetrize_lower<<< grid, threads, 0, magma_stream >>>( m, dA, ldda );
    }
    else {
        printf( "uplo has illegal value\n" );
        exit(1);
    }
}
