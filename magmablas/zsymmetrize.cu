#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal z -> s d c
       @author Mark Gates
*/
#include "common_magma.h"

/*
    Matrix is divided into 64 x m block rows.
    Each block has 64 threads.
    Each thread copies one row, iterating across all columns below diagonal.
    The bottom block of rows may be partially outside the matrix;
    if so, rows outside the matrix (i >= m) are disabled.
*/
__global__ void
zsymmetrize_kernel_lower( int m, hipDoubleComplex *A, int lda )
{
    // A iterates across row i and AT iterates down column i.
    int i = blockIdx.x*64 + threadIdx.x;
    hipDoubleComplex *AT = A;
    if ( i < m ) {
        A  += i;
        AT += i*lda;
        hipDoubleComplex *Aend = A + i*lda;
        while( A < Aend ) {
            *AT = hipConj(*A);
            A  += lda;
            AT += 1;
        }
    }
}


// only difference with _lower version is direction A=AT instead of AT=A.
__global__ void
zsymmetrize_kernel_upper( int m, hipDoubleComplex *A, int lda )
{
    // A iterates across row i and AT iterates down column i.
    int i = blockIdx.x*64 + threadIdx.x;
    hipDoubleComplex *AT = A;
    if ( i < m ) {
        A  += i;
        AT += i*lda;
        hipDoubleComplex *Aend = A + i*lda;
        while( A < Aend ) {
            *A = hipConj(*AT);
            A  += lda;
            AT += 1;
        }
    }
}


extern "C" void
magmablas_zsymmetrize( char uplo, int m, hipDoubleComplex *A, int lda )
{
/*
  Purpose
  =======

  ZSYMMETRIZE copies lower triangle to upper triangle, or vice-versa,
  to make A a general representation of a symmetric matrix.

  Arguments
  =========

  UPLO    (input) CHARACTER*1
          Specifies the part of the matrix A that is valid on input.
          = 'U':      Upper triangular part
          = 'L':      Lower triangular part

  M       (input) INTEGER
          The number of rows of the matrix A.  M >= 0.

  A       (input/output) COMPLEX DOUBLE PRECISION array, dimension (LDA,N)
          The m by m matrix A.

  LDA     (input) INTEGER
          The leading dimension of the array A.  LDA >= max(1,M).

  =====================================================================   */

    dim3 threads( 64 );
    dim3 grid( m/64 + (m%64 != 0) );
    
    //printf( "m %d, grid %d, threads %d\n", m, grid.x, threads.x );
    if ( m == 0 )
        return;
    
    if ( (uplo == 'U') || (uplo == 'u') ) {
        zsymmetrize_kernel_upper<<< grid, threads, 0, magma_stream >>>( m, A, lda );
    }
    else if ( (uplo == 'L') || (uplo == 'l') ) {
        zsymmetrize_kernel_lower<<< grid, threads, 0, magma_stream >>>( m, A, lda );
    }
    else {
        printf( "uplo has illegal value\n" );
        exit(1);
    }
}
