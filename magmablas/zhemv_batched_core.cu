#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c

       @author Ahmad Abdelfattah
       
*/
#include "magma_internal.h"
#include "batched_kernel_param.h"

#define PRECISION_z
#include "hemv_template_kernel_batched.cuh"

///////////////////////////////////////////////////////////////////////////////////////////////////
extern "C" void 
magmablas_zhemv_batched_core(
        magma_uplo_t uplo, magma_int_t n, 
        magmaDoubleComplex alpha, magmaDoubleComplex **dA_array, magma_int_t ldda,
                                  magmaDoubleComplex **dX_array, magma_int_t incx,
        magmaDoubleComplex beta,  magmaDoubleComplex **dY_array, magma_int_t incy,
        magma_int_t offA, magma_int_t offX, magma_int_t offY, 
        magma_int_t batchCount, magma_queue_t queue )
{
    if(uplo == MagmaLower){
        const int param[] = {ZHEMV_BATCHED_LOWER};
        const int nb = param[0];
        hemv_diag_template_batched<magmaDoubleComplex, ZHEMV_BATCHED_LOWER>
                ( uplo, n, 
                  alpha, dA_array, ldda, 
                         dX_array, incx, 
                  beta,  dY_array, incy, 
                  offA, offX, offY, batchCount, queue);
        if(n > nb){
            hemv_lower_template_batched<magmaDoubleComplex, ZHEMV_BATCHED_LOWER>
                ( n, alpha, 
                  dA_array, ldda, 
                  dX_array, incx, 
                  dY_array, incy, 
                  offA, offX, offY, batchCount, queue);
        }
    }
    else{    // upper
        const int param[] = {ZHEMV_BATCHED_UPPER};
        const int nb = param[0];
        hemv_diag_template_batched<magmaDoubleComplex, ZHEMV_BATCHED_UPPER>
                ( uplo, n, 
                  alpha, dA_array, ldda, 
                         dX_array, incx, 
                  beta,  dY_array, incy, 
                  offA, offX, offY, batchCount, queue);
        if(n > nb){
            hemv_upper_template_batched<magmaDoubleComplex, ZHEMV_BATCHED_UPPER>
                ( n, alpha, 
                  dA_array, ldda, 
                  dX_array, incx, 
                  dY_array, incy, 
                  offA, offX, offY, batchCount, queue);
        }
    }
}
/***************************************************************************//**
    Purpose
    -------
    ZHEMV performs the matrix-vector operation:

        y := alpha*A*x + beta*y,

    where alpha and beta are scalars, x and y are n element vectors and
    A is an n by n Hermitian matrix.
    This is the fixed size batched version of the operation. 

    Arguments
    ----------
    @param[in]
    uplo    magma_uplo_t.
            On entry, UPLO specifies whether the upper or lower
            triangular part of the array A is to be referenced as
            follows:
      -     = MagmaUpper:  Only the upper triangular part of A is to be referenced.
      -     = MagmaLower:  Only the lower triangular part of A is to be referenced.

    @param[in]
    n       INTEGER.
            On entry, N specifies the order of each matrix A.
            N must be at least zero.

    @param[in]
    alpha   COMPLEX_16.
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    dA_array    Array of pointers, dimension(batchCount). 
            Each is a COMPLEX_16 array A of DIMENSION ( LDDA, n ).
            Before entry with UPLO = MagmaUpper, the leading n by n
            upper triangular part of the array A must contain the upper
            triangular part of the Hermitian matrix and the strictly
            lower triangular part of A is not referenced.
            Before entry with UPLO = MagmaLower, the leading n by n
            lower triangular part of the array A must contain the lower
            triangular part of the Hermitian matrix and the strictly
            upper triangular part of A is not referenced.
            Note that the imaginary parts of the diagonal elements need
            not be set and are assumed to be zero.

    @param[in]
    ldda    INTEGER.
            On entry, LDDA specifies the first dimension of each A as declared
            in the calling (sub) program. LDDA must be at least
            max( 1, n ).
            It is recommended that ldda is multiple of 16. Otherwise
            performance would be deteriorated as the memory accesses
            would not be fully coalescent.

    @param[in]
    dX_array    Array of pointers, dimension(batchCount). 
            Each is a COMPLEX_16 array X of dimension at least
            ( 1 + ( n - 1 )*abs( INCX ) ).
            Before entry, the incremented array X must contain the n
            element vector X.

    @param[in]
    incx    INTEGER.
            On entry, INCX specifies the increment for the elements of
            X. INCX must not be zero.

    @param[in]
    beta    COMPLEX_16.
            On entry, BETA specifies the scalar beta. When BETA is
            supplied as zero then Y need not be set on input.

    @param[in,out]
    dY_array    Array of pointers, dimension(batchCount). 
            Each is a COMPLEX_16 array Y of dimension at least
            ( 1 + ( n - 1 )*abs( INCY ) ).
            Before entry, the incremented array Y must contain the n
            element vector Y. On exit, Y is overwritten by the updated
            vector Y.

    @param[in]
    incy    INTEGER.
            On entry, INCY specifies the increment for the elements of
            Y. INCY must not be zero.

    @param[in]
    batchCount    INTEGER.
            The number of problems to operate on. 
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_hemv_batched
*******************************************************************************/
extern "C" void 
magmablas_zhemv_batched(
        magma_uplo_t uplo, magma_int_t n, 
        magmaDoubleComplex alpha, magmaDoubleComplex **dA_array, magma_int_t ldda,
                                  magmaDoubleComplex **dX_array, magma_int_t incx,
        magmaDoubleComplex beta,  magmaDoubleComplex **dY_array, magma_int_t incy,
        magma_int_t batchCount, magma_queue_t queue )
{
    magma_int_t info = 0;
    if ( uplo != MagmaLower && uplo != MagmaUpper ) {
        info = -1;
    } else if ( n < 0 ) {
        info = -2;
    } else if ( ldda < max(1, n) ) {
        info = -5;
    } else if ( incx == 0 ) {
        info = -7;
    } else if ( incy == 0 ) {
        info = -10;
    } else if ( batchCount < 0 )
        info = -11;
    
    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;
    }

    if ( (n == 0) || ( MAGMA_Z_EQUAL(alpha, MAGMA_Z_ZERO) && MAGMA_Z_EQUAL(beta, MAGMA_Z_ONE) ) )
        return;    
    
    magmablas_zhemv_batched_core( 
            uplo, n, 
            alpha, dA_array, ldda, 
                   dX_array, incx,
            beta,  dY_array, incy,  
            0, 0, 0, 
            batchCount, queue );
}
///////////////////////////////////////////////////////////////////////////////////////////////////
