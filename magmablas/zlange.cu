#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c
       @author Mark Gates
*/
#include "common_magma.h"

/* Computes row sums dwork[i] = sum( abs( A(i,:) )), i=0:m-1, for || A ||_inf,
 * where m and n are any size.
 * Has ceil( m/64 ) blocks of 64 threads. Each thread does one row. */
extern "C" __global__ void
zlange_inf_kernel(
    int m, int n, const magmaDoubleComplex *A, int lda, double *dwork )
{
    int i = blockIdx.x*64 + threadIdx.x;
    double Cb[4] = {0, 0, 0, 0};
    int n_mod_4 = n % 4;
    n -= n_mod_4;
    
    // if beyond last row, skip row
    if ( i < m ) {
        A += i;
        
        if ( n >= 4 ) {
            const magmaDoubleComplex *Aend = A + lda*n;
            magmaDoubleComplex rA[4] = { A[0], A[lda], A[2*lda], A[3*lda] };
            A += 4*lda;
            
            while( A < Aend ) {
                Cb[0] += hipCabs( rA[0] );  rA[0] = A[0];
                Cb[1] += hipCabs( rA[1] );  rA[1] = A[lda];
                Cb[2] += hipCabs( rA[2] );  rA[2] = A[2*lda];
                Cb[3] += hipCabs( rA[3] );  rA[3] = A[3*lda];
                A += 4*lda;
            }
            
            Cb[0] += hipCabs( rA[0] );
            Cb[1] += hipCabs( rA[1] );
            Cb[2] += hipCabs( rA[2] );
            Cb[3] += hipCabs( rA[3] );
        }
    
        /* clean up code */
        switch( n_mod_4 ) {
            case 0:
                break;
    
            case 1:
                Cb[0] += hipCabs( A[0] );
                break;
    
            case 2:
                Cb[0] += hipCabs( A[0]   );
                Cb[1] += hipCabs( A[lda] );
                break;
    
            case 3:
                Cb[0] += hipCabs( A[0]     );
                Cb[1] += hipCabs( A[lda]   );
                Cb[2] += hipCabs( A[2*lda] );
                break;
        }
    
        /* compute final result */
        dwork[i] = Cb[0] + Cb[1] + Cb[2] + Cb[3];
    }
}

/**
    Purpose
    -------
    ZLANGE  returns the value of the one norm, or the Frobenius norm, or
    the  infinity norm, or the  element of  largest absolute value  of a
    real matrix A.
    
    Description
    -----------
    ZLANGE returns the value
    
       ZLANGE = ( max(abs(A(i,j))), NORM = 'M' or 'm'            ** not yet supported
                (
                ( norm1(A),         NORM = '1', 'O' or 'o'       ** not yet supported
                (
                ( normI(A),         NORM = 'I' or 'i'
                (
                ( normF(A),         NORM = 'F', 'f', 'E' or 'e'  ** not yet supported
    
    where norm1 denotes the one norm of a matrix (maximum column sum),
    normI denotes the infinity norm of a matrix (maximum row sum) and
    normF denotes the Frobenius norm of a matrix (square root of sum of
    squares). Note that max(abs(A(i,j))) is not a consistent matrix norm.
    
    Arguments
    ---------
    @param[in]
    norm    CHARACTER*1
            Specifies the value to be returned in ZLANGE as described
            above.
    
    @param[in]
    m       INTEGER
            The number of rows of the matrix A.  M >= 0.  When M = 0,
            ZLANGE is set to zero.
    
    @param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.  When N = 0,
            ZLANGE is set to zero.
    
    @param[in]
    A       DOUBLE PRECISION array on the GPU, dimension (LDA,N)
            The m by n matrix A.
    
    @param[in]
    lda     INTEGER
            The leading dimension of the array A.  LDA >= max(M,1).
    
    @param
    dwork   (workspace) DOUBLE PRECISION array on the GPU, dimension (MAX(1,LWORK)),
            where LWORK >= M when NORM = 'I'; otherwise, WORK is not
            referenced.

    @ingroup magma_zaux2
    ********************************************************************/
extern "C" double
magmablas_zlange(
    magma_norm_t norm, magma_int_t m, magma_int_t n,
    const magmaDoubleComplex *A, magma_int_t lda, double *dwork )
{
    magma_int_t info = 0;
    if ( norm != MagmaInfNorm )
        info = -1;
    else if ( m < 0 )
        info = -2;
    else if ( n < 0 )
        info = -3;
    else if ( lda < m )
        info = -5;
    
    if ( info != 0 ) {
        magma_xerbla( __func__, -(info) );
        return info;
    }
    
    /* Quick return */
    if ( m == 0 || n == 0 )
        return 0;
    
    dim3 threads( 64 );
    dim3 grid( (m-1)/64 + 1 );
    zlange_inf_kernel<<< grid, threads, 0, magma_stream >>>( m, n, A, lda, dwork );
    int i = hipblasIdamax( m, dwork, 1 ) - 1;
    double res;
    hipMemcpy( &res, &dwork[i], sizeof(double), hipMemcpyDeviceToHost );
    return res;
}
