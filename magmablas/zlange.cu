#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c
       @author Mark Gates
*/
#include "common_magma.h"
#include "magma_templates.h"


/* Computes row sums dwork[i] = sum( abs( A(i,:) )), i=0:m-1, for || A ||_inf,
 * where m and n are any size.
 * Has ceil( m/64 ) blocks of 64 threads. Each thread does one row.
 * See also zlange_max_kernel code, below. */
extern "C" __global__ void
zlange_inf_kernel(
    int m, int n, const magmaDoubleComplex *A, int lda, double *dwork )
{
    int i = blockIdx.x*64 + threadIdx.x;
    double rsum[4] = {0, 0, 0, 0};
    int n_mod_4 = n % 4;
    n -= n_mod_4;
    
    // if beyond last row, skip row
    if ( i < m ) {
        A += i;
        
        if ( n >= 4 ) {
            const magmaDoubleComplex *Aend = A + lda*n;
            magmaDoubleComplex rA[4] = { A[0], A[lda], A[2*lda], A[3*lda] };
            A += 4*lda;
            
            while( A < Aend ) {
                rsum[0] += MAGMA_Z_ABS( rA[0] );  rA[0] = A[0];
                rsum[1] += MAGMA_Z_ABS( rA[1] );  rA[1] = A[lda];
                rsum[2] += MAGMA_Z_ABS( rA[2] );  rA[2] = A[2*lda];
                rsum[3] += MAGMA_Z_ABS( rA[3] );  rA[3] = A[3*lda];
                A += 4*lda;
            }
            
            rsum[0] += MAGMA_Z_ABS( rA[0] );
            rsum[1] += MAGMA_Z_ABS( rA[1] );
            rsum[2] += MAGMA_Z_ABS( rA[2] );
            rsum[3] += MAGMA_Z_ABS( rA[3] );
        }
    
        /* clean up code */
        switch( n_mod_4 ) {
            case 0:
                break;
    
            case 1:
                rsum[0] += MAGMA_Z_ABS( A[0] );
                break;
    
            case 2:
                rsum[0] += MAGMA_Z_ABS( A[0]   );
                rsum[1] += MAGMA_Z_ABS( A[lda] );
                break;
    
            case 3:
                rsum[0] += MAGMA_Z_ABS( A[0]     );
                rsum[1] += MAGMA_Z_ABS( A[lda]   );
                rsum[2] += MAGMA_Z_ABS( A[2*lda] );
                break;
        }
    
        /* compute final result */
        dwork[i] = rsum[0] + rsum[1] + rsum[2] + rsum[3];
    }
}


/* Computes max of row dwork[i] = max( abs( A(i,:) )), i=0:m-1, for || A ||_max,
 * where m and n are any size.
 * Has ceil( m/64 ) blocks of 64 threads. Each thread does one row.
 * Based on zlange_inf_kernel code, above. */
extern "C" __global__ void
zlange_max_kernel(
    int m, int n, const magmaDoubleComplex *A, int lda, double *dwork )
{
    int i = blockIdx.x*64 + threadIdx.x;
    double rmax[4] = {0, 0, 0, 0};
    int n_mod_4 = n % 4;
    n -= n_mod_4;
    
    // if beyond last row, skip row
    if ( i < m ) {
        A += i;
        
        if ( n >= 4 ) {
            const magmaDoubleComplex *Aend = A + lda*n;
            magmaDoubleComplex rA[4] = { A[0], A[lda], A[2*lda], A[3*lda] };
            A += 4*lda;
            
            while( A < Aend ) {
                rmax[0] = max_nan( rmax[0], MAGMA_Z_ABS( rA[0] ));  rA[0] = A[0];
                rmax[1] = max_nan( rmax[1], MAGMA_Z_ABS( rA[1] ));  rA[1] = A[lda];
                rmax[2] = max_nan( rmax[2], MAGMA_Z_ABS( rA[2] ));  rA[2] = A[2*lda];
                rmax[3] = max_nan( rmax[3], MAGMA_Z_ABS( rA[3] ));  rA[3] = A[3*lda];
                A += 4*lda;
            }
            
            rmax[0] = max_nan( rmax[0], MAGMA_Z_ABS( rA[0] ));
            rmax[1] = max_nan( rmax[1], MAGMA_Z_ABS( rA[1] ));
            rmax[2] = max_nan( rmax[2], MAGMA_Z_ABS( rA[2] ));
            rmax[3] = max_nan( rmax[3], MAGMA_Z_ABS( rA[3] ));
        }
    
        /* clean up code */
        switch( n_mod_4 ) {
            case 0:
                break;
    
            case 1:
                rmax[0] = max_nan( rmax[0], MAGMA_Z_ABS( A[0] ));
                break;                          
                                                
            case 2:                             
                rmax[0] = max_nan( rmax[0], MAGMA_Z_ABS( A[  0] ));
                rmax[1] = max_nan( rmax[1], MAGMA_Z_ABS( A[lda] ));
                break;                          
                                                
            case 3:                             
                rmax[0] = max_nan( rmax[0], MAGMA_Z_ABS( A[    0] ));
                rmax[1] = max_nan( rmax[1], MAGMA_Z_ABS( A[  lda] ));
                rmax[2] = max_nan( rmax[2], MAGMA_Z_ABS( A[2*lda] ));
                break;
        }
    
        /* compute final result */
        dwork[i] = max_nan( max_nan( max_nan( rmax[0], rmax[1] ), rmax[2] ), rmax[3] );
    }
}


/* Computes col sums dwork[j] = sum( abs( A(:,j) )), j=0:n-1, for || A ||_one,
 * where m and n are any size.
 * Has n blocks of NB threads each. Block j sums one column, A(:,j) into dwork[j].
 * Thread i accumulates A(i,j) + A(i+NB,j) + A(i+2*NB,j) + ... into ssum[i],
 * then threads collectively do a sum-reduction of ssum,
 * and finally thread 0 saves to dwork[j]. */
extern "C" __global__ void
zlange_one_kernel(
    int m, int n, const magmaDoubleComplex *A, int lda, double *dwork )
{
    __shared__ double ssum[64];
    int tx = threadIdx.x;
    
    A += blockIdx.x*lda;  // column j
    
    ssum[tx] = 0;
    for( int i = tx; i < m; i += 64 ) {
        ssum[tx] += MAGMA_Z_ABS( A[i] );
    }
    magma_sum_reduce< 64 >( tx, ssum );
    if ( tx == 0 ) {
        dwork[ blockIdx.x ] = ssum[0];
    }
}


/**
    Purpose
    -------
    ZLANGE  returns the value of the one norm, or the Frobenius norm, or
    the  infinity norm, or the  element of  largest absolute value  of a
    real matrix A.
    
    Description
    -----------
    ZLANGE returns the value
    
       ZLANGE = ( max(abs(A(i,j))), NORM = 'M' or 'm'
                (
                ( norm1(A),         NORM = '1', 'O' or 'o'
                (
                ( normI(A),         NORM = 'I' or 'i'
                (
                ( normF(A),         NORM = 'F', 'f', 'E' or 'e'  ** not yet supported
    
    where norm1 denotes the one norm of a matrix (maximum column sum),
    normI denotes the infinity norm of a matrix (maximum row sum) and
    normF denotes the Frobenius norm of a matrix (square root of sum of
    squares). Note that max(abs(A(i,j))) is not a consistent matrix norm.
    
    Arguments
    ---------
    @param[in]
    norm    CHARACTER*1
            Specifies the value to be returned in ZLANGE as described
            above.
    
    @param[in]
    m       INTEGER
            The number of rows of the matrix A.  M >= 0.  When M = 0,
            ZLANGE is set to zero.
    
    @param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.  When N = 0,
            ZLANGE is set to zero.
    
    @param[in]
    dA      DOUBLE PRECISION array on the GPU, dimension (LDDA,N)
            The m by n matrix A.
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array A.  LDDA >= max(M,1).
    
    @param
    dwork   (workspace) DOUBLE PRECISION array on the GPU, dimension (LWORK).
    
@cond
TODO add lwork parameter
    @param[in]
    lwork   INTEGER
            The dimension of the array WORK.
            If NORM = 'I' or 'M', LWORK >= max( 1, M ).
            If NORM = '1',        LWORK >= max( 1, N ).
            Note this is different than LAPACK, which requires WORK only for
            NORM = 'I', and does not pass LWORK.
@endcond

    @ingroup magma_zaux2
    ********************************************************************/
extern "C" double
magmablas_zlange(
    magma_norm_t norm, magma_int_t m, magma_int_t n,
    magmaDoubleComplex_const_ptr dA, magma_int_t ldda,
    magmaDouble_ptr dwork )  //, magma_int_t lwork )
{
    magma_int_t info = 0;
    if ( ! (norm == MagmaInfNorm || norm == MagmaMaxNorm || norm == MagmaOneNorm) )
        info = -1;
    else if ( m < 0 )
        info = -2;
    else if ( n < 0 )
        info = -3;
    else if ( ldda < m )
        info = -5;
    //else if ( ((norm == MagmaInfNorm || norm == MagmaMaxNorm) && (lwork < m)) ||
    //          ((norm == MagmaOneNorm) && (lwork < n)) )
    //    info = -7;

    if ( info != 0 ) {
        magma_xerbla( __func__, -(info) );
        return info;
    }
    
    /* Quick return */
    if ( m == 0 || n == 0 )
        return 0;
    
    //int i;
    dim3 threads( 64 );
    double result = -1;
    if ( norm == MagmaInfNorm ) {
        dim3 grid( magma_ceildiv( m, 64 ) );
        zlange_inf_kernel<<< grid, threads, 0, magma_stream >>>( m, n, dA, ldda, dwork );
        magma_max_nan_kernel<<< 1, 512, 0, magma_stream >>>( m, dwork );
        hipMemcpy( &result, &dwork[0], sizeof(double), hipMemcpyDeviceToHost );
    }
    else if ( norm == MagmaMaxNorm ) {
        dim3 grid( magma_ceildiv( m, 64 ) );
        zlange_max_kernel<<< grid, threads, 0, magma_stream >>>( m, n, dA, ldda, dwork );
        magma_max_nan_kernel<<< 1, 512, 0, magma_stream >>>( m, dwork );
        hipMemcpy( &result, &dwork[0], sizeof(double), hipMemcpyDeviceToHost );
    }
    else if ( norm == MagmaOneNorm ) {
        dim3 grid( n );
        zlange_one_kernel<<< grid, threads, 0, magma_stream >>>( m, n, dA, ldda, dwork );
        magma_max_nan_kernel<<< 1, 512, 0, magma_stream >>>( n, dwork );  // note N instead of M
        hipMemcpy( &result, &dwork[0], sizeof(double), hipMemcpyDeviceToHost );
    }
    
    return result;
}
