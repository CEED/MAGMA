#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 0.1) --
	Univ. of Tennessee, Knoxville
	Univ. of California, Berkeley
	Univ. of Colorado, Denver
	June 2009
*/

#include "hipblas.h"
#include "magma.h"
#include <stdio.h>

#define BLOCK_SIZE 32

__global__ void
strmm_kernel (int M, int N, float *A, int lda, float *x, int ldx)
{
	int i, k;
	int inb;
	int tyb;
	float Ystx=0;
	float *Ast, *At, *Xst;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int bmx = blockDim.y;

	__shared__ float As[BLOCK_SIZE*BLOCK_SIZE];
	__shared__ float Xs[BLOCK_SIZE*512/BLOCK_SIZE];


	tyb = ty*BLOCK_SIZE;
	Xst = Xs+tyb;
	Ast = As+tyb+tx;
	At = A+ty*lda+tx;

	inb = bmx;

	// load A
	#pragma unroll
	for (i=0; i<(M/inb); i++)
		Ast[i*BLOCK_SIZE*inb] = At[i*inb*lda];
	
	for (k=0; k<N; k+=bmx)
	{
		// load b and x
		Xst[tx] = x[bx*ldx*N+(k+ty)*ldx+tx];

		// Synchronize to make sure the matrices are loaded
		__syncthreads();

		for (i=0; i<=BLOCK_SIZE; i++)
			if (tx >=i)
				Ystx += As[i*BLOCK_SIZE+tx]*Xs[tyb+i];

		// write back y
		x[bx*ldx*N+(k+ty)*ldx+tx] = Ystx;
		Ystx = 0;
	}
}



__global__ void
diag_strtri_kernel (char uplo, char diag, float *A, float *d_dinvA, int lda)
{
	int i,j;
	float Ystx=0;
	float *Bw=NULL, *x=NULL, *y=NULL, *Aoff=NULL;
	float *my_d_dinvA;

	// Thread index
	int tx = threadIdx.x;
	int txw;

	// Block index
	int bx = blockIdx.x;
		
	Aoff = A+bx*lda*BLOCK_SIZE+bx*BLOCK_SIZE;
	my_d_dinvA = d_dinvA+bx*BLOCK_SIZE*BLOCK_SIZE;

	__shared__ float As[BLOCK_SIZE*BLOCK_SIZE];
	__shared__ float Bs[BLOCK_SIZE*BLOCK_SIZE];

	// load A
	#pragma unroll
	for (i=0; i<BLOCK_SIZE; i++)
           // read in the whole square block of my A
	   Bs[i*BLOCK_SIZE+tx] = As[i*BLOCK_SIZE+tx] = *(Aoff+i*lda+tx);
	
        // not the upper or lower diagonal	
	// Synchronize to make sure the matrices are loaded
	__syncthreads();

	Bs[tx*BLOCK_SIZE+tx] = ((diag=='u' || diag=='U')?1:(1/As[tx*BLOCK_SIZE+tx]));

	if (uplo == 'l' || uplo == 'L')
	{
		/*
		 * the lower case
		 */
		for (i=BLOCK_SIZE-2; i>=0; i--)
		{
			Ystx = 0;
			if (tx>i)
			{
				//strmv
				Bw = Bs+(i+1)*BLOCK_SIZE+i+1;
				x = As+i*BLOCK_SIZE+i+1;
				y = Bs+i*BLOCK_SIZE+i+1;

				txw = tx-i-1;
				#pragma unroll
				for (j=0; j<txw+1; j++)
					Ystx += *(Bw+j*BLOCK_SIZE+txw)*x[j];

				//sscal
				y[txw] = Ystx*(-Bs[i*BLOCK_SIZE+i]);
			}
			__syncthreads();
		}

	}
	else
	{
		/*
		 * the upper case
		 */
		for (i=0; i<BLOCK_SIZE; i++)
		{
			Ystx = 0;
			if (tx<i)
			{
				//strmv
				x = As+i*BLOCK_SIZE;
				y = Bs+i*BLOCK_SIZE;

				#pragma unroll
				for (j=tx; j<i; j++)
					Ystx += *(Bs+j*BLOCK_SIZE+tx)*x[j];

				//sscal
				y[tx] = Ystx*(-Bs[i*BLOCK_SIZE+i]);
			}
			__syncthreads();
		}


	}
		
	// write back A
	#pragma unroll
	for (i=0; i<BLOCK_SIZE; i++)
		*(my_d_dinvA+i*BLOCK_SIZE+tx) = Bs[i*BLOCK_SIZE+tx];
}

#define NUM_OF_SM 30 


extern "C" void
magmablas_strsm1(char side, char uplo, char tran, char diag, 
                int M, int N, float* A, int lda, float* b, int ldb)
{
    int status ;
    int i, nblocks;
    dim3 dimBlock;
    float *d_dinvA;

    if ((M%BLOCK_SIZE) != 0)
	{
	  printf ("warning: M=%d not divisable by BLOCK_SIZE=%d\n", M, BLOCK_SIZE);
	  exit(0);
	}

    if (side == 'l' || side == 'L')
	{
          /* inverse the diagonals
	   * Allocate device memory for the inversed diagonal blocks, size=m*BLOCK_SIZE 
	   */
	  status  = hipMalloc((void**)&d_dinvA, BLOCK_SIZE*M*sizeof(float));
          if( status != HIPBLAS_STATUS_SUCCESS) {
	    fprintf (stderr, "!!!! device memory allocation error (dipiv)\n");
   	    return ;
          }
 
	  nblocks = M/BLOCK_SIZE;
	  diag_strtri_kernel<<<nblocks, BLOCK_SIZE>>>(uplo, diag, A, d_dinvA, lda);

	  if (tran == 'N' || tran == 'n')
	  /* the non-transpose case */
	  {
		if (uplo == 'L' || uplo == 'l')
		{
		/* the lower case */
  		   for (i=0; i<M; i+=BLOCK_SIZE)
			{
			  hipblasStrmm ('L', 'L', 'N', diag, BLOCK_SIZE, N, 1.0, 
                                        d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb);
			  if (i+BLOCK_SIZE>=M)
			     break;

			  hipblasSgemm ('N', 'N', M-i-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, 
                                    A+i*lda+i+BLOCK_SIZE, lda, b+i, ldb, 1.0, b+i+BLOCK_SIZE, ldb);
			}
		}
		else
		{
		   /* the upper case */
		   for (i=M-BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
			{
			  hipblasStrmm ('L', 'U', 'N', diag, BLOCK_SIZE, N, 1.0, 
                                       d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb);

			  if (i-BLOCK_SIZE<0)
			    break;

		 	  hipblasSgemm ('N', 'N', i, N, BLOCK_SIZE, -1.0, A+i*lda, 
                                       lda, b+i, ldb, 1.0, b, ldb);
			}
		}
		}
		else
		/* the transpose case */
		{
		  if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
			for (i=M-BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
			{
			  hipblasStrmm (side, uplo, tran, diag, BLOCK_SIZE, N, 1.0, 
                                       d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb);

			  if (i-BLOCK_SIZE<0)
				break;

			  hipblasSgemm ('T', 'N', i, N, BLOCK_SIZE, -1.0, A+i, lda, b+i, 
                                       ldb, 1.0, b, ldb);
			}
		}
		else
		{
		/* the upper case */
		for (i=0; i<M; i+=BLOCK_SIZE)
		  {
		     hipblasStrmm (side, uplo, tran, diag, BLOCK_SIZE, N, 1.0, 
                                  d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb);

		     if (i+BLOCK_SIZE>=M)
			break;

		     hipblasSgemm ('T', 'N', M-i-BLOCK_SIZE, N, BLOCK_SIZE, 
                            -1.0, A+(i+BLOCK_SIZE)*lda+i, lda, b+i, ldb, 1.0, b+i+BLOCK_SIZE, ldb);
		   }
		}
	}
      }
      else
      {
	/* inverse the diagonals
	 * Allocate device memory for the inversed diagonal blocks, size=m*BLOCK_SIZE 
	 */
	status = hipMalloc((void**)&d_dinvA, BLOCK_SIZE*N*sizeof(float));
          if( status != HIPBLAS_STATUS_SUCCESS) {
	    fprintf (stderr, "!!!! device memory allocation error (dipiv)\n");
   	    return ;
          }
	nblocks = N/BLOCK_SIZE;
	diag_strtri_kernel<<<nblocks, BLOCK_SIZE>>>(uplo, diag, A, d_dinvA, lda);

	if (tran == 'N' || tran == 'n')
	/* the non-transpose case */
	{
	   if (uplo == 'L' || uplo == 'l')
		{
		/* the lower case */
		for (i=N-BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
		   {
			hipblasStrmm ('R', 'L', 'N', diag, M, BLOCK_SIZE, 1.0, 
                                     d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+ldb*i, ldb);

			if (i-BLOCK_SIZE<0)
 		 	   break;

			hipblasSgemm ('N', 'N', M, i, BLOCK_SIZE, -1.0, b+ldb*i, ldb, 
                                     A+i, lda, 1.0, b, ldb);
		   }
		}
		else
		{
		  /* the upper case */
		  for (i=0; i<N; i+=BLOCK_SIZE)
			{
			   hipblasStrmm ('R', 'U', 'N', diag, M, BLOCK_SIZE, 1.0, 
                                         d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+ldb*i, ldb);

			   if (i+BLOCK_SIZE>=N)
				break;

			   hipblasSgemm ('N', 'N', M, N-i-BLOCK_SIZE, BLOCK_SIZE, -1.0, b+i*ldb, 
                                 ldb, A+(i+BLOCK_SIZE)*lda+i, lda, 1.0, b+(i+BLOCK_SIZE)*ldb, ldb);
			}
		}
	}
	else
	/* the transpose case */
	{
	   if (uplo == 'L' || uplo == 'l')
	   {
		  /* the lower case */
		  for (i=0; i<N; i+=BLOCK_SIZE)
		     {
			hipblasStrmm ('R', 'L', 'T', diag, M, BLOCK_SIZE, 1.0, 
                                     d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+ldb*i, ldb);

			if (i+BLOCK_SIZE>=N)
			  break;

			hipblasSgemm ('N', 'T', M, N-i-BLOCK_SIZE, BLOCK_SIZE, -1.0, b+ldb*i, 
                                   ldb, A+i*lda+BLOCK_SIZE+i, lda, 1.0, b+(i+BLOCK_SIZE)*ldb, ldb);
		      }
	   }
	   else
	   {
		/* the upper case */
		for (i=N-BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
		{
		   hipblasStrmm ('R', 'U', 'T', diag, M, BLOCK_SIZE, 1.0, 
                                d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+ldb*i, ldb);

		   if (i-BLOCK_SIZE<0)
			break;

       		   hipblasSgemm ('N', 'T', M, i, BLOCK_SIZE, -1.0, b+i*ldb, ldb, 
                                A+i*lda, lda, 1.0, b, ldb);
		}
	   }
	}
   }
   hipFree(d_dinvA);
}

#undef BLOCK_SIZE
