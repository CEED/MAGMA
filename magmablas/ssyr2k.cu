#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

*/
#include "common_magma.h"

#define block_M 32
#define block_N 32
#define thread_x 32
#define thread_y 2
#define unroll_f 16

/*
 * saxpy computes c += alpha*b, where b and c are 16-element vectors.
 */
static __device__ void saxpy(
    float alpha,
    const float * __restrict__ b,
    float       * __restrict__ c )
{
    #pragma unroll
    for (int i = 0; i < unroll_f; i++)
        c[i] += alpha * b[i];
}

__global__ void
ssyr2k_kernel_even_generic(
    float *C, const float *A, const float *B,
    int m, int in, int k,
    int lda, int ldb, int ldc,
    float alpha, float beta)
{
    int tx = threadIdx.x;
    const int ty = threadIdx.y;
    int ibx = blockIdx.x;
    int iby = blockIdx.y;
    iby = (iby + ibx + 3) % gridDim.y;
    const int minor = iby&1;
    const bool bottom = ibx > iby;
    ibx = ( bottom ) ? (ibx-1) : ( iby + gridDim.y );
    iby = ( bottom ) ?  iby    : ( blockIdx.x + minor + gridDim.y );
    if ( iby > ibx )
        iby = in;
    ibx = ibx * block_M;
    iby = iby * block_N;

    const float *A1 = A;
    const float *B1 = B;
    
    B += iby + tx;
    B += __mul24( ty, ldb );
    A += ibx + tx;
    C += ibx + tx + __mul24( iby + ty*unroll_f, ldc );

    float Ap[4];
    Ap[0] = A[0];
    Ap[1] = A[lda];
    Ap[2] = A[2*lda];
    Ap[3] = A[3*lda];

    float b = B[0];
    float b2 = B[2*ldb];
    const float *Bend = B + ldb*k;
    B += 4*ldb;
    A += 4*lda;
    __shared__ float Bb[4][block_N];
    float Cb[unroll_f] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
    do {
        float Ab[4] = { Ap[0], Ap[1], Ap[2], Ap[3] };
        Bb[ty][tx] = b;
        Bb[ty+2][tx] = b2;
        __syncthreads();
        Ap[0] = A[0];
        Ap[1] = A[lda];
        Ap[2] = A[2*lda];
        Ap[3] = A[3*lda];
        b = B[0];
        b2 = B[2*ldb];
        saxpy( Ab[0], &Bb[0][ty*unroll_f], Cb );
        saxpy( Ab[1], &Bb[1][ty*unroll_f], Cb );
        saxpy( Ab[2], &Bb[2][ty*unroll_f], Cb );
        saxpy( Ab[3], &Bb[3][ty*unroll_f], Cb );
        A += 4*lda;
        B += 4*ldb;
        __syncthreads();
    } while (B < Bend);
    Bb[ty][tx] = b;
    Bb[ty+2][tx] = b2;
    __syncthreads();
    saxpy( Ap[0], &Bb[0][ty*unroll_f], Cb );
    saxpy( Ap[1], &Bb[1][ty*unroll_f], Cb );
    saxpy( Ap[2], &Bb[2][ty*unroll_f], Cb );
    saxpy( Ap[3], &Bb[3][ty*unroll_f], Cb );
    __syncthreads();
     
    // -- 2nd Half
    B = A1;
    A = B1;
    int tlda = lda; lda = ldb; ldb = tlda;
    B += iby + tx;
    B += __mul24( ty, ldb );
    A += ibx + tx;
    Ap[0] = A[0];
    Ap[1] = A[lda];
    Ap[2] = A[2*lda];
    Ap[3] = A[3*lda];
    b = B[0];
    b2 = B[2*ldb];
    const float *Bend1 = B + ldb*k;
    B += 4*ldb;
    A += 4*lda;
    do {
        float Ab[4] = { Ap[0], Ap[1], Ap[2], Ap[3] };
        Bb[ty][tx] = b;
        Bb[ty+2][tx] = b2;
        __syncthreads();
        Ap[0] = A[0];
        Ap[1] = A[lda];
        Ap[2] = A[2*lda];
        Ap[3] = A[3*lda];
        b = B[0];
        b2 = B[2*ldb];
        saxpy( Ab[0], &Bb[0][ty*unroll_f], Cb );
        saxpy( Ab[1], &Bb[1][ty*unroll_f], Cb );
        saxpy( Ab[2], &Bb[2][ty*unroll_f], Cb );
        saxpy( Ab[3], &Bb[3][ty*unroll_f], Cb );
        A += 4*lda;
        B += 4*ldb;
        __syncthreads();
    } while (B < Bend1);
    Bb[ty][tx] = b;
    Bb[ty+2][tx] = b2;
    __syncthreads();
    saxpy( Ap[0], &Bb[0][ty*unroll_f], Cb );
    saxpy( Ap[1], &Bb[1][ty*unroll_f], Cb );
    saxpy( Ap[2], &Bb[2][ty*unroll_f], Cb );
    saxpy( Ap[3], &Bb[3][ty*unroll_f], Cb );

    lda = 0;

    if ( iby < ibx ) {
        tx = 15;
    }
    else {
        if ( tx > 15 ) {
            if ( ty == 0 ) {
                lda = 1;
                tx  = 15;
            }
            else {
                lda = 1;
                tx -= 16;
            }
        }
        else {
            if ( ty == 0 ) {
                lda = 1;
            }
            else {
                lda = 2;
                tx  = 32;
            }
        }
    }
    if ( (ibx + threadIdx.x) >= m )
        tx = -1;
    
    switch(tx) {
        case 0:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                break;
        case 1:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                break;
        case 2:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                break;
        case 3:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                break;
        case 4:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                break;
        case 5:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                break;
        case 6:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                break;
        case 7:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                break;
        case 8:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                break;
        case 9:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                break;
        case 10:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                break;
        case 11:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[11] + beta*C[0];  C += ldc;
                break;
        case 12:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[11] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[12] + beta*C[0];  C += ldc;
                break;
        case 13:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[11] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[12] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[13] + beta*C[0];  C += ldc;
                break;
        case 14:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[11] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[12] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[13] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[14] + beta*C[0];  C += ldc;
                break;
        case 15:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[11] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[12] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[13] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[14] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[15] + beta*C[0];  C += ldc;
                break;
        default:
                break;
    }
}

__global__ void
ssyr2k_kernel_odd_generic(
    float *C, const float *A, const float *B,
    int m, int in, int k,
    int lda, int ldb, int ldc,
    float alpha, float beta)
{
    int tx = threadIdx.x;
    const int ty = threadIdx.y;
    int ibx = blockIdx.x;
    int iby = blockIdx.y;
    iby = (iby + ibx) % gridDim.y;
    int minor = iby &1;
    bool bottom = ibx >= iby;
    ibx = ( bottom ) ? ibx : ( iby + gridDim.y - 1 );
    iby = ( bottom ) ? iby : ( blockIdx.x + minor + gridDim.y );
    if ( iby > ibx )
        iby = in + 1;
    ibx = ibx * block_M;
    iby = iby * block_N;

    const float *A1 = A;
    const float *B1 = B;

    B += iby + tx;
    B += __mul24( ty, ldb );
    A += ibx + tx;
    C += ibx + tx + __mul24( iby + ty*unroll_f, ldc );
    float Ap[4] = { A[0], A[lda], A[2*lda], A[3*lda] };

    float b = B[0];
    float b2 = B[2*ldb];

    const float *Bend = B + ldb*k;
    B += 4*ldb;
    A += 4*lda;
    __shared__ float Bb[4][block_N];
    float Cb[unroll_f] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
    do {
        float Ab[4] = { Ap[0], Ap[1], Ap[2], Ap[3] };
        Bb[ty][tx] = b;
        Bb[ty+2][tx] = b2;
        __syncthreads();
        Ap[0] = A[0];
        Ap[1] = A[lda];
        Ap[2] = A[2*lda];
        Ap[3] = A[3*lda];
        b = B[0];
        b2 = B[2*ldb];
        saxpy( Ab[0], &Bb[0][ty*unroll_f], Cb );
        saxpy( Ab[1], &Bb[1][ty*unroll_f], Cb );
        saxpy( Ab[2], &Bb[2][ty*unroll_f], Cb );
        saxpy( Ab[3], &Bb[3][ty*unroll_f], Cb );
        A += 4*lda;
        B += 4*ldb;
        __syncthreads();
    } while (B < Bend);
    Bb[ty][tx] = b;
    Bb[ty+2][tx] = b2;
    __syncthreads();
    saxpy( Ap[0], &Bb[0][ty*unroll_f], Cb );
    saxpy( Ap[1], &Bb[1][ty*unroll_f], Cb );
    saxpy( Ap[2], &Bb[2][ty*unroll_f], Cb );
    saxpy( Ap[3], &Bb[3][ty*unroll_f], Cb );
    __syncthreads();

    B = A1;
    A = B1;
    int tlda = lda; lda = ldb; ldb = tlda;
    B += iby + tx;
    B += __mul24( ty, ldb );
    A += ibx + tx;
    Ap[0] = A[0];
    Ap[1] = A[lda];
    Ap[2] = A[2*lda];
    Ap[3] = A[3*lda];
    b = B[0];
    b2 = B[2*ldb];
    const float *Bend1 = B + ldb*k;
    B += 4*ldb;
    A += 4*lda;
    do {
        float Ab[4] = { Ap[0], Ap[1], Ap[2], Ap[3] };
        Bb[ty][tx] = b;
        Bb[ty+2][tx] = b2;
        __syncthreads();
        Ap[0] = A[0];
        Ap[1] = A[lda];
        Ap[2] = A[2*lda];
        Ap[3] = A[3*lda];
        b  = B[0];
        b2 = B[2*ldb];
        saxpy( Ab[0], &Bb[0][ty*unroll_f], Cb );
        saxpy( Ab[1], &Bb[1][ty*unroll_f], Cb );
        saxpy( Ab[2], &Bb[2][ty*unroll_f], Cb );
        saxpy( Ab[3], &Bb[3][ty*unroll_f], Cb );
        A += 4*lda;
        B += 4*ldb;
        __syncthreads();
    } while (B < Bend1);
    Bb[ty][tx] = b;
    Bb[ty+2][tx] = b2;
    __syncthreads();
    saxpy( Ap[0], &Bb[0][ty*unroll_f], Cb );
    saxpy( Ap[1], &Bb[1][ty*unroll_f], Cb );
    saxpy( Ap[2], &Bb[2][ty*unroll_f], Cb );
    saxpy( Ap[3], &Bb[3][ty*unroll_f], Cb );
    __syncthreads();

    lda = 0;
    
    if ( iby < ibx ) {
        tx = 15;
    }
    else {
        if ( tx > 15 ) {
            if ( ty == 0 ) {
                lda = 1;
                tx = 15;
            }
            else {
                lda = 1;
                tx -= 16;
            }
        }
        else {
            if ( ty == 0 ) {
                lda = 1;
            }
            else {
                lda = 2;
                tx = 32;
            }
        }
    }
    if ( (ibx + threadIdx.x) >= m )
        tx = -1;

    switch(tx) {
        case 0:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                break;
        
        case 1:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                break;
        
        case 2:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                break;
        
        case 3:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                break;
        
        case 4:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                break;
        
        case 5:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                break;
        case 6:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                break;
        case 7:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                break;
        case 8:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                break;
        case 9:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                break;
        case 10:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                break;
        case 11:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[11] + beta*C[0];  C += ldc;
                break;
        case 12:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[11] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[12] + beta*C[0];  C += ldc;
                break;
        case 13:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[11] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[12] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[13] + beta*C[0];  C += ldc;
                break;
        case 14:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[11] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[12] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[13] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[14] + beta*C[0];  C += ldc;
                break;
        case 15:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[11] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[12] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[13] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[14] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[15] + beta*C[0];  C += ldc;
                break;
        default:
                break;
    }
}

__global__ void
ssyr2k_kernel_even_special(
    int flag,
    float *C, const float *A, const float *B,
    int m, int in, int k,
    int lda, int ldb, int ldc,
    float alpha, float beta)
{
    int tx = threadIdx.x;
    const int ty = threadIdx.y;
    int ibx = blockIdx.x;
    int iby = blockIdx.y;
    if ( flag == 1 )
        iby = (iby + ibx) % gridDim.y;
    const int minor = iby&1;
    const bool bottom = ibx > iby;
    ibx = ( bottom ) ? (ibx-1) : ( iby + gridDim.y );
    iby = ( bottom ) ?  iby    : ( blockIdx.x + minor + gridDim.y );
    if ( iby > ibx )
        iby = in;
    ibx = ibx * block_M;
    iby = iby * block_N;

    const float *A1 = A;
    const float *B1 = B;
    
    B += iby + tx;
    B += __mul24( ty, ldb );
    A += ibx + tx;
    C += ibx + tx + __mul24( iby + ty*unroll_f, ldc );
    float Ap[4] = { A[0], A[lda], A[2*lda], A[3*lda] };
    float b = B[0];
    float b2 = B[2*ldb];
    const float *Bend = B + ldb*k;
    B += 4*ldb;
    A += 4*lda;
    __shared__ float Bb[4][block_N];
    float Cb[unroll_f] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
    do {
        float Ab[4] = { Ap[0], Ap[1], Ap[2], Ap[3] };
        Bb[ty][tx] = b;
        Bb[ty+2][tx] = b2;
        __syncthreads();
        Ap[0] = A[0];
        Ap[1] = A[lda];
        Ap[2] = A[2*lda];
        Ap[3] = A[3*lda];
        b  = B[0];
        b2 = B[2*ldb];
        saxpy( Ab[0], &Bb[0][ty*unroll_f], Cb );
        saxpy( Ab[1], &Bb[1][ty*unroll_f], Cb );
        saxpy( Ab[2], &Bb[2][ty*unroll_f], Cb );
        saxpy( Ab[3], &Bb[3][ty*unroll_f], Cb );
        A += 4*lda;
        B += 4*ldb;
        __syncthreads();
    } while (B < Bend);
    Bb[ty][tx] = b;
    Bb[ty+2][tx] = b2;
    __syncthreads();
    saxpy( Ap[0], &Bb[0][ty*unroll_f], Cb );
    saxpy( Ap[1], &Bb[1][ty*unroll_f], Cb );
    saxpy( Ap[2], &Bb[2][ty*unroll_f], Cb );
    saxpy( Ap[3], &Bb[3][ty*unroll_f], Cb );

    // -- 2nd Half
    B = A1;
    A = B1;
    int tlda = lda; lda = ldb; ldb = tlda;

    B += iby + tx;
    B += __mul24( ty, ldb );
    A += ibx + tx;
    Ap[0] = A[0];
    Ap[1] = A[lda];
    Ap[2] = A[2*lda];
    Ap[3] = A[3*lda];
    b = B[0];
    b2 = B[2*ldb];
    const float *Bend1 = B + ldb*k;
    B += 4*ldb;
    A += 4*lda;
    do {
        float Ab[4] = { Ap[0], Ap[1], Ap[2], Ap[3] };
        Bb[ty][tx] = b;
        Bb[ty+2][tx] = b2;
        __syncthreads();
        Ap[0] = A[0];
        Ap[1] = A[lda];
        Ap[2] = A[2*lda];
        Ap[3] = A[3*lda];
        b = B[0];
        b2 = B[2*ldb];
        saxpy( Ab[0], &Bb[0][ty*unroll_f], Cb );
        saxpy( Ab[1], &Bb[1][ty*unroll_f], Cb );
        saxpy( Ab[2], &Bb[2][ty*unroll_f], Cb );
        saxpy( Ab[3], &Bb[3][ty*unroll_f], Cb );
        A += 4*lda;
        B += 4*ldb;
        __syncthreads();
    } while (B < Bend1);
    Bb[ty][tx] = b;
    Bb[ty+2][tx] = b2;
    __syncthreads();
    saxpy( Ap[0], &Bb[0][ty*unroll_f], Cb );
    saxpy( Ap[1], &Bb[1][ty*unroll_f], Cb );
    saxpy( Ap[2], &Bb[2][ty*unroll_f], Cb );
    saxpy( Ap[3], &Bb[3][ty*unroll_f], Cb );

    lda = 0;
    
    if ( iby < ibx ) {
        /*
        #pragma unroll 16
        for (int i = 0; i < unroll_f; i++, C += ldc)
            C[0] = alpha*Cb[i] + beta*C[0];
        */
        tx = 15;
    }
    else {
        if ( tx > 15 ) {
            if ( ty == 0 ) {
                lda = 1;
                tx = 15;
            }
            else {
                lda = 1;
                tx -= 16;
            }
        }
        else {
            if ( ty == 0 ) {
                lda = 1;
            }
            else {
                lda = 2;
                tx = 32;
            }
        }
    }

    switch(tx) {
        case 0:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                break;
        case 1:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                break;
        case 2:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                break;
        case 3:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                break;
        case 4:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                break;
        case 5:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                break;
        case 6:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                break;
        case 7:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                break;
        case 8:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                break;
        case 9:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                break;
        case 10:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                break;
        case 11:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[11] + beta*C[0];  C += ldc;
                break;
        case 12:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[11] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[12] + beta*C[0];  C += ldc;
                break;
        case 13:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[11] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[12] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[13] + beta*C[0];  C += ldc;
                break;
        case 14:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[11] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[12] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[13] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[14] + beta*C[0];  C += ldc;
                break;
        case 15:
                C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[11] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[12] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[13] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[14] + beta*C[0];  C += ldc;
                C[0] = alpha*Cb[15] + beta*C[0];  C += ldc;
                break;
        default:
                break;
    }
}

__global__ void
ssyr2k_kernel_odd_special(
    int flag,
    float *C, const float *A, const float *B,
    int m, int in, int k,
    int lda, int ldb, int ldc,
    float alpha, float beta)
{
    int tx = threadIdx.x;
    const int ty = threadIdx.y;
    int ibx = blockIdx.x;
    int iby = blockIdx.y;
    if ( flag == 1 )
        iby = (iby + ibx) % gridDim.y;
    int minor = iby & 1;
    bool bottom = (ibx >= iby);
    ibx = ( bottom ) ? ibx : ( iby + gridDim.y - 1 );
    iby = ( bottom ) ? iby : ( blockIdx.x + minor + gridDim.y );
    if ( iby > ibx )
        iby = in + 1;
    ibx = ibx * block_M;
    iby = iby * block_N;

    const float *A1 = A;
    const float *B1 = B;

    if ( iby > ibx ) {
        return;
    }
    else {
        B += iby + tx;
        B += __mul24( ty, ldb );
        A += ibx + tx;
        C += ibx + tx + __mul24( iby + ty*unroll_f, ldc );
        float Ap[4] = { A[0], A[lda], A[2*lda], A[3*lda] };

        float b = B[0];
        float b2 = B[2*ldb];

        const float *Bend = B + ldb*k;
        B += 4*ldb;
        A += 4*lda;
        __shared__ float Bb[4][block_N];
        float Cb[unroll_f] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
        do {
            float Ab[4] = { Ap[0], Ap[1], Ap[2], Ap[3] };
            Bb[ty][tx] = b;
            Bb[ty+2][tx] = b2;
            __syncthreads();
            Ap[0] = A[0];
            Ap[1] = A[lda];
            Ap[2] = A[2*lda];
            Ap[3] = A[3*lda];
            b = B[0];
            b2 = B[2*ldb];
            saxpy( Ab[0], &Bb[0][ty*unroll_f], Cb );
            saxpy( Ab[1], &Bb[1][ty*unroll_f], Cb );
            saxpy( Ab[2], &Bb[2][ty*unroll_f], Cb );
            saxpy( Ab[3], &Bb[3][ty*unroll_f], Cb );
            A += 4*lda;
            B += 4*ldb;
            __syncthreads();
        } while (B < Bend);
        Bb[ty][tx] = b;
        Bb[ty+2][tx] = b2;
        __syncthreads();
        saxpy( Ap[0], &Bb[0][ty*unroll_f], Cb );
        saxpy( Ap[1], &Bb[1][ty*unroll_f], Cb );
        saxpy( Ap[2], &Bb[2][ty*unroll_f], Cb );
        saxpy( Ap[3], &Bb[3][ty*unroll_f], Cb );
        B = A1;
        A = B1;
        int tlda = lda; lda = ldb; ldb = tlda;
        B += iby + tx;
        B += __mul24( ty, ldb );
        A += ibx + tx;
        Ap[0] = A[0];
        Ap[1] = A[lda];
        Ap[2] = A[2*lda];
        Ap[3] = A[3*lda];
        b = B[0];
        b2 = B[2*ldb];
        const float *Bend1 = B + ldb*k;
        B += 4*ldb;
        A += 4*lda;
        do {
            float Ab[4] = { Ap[0], Ap[1], Ap[2], Ap[3] };
            Bb[ty][tx] = b;
            Bb[ty+2][tx] = b2;
            __syncthreads();
            Ap[0] = A[0];
            Ap[1] = A[lda];
            Ap[2] = A[2*lda];
            Ap[3] = A[3*lda];
            b = B[0];
            b2 = B[2*ldb];
            saxpy( Ab[0], &Bb[0][ty*unroll_f], Cb );
            saxpy( Ab[1], &Bb[1][ty*unroll_f], Cb );
            saxpy( Ab[2], &Bb[2][ty*unroll_f], Cb );
            saxpy( Ab[3], &Bb[3][ty*unroll_f], Cb );
            A += 4*lda;
            B += 4*ldb;
            __syncthreads();
        } while (B < Bend1);
        Bb[ty][tx] = b;
        Bb[ty+2][tx] = b2;
        __syncthreads();
        saxpy( Ap[0], &Bb[0][ty*unroll_f], Cb );
        saxpy( Ap[1], &Bb[1][ty*unroll_f], Cb );
        saxpy( Ap[2], &Bb[2][ty*unroll_f], Cb );
        saxpy( Ap[3], &Bb[3][ty*unroll_f], Cb );

        lda = 0;
        
        if ( iby < ibx ) {
            /*
            #pragma unroll 16
            for( int i = 0; i < unroll_f; i++, C += ldc )
                C[0] = alpha*Cb[i] + beta*C[0];
            */
            tx = 15;
        }
        else {
            if ( tx > 15 ) {
                if ( ty == 0 ) {
                    lda = 1;
                    tx = 15;
                }
                else {
                    lda = 1;
                    tx -= 16;
                }
            }
            else {
                if ( ty == 0 ) {
                    lda = 1;
                }
                else {
                    lda = 2;
                    tx = 32;
                }
            }
        }

        switch( tx ) {
            case 0:
                    C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                    break;
            
            case 1:
                    C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                    break;
            
            case 2:
                    C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                    break;
            
            case 3:
                    C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                    break;
            
            case 4:
                    C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                    break;
            
            case 5:
                    C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                    break;
            case 6:
                    C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                    break;
            case 7:
                    C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                    break;
            case 8:
                    C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                    break;
            case 9:
                    C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                    break;
            case 10:
                    C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                    break;
            case 11:
                    C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[11] + beta*C[0];  C += ldc;
                    break;
            case 12:
                    C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[11] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[12] + beta*C[0];  C += ldc;
                    break;
            case 13:
                    C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[11] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[12] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[13] + beta*C[0];  C += ldc;
                    break;
            case 14:
                    C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[11] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[12] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[13] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[14] + beta*C[0];  C += ldc;
                    break;
            case 15:
                    C[0] = alpha*Cb[0] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[1] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[2] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[3] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[4] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[5] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[6] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[7] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[8] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[9] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[10] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[11] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[12] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[13] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[14] + beta*C[0];  C += ldc;
                    C[0] = alpha*Cb[15] + beta*C[0];  C += ldc;
                    break;
            default:
                        break;
        }
    }
}

/**
    @deprecated
    
    Purpose
    -------
    SSYR2K performs one of the symmetric rank 2k operations
        C := alpha*A*B^T + alpha*B*A^T + beta*C,
    or
        C := alpha*A^T*B + alpha*B^T*A + beta*C,

    where alpha and beta are scalars, C is an n by n symmetric matrix
    and A and B are n by k matrices in the first case and k by n
    matrices in the second case.

    This implementation is for UPLO == MagmaLower and TRANS == MagmaNoTrans.

    Assumptions
    -----------
    Both lda and ldb must be multiple of 32.
    Parameter k must be divisible by 8 - note that this algorithm was developed
    for the tridiagonal factorization and k in that case would be the blocking size.
    We always request the blocking size to be divisible by at least 16.

    This kernel goes to about 300 GFlop/s on the GTX280.
    
    Arguments
    ---------
    @param[in]
    uplo    magma_uplo_t
            On entry, UPLO specifies whether the upper or lower
            triangular part of the array C is to be referenced as
            follows:
      -        MagmaUpper   Only the upper triangular part of C is referenced.
      -        MagmaLower   Only the lower triangular part of C is referenced. -- not implemented.
            \n
            Only MagmaLower is implemented.
   
    @param[in]
    trans   magma_trans_t
            On entry, TRANS specifies the operation to be performed as
            follows:
      -        TRANS = MagmaNoTrans     C := alpha*A*B**T + alpha*B*A**T + beta*C.
      -        TRANS = MagmaTrans       C := alpha*A**T*B + alpha*B**T*A + beta*C. -- not implemented.
      -        TRANS = MagmaConjTrans   C := alpha*A**T*B + alpha*B**T*A + beta*C. -- not implemented.
            \n
            Only MagmaNoTrans is implemented.

    @param[in]
    n       INTEGER
            On entry, N specifies the order of the matrix C. N must be
            at least zero.
   
    @param[in]
    k       INTEGER
            On entry with TRANS = MagmaNoTrans, K specifies the number
            of columns of the matrices A and B, and on entry with
            TRANS = MagmaTrans or MagmaConjTrans, K specifies the number
            of rows of the matrices A and B. K must be at least zero.
            \n
            Assumption: k must be divisible by 8.
   
    @param[in]
    alpha   REAL
            On entry, ALPHA specifies the scalar alpha.
   
    @param[in]
    A       A is REAL array of DIMENSION ( LDA, ka ), where ka is
            k when TRANS = MagmaNoTrans, and is n otherwise.
            Before entry with TRANS = MagmaNoTrans, the leading n by k
            part of the array A must contain the matrix A, otherwise
            the leading k by n part of the array A must contain the
            matrix A.
   
    @param[in]
    lda     INTEGER
            On entry, LDA specifies the first dimension of A as declared
            in the calling (sub) program. When TRANS = MagmaNoTrans
            then LDA must be at least max( 1, n ), otherwise LDA must
            be at least max( 1, k ).
            \n
            Assumption: lda must be divisible by 32.
   
    @param[in]
    B       REAL array of DIMENSION ( LDB, kb ), where kb is
            k when TRANS = MagmaNoTrans, and is n otherwise.
            Before entry with TRANS = MagmaNoTrans, the leading n by k
            part of the array B must contain the matrix B, otherwise
            the leading k by n part of the array B must contain the
            matrix B.
   
    @param[in]
    ldb     INTEGER
            On entry, LDB specifies the first dimension of B as declared
            in the calling (sub) program. When TRANS = MagmaNoTrans
            then LDB must be at least max( 1, n ), otherwise LDB must
            be at least max( 1, k ).
            \n
            Assumption: ldb must be divisible by 32.
   
    @param[in]
    BETA    REAL
            On entry, BETA specifies the scalar beta.

    @param[in,out]
    C       REAL array of DIMENSION ( LDC, n ).
            Before entry with UPLO = MagmaUpper, the leading n by n
            upper triangular part of the array C must contain the upper
            triangular part of the symmetric matrix and the strictly
            lower triangular part of C is not referenced. On exit, the
            upper triangular part of the array C is overwritten by the
            upper triangular part of the updated matrix.
            \n
            Before entry with UPLO = MagmaLower, the leading n by n
            lower triangular part of the array C must contain the lower
            triangular part of the symmetric matrix and the strictly
            upper triangular part of C is not referenced. On exit, the
            lower triangular part of the array C is overwritten by the
            lower triangular part of the updated matrix.
   
    @param[in]
    LDC     INTEGER
            On entry, LDC specifies the first dimension of C as declared
            in the calling (sub) program. LDC must be at least max( 1, n ).

    @ingroup magma_sblas3
    ********************************************************************/
extern "C" void
magmablas_ssyr2k(
    magma_uplo_t uplo, magma_trans_t trans, magma_int_t n, magma_int_t k,
    float alpha,
    const float *A, magma_int_t lda,
    const float *B, magma_int_t ldb,
    float beta,
    float *C, magma_int_t ldc)
{
    // only uplo=Lower && trans=NoTrans implemented.
    magma_int_t info = 0;
    if ( uplo != MagmaLower )  /*&& uplo != MagmaUpper*/
        info = -1;
    else if ( trans != MagmaNoTrans )  /*&& trans != MagmaTrans && trans != MagmaConjTrans*/
        info = -2;
    else if ( n < 0 )
        info = -3;
    else if ( k < 0 || k % 8 != 0 )
        info = -4;
    else if ( (trans == MagmaNoTrans ? lda < n : lda < k) || (lda % 32 != 0) )
        info = -7;
    else if ( (trans == MagmaNoTrans ? ldb < n : lda < k) || (ldb % 32 != 0) )
        info = -9;
    else if ( ldc < n )
        info = -12;
    
    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }
    
    magma_int_t in = n / block_M;
    magma_int_t flag = 1;
    if ( lda >= 1024 && lda % 256 == 0 )
        flag = 1; // It was kept to reorder the GPUs internal scheduling of thread blocks.
    
    if ( n % block_M == 0 ) {
        if ( in & 1 ) {
            dim3 grid( in, (in/2 + 1));
            dim3 threads( thread_x, thread_y );
            ssyr2k_kernel_odd_special<<< grid, threads, 0, magma_stream >>>
                (flag, C, A, B, n, in/2, k, lda, ldb, ldc, alpha, beta);
        }
        else {
            dim3 grid( in + 1, (in/2));
            dim3 threads( thread_x, thread_y );
            ssyr2k_kernel_even_special<<< grid, threads, 0, magma_stream >>>
                (flag, C, A, B, n, in/2, k, lda, ldb, ldc, alpha, beta);
        }
    }
    else {
        in += 1;
        if ( in & 1 ) {
            dim3 grid( in, (in/2 + 1));
            dim3 threads( thread_x, thread_y );
            ssyr2k_kernel_odd_generic<<< grid, threads, 0, magma_stream >>>
                (C, A, B, n, in/2, k, lda, ldb, ldc, alpha, beta);
        }
        else {
            dim3 grid( in + 1, (in/2));
            dim3 threads( thread_x, thread_y );
            ssyr2k_kernel_even_generic<<< grid, threads, 0, magma_stream >>>
                (C, A, B, n, in/2, k, lda, ldb, ldc, alpha, beta);
        }
    }
}
