
#include <hip/hip_runtime.h>
/*
    -- MAGMA (version 0.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
	Univ. of Colorado, Denver
       November 2009
*/

__global__ void dtranspose_32( double *B, int ldb, double *A, int lda )
{	
	__shared__ double a[32][33];
	
	int inx = threadIdx.x;
	int iny = threadIdx.y;
	int ibx = blockIdx.x*32;
	int iby = blockIdx.y*32;
	
	A += ibx + inx + __mul24( iby + iny, lda );
	B += iby + inx + __mul24( ibx + iny, ldb );
	
	a[iny+0][inx] = A[0*lda];
	a[iny+8][inx] = A[8*lda];
	a[iny+16][inx] = A[16*lda];
	a[iny+24][inx] = A[24*lda];
	
	__syncthreads();
	
	B[0*ldb] = a[inx][iny+0];
	B[8*ldb] = a[inx][iny+8];
	B[16*ldb] = a[inx][iny+16];
	B[24*ldb] = a[inx][iny+24];
} 

//
//	m, n - dimensions in the source matrix
//
extern "C" void 
magmablas_dtranspose(double *odata, int ldo, 
                    double *idata, int ldi, 
                    int m, int n )
{
	//assert( (m%32) == 0 && (n%32) == 0, "misaligned transpose" );
	dim3 threads( 32, 8, 1 );
	dim3 grid( m/32, n/32, 1 );
	dtranspose_32<<< grid, threads >>>( odata, ldo, idata, ldi );
}
