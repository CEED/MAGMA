#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c

*/
#include "common_magma.h"

#define BLOCK_SIZE 64

// copy & conjugate a single vector of length n.
// TODO: this was modeled on the old zswap routine. Update to new zlacpy code for 2D matrix?

__global__ void zlacpy_cnjg_kernel(
    int n,
    magmaDoubleComplex *A1, int lda1,
    magmaDoubleComplex *A2, int lda2 )
{
    int x = threadIdx.x + blockDim.x*blockIdx.x;
    int offset1 = x*lda1;
    int offset2 = x*lda2;
    if ( x < n )
    {
        A2[offset2] = MAGMA_Z_CNJG( A1[offset1] );
    }
}


extern "C" void 
magmablas_zlacpy_cnjg_q(
    magma_int_t n,
    magmaDoubleComplex_ptr dA1, magma_int_t lda1, 
    magmaDoubleComplex_ptr dA2, magma_int_t lda2,
    magma_queue_t queue )
{
    dim3 threads( BLOCK_SIZE );
    dim3 blocks( magma_ceildiv( n, BLOCK_SIZE ) );
    zlacpy_cnjg_kernel<<< blocks, threads, 0, queue >>>( n, dA1, lda1, dA2, lda2 );
}


extern "C" void 
magmablas_zlacpy_cnjg(
    magma_int_t n,
    magmaDoubleComplex_ptr dA1, magma_int_t lda1, 
    magmaDoubleComplex_ptr dA2, magma_int_t lda2)
{
    magmablas_zlacpy_cnjg_q( n, dA1, lda1, dA2, lda2, magma_stream );
}
