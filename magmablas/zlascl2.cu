#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c

       @author Theo Mary
*/
#include "common_magma.h"

#define NB 64


// each thread block does one NB x n block row of A.
// each thread does one row, starting from left edge and moving right.
__global__ void
zlascl2_full(int m, int n, const double* D, magmaDoubleComplex* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    double mul = D[ind];
    A += ind;
    if (ind < m) {
        for (int j=0; j < n; j++ )
            A[j*lda] *= mul;
    }
}


// each thread block does one NB x n block row of A.
// each thread does one row, starting from left edge and moving right to diagonal.
__global__ void
zlascl2_lower(int m, int n, const double* D, magmaDoubleComplex* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    int break_d = (ind < n) ? ind : n-1;

    double mul = D[ind];
    A += ind;
    if (ind < m) {
        for (int j=0; j <= break_d; j++ )
            A[j*lda] *= mul;
    }
}


// each thread block does one NB x n block row of A.
// each thread does one row, starting from right edge and moving left to diagonal.
__global__ void
zlascl2_upper(int m, int n, const double *D, magmaDoubleComplex* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    double mul = D[ind];
    A += ind;
    if (ind < m) {
        for (int j=n-1; j >= ind; j--)
            A[j*lda] *= mul;
    }
}


/**
    Purpose
    -------
    ZLASCL2 scales the M by N complex matrix A by the real diagonal matrix dD.
    TYPE specifies that A may be full, upper triangular, lower triangular.

    Arguments
    ---------
    @param[in]
    type    magma_type_t
            TYPE indices the storage type of the input matrix A.
            = MagmaFull:   full matrix.
            = MagmaLower:  lower triangular matrix.
            = MagmaUpper:  upper triangular matrix.
            Other formats that LAPACK supports, MAGMA does not currently support.

    @param[in]
    m       INTEGER
            The number of rows of the matrix A.  M >= 0.

    @param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.

    @param[in]
    dD      DOUBLE PRECISION vector, dimension (M)
            The diagonal matrix containing the scalar factors. Stored as a vector.

    @param[in,out]
    dA      COMPLEX*16 array, dimension (LDDA,N)
            The matrix to be scaled by dD.  See TYPE for the
            storage type.

    @param[in]
    ldda    INTEGER
            The leading dimension of the array A.  LDDA >= max(1,M).

    @param[out]
    info    INTEGER
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value.
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_zaux2
    ********************************************************************/
extern "C" void
magmablas_zlascl2_q(
    magma_type_t type, magma_int_t m, magma_int_t n,
    magmaDouble_const_ptr dD,
    magmaDoubleComplex_ptr dA, magma_int_t ldda,
    magma_queue_t queue,
    magma_int_t *info )
{
    *info = 0;
    if ( type != MagmaLower && type != MagmaUpper && type != MagmaFull )
        *info = -1;
    else if ( m < 0 )
        *info = -2;
    else if ( n < 0 )
        *info = -3;
    else if ( ldda < max(1,m) )
        *info = -5;
    
    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return;  //info;
    }
    
    dim3 grid( magma_ceildiv( m, NB ) );
    dim3 threads( NB );
    
    if (type == MagmaLower) {
        zlascl2_lower <<< grid, threads, 0, queue >>> (m, n, dD, dA, ldda);
    }
    else if (type == MagmaUpper) {
        zlascl2_upper <<< grid, threads, 0, queue >>> (m, n, dD, dA, ldda);
    }
    else if (type == MagmaFull) {
        zlascl2_full  <<< grid, threads, 0, queue >>> (m, n, dD, dA, ldda);
    }
}


/**
    @see magmablas_zlascl2_q
    @ingroup magma_zaux2
    ********************************************************************/
extern "C" void
magmablas_zlascl2(
    magma_type_t type, magma_int_t m, magma_int_t n,
    magmaDouble_const_ptr dD,
    magmaDoubleComplex_ptr dA, magma_int_t ldda, magma_int_t *info )
{
    magmablas_zlascl2_q( type, m, n, dD, dA, ldda, magma_stream, info );
}
