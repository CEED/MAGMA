#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c

*/
#include "common_magma.h"


/*********************************************************/
/*
 *  Swap diagonal blocks of two matrices.
 *  Each thread block swaps one diagonal block.
 *  Each thread iterates across one row of the block.
 */

__global__ void 
zswapdblk_kernel( int nb,
                  magmaDoubleComplex *dA, int ldda, int inca,
                  magmaDoubleComplex *dB, int lddb, int incb )
{
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    dA += tx + bx * nb * (ldda + inca);
    dB += tx + bx * nb * (lddb + incb);

    magmaDoubleComplex tmp;

    #pragma unroll
    for( int i = 0; i < nb; i++ ){
        tmp        = dA[i*ldda];
        dA[i*ldda] = dB[i*lddb];
        dB[i*lddb] = tmp;
    }
}


/**
    Purpose
    -------
    zswapdblk swaps diagonal blocks of size nb x nb between matrices
    dA and dB on the GPU. It swaps nblocks = n/nb blocks.
    For i = 1 .. nblocks, submatrices
    dA( i*nb*inca, i*nb ) and
    dB( i*nb*incb, i*nb ) are swapped.
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of columns of the matrices dA and dB.  N >= 0.

    @param[in]
    nb      INTEGER
            The size of diagonal blocks.
            NB > 0 and NB <= maximum threads per CUDA block (512 or 1024).

    @param[in,out]
    dA      COMPLEX_16 array, dimension (LDDA,N)
            The matrix dA.

    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.
            LDDA >= (nblocks - 1)*nb*inca + nb.

    @param[in]
    inca    INTEGER
            The row increment between diagonal blocks of dA. inca >= 0. For example,
            inca = 1 means blocks are stored on the diagonal at dA(i*nb, i*nb),
            inca = 0 means blocks are stored side-by-side    at dA(0,    i*nb).

    @param[in,out]
    dB      COMPLEX_16 array, dimension (LDDB,N)
            The matrix dB.

    @param[in]
    lddb    INTEGER
            The leading dimension of the array db.
            LDDB >= (nblocks - 1)*nb*incb + nb.

    @param[in]
    incb    INTEGER
            The row increment between diagonal blocks of dB. incb >= 0. See inca.
    
    @ingroup magma_zaux2
    ********************************************************************/
extern "C" void 
magmablas_zswapdblk_q(
    magma_int_t n, magma_int_t nb,
    magmaDoubleComplex *dA, magma_int_t ldda, magma_int_t inca,
    magmaDoubleComplex *dB, magma_int_t lddb, magma_int_t incb,
    magma_queue_t queue )
{
    magma_int_t nblocks = n / nb;
    
    magma_int_t info = 0;
    if (n < 0) {
        info = -1;
    } else if (nb < 1 || nb > 1024) {
        info = -2;
    } else if (ldda < (nblocks-1)*nb*inca + nb) {
        info = -4;
    } else if (inca < 0) {
        info = -5;
    } else if (lddb < (nblocks-1)*nb*incb + nb) {
        info = -7;
    } else if (incb < 0) {
        info = -8;
    }

    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }

    if ( nblocks > 0 ) {
        zswapdblk_kernel<<< nblocks, nb, 0, queue >>>
            ( nb, dA, ldda, inca,
                  dB, lddb, incb );
    }
}


/**
    @see magmablas_zswapdblk_q
    @ingroup magma_zaux2
    ********************************************************************/
extern "C" void 
magmablas_zswapdblk(
    magma_int_t n, magma_int_t nb,
    magmaDoubleComplex *dA, magma_int_t ldda, magma_int_t inca,
    magmaDoubleComplex *dB, magma_int_t lddb, magma_int_t incb )
{
    magmablas_zswapdblk_q( n, nb, dA, ldda, inca, dB, lddb, incb, magma_stream );
}
