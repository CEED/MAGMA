#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 0.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       June 2009
*/

#include "hipblas.h"
#include "magma.h"

extern "C" void
magmablas_ztrmm(char side, char uplo, char transa, char diag, 
		int m, int n, double2 alpha, 
		double2 *A, int lda,
		double2 *B, int ldb) {
/*  -- MAGMA (version 0.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       June 2009

    Purpose   
    =======   

    ZTRMM  performs one of the matrix-matrix operations   

       B := alpha*op( A )*B,   or   B := alpha*B*op( A )   

    where  alpha  is a scalar,  B  is an m by n matrix,  A  is a unit, or   
    non-unit,  upper or lower triangular matrix  and  op( A )  is one  of 

       op( A ) = A   or   op( A ) = A'   or   op( A ) = conjg( A' ).   

    Parameters   
    ==========   

    SIDE   - CHARACTER   
             On entry,  SIDE specifies whether  op( A ) multiplies B from 
             the left or right as follows:   
                SIDE = 'L' or 'l'   B := alpha*op( A )*B.   
                SIDE = 'R' or 'r'   B := alpha*B*op( A ).   
             Unchanged on exit.   

    UPLO   - CHARACTER
             On entry, UPLO specifies whether the matrix A is an upper or 
             lower triangular matrix as follows:   
                UPLO = 'U' or 'u'   A is an upper triangular matrix.   
                UPLO = 'L' or 'l'   A is a lower triangular matrix.   
             Unchanged on exit.   

    TRANSA - CHARACTER
             On entry, TRANSA specifies the form of op( A ) to be used in 
             the matrix multiplication as follows:   
                TRANSA = 'N' or 'n'   op( A ) = A.   
                TRANSA = 'T' or 't'   op( A ) = A'.   
                TRANSA = 'C' or 'c'   op( A ) = conjg( A' ).   
             Unchanged on exit.   

    DIAG   - CHARACTER
             On entry, DIAG specifies whether or not A is unit triangular 
             as follows:   
                DIAG = 'U' or 'u'   A is assumed to be unit triangular.   
                DIAG = 'N' or 'n'   A is not assumed to be unit   
                                    triangular.   
             Unchanged on exit.   

    M      - INTEGER
             On entry, M specifies the number of rows of B. M must be at 
             least zero.   
             Unchanged on exit.   

    N      - INTEGER   
             On entry, N specifies the number of columns of B.  N must be 
             at least zero.   
             Unchanged on exit.   

    ALPHA  - DOUBLE COMPLEX  
             On entry,  ALPHA specifies the scalar  alpha. When  alpha is 
             zero then  A is not referenced and  B need not be set before 
             entry.   
             Unchanged on exit.   

    A      - DOUBLE COMPLEX array of DIMENSION ( LDA, k ), where k is m 
             when  SIDE = 'L' or 'l'  and is  n  when  SIDE = 'R' or 'r'. 
             Before entry  with  UPLO = 'U' or 'u',  the  leading  k by k 
             upper triangular part of the array  A must contain the upper 
             triangular matrix  and the strictly lower triangular part of 
             A is not referenced.   
             Before entry  with  UPLO = 'L' or 'l',  the  leading  k by k 
             lower triangular part of the array  A must contain the lower 
             triangular matrix  and the strictly upper triangular part of 
             A is not referenced.   
             Note that when  DIAG = 'U' or 'u',  the diagonal elements of 
             A  are not referenced either,  but are assumed to be  unity. 
             Unchanged on exit.   

    LDA    - INTEGER
             On entry, LDA specifies the first dimension of A as declared 
             in the calling (sub) program.  When  SIDE = 'L' or 'l'  then 
             LDA  must be at least  max( 1, m ),  when  SIDE = 'R' or 'r' 
             then LDA must be at least max( 1, n ).   
             Unchanged on exit.   

    B      - DOUBLE COMPLEX array of DIMENSION ( LDB, n ).   
             Before entry,  the leading  m by n part of the array  B must 
             contain the matrix  B,  and  on exit  is overwritten  by the 
             transformed matrix.   

    LDB    - INTEGER
             On entry, LDB specifies the first dimension of B as declared 
             in  the  calling  (sub)  program.   LDB  must  be  at  least 
             max( 1, m ).   
             Unchanged on exit.   

    Level 3 Blas routine.   

    ===================================================================== */

    int k;
    if (side == 'L' || side == 'l')
       k = m;
    else 
       k = n;

    double2 *a = (double2*)malloc(k*k * sizeof(double2));
    double2 *b = (double2*)malloc(m*n * sizeof(double2));

    hipblasGetMatrix(k, k, sizeof(double2), A, lda, a, k);
    hipblasGetMatrix(m, n, sizeof(double2), B, ldb, b, m);

    ztrmm_(&side, &uplo, &transa, &diag,
           &m, &n, &alpha, a, &k, b, &m);     

    hipblasSetMatrix(m, n, sizeof(double2), b, m, B, ldb);
   
    free(a);
    free(b);
}
