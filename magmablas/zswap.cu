#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal z -> s d c

*/
#include "common_magma.h"

#define BLOCK_SIZE 64

/*********************************************************
 *
 * SWAP BLAS: permute to set of N elements
 *
 ********************************************************/
/*
 *  First version: line per line
 */
typedef struct {
    magmaDoubleComplex *A1;
    magmaDoubleComplex *A2;
    int n, lda1, lda2;
} magmagpu_zswap_params_t;

__global__ void magmagpu_zswap( magmagpu_zswap_params_t params )
{
    unsigned int x = threadIdx.x + __mul24(blockDim.x, blockIdx.x);
    unsigned int offset1 = __mul24( x, params.lda1);
    unsigned int offset2 = __mul24( x, params.lda2);
    if( x < params.n )
    {
        magmaDoubleComplex *A1  = params.A1 + offset1;
        magmaDoubleComplex *A2  = params.A2 + offset2;
        magmaDoubleComplex temp = *A1;
        *A1 = *A2;
        *A2 = temp;
    }
}

extern "C" void 
magmablas_zswap( magma_int_t n, magmaDoubleComplex *dA1T, magma_int_t lda1, 
                 magmaDoubleComplex *dA2T, magma_int_t lda2)
{
    int blocksize = 64;
    dim3 blocks( (n+blocksize-1) / blocksize, 1, 1);
    magmagpu_zswap_params_t params = { dA1T, dA2T, n, lda1, lda2 };
    magmagpu_zswap<<< blocks, blocksize, 0, magma_stream >>>( params );
}

