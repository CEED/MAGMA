#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date
       
       @author Mark Gates

       @precisions normal z -> s d c

*/
#include "common_magma.h"

#define NB 64


/* Vector is divided into ceil(n/nb) blocks.
   Each thread swaps one element, x[tid] <---> y[tid].
*/
__global__ void zswap_kernel(
    int n,
    magmaDoubleComplex *x, int incx,
    magmaDoubleComplex *y, int incy )
{
    magmaDoubleComplex tmp;
    int ind = threadIdx.x + blockDim.x*blockIdx.x;
    if ( ind < n ) {
        x += ind*incx;
        y += ind*incy;
        tmp = *x;
        *x  = *y;
        *y  = tmp;
    }
}


/**
    Purpose:
    =============
    Swap vector x and y; \f$ x <-> y \f$.

    @param[in]
    n       Number of elements in vector x and y. n >= 0.

    @param[in,out]
    dx      COMPLEX_16 array on GPU device.
            The n element vector x of dimension (1 + (n-1)*incx).

    @param[in]
    incx    Stride between consecutive elements of dx. incx != 0.

    @param[in,out]
    dy      COMPLEX_16 array on GPU device.
            The n element vector y of dimension (1 + (n-1)*incy).

    @param[in]
    incy    Stride between consecutive elements of dy. incy != 0.

    @ingroup magma_zblas1
    ********************************************************************/
extern "C" void 
magmablas_zswap_q(
    magma_int_t n,
    magmaDoubleComplex_ptr dx, magma_int_t incx, 
    magmaDoubleComplex_ptr dy, magma_int_t incy,
    magma_queue_t queue )
{
    dim3 blocks( (n+NB-1) / NB );
    zswap_kernel<<< blocks, NB, 0, queue >>>( n, dx, incx, dy, incy );
}


/**
    @see magmablas_zswap_q
    @ingroup magma_zblas1
    ********************************************************************/
extern "C" void 
magmablas_zswap(
    magma_int_t n,
    magmaDoubleComplex_ptr dx, magma_int_t incx, 
    magmaDoubleComplex_ptr dy, magma_int_t incy)
{
    magmablas_zswap_q( n, dx, incx, dy, incy, magma_stream );
}
