#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

       @precisions normal z -> s d c

*/

#include <stdio.h>
#include <hipblas.h>

#define BLOCK_SIZE 64

/*********************************************************
 *
 * SWAP BLAS: permute to set of N elements
 *
 ********************************************************/
/*
 *  First version: line per line
 */
typedef struct {
    hipDoubleComplex *A1;
    hipDoubleComplex *A2;
    int n, lda1, lda2;
} zswap_params_t;

__global__ void myzswap( zswap_params_t params )
{
    unsigned int x = threadIdx.x + __mul24(blockDim.x, blockIdx.x);
    unsigned int offset1 = __mul24( x, params.lda1);
    unsigned int offset2 = __mul24( x, params.lda2);
    if( x < params.n )
    {
        hipDoubleComplex *A1  = params.A1 + offset1;
        hipDoubleComplex *A2  = params.A2 + offset2;
        hipDoubleComplex temp = *A1;
        *A1 = *A2;
        *A2 = temp;
    }
}

extern "C" void 
magmablas_zswap( int n, hipDoubleComplex *dA1T, int lda1, 
                 hipDoubleComplex *dA2T, int lda2)
{
    int blocksize = 64;
    dim3 blocks = ( (params.n+blocksize-1) / blocksize, 1, 1);
    zswap_params_t params = { dA1T, dA2T, n, lda1, lda2 };
    myzswap<<< blocks, blocksize >>>( params );
}

