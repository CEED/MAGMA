#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

       @precisions normal z -> s d c

*/
#include "common_magma.h"

#define BLOCK_SIZE 64

/*********************************************************
 *
 * SWAP BLAS: permute to set of N elements
 *
 ********************************************************/
/*
 *  First version: line per line
 */
typedef struct {
    hipDoubleComplex *A1;
    hipDoubleComplex *A2;
    int n, lda1, lda2;
} magmagpu_zswap_params_t;

__global__ void magmagpu_zswap( magmagpu_zswap_params_t params )
{
    unsigned int x = threadIdx.x + __mul24(blockDim.x, blockIdx.x);
    unsigned int offset1 = __mul24( x, params.lda1);
    unsigned int offset2 = __mul24( x, params.lda2);
    if( x < params.n )
    {
        hipDoubleComplex *A1  = params.A1 + offset1;
        hipDoubleComplex *A2  = params.A2 + offset2;
        hipDoubleComplex temp = *A1;
        *A1 = *A2;
        *A2 = temp;
    }
}

extern "C" void 
magmablas_zswap( int n, hipDoubleComplex *dA1T, int lda1, 
                 hipDoubleComplex *dA2T, int lda2)
{
    int  blocksize = 64;
    dim3 blocks( (n+blocksize-1) / blocksize, 1, 1);
    magmagpu_zswap_params_t params = { dA1T, dA2T, n, lda1, lda2 };
    magmagpu_zswap<<< blocks, blocksize >>>( params );
}

