#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010
*/

#include "hipblas.h"
#include "magma.h"

__device__ void saxpy(double a,double *b, double *c) {
	c[0] += a * b[0];
	c[1] += a * b[1];
	c[2] += a * b[2];
	c[3] += a * b[3];
	c[4] += a * b[4];
	c[5] += a * b[5];
	c[6] += a * b[6];
	c[7] += a * b[7];
	c[8] += a * b[8];
	c[9] += a * b[9];
	c[10] += a * b[10];
	c[11] += a * b[11];
	c[12] += a * b[12];
	c[13] += a * b[13];
	c[14] += a * b[14];
	c[15] += a * b[15];
}


extern "C" __global__ void 
dgemm_kernel_N_T_64_16_4_16_4(double *C, const double *A, const double *B, 
                              int m, int n, int k, 
                              int lda, int ldb, int ldc, 
                              double alpha, double beta) 
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose:
    ========
    This routine computes
       C = alpha* A*B^T  + beta * C

    B is put into shared memory
    Parameters Used:
        blk_M=64 blk_N=16 blk_K=4 nthd_x=16 nthd_y=4

    This code should run for any matrix size.
    ===============================================================  */

	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	const int ibx = blockIdx.x * 64;
	const int iby = blockIdx.y *16;
	

	const int idt = ty * 16 + tx;

	if( iby + tx >=n )
		B+= iby+0;
	else
		B+= iby+tx;
	/*
		Taking care of boundary cases where K<4.
	*/
	if( ty >=k ) 
		B+= __mul24( 0,ldb);
	else
		B+= __mul24( ty,ldb);
		

	if( ibx + idt >= m ) 
		A += ibx + 0 ; 
	else
		A += ibx + idt;


	int s2=lda, s3=2*lda, s4=3*lda ; 

	switch (k){
		case 1: 
			s2=0; s3=0;s4=0 ; 
			break ; 
		case 2:
			s2=lda; s3=0;s4=0 ; 
			break ; 
		case 3:  
			s2=lda; s3=2*lda;s4=0 ; 
			break ;
	}
	
	C += ibx +idt +__mul24( iby,ldc);

	double Ap[4]={A[0], A[s2], A[s3], A[s4]};

	double b=B[0];

	const double *Bend = B + ldb*(k-k%4);

	B+=4*ldb;
	A+=4*lda;

	__shared__ double Bb[4][16];

	double Cb[16] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

	if(k>7)
	do {

		double Ab[4] = {Ap[0], Ap[1], Ap[2], Ap[3]};

		Bb[ty][tx]=b;

		__syncthreads();

		Ap[0] = A[0];
		Ap[1] = A[s2];
		Ap[2] = A[s3];
		Ap[3] = A[s4];

		 b=B[0];


		saxpy(Ab[0], &Bb[0][0], Cb);
		saxpy(Ab[1], &Bb[1][0], Cb);
		saxpy(Ab[2], &Bb[2][0], Cb);
		saxpy(Ab[3], &Bb[3][0], Cb);

		A+=4*lda;
		B += 4*ldb;

		__syncthreads();
	} while (B < Bend);

	if(k>3){

		Bb[ty][tx]=b;
		int k1 = k-k%4;

		if( (k1+ty) >=k)
		    B-=4*ldb;
		else 
		    B-=0*ldb;

		if( (k1+0) >= k ) {s2=0  ;s3=0*lda;s4=0;A-=4*lda;} else
		if( (k1+1) >= k ) {s2=0  ;s3=0*lda;s4=0;A-=0*lda;} else
		if( (k1+2) >= k ) {s2=lda;s3=0*lda;s4=0;A-=0*lda;} else
		if( (k1+3) >= k ) {s2=lda;s3=2*lda;s4=0;A-=0*lda;} 
			
		__syncthreads();


	         b=B[0];

		saxpy(Ap[0], &Bb[0][0], Cb);        Ap[0] = A[0];
		saxpy(Ap[1], &Bb[1][0], Cb);        Ap[1] = A[s2];
		saxpy(Ap[2], &Bb[2][0], Cb);        Ap[2] = A[s3];
		saxpy(Ap[3], &Bb[3][0], Cb);        Ap[3] = A[s4];
	
	}

	k=k%4;

	if ( k!=0){

		__syncthreads();

		Bb[ty][tx]=b;

		__syncthreads();

		for(int i=0;i<k;i++){
			saxpy(Ap[i],&Bb[i][0], Cb);
		}
	}



	if( (iby+16)>=n) { 
		lda = n-iby;
	}
	else{
		lda = 16;
	}

	if( (ibx+idt) >= m )
		lda = 0 ;
	else lda = lda ;


        
	switch(lda){
		case 16:
	
			C[0] =alpha*Cb[0] + beta * C[0];
			C[1*ldc] =alpha*Cb[1] + beta * C[1*ldc];
			C[2*ldc] =alpha*Cb[2] + beta * C[2*ldc];
			C[3*ldc] =alpha*Cb[3] + beta * C[3*ldc];
			C[4*ldc] =alpha*Cb[4] + beta * C[4*ldc];
			C[5*ldc] =alpha*Cb[5] + beta * C[5*ldc];
			C[6*ldc] =alpha*Cb[6] + beta * C[6*ldc];
			C[7*ldc] =alpha*Cb[7] + beta * C[7*ldc];
			C[8*ldc] =alpha*Cb[8] + beta * C[8*ldc];
			C[9*ldc] =alpha*Cb[9] + beta * C[9*ldc];
			C[10*ldc] =alpha*Cb[10] + beta * C[10*ldc];
			C[11*ldc] =alpha*Cb[11] + beta * C[11*ldc];
			C[12*ldc] =alpha*Cb[12] + beta * C[12*ldc];
			C[13*ldc] =alpha*Cb[13] + beta * C[13*ldc];
			C[14*ldc] =alpha*Cb[14] + beta * C[14*ldc];
			C[15*ldc] =alpha*Cb[15] + beta * C[15*ldc];

			break;
		case 15:
			C[0] =alpha*Cb[0] + beta * C[0];
			C[1*ldc] =alpha*Cb[1] + beta * C[1*ldc];
			C[2*ldc] =alpha*Cb[2] + beta * C[2*ldc];
			C[3*ldc] =alpha*Cb[3] + beta * C[3*ldc];
			C[4*ldc] =alpha*Cb[4] + beta * C[4*ldc];
			C[5*ldc] =alpha*Cb[5] + beta * C[5*ldc];
			C[6*ldc] =alpha*Cb[6] + beta * C[6*ldc];
			C[7*ldc] =alpha*Cb[7] + beta * C[7*ldc];
			C[8*ldc] =alpha*Cb[8] + beta * C[8*ldc];
			C[9*ldc] =alpha*Cb[9] + beta * C[9*ldc];
			C[10*ldc] =alpha*Cb[10] + beta * C[10*ldc];
			C[11*ldc] =alpha*Cb[11] + beta * C[11*ldc];
			C[12*ldc] =alpha*Cb[12] + beta * C[12*ldc];
			C[13*ldc] =alpha*Cb[13] + beta * C[13*ldc];
			C[14*ldc] =alpha*Cb[14] + beta * C[14*ldc];
			break;
		case 14:
			C[0] =alpha*Cb[0] + beta * C[0];
			C[1*ldc] =alpha*Cb[1] + beta * C[1*ldc];
			C[2*ldc] =alpha*Cb[2] + beta * C[2*ldc];
			C[3*ldc] =alpha*Cb[3] + beta * C[3*ldc];
			C[4*ldc] =alpha*Cb[4] + beta * C[4*ldc];
			C[5*ldc] =alpha*Cb[5] + beta * C[5*ldc];
			C[6*ldc] =alpha*Cb[6] + beta * C[6*ldc];
			C[7*ldc] =alpha*Cb[7] + beta * C[7*ldc];
			C[8*ldc] =alpha*Cb[8] + beta * C[8*ldc];
			C[9*ldc] =alpha*Cb[9] + beta * C[9*ldc];
			C[10*ldc] =alpha*Cb[10] + beta * C[10*ldc];
			C[11*ldc] =alpha*Cb[11] + beta * C[11*ldc];
			C[12*ldc] =alpha*Cb[12] + beta * C[12*ldc];
			C[13*ldc] =alpha*Cb[13] + beta * C[13*ldc];
			break;
		case 13:
			C[0] =alpha*Cb[0] + beta * C[0];
			C[1*ldc] =alpha*Cb[1] + beta * C[1*ldc];
			C[2*ldc] =alpha*Cb[2] + beta * C[2*ldc];
			C[3*ldc] =alpha*Cb[3] + beta * C[3*ldc];
			C[4*ldc] =alpha*Cb[4] + beta * C[4*ldc];
			C[5*ldc] =alpha*Cb[5] + beta * C[5*ldc];
			C[6*ldc] =alpha*Cb[6] + beta * C[6*ldc];
			C[7*ldc] =alpha*Cb[7] + beta * C[7*ldc];
			C[8*ldc] =alpha*Cb[8] + beta * C[8*ldc];
			C[9*ldc] =alpha*Cb[9] + beta * C[9*ldc];
			C[10*ldc] =alpha*Cb[10] + beta * C[10*ldc];
			C[11*ldc] =alpha*Cb[11] + beta * C[11*ldc];
			C[12*ldc] =alpha*Cb[12] + beta * C[12*ldc];
			break;
		case 12:
			C[0] =alpha*Cb[0] + beta * C[0];
			C[1*ldc] =alpha*Cb[1] + beta * C[1*ldc];
			C[2*ldc] =alpha*Cb[2] + beta * C[2*ldc];
			C[3*ldc] =alpha*Cb[3] + beta * C[3*ldc];
			C[4*ldc] =alpha*Cb[4] + beta * C[4*ldc];
			C[5*ldc] =alpha*Cb[5] + beta * C[5*ldc];
			C[6*ldc] =alpha*Cb[6] + beta * C[6*ldc];
			C[7*ldc] =alpha*Cb[7] + beta * C[7*ldc];
			C[8*ldc] =alpha*Cb[8] + beta * C[8*ldc];
			C[9*ldc] =alpha*Cb[9] + beta * C[9*ldc];
			C[10*ldc] =alpha*Cb[10] + beta * C[10*ldc];
			C[11*ldc] =alpha*Cb[11] + beta * C[11*ldc];
			break;
		case 11:
			C[0] =alpha*Cb[0] + beta * C[0];
			C[1*ldc] =alpha*Cb[1] + beta * C[1*ldc];
			C[2*ldc] =alpha*Cb[2] + beta * C[2*ldc];
			C[3*ldc] =alpha*Cb[3] + beta * C[3*ldc];
			C[4*ldc] =alpha*Cb[4] + beta * C[4*ldc];
			C[5*ldc] =alpha*Cb[5] + beta * C[5*ldc];
			C[6*ldc] =alpha*Cb[6] + beta * C[6*ldc];
			C[7*ldc] =alpha*Cb[7] + beta * C[7*ldc];
			C[8*ldc] =alpha*Cb[8] + beta * C[8*ldc];
			C[9*ldc] =alpha*Cb[9] + beta * C[9*ldc];
			C[10*ldc] =alpha*Cb[10] + beta * C[10*ldc];
			break;
		case 10:
			C[0] =alpha*Cb[0] + beta * C[0];
			C[1*ldc] =alpha*Cb[1] + beta * C[1*ldc];
			C[2*ldc] =alpha*Cb[2] + beta * C[2*ldc];
			C[3*ldc] =alpha*Cb[3] + beta * C[3*ldc];
			C[4*ldc] =alpha*Cb[4] + beta * C[4*ldc];
			C[5*ldc] =alpha*Cb[5] + beta * C[5*ldc];
			C[6*ldc] =alpha*Cb[6] + beta * C[6*ldc];
			C[7*ldc] =alpha*Cb[7] + beta * C[7*ldc];
			C[8*ldc] =alpha*Cb[8] + beta * C[8*ldc];
			C[9*ldc] =alpha*Cb[9] + beta * C[9*ldc];
			break;
		case 9:
			C[0] =alpha*Cb[0] + beta * C[0];
			C[1*ldc] =alpha*Cb[1] + beta * C[1*ldc];
			C[2*ldc] =alpha*Cb[2] + beta * C[2*ldc];
			C[3*ldc] =alpha*Cb[3] + beta * C[3*ldc];
			C[4*ldc] =alpha*Cb[4] + beta * C[4*ldc];
			C[5*ldc] =alpha*Cb[5] + beta * C[5*ldc];
			C[6*ldc] =alpha*Cb[6] + beta * C[6*ldc];
			C[7*ldc] =alpha*Cb[7] + beta * C[7*ldc];
			C[8*ldc] =alpha*Cb[8] + beta * C[8*ldc];
			break;
		case 8:
			C[0] =alpha*Cb[0] + beta * C[0];
			C[1*ldc] =alpha*Cb[1] + beta * C[1*ldc];
			C[2*ldc] =alpha*Cb[2] + beta * C[2*ldc];
			C[3*ldc] =alpha*Cb[3] + beta * C[3*ldc];
			C[4*ldc] =alpha*Cb[4] + beta * C[4*ldc];
			C[5*ldc] =alpha*Cb[5] + beta * C[5*ldc];
			C[6*ldc] =alpha*Cb[6] + beta * C[6*ldc];
			C[7*ldc] =alpha*Cb[7] + beta * C[7*ldc];
			break;
		case 7:
			C[0] =alpha*Cb[0] + beta * C[0];
			C[1*ldc] =alpha*Cb[1] + beta * C[1*ldc];
			C[2*ldc] =alpha*Cb[2] + beta * C[2*ldc];
			C[3*ldc] =alpha*Cb[3] + beta * C[3*ldc];
			C[4*ldc] =alpha*Cb[4] + beta * C[4*ldc];
			C[5*ldc] =alpha*Cb[5] + beta * C[5*ldc];
			C[6*ldc] =alpha*Cb[6] + beta * C[6*ldc];
			break;
		case 6:
			C[0] =alpha*Cb[0] + beta * C[0];
			C[1*ldc] =alpha*Cb[1] + beta * C[1*ldc];
			C[2*ldc] =alpha*Cb[2] + beta * C[2*ldc];
			C[3*ldc] =alpha*Cb[3] + beta * C[3*ldc];
			C[4*ldc] =alpha*Cb[4] + beta * C[4*ldc];
			C[5*ldc] =alpha*Cb[5] + beta * C[5*ldc];
			break;
		case 5:
			C[0] =alpha*Cb[0] + beta * C[0];
			C[1*ldc] =alpha*Cb[1] + beta * C[1*ldc];
			C[2*ldc] =alpha*Cb[2] + beta * C[2*ldc];
			C[3*ldc] =alpha*Cb[3] + beta * C[3*ldc];
			C[4*ldc] =alpha*Cb[4] + beta * C[4*ldc];
			break;
		case 4:
			C[0] =alpha*Cb[0] + beta * C[0];
			C[1*ldc] =alpha*Cb[1] + beta * C[1*ldc];
			C[2*ldc] =alpha*Cb[2] + beta * C[2*ldc];
			C[3*ldc] =alpha*Cb[3] + beta * C[3*ldc];
			break;
		case 3:
			C[0] =alpha*Cb[0] + beta * C[0];
			C[1*ldc] =alpha*Cb[1] + beta * C[1*ldc];
			C[2*ldc] =alpha*Cb[2] + beta * C[2*ldc];
			break;
		case 2:
			C[0] =alpha*Cb[0] + beta * C[0];
			C[1*ldc] =alpha*Cb[1] + beta * C[1*ldc];
			break;
		case 1:
			C[0] =alpha*Cb[0] + beta * C[0];
			break;
		case 0:
			break;
	}

}

extern "C" void
magmablas_dgemm_kernel_N_T_64_16_4_16_4(double *C,
                                        const double *A,
                                        const double *B,
                                        int m, int n, int k, 
                                        int lda, int ldb, int ldc,
                                        double alpha, double beta)
{	
        dim3 threads( 16, 4 );
        dim3 grid(m/64+(m%64!=0),n/16+(n%16!=0));
        dgemm_kernel_N_T_64_16_4_16_4<<< grid, threads >>>(C, A, B, 
                                                           m, n, k, 
                                                           lda, ldb, ldc, 
                                                           alpha, beta);
}

