#include "hip/hip_runtime.h"
/*
  -- MAGMA (version 1.0) --
  Univ. of Tennessee, Knoxville
  Univ. of California, Berkeley
  Univ. of Colorado, Denver
  November 2010

  @precisions mixed zc -> ds

*/
#include "common_magma.h"

extern "C" __global__ void
zcaxpycp_special(hipFloatComplex *R, hipDoubleComplex *X, magma_int_t M, hipDoubleComplex *B,hipDoubleComplex *W )
{
    const magma_int_t ibx = blockIdx.x * 64;
    const magma_int_t idt = threadIdx.x;
    X += ibx+idt;
    R += ibx+idt;
    B += ibx+idt;
    W += ibx+idt;
    X[0] = MAGMA_Z_ADD( X[0], hipComplexFloatToDouble(R[0]) );
    W[0] = B[0];
}

extern "C" __global__ void
zaxpycp_special(hipDoubleComplex *R, hipDoubleComplex *X, magma_int_t M, hipDoubleComplex *B)
{
    const magma_int_t ibx = blockIdx.x * 64;
    const magma_int_t idt = threadIdx.x;
    X += ibx+idt;
    R += ibx+idt;
    B += ibx+idt;
    X[0] = MAGMA_Z_ADD( X[0], R[0] );
    R[0] = B[0];
}

extern "C" __global__ void
zcaxpycp_generic(hipFloatComplex *R, hipDoubleComplex *X, magma_int_t M, hipDoubleComplex *B,hipDoubleComplex *W )
{
    const magma_int_t ibx = blockIdx.x * 64;
    const magma_int_t idt = threadIdx.x;
    if( ( ibx + idt ) < M ) {
        X += ibx+idt;
        R += ibx+idt;
        B += ibx+idt;
        W += ibx+idt;
    }
    else{
        X +=(M-1);
        R +=(M-1);
        B +=(M-1);
        W +=(M-1);
    }
    X[0] = MAGMA_Z_ADD( X[0], hipComplexFloatToDouble( R[0] ) );
    W[0] = B[0];
}

extern "C" __global__ void
zaxpycp_generic(hipDoubleComplex *R, hipDoubleComplex *X, magma_int_t M, hipDoubleComplex *B)
{
    const magma_int_t ibx = blockIdx.x * 64;
    const magma_int_t idt = threadIdx.x;
    if( ( ibx + idt ) < M ) {
        X += ibx+idt;
        R += ibx+idt;
        B += ibx+idt;
    }
    else{
        X +=(M-1);
        R +=(M-1);
        B +=(M-1);
    }
    X[0] = MAGMA_Z_ADD( X[0], R[0] );
    R[0] = B[0];
}


extern "C" void
magmablas_zcaxpycp(hipFloatComplex *R, hipDoubleComplex *X, magma_int_t M, hipDoubleComplex *B, hipDoubleComplex *W)
{
    dim3 threads( 64, 1 );
    dim3 grid(M/64+(M%64!=0),1);
    if( M %64 == 0 ) {
        zcaxpycp_special <<< grid, threads >>> ( R, X, M, B, W) ;
    }
    else{
        zcaxpycp_generic <<< grid, threads >>> ( R, X, M, B, W) ;
    }
}

extern "C" void
magmablas_zaxpycp(hipDoubleComplex *R, hipDoubleComplex *X, magma_int_t M, hipDoubleComplex *B)
{
    dim3 threads( 64, 1 );
    dim3 grid(M/64+(M%64!=0),1);
    if( M %64 == 0 ) {
        zaxpycp_special <<< grid, threads >>> ( R, X, M, B) ;
    }
    else{
        zaxpycp_generic <<< grid, threads >>> ( R, X, M, B) ;
    }
}
