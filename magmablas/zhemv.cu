#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date
       
       zsymv.cu is nearly identical to zhemv.cu, just change names and drop MAGMA_Z_CNJG.
       
       zhemv_kernel_U (upper) in zhemv_upper.cu is very similar to
       zhemv_kernel_L (lower) in zhemv.cu; diff the two files to compare.
       
       @precisions normal z -> s d c
       
       @author Mark Gates
*/
#include "common_magma.h"
#include "commonblas_z.h"

#define PRECISION_z

#define NB_X         64
#define NB_Y          4
#define bank_shift   33
#define quarter_NB_X 16
#define half_NB_X    32


/*******************************************************************************
    Lower case, compute block multiply, work = A*x, for any size n:
    
           [ (A11*x1)   (A21^H*x2)          (A31^H*x3)                 ]   [ A11  A21^H  A31^H ]   [ x1 ]
    work = [   ---      (A21*x1 + A22*x2)   (A32^H*x3)                 ] = [ A21  A22    A32^H ] * [ x2 ]
           [   ---        ---               (A31*x1 + A32*x2 + A33*x3) ]   [ A31  A32    A33   ]   [ x3 ]
    
    Uses a 64x4 thread block.
    For     diagonal tiles, covers a 64x64 tile using three 32x32 tiles (plus one gets transposed).
    For off-diagonal tiles, covers a 64x64 tile using four  64x16 tiles.
    In both cases, each thread multiplies 4 elements.
    
    For rows past the bottom of the matrix, the A pointer is adjusted to be the
    last valid row of A, which multiple threads will read.
    Extra rows are ignored when saving results to work.
    Columns past the right edge are explicitly ignored when loading.
    x values past the bottom are set to zero, thus, extra columns are zeroed
    when multiplying.
    
    Previously:
           [ (A11*x1)       ---                                          ]
    work = [ (A21^H*x2)   (A21*x1 + A22*x2)     ---                      ]
           [ (A31^H*x3)   (A32^H*x3)          (A31*x1 + A32*x2 + A33*x3) ]
    which doesn't work as well because that has dimension blocks*NB by blocks,
    where blocks*NB >= n, and it can be that blocks*NB > lda, so it won't fit in
    lda*blocks space. This is why it used to need lwork = lda*(blocks + 1).
    ********************************************************************/
__global__ void
zhemv_kernel_L(
    int n,
    magmaDoubleComplex const * __restrict__ A, int lda,
    magmaDoubleComplex const * __restrict__ x, int incx,
    magmaDoubleComplex       * __restrict__ work)
{
#if defined(PRECISION_s) || defined(PRECISION_d) || defined(PRECISION_c) || (__CUDA_ARCH__ >= 200)

    // treats sA as 16x64 block
    #define sA16(i_, j_) (sA[(i_)][(j_)])  // i.e., sA[ (i_)*(NB_X+3) + (j_) ]
    
    // treats sA as 32x32 block
    #define sA32(i_, j_) (sA[0][(i_) + bank_shift*(j_)])
    
    // 64x4 thread block
    const int tx  = threadIdx.x;
    const int ty  = threadIdx.y;
    const int blk = blockIdx.x;
    const int blk_ind = NB_X * blk;
    const int td  = NB_X * ty + tx;

    // 32x8 thread block
    const int tx2 = td % half_NB_X;
    const int ty2 = td / half_NB_X;

    // If this blk has fewer than NB_X rows, partial is the number of valid rows,
    // so tx = 0, ..., partial-1 are valid rows, and tx >= partial are invalid.
    // Else, partial == 0.
    const int partial = (blk == gridDim.x - 1 ? (n % NB_X) : 0);
    
    magmaDoubleComplex psum, psum_t;
    magmaDoubleComplex total = MAGMA_Z_ZERO;

    // sA is used as a 32x32 block, sA32(i,j),
    // and as a 16x64 block, sA16(i,j), in different parts of the code.
    // sA must be at least half_NB_X*bank_shift = 32x33 = 1056;
    // quarter_NB_X*(NB_X + 2) = 16*(64 + 2) = 1056
    __shared__ magmaDoubleComplex sA [quarter_NB_X][NB_X + 3]; /* Why +3? seems it only needs +2. Does +3 reduce bank conflicts? */
    __shared__ magmaDoubleComplex sx_blk[NB_X];  // for x[ blk ]
    __shared__ magmaDoubleComplex sx_jj [NB_X];  // for x[ jj ], which cycles over all blocks left of diag

    magmaDoubleComplex rA[4];
    magmaDoubleComplex psums_t[4];

    // --------------------
    // load 64x1 block x(blk_ind + 0:63) into sx_blk
    x += (blk_ind + tx)*incx;  // x is x(blk_ind + tx)
    if ( ty == 0 ) {
        if ( partial == 0 || tx < partial ) {
            sx_blk[tx] = x[0];
        }
        else {
            sx_blk[tx] = MAGMA_Z_ZERO;
        }
    }

    // --------------------
    // move to block row
    work += blk*lda;     // work is work(0, blk)
    
    A += blk_ind;        // A is A(blk_ind, 0)
    A += ty2*lda + tx2;  // A is A(blk_ind + tx2, ty2)
    
    // move to 32x32 diag block
    A += blk_ind*lda;    // A is A(blk_ind + tx2, blk_ind + ty2)

    // load 32x32 diag block A(blk_ind + 0:31, blk_ind + 0:31) into sA,
    // as four 32x8 sections one after another:
    // columns 0:7, then 8:15, then 16:23, then 24:31
    if ( partial ) {
        if ( tx2 >= partial ) {
            A = A - tx2 + (partial - 1);  // A is A(blk_ind + partial-1, blk_ind + ty2), the bottom-most valid row
        }
        #pragma unroll
        for (int j=0; j < half_NB_X; j += 8) {
            if ( ty2+j < partial ) {
                sA32(tx2, ty2 + j) = A[j*lda];
            }
            else {
                sA32(tx2, ty2 + j) = MAGMA_Z_ZERO;
            }
        }
        if ( tx2 >= partial ) {
            A = A + tx2 - (partial - 1);  // A is A(blk_ind + tx2, blk_ind + ty2)
        }
    }
    else {
        #pragma unroll
        for (int j=0; j < half_NB_X; j += 8) {
            sA32(tx2, ty2 + j) = A[j*lda];
        }
    }
    __syncthreads();

    // symmetrize 32x32 diag block, copying lower to upper triangle,
    // as four 32x8 sections in parallel:
    // columns 0,4,8,12,16,20,24,28; then 1,5,...,29; then 2,6,...,30, then 3,7,...,31
    #pragma unroll
    for (int j=ty2*4; j < ty2*4 + 4; j++) {
        if ( j < tx2 ) {
            sA32(j, tx2) = MAGMA_Z_CNJG( sA32(tx2, j) );
        }
    }
    __syncthreads();

    // multiply 32x32 diag block * x
    // each thread does partial row sA(tx2, ty2*4 : ty2*4 + 3)
    psum = MAGMA_Z_ZERO;
    #pragma unroll
    for (int j=0; j < 4; j++) {
        psum += sA32(tx2, ty2*4 + j) * sx_blk[ty2*4 + j];
    }
    __syncthreads();

    // store partial row sums
    sA32(ty2, tx2) = psum;
    __syncthreads();

    // sum up partial row sums, so thread (tx2,0) has total for row (blk_ind + tx2)
    if ( ty2 == 0 ) {
        total = sA32(0, tx2) + sA32(1, tx2)
              + sA32(2, tx2) + sA32(3, tx2)
              + sA32(4, tx2) + sA32(5, tx2)
              + sA32(6, tx2) + sA32(7, tx2);
    }
    __syncthreads();

    // --------------------
    // move to next 32x32 diag block, then repeat steps from first diag block
    A += half_NB_X + half_NB_X*lda;  // A is A(blk_ind + NB/2 + tx2, blk_ind + NB/2 + ty2)

    // load 32x32 diag block A[block + 0:31, block + 0:31] into sA
    if ( partial ) {
        if ( tx2 + half_NB_X >= partial ) {
            A = A - (tx2 + half_NB_X) + (partial - 1);
        }
        #pragma unroll
        for (int j=0; j < half_NB_X; j += 8) {
            if ( ty2+j + half_NB_X < partial ) {
                sA32(tx2, ty2 + j) = A[j*lda];
            }
            else {
                sA32(tx2, ty2 + j) = MAGMA_Z_ZERO;
            }
        }
        if ( tx2 + half_NB_X >= partial ) {
            A = A + (tx2 + half_NB_X) - (partial - 1);
        }
    }
    else {
        #pragma unroll
        for (int j=0; j < half_NB_X; j += 8) {
            sA32(tx2, ty2 + j) = A[j*lda];
        }
    }
    __syncthreads();

    // symmetrize 32x32 diag block, copying lower to upper triangle
    #pragma unroll
    for (int j=ty2*4; j < ty2*4 + 4; j++) {
        if ( j < tx2 ) {
            sA32(j, tx2) = MAGMA_Z_CNJG( sA32(tx2, j) );
        }
    }
    __syncthreads();

    // multiply 32x32 diag block * x
    psum = MAGMA_Z_ZERO;
    #pragma unroll
    for (int j=0; j < 4; j++) {
        psum += sA32(tx2, ty2*4 + j) * sx_blk[half_NB_X + ty2*4 + j];
    }
    __syncthreads();
    
    // store partial row sums
    sA32(ty2, tx2) = psum;
    __syncthreads();

    // sum up partial row sums, so thread (tx2,1) has total for row (blk_ind + NB/2 + tx2)
    if ( ty2 == 1 ) {
        total = sA32(0, tx2) + sA32(1, tx2)
              + sA32(2, tx2) + sA32(3, tx2)
              + sA32(4, tx2) + sA32(5, tx2)
              + sA32(6, tx2) + sA32(7, tx2);
    }
    __syncthreads();

    // --------------------
    // move to off-diag 32x32 block
    A -= half_NB_X*lda;  // A is A(blk_ind + NB/2 + tx2, blk_ind + ty2)

    // load 32x32 block of A into sA,
    // as four 32x8 sections one after another:
    // columns 0:7, then 8:15, then 16:23, then 24:31
    if ( partial ) {
        if ( tx2 + half_NB_X >= partial ) {
            A = A - (tx2 + half_NB_X) + (partial - 1);
        }
        #pragma unroll
        for (int j=0; j < half_NB_X; j += 8) {
            if ( ty2+j < partial ) {
                sA32(tx2, ty2 + j) = A[j*lda];
            }
            else {
                sA32(tx2, ty2 + j) = MAGMA_Z_ZERO;
            }
        }
        if ( tx2 + half_NB_X >= partial ) {
            A = A + (tx2 + half_NB_X) - (partial - 1);
        }
    }
    else {
        #pragma unroll
        for (int j=0; j < half_NB_X; j += 8) {
            sA32(tx2, ty2 + j) = A[j*lda];
        }
    }
    __syncthreads();

    // multiply 32x32 block (below diag)
    psum = MAGMA_Z_ZERO;
    #pragma unroll
    for (int j=0; j < 4; j++) {
        psum += sA32(tx2, ty2 + j*8) * sx_blk[j*8 + ty2];
    }
    //__syncthreads();  // no sync needed here

    // multiply transposed 32x32 block (above diag)
    psum_t = MAGMA_Z_ZERO;
    #pragma unroll
    for (int j=0; j < 4; j++) {
        psum_t += MAGMA_Z_CNJG( sA32(ty2*4 + j, tx2) ) * sx_blk[half_NB_X + ty2*4 + j];
    }
    __syncthreads();

    // store partial sums for non-transposed 32x32 block
    sA32(ty2, tx2) = psum;
    __syncthreads();
    
    // sum up partial row sums, so thread (tx2,1) has total for row (blk_ind + NB/2 + tx2)
    if ( ty2 == 1 ) {
        total = total
              + sA32(0, tx2) + sA32(1, tx2)
              + sA32(2, tx2) + sA32(3, tx2)
              + sA32(4, tx2) + sA32(5, tx2)
              + sA32(6, tx2) + sA32(7, tx2);
    }
    __syncthreads();

    // store partial sums for transposed 32x32 block
    sA32(ty2, tx2) = psum_t;
    __syncthreads();
    
    // sum up partial row sums, so thread (tx2,0) has total for row (blk_ind + tx2)
    if ( ty2 == 0 ) {
        total = total
              + sA32(0, tx2) + sA32(1, tx2)
              + sA32(2, tx2) + sA32(3, tx2)
              + sA32(4, tx2) + sA32(5, tx2)
              + sA32(6, tx2) + sA32(7, tx2);
    }
    __syncthreads();
    
    // --------------------
    // move to leftmost 64x64 block in block row, and
    // switch thread offset from (tx2,ty2) 32x8 block to (tx,ty) 64x4 block
    A -= half_NB_X;      // A is A(blk_ind + tx2, blk_ind + ty2)
    A -= blk_ind*lda;    // A is A(blk_ind + tx2,           ty2)
    A -= ty2*lda + tx2;  // A is A(blk_ind, 0)
    A += 4*ty*lda + tx;  // A is A(blk_ind + tx, 4*ty)
    
    if ( partial && tx >= partial ) {
        A = A - tx + (partial - 1);  // A is A(blk_ind + partial-1, 4*ty), the bottom-most valid row
    }
    
    x -= blk_ind*incx;  // x is x(tx)

    // 16x16 thread block
    const int tx4 = td % quarter_NB_X;
    const int ty4 = td / quarter_NB_X;

    // cycle over blocks jj left of diagonal, in block row blk
    for (int jj=0; jj < blk; ++jj) {
        // load 64x1 block x(jj_ind + 0:63) into sx_jj
        // since this block is left of diagonal, x must have all NB rows
        if ( ty == 0 ) {
            sx_jj[tx] = x[jj*NB_X*incx];
        }
        __syncthreads();

        for (int k=0; k < 4; k++) {
            // load 64x16 block of A into rA, 4 elements per thread,
            // as four 64x4 sections in parallel:
            // columns 0,4,8,12; then 1,5,9,13; then 2,6,10,14; then 3,7,11,15
            // since this block is left of diagonal, it has all NB columns,
            // and block of x must have all NB rows.
            #pragma unroll
            for (int j=0; j < 4; j++) {
                rA[j] = A[j*lda];
            }

            // 1) multiply 64x16 block A_{blk,jj} * x_jj
            //    each thread does partial row rA(tx + 16*k, ty*4 + 16*k : ty*4 + 3 + 16*k)
            // 2) multiply transposed 16x64 block A_{blk,jj}^H * x_blk,
            //    storing each product Aji*xi to sA(j,i)
            #pragma unroll
            for (int j=0; j < 4; j++) {
                total += rA[j] * sx_jj[quarter_NB_X*k + ty*4 + j];  // y_blk = A_{blk,jj}   * x_jj
                sA16(ty*4 + j, tx) = MAGMA_Z_CNJG( rA[j] ) * sx_blk[tx];  // y_jj  = A_{blk,jj}^H * x_blk
            }
            __syncthreads();

            // do partial row sums for transposed 16x64 result
            // use 16x16 thread grid (tx4, ty4) instead of 64x4 (tx, ty)
            // sum sixteen 16x4 sections in parallel:
            // columns 0,4,8,...,60; then 1,5,...,61; then 2,6,...,62; then 3,7,...,63
            psum_t = MAGMA_Z_ZERO;
            #pragma unroll
            for (int j=0; j < 4; j++) {
                psum_t += sA16(tx4, ty4*4 + j);
            }
            __syncthreads();

            // store partial row sums of transposed result, y_jj (locally)
            psums_t[k] = psum_t;

            // move right to next 64x16 block
            A += lda * quarter_NB_X;  // A is A(blk_ind + tx#, jj*NB_x + (k+1)*NB_X/4 + 4*ty), # tx or partial
        }
        // already at next 64x64 block
        // A is A(blk_ind + tx#, (jj+1)*NB_x + 4*ty), # tx or partial

        // store partial row sums of transposed result, y_jj
        #pragma unroll
        for (int k=0; k < 4; k++) {
            sA16(tx4, ty4 + quarter_NB_X*k) = psums_t[k];
        }
        __syncthreads();
        
        // sum up partial row sums of transposed result, y_jj, and store final total to workspace
        // thread (tx4,ty4) where ty4 < 4 sums row tx4 + ty4*16
        // since this is the transposed block above the diagonal, it must have all NB rows
        if ( ty4 < 4 ) {
            int ty4_nb4 = ty4*quarter_NB_X;
            psum_t = sA16(tx4,  0 + ty4_nb4) + sA16(tx4,  1 + ty4_nb4)
                   + sA16(tx4,  2 + ty4_nb4) + sA16(tx4,  3 + ty4_nb4)
                   + sA16(tx4,  4 + ty4_nb4) + sA16(tx4,  5 + ty4_nb4)
                   + sA16(tx4,  6 + ty4_nb4) + sA16(tx4,  7 + ty4_nb4)
                   + sA16(tx4,  8 + ty4_nb4) + sA16(tx4,  9 + ty4_nb4)
                   + sA16(tx4, 10 + ty4_nb4) + sA16(tx4, 11 + ty4_nb4)
                   + sA16(tx4, 12 + ty4_nb4) + sA16(tx4, 13 + ty4_nb4)
                   + sA16(tx4, 14 + ty4_nb4) + sA16(tx4, 15 + ty4_nb4);
            work[jj*NB_X + tx4 + ty4_nb4] = psum_t;  // store at work( jj*NB_X + tx4 + ty4*16, blk )
        }
        __syncthreads();
    }

    // store row sums
    sA16(ty, tx) = total;
    __syncthreads();
    
    // sum up final total, y_blk, for row tx
    if ( ty == 0 && (partial == 0 || tx < partial) ) {
        total = sA16(0, tx)
              + sA16(1, tx)
              + sA16(2, tx)
              + sA16(3, tx);
        work[blk*NB_X + tx] = total;  // store at work( blk*NB_X + tx, blk )
    }
#endif  /* PRECISION_[sdc] || (__CUDA_ARCH__ >= 200) */
}
// end zhemv_kernel_L


/**************************************************************
    Lower case, sum up final results
    Each block sums one block row; each thread sums one row.
    
    On input (for 3 blocks):
           [ (A11*x1)   (A21^H*x2)          (A31^H*x3)                 ]
    work = [   ---      (A21*x1 + A22*x2)   (A32^H*x3)                 ]
           [   ---        ---               (A31*x1 + A32*x2 + A33*x3) ]
    
    On output:
              [ (A11*x1) + (A21^H*x2) + (A31^H*x3) ]
    y = alpha*[ (A21*x1 + A22*x2)     + (A32^H*x3) ] + beta*y
              [ (A21*x1 + A22*x2 + A33*x3)         ]
    ********************************************************************/
__global__ void
zhemv_kernel_L_sum(
    int n,
    magmaDoubleComplex alpha,
    int lda,
    magmaDoubleComplex beta,
    magmaDoubleComplex       * __restrict__ y, int incy,
    magmaDoubleComplex const * __restrict__ work )
{
    int tx  = threadIdx.x;
    int blk = blockIdx.x;
    int blk_ind = blk * NB_X;
    int ind     = blk_ind + tx;
    int blocks  = gridDim.x;
    
    // Don't write outside [0, ..., n)
    if ( ind < n ) {
        work += ind + blk*lda;
        magmaDoubleComplex Ax = MAGMA_Z_ZERO;
        for (int j = blk; j < blocks; ++j) {
            Ax += work[0];
            work += lda;
        }
        y[ind * incy] = beta*y[ind * incy] + alpha*Ax;
    }
}


/**
    Purpose
    -------
    magmablas_zhemv_work performs the matrix-vector operation:

        y := alpha*A*x + beta*y,

    where alpha and beta are scalars, x and y are n element vectors and
    A is an n by n Hermitian matrix.

    Arguments
    ----------
    @param[in]
    uplo    magma_uplo_t.
            On entry, UPLO specifies whether the upper or lower
            triangular part of the array A is to be referenced as
            follows:
      -     = MagmaUpper:  Only the upper triangular part of A is to be referenced.
      -     = MagmaLower:  Only the lower triangular part of A is to be referenced.

    @param[in]
    n       INTEGER.
            On entry, N specifies the order of the matrix A.
            N must be at least zero.

    @param[in]
    alpha   COMPLEX_16.
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    dA      COMPLEX_16 array of DIMENSION ( LDDA, n ).
            Before entry with UPLO = MagmaUpper, the leading n by n
            upper triangular part of the array A must contain the upper
            triangular part of the Hermitian matrix and the strictly
            lower triangular part of A is not referenced.
            Before entry with UPLO = MagmaLower, the leading n by n
            lower triangular part of the array A must contain the lower
            triangular part of the Hermitian matrix and the strictly
            upper triangular part of A is not referenced.
            Note that the imaginary parts of the diagonal elements need
            not be set and are assumed to be zero.

    @param[in]
    ldda    INTEGER.
            On entry, LDDA specifies the first dimension of A as declared
            in the calling (sub) program. LDDA must be at least
            max( 1, n ).
            It is recommended that ldda is multiple of 16. Otherwise
            performance would be deteriorated as the memory accesses
            would not be fully coalescent.

    @param[in]
    dx      COMPLEX_16 array of dimension at least
            ( 1 + ( n - 1 )*abs( INCX ) ).
            Before entry, the incremented array X must contain the n
            element vector x.

    @param[in]
    incx    INTEGER.
            On entry, INCX specifies the increment for the elements of
            X. INCX must not be zero.

    @param[in]
    beta    COMPLEX_16.
            On entry, BETA specifies the scalar beta. When BETA is
            supplied as zero then Y need not be set on input.

    @param[in,out]
    dy      COMPLEX_16 array of dimension at least
            ( 1 + ( n - 1 )*abs( INCY ) ).
            Before entry, the incremented array Y must contain the n
            element vector y. On exit, Y is overwritten by the updated
            vector y.

    @param[in]
    incy    INTEGER.
            On entry, INCY specifies the increment for the elements of
            Y. INCY must not be zero.

    @param[in]
    dwork   (workspace) COMPLEX_16 array on the GPU, dimension (MAX(1, LWORK)),

    @param[in]
    lwork   INTEGER.
            The dimension of the array DWORK. LWORK >= LDDA * ceil( N / NB_X ),
            where NB_X = 64.
    
    @param[in]
    queue   magma_queue_t.
            Queue to execute in.

    MAGMA implements zhemv through two steps:
    1)  perform the multiplication in each thread block and put the
        intermediate value in dwork.
    2)  sum the intermediate values and store the final result in y.
    
    magamblas_zhemv_work requires users to provide a workspace, while
    magmablas_zhemv is a wrapper routine allocating the workspace inside the
    routine and provides the same interface as cublas.
    
    If users need to call zhemv frequently, we suggest using
    magmablas_zhemv_work instead of magmablas_zhemv. As the overhead to
    allocate and free in device memory in magmablas_zhemv would hurt performance.
    Our tests show that this penalty is about 10 Gflop/s when the matrix
    size is around 10000.

    @ingroup magma_zblas2
    ********************************************************************/
extern "C"
magma_int_t
magmablas_zhemv_work(
    magma_uplo_t uplo, magma_int_t n,
    magmaDoubleComplex alpha,
    magmaDoubleComplex_const_ptr dA, magma_int_t ldda,
    magmaDoubleComplex_const_ptr dx, magma_int_t incx,
    magmaDoubleComplex beta,
    magmaDoubleComplex_ptr dy, magma_int_t incy,
    magmaDoubleComplex_ptr dwork, magma_int_t lwork,
    magma_queue_t queue )
{
#if defined(PRECISION_z)
    // z precision requires CUDA ARCH 2.x; call CUBLAS version instead.
    magma_int_t arch = magma_getdevice_arch();
    if ( arch < 200 ) {
        magma_zhemv( uplo, n, alpha, dA, ldda, dx, incx, beta, dy, incy );
        return MAGMA_SUCCESS;
    }
#endif

    // --------------------
    // [sdc] precisions, or z precision with CUDA ARCH 2.x
    int upper = (uplo == MagmaUpper);

    magma_int_t blocks = magma_ceildiv( n, NB_X );
    magma_int_t lwmin  = ldda*blocks;

    /*
     * Test the input parameters.
     */
    magma_int_t info = 0;
    if ((! upper) && (uplo != MagmaLower)) {
        info = -1;
    } else if ( n < 0 ) {
        info = -2;
    } else if ( ldda < max(1, n) ) {
        info = -5;
    } else if ( incx == 0 ) {
        info = -7;
    } else if ( incy == 0 ) {
        info = -10;
    } else if ( lwork < lwmin ) {
        info = -12;
    }
    
    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return info;
    }

    /*
     * Quick return if possible.
     */
    if ( (n == 0) || ( MAGMA_Z_EQUAL(alpha, MAGMA_Z_ZERO) && MAGMA_Z_EQUAL(beta, MAGMA_Z_ONE) ) )
        return info;

    dim3 grid( blocks, 1, 1 );
    dim3 threads( NB_X, NB_Y, 1 );
    dim3 threads_sum( NB_X, 1, 1 );

    if ( upper ) {
        zhemv_kernel_U<<< grid, threads, 0, queue->hip_stream() >>>
            (n, dA, ldda, dx, incx, dwork);
        
        zhemv_kernel_U_sum<<< grid, threads_sum, 0, queue->hip_stream() >>>
            (n, alpha, ldda, beta, dy, incy, dwork);
    }
    else {
        zhemv_kernel_L<<< grid, threads, 0, queue->hip_stream() >>>
            (n, dA, ldda, dx, incx, dwork);
        
        zhemv_kernel_L_sum<<< grid, threads_sum, 0, queue->hip_stream() >>>
            (n, alpha, ldda, beta, dy, incy, dwork);
    }
    return info;
}
// end magmablas_zhemv_work


/**
    Purpose
    -------
    magmablas_zhemv performs the matrix-vector operation:

        y := alpha*A*x + beta*y,

    where alpha and beta are scalars, x and y are n element vectors and
    A is an n by n Hermitian matrix.

    Arguments
    ----------
    @param[in]
    uplo    magma_uplo_t.
            On entry, UPLO specifies whether the upper or lower
            triangular part of the array A is to be referenced as
            follows:
      -     = MagmaUpper:  Only the upper triangular part of A is to be referenced.
      -     = MagmaLower:  Only the lower triangular part of A is to be referenced.

    @param[in]
    n       INTEGER.
            On entry, N specifies the order of the matrix A.
            N must be at least zero.

    @param[in]
    alpha   COMPLEX_16.
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    dA      COMPLEX_16 array of DIMENSION ( LDDA, n ).
            Before entry with UPLO = MagmaUpper, the leading n by n
            upper triangular part of the array A must contain the upper
            triangular part of the Hermitian matrix and the strictly
            lower triangular part of A is not referenced.
            Before entry with UPLO = MagmaLower, the leading n by n
            lower triangular part of the array A must contain the lower
            triangular part of the Hermitian matrix and the strictly
            upper triangular part of A is not referenced.
            Note that the imaginary parts of the diagonal elements need
            not be set and are assumed to be zero.

    @param[in]
    ldda    INTEGER.
            On entry, LDDA specifies the first dimension of A as declared
            in the calling (sub) program. LDDA must be at least
            max( 1, n ).
            It is recommended that ldda is multiple of 16. Otherwise
            performance would be deteriorated as the memory accesses
            would not be fully coalescent.

    @param[in]
    dx      COMPLEX_16 array of dimension at least
            ( 1 + ( n - 1 )*abs( INCX ) ).
            Before entry, the incremented array X must contain the n
            element vector x.

    @param[in]
    incx    INTEGER.
            On entry, INCX specifies the increment for the elements of
            X. INCX must not be zero.

    @param[in]
    beta    COMPLEX_16.
            On entry, BETA specifies the scalar beta. When BETA is
            supplied as zero then Y need not be set on input.

    @param[in,out]
    dy      COMPLEX_16 array of dimension at least
            ( 1 + ( n - 1 )*abs( INCY ) ).
            Before entry, the incremented array Y must contain the n
            element vector y. On exit, Y is overwritten by the updated
            vector y.

    @param[in]
    incy    INTEGER.
            On entry, INCY specifies the increment for the elements of
            Y. INCY must not be zero.

    @ingroup magma_zblas2
    ********************************************************************/
extern "C"
magma_int_t
magmablas_zhemv(
    magma_uplo_t uplo, magma_int_t n,
    magmaDoubleComplex alpha,
    magmaDoubleComplex_const_ptr dA, magma_int_t ldda,
    magmaDoubleComplex_const_ptr dx, magma_int_t incx,
    magmaDoubleComplex beta,
    magmaDoubleComplex_ptr dy, magma_int_t incy)
{
#if defined(PRECISION_z)
    // z precision requires CUDA ARCH 2.x; call CUBLAS version instead.
    magma_int_t arch = magma_getdevice_arch();
    if ( arch < 200 ) {
        magma_zhemv( uplo, n, alpha, dA, ldda, dx, incx, beta, dy, incy );
        return MAGMA_SUCCESS;
    }
#endif

    // --------------------
    // [sdc] precisions, or z precision with CUDA ARCH 2.x
    int upper = (uplo == MagmaUpper);

    /*
     * Test the input parameters.
     */
    magma_int_t info = 0;
    if ((! upper) && (uplo != MagmaLower)) {
        info = -1;
    } else if ( n < 0 ) {
        info = -2;
    } else if ( ldda < max(1, n) ) {
        info = -5;
    } else if ( incx == 0 ) {
        info = -7;
    } else if ( incy == 0 ) {
        info = -10;
    }
    
    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return info;
    }

    /*
     * Quick return if possible.
     */
    if ( (n == 0) || ( MAGMA_Z_EQUAL(alpha, MAGMA_Z_ZERO) && MAGMA_Z_EQUAL(beta, MAGMA_Z_ONE) ) )
        return info;

    magmaDoubleComplex_ptr dwork;
    magma_int_t blocks = magma_ceildiv( n, NB_X );
    magma_int_t lwork  = ldda*blocks;

    magma_zmalloc( &dwork, lwork );
    if ( dwork == NULL ) {
        info = MAGMA_ERR_DEVICE_ALLOC;
        magma_xerbla( __func__, -(info) );
        return info;
    }
    
    magmablas_zhemv_work( uplo, n, alpha, dA, ldda, dx, incx, beta, dy, incy,
                          dwork, lwork, magmablasGetQueue() );
    
    magma_free( dwork );
    
    return info;
}
// end magmablas_zhemv
