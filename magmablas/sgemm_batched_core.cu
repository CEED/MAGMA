#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal s

       @author Jakub Kurzak
       @author Stan Tomov
       @author Mark Gates
       @author Azzam Haidar
       @author Ahmad Abdelfattah
       
*/

#include "common_magma.h"
#define PRECISION_s

#include "gemm_template_kernel_batched.cuh"
#include "gemm_config/sgemm_param_nn.h"
#include "gemm_config/sgemm_param_nt.h"
#include "gemm_config/sgemm_param_tn.h"
#include "gemm_config/sgemm_param_tt.h"

#define version(s,v) s ## _V_ ## v
/**
    Purpose
    -------
    SGEMM performs one of the matrix-matrix operations
    
        C = alpha*op( A )*op( B ) + beta*C,
    
    where op( X ) is one of
    
        op( X ) = X   or   op( X ) = X**T   or   op( X ) = X**H,
    
    alpha and beta are scalars, and A, B and C are matrices, with
    op( A ) an m by k matrix, op( B ) a k by n matrix and C an m by n matrix.
    
    Parameters
    ----------
    @param[in]
    transA  CHARACTER*1.
            On entry, transA specifies the form of op( A ) to be used in
            the matrix multiplication as follows:
      -     = 'N':  op( A ) = A.
      -     = 'T':  op( A ) = A**T.
      -     = 'C':  op( A ) = A**H.
    
    @param[in]
    transB  CHARACTER*1.
            On entry, transB specifies the form of op( B ) to be used in
            the matrix multiplication as follows:
      -     = 'N':  op( B ) = B.
      -     = 'T':  op( B ) = B**T.
      -     = 'C':  op( B ) = B**H.
    
    @param[in]
    m       INTEGER.
            On entry,  M  specifies  the number  of rows  of the  matrix
            op( dA )  and of the  matrix dC.  M  must  be at least  zero.
    
    @param[in]
    n       INTEGER.
            On entry,  N  specifies the number  of columns of the matrix
            op( dB ) and the number of columns of the matrix dC. N must be
            at least zero.
    
    @param[in]
    k       INTEGER.
            On entry,  K  specifies  the number of columns of the matrix
            op( dA ) and the number of rows of the matrix op( dB ). K must
            be at least  zero.
    
    @param[in]
    alpha   REAL
            On entry, ALPHA specifies the scalar alpha.
    
    @param[in]
    dA      REAL array of DIMENSION ( LDA, ka ), where ka is
            k  when  transA = MagmaNoTrans,  and is  m  otherwise.
            Before entry with  transA = MagmaNoTrans,  the leading  m by k
            part of the array dA must contain the matrix dA, otherwise
            the leading  k by m  part of the array dA must contain  the
            matrix dA.
    
    @param[in]
    ldda    INTEGER.
            On entry, LDA specifies the first dimension of A as declared
            in the calling (sub) program. When  transA = MagmaNoTrans then
            LDA must be at least  max( 1, m ), otherwise  LDA must be at
            least  max( 1, k ).
    
    @param[in]
    dB      REAL array of DIMENSION ( LDB, kb ), where kb is
            n  when  transB = MagmaNoTrans,  and is  k  otherwise.
            Before entry with  transB = MagmaNoTrans,  the leading  k by n
            part of the array dB must contain the matrix dB, otherwise
            the leading  n by k  part of the array dB must contain  the
            matrix dB.
    
    @param[in]
    lddb    INTEGER.
            On entry, LDB specifies the first dimension of dB as declared
            in the calling (sub) program. When  transB = MagmaNoTrans then
            LDB must be at least  max( 1, k ), otherwise  LDB must be at
            least  max( 1, n ).
    
    @param[in]
    beta    REAL.
            On entry,  BETA  specifies the scalar  beta.  When  BETA  is
            supplied as zero then dC need not be set on input.
    
    @param[in,out]
    dC      REAL array of DIMENSION ( LDC, n ).
            Before entry, the leading  m by n  part of the array  dC must
            contain the matrix  dC,  except when  beta  is zero, in which
            case dC need not be set on entry.
            On exit, the array  dC  is overwritten by the  m by n  matrix
            ( alpha*op( dA )*op( dB ) + beta*dC ).
    
    @param[in]
    lddc    INTEGER.
            On entry, LDC specifies the first dimension of dC as declared
            in  the  calling  (sub)  program.   LDC  must  be  at  least
            max( 1, m ).

    @ingroup magma_sblas3
    ********************************************************************/
void
magmablas_sgemm_batched(
    magma_trans_t transA, magma_trans_t transB, 
    magma_int_t m, magma_int_t n, magma_int_t k,
    float alpha,
    float const * const * dA_array, magma_int_t ldda,
    float const * const * dB_array, magma_int_t lddb,
    float beta,
    float **dC_array, magma_int_t lddc, 
    magma_int_t batchCount, magma_queue_t queue )
{
    magma_int_t info = 0;
    if      ( transA != MagmaNoTrans && transA != MagmaTrans && transA != MagmaConjTrans )
        info = -1;
    else if ( transB != MagmaNoTrans && transB != MagmaTrans && transB != MagmaConjTrans )
        info = -2;
    else if ( m < 0 )
        info = -3;
    else if ( n < 0 )
        info = -4;
    else if ( k < 0 )
        info = -5;
    else if ( transA == MagmaNoTrans ? ldda < m : ldda < k )
        info = -8;
    else if ( transB == MagmaNoTrans ? lddb < k : lddb < n )
        info = -10;
    else if ( lddc < m )
        info = -13;
    
    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }
    
    magma_int_t arch = magma_getdevice_arch();
    if ( arch < 200  ) {
        printf("arch < 200 not supported \n"); // TODO call cublas
        return;
    }
    
    if ( m <= 0 || n <= 0 || k <= 0 )
        return;

    magma_int_t shape;
    if      (transA == MagmaNoTrans   && transB == MagmaNoTrans)   { shape = 0; } // nn
    else if (transA == MagmaNoTrans   && transB == MagmaTrans)     { shape = 1; } // nt
    else if (transA == MagmaNoTrans   && transB == MagmaConjTrans) { shape = 2; } // nc
    else if (transA == MagmaTrans     && transB == MagmaNoTrans)   { shape = 3; } // tn
    else if (transA == MagmaTrans     && transB == MagmaTrans)     { shape = 4; } // tt
    else if (transA == MagmaTrans     && transB == MagmaConjTrans) { shape = 5; } // tc
    else if (transA == MagmaConjTrans && transB == MagmaNoTrans)   { shape = 6; } // cn
    else if (transA == MagmaConjTrans && transB == MagmaTrans)     { shape = 7; } // ct
    else if (transA == MagmaConjTrans && transB == MagmaConjTrans) { shape = 8; } // cc
    
    //TODO: probably the texture init code should be placed here

    size_t offsetA = 0;
    size_t offsetB = 0;
    offsetA = offsetA/sizeof(float);
    offsetB = offsetB/sizeof(float);
    
    switch(shape)
    {
        case 0: // nn
            {
                if (k < 64)
                {
                    if (k == 8 && n == 24)
                        gemm_template_batched_nn<float, version(NN,512), 0, 0>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
                    else if (n < 32)
                        gemm_template_batched_nn<float, version(NN,510), 0, 0>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
                    else
                        gemm_template_batched_nn<float, version(NN,504), 0, 0>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
                }
                else
                {
                    gemm_template_batched_nn<float, version(NN,518), 0, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
                }
            }
            break;
        case 1: // nt
            {
                gemm_template_batched_nt<float, version(NT,734), 0, 0>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
            }
            break;
        case 2: // nc
            {
                gemm_template_batched_nt<float, version(NT,734), 0, 1>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
            }
            break;
        case 3: // tn
            {
                if (k < 64)
                {
                    gemm_template_batched_tn<float, version(TN,654), 0, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
                }
                else
                {
                    gemm_template_batched_tn<float, version(TN,666), 0, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
                }
            }
            break;
        case 6: // cn
            {
                if (k < 64)
                {
                    gemm_template_batched_tn<float, version(TN,654), 1, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
                }
                else
                {
                    gemm_template_batched_tn<float, version(TN,666), 1, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
                }
            }
            break;
        case 4: // tt
            {
                if (k < 128)
                {
                    if (m < 128)
                    {
                        gemm_template_batched_tt<float, version(TT,275), 0, 0>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
                    }
                    else
                    {
                        gemm_template_batched_tt<float, version(TT,312), 0, 0>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
                    }
                }
                else
                {
                    gemm_template_batched_tt<float, version(TT,312), 0, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
                }
            }
            break;
        case 5: // tc
            {
                if (k < 128)
                {
                    if (m < 128)
                    {
                        gemm_template_batched_tt<float, version(TT,275), 0, 1>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
                    }
                    else
                    {
                        gemm_template_batched_tt<float, version(TT,312), 0, 1>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
                    }
                }
                else
                {
                    gemm_template_batched_tt<float, version(TT,312), 0, 1>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
                }
            }
            break;
        case 7: // ct
            {
                if (k < 128)
                {
                    if (m < 128)
                    {
                        gemm_template_batched_tt<float, version(TT,275), 1, 0>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
                    }
                    else
                    {
                        gemm_template_batched_tt<float, version(TT,312), 1, 0>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
                    }
                }
                else
                {
                    gemm_template_batched_tt<float, version(TT,312), 1, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
                }
            }
            break;
        case 8: // cc
            {
                if (k < 128)
                {
                    if (m < 128)
                    {
                        gemm_template_batched_tt<float, version(TT,275), 1, 1>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
                    }
                    else
                    {
                        gemm_template_batched_tt<float, version(TT,312), 1, 1>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
                    }
                }
                else
                {
                    gemm_template_batched_tt<float, version(TT,312), 1, 1>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, offsetA, offsetB, batchCount, queue);
                }
            }
            break;
        default:; // propose something
    }
}
///////////////////////////////////////////////////////////////////////////////////////////////////
