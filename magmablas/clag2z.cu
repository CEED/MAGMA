#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions mixed zc -> ds
       @author Mark Gates
*/
#include "magma_internal.h"

#define BLK_X 64
#define BLK_Y 32


/*
    Divides matrix into ceil( m/BLK_X ) x ceil( n/BLK_Y ) blocks.
    Each block has BLK_X threads.
    Each thread loops across one row, updating BLK_Y entries.
    
    Code similar to clat2z and zlaset.
*/
__global__
void clag2z_kernel(
    int m, int n,
    const magmaFloatComplex *SA, int ldsa,
    magmaDoubleComplex       *A, int lda )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    /* check if full block-column */
    bool full = (iby + BLK_Y <= n);
    /* do only rows inside matrix */
    if ( ind < m ) {
        A  += ind + iby*lda;
        SA += ind + iby*ldsa;
        if ( full ) {
            // full block-column
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                A[j*lda] = MAGMA_Z_MAKE( MAGMA_C_REAL( SA[j*ldsa] ), MAGMA_C_IMAG( SA[j*ldsa] ));
            }
        }
        else {
            // partial block-column
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
                A[j*lda] = MAGMA_Z_MAKE( MAGMA_C_REAL( SA[j*ldsa] ), MAGMA_C_IMAG( SA[j*ldsa] ));
            }
        }
    }
}


/***************************************************************************//**
    Purpose
    -------
    CLAG2Z converts a single-complex matrix, SA,
                 to a double-complex matrix, A.

    Note that while it is possible to overflow while converting
    from double to single, it is not possible to overflow when
    converting from single to double.

    Arguments
    ---------
    @param[in]
    m       INTEGER
            The number of lines of the matrix A.  M >= 0.

    @param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.

    @param[in]
    SA      COMPLEX array, dimension (LDSA,N)
            On entry, the M-by-N coefficient matrix SA.

    @param[in]
    ldsa    INTEGER
            The leading dimension of the array SA.  LDSA >= max(1,M).

    @param[out]
    A       COMPLEX_16 array, dimension (LDA,N)
            On exit, the M-by-N coefficient matrix A.

    @param[in]
    lda     INTEGER
            The leading dimension of the array A.  LDA >= max(1,M).

    @param[out]
    info    INTEGER
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.
    
    @ingroup magma_lag2
*******************************************************************************/
extern "C" void
magmablas_clag2z(
    magma_int_t m, magma_int_t n,
    magmaFloatComplex_const_ptr SA, magma_int_t ldsa,
    magmaDoubleComplex_ptr       A, magma_int_t lda,
    magma_queue_t queue,
    magma_int_t *info )
{
    *info = 0;
    if ( m < 0 )
        *info = -1;
    else if ( n < 0 )
        *info = -2;
    else if ( ldsa < max(1,m) )
        *info = -4;
    else if ( lda < max(1,m) )
        *info = -6;

    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return; //*info;
    }

    /* quick return */
    if ( m == 0 || n == 0 ) {
        return;
    }

    dim3 threads( BLK_X, 1 );
    dim3 grid( magma_ceildiv( m, BLK_X ), magma_ceildiv( n, BLK_Y ) );
    clag2z_kernel<<< grid, threads, 0, queue->hip_stream() >>> ( m, n, SA, ldsa, A, lda );
}
