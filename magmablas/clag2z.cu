#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions mixed zc -> ds

*/
#include "common_magma.h"

__global__ void 
clag2z_generic(int M, int N, 
               const magmaFloatComplex *SA, int LDSA, 
               magmaDoubleComplex       *A, int LDA ) 
{ 
    int ibx = blockIdx.x * 64;

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int idt = ty * 16 + tx;
        
    if( (ibx+idt) >= M ){
        SA += (M-1);
        A  += (M-1);
    }
    else{
        SA += ibx+idt;
        A  += ibx+idt;
    }
    const magmaFloatComplex * SAend = SA+LDSA*N;
    magmaDoubleComplex Ap[1]={ hipComplexFloatToDouble(SA[0]) };
    do {
        SA  += LDSA;
        A[0] = Ap[0];
        Ap[0]= hipComplexFloatToDouble(SA[0]);
        A   += LDA;

    } while (SA < SAend);

    A[0] = Ap[0];
}

__global__ void 
clag2z_special(int M, int N, 
               const magmaFloatComplex *SA, int LDSA, 
               magmaDoubleComplex       *A, int LDA ) 
{ 
    int ibx = blockIdx.x * 64;

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int idt = ty * 16 + tx;
        
    if( (ibx+idt) >= M ){
        SA += (M-1);
        A  += (M-1);
    }
    else{
        SA += ibx+idt;
        A  += ibx+idt;
    }
    magmaDoubleComplex Ap[1] = { hipComplexFloatToDouble(SA[0]) };
    A[0] = Ap[0];
}

extern "C" void 
magmablas_clag2z_64_64_16_4_v2( magma_int_t M, magma_int_t N, 
                                const magmaFloatComplex *SA, magma_int_t LDSA, 
                                magmaDoubleComplex       *A, magma_int_t LDA )
{
    if( M == 0 || N==0 ) {
        printf("One of the dimension is ZERO\n");
        exit(-1);
    }
    dim3 threads( 16, 4 );
    dim3 grid(M/64+(M%64!=0),1);
    if( N > 1 ) {
        clag2z_generic<<< grid, threads, 0, magma_stream >>> (  M, N, SA, LDSA, A, LDA ) ;
    }
    else{
        clag2z_special<<< grid, threads, 0, magma_stream >>> (  M, N, SA, LDSA, A, LDA ) ;
    }
}

/**
    Purpose
    -------
    
    CLAG2Z converts a SINGLE PRECISION matrix, SA,
                 to a DOUBLE PRECISION matrix, A.
    
    Note that while it is possible to overflow while converting
    from double to single, it is not possible to overflow when
    converting from single to double.
        
    Arguments
    ---------
    
    @param[in]
    m       INTEGER
            The number of lines of the matrix A.  M >= 0.
    
    @param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.
    
    @param[in]
    SA      REAL array, dimension (LDSA,N)
            On entry, the M-by-N coefficient matrix SA.
    
    @param[in]
    ldsa    INTEGER
            The leading dimension of the array SA.  LDSA >= max(1,M).
    
    @param[out]
    A       DOUBLE PRECISION array, dimension (LDA,N)
            On exit, the M-by-N coefficient matrix A.
    
    @param[in]
    lda     INTEGER
            The leading dimension of the array A.  LDA >= max(1,M).
    
    @param[out]
    info    INTEGER
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value

    @ingroup magma_caux2
    ********************************************************************/
extern "C" void 
magmablas_clag2z(
    magma_int_t m, magma_int_t n,
    const magmaFloatComplex *SA, magma_int_t ldsa,
    magmaDoubleComplex       *A, magma_int_t lda,
    magma_int_t *info)
{
    *info = 0;
    if ( m < 0 )
        *info = -1;
    else if ( n < 0 )
        *info = -2;
    else if ( ldsa < max(1,m) )
        *info = -4;
    else if ( lda < max(1,m) )
        *info = -6;
    
    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        //return *info;
    }
    
    magmablas_clag2z_64_64_16_4_v2( m, n, SA, ldsa, A, lda );
}        
