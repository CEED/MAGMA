#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

       @precisions normal z -> s d c

*/

#include <stdio.h>
#include <hipblas.h>

#define BLOCK_SIZE 64

typedef struct {
        double2 *A;
        int n, lda, j0;
        short ipiv[BLOCK_SIZE];
} zlaswp_params_t;

typedef struct {
        double2 *A;
        int n, lda, j0, npivots;
        short ipiv[BLOCK_SIZE];
} zlaswp_params_t2;

/*********************************************************
 *
 * SWAP BLAS: permute to set of N elements
 *
 ********************************************************/
/*
 *  First version: line per line
 */
typedef struct {
    hipDoubleComplex *A1;
    hipDoubleComplex *A2;
    int n; /*, lda1, lda2;*/
} zswap_params_t;

__global__ void myzswap( zswap_params_t params )
{
    unsigned int x = threadIdx.x + blockDim.x*blockIdx.x;
    /* unsigned int offset1 = __mul24( x, params.lda1); */
    /* unsigned int offset2 = __mul24( x, params.lda2); */
    if( x < params.n )
    {
        hipDoubleComplex *A1  = params.A1 + x; /*offset1;*/
        hipDoubleComplex *A2  = params.A2 + x; /*offset2;*/
        hipDoubleComplex temp = *A1;
        *A1 = *A2;
        *A2 = temp;
    }
}

extern "C" void zswap( zswap_params_t &params )
{
    int blocksize = 64;
    dim3 blocks = (params.n+blocksize-1) / blocksize;
    myzswap<<< blocks, blocksize >>>( params );
}


extern "C" void 
magmablas_zswap( int n, hipDoubleComplex *dA1T, int lda1, 
                 hipDoubleComplex *dA2T, int lda2)
{
    zswap_params_t params = { dA1T, dA2T, n }; /*, lda1, lda2 };*/
    int  blocksize = 64;
    int  blocks = (params.n+blocksize-1) / blocksize;
    
    myzswap<<< blocks, blocksize >>>( params );
}

/*********************************************************/
/*
*  Blocked version: swap several pair of line
 */
typedef struct {
    hipDoubleComplex *A1;
    hipDoubleComplex *A2;
    int n, ldx1, ldx2, ldy1, ldy2, npivots;
    short ipiv[BLOCK_SIZE];
} zswapblk_params_t;

__global__ void myzswapblk( zswapblk_params_t params )
{
    unsigned int y = threadIdx.x + blockDim.x*blockIdx.x;
    /* unsigned int offset1 = __mul24( y, params.ldy1); */
    /* unsigned int offset2 = __mul24( y, params.ldy2); */
    if( y < params.n )
    {
        hipDoubleComplex *A1 = params.A1 + y - params.ldx1; /*offset1*/
        hipDoubleComplex *A2 = params.A2 + y;               /*offset2*/
      
        for( int i = 0; i < params.npivots; i++ )
        {
            A1 += params.ldx1;
            if ( params.ipiv[i] == -1 )
                continue;
            hipDoubleComplex tmp1  = *A1;
            hipDoubleComplex *tmp2 = A2 + params.ipiv[i]*params.ldx2;
            *A1   = *tmp2;
            *tmp2 = tmp1;
        }
    }
}

extern "C" void zswapblk( zswapblk_params_t &params )
{
    int blocksize = 64;
    dim3 blocks = (params.n+blocksize-1) / blocksize;
    myzswapblk<<< blocks, blocksize >>>( params );
}


extern "C" void 
magmablas_zswapblk( int n, hipDoubleComplex *dA1T, int ldx1, int ldy1, 
                    hipDoubleComplex *dA2T, int ldx2, int ldy2,
                    int i1, int i2, int *ipiv, int inci, int offset )
{
    int  k, im;
    for( k=(i1-1); k<i2; k+=BLOCK_SIZE )
    {
        int sb = min(BLOCK_SIZE, i2-k);
        zswapblk_params_t params = { dA1T+k*ldx1, dA2T, n, ldx1, ldx2, ldy1, ldy2, sb };
        for( int j = 0; j < sb; j++ )
        {
            im = ipiv[(k+j)*inci] - 1;
            if ( (k+j) == im)
                params.ipiv[j] = -1;
            else
                params.ipiv[j] = im - offset;
        }
        zswapblk ( params );
    }
}

/*********************************************************
 *
 * LAPACK Swap: permute a set of lines following ipiv
 *
 ********************************************************/
typedef struct {
    double2 *A;
    int n, ldx, ldy, j0, npivots;
    short ipiv[BLOCK_SIZE];
} zlaswpx_params_t;

__global__ void myzlaswpx( zlaswpx_params_t params )
{
    unsigned int y = threadIdx.x + __mul24(blockDim.x, blockIdx.x);
    unsigned int offset1 = __mul24( y, params.ldy);
    if( y < params.n )
    {
        int ldx = params.ldx;
        hipDoubleComplex *A = params.A + offset1 + ldx * params.j0;
        hipDoubleComplex *Ai = A;
        
        for( int i = 0; i < params.npivots; i++ )
        {
            int j = params.ipiv[i];
            hipDoubleComplex *p2 = A + j*ldx;
            hipDoubleComplex temp = *Ai;
            *Ai = *p2;
            *p2 = temp;
            Ai += ldx;
        }
    }
}

extern "C" void zlaswpx( zlaswpx_params_t &params )
{
 	int blocksize = 64;
	dim3 blocks = (params.n+blocksize-1) / blocksize;
	myzlaswpx<<< blocks, blocksize >>>( params );
}

/*
 * Old version
 */
__global__ void myzlaswp2( zlaswp_params_t2 params )
{
        unsigned int tid = threadIdx.x + __mul24(blockDim.x, blockIdx.x);
        if( tid < params.n )
	{
                int lda = params.lda;
		double2 *A = params.A + tid + lda * params.j0;

		for( int i = 0; i < params.npivots; i++ )
		{
                 	int j = params.ipiv[i];
			double2 *p1 = A + i*lda;
			double2 *p2 = A + j*lda;
			double2 temp = *p1;
			*p1 = *p2;
			*p2 = temp;
		}
	}
}

extern "C" void zlaswp2( zlaswp_params_t &params );

extern "C" void zlaswp3( zlaswp_params_t2 &params )
{
 	int blocksize = 64;
	dim3 blocks = (params.n+blocksize-1) / blocksize;
	myzlaswp2<<< blocks, blocksize >>>( params );
}


extern "C" void 
magmablas_zpermute_long2( double2 *dAT, int lda, int *ipiv, int nb, int ind )
{
        int k;

        for( k = 0; k < nb-BLOCK_SIZE; k += BLOCK_SIZE )
        {
                //zlaswp_params_t params = { dAT, lda, lda, ind + k };
                zlaswp_params_t2 params = { dAT, lda, lda, ind + k, BLOCK_SIZE };
                for( int j = 0; j < BLOCK_SIZE; j++ )
                {
                        params.ipiv[j] = ipiv[ind + k + j] - k - 1;
                        ipiv[ind + k + j] += ind;
                }
                //zlaswp2( params );
	        zlaswp3( params );
        }

	int num_pivots = nb - k;

        zlaswp_params_t2 params = { dAT, lda, lda, ind + k, num_pivots};
        for( int j = 0; j < num_pivots; j++ )
        {
            params.ipiv[j] = ipiv[ind + k + j] - k - 1;
            ipiv[ind + k + j] += ind;
        }
        zlaswp3( params );
}

extern "C" void 
magmablas_zlaswp( int n, hipDoubleComplex *dAT, int lda, 
                  int i1, int i2, int *ipiv, int inci )
{
  int k;
  
  for( k=(i1-1); k<i2; k+=BLOCK_SIZE )
    {
      int sb = min(BLOCK_SIZE, i2-k);
      //zlaswp_params_t params = { dAT, lda, lda, ind + k };
      zlaswp_params_t2 params = { dAT+k*lda, n, lda, 0, sb };
      for( int j = 0; j < sb; j++ )
        {
          params.ipiv[j] = ipiv[(k+j)*inci] - k - 1;
        }
      zlaswp3( params );
    }
}

extern "C" void 
magmablas_zlaswpx( int n, hipDoubleComplex *dAT, int ldx, int ldy, 
                   int i1, int i2, int *ipiv, int inci )
{
  int k;
  
  for( k=(i1-1); k<i2; k+=BLOCK_SIZE )
    {
      int sb = min(BLOCK_SIZE, i2-k);
      //zlaswp_params_t params = { dAT, lda, lda, ind + k };
      zlaswpx_params_t params = { dAT+k*ldx, n, ldx, ldy, 0, sb };
      for( int j = 0; j < sb; j++ )
        {
          params.ipiv[j] = ipiv[(k+j)*inci] - k - 1;
        }
      zlaswpx( params );
    }
}

#undef BLOCK_SIZE
