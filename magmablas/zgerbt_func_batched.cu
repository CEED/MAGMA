#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c

       @author Adrien Remy
       @author Azzam Haidar
*/
#include "magma_internal.h"
#include "zgerbt.h"

#define block_height  32
#define block_width  4
#define block_length 256
#define NB 64

/***************************************************************************//**
    Purpose
    -------
    ZPRBT_MVT compute B = UTB to randomize B
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of values of db.  n >= 0.

    @param[in]
    du     COMPLEX_16 array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in,out]
    db     COMPLEX_16 array, dimension (n)
            The n vector db computed by ZGESV_NOPIV_GPU
            On exit db = du*db
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.
*******************************************************************************/
extern "C" void
magmablas_zprbt_mtv_batched(
    magma_int_t n, 
    magmaDoubleComplex *du, magmaDoubleComplex **db_array,
    magma_int_t batchCount, magma_queue_t queue)
{
    magma_int_t threads = block_length;
    dim3 grid( magma_ceildiv( n, 4*block_length ), batchCount);

    magmablas_zapply_transpose_vector_kernel_batched<<< grid, threads, 0, queue->hip_stream() >>>(n/2, du, n, db_array, 0);
    magmablas_zapply_transpose_vector_kernel_batched<<< grid, threads, 0, queue->hip_stream() >>>(n/2, du, n+n/2, db_array, n/2);

    threads = block_length;
    grid = magma_ceildiv( n, 2*block_length );
    magmablas_zapply_transpose_vector_kernel_batched<<< grid, threads, 0, queue->hip_stream() >>>(n, du, 0, db_array, 0);
}


/***************************************************************************//**
    Purpose
    -------
    ZPRBT_MV compute B = VB to obtain the non randomized solution
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of values of db.  n >= 0.
    
    @param[in,out]
    db      COMPLEX_16 array, dimension (n)
            The n vector db computed by ZGESV_NOPIV_GPU
            On exit db = dv*db
    
    @param[in]
    dv      COMPLEX_16 array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.
*******************************************************************************/
extern "C" void
magmablas_zprbt_mv_batched(
    magma_int_t n, 
    magmaDoubleComplex *dv, magmaDoubleComplex **db_array,
    magma_int_t batchCount, magma_queue_t queue)
{
    magma_int_t threads = block_length;
    dim3 grid ( magma_ceildiv( n, 2*block_length ), batchCount);

    magmablas_zapply_vector_kernel_batched<<< grid, threads, 0, queue->hip_stream() >>>(n, dv, 0, db_array, 0);

    threads = block_length;
    grid = magma_ceildiv( n, 4*block_length );

    magmablas_zapply_vector_kernel_batched<<< grid, threads, 0, queue->hip_stream() >>>(n/2, dv, n, db_array, 0);
    magmablas_zapply_vector_kernel_batched<<< grid, threads, 0, queue->hip_stream() >>>(n/2, dv, n+n/2, db_array, n/2);
}


/***************************************************************************//**
    Purpose
    -------
    ZPRBT randomize a square general matrix using partial randomized transformation
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of columns and rows of the matrix dA.  n >= 0.
    
    @param[in,out]
    dA      COMPLEX_16 array, dimension (n,ldda)
            The n-by-n matrix dA
            On exit dA = duT*dA*d_V
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDA >= max(1,n).
    
    @param[in]
    du      COMPLEX_16 array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix U
    
    @param[in]
    dv      COMPLEX_16 array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.
*******************************************************************************/
extern "C" void 
magmablas_zprbt_batched(
    magma_int_t n, 
    magmaDoubleComplex **dA_array, magma_int_t ldda, 
    magmaDoubleComplex *du, magmaDoubleComplex *dv,
    magma_int_t batchCount, magma_queue_t queue)
{
    du += ldda;
    dv += ldda;

    dim3 threads(block_height, block_width);
    dim3 grid( magma_ceildiv( n, 4*block_height ), 
               magma_ceildiv( n, 4*block_width  ),
               batchCount );

    magmablas_zelementary_multiplication_kernel_batched<<< grid, threads, 0, queue->hip_stream() >>>(n/2, dA_array,            0, ldda, du,   0, dv,   0);
    magmablas_zelementary_multiplication_kernel_batched<<< grid, threads, 0, queue->hip_stream() >>>(n/2, dA_array,     ldda*n/2, ldda, du,   0, dv, n/2);
    magmablas_zelementary_multiplication_kernel_batched<<< grid, threads, 0, queue->hip_stream() >>>(n/2, dA_array,          n/2, ldda, du, n/2, dv,   0);
    magmablas_zelementary_multiplication_kernel_batched<<< grid, threads, 0, queue->hip_stream() >>>(n/2, dA_array, ldda*n/2+n/2, ldda, du, n/2, dv, n/2);

    dim3 threads2(block_height, block_width);
    dim3 grid2( magma_ceildiv( n, 2*block_height ), 
                magma_ceildiv( n, 2*block_width  ),
                batchCount );
    magmablas_zelementary_multiplication_kernel_batched<<< grid2, threads2, 0, queue->hip_stream() >>>(n, dA_array, 0, ldda, du, -ldda, dv, -ldda);
}
