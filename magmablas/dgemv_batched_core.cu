#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal d
       
       @author Tingxing Dong
       @author Azzam Haidar

*/
#include "common_magma.h"
#include "commonblas_d.h"
#include "magma_templates.h"

#define PRECISION_d

#include "gemv_template_kernel_batched.cuh"
#include "gemv_config/gemvn_param.h"
#include "gemv_config/gemvt_param.h"
#define version(s,v) s ## _V_ ## v



/**
    Purpose
    -------
    DGEMV performs one of the matrix-vector operations
    
        y := alpha*A*x    + beta*y,   or
        y := alpha*A**T*x + beta*y,   or
        y := alpha*A**H*x + beta*y,
    
    where alpha and beta are scalars, x and y are vectors and A is an
    m by n matrix.

    Arguments
    ----------
    @param[in]
    trans   magma_trans_t
            On entry, TRANS specifies the operation to be performed as
            follows:
      -     = MagmaNoTrans:    y := alpha*A  *x + beta*y
      -     = MagmaTrans:      y := alpha*A^T*x + beta*y
      -     = MagmaConjTrans:  y := alpha*A^H*x + beta*y

    @param[in]
    m       INTEGER
            On entry, m specifies the number of rows of the matrix A.

    @param[in]
    n       INTEGER
            On entry, n specifies the number of columns of the matrix A
 
    @param[in]
    alpha   DOUBLE_PRECISION
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    dA      DOUBLE_PRECISION array of dimension ( LDDA, n ) on the GPU.
   
    @param[in]
    ldda    INTEGER
            LDDA specifies the leading dimension of A.

    @param[in]
    dx      DOUBLE_PRECISION array of dimension
            n if trans == MagmaNoTrans
            m if trans == MagmaTrans or MagmaConjTrans
     
    @param[in]
    incx    Specifies the increment for the elements of X.
            INCX must not be zero.
  
    @param[in]
    beta    DOUBLE REAL
            On entry, BETA specifies the scalar beta. When BETA is
            supplied as zero then Y need not be set on input.

    @param[out]
    dy      DOUBLE PRECISION array of dimension
            m if trans == MagmaNoTrans
            n if trans == MagmaTrans or MagmaConjTrans

    @param[in]
    incy    Specifies the increment for the elements of Y.
            INCY must not be zero.

    @ingroup magma_dblas2
    ********************************************************************/

extern "C" void
magmablas_dgemv_batched(
    magma_trans_t trans, magma_int_t m, magma_int_t n, 
    double alpha,
    magmaDouble_ptr dA_array[], magma_int_t ldda, 
    magmaDouble_ptr dx_array[], magma_int_t incx,
    double beta,
    magmaDouble_ptr dy_array[], magma_int_t incy, 
    magma_int_t batchCount, magma_queue_t queue)
{
    magma_int_t info = 0;
    if ( trans != MagmaNoTrans && trans != MagmaTrans && trans != MagmaConjTrans )
        info = -1;
    else if ( m < 0 )
        info = -2;
    else if ( n < 0 )
        info = -3;
    else if ( ldda < m )
        info = -6;
    else if ( incx == 0 )
        info = -8;
    else if ( incy == 0 )
        info = -11;
    
    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }    
                                                                                   
    if ( trans == MagmaNoTrans ) {                                                   
        if(max(m, n) <= 96){ // small size                         
            if(m < n){ // Fat or square matrix
                if( m<=16) 
                {
                    gemvn_template_batched<double, version(N, 72)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else if( m<=32)            
                {
                    gemvn_template_batched<double, version(N, 100)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else if( m<=64)            
                {
                    gemvn_template_batched<double, version(N, 122)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else
                {
                    gemvn_template_batched<double, version(N, 135)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }  
            }else{   // Tall or square matrix
                if ( n<=16) 
                {    
                    gemvn_template_batched<double, version(N, 128)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else if ( n<=64) 
                {
                    gemvn_template_batched<double, version(N, 132)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }  
                else 
                {
                    gemvn_template_batched<double, version(N, 135)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }  
            }
        }
        else{ // big size
            if(m < n){ // Fat matrix
                if(m <= 8)
                {
                    gemvn_template_batched<double, version(N, 79)>              
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else if(m <= 16)
                {
                    gemvn_template_batched<double, version(N, 70)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else if(m <= 32)
                {
                    gemvn_template_batched<double, version(N, 104)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else if(m <= 32)
                {
                    gemvn_template_batched<double, version(N, 124)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else
                {
                    gemvn_template_batched<double, version(N, 135)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
            }
            else{ // (m > n) Tall matrix
                if(m <= 256)
                {
                    gemvn_template_batched<double, version(N, 137)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else
                {
                    gemvn_template_batched<double, version(N, 140)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
            }
        }// big size        
     }
     else{                                                 

        magma_int_t CONJA = -1;

        if ( trans == MagmaConjTrans ) {
            CONJA = 1;
        }else {
            CONJA = 0;
        }


        if(max(m, n) <= 96){ // small size                              
            if(m <= 16)
            {
                gemvc_template_batched<double, version(T, 42)>             
                        (m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, CONJA, batchCount, queue );
            }
            else
            {
                gemvc_template_batched<double, version(T, 47)>             
                        (m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, CONJA, batchCount, queue );
            }

        }else{ // big size
            if(m <= n){ //  Fat or square matrix
                if(m <=64)
                {
                    gemvc_template_batched<double, version(T, 47)>             
                        (m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, CONJA, batchCount, queue );
                }
                else
                {
                    gemvc_template_batched<double, version(T, 91)>             
                        (m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, CONJA, batchCount, queue );
                }

            }else{ // (m > n) Tall matrix
                if(n <= 64)
                {
                    gemvc_template_batched<double, version(T, 90)>             
                        (m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, CONJA, batchCount, queue );
                }
                else
                {
                    gemvc_template_batched<double, version(T, 91)>             
                        (m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, CONJA, batchCount, queue );
                }
            }
        }        
     }                   

}
///////////////////////////////////////////////////////////////////////////////////////////////////
