#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal z -> s d c

*/
#include "common_magma.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512

// ----------------------------------------
// Does sum reduction of array x, leaving total in x[0].
// Contents of x are destroyed in the process.
// With k threads, can reduce array up to 2*k in size.
// Assumes number of threads <= 1024 (which is max number of threads up to CUDA capability 3.0)
// Having n as template parameter allows compiler to evaluate some conditions at compile time.
template< int n >
__device__ void sum_reduce( /*int n,*/ int i, hipDoubleComplex* x )
{
    __syncthreads();
    if ( n > 1024 ) { if ( i < 1024 && i + 1024 < n ) { x[i] += x[i+1024]; }  __syncthreads(); }
    if ( n >  512 ) { if ( i <  512 && i +  512 < n ) { x[i] += x[i+ 512]; }  __syncthreads(); }
    if ( n >  256 ) { if ( i <  256 && i +  256 < n ) { x[i] += x[i+ 256]; }  __syncthreads(); }
    if ( n >  128 ) { if ( i <  128 && i +  128 < n ) { x[i] += x[i+ 128]; }  __syncthreads(); }
    if ( n >   64 ) { if ( i <   64 && i +   64 < n ) { x[i] += x[i+  64]; }  __syncthreads(); }
    if ( n >   32 ) { if ( i <   32 && i +   32 < n ) { x[i] += x[i+  32]; }  __syncthreads(); }
    // probably don't need __syncthreads for < 16 threads
    // because of implicit warp level synchronization.
    if ( n >   16 ) { if ( i <   16 && i +   16 < n ) { x[i] += x[i+  16]; }  __syncthreads(); }
    if ( n >    8 ) { if ( i <    8 && i +    8 < n ) { x[i] += x[i+   8]; }  __syncthreads(); }
    if ( n >    4 ) { if ( i <    4 && i +    4 < n ) { x[i] += x[i+   4]; }  __syncthreads(); }
    if ( n >    2 ) { if ( i <    2 && i +    2 < n ) { x[i] += x[i+   2]; }  __syncthreads(); }
    if ( n >    1 ) { if ( i <    1 && i +    1 < n ) { x[i] += x[i+   1]; }  __syncthreads(); }
}
// end sum_reduce


__global__
void magma_zlarf_kernel( int m, hipDoubleComplex *v, hipDoubleComplex *tau,
                         hipDoubleComplex *c, int ldc, double *xnorm )
{
    if ( !MAGMA_Z_EQUAL(*tau, MAGMA_Z_ZERO) ) {
        const int i = threadIdx.x;
        hipDoubleComplex *dc = c + blockIdx.x * ldc;//, alpha;

        __shared__ hipDoubleComplex sum[ BLOCK_SIZE ];

        if (i==0){
            //alpha = v[0];
            v[0]  = MAGMA_Z_ONE;
        } 
        __syncthreads();

        /*  w := v' * C  */
        sum[i] = MAGMA_Z_ZERO;
        for( int j = i; j < m; j += BLOCK_SIZE )
            sum[i] += MAGMA_Z_MUL( MAGMA_Z_CNJG( v[j] ), dc[j] );
        sum_reduce< BLOCK_SIZE >( i, sum );

        /*  C := C - v * w  */
        __syncthreads();
        hipDoubleComplex z__1 = - MAGMA_Z_CNJG(*tau) * sum[0];
        for( int j = i; j < m; j += BLOCK_SIZE ) {
                dc[j] += z__1 * v[j];
        }
        
        if (i==0){
            //v[0] = alpha;

            double temp = MAGMA_Z_ABS( dc[0] ) / xnorm[blockIdx.x];
            temp = (temp + 1.) * (1. - temp);
            xnorm[blockIdx.x] = xnorm[blockIdx.x] * sqrt(temp); 
        }
    }
}

/*
    Apply a complex elementary reflector H to a complex M-by-N
    matrix C from the left. H is represented in the form
          H = I - tau * v * v'
    where tau is a complex scalar and v is a complex vector.
    If tau = 0, then H is taken to be the unit matrix.

    To apply H' (the conjugate transpose of H), supply conjg(tau) 
    instead tau.
 */
extern "C" void
magma_zlarf_gpu(int m, int n, hipDoubleComplex *v, hipDoubleComplex *tau,
                hipDoubleComplex *c, int ldc, double *xnorm)
{
    dim3  blocks( n );
    dim3 threads( BLOCK_SIZE );

    magma_zlarf_kernel<<< blocks, threads >>>( m, v, tau, c, ldc, xnorm);
}
