#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c
       @author Azzam Haidar

*/
#include "common_magma.h"
#include "magma_templates.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512

#define BLOCK_SIZEx  32
#define BLOCK_SIZEy  16


//==============================================================================
//==============================================================================

__global__
void magma_zlarf_kernel( int m, magmaDoubleComplex *v, magmaDoubleComplex *tau,
                         magmaDoubleComplex *c, int ldc )
{
    if ( !MAGMA_Z_EQUAL(*tau, MAGMA_Z_ZERO) ) {
        const int tx = threadIdx.x;
        magmaDoubleComplex *dc = c + blockIdx.x * ldc;

        __shared__ magmaDoubleComplex sum[ BLOCK_SIZE ];
        magmaDoubleComplex lsum;

        /* perform  w := v' * C  */
        if (tx==0)
            lsum = dc[0]; //since V[0] should be one
        else
            lsum = MAGMA_Z_ZERO;
        for( int j = tx+1; j < m; j += BLOCK_SIZE ){
            lsum += MAGMA_Z_MUL( MAGMA_Z_CNJG( v[j] ), dc[j] );
        }
        sum[tx] = lsum;
        magma_sum_reduce< BLOCK_SIZE >( tx, sum );

        /*  C := C - v * w  */
        __syncthreads();
        magmaDoubleComplex z__1 = - MAGMA_Z_CNJG(*tau) * sum[0];
        for( int j = m-tx-1; j>0 ; j -= BLOCK_SIZE )
             dc[j] += z__1 * v[j];

        if(tx==0) dc[0] += z__1;
    }
}

//==============================================================================
//==============================================================================

__global__
void magma_zlarf_smkernel( int m, int n, magmaDoubleComplex *v, magmaDoubleComplex *tau,
                           magmaDoubleComplex *c, int ldc )
{
    if ( ! MAGMA_Z_EQUAL(*tau, MAGMA_Z_ZERO) ) {
        const int i = threadIdx.x, col= threadIdx.y;

        for( int k = col; k < n; k += BLOCK_SIZEy ) {
            magmaDoubleComplex *dc = c + k * ldc;
    
            __shared__ magmaDoubleComplex sum[ BLOCK_SIZEx ][ BLOCK_SIZEy + 1];
            magmaDoubleComplex lsum;
    
            /*  w := v' * C  */
            lsum = MAGMA_Z_ZERO;
            for( int j = i; j < m; j += BLOCK_SIZEx ){
                if (j==0)
                   lsum += MAGMA_Z_MUL( MAGMA_Z_ONE, dc[j] );
                else
                   lsum += MAGMA_Z_MUL( MAGMA_Z_CNJG( v[j] ), dc[j] );
            }
            sum[i][col] = lsum;
            magma_sum_reduce_2d< BLOCK_SIZEx, BLOCK_SIZEy+1 >( i, col, sum );
    
            /*  C := C - v * w  */
            __syncthreads();
            magmaDoubleComplex z__1 = - MAGMA_Z_CNJG(*tau) * sum[0][col];
            for( int j = m-i-1; j>=0 ; j -= BLOCK_SIZEx ) {
                 if (j==0)
                    dc[j] += z__1;
                 else
                    dc[j] += z__1 * v[j];
            }
        }
    }
}

//==============================================================================

/*
    Apply a complex elementary reflector H to a complex M-by-N
    matrix C from the left. H is represented in the form
          H = I - tau * v * v'
    where tau is a complex scalar and v is a complex vector.
    If tau = 0, then H is taken to be the unit matrix.

    To apply H' (the conjugate transpose of H), supply conjg(tau)
    instead tau.

    This routine uses only one SM (block).
 */
extern "C" void
magma_zlarf_sm(int m, int n, magmaDoubleComplex *v, magmaDoubleComplex *tau,
               magmaDoubleComplex *c, int ldc)
{
    dim3  blocks( 1 );
    dim3 threads( BLOCK_SIZEx, BLOCK_SIZEy );

    magma_zlarf_smkernel<<< blocks, threads, 0, magma_stream >>>( m, n, v, tau, c, ldc );
}
//==============================================================================
/*
    Apply a complex elementary reflector H to a complex M-by-N
    matrix C from the left. H is represented in the form
          H = I - tau * v * v'
    where tau is a complex scalar and v is a complex vector.
    If tau = 0, then H is taken to be the unit matrix.

    To apply H' (the conjugate transpose of H), supply conjg(tau) 
    instead tau.

 */

extern "C" magma_int_t
magma_zlarf_gpu(
    magma_int_t m,  magma_int_t n,
    magmaDoubleComplex *v, magmaDoubleComplex *tau,
    magmaDoubleComplex *c,  magma_int_t ldc)
{
    dim3 grid( n, 1, 1 );
    dim3 threads( BLOCK_SIZE );
    if ( n>0 ){
        magma_zlarf_kernel<<< grid, threads, 0, magma_stream >>>( m, v, tau, c, ldc);
    }

    // The computation can be done on 1 SM with the following routine.
    // magma_zlarf_sm(m, n, v, tau, c, ldc);

    return MAGMA_SUCCESS;
}

//==============================================================================
