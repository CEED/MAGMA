#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal z -> s d c

*/
#include "common_magma.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512

#define BLOCK_SIZEx  32
#define BLOCK_SIZEy  16

// ----------------------------------------
// Does sum reduction of array x, leaving total in x[0].
// Contents of x are destroyed in the process.
// With k threads, can reduce array up to 2*k in size.
// Assumes number of threads <= 1024 (which is max number of threads up to CUDA capability 3.0)
// Having n as template parameter allows compiler to evaluate some conditions at compile time.
template< int n >
__device__ void sum_reduce( /*int n,*/ int i, magmaDoubleComplex* x )
{
    __syncthreads();
    if ( n > 1024 ) { if ( i < 1024 && i + 1024 < n ) { x[i] += x[i+1024]; }  __syncthreads(); }
    if ( n >  512 ) { if ( i <  512 && i +  512 < n ) { x[i] += x[i+ 512]; }  __syncthreads(); }
    if ( n >  256 ) { if ( i <  256 && i +  256 < n ) { x[i] += x[i+ 256]; }  __syncthreads(); }
    if ( n >  128 ) { if ( i <  128 && i +  128 < n ) { x[i] += x[i+ 128]; }  __syncthreads(); }
    if ( n >   64 ) { if ( i <   64 && i +   64 < n ) { x[i] += x[i+  64]; }  __syncthreads(); }
    if ( n >   32 ) { if ( i <   32 && i +   32 < n ) { x[i] += x[i+  32]; }  __syncthreads(); }
    // probably don't need __syncthreads for < 16 threads
    // because of implicit warp level synchronization.
    if ( n >   16 ) { if ( i <   16 && i +   16 < n ) { x[i] += x[i+  16]; }  __syncthreads(); }
    if ( n >    8 ) { if ( i <    8 && i +    8 < n ) { x[i] += x[i+   8]; }  __syncthreads(); }
    if ( n >    4 ) { if ( i <    4 && i +    4 < n ) { x[i] += x[i+   4]; }  __syncthreads(); }
    if ( n >    2 ) { if ( i <    2 && i +    2 < n ) { x[i] += x[i+   2]; }  __syncthreads(); }
    if ( n >    1 ) { if ( i <    1 && i +    1 < n ) { x[i] += x[i+   1]; }  __syncthreads(); }
}
// end sum_reduce


template< int n >
__device__ void sum_reduce_2d( /*int n,*/ int i, int c, magmaDoubleComplex x[][BLOCK_SIZEy+1] )
{
    __syncthreads();
    if ( n > 1024 ) { if ( i < 1024 && i + 1024 < n ) { x[i][c] += x[i+1024][c]; }  __syncthreads(); }
    if ( n >  512 ) { if ( i <  512 && i +  512 < n ) { x[i][c] += x[i+ 512][c]; }  __syncthreads(); }
    if ( n >  256 ) { if ( i <  256 && i +  256 < n ) { x[i][c] += x[i+ 256][c]; }  __syncthreads(); }
    if ( n >  128 ) { if ( i <  128 && i +  128 < n ) { x[i][c] += x[i+ 128][c]; }  __syncthreads(); }
    if ( n >   64 ) { if ( i <   64 && i +   64 < n ) { x[i][c] += x[i+  64][c]; }  __syncthreads(); }
    if ( n >   32 ) { if ( i <   32 && i +   32 < n ) { x[i][c] += x[i+  32][c]; }  __syncthreads(); }
    // probably don't need __syncthreads for < 16 threads
    // because of implicit warp level synchronization.
    if ( n >   16 ) { if ( i <   16 && i +   16 < n ) { x[i][c] += x[i+  16][c]; }  __syncthreads(); }
    if ( n >    8 ) { if ( i <    8 && i +    8 < n ) { x[i][c] += x[i+   8][c]; }  __syncthreads(); }
    if ( n >    4 ) { if ( i <    4 && i +    4 < n ) { x[i][c] += x[i+   4][c]; }  __syncthreads(); }
    if ( n >    2 ) { if ( i <    2 && i +    2 < n ) { x[i][c] += x[i+   2][c]; }  __syncthreads(); }
    if ( n >    1 ) { if ( i <    1 && i +    1 < n ) { x[i][c] += x[i+   1][c]; }  __syncthreads(); }
}
// end sum_reduce

//==============================================================================

__global__
void magma_zlarf_kernel( int m, magmaDoubleComplex *v, magmaDoubleComplex *tau,
                         magmaDoubleComplex *c, int ldc, double *xnorm )
{
    if ( !MAGMA_Z_EQUAL(*tau, MAGMA_Z_ZERO) ) {
        const int i = threadIdx.x;
        magmaDoubleComplex *dc = c + blockIdx.x * ldc;

        __shared__ magmaDoubleComplex sum[ BLOCK_SIZE ];
        magmaDoubleComplex lsum;

        /*  w := v' * C  */
        lsum = MAGMA_Z_ZERO;
        for( int j = i; j < m; j += BLOCK_SIZE ){
            if (j==0)
               lsum += MAGMA_Z_MUL( MAGMA_Z_ONE, dc[j] );
            else
               lsum += MAGMA_Z_MUL( MAGMA_Z_CNJG( v[j] ), dc[j] );
        }
        sum[i] = lsum;
        sum_reduce< BLOCK_SIZE >( i, sum );

        /*  C := C - v * w  */
        __syncthreads();
        magmaDoubleComplex z__1 = - MAGMA_Z_CNJG(*tau) * sum[0];
        for( int j = m-i-1; j>=0 ; j -= BLOCK_SIZE ) {
             if (j==0)
                dc[j] += z__1;
             else
                dc[j] += z__1 * v[j];
        }
        __syncthreads();

        /* Adjust the rest of the column norms */
        if (i==0){
            double temp = MAGMA_Z_ABS( dc[0] ) / xnorm[blockIdx.x];
            temp = (temp + 1.) * (1. - temp);
            xnorm[blockIdx.x] = xnorm[blockIdx.x] * sqrt(temp); 
        }
    }
}

//==============================================================================

__global__
void magma_zlarf_smkernel( int m, int n, magmaDoubleComplex *v, magmaDoubleComplex *tau,
                           magmaDoubleComplex *c, int ldc, double *xnorm )
{
    if ( !MAGMA_Z_EQUAL(*tau, MAGMA_Z_ZERO) ) {
        const int i = threadIdx.x, col= threadIdx.y;

        for( int k = col; k < n; k+= BLOCK_SIZEy)
        {
        magmaDoubleComplex *dc = c + k * ldc;

        __shared__ magmaDoubleComplex sum[ BLOCK_SIZEx ][ BLOCK_SIZEy + 1];
        magmaDoubleComplex lsum;

        /*  w := v' * C  */
        lsum = MAGMA_Z_ZERO;
        for( int j = i; j < m; j += BLOCK_SIZEx ){
            if (j==0)
               lsum += MAGMA_Z_MUL( MAGMA_Z_ONE, dc[j] );
            else
               lsum += MAGMA_Z_MUL( MAGMA_Z_CNJG( v[j] ), dc[j] );
        }
        sum[i][col] = lsum;
        sum_reduce_2d< BLOCK_SIZEx >( i, col, sum );

        /*  C := C - v * w  */
        __syncthreads();
        magmaDoubleComplex z__1 = - MAGMA_Z_CNJG(*tau) * sum[0][col];
        for( int j = m-i-1; j>=0 ; j -= BLOCK_SIZEx ) {
             if (j==0)
                dc[j] += z__1;
             else
                dc[j] += z__1 * v[j];
        }
        __syncthreads();

        /* Adjust the rest of the column norms */
        if (i==0){
            double temp = MAGMA_Z_ABS( dc[0] ) / xnorm[k];
            temp = (temp + 1.) * (1. - temp);
            xnorm[k] = xnorm[k] * sqrt(temp);
        }
        }
    }
}

//==============================================================================

/*
    Apply a complex elementary reflector H to a complex M-by-N
    matrix C from the left. H is represented in the form
          H = I - tau * v * v'
    where tau is a complex scalar and v is a complex vector.
    If tau = 0, then H is taken to be the unit matrix.

    To apply H' (the conjugate transpose of H), supply conjg(tau)
    instead tau.

    This routine uses only one SM (block).
 */
extern "C" void
magma_zlarf_sm(int m, int n, magmaDoubleComplex *v, magmaDoubleComplex *tau,
               magmaDoubleComplex *c, int ldc, double *xnorm)
{
    dim3  blocks( 1 );
    dim3 threads( BLOCK_SIZEx, BLOCK_SIZEy );

    magma_zlarf_smkernel<<< blocks, threads, 0, magma_stream >>>( m, n, v, tau, c, ldc, xnorm);
}

//==============================================================================
/*
    Apply a complex elementary reflector H to a complex M-by-N
    matrix C from the left. H is represented in the form
          H = I - tau * v * v'
    where tau is a complex scalar and v is a complex vector.
    If tau = 0, then H is taken to be the unit matrix.

    To apply H' (the conjugate transpose of H), supply conjg(tau) 
    instead tau.

    The norms of v(:, 1:n) are given as input in xnorm(1:n). On exit, the norms
    are adjusted to hold the norms of v(2:m,2:n). This is a difference with the 
    LAPACK's zlarf routine. 
 */

extern "C" magma_int_t
magma_zlarf_gpu(
    magma_int_t m,  magma_int_t n,
    magmaDoubleComplex *v, magmaDoubleComplex *tau,
    magmaDoubleComplex *c,  magma_int_t ldc, double *xnorm)
{
    dim3  blocks( n );
    dim3 threads( BLOCK_SIZE );

    magma_zlarf_kernel<<< blocks, threads, 0, magma_stream >>>( m, v, tau, c, ldc, xnorm);

    // The computation can be done on 1 SM with the following routine.
    // magma_zlarf_sm(m, n, v, tau, c, ldc, xnorm);

    return MAGMA_SUCCESS;
}

//==============================================================================
