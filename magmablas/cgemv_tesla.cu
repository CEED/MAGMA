/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal d

*/
#include "common_magma.h"

#define magmablas_cgemv_tesla magmablas_cgemv

extern "C" void
magmablas_cgemv_tesla(char trans, magma_int_t m, magma_int_t n, 
                      hipFloatComplex alpha, const hipFloatComplex *A, magma_int_t lda, 
                                            const hipFloatComplex *x, magma_int_t incx, 
                      hipFloatComplex beta,  hipFloatComplex       *y, magma_int_t incy) 
{
    hipblasCgemv(trans, m, n, alpha, A, lda, x, incx, beta, y, incy);
}
