/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

*/
#include "common_magma.h"

#define magmablas_cgemv_tesla magmablas_cgemv

extern "C" void
magmablas_cgemv_tesla(char trans, int m, int n, 
                      hipFloatComplex alpha, hipFloatComplex *A, int lda, 
                                            hipFloatComplex *x, int incx, 
                      hipFloatComplex beta,  hipFloatComplex *y, int incy) 
{
    hipblasCgemv(trans, m, n, alpha, A, lda, x, incx, beta, y, incy);
}
