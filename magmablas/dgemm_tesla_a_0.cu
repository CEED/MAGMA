#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010
*/

#include <hipblas.h>
#include "magmablas.h"
#include "commonblas_d.h"

extern "C" __global__ void  
dgemm_kernel_a_0(double *C, const double *A, const double *B,
                 int m, int n, int k, 
                 int lda, int ldb, int ldc,
                 double alpha, double beta)
{
        const int tx = threadIdx.x;
        const int ty = threadIdx.y;

	int ibx = blockIdx.x * 64;
	int iby = blockIdx.y *16;

	const int idt = ty * 16 + tx;


	C += ibx +idt +__mul24(iby,ldc);

        ibx = ibx+idt - m  ;
	
	if( (iby+16)>=n) { 
		lda = n-iby;
	}
	else    {
		lda = 16;
	}
	if( ibx >= 0 )
		lda = 0 ;
	else lda = lda ;

	switch(lda){
		case 16:
			C[ 0    ] *=beta;
			C[ 1*ldc] *=beta;
			C[ 2*ldc] *=beta;
			C[ 3*ldc] *=beta;
			C[ 4*ldc] *=beta;
			C[ 5*ldc] *=beta;
			C[ 6*ldc] *=beta;
			C[ 7*ldc] *=beta;
			C[ 8*ldc] *=beta;
			C[ 9*ldc] *=beta;
			C[10*ldc] *=beta;
			C[11*ldc] *=beta;
			C[12*ldc] *=beta;
			C[13*ldc] *=beta;
			C[14*ldc] *=beta;;
			C[15*ldc] *=beta;
			break;
		case 0:
			break;
		case 15:
			C[ 0    ] *=beta;
			C[ 1*ldc] *=beta;
			C[ 2*ldc] *=beta;
			C[ 3*ldc] *=beta;
			C[ 4*ldc] *=beta;
			C[ 5*ldc] *=beta;
			C[ 6*ldc] *=beta;
			C[ 7*ldc] *=beta;
			C[ 8*ldc] *=beta;
			C[ 9*ldc] *=beta;
			C[10*ldc] *=beta;
			C[11*ldc] *=beta;
			C[12*ldc] *=beta;
			C[13*ldc] *=beta;
			C[14*ldc] *=beta;
			break;
		case 14:
			C[ 0    ] *=beta;
			C[ 1*ldc] *=beta;
			C[ 2*ldc] *=beta;
			C[ 3*ldc] *=beta;
			C[ 4*ldc] *=beta;
			C[ 5*ldc] *=beta;
			C[ 6*ldc] *=beta;
			C[ 7*ldc] *=beta;
			C[ 8*ldc] *=beta;
			C[ 9*ldc] *=beta;
			C[10*ldc] *=beta;
			C[11*ldc] *=beta;
			C[12*ldc] *=beta;
			C[13*ldc] *=beta;
			break;
		case 13:
			C[ 0    ] *=beta;
			C[ 1*ldc] *=beta;
			C[ 2*ldc] *=beta;
			C[ 3*ldc] *=beta;
			C[ 4*ldc] *=beta;
			C[ 5*ldc] *=beta;
			C[ 6*ldc] *=beta;
			C[ 7*ldc] *=beta;
			C[ 8*ldc] *=beta;
			C[ 9*ldc] *=beta;
			C[10*ldc] *=beta;
			C[11*ldc] *=beta;
			C[12*ldc] *=beta;
			break;
		case 12:
			C[ 0    ] *=beta;
			C[ 1*ldc] *=beta;
			C[ 2*ldc] *=beta;
			C[ 3*ldc] *=beta;
			C[ 4*ldc] *=beta;
			C[ 5*ldc] *=beta;
			C[ 6*ldc] *=beta;
			C[ 7*ldc] *=beta;
			C[ 8*ldc] *=beta;
			C[ 9*ldc] *=beta;
			C[10*ldc] *=beta;
			C[11*ldc] *=beta;
			break;
		case 11:
			C[ 0    ] *=beta;
			C[ 1*ldc] *=beta;
			C[ 2*ldc] *=beta;
			C[ 3*ldc] *=beta;
			C[ 4*ldc] *=beta;
			C[ 5*ldc] *=beta;
			C[ 6*ldc] *=beta;
			C[ 7*ldc] *=beta;
			C[ 8*ldc] *=beta;
			C[ 9*ldc] *=beta;
			C[10*ldc] *=beta;
			break;
		case 10:
			C[ 0    ] *=beta;
			C[ 1*ldc] *=beta;
			C[ 2*ldc] *=beta;
			C[ 3*ldc] *=beta;
			C[ 4*ldc] *=beta;
			C[ 5*ldc] *=beta;
			C[ 6*ldc] *=beta;
			C[ 7*ldc] *=beta;
			C[ 8*ldc] *=beta;
			C[ 9*ldc] *=beta;
			break;
		case 9:
			C[ 0    ] *=beta;
			C[ 1*ldc] *=beta;
			C[ 2*ldc] *=beta;
			C[ 3*ldc] *=beta;
			C[ 4*ldc] *=beta;
			C[ 5*ldc] *=beta;
			C[ 6*ldc] *=beta;
			C[ 7*ldc] *=beta;
			C[ 8*ldc] *=beta;
			break;
		case 8:
			C[ 0    ] *=beta;
			C[ 1*ldc] *=beta;
			C[ 2*ldc] *=beta;
			C[ 3*ldc] *=beta;
			C[ 4*ldc] *=beta;
			C[ 5*ldc] *=beta;
			C[ 6*ldc] *=beta;
			C[ 7*ldc] *=beta;
			break;
		case 7:
			C[ 0    ] *=beta;
			C[ 1*ldc] *=beta;
			C[ 2*ldc] *=beta;
			C[ 3*ldc] *=beta;
			C[ 4*ldc] *=beta;
			C[ 5*ldc] *=beta;
			C[ 6*ldc] *=beta;
			break;
		case 6:
			C[ 0    ] *=beta;
			C[ 1*ldc] *=beta;
			C[ 2*ldc] *=beta;
			C[ 3*ldc] *=beta;
			C[ 4*ldc] *=beta;
			C[ 5*ldc] *=beta;
			break;
		case 5:
			C[ 0    ] *=beta;
			C[ 1*ldc] *=beta;
			C[ 2*ldc] *=beta;
			C[ 3*ldc] *=beta;
			C[ 4*ldc] *=beta;
			break;
		case 4:
			C[ 0    ] *=beta;
			C[ 1*ldc] *=beta;
			C[ 2*ldc] *=beta;
			C[ 3*ldc] *=beta;
			break;
		case 3:
			C[ 0    ] *=beta;
			C[ 1*ldc] *=beta;
			C[ 2*ldc] *=beta;
			break;
		case 2:
			C[ 0    ] *=beta;
			C[ 1*ldc] *=beta;
			break;
		case 1:
			C[ 0    ] *=beta;
			break;
	}
}

extern "C" void
magmablas_dgemm_kernel_a_0(double *C, const double *A, const double *B,
                           int m, int n, int k, 
                           int lda, int ldb, int ldc, 
                           double alpha, double beta)
{
        dim3 threads( 16, 4 );
        dim3 grid(m/64+(m%64!=0),n/16+(n%16!=0));
        dgemm_kernel_a_0<<< grid, threads >>>(C, A, B, 
                                              m, n, k, 
                                              lda , ldb , ldc , 
                                              alpha , beta ) ;
}
