/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date
*/

#include "common_magma.h"

magma_queue_t magma_stream = 0;


/**
    Purpose
    -------
    magmablasSetKernelStream sets the CUDA stream that all MAGMA BLAS and
    CUBLAS routines use.
    
    In a multi-threaded application, be careful to avoid race conditions
    when using this. For instance, if calls are executed in this order:
    
    @verbatim
        thread 1                            thread 2
        ------------------------------      ------------------------------
    1.  magmablasSetKernelStream( s1 )         
    2.                                      magmablasSetKernelStream( s2 )
    3.  magma_dgemm( ... )
    4.                                      magma_dgemm( ... )
    @endverbatim
    
    both magma_dgemm would occur on stream s2. A lock should be used to prevent
    this, so the dgemm in thread 1 uses stream s1, and the dgemm in thread 2
    uses s2:
    
    @verbatim
        thread 1                            thread 2
        ------------------------------      ------------------------------
    1.  lock()                                  
    2.  magmablasSetKernelStream( s1 )          
    3.  magma_dgemm( ... )                      
    4.  unlock()                                
    5.                                      lock()
    6.                                      magmablasSetKernelStream( s2 )
    7.                                      magma_dgemm( ... )
    8.                                      unlock()
    @endverbatim
    
    Most BLAS calls in MAGMA, such as magma_dgemm, are asynchronous, so the lock
    will only have to wait until dgemm is queued, not until it is finished.
    
    Arguments
    ---------
    @param[in]
    stream  magma_queue_t
            The CUDA stream.

    @ingroup magma_util
    ********************************************************************/
extern "C"
hipblasStatus_t magmablasSetKernelStream( magma_queue_t stream )
{
    magma_stream = stream;
    return cublasSetKernelStream( stream );
}


/**
    Purpose
    -------
    magmablasGetKernelStream gets the CUDA stream that all MAGMA BLAS
    routines use.

    Arguments
    ---------
    @param[out]
    stream  magma_queue_t
            The CUDA stream.

    @ingroup magma_util
    ********************************************************************/
extern "C"
hipblasStatus_t magmablasGetKernelStream( magma_queue_t *stream )
{
    *stream = magma_stream;
    return HIPBLAS_STATUS_SUCCESS;
}
