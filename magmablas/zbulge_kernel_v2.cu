#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2011      The University of Tennessee and The University
 *                         of Tennessee Research Foundation.  All rights
 *                         reserved.
 *
 *
 *     @author Azzam Haidar
 *     @author Stan Tomov
 *
 *     @precisions normal z -> s d c
 *
 */

#include "common_magma.h"
#include "magma_bulge.h"
#include <cblas.h>

//#include "magma_zbulgeinc.h"
// === Define what BLAS to use ============================================
#define PRECISION_z

// nb is assumed to be < BLOCK_SIZE; if not, increase BLOCK_SIZE
// NOTE THAT BLOCK_SIZE should be equal BLOCK_SIZEx*BLOCK_SIZEy
// and BLOCK_SIZEy <= BLOCK_SIZEx

// Requested SHARED MEMORY GPU HAS 48KB
// +MAX_NB for zlarfg could be added in case the compiler do inline of 2 calls.
// for zlarfxsym:  (SIZEx*(SIZEx+1)) + (MAX_NB*(SIZEy+1)) + MAX_NB + 1 
//         ==> for double precision if MAX_NB=128: block_x=64 ==> block_y <=8  ==> block <=512
//         ==> for double precision if MAX_NB=128: block_x=32 ==> block_y <=8  ==> block <=512

// for zlarfrgl:   (SIZEx*(SIZEx+1)) + (SIZEx*(SIZEy+1)) + SIZEx +- MAX_NB for zlarfg: used when NB<SIZEx ==> for block_x=64 it allow basically block_y=upto 16
// for zlarfr: (BLKD1*(BLKD2+1)) + (BLKD1*(MAX_NB+1)) + MAX_NB
// for zlarfl: (BLKD1*BLKD2) + (MAX_NB*(BLKD1+1)) + MAX_NB  < zlarfr in case BLKD1<MAX_NB


#define BLOCK_SIZE  512
// MAX NB SHOULD BE ALWAYS powerof 2 and less than BLOCK_SIZE because of sum reduce and maybe other kernel
#define MAX_NB      128

//BLOCK_SIZEx*BLOCK_SIZEy = BLOCK_SIZE
#define BLOCK_SIZEx  64
#define BLOCK_SIZEy  8

//BLKD1_SIZE*BLKD2_SIZE = BLOCK_SIZE
#define BLKD1_SIZE  32 // should always be smaller= than 32 if MAX_NB <128 and less than 16 if MAX_NB 256
#define BLKD2_SIZE  16

 

 
// === End defining what BLAS to use ======================================

extern "C" {

    void magma_zlarfxsym_v2(magma_int_t n, hipDoubleComplex *A, magma_int_t lda, hipDoubleComplex *V, hipDoubleComplex *TAU, hipDoubleComplex *work);

}

__device__ void zsum_reduce( int n, int i, hipDoubleComplex* x )
{
    __syncthreads();
    if ( n >  128 ) { if ( i <  128 && i +  128 < n ) { x[i] += x[i+ 128]; }  __syncthreads(); }
    if ( n >   64 ) { if ( i <   64 && i +   64 < n ) { x[i] += x[i+  64]; }  __syncthreads(); }
    if ( n >   32 ) { if ( i <   32 && i +   32 < n ) { x[i] += x[i+  32]; }  __syncthreads(); }
    // probably don't need __syncthreads for < 16 threads
    // because of implicit warp level synchronization.
    if ( n >   16 ) { if ( i <   16 && i +   16 < n ) { x[i] += x[i+  16]; }  __syncthreads(); }
    if ( n >    8 ) { if ( i <    8 && i +    8 < n ) { x[i] += x[i+   8]; }  __syncthreads(); }
    if ( n >    4 ) { if ( i <    4 && i +    4 < n ) { x[i] += x[i+   4]; }  __syncthreads(); }
    if ( n >    2 ) { if ( i <    2 && i +    2 < n ) { x[i] += x[i+   2]; }  __syncthreads(); }
    if ( n >    1 ) { if ( i <    1 && i +    1 < n ) { x[i] += x[i+   1]; }  __syncthreads(); }
}


 __device__ void sum_reduce(int n, int i, double* x )
{
    __syncthreads();
    if ( n >  128 ) { if ( i <  128 && i +  128 < n ) { x[i] += x[i+ 128]; }  __syncthreads(); }
    if ( n >   64 ) { if ( i <   64 && i +   64 < n ) { x[i] += x[i+  64]; }  __syncthreads(); }
    if ( n >   32 ) { if ( i <   32 && i +   32 < n ) { x[i] += x[i+  32]; }  __syncthreads(); }
    // probably don't need __syncthreads for < 16 threads
    // because of implicit warp level synchronization.
    if ( n >   16 ) { if ( i <   16 && i +   16 < n ) { x[i] += x[i+  16]; }  __syncthreads(); }
    if ( n >    8 ) { if ( i <    8 && i +    8 < n ) { x[i] += x[i+   8]; }  __syncthreads(); }
    if ( n >    4 ) { if ( i <    4 && i +    4 < n ) { x[i] += x[i+   4]; }  __syncthreads(); }
    if ( n >    2 ) { if ( i <    2 && i +    2 < n ) { x[i] += x[i+   2]; }  __syncthreads(); }
    if ( n >    1 ) { if ( i <    1 && i +    1 < n ) { x[i] += x[i+   1]; }  __syncthreads(); }
}

__device__ void sum_rowreduce_1d( int n, int thxid, int thyid, hipDoubleComplex x[][BLOCK_SIZEy+1] )// +1 is bad here but this function is used to reduce only 1 column so its OK
{
    __syncthreads();
    if ( n > 1024 ) { if ( thxid < 1024 && thxid + 1024 < n ) { x[thxid][thyid] += x[thxid+1024][thyid]; }  __syncthreads(); }
    if ( n >  512 ) { if ( thxid <  512 && thxid +  512 < n ) { x[thxid][thyid] += x[thxid+ 512][thyid]; }  __syncthreads(); }
    if ( n >  256 ) { if ( thxid <  256 && thxid +  256 < n ) { x[thxid][thyid] += x[thxid+ 256][thyid]; }  __syncthreads(); }
    if ( n >  128 ) { if ( thxid <  128 && thxid +  128 < n ) { x[thxid][thyid] += x[thxid+ 128][thyid]; }  __syncthreads(); }
    if ( n >   64 ) { if ( thxid <   64 && thxid +   64 < n ) { x[thxid][thyid] += x[thxid+  64][thyid]; }  __syncthreads(); }
    if ( n >   32 ) { if ( thxid <   32 && thxid +   32 < n ) { x[thxid][thyid] += x[thxid+  32][thyid]; }  __syncthreads(); }
    // probably don't need __syncthreads for < 16 threads
    // because of implicit warp level synchronization.
    if ( n >   16 ) { if ( thxid <   16 && thxid +   16 < n ) { x[thxid][thyid] += x[thxid+  16][thyid]; }  __syncthreads(); }
    if ( n >    8 ) { if ( thxid <    8 && thxid +    8 < n ) { x[thxid][thyid] += x[thxid+   8][thyid]; }  __syncthreads(); }
    if ( n >    4 ) { if ( thxid <    4 && thxid +    4 < n ) { x[thxid][thyid] += x[thxid+   4][thyid]; }  __syncthreads(); }
    if ( n >    2 ) { if ( thxid <    2 && thxid +    2 < n ) { x[thxid][thyid] += x[thxid+   2][thyid]; }  __syncthreads(); }
    if ( n >    1 ) { if ( thxid <    1 && thxid +    1 < n ) { x[thxid][thyid] += x[thxid+   1][thyid]; }  __syncthreads(); }
}
__device__ void sum_colreduce_2d(int ncol, int thxid, int thyid, hipDoubleComplex x[BLOCK_SIZEx][BLOCK_SIZEy+1] )
{
    __syncthreads();
    if ( ncol > 1024 ) { if ( thyid < 1024 && thyid + 1024 < ncol ) { x[thxid][thyid] += x[thxid][thyid+1024]; }  __syncthreads(); }
    if ( ncol >  512 ) { if ( thyid <  512 && thyid +  512 < ncol ) { x[thxid][thyid] += x[thxid][thyid+ 512]; }  __syncthreads(); }
    if ( ncol >  256 ) { if ( thyid <  256 && thyid +  256 < ncol ) { x[thxid][thyid] += x[thxid][thyid+ 256]; }  __syncthreads(); }
    if ( ncol >  128 ) { if ( thyid <  128 && thyid +  128 < ncol ) { x[thxid][thyid] += x[thxid][thyid+ 128]; }  __syncthreads(); }
    if ( ncol >   64 ) { if ( thyid <   64 && thyid +   64 < ncol ) { x[thxid][thyid] += x[thxid][thyid+  64]; }  __syncthreads(); }
    if ( ncol >   32 ) { if ( thyid <   32 && thyid +   32 < ncol ) { x[thxid][thyid] += x[thxid][thyid+  32]; }  __syncthreads(); }
    // probably don't need __syncthreads for < 16 threads
    // because of implicit warp level synchronization.
    if ( ncol >   16 ) { if ( thyid <   16 && thyid +   16 < ncol ) { x[thxid][thyid] += x[thxid][thyid+  16]; }  __syncthreads(); }
    if ( ncol >    8 ) { if ( thyid <    8 && thyid +    8 < ncol ) { x[thxid][thyid] += x[thxid][thyid+   8]; }  __syncthreads(); }
    if ( ncol >    4 ) { if ( thyid <    4 && thyid +    4 < ncol ) { x[thxid][thyid] += x[thxid][thyid+   4]; }  __syncthreads(); }
    if ( ncol >    2 ) { if ( thyid <    2 && thyid +    2 < ncol ) { x[thxid][thyid] += x[thxid][thyid+   2]; }  __syncthreads(); }
    if ( ncol >    1 ) { if ( thyid <    1 && thyid +    1 < ncol ) { x[thxid][thyid] += x[thxid][thyid+   1]; }  __syncthreads(); }
}
__device__ void sum_colreduce_2de(int mrow, int ncol, int thxid, int thyid, int blkx, hipDoubleComplex x[][BLOCK_SIZEy+1] )
{
    __syncthreads();
    for(int k=0; k<mrow; k+=blkx){
        if ( ncol > 1024 ) { if ( thyid < 1024 && thyid + 1024 < ncol ) { x[thxid+k][thyid] += x[thxid+k][thyid+1024]; }  __syncthreads(); }
        if ( ncol >  512 ) { if ( thyid <  512 && thyid +  512 < ncol ) { x[thxid+k][thyid] += x[thxid+k][thyid+ 512]; }  __syncthreads(); }
        if ( ncol >  256 ) { if ( thyid <  256 && thyid +  256 < ncol ) { x[thxid+k][thyid] += x[thxid+k][thyid+ 256]; }  __syncthreads(); }
        if ( ncol >  128 ) { if ( thyid <  128 && thyid +  128 < ncol ) { x[thxid+k][thyid] += x[thxid+k][thyid+ 128]; }  __syncthreads(); }
        if ( ncol >   64 ) { if ( thyid <   64 && thyid +   64 < ncol ) { x[thxid+k][thyid] += x[thxid+k][thyid+  64]; }  __syncthreads(); }
        if ( ncol >   32 ) { if ( thyid <   32 && thyid +   32 < ncol ) { x[thxid+k][thyid] += x[thxid+k][thyid+  32]; }  __syncthreads(); }
        // probably don't need __syncthreads for < 16 threads
        // because of implicit warp level synchronization.
        if ( ncol >   16 ) { if ( thyid <   16 && thyid +   16 < ncol ) { x[thxid+k][thyid] += x[thxid+k][thyid+  16]; }  __syncthreads(); }
        if ( ncol >    8 ) { if ( thyid <    8 && thyid +    8 < ncol ) { x[thxid+k][thyid] += x[thxid+k][thyid+   8]; }  __syncthreads(); }
        if ( ncol >    4 ) { if ( thyid <    4 && thyid +    4 < ncol ) { x[thxid+k][thyid] += x[thxid+k][thyid+   4]; }  __syncthreads(); }
        if ( ncol >    2 ) { if ( thyid <    2 && thyid +    2 < ncol ) { x[thxid+k][thyid] += x[thxid+k][thyid+   2]; }  __syncthreads(); }
        if ( ncol >    1 ) { if ( thyid <    1 && thyid +    1 < ncol ) { x[thxid+k][thyid] += x[thxid+k][thyid+   1]; }  __syncthreads(); }
    }
}



__device__ void sum_rowreduce_2dn( int nrow, int thxid, int thyid, hipDoubleComplex x[BLKD2_SIZE][BLKD1_SIZE] )
{
    __syncthreads();
    if ( nrow > 1024 ) { if ( thxid < 1024 && thxid + 1024 < nrow ) { x[thxid][thyid] += x[thxid+1024][thyid]; }  __syncthreads(); }
    if ( nrow >  512 ) { if ( thxid <  512 && thxid +  512 < nrow ) { x[thxid][thyid] += x[thxid+ 512][thyid]; }  __syncthreads(); }
    if ( nrow >  256 ) { if ( thxid <  256 && thxid +  256 < nrow ) { x[thxid][thyid] += x[thxid+ 256][thyid]; }  __syncthreads(); }
    if ( nrow >  128 ) { if ( thxid <  128 && thxid +  128 < nrow ) { x[thxid][thyid] += x[thxid+ 128][thyid]; }  __syncthreads(); }
    if ( nrow >   64 ) { if ( thxid <   64 && thxid +   64 < nrow ) { x[thxid][thyid] += x[thxid+  64][thyid]; }  __syncthreads(); }
    if ( nrow >   32 ) { if ( thxid <   32 && thxid +   32 < nrow ) { x[thxid][thyid] += x[thxid+  32][thyid]; }  __syncthreads(); }
    // probably don't need __syncthreads for < 16 threads
    // because of implicit warp level synchronization.
    if ( nrow >   16 ) { if ( thxid <   16 && thxid +   16 < nrow ) { x[thxid][thyid] += x[thxid+  16][thyid]; }  __syncthreads(); }
    if ( nrow >    8 ) { if ( thxid <    8 && thxid +    8 < nrow ) { x[thxid][thyid] += x[thxid+   8][thyid]; }  __syncthreads(); }
    if ( nrow >    4 ) { if ( thxid <    4 && thxid +    4 < nrow ) { x[thxid][thyid] += x[thxid+   4][thyid]; }  __syncthreads(); }
    if ( nrow >    2 ) { if ( thxid <    2 && thxid +    2 < nrow ) { x[thxid][thyid] += x[thxid+   2][thyid]; }  __syncthreads(); }
    if ( nrow >    1 ) { if ( thxid <    1 && thxid +    1 < nrow ) { x[thxid][thyid] += x[thxid+   1][thyid]; }  __syncthreads(); }
}

__device__ void sum_colreduce_2dn(int ncol, int thxid, int thyid, hipDoubleComplex x[BLKD1_SIZE][BLKD2_SIZE+1] )
{
    __syncthreads();
    if ( ncol > 1024 ) { if ( thyid < 1024 && thyid + 1024 < ncol ) { x[thxid][thyid] += x[thxid][thyid+1024]; }  __syncthreads(); }
    if ( ncol >  512 ) { if ( thyid <  512 && thyid +  512 < ncol ) { x[thxid][thyid] += x[thxid][thyid+ 512]; }  __syncthreads(); }
    if ( ncol >  256 ) { if ( thyid <  256 && thyid +  256 < ncol ) { x[thxid][thyid] += x[thxid][thyid+ 256]; }  __syncthreads(); }
    if ( ncol >  128 ) { if ( thyid <  128 && thyid +  128 < ncol ) { x[thxid][thyid] += x[thxid][thyid+ 128]; }  __syncthreads(); }
    if ( ncol >   64 ) { if ( thyid <   64 && thyid +   64 < ncol ) { x[thxid][thyid] += x[thxid][thyid+  64]; }  __syncthreads(); }
    if ( ncol >   32 ) { if ( thyid <   32 && thyid +   32 < ncol ) { x[thxid][thyid] += x[thxid][thyid+  32]; }  __syncthreads(); }
    // probably don't need __syncthreads for < 16 threads
    // because of implicit warp level synchronization.
    if ( ncol >   16 ) { if ( thyid <   16 && thyid +   16 < ncol ) { x[thxid][thyid] += x[thxid][thyid+  16]; }  __syncthreads(); }
    if ( ncol >    8 ) { if ( thyid <    8 && thyid +    8 < ncol ) { x[thxid][thyid] += x[thxid][thyid+   8]; }  __syncthreads(); }
    if ( ncol >    4 ) { if ( thyid <    4 && thyid +    4 < ncol ) { x[thxid][thyid] += x[thxid][thyid+   4]; }  __syncthreads(); }
    if ( ncol >    2 ) { if ( thyid <    2 && thyid +    2 < ncol ) { x[thxid][thyid] += x[thxid][thyid+   2]; }  __syncthreads(); }
    if ( ncol >    1 ) { if ( thyid <    1 && thyid +    1 < ncol ) { x[thxid][thyid] += x[thxid][thyid+   1]; }  __syncthreads(); }
}


///////////////////////////////////////////////////////////
//// add -1 because of C
#define dA(i,j)   &(dA[((i)-(j)) + ldda*((j)-1)])
#define dAC(i,j)   &(dA[(i) + ldda*(j)])

#define dV(i)     &(dV[(i)])
#define dTAU(i)   &(dTAU[(i)])

__device__ void zlarfxsym_v2(magma_int_t n, 
                             hipDoubleComplex *dA, magma_int_t ldda, 
                             hipDoubleComplex *dV, hipDoubleComplex *dTAU) 
{
/*
    WORK (workspace) double complex array, dimension N
*/

    magma_int_t j,nint,gbrow,gbcol,blkjcol;
    hipDoubleComplex dtmp     = MAGMA_Z_ZERO;
    hipDoubleComplex c_half   =  MAGMA_Z_HALF;
    const int myrow = threadIdx.x % BLOCK_SIZEx, mycol= threadIdx.x / BLOCK_SIZEx,  thid = threadIdx.x;

    __shared__ hipDoubleComplex loctau;
    __shared__ hipDoubleComplex locv[ MAX_NB ];
    __shared__ hipDoubleComplex loca[ BLOCK_SIZEx ][ BLOCK_SIZEx+1 ];
    __shared__ hipDoubleComplex sum[ MAX_NB ][ BLOCK_SIZEy+1];

    __syncthreads();
    if(thid<n)
       locv[thid] = dV[thid];
    if(thid==0) loctau     = dTAU[0];
    __syncthreads();
   
    // initialize all the column of sum (BLOCK_SIZEy col) to zero
    for( j = myrow; j < MAX_NB; j+= BLOCK_SIZEx)
    {
        sum[j][mycol] = MAGMA_Z_ZERO;
    }
    /*
    if(thid<MAX_NB){
       for( j = 0; j < BLOCK_SIZEy; j++){
            sum[thid][j] = MAGMA_Z_ZERO;
       }  
    }
    */
    __syncthreads();



    /* 
        X = tau A V 
        blasf77_zhemv("L", &n, TAU, A, &lda, V, &ione, &c_zero, work, &ione);
    */

    j = n%BLOCK_SIZEx;
    nint = j == 0 ? n : n - j; 
    //printf("me %d nint %d\n",thid,myrow, nint);
    
    // go over the blocki (vertical down) excluding the last block in case of padding required
    for(gbrow = myrow; gbrow<nint; gbrow+=BLOCK_SIZEx){
        //if(thid==0)printf("%d  ===============  HELLO FROM THE MAIN LOOP  ================= \n", thid); __syncthreads();

        // go over the blockj (horizontal left to right)
        // excluding diagonal block which is treated after it
        blkjcol = (gbrow/BLOCK_SIZEx)*BLOCK_SIZEx;
        for( gbcol = 0; gbcol<blkjcol; gbcol+=BLOCK_SIZEx){
            //if(thid==0)printf("%d ===============> MAIN LOOP  offDIAG BLOCK gbrow %d   gbcol %d \n", thid, gbrow, gbcol); __syncthreads();

            // for non diag block, copy the matrix to shared,
            // and directly do the first GEMV (threads horizontal reading)  
            // then another loop will do the second GEMV 
            // with the transpose (vertical reading)  
            for( j = mycol; j < BLOCK_SIZEx; j+= BLOCK_SIZEy)
            {
                loca[myrow][j] = *(dAC(gbrow, (gbcol+j))) ;  
                sum[gbrow][mycol] += loca[myrow][j] * locv[gbcol+j];
            }
            __syncthreads();
            for( j = mycol; j < BLOCK_SIZEx; j+= BLOCK_SIZEy)
            {
                sum[gbcol+myrow][mycol] += MAGMA_Z_CNJG(loca[j][myrow]) * locv[blkjcol+j];
            }
            __syncthreads();
        }
        // the diagonal block
        gbcol = blkjcol;
        //if(thid==0)printf("%d  ===============>  DIAG BLOCK  myrow %d mycol %d gbrow %d   gbcol %d \n", thid,myrow, mycol, gbrow, gbcol); __syncthreads();
        for( j = mycol; j <= myrow; j+= BLOCK_SIZEy)
        {
            loca[myrow][j] = *(dAC(gbrow, (gbcol+j))) ; 
            loca[j][myrow] =  MAGMA_Z_CNJG( loca[myrow][j] );
        }
        __syncthreads();

        for( j = mycol; j < BLOCK_SIZEx; j+= BLOCK_SIZEy)
        {
            sum[gbrow][mycol] += loca[myrow][j] * locv[gbcol+j];
        }
        __syncthreads();
    }
    // In case where a padding should exist and is not, so let do the last block in case of its size < BLOCK_SIZEx independently


    if(nint<n){
        gbrow = nint+myrow;
        blkjcol = (gbrow/BLOCK_SIZEx)*BLOCK_SIZEx;    
        for( gbcol = 0; gbcol<blkjcol; gbcol+=BLOCK_SIZEx){
            //printf("%d  LAST LOOP  gbrow %d   gbcol %d \n", thid, gbrow, gbcol);
            if(gbrow<n){
                for( j = mycol; j < BLOCK_SIZEx; j+= BLOCK_SIZEy)
                {
                    loca[myrow][j] = *(dAC(gbrow, (gbcol+j))) ;  
                    sum[gbrow][mycol] += loca[myrow][j] * locv[gbcol+j];
                }
            }
            __syncthreads();

            for( j = mycol; j < n-nint; j+= BLOCK_SIZEy)
            {
                sum[gbcol+myrow][mycol] += MAGMA_Z_CNJG(loca[j][myrow]) * locv[blkjcol+j];
            }
            __syncthreads();
        }
        // the diagonal block
        gbcol = blkjcol;
        //printf("%d  LAST DIAG BLOCK  gbrow %d   gbcol %d \n", thid, gbrow, gbcol);
        if(gbrow<n){
            for( j = mycol; j <= myrow; j+= BLOCK_SIZEy)
            {
                loca[myrow][j] = *(dAC(gbrow, (gbcol+j))) ; 
                loca[j][myrow] =  MAGMA_Z_CNJG( loca[myrow][j] );
            }
        }
        __syncthreads();
        if(gbrow<n){
            for( j = mycol; j <  n-nint; j+= BLOCK_SIZEy)
            {
                sum[gbrow][mycol] += loca[myrow][j] * locv[gbcol+j];
            }
        }
        __syncthreads();
    }
    
    // The result of the GEMV is now in sum[1:n][BLOCK_SIZEy]
    // and need to be summed over the BLOCK_SIZEy.
    // each thread go over the BLOCK_SIZEy and summ it to its sum[thid][0]
    //sum_colreduce_2de(MAX_NB, BLOCK_SIZEy, myrow, mycol, BLOCK_SIZEx, sum);
    if(thid<n){
        
        for( j = 1; j < BLOCK_SIZEy; j++){
            sum[thid][0] += sum[thid][j];
        }
        
        sum[thid][1] = loctau * sum[thid][0];
        /* compute dtmp= X'*V */
        sum[thid][0] = MAGMA_Z_CNJG( sum[thid][1] ) * locv[thid];
    }
    sum_rowreduce_1d(n, thid, 0, sum);

    if(thid<n){
        /* compute 1/2 X'*V*t = 1/2*dtmp*tau  */
        dtmp = sum[0][0] * c_half * loctau;
        /*
           compute W=X-1/2VX'Vt = X - dtmp*V 
           blasf77_zaxpy(&n, &dtmp, V, &ione, work, &ione); 
        */
        sum[thid][1] -= dtmp * locv[thid]; 
    }
    __syncthreads();    

//=======================================================================
//=======================================================================
//=======================================================================
    // still need to be optimized using all thread 2D writing back data same as i read it.
//=======================================================================
//=======================================================================
//=======================================================================
    /* 
       performs the symmetric rank 2 operation A := alpha*x*y' + alpha*y*x' + A 
       blasf77_zher2("L", &n, &c_neg_one, work, &ione, V, &ione, A, &lda);
    */
    if(thid<n){
        if( n <= BLOCK_SIZEx){ // meaning that the matrix is fully loaded into shared so use it
            for(j=0; j<=thid; j++)
               *dAC(thid, j) = loca[thid][j] - sum[thid][1]*MAGMA_Z_CNJG( locv[j] ) - locv[thid]*MAGMA_Z_CNJG( sum[j][1] ); 
        }else{        
            for(j=0; j<=thid; j++)
               *dAC(thid, j) -= sum[thid][1]*MAGMA_Z_CNJG( locv[j] ) + locv[thid]*MAGMA_Z_CNJG( sum[j][1] ); 
        }
    }
    



    // synch the routine
    __syncthreads();

}

///////////////////////////////////////////////////////////
//                  TYPE 1-BAND Householder
///////////////////////////////////////////////////////////
__device__ void zlarfg(int n, hipDoubleComplex *dA, hipDoubleComplex *dx,
                       hipDoubleComplex *dtau)
{
    const int i = threadIdx.x;
    __shared__ hipDoubleComplex scale;
    __shared__ double dsum[ MAX_NB ], beta;
    hipDoubleComplex alpha;

#if (defined(PRECISION_s) || defined(PRECISION_d))
#else
    double alphar;
    __shared__ double alphai;
#endif



    __syncthreads();
#if (defined(PRECISION_s) || defined(PRECISION_d))
    if( n <= 1 ) {
#else
    if( n <= 0 ) {
#endif
        *dtau = MAGMA_Z_ZERO;
        return;
    }


    // PAY ATTENTION ALL THREADS CAM HERE BUT DSUM IS OF SIZE MAX_NB SO NOT ALL SHOULD WRITE IT
    // MAX NB SHOULD BE ALWAYS power OF 2
    if(i<MAX_NB)
        dsum[i] = MAGMA_D_ZERO;

    /* Compute the norm of dx
      XNORM = DZNRM2( N-1, X, INCX )
    */
    if (i<n-1){
#if (defined(PRECISION_s) || defined(PRECISION_d))
         {
         double re = dx[i];
         dsum[i] = re*re;
         }
#else
         {
         double re = MAGMA_Z_REAL(dx[i]), im = MAGMA_Z_IMAG(dx[i]);
         dsum[i] = re*re + im*im;
         }
#endif
    }
    // we need a sync here but because sum_reduce has a sync implicitly at the top so we comment it
    //__syncthreads();
    sum_reduce( n-1, i, dsum );


    if ( i == 0 ) {
    alpha = *dA;
#if (defined(PRECISION_s) || defined(PRECISION_d))
    beta = sqrt(dsum[0]);
#else
    alphar = MAGMA_Z_REAL(alpha);
    alphai = MAGMA_Z_IMAG(alpha);
    if ( n == 1 )
        beta = MAGMA_D_ZERO;
    else
        beta = sqrt(dsum[0]);
#endif 
    }
    __syncthreads();



#if (defined(PRECISION_s) || defined(PRECISION_d))
    if( beta == 0) {
#else
    if( beta == 0 && alphai == 0) {
#endif
        *dtau = MAGMA_Z_ZERO;
        return;
    }

    if ( i == 0 ) {
#if (defined(PRECISION_s) || defined(PRECISION_d))
            beta  = beta*beta + alpha*alpha;
            beta  = sqrt(beta);
            beta  = -copysign( beta, alpha );

            // todo: deal with badly scaled vectors (see lapack's larfg)
            *dtau = (beta - alpha) / beta;
            *dA = beta;

            scale = 1. / (alpha - beta);
#else
            beta  = beta*beta + alphar*alphar + alphai*alphai;
            beta  = sqrt(beta);
            beta  = -copysign( beta, alphar );

            // todo: deal with badly scaled vectors (see lapack's larfg)
            *dtau = MAGMA_Z_MAKE((beta - alphar)/beta, -alphai/beta);

            *dA = MAGMA_Z_MAKE(beta, 0.);

            alpha = MAGMA_Z_MAKE( MAGMA_Z_REAL(alpha) - beta, MAGMA_Z_IMAG(alpha));
            scale = MAGMA_Z_DIV( MAGMA_Z_ONE, alpha);
#endif
    }

    // scale x
    __syncthreads();
    if ( i < n-1)
        dx[i] = MAGMA_Z_MUL(dx[i], scale);



    // synch the routine
    __syncthreads();
    
}


__global__
void magma_ztrdtype1cbHLsym_withQ_v2_gpu_kernel(hipDoubleComplex *dA, int ldda,
                                                hipDoubleComplex *dV, hipDoubleComplex *dTAU,
                                                int st, int len)
{
       const int thid = threadIdx.x;

          /*
             V(0)  = c_one;
             cblas_zcopy(len-1, A(st+1, st-1), ione, V(1), ione);
             memset(A(st+1, st-1), 0, (len-1)*sizeof(hipDoubleComplex));
          */
          if (thid==0){
             dV[0] = MAGMA_Z_ONE;
          } else if(thid < len){
             dV[thid] = *dA(st+thid, st-1);
             *dA(st+thid, st-1) = MAGMA_Z_ZERO;
          }
       
          /*
             Eliminate the col  at st-1
             lapackf77_zlarfg( &len, A(st, st-1), V(1), &ione, TAU );
          */
          zlarfg(len, dA(st,st-1), dV(1), dTAU);

          /*
             apply left and right on A(st:ed,st:ed)
             magma_zlarfxsym_v2(len, A(st,st), lda-1, V, TAU, work);
          */
          zlarfxsym_v2(len, dA(st,st), ldda-1, dV, dTAU);
}

extern "C" void
magma_ztrdtype1cbHLsym_withQ_v2_gpu(magma_int_t n, magma_int_t nb, 
                                    hipDoubleComplex *dA, magma_int_t ldda, 
                                    hipDoubleComplex *dV, magma_int_t lddv, 
                                    hipDoubleComplex *dTAU,
                                    magma_int_t st, magma_int_t ed, magma_int_t sweep, 
                                    magma_int_t Vblksiz, hipDoubleComplex *dwork) 
{
/*
    WORK (workspace) double complex array, dimension N
*/
    magma_int_t vpos, taupos, len;
    //magma_int_t lddx = ldda-1;

    if (nb > BLOCK_SIZE)
       printf("magma_ztrdtype1cbHLsym_withQ_v2_gpu: BLOCK_SIZE should be > %d\n", nb);
 
    magma_bulge_findVTAUpos(n, nb, Vblksiz, sweep-1, st-1, lddv, &vpos, &taupos);
    //printf("voici vpos %d taupos %d  tpos %d  blkid %d \n", vpos, taupos, tpos, blkid);

    len = ed-st+1;

    /* === Compute the following using one multiprocessor with BLOCK_SIZE threads ===
       *V(vpos)  = c_one;
       cblas_zcopy(len-1, A(st+1, st-1), ione, V(vpos+1), ione);
       memset(A(st+1, st-1), 0, (len-1)*sizeof(hipDoubleComplex));

       // Eliminate the col  at st-1 
       lapackf77_zlarfg( &len, A(st, st-1), V(vpos+1), &ione, TAU(taupos) );

       // apply left and right on A(st:ed,st:ed)
       magma_zlarfxsym_v2(len, A(st,st), lda-1, V(vpos), TAU(taupos), work);
    */ 
    magma_ztrdtype1cbHLsym_withQ_v2_gpu_kernel<<<1, BLOCK_SIZE>>>(dA, ldda, dV+vpos,
                                                                  dTAU+taupos,
                                                                  st, len);
}
#undef dA
#undef dAC
#undef dV
#undef dTAU
























///////////////////////////////////////////////////////////
//                  TYPE 2-LPK Householder
///////////////////////////////////////////////////////////
//// add -1 because of C
#define dA(i,j)    &(dA[((i)-(j)) + ldda*((j)-1)])
#define dAC(i,j)   &(dA[(i) + ldda*(j)])

#define   dV(i)     &(dV[(i)])
#define dTAU(i)   &(dTAU[(i)])


/* Applies a complex elementary reflector H to a complex m by n
   matrix C, from the right. H is represented in the form

        H = I - tau * v * v'

   where tau is a complex scalar and v is a complex vector.
   If tau = 0, then H is taken to be the unit matrix              */
//====================================================================================================
//    RIGHT + ZLARFG + LEFT 2D BLOCKED when matrix fir into shared
//====================================================================================================
__device__ void zlarfrgl(int m, int n, hipDoubleComplex *vr, hipDoubleComplex dtaur, 
                       hipDoubleComplex *vl, hipDoubleComplex *dtaul,
                       hipDoubleComplex *c, int ldc)
{

   const int thid = threadIdx.x;
   __shared__ hipDoubleComplex loctau;
   __syncthreads();

   if(thid==0) loctau     = dtaur;
   __syncthreads();


   if ( !MAGMA_Z_EQUAL(loctau, MAGMA_Z_ZERO) ) {

       hipDoubleComplex dalpha = MAGMA_Z_ZERO;
       hipDoubleComplex lsum = MAGMA_Z_ZERO;
       magma_int_t j;
       const int myrow = threadIdx.x % BLOCK_SIZEx, mycol= threadIdx.x / BLOCK_SIZEx;
       __shared__ hipDoubleComplex locv[ BLOCK_SIZEx ];
       __shared__ hipDoubleComplex loca[ BLOCK_SIZEx ][ BLOCK_SIZEx+1 ];
       __shared__ hipDoubleComplex sum[ BLOCK_SIZEx ][ BLOCK_SIZEy+1];

       
       //__shared__ hipDoubleComplex sumrow[ BLOCK_SIZEy ][ BLOCK_SIZEx];

 
       if(thid<n)
           locv[thid] = vr[thid];
       __syncthreads();




       //===========================================
       //        DO the RIGHT UPDATE
       //===========================================
     
       sum[myrow][mycol] = MAGMA_Z_ZERO;
       // read a block of size BLKD1_SIZE x BLKD2_SIZE and do the GEMV
       // w := C  * v  
       if(myrow<m){
           // Read C(m,n) and store it into loca    
           for( j = mycol; j < n; j+= BLOCK_SIZEy)
               loca[myrow][j]     = c[myrow+j*ldc];
           for( j = mycol; j < n; j+= BLOCK_SIZEy)
               sum[myrow][mycol] += loca[myrow][j] * locv[j] ;
       }
       sum_colreduce_2d(BLOCK_SIZEy, myrow, mycol, sum);
     
       //  C := C - tau * w * v' 
       if(myrow<m){
           lsum = -loctau * sum[myrow][0];
           for( j = mycol; j < n; j+= BLOCK_SIZEy)
               loca[myrow][j] += lsum * MAGMA_Z_CNJG( locv[j] );
       }
       __syncthreads();
       //===========================================
       //===========================================
       //        IN CASE OF BULGE CREATED 
       //      remove it and do a LEFT UPDATE
       //===========================================



       if(m>1){
           if(thid<m){
               locv[thid] = loca[thid][0]; // copy first column of A to annhiliate it
           }
           zlarfg(m, &(locv[0]), &(locv[1]), &(loctau));
           dalpha = locv[0];
           if(thid==0) locv[0]=MAGMA_Z_ONE;
           __syncthreads();

           //zlarfg(m, &(loca[0][0]), &(locv[1]), &(loctau)); //if used pay attention to the writing of 
           // first column of loca when applying the left below, so need to put an if condition
           // note that here I am writing the first column of loca 
           // which should be just annhiliated, by some scratch
           // then later when I finish i will put it good value.


           if ( !MAGMA_Z_EQUAL(loctau, MAGMA_Z_ZERO) ) {
               // w := v' * C 
               
               sum[myrow][mycol] = MAGMA_Z_ZERO;
               if(myrow<n){
                   for( j = mycol; j < m; j+= BLOCK_SIZEy)
                       sum[myrow][mycol] += loca[j][myrow] * MAGMA_Z_CNJG( locv[j] );
               }
               sum_colreduce_2d(BLOCK_SIZEy, myrow, mycol, sum);
               //sum_rowreduce_2dn(BLKD2_SIZE, mycol, myrwo, sumrow);
          
               //  C := C - tau * v * w 
               if(myrow<n){
                   lsum = -MAGMA_Z_CNJG(loctau) * sum[myrow][0];          
                   for( j = mycol; j < m; j+= BLOCK_SIZEy)
                       loca[j][myrow]  += lsum * locv[j];                  
               }
               __syncthreads();
               /*
               if( (thid<n)) {
                  //  w := v'  * C  
                  lsum = loca[0][thid];
                  for( j = 1; j < m; j ++ )
                     lsum +=  loca[j][thid]* MAGMA_Z_CNJG(locv[j]);
          
          
                  //  C := C - tau * v * w
                  lsum = - loctau * lsum;
                  loca[0][thid] += lsum;
                  for( j = 1; j < m; j ++ )
                      loca[j][thid]  += lsum *  locv[j];
          
               }
               __syncthreads();
               */
           }
      
          
           // if bulge created write back the new V and tau
           // and fix the first column of loca
           if (thid==0){
                loca[0][0] = dalpha;
                vl[0]      = MAGMA_Z_ONE;
                dtaul[0]   = loctau;
           } else if (thid<m) {
                vl[thid] = locv[thid];
                loca[thid][0] = MAGMA_Z_ZERO;
           }
           __syncthreads();
       }
       //===========================================
       // write back the matrix loca to dA
       if(myrow<m){
              for( j = mycol; j < n; j+= BLOCK_SIZEy)
                  c[myrow+j*ldc]  = loca[myrow][j];
       }
   }
  // synch the routine
  __syncthreads();
}
//====================================================================================================



/* Applies a complex elementary reflector H to a complex m by n
   matrix C, from the right. H is represented in the form

        H = I - tau * v * v'

   where tau is a complex scalar and v is a complex vector.
   If tau = 0, then H is taken to be the unit matrix              */
//====================================================================================================
//    RIGHT UPDATE 2D BLOCKED
//====================================================================================================
__device__ void zlarfr(int m, int n, hipDoubleComplex *v, hipDoubleComplex dtau, 
                       hipDoubleComplex *c, int ldc)
{

   const int thid = threadIdx.x;
   __shared__ hipDoubleComplex loctau;
   __syncthreads();


   if(thid==0) loctau     = dtau;
   __syncthreads();

   if ( !MAGMA_Z_EQUAL(loctau, MAGMA_Z_ZERO) ) {
      hipDoubleComplex lsum = MAGMA_Z_ZERO;
      magma_int_t j, gbrow, mpad;
      const int myrow = threadIdx.x % BLKD1_SIZE, mycol= threadIdx.x / BLKD1_SIZE;
      __shared__ hipDoubleComplex locv[ MAX_NB ];
      __shared__ hipDoubleComplex loca[ BLKD1_SIZE ][ MAX_NB+1 ];
      __shared__ hipDoubleComplex sum[ BLKD1_SIZE ][ BLKD2_SIZE+1];

      if(thid<n)
          locv[thid] = v[thid];
      __syncthreads();
       
      mpad = ((m+BLKD1_SIZE-1)/BLKD1_SIZE)*BLKD1_SIZE;
      // go over the blocki (vertical down)
      for(gbrow = myrow; gbrow<mpad; gbrow+=BLKD1_SIZE){
          sum[myrow][mycol] = MAGMA_Z_ZERO;
          // read a block of size BLKD1_SIZE x BLKD2_SIZE and do the GEMV
          // w := C  * v  
          if(gbrow<m){
              for( j = mycol; j < n; j+= BLKD2_SIZE)
                  loca[myrow][j]     = c[gbrow+j*ldc];
              for( j = mycol; j < n; j+= BLKD2_SIZE)
                  sum[myrow][mycol] += loca[myrow][j] * locv[j] ;
          }
          sum_colreduce_2dn(BLKD2_SIZE, myrow, mycol, sum);

          //  C := C - tau * w * v' 
          if(gbrow<m){
              /*
              if(mycol == 0){
                  sum[myrow][0] = -loctau *sum[myrow][0];
              }
              __syncthreads();
              for( j = mycol; j < n; j+= BLKD2_SIZE)
                  c[gbrow+j*ldc]  = loca[myrow][j]  + sum[myrow][0] * MAGMA_Z_CNJG( locv[j] );
              */
                  
              lsum = -loctau * sum[myrow][0];
              /*
              for( j = mycol; j < n; j+= BLKD2_SIZE)
                  loca[myrow][j] += lsum * MAGMA_Z_CNJG( locv[j] );
              for( j = mycol; j < n; j+= BLKD2_SIZE)
                  c[gbrow+j*ldc]  = loca[myrow][j];               */
              for( j = mycol; j < n; j+= BLKD2_SIZE)
                  c[gbrow+j*ldc]  = loca[myrow][j] + lsum * MAGMA_Z_CNJG( locv[j] );
                  
          }
          // sync between the blocki but ithink i don't need it here because every thread work on its same loca
          __syncthreads();
      }
  }
  // synch the routine
  __syncthreads();
}
//====================================================================================================

//====================================================================================================
//    LEFT UPDATE 2D BLOCKED
//====================================================================================================
__device__ void zlarfl(int m, int n, hipDoubleComplex *v, hipDoubleComplex dtau, 
                       hipDoubleComplex *c, int ldc)
{

   const int thid = threadIdx.x;
   __shared__ hipDoubleComplex loctau;
   __syncthreads();


   if(thid==0) loctau     = dtau;
   __syncthreads();

   if ( !MAGMA_Z_EQUAL(loctau, MAGMA_Z_ZERO) ) {
      hipDoubleComplex lsum = MAGMA_Z_ZERO;
      magma_int_t j, gbcol, npad;
      magma_int_t idlastblk,blkid,blknbcol,gbrow,blkjcol;
      //magma_int_t irow,icol,blksize;
      const int mycol = threadIdx.x % BLKD1_SIZE, myrow= threadIdx.x / BLKD1_SIZE;
      __shared__ hipDoubleComplex locv[ MAX_NB ];
      __shared__ hipDoubleComplex loca[ MAX_NB ] [ BLKD1_SIZE +1];
      __shared__ hipDoubleComplex sum[ BLKD2_SIZE] [ BLKD1_SIZE ];

      if(thid<m)
          locv[thid] = v[thid];
      __syncthreads();

       
      npad = ((n+BLKD1_SIZE-1)/BLKD1_SIZE)*BLKD1_SIZE;
      idlastblk = (npad/BLKD1_SIZE)-1;
      // go over the blocki (vertical down)
      for(gbcol = mycol; gbcol<npad; gbcol+=BLKD1_SIZE){
          sum[myrow][mycol] = MAGMA_Z_ZERO;
/*
          blkid    = gbcol/BLKD1_SIZE;
          blknbcol = blkid == idlastblk?  n-(idlastblk*BLKD1_SIZE) : BLKD1_SIZE;
          blksize  = blknbcol * m; 

          for( j = thid; j < blksize; j+= BLOCK_SIZE) { 
              irow = j%m;
              icol = j/m;
              loca[irow][icol]     = c[irow+(blkid*BLKD1_SIZE+icol)*ldc];
          }
          __syncthreads();
*/
          blkid    = gbcol/BLKD1_SIZE;
          blknbcol = blkid == idlastblk?  n-(idlastblk*BLKD1_SIZE) : BLKD1_SIZE;
          blkjcol  = blkid*BLKD1_SIZE;
          for(gbrow = mycol; gbrow<m; gbrow+=BLKD1_SIZE){
              for( j = myrow; j < blknbcol; j+= BLKD2_SIZE)
                  loca[gbrow][j]     = c[gbrow+(blkjcol+j)*ldc];
          }
          __syncthreads();

          // read a block of size BLKD1_SIZE x BLKD2_SIZE and do the GEMV
          // w := C  * v  
          if(gbcol<n){
              for( j = myrow; j < m; j+= BLKD2_SIZE)
              {
                  //loca[j][mycol]     = c[j+gbcol*ldc];
                  sum[myrow][mycol] += loca[j][mycol] * MAGMA_Z_CNJG( locv[j] );
              }
          }
          sum_rowreduce_2dn(BLKD2_SIZE, myrow, mycol, sum);

          //  C := C - tau * w * v' 
          if(gbcol<n){
              /*
              if(myrow == 0){
                  sum[0][mycol] = -loctau *sum[0][mycol];
              }
              __syncthreads();
              for( j = myrow; j < m; j+= BLKD2_SIZE)
                  c[j+gbcol*ldc]  = loca[j][mycol]  + sum[0][mycol] * locv[j];
              */
                  
              lsum = -loctau * sum[0][mycol];
              for( j = myrow; j < m; j+= BLKD2_SIZE)
                  loca[j][mycol]  += lsum * locv[j];                  
          }
          __syncthreads();
/*
          for( j = thid; j < blksize; j+= BLOCK_SIZE) { 
              irow = j%m;
              icol = j/m;
              c[irow+(blkid*BLKD1_SIZE+icol)*ldc] = loca[irow][icol];
          }
          __syncthreads();
*/
          for(gbrow = mycol; gbrow<m; gbrow+=BLKD1_SIZE){
              for( j = myrow; j < blknbcol; j+= BLKD2_SIZE)
                  c[gbrow+(blkjcol+j)*ldc] = loca[gbrow][j];
          }
          // sync between the blocki but ithink i don't need it here because every thread work on its same loca
          __syncthreads();
      }
  }
  // synch the routine
  __syncthreads();
}
//====================================================================================================
__global__ void
magma_zlarfrgl_gpu_kernel(int lem, int len, hipDoubleComplex *dVR, hipDoubleComplex *dTAUR,
                        hipDoubleComplex *dVL, hipDoubleComplex *dTAUL,
                        hipDoubleComplex *dA, int ldda)
{
     zlarfrgl(lem, len, dVR, dTAUR[0], dVL, dTAUL, dA, ldda);
}
//====================================================================================================
__global__ void
magma_zlarfr_gpu_kernel(int lem, int len, hipDoubleComplex *dV, hipDoubleComplex *dTAU,
                        hipDoubleComplex *dA, int ldda)
{
    zlarfr(lem, len, dV, dTAU[0], dA, ldda);
}
//====================================================================================================
__global__ void
magma_ztrdtype2cbHLsym_withQ_v2_gpu_kernel(int lem, int len,
                                           hipDoubleComplex *dA, int ldda,
                                           hipDoubleComplex *dV, hipDoubleComplex *dTAU,
                                           int st, int ed)
{
     const int thid = threadIdx.x;


     if (lem > 0) {
        if (thid==0){
             dV[0] = MAGMA_Z_ONE;
        } else if (thid<lem) {
             dV[thid] = *dA(ed+1+thid, st);
             *dA(ed+1+thid, st) = MAGMA_Z_ZERO;
        }
        zlarfg(lem, dA(ed+1, st), dV(1), dTAU);
     }
     // note that all htreads need to call this function
     zlarfl(lem, len-1, dV, MAGMA_Z_CNJG( dTAU[0] ), dA(ed+1, st+1), ldda-1);
}
//============================================================================

extern "C" void
magma_ztrdtype2cbHLsym_withQ_v2_gpu(magma_int_t n, magma_int_t nb, 
                                    hipDoubleComplex *dA, magma_int_t ldda, 
                                    hipDoubleComplex *dV, magma_int_t lddv, 
                                    hipDoubleComplex *dTAU,
                                    magma_int_t st, magma_int_t ed, magma_int_t sweep, 
                                    magma_int_t Vblksiz) 
{
    magma_int_t vposr=-1, tauposr=-1, vposl=-1, tauposl=-1;

    magma_int_t lddx = ldda-1;
    magma_int_t len = ed - st + 1;
    magma_int_t lem = min(ed+nb, n) - ed;

    if (nb > BLOCK_SIZE)
       printf("magma_ztrdtype2cbHLsym_withQ_v2_gpu: BLOCK_SIZE should be > %d\n", nb);

    if(lem>0){
        magma_bulge_findVTAUpos(n, nb, Vblksiz, sweep-1, st-1, lddv, &vposr, &tauposr);
        if(lem>1) magma_bulge_findVTAUpos(n, nb, Vblksiz, sweep-1, ed, lddv, &vposl, &tauposl);

        if(len>BLOCK_SIZEx){
            // Apply Right 
            magma_zlarfr_gpu_kernel<<< 1, BLOCK_SIZE >>>(lem, len, dV+vposr, dTAU+tauposr, 
                                                         dA(ed+1, st), lddx);
            magma_ztrdtype2cbHLsym_withQ_v2_gpu_kernel<<<1, BLOCK_SIZE>>>(lem, len,
                                                                          dA, ldda,
                                                                          dV+vposl, dTAU+tauposl,
                                                                          st, ed);
        }else{
            magma_zlarfrgl_gpu_kernel<<< 1, BLOCK_SIZE >>>(lem, len, 
                                                     dV+vposr, dTAU+tauposr,
                                                     dV+vposl, dTAU+tauposl,        
                                                     dA(ed+1, st), lddx);
        }
    }

}
#undef dA
#undef dAC
#undef dV
#undef dTAU
//====================================================================================================




///////////////////////////////////////////////////////////
//                  TYPE 3-LPK Householder
///////////////////////////////////////////////////////////
//// add -1 because of C
#define dA(i,j)   &(dA[((i)-(j)) + ldda*((j)-1)])
#define dAC(i,j)   &(dA[(i) + ldda*(j)])
#define dV(i)     &(dV[(i)])
#define dTAU(i)   &(dTAU[(i)])

__global__
void magma_ztrdtype3cbHLsym_withQ_v2_gpu_kernel(hipDoubleComplex *dA, int ldda,
                                                hipDoubleComplex *dV, hipDoubleComplex *dTAU,
                                                int st, int len)
{
          /*
             apply left and right on A(st:ed,st:ed)
             magma_zlarfxsym_v2(len, A(st,st), lda-1, V, TAU, work);
          */
          zlarfxsym_v2(len, dA(st,st), ldda-1, dV, dTAU);
}

extern "C" void
magma_ztrdtype3cbHLsym_withQ_v2_gpu(magma_int_t n, magma_int_t nb, 
                                    hipDoubleComplex *dA, magma_int_t ldda, 
                                    hipDoubleComplex *dV, magma_int_t lddv, 
                                    hipDoubleComplex *dTAU,
                                    magma_int_t st, magma_int_t ed, magma_int_t sweep, 
                                    magma_int_t Vblksiz) 
{
/*
    WORK (workspace) double complex array, dimension N
*/
    magma_int_t vpos, taupos, len;
    //magma_int_t lddx = ldda-1;

    if (nb > BLOCK_SIZE)
       printf("magma_ztrdtype1cbHLsym_withQ_v2_gpu: BLOCK_SIZE should be > %d\n", nb);
 
    magma_bulge_findVTAUpos(n, nb, Vblksiz, sweep-1, st-1, lddv, &vpos, &taupos);
    len = ed-st+1;

    magma_ztrdtype3cbHLsym_withQ_v2_gpu_kernel<<<1, BLOCK_SIZE>>>(dA, ldda, dV+vpos,
                                                                  dTAU+taupos,
                                                                  st, len);



}
#undef dA
#undef dAC
#undef dV
#undef dTAU







