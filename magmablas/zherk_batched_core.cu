#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z

       @author Jakub Kurzak
       @author Stan Tomov
       @author Mark Gates
       @author Azzam Haidar
       @author Ahmad Abdelfattah
       
*/

#include "common_magma.h"
#define PRECISION_z

#include "herk_template_kernel_batched.cuh"
#include "gemm_config/zgemm_param_nn.h"
#include "gemm_config/zgemm_param_nt.h"
#include "gemm_config/zgemm_param_tn.h"
#include "gemm_config/zgemm_param_tt.h"
#define version(s,v) s ## _V_ ## v

///////////////////////////////////////////////////////////////////////////////////////////////////
/**
    Purpose
    -------
    ZHERK performs one of the hermitian rank k operations

    C := alpha*A*A**H + beta*C,

    or

    C := alpha*A**H*A + beta*C,

    where alpha and beta are real scalars, C is an n by n hermitian
    matrix and A is an n by k matrix in the first case and a k by n
    matrix in the second case.
    
    Parameters
    ----------

    @param[in]
    uplo    CHARACTER*1.
           On entry, uplo specifies whether the upper or lower
           triangular part of the array C is to be referenced as
           follows:

           uplo = 'U' or 'u' Only the upper triangular part of C
           is to be referenced.

           uplo = 'L' or 'l' Only the lower triangular part of C
           is to be referenced.
    
    @param[in]
    trans   CHARACTER*1.
            On entry, trans specifies the operation to be performed as
            follows:

            trans = 'N' or 'n' C := alpha*A*A**H + beta*C.

            trans = 'C' or 'c' C := alpha*A**H*A + beta*C.

    @param[in]
    n       INTEGER.
            On entry,  specifies the order of the matrix C. N must be
            at least zero.
    
    @param[in]
    k       INTEGER.
            On entry with trans = 'N' or 'n', k specifies the number
            of columns of the matrix A, and on entry with
            trans = 'C' or 'c', k specifies the number of rows of the
            matrix A. K must be at least zero.

    @param[in]
    alpha   DOUBLE PRECISION
            On entry, ALPHA specifies the scalar alpha.
    
    @param[in]
    dA_array      Array of pointers, dimension (batchCount). 
             Each is a COMPLEX_16 array A of DIMENSION ( ldda, ka ), where ka is
             k  when  trans = MagmaNoTrans,  and is  n  otherwise.
             Before entry with  trans = MagmaNoTrans,  the leading  m by k
             part of the array A must contain the matrix A, otherwise
             the leading  k by m  part of the array A must contain  the
             matrix A.
    
    @param[in]
    ldda    INTEGER.
            On entry, ldda specifies the first dimension of each array A as declared
            in the calling (sub) program. When  trans = MagmaNoTrans then
            ldda must be at least  max( 1, n ), otherwise  ldda must be at
            least  max( 1, k ).
    
    @param[in]
    beta    DOUBLE PRECISION.
            On entry,  BETA  specifies the scalar  beta.  When  BETA  is
            supplied as zero then C need not be set on input.
    
    @param[in,out]
    dC_array      Array of pointers, dimension (batchCount).
             Each is a COMPLEX_16 array C of DIMENSION ( lddc, n ).
             Before entry with uplo = 'U' or 'u', the leading n by n
             upper triangular part of the array C must contain the upper
             triangular part of the hermitian matrix and the strictly
             lower triangular part of C is not referenced. On exit, the
             upper triangular part of the array C is overwritten by the
             upper triangular part of the updated matrix.
             Before entry with uplo = 'L' or 'l', the leading n by n
             lower triangular part of the array C must contain the lower
             triangular part of the hermitian matrix and the strictly
             upper triangular part of C is not referenced. On exit, the
             lower triangular part of the array C is overwritten by the
             lower triangular part of the updated matrix.
             Note that the imaginary parts of the diagonal elements need
             not be set, they are assumed to be zero, and on exit they
             are set to zero.

    @param[in]
    lddc    INTEGER.
            On entry, lddc specifies the first dimension of each array C as declared
            in  the  calling  (sub)  program.   lddc  must  be  at  least
            max( 1, m ).
    
    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_zblas3
    ********************************************************************/
void
magmablas_zherk_batched(
    magma_uplo_t uplo, magma_trans_t trans, 
    magma_int_t n, magma_int_t k,
    double alpha,
    magmaDoubleComplex const * const * dA_array, magma_int_t ldda,
    double beta,
    magmaDoubleComplex **dC_array, magma_int_t lddc, 
    magma_int_t batchCount, magma_queue_t queue )
{
    magmaDoubleComplex cbeta  = MAGMA_Z_MAKE( beta, 0. );
    magmaDoubleComplex calpha = MAGMA_Z_MAKE( alpha, 0. );

    magma_int_t info = 0;
    if      ( uplo != MagmaUpper && uplo != MagmaLower )
        info = -1;
    #if defined(PRECISION_c) || defined(PRECISION_z) 
    else if ( trans != MagmaNoTrans && trans != MagmaConjTrans )
    #else 
    else if ( trans != MagmaNoTrans && trans != MagmaTrans && trans != MagmaConjTrans )
    #endif
        info = -2;
    else if ( n < 0 )
        info = -3;
    else if ( k < 0 )
        info = -4;
    else if ( trans == MagmaNoTrans ? ldda < n : ldda < k )
        info = -7;
    else if ( lddc < n )
        info = -10;

    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }
    
    magma_int_t arch = magma_getdevice_arch();
    if ( arch < 200  ) {
        printf("not supported \n"); // TODO call cublas
        return;
    }
    
    // --------------------
    // CUDA ARCH 2.x (Fermi) version
    if ( n <= 0 || k <= 0 )
        return;

    // we have two shapes only (nc or cn)
    magma_int_t shape = 0;
    if      (trans == MagmaNoTrans)   { shape = 0; } // nc
    else                              { shape = 1; } // cn
        
    //TODO: probably the texture init code should be placed here

    size_t offsetA = 0;
    size_t offsetB = 0;
    offsetA = offsetA/sizeof(magmaDoubleComplex);
    offsetB = offsetB/sizeof(magmaDoubleComplex);
    
    switch(shape)
    {
        case 0: // nc
            {
                if (k <= 8)
                {
                    // version 58
                    herk_template_batched_nt<magmaDoubleComplex, version(NT,58), 0, 1>
                    (uplo, n, k, dA_array, ldda, dC_array, lddc, calpha, cbeta, offsetA, offsetB, batchCount, queue);
                }
                else
                {
                    // version 29
                    herk_template_batched_nt<magmaDoubleComplex, version(NT,29), 0, 1>
                    (uplo, n, k, dA_array, ldda, dC_array, lddc, calpha, cbeta, offsetA, offsetB, batchCount, queue);
                }
            }
            break;
        case 1: // cn
            {
                // version 72
                herk_template_batched_tn<magmaDoubleComplex, version(TN,72), 1, 0>
                (uplo, n, k, dA_array, ldda, dC_array, lddc, calpha, cbeta, offsetA, offsetB, batchCount, queue);
            }
            break;
        default:; // propose something
    }
}
///////////////////////////////////////////////////////////////////////////////////////////////////
