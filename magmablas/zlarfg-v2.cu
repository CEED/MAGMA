#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal z -> s d c

*/
#include "common_magma.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512

#define PRECISION_z

__global__
void magma_zlarfg_gpu_kernel( int n, hipDoubleComplex* dx0, hipDoubleComplex* dx, 
                              hipDoubleComplex *dtau, double *dxnorm )
{
    const int i = threadIdx.x;
    const int j = i + BLOCK_SIZE * blockIdx.x;
    __shared__ hipDoubleComplex scale;
    __shared__ double xnorm;    
  
    hipDoubleComplex dxi;

    if ( j < n-1)
        dxi = dx[j];
  
    if ( i == 0 ) {
        xnorm = *dxnorm;
        if ( xnorm == 0 ) {
            *dtau = MAGMA_Z_ZERO;
        }
        else {

#if (defined(PRECISION_s) || defined(PRECISION_d))
            double alpha = *dx0;

            // no need to compute the norm as it is passed as input
            double beta  = xnorm; // sqrt( alpha*alpha + xnorm*xnorm );
            beta  = -copysign( beta, alpha );
 
            // todo: deal with badly scaled vectors (see lapack's larfg)
            *dtau = (beta - alpha) / beta;
            *dx0  = beta;

            scale = 1. / (alpha - beta);
#else
            hipDoubleComplex alpha = *dx0;
            double alphar =  MAGMA_Z_REAL(alpha), alphai = MAGMA_Z_IMAG(alpha);

            // no need to compute the norm as it is passed as input
            double beta  = xnorm; // sqrt( alphar*alphar + alphai*alphai + xnorm*xnorm );
            beta  = -copysign( beta, alphar );

            // todo: deal with badly scaled vectors (see lapack's larfg)
            *dtau = MAGMA_Z_MAKE((beta - alphar)/beta, -alphai/beta);
            *dx0  = MAGMA_Z_MAKE(beta, 0.);
            
            alpha = MAGMA_Z_MAKE( MAGMA_Z_REAL(alpha) - beta, MAGMA_Z_IMAG(alpha));
            scale = MAGMA_Z_DIV( MAGMA_Z_ONE, alpha);
#endif
        }
    }

    // scale x
    __syncthreads();
    if ( xnorm != 0 && j < n-1)
        dx[j] = MAGMA_Z_MUL(dxi, scale);
}

/*
   Generates Householder elementary reflector H = I - tau v v^T to reduce
     H [ dx0 ] = [ beta ]
       [ dx  ]   [ 0    ]
   with beta = ±norm( [dx0, dx] ) = ±dxnorm[0].
   Stores v over dx; first element of v is 1 and is not stored.
   Stores beta over dx0.
   Stores tau.  

   The difference with LAPACK's zlarfg is that the norm of dx, and hance beta,
   are computed outside the routine and passed to it in dxnorm (array on the GPU).
*/
extern "C" void
magma_zlarfg_gpu(int n, hipDoubleComplex *dx0, hipDoubleComplex *dx, 
                 hipDoubleComplex *dtau, double *dxnorm)
{
    dim3 blocks((n+BLOCK_SIZE-1) / BLOCK_SIZE);
    dim3 threads( BLOCK_SIZE );

    magma_zlarfg_gpu_kernel<<< blocks, threads >>>( n, dx0, dx, dtau, dxnorm );
}
