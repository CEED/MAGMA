#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c

*/
#include "common_magma.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512

#define PRECISION_z


__global__
void magma_zlarfg_gpu_kernel( int n, magmaDoubleComplex* dx0, magmaDoubleComplex* dx,
                              magmaDoubleComplex *dtau, double *dxnorm, magmaDoubleComplex* dAkk)
{
    const int i = threadIdx.x;
    const int j = i + BLOCK_SIZE * blockIdx.x;
    __shared__ magmaDoubleComplex scale;
    double xnorm;

    magmaDoubleComplex dxi;

#if (defined(PRECISION_s) || defined(PRECISION_d))
    if( n <= 1 ) {
#else
    if( n <= 0 ) {
#endif
        *dtau = MAGMA_Z_ZERO;
        *dAkk = *dx0;
        return;
    }

    if ( j < n-1)
        dxi = dx[j];

    xnorm = *dxnorm;
    magmaDoubleComplex alpha = *dx0;

#if (defined(PRECISION_s) || defined(PRECISION_d))
    if ( xnorm != 0 ) {
        if (i == 0) {  
            double beta  = sqrt( alpha*alpha + xnorm*xnorm );
            beta  = -copysign( beta, alpha );

            // todo: deal with badly scaled vectors (see lapack's larfg)
            *dtau = (beta - alpha) / beta;
            *dAkk  = beta;

            scale = 1. / (alpha - beta);
        }
#else
    double alphar = MAGMA_Z_REAL(alpha);
    double alphai = MAGMA_Z_IMAG(alpha);
    if ( xnorm != 0 || alphai != 0) {
        if (i == 0) {
            double beta  = sqrt( alphar*alphar + alphai*alphai + xnorm*xnorm );
            beta  = -copysign( beta, alphar );

            // todo: deal with badly scaled vectors (see lapack's larfg)
            *dtau = MAGMA_Z_MAKE((beta - alphar)/beta, -alphai/beta);
            *dAkk = MAGMA_Z_MAKE(beta, 0.);

            alpha = MAGMA_Z_MAKE( MAGMA_Z_REAL(alpha) - beta, MAGMA_Z_IMAG(alpha));
            scale = MAGMA_Z_DIV( MAGMA_Z_ONE, alpha);
        }
#endif

        // scale x
        __syncthreads();
        if ( xnorm != 0 && j < n-1)
            dx[j] = MAGMA_Z_MUL(dxi, scale);

    } else {
        *dtau = MAGMA_Z_ZERO;
        *dAkk = *dx0; 
    }
}


/*
    Generates Householder elementary reflector H = I - tau v v^T to reduce
        H [ dx0 ] = [ beta ]
          [ dx  ]   [ 0    ]
    with beta = ±norm( [dx0, dx] ) = ±dxnorm[0].
    Stores v over dx; first element of v is 1 and is not stored.
    Stores beta over dx0.
    Stores tau.  
    
    The difference with LAPACK's zlarfg is that the norm of dx, and hence beta,
    are computed outside the routine and passed to it in dxnorm (array on the GPU).
*/
extern "C" void
magma_zlarfg_gpu(
    magma_int_t n,
    magmaDoubleComplex_ptr dx0,
    magmaDoubleComplex_ptr dx,
    magmaDoubleComplex_ptr dtau,
    magmaDouble_ptr        dxnorm,
    magmaDoubleComplex_ptr dAkk)
{
    dim3 blocks( magma_ceildiv( n, BLOCK_SIZE ) );
    dim3 threads( BLOCK_SIZE );

    /* recomputing the norm */
    //magmablas_dznrm2_cols(n, 1, dx0, n, dxnorm);
    magmablas_dznrm2_cols(n-1, 1, dx0+1, n, dxnorm);

    magma_zlarfg_gpu_kernel<<< blocks, threads,
                               0, magma_stream >>>(n, dx0, dx, dtau, dxnorm, dAkk);
}
