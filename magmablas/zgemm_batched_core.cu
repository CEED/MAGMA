#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @author Jakub Kurzak
       @author Stan Tomov
       @author Mark Gates
       @author Azzam Haidar
       @author Ahmad Abdelfattah
       
*/
#include "magma_internal.h"

#define PRECISION_z

#include "gemm_template_kernel_batched.cuh"
#include "gemm_config/zgemm_param_nn.h"
#include "gemm_config/zgemm_param_nt.h"
#include "gemm_config/zgemm_param_tn.h"
#include "gemm_config/zgemm_param_tt.h"

#define version(s,v) s ## _V_ ## v

/***************************************************************************//**
    Purpose
    -------
    ZGEMM performs one of the matrix-matrix operations
    
        C = alpha*op( A )*op( B ) + beta*C,
    
    where op( X ) is one of
    
        op( X ) = X      or
        op( X ) = X**T   or
        op( X ) = X**H,
    
    alpha and beta are scalars, and A, B and C are matrices, with
    op( A ) an m by k matrix, op( B ) a k by n matrix and C an m by n matrix.
    
    Parameters
    ----------
    @param[in]
    transA  magma_trans_t.
            On entry, transA specifies the form of op( A ) to be used in
            the matrix multiplication as follows:
      -     = MagmaNoTrans:    op( A ) = A.
      -     = MagmaTrans:      op( A ) = A**T.
      -     = MagmaConjTrans:  op( A ) = A**H.
    
    @param[in]
    transB  magma_trans_t.
            On entry, transB specifies the form of op( B ) to be used in
            the matrix multiplication as follows:
      -     = MagmaNoTrans:    op( B ) = B.
      -     = MagmaTrans:      op( B ) = B**T.
      -     = MagmaConjTrans:  op( B ) = B**H.
    
    @param[in]
    m       INTEGER.
            On entry,  M  specifies  the number  of rows  of the  matrix
            op( A )  and of the  matrix C.  M  must  be at least  zero.
    
    @param[in]
    n       INTEGER.
            On entry,  N  specifies the number  of columns of the matrix
            op( B ) and the number of columns of the matrix C. N must be
            at least zero.
    
    @param[in]
    k       INTEGER.
            On entry,  K  specifies  the number of columns of the matrix
            op( A ) and the number of rows of the matrix op( B ). K must
            be at least  zero.
    
    @param[in]
    alpha   COMPLEX_16
            On entry, ALPHA specifies the scalar alpha.
    
    @param[in]
    dA_array      Array of pointers, dimension (batchCount).
             Each is a COMPLEX_16 array A of DIMENSION ( ldda, ka ), where ka is
             k  when  transA = MagmaNoTrans,  and is  m  otherwise.
             Before entry with  transA = MagmaNoTrans,  the leading  m by k
             part of the array A must contain the matrix A, otherwise
             the leading  k by m  part of the array A must contain  the
             matrix A.
    
    @param[in]
    ldda    INTEGER.
            On entry, ldda specifies the first dimension of each array A as declared
            in the calling (sub) program. When  transA = MagmaNoTrans then
            ldda must be at least  max( 1, m ), otherwise  ldda must be at
            least  max( 1, k ).
    
    @param[in]
    dB_array      Array of pointers, dimension (batchCount).
             Each is a COMPLEX_16 array B of DIMENSION ( lddb, kb ), where kb is
             n  when  transB = MagmaNoTrans,  and is  k  otherwise.
             Before entry with  transB = MagmaNoTrans,  the leading  k by n
             part of the array B must contain the matrix B, otherwise
             the leading  n by k  part of the array B must contain  the
             matrix B.
    
    @param[in]
    lddb    INTEGER.
            On entry, lddb specifies the first dimension of each array B as declared
            in the calling (sub) program. When  transB = MagmaNoTrans then
            lddb must be at least  max( 1, k ), otherwise  lddb must be at
            least  max( 1, n ).
    
    @param[in]
    beta    COMPLEX_16.
            On entry,  BETA  specifies the scalar  beta.  When  BETA  is
            supplied as zero then C need not be set on input.
    
    @param[in,out]
    dC_array      Array of pointers, dimension (batchCount).
             Each is a COMPLEX_16 array C of DIMENSION ( lddc, n ).
             Before entry, the leading  m by n  part of the array  C must
             contain the matrix  C,  except when  beta  is zero, in which
             case C need not be set on entry.
             On exit, the array  C  is overwritten by the  m by n  matrix
             ( alpha*op( A )*op( B ) + beta*C ).
    
    @param[in]
    lddc    INTEGER.
            On entry, lddc specifies the first dimension of each array C as declared
            in  the  calling  (sub)  program.   lddc  must  be  at  least
            max( 1, m ).
    
    @param[in]
    batchCount  INTEGER
                The number of matrices to operate on.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_gemm_batched
*******************************************************************************/
void
magmablas_zgemm_batched_core(
    magma_trans_t transA, magma_trans_t transB,
    magma_int_t m, magma_int_t n, magma_int_t k,
    magmaDoubleComplex alpha,
    magmaDoubleComplex const * const * dA_array, magma_int_t ldda,
    magmaDoubleComplex const * const * dB_array, magma_int_t lddb,
    magmaDoubleComplex beta,
    magmaDoubleComplex **dC_array, magma_int_t lddc,
    magma_int_t roffA, magma_int_t coffA,
    magma_int_t roffB, magma_int_t coffB,
    magma_int_t roffC, magma_int_t coffC,
    magma_int_t batchCount, magma_queue_t queue )
{
    magma_int_t info = 0;
    if      ( transA != MagmaNoTrans && transA != MagmaTrans && transA != MagmaConjTrans )
        info = -1;
    else if ( transB != MagmaNoTrans && transB != MagmaTrans && transB != MagmaConjTrans )
        info = -2;
    else if ( m < 0 )
        info = -3;
    else if ( n < 0 )
        info = -4;
    else if ( k < 0 )
        info = -5;
    else if ( transA == MagmaNoTrans ? ldda < m : ldda < k )
        info = -8;
    else if ( transB == MagmaNoTrans ? lddb < k : lddb < n )
        info = -10;
    else if ( lddc < m )
        info = -13;
    
    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }
    
    magma_int_t arch = magma_getdevice_arch();
    if ( arch < 200  ) {
        printf("arch < 200 not supported \n"); // TODO call cublas
        return;
    }
    
    if ( m <= 0 || n <= 0 || k <= 0 )
        return;

    magma_int_t shape = 0;
    if      (transA == MagmaNoTrans   && transB == MagmaNoTrans)   { shape = 0; } // nn
    else if (transA == MagmaNoTrans   && transB == MagmaTrans)     { shape = 1; } // nt
    else if (transA == MagmaNoTrans   && transB == MagmaConjTrans) { shape = 2; } // nc
    else if (transA == MagmaTrans     && transB == MagmaNoTrans)   { shape = 3; } // tn
    else if (transA == MagmaTrans     && transB == MagmaTrans)     { shape = 4; } // tt
    else if (transA == MagmaTrans     && transB == MagmaConjTrans) { shape = 5; } // tc
    else if (transA == MagmaConjTrans && transB == MagmaNoTrans)   { shape = 6; } // cn
    else if (transA == MagmaConjTrans && transB == MagmaTrans)     { shape = 7; } // ct
    else if (transA == MagmaConjTrans && transB == MagmaConjTrans) { shape = 8; } // cc
    
    switch(shape)
    {
        case 0: // nn
            {
                gemm_template_batched_nn<magmaDoubleComplex, version(NN,18), 0, 0>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, roffA, coffA, roffB, coffB, roffC, coffC, batchCount, queue);
            }
            break;
        case 1: // nt
            {
                if (k <= 8)
                {
                    // version 58
                    gemm_template_batched_nt<magmaDoubleComplex, version(NT,58), 0, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, roffA, coffA, roffB, coffB, roffC, coffC, batchCount, queue);
                }
                else
                {
                    // version 29
                    gemm_template_batched_nt<magmaDoubleComplex, version(NT,29), 0, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, roffA, coffA, roffB, coffB, roffC, coffC, batchCount, queue);
                }
            }
            break;
        case 2: // nc
            {
                if (k <= 8)
                {
                    // version 58
                    gemm_template_batched_nt<magmaDoubleComplex, version(NT,58), 0, 1>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, roffA, coffA, roffB, coffB, roffC, coffC, batchCount, queue);
                }
                else
                {
                    // version 29
                    gemm_template_batched_nt<magmaDoubleComplex, version(NT,29), 0, 1>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, roffA, coffA, roffB, coffB, roffC, coffC, batchCount, queue);
                }
            }
            break;
        case 3: // tn
            {
                // version 72
                gemm_template_batched_tn<magmaDoubleComplex, version(TN,72), 0, 0>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, roffA, coffA, roffB, coffB, roffC, coffC, batchCount, queue);
            }
            break;
        case 6: // cn
            {
                // version 72
                gemm_template_batched_tn<magmaDoubleComplex, version(TN,72), 1, 0>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, roffA, coffA, roffB, coffB, roffC, coffC, batchCount, queue);
            }
            break;
        case 4: // tt
            {
                // version 13
                gemm_template_batched_tt<magmaDoubleComplex, version(TT,13), 0, 0>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, roffA, coffA, roffB, coffB, roffC, coffC, batchCount, queue);
            }
            break;
        case 5: // tc
            {
                // version 13
                gemm_template_batched_tt<magmaDoubleComplex, version(TT,13), 0, 1>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, roffA, coffA, roffB, coffB, roffC, coffC, batchCount, queue);
            }
            break;
        case 7: // ct
            {
                // version 13
                gemm_template_batched_tt<magmaDoubleComplex, version(TT,13), 1, 0>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, roffA, coffA, roffB, coffB, roffC, coffC, batchCount, queue);
            }
            break;
        case 8: // cc
            {
                // version 13
                gemm_template_batched_tt<magmaDoubleComplex, version(TT,13), 1, 1>
                (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, roffA, coffA, roffB, coffB, roffC, coffC, batchCount, queue);
            }
            break;
        default:; // propose something
    }
}
