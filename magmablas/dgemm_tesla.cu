#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal d -> s
*/
#include "common_magma.h"
#include "commonblas_d.h"

extern "C" void
magmablas_dgemm_tesla(
    magma_trans_t transA, magma_trans_t transB, magma_int_t m, magma_int_t n, magma_int_t k,
    double alpha,
    const double *A, magma_int_t lda,
    const double *B, magma_int_t ldb,
    double beta,
    double *C, magma_int_t ldc )
{
/*  -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

    Purpose
    =======
    
    DGEMM performs one of the matrix-matrix operations
    
        C = alpha*op( A )*op( B ) + beta*C,
    
    where op( X ) is one of
    
        op( X ) = X   or   op( X ) = X**T,
    
    alpha and beta are scalars, and A, B and C are matrices, with op( A )
    an m by k matrix,  op( B ) a k by n matrix and C an m by n matrix.
    
    Parameters
    ==========
    
    transA  CHARACTER*1.
            On entry, transA specifies the form of op( A ) to be used in
            the matrix multiplication as follows:
    
                transA = 'N' or 'n',  op( A ) = A.
    
                transA = 'T' or 't',  op( A ) = A**T.
    
                transA = 'C' or 'c',  op( A ) = A**T.
    
            Unchanged on exit.
    
    transB  CHARACTER*1.
            On entry, transB specifies the form of op( B ) to be used in
            the matrix multiplication as follows:
    
                transB = 'N' or 'n',  op( B ) = B.
    
                transB = 'T' or 't',  op( B ) = B**T.
    
                transB = 'C' or 'c',  op( B ) = B**T.
    
            Unchanged on exit.
    
    M       INTEGER.
            On entry,  M  specifies  the number  of rows  of the  matrix
            op( A )  and of the  matrix  C.  M  must  be at least  zero.
            Unchanged on exit.
    
    N       INTEGER.
            On entry,  N  specifies the number  of columns of the matrix
            op( B ) and the number of columns of the matrix C. N must be
            at least zero.
            Unchanged on exit.
    
    K       INTEGER.
            On entry,  K  specifies  the number of columns of the matrix
            op( A ) and the number of rows of the matrix op( B ). K must
            be at least  zero.
            Unchanged on exit.
    
    ALPHA   DOUBLE PRECISION.
            On entry, ALPHA specifies the scalar alpha.
            Unchanged on exit.
    
    A       DOUBLE PRECISION array of DIMENSION ( LDA, ka ), where ka is
            k  when  transA = 'N' or 'n',  and is  m  otherwise.
            Before entry with  transA = 'N' or 'n',  the leading  m by k
            part of the array  A  must contain the matrix  A,  otherwise
            the leading  k by m  part of the array  A  must contain  the
            matrix A.
            Unchanged on exit.
    
    LDA     INTEGER.
            On entry, LDA specifies the first dimension of A as declared
            in the calling (sub) program. When  transA = 'N' or 'n' then
            LDA must be at least  max( 1, m ), otherwise  LDA must be at
            least  max( 1, k ).
            Unchanged on exit.
    
    B       DOUBLE PRECISION array of DIMENSION ( LDB, kb ), where kb is
            n  when  transB = 'N' or 'n',  and is  k  otherwise.
            Before entry with  transB = 'N' or 'n',  the leading  k by n
            part of the array  B  must contain the matrix  B,  otherwise
            the leading  n by k  part of the array  B  must contain  the
            matrix B.
            Unchanged on exit.
    
    LDB     INTEGER.
            On entry, LDB specifies the first dimension of B as declared
            in the calling (sub) program. When  transB = 'N' or 'n' then
            LDB must be at least  max( 1, k ), otherwise  LDB must be at
            least  max( 1, n ).
            Unchanged on exit.
    
    BETA    DOUBLE PRECISION.
            On entry,  BETA  specifies the scalar  beta.  When  BETA  is
            supplied as zero then C need not be set on input.
            Unchanged on exit.
    
    C       DOUBLE PRECISION array of DIMENSION ( LDC, n ).
            Before entry, the leading  m by n  part of the array  C must
            contain the matrix  C,  except when  beta  is zero, in which
            case C need not be set on entry.
            On exit, the array  C  is overwritten by the  m by n  matrix
            ( alpha*op( A )*op( B ) + beta*C ).
    
    LDC     INTEGER.
            On entry, LDC specifies the first dimension of C as declared
            in  the  calling  (sub)  program.   LDC  must  be  at  least
            max( 1, m ).
            Unchanged on exit.
    =====================================================================    */

    if ( m == 0 || n == 0 || ((alpha == 0.0 || k == 0) && beta == 1.0) ) {
        return;
    }
    transA = (magma_trans_t)toupper( transA );
    transB = (magma_trans_t)toupper( transB );
    if ( alpha == 0.0 ) {
        if ( beta == 0.0 ) {
            magmablas_dgemm_ab_0(
                C, A, B, m, n, k, lda, ldb, ldc, alpha, beta );
            return;
        }
        else {
            magmablas_dgemm_a_0(
                C, A, B, m, n, k, lda, ldb, ldc, alpha, beta );
            return;
        }
    }
    
    if ( ldc < m ) return;  /* TODO: error */
    if ( transA == 'N' ) {
        if ( transB == 'N' ) {
            if ( lda < m ) return;  /* TODO: error */
            if ( ldb < k ) return;  /* TODO: error */
            /*=======================================================================
              ===================C = alpha * A * B + beta * C =======================
              =======================================================================*/
            if ( m > 512 && n > 512 ) {
                if ( m % 64 == 0 && n % 16 == 0 && k % 16 == 0 )
                    magmablas_dgemm_N_N_64_16_16_16_4_special(
                        C, A, B, m, n, k, lda, ldb, ldc, alpha, beta );
                else
                    magmablas_dgemm_N_N_64_16_16_16_4(
                        C, A, B, m, n, k, lda, ldb, ldc, alpha, beta );
            }
            else {
                hipblasDgemm(
                    transA, transB, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc );
            }
        }
        else {
            if ( lda < m ) return;  /* TODO: error */
            if ( ldb < n ) return;  /* TODO: error */
            /*=======================================================================
              ===================C = alpha * A * B^T + beta * C======================
              =======================================================================*/
            if ( m > 512 && n > 512 ) {
                //if ( m % 64 == 0 && n % 16 == 0 && k % 4 == 0 )
                //    magmablas_dgemm_N_T_64_16_4_16_4(
                //        C, A, B, m, n, k, lda, ldb, ldc, alpha, beta );
                //else
                    magmablas_dgemm_N_T_64_16_4_16_4(
                        C, A, B, m, n, k, lda, ldb, ldc, alpha, beta );
            }
            else {
                hipblasDgemm(
                    transA, transB, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc );
            }
        }
    }
    else {
        if ( transB == 'N' ) {
            if ( lda < k ) return;  /* TODO: error */
            if ( ldb < k ) return;  /* TODO: error */
            /*=======================================================================
              ===================C = alpha * A^T * B + beta * C======================
              =======================================================================*/
            if ( m > 512 && n > 512 ) {
                //if ( m % 32 == 0 && n % 32 == 0 && k % 8 == 0 )
                //    magmablas_dgemm_T_N_32_32_8_8_8(
                //        C, A, B, m, n, k, lda, ldb, ldc, alpha, beta );
                //else
                    magmablas_dgemm_T_N_32_32_8_8_8(
                        C, A, B, m, n, k, lda, ldb, ldc, alpha, beta );
            }
            else {
                hipblasDgemm(
                    transA, transB, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc );
            }
        }
        else {
            if ( lda < k ) return;  /* TODO: error */
            if ( ldb < n ) return;  /* TODO: error */
            /*=======================================================================
              ===================C = alpha * A^T * B^T + beta * C====================
              =======================================================================*/
            if ( m > 512 && n > 512 ) {
                if ( m % 64 == 0 && n % 16 == 0 && k % 16 == 0 )
                    magmablas_dgemm_T_T_64_16_16_16_4_special(
                        C, B, A, n, m, k, ldb, lda, ldc, alpha, beta );
                else
                    magmablas_dgemm_T_T_64_16_16_16_4(
                        C, B, A, n, m, k, ldb, lda, ldc, alpha, beta );
            }
            else {
                hipblasDgemm(
                    transA, transB, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc );
            }
        }
    }
}
