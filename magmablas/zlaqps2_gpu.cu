#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c

*/

#include "common_magma.h"
#include "commonblas_z.h"

#define PRECISION_z

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512


/* --------------------------------------------------------------------------- */
/**
    Purpose
    -------
    ZLAQPS computes a step of QR factorization with column pivoting
    of a complex M-by-N matrix A by using Blas-3.  It tries to factorize
    NB columns from A starting from the row OFFSET+1, and updates all
    of the matrix with Blas-3 xGEMM.

    In some cases, due to catastrophic cancellations, it cannot
    factorize NB columns.  Hence, the actual number of factorized
    columns is returned in KB.

    Block A(1:OFFSET,1:N) is accordingly pivoted, but not factorized.

    Arguments
    ---------
    @param[in]
    m       INTEGER
            The number of rows of the matrix A. M >= 0.

    @param[in]
    n       INTEGER
            The number of columns of the matrix A. N >= 0

    @param[in]
    offset  INTEGER
            The number of rows of A that have been factorized in
            previous steps.

    @param[in]
    nb      INTEGER
            The number of columns to factorize.

    @param[out]
    kb      INTEGER
            The number of columns actually factorized.

    @param[in,out]
    dA      COMPLEX*16 array, dimension (LDDA,N)
            On entry, the M-by-N matrix A.
            On exit, block A(OFFSET+1:M,1:KB) is the triangular
            factor obtained and block A(1:OFFSET,1:N) has been
            accordingly pivoted, but no factorized.
            The rest of the matrix, block A(OFFSET+1:M,KB+1:N) has
            been updated.

    @param[in]
    ldda    INTEGER
            The leading dimension of the array A. LDDA >= max(1,M).

    @param[in,out]
    jpvt    INTEGER array, dimension (N)
            JPVT(I) = K <==> Column K of the full matrix A has been
            permuted into position I in AP.

    @param[out]
    dtau    COMPLEX*16 array, dimension (KB)
            The scalar factors of the elementary reflectors.

    @param[in,out]
    dvn1    DOUBLE PRECISION array, dimension (N)
            The vector with the partial column norms.

    @param[in,out]
    dvn2    DOUBLE PRECISION array, dimension (N)
            The vector with the exact column norms.

    @param[in,out]
    dauxv   COMPLEX*16 array, dimension (NB)
            Auxiliar vector.

    @param[in,out]
    dF      COMPLEX*16 array, dimension (LDDF,NB)
            Matrix F**H = L * Y**H * A.

    @param[in]
    lddf    INTEGER
            The leading dimension of the array F. LDDF >= max(1,N).

    @ingroup magma_zgeqp3_aux
    ********************************************************************/
extern "C" magma_int_t
magma_zlaqps2_gpu(
    magma_int_t m, magma_int_t n, magma_int_t offset,
    magma_int_t nb, magma_int_t *kb,
    magmaDoubleComplex_ptr dA,  magma_int_t ldda,
    magma_int_t *jpvt,
    magmaDoubleComplex_ptr dtau, 
    magmaDouble_ptr dvn1, magmaDouble_ptr dvn2,
    magmaDoubleComplex_ptr dauxv,
    magmaDoubleComplex_ptr dF,  magma_int_t lddf)
{
#define dA(i_, j_) (dA + (i_) + (j_)*(ldda))
#define dF(i_, j_) (dF + (i_) + (j_)*(lddf))

    magmaDoubleComplex c_zero    = MAGMA_Z_MAKE( 0.,0.);
    magmaDoubleComplex c_one     = MAGMA_Z_MAKE( 1.,0.);
    magmaDoubleComplex c_neg_one = MAGMA_Z_MAKE(-1.,0.);
    magma_int_t ione = 1;
    
    magma_int_t i__1, i__2;
    
    magma_int_t k, rk;
    magmaDoubleComplex tauk;
    magma_int_t pvt, itemp;
    double tol3z;

    magmaDoubleComplex_ptr dAkk = dauxv;
    dauxv += nb;

    double lsticc, *lsticcs;
    magma_dmalloc( &lsticcs, 1+256*(n+255)/256 );

    tol3z = magma_dsqrt( lapackf77_dlamch("Epsilon"));

    lsticc = 0;
    k = 0;
    while( k < nb && lsticc == 0 ) {
        rk = offset + k;

        /* Determine ith pivot column and swap if necessary */
        pvt = k - 1 + magma_idamax( n-k, &dvn1[k], ione );

        if (pvt != k) {
            magmablas_zswap( k+1, dF(pvt,0), lddf, dF(k,0), lddf);

            itemp     = jpvt[pvt];
            jpvt[pvt] = jpvt[k];
            jpvt[k]   = itemp;
            magma_dswap( 2, &dvn1[pvt], n+offset, &dvn1[k], n+offset );

            magmablas_zswap( m, dA(0,pvt), ione, dA(0, k), ione );
        }

        /* Apply previous Householder reflectors to column K:
           A(RK:M,K) := A(RK:M,K) - A(RK:M,1:K-1)*F(K,1:K-1)'.
           Optimization: multiply with beta=0; wait for vector and subtract */
        if (k > 0) {
            magmablas_zgemv_conjv( m-rk, k,
                                   c_neg_one, dA(rk, 0), ldda,
                                              dF(k,  0), lddf,
                                   c_one,     dA(rk, k), ione );
        }

        /*  Generate elementary reflector H(k). */
        magma_zlarfg_gpu(m-rk, dA(rk, k), dA(rk + 1, k), &dtau[k], &dvn1[k], &dAkk[k]);
        magma_zsetvector( 1, &c_one,   1, dA(rk, k), 1 );

        /* Compute Kth column of F:
           Compute  F(K+1:N,K) := tau(K)*A(RK:M,K+1:N)'*A(RK:M,K) on the GPU */
        if (k < n-1 || k > 0 ) magma_zgetvector( 1, &dtau[k], 1, &tauk, 1 );
        if (k < n-1) {
            magma_zgemv( MagmaConjTrans, m-rk, n-k-1,
                     tauk,   dA( rk,  k+1 ), ldda,
                             dA( rk,  k   ), 1,
                     c_zero, dF( k+1, k   ), 1 );
        }

        /* Incremental updating of F:
           F(1:N,K) := F(1:N,K) - tau(K)*F(1:N,1:K-1)*A(RK:M,1:K-1)'*A(RK:M,K). 
           F(1:N,K) := tau(K)*A(RK:M,K+1:N)'*A(RK:M,K) - tau(K)*F(1:N,1:K-1)*A(RK:M,1:K-1)'*A(RK:M,K)
                    := tau(K)(A(RK:M,K+1:N)' - F(1:N,1:K-1)*A(RK:M,1:K-1)') A(RK:M,K)  
           so, F is (updated A)*V */
        if (k > 0) {
            /*z__1 = MAGMA_Z_NEGATE( tauk );
            magma_zgemv( MagmaConjTrans, m-rk, k,
                         z__1,   dA(rk, 0), ldda,
                                 dA(rk, k), ione,
                         c_zero, dauxv, ione ); */

            magma_zgemv_kernel3<<< k, BLOCK_SIZE, 0, magma_stream >>>(m-rk, dA(rk, 0), ldda,
                                                                      dA(rk, k), dauxv, dtau+k);

            /* I think we only need stricly lower-triangular part */
            magma_zgemv( MagmaNoTrans, n-k-1, k,
                         c_one, dF(k+1,0), lddf,
                                dauxv,     ione,
                         c_one, dF(k+1,k), ione );
        }

       /* Update the current row of A:
           A(RK,K+1:N) := A(RK,K+1:N) - A(RK,1:K)*F(K+1:N,1:K)'.               */
        if (k < n-1) {
            i__1 = n - k - 1;
            i__2 = k + 1;
            /* left-looking update of rows,                     *
             * since F=A**H v with original A, so no right-looking */
            magma_zgemm( MagmaNoTrans, MagmaConjTrans, ione, i__1, i__2,
                         c_neg_one, dA(rk, 0  ), ldda,
                                    dF(k+1,0  ), lddf,
                         c_one,     dA(rk, k+1), ldda ); 
        }

        /* Update partial column norms. */
        if (rk < min(m, n+offset)-1) {
            magmablas_dznrm2_row_check_adjust(n-k-1, tol3z, &dvn1[k+1], 
                                              &dvn2[k+1], dA(rk,k+1), ldda, lsticcs); 
            
            magma_dgetvector( 1, &lsticcs[0], 1, &lsticc, 1 );
        }

        //*dA(rk, k) = Akk;
        //magma_zsetvector( 1, &Akk, 1, dA(rk, k), 1 );
        //magmablas_zlacpy(MagmaUpperLower, 1, 1, dAkk, 1, dA(rk, k), 1);

        ++k;
    }
    // restore the diagonals
    magma_zcopymatrix( 1, k, dAkk, 1, dA(offset, 0), ldda+1 );

    // leave k as the last column done
    --k;
    *kb = k + 1;
    rk = offset + *kb - 1;

    /* Apply the block reflector to the rest of the matrix:
       A(OFFSET+KB+1:M,KB+1:N) := A(OFFSET+KB+1:M,KB+1:N) - 
                                  A(OFFSET+KB+1:M,1:KB)*F(KB+1:N,1:KB)'  */
    if (*kb < min(n, m - offset)) {
        i__1 = m - rk - 1;
        i__2 = n - *kb;

        magma_zgemm( MagmaNoTrans, MagmaConjTrans, i__1, i__2, *kb,
                     c_neg_one, dA(rk+1, 0  ), ldda,
                                dF(*kb,  0  ), lddf,
                     c_one,     dA(rk+1, *kb), ldda );
    }

    /* Recomputation of difficult columns. */
    if ( lsticc > 0 ) {
        // printf( " -- recompute dnorms --\n" );
        magmablas_dznrm2_check(m-rk-1, n-*kb, dA(rk+1,*kb), ldda,
                               &dvn1[*kb], lsticcs);
        magma_dcopymatrix( n-*kb, 1, &dvn1[*kb], n, &dvn2[*kb], n );
    }
    magma_free(lsticcs);
    
    return MAGMA_SUCCESS;
} /* magma_zlaqps */
