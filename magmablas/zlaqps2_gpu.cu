#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c

*/

#include "common_magma.h"

#define PRECISION_z

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE 512

__global__ void magma_zgemv_kernel3(int m, const magmaDoubleComplex * __restrict__ V, int ldv,
                                    magmaDoubleComplex *c, magmaDoubleComplex *dwork,
                                    magmaDoubleComplex *tau);

/* --------------------------------------------------------------------------- */
/**
    Purpose
    -------
    ZLAQPS computes a step of QR factorization with column pivoting
    of a complex M-by-N matrix A by using Blas-3.  It tries to factorize
    NB columns from A starting from the row OFFSET+1, and updates all
    of the matrix with Blas-3 xGEMM.

    In some cases, due to catastrophic cancellations, it cannot
    factorize NB columns.  Hence, the actual number of factorized
    columns is returned in KB.

    Block A(1:OFFSET,1:N) is accordingly pivoted, but not factorized.

    Arguments
    ---------
    @param[in]
    m       INTEGER
            The number of rows of the matrix A. M >= 0.

    @param[in]
    n       INTEGER
            The number of columns of the matrix A. N >= 0

    @param[in]
    offset  INTEGER
            The number of rows of A that have been factorized in
            previous steps.

    @param[in]
    NB      INTEGER
            The number of columns to factorize.

    @param[out]
    kb      INTEGER
            The number of columns actually factorized.

    @param[in,out]
    A       COMPLEX*16 array, dimension (LDA,N)
            On entry, the M-by-N matrix A.
            On exit, block A(OFFSET+1:M,1:KB) is the triangular
            factor obtained and block A(1:OFFSET,1:N) has been
            accordingly pivoted, but no factorized.
            The rest of the matrix, block A(OFFSET+1:M,KB+1:N) has
            been updated.

    @param[in]
    lda     INTEGER
            The leading dimension of the array A. LDA >= max(1,M).

    @param[in,out]
    jpvt    INTEGER array, dimension (N)
            JPVT(I) = K <==> Column K of the full matrix A has been
            permuted into position I in AP.

    @param[out]
    tau     COMPLEX*16 array, dimension (KB)
            The scalar factors of the elementary reflectors.

    @param[in,out]
    VN1     DOUBLE PRECISION array, dimension (N)
            The vector with the partial column norms.

    @param[in,out]
    VN2     DOUBLE PRECISION array, dimension (N)
            The vector with the exact column norms.

    @param[in,out]
    AUXV    COMPLEX*16 array, dimension (NB)
            Auxiliar vector.

    @param[in,out]
    F       COMPLEX*16 array, dimension (LDF,NB)
            Matrix F' = L*Y'*A.

    @param[in]
    ldf     INTEGER
            The leading dimension of the array F. LDF >= max(1,N).

    @ingroup magma_zgeqp3_aux
    ********************************************************************/
extern "C" magma_int_t
magma_zlaqps2_gpu(magma_int_t m, magma_int_t n, magma_int_t offset,
             magma_int_t nb, magma_int_t *kb,
             magmaDoubleComplex *A,  magma_int_t lda,
             magma_int_t *jpvt, magmaDoubleComplex *tau, 
             double *vn1, double *vn2,
             magmaDoubleComplex *auxv,
             magmaDoubleComplex *F,  magma_int_t ldf)
{
#define  A(i, j) (A  + (i) + (j)*(lda ))
#define  F(i, j) (F  + (i) + (j)*(ldf ))

    magmaDoubleComplex c_zero    = MAGMA_Z_MAKE( 0.,0.);
    magmaDoubleComplex c_one     = MAGMA_Z_MAKE( 1.,0.);
    magmaDoubleComplex c_neg_one = MAGMA_Z_MAKE(-1.,0.);
    magma_int_t ione = 1;
    
    magma_int_t i__1, i__2;
    
    magma_int_t k, rk;
    magmaDoubleComplex tauk;
    magma_int_t pvt, itemp;
    double tol3z;

    magmaDoubleComplex *dAkk = auxv;
    auxv+=nb;

    double lsticc, *lsticcs;
    magma_dmalloc( &lsticcs, 1+256*(n+255)/256 );

    tol3z = magma_dsqrt( lapackf77_dlamch("Epsilon"));

    lsticc = 0;
    k = 0;
    while( k < nb && lsticc == 0 ) {
        rk = offset + k;
        
        /* Determine ith pivot column and swap if necessary */
        pvt = k - 1 + magma_idamax( n-k, &vn1[k], ione );
        
        if (pvt != k) {
            magmablas_zswap( k, F(pvt,0), ldf, F(k,0), ldf);
            itemp     = jpvt[pvt];
            jpvt[pvt] = jpvt[k];
            jpvt[k]   = itemp;
            #if (defined(PRECISION_d) || defined(PRECISION_z))
                //magma_dswap( 1, &vn1[pvt], 1, &vn1[k], 1 );
                //magma_dswap( 1, &vn2[pvt], 1, &vn2[k], 1 );
                magma_dswap( 2, &vn1[pvt], n+offset, &vn1[k], n+offset);
            #else
                //magma_sswap( 1, &vn1[pvt], 1, &vn1[k], 1 );
                //magma_sswap( 1, &vn2[pvt], 1, &vn2[k], 1 );
                magma_sswap(2, &vn1[pvt], n+offset, &vn1[k], n+offset);
            #endif

            magmablas_zswap( m, A(0,pvt), ione, A(0, k), ione );
        }

        /* Apply previous Householder reflectors to column K:
           A(RK:M,K) := A(RK:M,K) - A(RK:M,1:K-1)*F(K,1:K-1)'.
           Optimization: multiply with beta=0; wait for vector and subtract */
        if (k > 0) {
            /*#if (defined(PRECISION_c) || defined(PRECISION_z))
            for (j = 0; j < k; ++j){
                *F(k,j) = MAGMA_Z_CNJG( *F(k,j) );
            }
            #endif*/

            magmablas_zgemv( MagmaNoTrans, m-rk, k,
                             c_neg_one, A(rk, 0), lda,
                                        F(k,  0), ldf,
                             c_one,     A(rk, k), ione );

            /*#if (defined(PRECISION_c) || defined(PRECISION_z))
            for (j = 0; j < k; ++j) {
                *F(k,j) = MAGMA_Z_CNJG( *F(k,j) );
            }
            #endif*/
        }
        
        /*  Generate elementary reflector H(k). */
        magma_zlarfg_gpu(m-rk, A(rk, k), A(rk + 1, k), &tau[k], &vn1[k], &dAkk[k]);
                
        //Akk = *A(rk, k);
        //*A(rk, k) = c_one;
        //magma_zgetvector( 1, A(rk, k), 1, &Akk,     1 );
        // this needs to be done outside zlarfg to avoid the race condition.
        magma_zsetvector( 1, &c_one,   1, A(rk, k), 1 );

        /* Compute Kth column of F:
           Compute  F(K+1:N,K) := tau(K)*A(RK:M,K+1:N)'*A(RK:M,K) on the GPU */
        if (k < n-1 || k > 0 ) magma_zgetvector( 1, &tau[k], 1, &tauk, 1 );
        if (k < n-1) {
            magmablas_zgemv( MagmaConjTrans, m-rk, n-k-1,
                         tauk,   A( rk,  k+1 ), lda,
                                 A( rk,  k   ), 1,
                         c_zero, F( k+1, k   ), 1 );
        }
        
        /* Incremental updating of F:
           F(1:N,K) := F(1:N,K) - tau(K)*F(1:N,1:K-1)*A(RK:M,1:K-1)'*A(RK:M,K). 
           F(1:N,K) := tau(K)*A(RK:M,K+1:N)'*A(RK:M,K) - tau(K)*F(1:N,1:K-1)*A(RK:M,1:K-1)'*A(RK:M,K)
                    := tau(K)(A(RK:M,K+1:N)' - F(1:N,1:K-1)*A(RK:M,1:K-1)') A(RK:M,K)  
           so, F is (updated A)*V */
        if (k > 0) {
            /*z__1 = MAGMA_Z_NEGATE( tauk );
            magmablas_zgemv( MagmaConjTrans, m-rk, k,
                             z__1,   A(rk, 0), lda,
                                     A(rk, k), ione,
                             c_zero, auxv, ione );*/

            magma_zgemv_kernel3<<< k, BLOCK_SIZE, 0, magma_stream >>>(m-rk, A(rk, 0), lda,
                                                                      A(rk, k), auxv, tau+k);

            /* I think we only need stricly lower-triangular part */
            magmablas_zgemv( MagmaNoTrans, n-k-1, k,
                             c_one, F(k+1,0), ldf,
                                    auxv,     ione,
                             c_one, F(k+1,k), ione );
        }
        
        /* Update the current row of A:
           A(RK,K+1:N) := A(RK,K+1:N) - A(RK,1:K)*F(K+1:N,1:K)'.               */
        if (k < n-1) {
            i__1 = n - k - 1;
            i__2 = k + 1;
            /* left-looking update of rows,                     *
             * since F=A'v with original A, so no right-looking */
            magma_zgemm( MagmaNoTrans, MagmaConjTrans, ione, i__1, i__2,
                         c_neg_one, A(rk, 0  ), lda,
                                    F(k+1,0  ), ldf,
                         c_one,     A(rk, k+1), lda ); 
        }
        
        /* Update partial column norms. */
        if (rk < min(m, n+offset)-1){
           magmablas_dznrm2_row_check_adjust(n-k-1, tol3z, &vn1[k+1], 
                                             &vn2[k+1], A(rk,k+1), lda, lsticcs); 

           #if defined(PRECISION_d) || defined(PRECISION_z)
               magma_dgetvector( 1, &lsticcs[0], 1, &lsticc, 1 );
           #else
               magma_sgetvector( 1, &lsticcs[0], 1, &lsticc, 1 );
           #endif
        }

        //*A(rk, k) = Akk;
        //magma_zsetvector( 1, &Akk, 1, A(rk, k), 1 );
        //magmablas_zlacpy(MagmaUpperLower, 1, 1, dAkk, 1, A(rk, k), 1);

        ++k;
    }
    // restore the diagonals
    magma_zcopymatrix( 1, k, dAkk, 1, A(offset, 0), lda+1 );

    // leave k as the last column done
    --k;
    *kb = k + 1;
    rk = offset + *kb - 1;

    /* Apply the block reflector to the rest of the matrix:
       A(OFFSET+KB+1:M,KB+1:N) := A(OFFSET+KB+1:M,KB+1:N) - 
                                  A(OFFSET+KB+1:M,1:KB)*F(KB+1:N,1:KB)'  */
    if (*kb < min(n, m - offset)) {
        i__1 = m - rk - 1;
        i__2 = n - *kb;
        
        magma_zgemm( MagmaNoTrans, MagmaConjTrans, i__1, i__2, *kb,
                     c_neg_one, A(rk+1, 0  ), lda,
                                F(*kb,  0  ), ldf,
                     c_one,     A(rk+1, *kb), lda );
    }

    /* Recomputation of difficult columns. */
    if( lsticc > 0 ) {
        // printf( " -- recompute dnorms --\n" );
        magmablas_dznrm2_check(m-rk-1, n-*kb, A(rk+1,*kb), lda,
                               &vn1[*kb], lsticcs);
#if defined(PRECISION_d) || defined(PRECISION_z)
        magma_dcopymatrix( n-*kb, 1, &vn1[*kb], *kb, &vn2[*kb], *kb);
#else   
        magma_scopymatrix( n-*kb, 1, &vn1[*kb], *kb, &vn2[*kb], *kb);
#endif  
    }
    magma_free(lsticcs);
    
    return MAGMA_SUCCESS;
} /* magma_zlaqps */
