#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal s

*/
#include "common_magma.h"

#define magmablas_sgemv_fermi magmablas_sgemv

#define num_threads 128
#define sgemv_bs 32
#define threadSize 128


__global__ void 
sgemvn_kernel1_fermi(
    int n, int m, int n1, float alpha,
    const float* A, int lda,
    const float *x, float beta, 
    float *y)
{
  int ind = blockIdx.x*num_threads + threadIdx.x;

  A += ind;

  float res = 0.f;

  for(int i=0; i<n1; i += sgemv_bs ){

    #pragma unroll
    for(int j=0; j < sgemv_bs ; j++){
       res += A[0] * x[j];
       A   += lda;
    }
        x += sgemv_bs;
  }

  if (m>n1){

     for(int j=0; j<(m-n1); j++){
         res += A[0] * x[j];
         A   += lda;
     }
  }

  if (ind<n)
     y[ind] = alpha * res + beta * y[ind];

}

__global__ void 
sgemvn_kernel2_fermi(
    int n, int m, int n1, float alpha,
    const float* A, int lda,
    const float *x, float beta, 
    float *y)
{
  int ind = blockIdx.x*num_threads + threadIdx.x;

  A += ind;
  x += threadIdx.x;

  float res = 0.f;

  __shared__ float buff[num_threads];
  for(int i=0; i<n1; i += num_threads ){
    __syncthreads();
    buff[threadIdx.x]  = x[i];

    __syncthreads();
    #pragma unroll
    for(int j=0; j < num_threads ; j++){
       res+=A[0]*buff[j];
       A+=lda;
    }
  }
  __syncthreads();

  if (m>n1){
     buff[threadIdx.x]  = x[n1];

     __syncthreads();
     for(int j=0; j<(m-n1); j++){
         res += A[0]*buff[j];
         A+=lda;
     }
  }

  if (ind<n)
     y[ind] = alpha * res + beta * y[ind];
}

extern "C" void
magmablas_sgemvn_fermi(
    magma_int_t n, magma_int_t m, float alpha,
    const float *A, magma_int_t lda,
    const float *x, float beta,
    float *y)
{
/*  -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

    Purpose
    =======

    This routine computes Y = alpha A x on the GPU.

    N      - (input) INTEGER.
             On entry, N specifies the number of rows of the matrix A.

    M      - (input) INTEGER.
             On entry, M specifies the number of columns of the matrix A

    A      - (input) SINGLE PRECISION array of dimension ( LDA, m ) on the GPU.
   
    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension m.
     
    Y      - (output) SINGLE PRECISION array of        dimension m. 
             On exit Y = alpha A X.

    ===================================================================== */

    magma_int_t blocks;
    if (n % num_threads==0)
        blocks = n/num_threads;
    else
        blocks = n/num_threads + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(num_threads, 1, 1);
    if(n<=8500) 
                sgemvn_kernel1_fermi<<< grid, threads, 0, magma_stream >>>(n, m, (m / sgemv_bs)*sgemv_bs, 
                                                   alpha, A, lda, x, beta,  y);
    else
                sgemvn_kernel2_fermi<<< grid, threads, 0, magma_stream >>>(n, m, (m / num_threads)*num_threads, 
                                                   alpha, A, lda, x, beta, y);
}



__global__ void 
sgemvt_kernel1_fermi(
    int m, int n, float alpha, int n1,
    const float* A, int lda,
    const float *x, float beta,
    float *y)
{
        int tx = threadIdx.x;

        __shared__ float sdata[threadSize];
        
        volatile float *smem;

        float res;
        res = 0.0f;
     
        for(int i=0; i<n1; i+= threadSize)
        {
                res += A[tx + i + lda * blockIdx.y] * x[tx + i];
        }

        
        if(m > n1)
        {
                if( tx + n1 <  m )
                {
                        res  += A[tx + n1 + lda *blockIdx.y] * x[tx + n1];
                }
                else 
                {
                        res  += 0.0f;
                }
        }        

        sdata[tx] = res;
        __syncthreads();
    
        
        for(int s=blockDim.x/2;s>32;s>>=1) 
        {
                if(tx<s)
                {
                    sdata[tx] += sdata[tx + s];
                }
                 __syncthreads();
        }

        if(tx < 32)
        {
                smem = sdata;
                smem[tx] += smem[tx + 32];
                smem[tx] += smem[tx + 16];
                smem[tx] += smem[tx +  8];
                smem[tx] += smem[tx +  4];
                smem[tx] += smem[tx +  2];
                smem[tx] += smem[tx +  1];
        }

    if( tx == 0 ) 
        {

                if (blockIdx.y < n)
                {
                        y[blockIdx.y] = sdata[0] * alpha + beta * y[blockIdx.y];
                }
        }
}


__global__ void 
sgemvt_kernel2_fermi(
    int m, int n, float alpha, int n1,
    const float* A, int lda,
    const float *x, float beta, 
    float *y)
{
  const int inx = threadIdx.x;
  const int iny = threadIdx.y;

  int ind  = iny + blockIdx.x * 16;
  ind = inx + ind * lda;
  int ind2 = inx + iny * 16;
  if (ind2>31)
     ind2-=32;

  A += ind;
  x += ind2;

  float res = 0.f;

  __shared__ float buff[32];
  __shared__ float la[16][17];

  for(int i=0; i<n1; i += 32 ){
     buff[ind2]  = x[i];
     #pragma unroll
     for(int j=0; j<4; j++)
        la[iny + j * 4][inx] = A[j* 4 * lda];

     __syncthreads();
     #pragma unroll
     for(int j=0; j < 4; j++)
       res += la[inx][iny*4+j]*buff[j+iny*4];

     A += 16;

     __syncthreads();
     //===========================================
     #pragma unroll
     for(int j=0; j<4; j++)
         la[iny+ j * 4][inx] = A[j* 4 * lda];

     __syncthreads();

     #pragma unroll
     for(int j=0; j < 4; j++)
        res += la[inx][iny*4+j]*buff[j+16+iny*4];
     A += 16;
  }

  __syncthreads(); // 1
  if (n>n1){
     if (ind2>=(n-n1))
        buff[ind2]=0.;
     else
        buff[ind2]  = x[n1];

     __syncthreads();
     #pragma unroll
     for(int j=0; j<4; j++)
         if (inx>=(n-n1))
            la[iny + j * 4][inx] =  0.f;
         else
            la[iny + j * 4][inx] = A[j* 4 * lda];

     __syncthreads();
     if (n-n1>4){
        #pragma unroll
        for(int j=0; j < 4; j++){
           ind =  j+iny*4;
           res += la[inx][ind]*buff[ind];
        }
        A += 16;
        __syncthreads();
        #pragma unroll
        for(int j=0; j<4; j++)
          if (inx+16>=(n-n1))
             la[iny+ j * 4][inx] = 0.f;
          else
             la[iny+ j * 4][inx] = A[j* 4* lda];

        __syncthreads();

        #pragma unroll
        for(int j=0; j < 4; j++){
           ind = j+4*iny;
           res += la[inx][ind]*buff[16+ind];
        }
     }
     else {
        #pragma unroll
        for(int j=0; j < 4; j++){
          ind = j+iny*4;
          res += la[inx][ind]*buff[ind];
        }
     }
  }

  __syncthreads();
  ind = inx + blockIdx.x * 16;
  la[inx][iny]= res;
  __syncthreads();
  if (ind<n && iny==0){
     res = la[inx][0] + la[inx][1] + la[inx][2] + la[inx][3];
     y[ind] = alpha*res + beta * y[ind];
  }
}

extern "C" void
magmablas_sgemvt1_fermi(
    magma_int_t m, magma_int_t n, float alpha,
    const float *A, magma_int_t lda,
    const float *x, float beta,
    float *y)
{


    dim3 grid    ( 1,  n,  1);
    dim3 threads ( threadSize,   1,  1);

    sgemvt_kernel1_fermi<<< grid, threads, 0, magma_stream >>>( m, n, alpha, ( m / threadSize) * threadSize,
                                       A, lda, x, beta, y);

                                                                          
}

extern "C" void
magmablas_sgemvt2_fermi(
    magma_int_t m, magma_int_t n, float alpha,
    const float *A, magma_int_t lda,
    const float *x, float beta,
    float *y)
{

    magma_int_t blocks;

    if (n % 16==0)
        blocks = n/16;
    else
        blocks = n/16 + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(16, 4, 1);

    sgemvt_kernel2_fermi<<< grid, threads, 0, magma_stream >>>(m, n, alpha, (m / 32)*32,
                                      A, lda, x, beta, y);
}

extern "C" void
magmablas_sgemvt_fermi(
    magma_int_t m, magma_int_t n, float alpha,
    const float *A, magma_int_t lda, 
    const float *x, float beta,
    float *y)
{
/*  -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

    Purpose
    =======

    This routine computes y = alpha *  A^t *  x on the GPU.

    M      - (input) INTEGER.
             On entry, M specifies the number of rows of the matrix A.

    N      - (input) INTEGER.
             On entry, N specifies the number of columns of the matrix A

    A      - (input) SINGLE PRECISION array of dimension ( LDA, n ) on the GPU.

    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension m.

    Y      - (output) SINGLE PRECISION array of dimension n.
             On exit Y = alpha A^t X.

    ===================================================================== */

      magmablas_sgemvt1_fermi(m, n, alpha, A, lda, x, beta, y);
    

}


extern "C" void
magmablas_sgemv_fermi(char trans,
                      magma_int_t m, magma_int_t n,
                      float alpha, 
                      const float *A, magma_int_t lda, 
                      const float *x, magma_int_t incx,
                      float beta,
                      float *z, magma_int_t incz)
{
/*  -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

    Purpose
    =======
    This routine computes:
    1) z =       A   x    if trans == 'N' or 'n', alpha == 1, beta == 0, 
                          and incx == incz == 1 (using magmablas code)
    2) z = alpha A^t x    if trans == 'T' or 't', beta == 0,
                          and incx == incz == 1 (using magmablas code)
    3) z = alpha A^trans x + beta z
                          otherwise, using CUBLAS.

   Arguments
   ==========
    TRANS  - CHARACTER*1
             On entry, TRANS specifies the operation to be performed as
             follows:
               TRANS = 'N' or 'n'   z := alpha*A *x + beta*z
               TRANS = 'T' or 't'   z := alpha*A'*x + beta*z

    M      - (input) INTEGER
             On entry, m specifies the number of rows of the matrix A.

    N      - (input) INTEGER
             On entry, n specifies the number of columns of the matrix A
 
    ALPHA  - REAL
             On entry, ALPHA specifies the scalar alpha.
             Unchanged on exit.

    A      - (input) SINGLE PRECISION array of dimension ( LDA, n ) on the GPU.
   
    LDA    - (input) INTEGER
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension 
             n if trans == 'n'
             m if trans == 't'
     
    INCX   - (input) Specifies the increment for the elements of X.
             INCX must not be zero. Unchanged on exit.
  
    BETA   - REAL
             On entry, BETA specifies the scalar beta. When BETA is
             supplied as zero then Y need not be set on input.
             Unchanged on exit.

    Z      - (output) SINGLE PRECISION array of        dimension 
             m if trans == 'n'
             n if trans == 't' 

    INCZ  - (input) Specifies the increment for the elements of Z.
            INCZ must not be zero. Unchanged on exit.
    ===================================================================== */

    if (incx == 1 && incz == 1 ) {
       if (trans == 'n' || trans == 'N')
           magmablas_sgemvn_fermi(m,  n, alpha, A, lda, x, beta, z);
       else if (trans == 't' || trans == 'T')
          magmablas_sgemvt_fermi(m,  n, alpha, A, lda, x, beta, z);
       else
          printf("trans = %c in sgemv_fermi is not available\n", trans);               
    }
    else
       hipblasSgemv(trans, m, n, alpha, A, lda, x, incx, beta, z, incz);
}

#undef num_threads
#undef sgemv_bs
#undef threadSize 
