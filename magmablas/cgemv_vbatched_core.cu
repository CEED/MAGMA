#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal c
       
       @author Tingxing Dong
       @author Azzam Haidar

*/
#include "magma_internal.h"
#include "magma_templates.h"

#define PRECISION_c

#include "gemv_template_kernel_vbatched.cuh"
#include "gemv_config/gemvn_param.h"
#include "gemv_config/gemvt_param.h"

#define version(s,v) s ## _V_ ## v

extern "C" void
magmablas_cgemv_vbatched_max_nocheck(
    magma_trans_t trans, magma_int_t* m, magma_int_t* n, 
    magmaFloatComplex alpha,
    magmaFloatComplex_ptr dA_array[], magma_int_t* ldda, 
    magmaFloatComplex_ptr dx_array[], magma_int_t* incx,
    magmaFloatComplex beta,
    magmaFloatComplex_ptr dy_array[], magma_int_t* incy, 
    magma_int_t batchCount, 
    magma_int_t max_m, magma_int_t max_n, magma_queue_t queue)
{
    if ( trans == MagmaNoTrans ) {                                                   
        if (max(max_m, max_n) <= 96) { // small size                         
            if (max_m < max_n) { // Fat matrix
                if ( max_m <= 16) 
                {    
                    gemvn_template_vbatched<magmaFloatComplex, version(N, 70)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else if ( max_m <= 32) 
                {
                    gemvn_template_vbatched<magmaFloatComplex, version(N, 100)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else if ( max_m <= 64)            
                {
                    gemvn_template_vbatched<magmaFloatComplex, version(N, 117)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else
                {
                    gemvn_template_vbatched<magmaFloatComplex, version(N, 131)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }  
            }
            else {  // Tall or square matrix
                if ( max_n <= 32) 
                {    
                    gemvn_template_vbatched<magmaFloatComplex, version(N, 129)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else
                {
                    gemvn_template_vbatched<magmaFloatComplex, version(N, 131)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }  
            }
        }
        else { // big size
            if (max_m < max_n) { // Fat matrix
                if (max_m <= 8)
                {
                    gemvn_template_vbatched<magmaFloatComplex, version(N, 36)>              
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else if (max_m <= 16)
                {
                    gemvn_template_vbatched<magmaFloatComplex, version(N, 70)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else if (max_m <= 32)
                {
                    gemvn_template_vbatched<magmaFloatComplex, version(N, 100)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else if (max_m <= 32)
                {
                    gemvn_template_vbatched<magmaFloatComplex, version(N, 116)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else
                {
                    gemvn_template_vbatched<magmaFloatComplex, version(N, 133)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
            }
            else { // Tall or square matrix
                if (max_m <= 256)
                {
                    gemvn_template_vbatched<magmaFloatComplex, version(N, 137)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else
                {
                    gemvn_template_vbatched<magmaFloatComplex, version(N, 140)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
            }
        }// big size        
    } else {
        if (max(max_m, max_n) <= 96) // small size
        {
            if (max_n <= 8)
            {
                gemvc_template_vbatched<magmaFloatComplex, version(T, 42)>             
                        ( trans, m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
            }
            else
            {
                gemvc_template_vbatched<magmaFloatComplex, version(T, 46)>             
                        ( trans, m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
            }
        }
        else // big size
        {
            if (max_m <= max_n) //  Fat or square matrix
            {    
                if (max_m <= 64)
                {
                    gemvc_template_vbatched<magmaFloatComplex, version(T, 47)>             
                        ( trans, m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else
                {
                    gemvc_template_vbatched<magmaFloatComplex, version(T, 90)>             
                        ( trans, m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
            }                           
            else// (max_m > n) Tall matrix
            {
                if (max_n <= 8)
                {
                    gemvc_template_vbatched<magmaFloatComplex, version(T, 130)>             
                        ( trans, m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
                else
                {
                    gemvc_template_vbatched<magmaFloatComplex, version(T, 90)>             
                        ( trans, m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, max_m, max_n, batchCount, queue );
                }
            }
        }        
    }
}
///////////////////////////////////////////////////////////////////////////////////////////////////
