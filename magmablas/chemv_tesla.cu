#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       Note: [ds] precisions generated from csymv_tesla.cu
       
*/
#include "common_magma.h"
#define PRECISION_c

/* The version for fermi can be found in chemv_fermi.cu */

#define hemv_bs          64
#define thread_x         64
#define thread_y          4
#define bank_shift       33
#define quarter_thread_x 16
#define half_thread_x    32

/*******************************************************************************
 *    Lower case, where n is multiple of block size (hemv_bs)
 */

__global__ void
chemv_kernel_tesla_L_special(
    int n, magmaFloatComplex alpha,
    const magmaFloatComplex * __restrict__ A, int lda,
    const magmaFloatComplex * __restrict__ x, int incx,
    magmaFloatComplex  beta,
    magmaFloatComplex * __restrict__ y, int incy,
    magmaFloatComplex * __restrict__ WC)
{
    int tx   = threadIdx.x;
    int ty   = threadIdx.y;
    int blkc = blockIdx.x;

    magmaFloatComplex res  = MAGMA_C_ZERO;
    magmaFloatComplex res_ = MAGMA_C_ZERO;
    magmaFloatComplex res1 = MAGMA_C_ZERO;

    // la must be at least half_thread_x*bank_shift = 32x33 = 1056;
    // quarter_thread_x*(thread_x+2) = 16*(64+2) = 1056
    __shared__ magmaFloatComplex la   [quarter_thread_x][thread_x+3]; /* Why +3? */
    __shared__ magmaFloatComplex buff [thread_x];
    __shared__ magmaFloatComplex buff2[thread_x];

    magmaFloatComplex tr[4];
    magmaFloatComplex b[4];

    int break_d   =  thread_x * blkc;
    const int td  = (thread_x * ty) + tx;
    int       tx_ = td % half_thread_x;
    int       ty_ = td / half_thread_x;

    WC +=  break_d + tx;
    x  += (break_d + tx)*incx;
    A  +=  break_d * (lda+1);
    A  += ty_*lda + tx_;

    // load x[block] into buff
    if ( ty == 0 ) {
        buff[tx] = x[0];
    } // obtain the vector x store in buff;

    tx = tx_; ty = ty_;

    #pragma unroll
    for(int j=0; j < half_thread_x; j += 8)
        la[0][ bank_shift * (ty_+j) + tx_] = A[ j * lda];
    __syncthreads();

    #pragma unroll
    for(int i=ty_*4; i<(ty_ * 4 + 4); i++) {
        if ( i < tx_ ) {
            la[0][bank_shift * tx_ + i] = hipConjf( la[0][ i * bank_shift + tx_] );
        }
        else
            la[0][bank_shift * tx_ + i] = la[0][ bank_shift * tx_ + i];
    }
    __syncthreads();

    #pragma unroll
    for(int j=0; j < 4; j++)
        res += hipConjf( la[0][bank_shift * tx_ + j + ty_ * 4] ) * buff[j + ty_ * 4];
    __syncthreads();

    la[0][bank_shift*tx_+ty_] = res;
    __syncthreads();

    if ( ty_== 0 ) {
        res1 = la[0][tx_*bank_shift+0]+la[0][tx_*bank_shift+1]
             + la[0][tx_*bank_shift+2]+la[0][tx_*bank_shift+3]
             + la[0][tx_*bank_shift+4]+la[0][tx_*bank_shift+5]
             + la[0][tx_*bank_shift+6]+la[0][tx_*bank_shift+7];
    }
    else {
        res1 = MAGMA_C_ZERO;
    }
    __syncthreads();

    res = MAGMA_C_ZERO;

    A += half_thread_x + half_thread_x*lda;

    #pragma unroll
    for(int j=0; j < half_thread_x; j += 8)
        la[0][bank_shift*(ty_+j)+tx_] = A[ j * lda];
    __syncthreads();

    #pragma unroll
    for(int i=ty_*4; i<(4+ty_*4); i++) {
        if ( i < tx_ ) {
            la[0][bank_shift*tx_+i] = hipConjf( la[0][bank_shift*i+tx_] );
        }
        else
            la[0][bank_shift*tx_+i] = la[0][bank_shift*tx_+i];
    }
    __syncthreads();

    #pragma unroll
    for(int j=0; j < 4; j++)
        res += hipConjf( la[0][bank_shift*tx_+j+ty_*4] ) * buff[half_thread_x + j + 4 * ty_];
    __syncthreads();
    la[0][bank_shift*tx_+ty_] = res;
    __syncthreads();

    magmaFloatComplex res2;
    res2 = MAGMA_C_ZERO;
    if ( ty_== 1 ) {
        res2 = la[0][tx_*bank_shift+0]+la[0][tx_*bank_shift+1]
             + la[0][tx_*bank_shift+2]+la[0][tx_*bank_shift+3]
             + la[0][tx_*bank_shift+4]+la[0][tx_*bank_shift+5]
             + la[0][tx_*bank_shift+6]+la[0][tx_*bank_shift+7];
    }
    else {
        res2 = MAGMA_C_ZERO;
    }
    __syncthreads();

    res = MAGMA_C_ZERO;

    A -= half_thread_x*lda;

    res_ = MAGMA_C_ZERO;

    #pragma unroll
    for(int j=0; j < half_thread_x; j += 8)
        tr[j/8] = A[ j * lda];

    #pragma unroll
    for(int j=0; j < 4; j++) {
        res += tr[j] * buff[ j*8 + ty_];
        la[0][bank_shift*(ty_+j*8)+tx_] = tr[j];
    }
    __syncthreads();

    #pragma unroll
    for(int j=0; j < 4; j++)
        res_ += hipConjf(la[0][bank_shift*tx_+j+ty_*4]) * buff[half_thread_x +j+ty_*4];
    __syncthreads();

    la[0][bank_shift*tx_+ty_] = res;
    __syncthreads();
    if ( ty_ == 1 ) {
        res2 = res2
             + la[0][tx_*bank_shift+0]+la[0][tx_*bank_shift+1]
             + la[0][tx_*bank_shift+2]+la[0][tx_*bank_shift+3]
             + la[0][tx_*bank_shift+4]+la[0][tx_*bank_shift+5]
             + la[0][tx_*bank_shift+6]+la[0][tx_*bank_shift+7];
    }
    else {
        res2 = MAGMA_C_ZERO;
    }
    __syncthreads();

    la[0][bank_shift*tx_+ty_] = res_;
    __syncthreads();
    if ( ty_ == 0 ) {
        res1 = res1
             + la[0][tx_*bank_shift+0]+la[0][tx_*bank_shift+1]
             + la[0][tx_*bank_shift+2]+la[0][tx_*bank_shift+3]
             + la[0][tx_*bank_shift+4]+la[0][tx_*bank_shift+5]
             + la[0][tx_*bank_shift+6]+la[0][tx_*bank_shift+7];
    }
    else {
        res1 = MAGMA_C_ZERO;
    }
    A -= half_thread_x;

    __syncthreads();
    tx = threadIdx.x;
    ty = threadIdx.y;

    if ( ty_ == 0  && ty == 0  )
        res = res1;
    else if ( ty_ == 1  && ty == 0  )
        res = res2;
    else {
            res = MAGMA_C_ZERO;
        }

    A -= ty_* lda;
    A -= tx_;

    A = A - lda * blkc * thread_x;
    x = x - blkc * thread_x * incx;

    A += 4 * ty* lda;
    A += tx;

    int wc_c = 0;
    int count = 0;

    tx_ = td % quarter_thread_x;
    ty_ = td / quarter_thread_x;

    WC -= tx;
    WC += tx_;

    if ( blkc * thread_x >= thread_x ) {
        #pragma unroll
        for( int i=0; i < thread_x; i += thread_x ) {
            res_ = MAGMA_C_ZERO;
            count++;
            if ( ty == 0 )
                buff2[tx] = x[i*incx];
            __syncthreads();

            #pragma unroll
            for( int k=0; k < 4; k++ ) {
                #pragma unroll
                for(int j=0; j < 4; j++)
                    tr[j] = A[j*lda];

                #pragma unroll
                for(int j=0; j < 4; j++) {
                    res += tr[j] * buff2[ quarter_thread_x * k + ty*4 + j];
                    la[j + ty*4][tx] = hipConjf(tr[j]) * buff[tx];
                }
                __syncthreads();

                res_ = MAGMA_C_ZERO;

                #pragma unroll
                for(int j=0; j < 4; j++) {
                    res_ += la[tx_][ty_*4+j];
                }
                b[k] = res_;
                __syncthreads();

                A += lda * quarter_thread_x;
            }

            #pragma unroll
            for(int k=0; k < 4; k++) {
                la[tx_][ty_+quarter_thread_x*k] = b[k];
            }
            __syncthreads();
            if ( ty_ < 4 ) {
                int k = ty_*quarter_thread_x;
                res_ = la[tx_][0+k] + la[tx_][1+k]
                     + la[tx_][2+k] + la[tx_][3+k]
                     + la[tx_][4+k] + la[tx_][5+k]
                     + la[tx_][6+k] + la[tx_][7+k]
                     + la[tx_][8+k] + la[tx_][9+k]
                     + la[tx_][10+k]+ la[tx_][11+k]
                     + la[tx_][12+k]+ la[tx_][13+k]
                     + la[tx_][14+k]+ la[tx_][15+k];
                WC[k + wc_c*lda ] =   res_;
            }

            wc_c++;
            __syncthreads();
        }
    }
    
    for(int i=thread_x; i < (blkc * thread_x); i += thread_x ) {
        res_ = MAGMA_C_ZERO;
        count++;
        if ( ty == 0 )
            buff2[tx] = x[i*incx];
        __syncthreads();

        #pragma unroll
        for( int k=0; k < 4; k++ ) {
            #pragma unroll
            for(int j=0; j < 4; j++)
                tr[j] = A[j*lda];

            #pragma unroll
            for(int j=0; j < 4; j++) {
                res += tr[j] * buff2[ quarter_thread_x*k + ty*4 + j];
                la[j + ty*4][tx] = hipConjf( tr[j] ) * buff[tx];
            }
            __syncthreads();

            res_ = MAGMA_C_ZERO;

            #pragma unroll
            for(int j=0; j < 4; j++)
                res_ += la[tx_][ty_*4+j];

            b[k] = res_;
            __syncthreads();

            A += lda * quarter_thread_x;
        }

        #pragma unroll
        for(int k=0; k < 4; k++) {
            la[tx_][ty_+quarter_thread_x*k] = b[k];
        }
        __syncthreads();
        if ( ty_ < 4 ) {
            int k = ty_*quarter_thread_x;
            res_ = la[tx_][0+k] + la[tx_][1+k]
                 + la[tx_][2+k] + la[tx_][3+k]
                 + la[tx_][4+k] + la[tx_][5+k]
                 + la[tx_][6+k] + la[tx_][7+k]
                 + la[tx_][8+k] + la[tx_][9+k]
                 + la[tx_][10+k]+ la[tx_][11+k]
                 + la[tx_][12+k]+ la[tx_][13+k]
                 + la[tx_][14+k]+ la[tx_][15+k];
            WC[k + wc_c*lda ] =   res_;
        }

        wc_c++;
        __syncthreads();
    }

    WC += tx;
    WC -= tx_;

    la[ty][tx] = res;
    __syncthreads();
    if ( ty == 0 ) {
        res = la[0][tx]+ la[1][tx]
            + la[2][tx]+ la[3][tx];
        WC[0+lda*(blkc)  ] =  res;
    }
}

/**************************************************************
 *    Lower case for generic sizes
 */
__global__ void
chemv_kernel_tesla_L_generic(
    int n, magmaFloatComplex alpha,
    const magmaFloatComplex * __restrict__ A, int lda,
    const magmaFloatComplex * __restrict__ x, int incx,
    magmaFloatComplex beta,
    magmaFloatComplex * __restrict__ y, int incy,
    magmaFloatComplex * __restrict__ WC,
    int m_mod_thread_x)
{
    int tx   = threadIdx.x;
    int ty   = threadIdx.y;
    int blkc = blockIdx.x;

    magmaFloatComplex res  = MAGMA_C_ZERO;
    magmaFloatComplex res_ = MAGMA_C_ZERO;
    magmaFloatComplex res1 = MAGMA_C_ZERO;

    __shared__ magmaFloatComplex la   [quarter_thread_x][thread_x+3];
    __shared__ magmaFloatComplex buff [thread_x];
    __shared__ magmaFloatComplex buff2[thread_x];

    magmaFloatComplex tr[4];
    magmaFloatComplex b[8];

    int break_d   =  thread_x * blkc;
    const int td  = (thread_x * ty) + tx;
    int       tx_ = td % half_thread_x;
    int       ty_ = td / half_thread_x;

    WC +=  break_d + tx;
    x += (break_d + tx) * incx;
    A +=  break_d * (lda+1);
    A += lda * ty_;

    int trackA;
    if ( blkc == ( gridDim.x - 1 ) ) {
        if ( ty == 0 ) {
            if ( tx > m_mod_thread_x ) {
                buff[tx] = MAGMA_C_ZERO;
            }
            else
                buff[tx]  = x[0];
        }
        if ( tx_ > m_mod_thread_x )
            trackA=m_mod_thread_x;
        else
            trackA=tx_;
        A += trackA;
    }
    else {
        if ( ty == 0 ) {
            buff[tx]  = x[0];
        }
        trackA = tx_;
        A += trackA;
    }

    // Somehow merging these two if - else creates problem
    // It could be a potential bug -- from synchronization or from cuda or compiler
    if ( blkc == ( gridDim.x - 1 ) ) {
        #pragma unroll
        for(int j=0; j < half_thread_x; j += 8) {
            if ( ( ty_ + j ) > m_mod_thread_x ) {
                la[0][bank_shift*(ty_+j)+tx_] = MAGMA_C_MAKE( 9999, 0 );
            }
            else
                la[0][bank_shift*(ty_+j)+tx_] = A[ j * lda];
        }
        A -= trackA;
    }
    else {
        #pragma unroll
        for(int j=0; j < half_thread_x; j += 8) {
            la[0][bank_shift*(ty_+j)+tx_] = A[ j * lda];
        }
    }
    tx = tx_;
    ty = ty_;
    __syncthreads();

    #pragma unroll
    for(int i=ty_*4; i<(ty_*4+4); i++) {
        if ( i < tx_ ) {
            la[0][bank_shift*tx_+i] = hipConjf(la[0][i*bank_shift+tx_]);
        }
        else
            la[0][bank_shift*tx_+i] = la[0][bank_shift*tx_+i];
    }
    __syncthreads();

    #pragma unroll
    for(int j=0; j < 4; j++)
        res += hipConjf(la[0][bank_shift*tx_+j+ty_*4]) * buff[j+ty_*4];
    __syncthreads();

    la[0][bank_shift*tx_+ty_] = res;
    __syncthreads();
    if ( ty_== 0 ) {
        res1 = la[0][tx_*bank_shift+0]
             + la[0][tx_*bank_shift+1]
             + la[0][tx_*bank_shift+2]
             + la[0][tx_*bank_shift+3]
             + la[0][tx_*bank_shift+4]
             + la[0][tx_*bank_shift+5]
             + la[0][tx_*bank_shift+6]
             + la[0][tx_*bank_shift+7];
    }
    else {
        res1 = MAGMA_C_ZERO;
    }
    __syncthreads();

    res = MAGMA_C_ZERO;

    if ( blkc == ( gridDim.x - 1 ) ) {
        if ( (tx_+half_thread_x) > m_mod_thread_x )
            trackA = m_mod_thread_x;
        else
            trackA = tx_ + half_thread_x;
        A += trackA+half_thread_x*lda;

        #pragma unroll
        for(int j=0; j < half_thread_x; j += 8) {
            if ( ( ty_ + j+half_thread_x ) > m_mod_thread_x ) {
                la[0][bank_shift*(ty_+j)+tx_] = MAGMA_C_MAKE( 99999, 0 );
            }
            else
                la[0][bank_shift*(ty_+j)+tx_] = A[ j * lda];
        }

        A -= trackA+half_thread_x*lda;
        A += tx_;
        A += half_thread_x + half_thread_x*lda;
    }
    else {
        A += half_thread_x + half_thread_x*lda;

        #pragma unroll
        for(int j=0; j < half_thread_x; j += 8) {
            la[0][bank_shift*(ty_+j)+tx_] = A[ j * lda];
        }
    }

    __syncthreads();
    #pragma unroll
    for(int i=ty_*4; i<(4+ty_*4); i++) {
        if ( i < tx_ ) {
            la[0][bank_shift*tx_+i] = hipConjf(la[0][bank_shift*i+tx_]);
        }
        else
            la[0][bank_shift*tx_+i] = la[0][bank_shift*tx_+i];
    }
    __syncthreads();

    #pragma unroll
    for(int j=0; j < 4; j++)
        res += hipConjf(la[0][bank_shift*tx_+j+ty_*4]) * buff[half_thread_x + j + 4 * ty_];
    __syncthreads();

    la[0][bank_shift*tx_+ty_] = res;
    __syncthreads();

    magmaFloatComplex res2;
    res2 = MAGMA_C_ZERO;
    if ( ty_== 1 ) {
        res2 = la[0][tx_*bank_shift+0]
             + la[0][tx_*bank_shift+1]
             + la[0][tx_*bank_shift+2]
             + la[0][tx_*bank_shift+3]
             + la[0][tx_*bank_shift+4]
             + la[0][tx_*bank_shift+5]
             + la[0][tx_*bank_shift+6]
             + la[0][tx_*bank_shift+7];
    }
    else {
        res2 = MAGMA_C_ZERO;
    }
    __syncthreads();

    res = MAGMA_C_ZERO;
    res_ = MAGMA_C_ZERO;

    A -= half_thread_x*lda;
    if ( blkc == ( gridDim.x - 1 ) ) {
        A -= tx_;
        if ( tx_ > m_mod_thread_x )
            trackA=m_mod_thread_x;
        else
            trackA=tx_;
        A += trackA;

        #pragma unroll
        for(int j=0; j < half_thread_x; j += 8)
            if ( ( ty_ + j ) > m_mod_thread_x ) {
                tr[j/8] = MAGMA_C_MAKE( 99999, 0 );
            }
            else
                tr[j/8] = A[ j * lda];
        A -= trackA;
        A += tx_;
    }
    else {
        #pragma unroll
        for(int j=0; j < half_thread_x; j += 8)
            tr[j/8] = A[ j * lda];
    }
    __syncthreads();

    #pragma unroll
    for(int j=0; j < 4; j++) {
        res += tr[j] * buff[ j*8 + ty_];
        la[0][bank_shift*(ty_+j*8)+tx_] = tr[j];
    }
    __syncthreads();

    #pragma unroll
    for(int j=0; j < 4; j++)
        res_ += hipConjf(la[0][bank_shift*tx_+j+ty_*4]) * buff[half_thread_x +j+ty_*4];
    __syncthreads();

    la[0][bank_shift*tx_+ty_] = res;
    __syncthreads();
    if ( ty_ == 1 ) {
        res2 = res2
             + la[0][tx_*bank_shift+0]
             + la[0][tx_*bank_shift+1]
             + la[0][tx_*bank_shift+2]
             + la[0][tx_*bank_shift+3]
             + la[0][tx_*bank_shift+4]
             + la[0][tx_*bank_shift+5]
             + la[0][tx_*bank_shift+6]
             + la[0][tx_*bank_shift+7];
    }
    else {
        res2 = MAGMA_C_ZERO;
    }
    __syncthreads();

    la[0][bank_shift*tx_+ty_] = res_;
    __syncthreads();

    if ( ty_ == 0 ) {
        res1 = res1
             + la[0][tx_*bank_shift+0]
             + la[0][tx_*bank_shift+1]
             + la[0][tx_*bank_shift+2]
             + la[0][tx_*bank_shift+3]
             + la[0][tx_*bank_shift+4]
             + la[0][tx_*bank_shift+5]
             + la[0][tx_*bank_shift+6]
             + la[0][tx_*bank_shift+7];
    }
    else {
        res1 = MAGMA_C_ZERO;
    }
    A -= half_thread_x;

    __syncthreads();
    tx = threadIdx.x;
    ty = threadIdx.y;

    if ( ty_ == 0  && ty == 0  )
        res = res1;
    else if ( ty_ == 1  && ty == 0  )
        res = res2;
    else {
        res = MAGMA_C_ZERO;
    }

    A -= ty_* lda;
    A -= tx_;

    A = A - lda*break_d;
    x = x - break_d*incx;

    A += 4 * ty* lda;

    if ( blkc  == ( gridDim.x - 1 ) ) {
        if ( tx <= m_mod_thread_x )
            A += tx;
        else
            A += m_mod_thread_x;
    }
    else{
        A += tx;
    }

    int wc_c = 0;
    int count = 0;

    tx_ = td % quarter_thread_x;
    ty_ = td / quarter_thread_x;

    WC -= tx;
    WC += tx_;

    #pragma unroll
    for(int j=0; j < 4; j++)
        b[j] =  buff[ty_*4+j];

    if ( break_d > 0 )
        #pragma unroll
        for( int i=0; i < thread_x; i += thread_x ) {
            res_ = MAGMA_C_ZERO;
            count++;
            if ( ty == 0 ) {
                buff2[tx]  = x[i*incx];
            }
            __syncthreads();

            #pragma unroll
            for( int k=0; k < 4; k++ ) {
                #pragma unroll
                for(int j=0; j < 4; j++)
                    tr[j] = A[j*lda];

                #pragma unroll
                for(int j=0; j < 4; j++) {
                    res += tr[j]*buff2[quarter_thread_x*k + ty*4+(j)];
                    la[( (j)+ty*4)][tx] = hipConjf(tr[j]);
                }
                __syncthreads();

                res_ = MAGMA_C_ZERO;

                #pragma unroll
                for(int j=0; j < 4; j++)
                    res_ += la[tx_][ty_*4+j]* b[j];
                b[4+k] = res_;
                __syncthreads();
                A += lda* quarter_thread_x;
            }

            #pragma unroll
            for(int k=0; k < 4; k++) {
                la[tx_][ty_+quarter_thread_x*k] = b[4+k];
            }
            __syncthreads();

            if ( ty_ < 4 ) {
                int k = ty_*quarter_thread_x;
                res_ = la[tx_][0+k] + la[tx_][1+k]
                     + la[tx_][2+k] + la[tx_][3+k]
                     + la[tx_][4+k] + la[tx_][5+k]
                     + la[tx_][6+k] + la[tx_][7+k]
                     + la[tx_][8+k] + la[tx_][9+k]
                     + la[tx_][10+k]+ la[tx_][11+k]
                     + la[tx_][12+k]+ la[tx_][13+k]
                     + la[tx_][14+k]+ la[tx_][15+k];
                WC[k + wc_c*lda ] =   res_;
            }
            wc_c++;
            __syncthreads();
        }

    for( int i=thread_x; i < break_d; i += thread_x ) {
        res_ = MAGMA_C_ZERO;
        count++;
        if ( ty == 0 )
            buff2[tx]  = x[i*incx];
        __syncthreads();

        #pragma unroll
        for( int k=0; k < 4; k++ ) {
            #pragma unroll
            for(int j=0; j < 4; j++)
                tr[j] = A[j*lda];
            #pragma unroll
            for(int j=0; j < 4; j++) {
                res += tr[j]*buff2[quarter_thread_x*k + ty*4+(j)];
                la[( (j)+ty*4)][tx] = hipConjf(tr[j]);
            }
            __syncthreads();

            res_ = MAGMA_C_ZERO;

            #pragma unroll
            for(int j=0; j < 4; j++)
                res_ += la[tx_][ty_*4+j]* b[j];
            b[4+k] = res_;
            __syncthreads();
            A += lda* quarter_thread_x;
        }

        #pragma unroll
        for(int k=0; k < 4; k++) {
            la[tx_][ty_+quarter_thread_x*k] = b[4+k];
        }
        __syncthreads();

        if ( ty_ < 4 ) {
            int k = ty_*quarter_thread_x;
            res_ = la[tx_][0+k] + la[tx_][1+k]
                 + la[tx_][2+k] + la[tx_][3+k]
                 + la[tx_][4+k] + la[tx_][5+k]
                 + la[tx_][6+k] + la[tx_][7+k]
                 + la[tx_][8+k] + la[tx_][9+k]
                 + la[tx_][10+k]+ la[tx_][11+k]
                 + la[tx_][12+k]+ la[tx_][13+k]
                 + la[tx_][14+k]+ la[tx_][15+k];
            WC[k + wc_c*lda ] =   res_;
        }
        wc_c++;
        __syncthreads();
    }

    WC += tx;
    WC -= tx_;
    la[ty][tx] = res;
    __syncthreads();

    if ( ty == 0 ) {
        res=la[0][tx]+ la[1][tx]+ la[2][tx]+ la[3][tx];
        WC[0+lda*(blkc)] = res;
    }
}

__global__ void
chemv_kernel_tesla_L_update(
    int n, magmaFloatComplex alpha,
    const magmaFloatComplex * __restrict__ A, int lda,
    const magmaFloatComplex * __restrict__ x, int incx,
    magmaFloatComplex beta,
    magmaFloatComplex * __restrict__ y, int incy,
    magmaFloatComplex * __restrict__ WC )
{
    int i;
    int tx  = threadIdx.x;
    int ind = blockIdx.x * thread_x + tx;
    magmaFloatComplex Ca;

    Ca = MAGMA_C_ZERO;
    WC += ind + lda * blockIdx.x;

    for(i = blockIdx.x*thread_x; i < n; i += thread_x) {
        Ca += WC[0];
        WC += thread_x;
    }
    if ( ind < n )
        y[ind * incy] = beta * y[ind * incy]  + alpha * Ca;
}


extern "C"
void magmablas_chemv_tesla_L(
    magma_int_t n, magmaFloatComplex alpha,
    const magmaFloatComplex *A, magma_int_t lda,
    const magmaFloatComplex *x, magma_int_t incx,
    magmaFloatComplex beta,
    magmaFloatComplex *y, magma_int_t incy,
    magmaFloatComplex *dwork)
{
    magma_int_t blocks = (n - 1)/hemv_bs + 1;
    dim3 grid(blocks, 1, 1);
    dim3 threads(thread_x, thread_y, 1);

    /*
     * If matrix size is multiple of hemv_bs, we use a specific code.
     * otherwise, we call the generic case.
     */
    if ( n % hemv_bs == 0 ) {
        chemv_kernel_tesla_L_special<<< grid, threads, 0, magma_stream >>>
            (n, alpha, A, lda, x, incx, beta, y, incy, dwork);
    }
    else{
        magma_int_t m_mod_thread_x = (n % hemv_bs) - 1;
        chemv_kernel_tesla_L_generic<<< grid, threads, 0, magma_stream >>>
            (n, alpha, A, lda, x, incx, beta, y, incy, dwork, m_mod_thread_x);
    }

    dim3 threads_u(hemv_bs, 1, 1);
    chemv_kernel_tesla_L_update<<< grid, threads_u, 0, magma_stream >>>
        (n, alpha, A, lda, x, incx, beta, y, incy, dwork);
}

/**
    Purpose
    -------
    magmablas_chemv performs the matrix-vector operation:

        y := alpha*A*x + beta*y,

    where alpha and beta are scalars, x and y are n element vectors and
    A is an n by n hermitian matrix.

    Arguments
    ----------
    @param[in]
    uplo    CHARACTER*1.
            On entry, UPLO specifies whether the upper or lower
            triangular part of the array A is to be referenced as
            follows:
      -     = 'U':  Only the upper triangular part of A is to be referenced.
      -     = 'L':  Only the lower triangular part of A is to be referenced.


    @param[in]
    n       INTEGER.
            On entry, N specifies the order of the matrix A.
            N must be at least zero.

    @param[in]
    alpha   COMPLEX*16.
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    A       COMPLEX*16 array of DIMENSION ( LDA, n ).
            Before entry with UPLO = 'U' or 'u', the leading n by n
            upper triangular part of the array A must contain the upper
            triangular part of the hermitian matrix and the strictly
            lower triangular part of A is not referenced.
            Before entry with UPLO = 'L' or 'l', the leading n by n
            lower triangular part of the array A must contain the lower
            triangular part of the hermitian matrix and the strictly
            upper triangular part of A is not referenced.
            Note that the imaginary parts of the diagonal elements need
            not be set and are assumed to be zero.

    @param[in]
    lda     INTEGER.
            On entry, LDA specifies the first dimension of A as declared
            in the calling (sub) program. LDA must be at least
            max( 1, n ).
            It is recommended that lda is multiple of 16. Otherwise
            performance would be deteriorated as the memory accesses
            would not be fully coalescent.

    @param[in]
    x       COMPLEX*16 array of dimension at least
            ( 1 + ( n - 1 )*abs( INCX ) ).
            Before entry, the incremented array X must contain the n
            element vector x.

    @param[in]
    incx    INTEGER.
            On entry, INCX specifies the increment for the elements of
            X. INCX must not be zero.

    @param[in]
    beta    COMPLEX*16.
            On entry, BETA specifies the scalar beta. When BETA is
            supplied as zero then Y need not be set on input.

    @param[in,out]
    y       COMPLEX*16 array of dimension at least
            ( 1 + ( n - 1 )*abs( INCY ) ).
            Before entry, the incremented array Y must contain the n
            element vector y. On exit, Y is overwritten by the updated
            vector y.

    @param[in]
    incy    INTEGER.
            On entry, INCY specifies the increment for the elements of
            Y. INCY must not be zero.

    @ingroup magma_zblas2
    ********************************************************************/
extern "C"
magma_int_t
magmablas_chemv_tesla(
    magma_uplo_t uplo, magma_int_t n,
    magmaFloatComplex alpha,
    const magmaFloatComplex *A, magma_int_t lda,
    const magmaFloatComplex *x, magma_int_t incx,
    magmaFloatComplex beta,
    magmaFloatComplex *y, magma_int_t incy)
{
    int upper = (uplo == MagmaUpper);

    /*
     * Test the input parameters.
     */
    if ((! upper) && (uplo != MagmaLower)) {
        return -1;
    } else if ( n < 0 ) {
        return -2;
    } else if ( lda < max(1,n) ) {
        return -5;
    } else if ( incx == 0 ) {
        return -7;
    } else if ( incy == 0 ) {
        return -10;
    }

    /*
     * Quick return if possible.
     */
    if ( (n == 0) || ( MAGMA_C_EQUAL(alpha, MAGMA_C_ZERO) && MAGMA_C_EQUAL(beta, MAGMA_C_ONE) ) )
        return MAGMA_SUCCESS;

    /* TODO: Upper case is not implemented in MAGMA */
    if ( upper )
        hipblasChemv( lapacke_uplo_const(uplo), n, alpha, A, lda, x, incx, beta, y, incy);
    else {
        magmaFloatComplex *dwork;
        magma_int_t blocks = (n - 1)/thread_x + 1;
        magma_int_t lwork  = lda * (blocks + 1);

        // TODO deal with error
        magma_cmalloc( &dwork, lwork );

        magmablas_chemv_tesla_L(n, alpha, A, lda, x, incx, beta, y, incy, dwork);

        magma_free( dwork );
    }
    return MAGMA_SUCCESS;
}
