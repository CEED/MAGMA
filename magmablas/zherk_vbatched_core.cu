/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @author Jakub Kurzak
       @author Stan Tomov
       @author Mark Gates
       @author Azzam Haidar
       @author Ahmad Abdelfattah
       
*/

#include "magma_internal.h"
#include "magma_templates.h"

#define PRECISION_z

#include "herk_template_kernel_vbatched.cuh"
#include "gemm_config/zgemm_param_nn.h"
#include "gemm_config/zgemm_param_nt.h"
#include "gemm_config/zgemm_param_tn.h"
#include "gemm_config/zgemm_param_tt.h"

/******************************************************************************/
#define version(s,v) s ## _V_ ## v
template<int CONJ>
void
magmablas_zsyrkherk_vbatched(
    magma_uplo_t uplo, magma_trans_t trans, 
    magma_int_t* n, magma_int_t* k,
    magmaDoubleComplex alpha,
    magmaDoubleComplex const * const * dA_array, magma_int_t* ldda,
    magmaDoubleComplex const * const * dB_array, magma_int_t* lddb,
    magmaDoubleComplex beta,
    magmaDoubleComplex **dC_array, magma_int_t* lddc, 
    magma_int_t max_n, magma_int_t max_k, 
    magma_int_t batchCount, magma_queue_t queue )
{
    magmaDoubleComplex cbeta  = beta;
    magmaDoubleComplex calpha = alpha;

    // we have two shapes only
    magma_int_t shape;
    if   (trans == MagmaNoTrans) { shape = 0; } // nc or nt
    else                         { shape = 1; } // cn or tn
        
    switch(shape)
    {
        case 0: // nc or nt
            {
                if(max_k <= 8)
                {
                    // version 58
                    herk_template_vbatched_nt<magmaDoubleComplex, version(NT,58), 0, CONJ>
                    (uplo, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, calpha, cbeta, batchCount, queue, max_n);
                }
                else
                {
                    // version 29
                    herk_template_vbatched_nt<magmaDoubleComplex, version(NT,29), 0, CONJ>
                    (uplo, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, calpha, cbeta, batchCount, queue, max_n);
                }
            }
            break;
        case 1: // cn or tn
            {
                // version 72
                herk_template_vbatched_tn<magmaDoubleComplex, version(TN,72), CONJ, 0>
                (uplo, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, calpha, cbeta, batchCount, queue, max_n);
            }
            break;
        default:; // propose something
    }
}


/******************************************************************************/
extern "C" void
magmablas_zherk_internal_vbatched(
    magma_uplo_t uplo, magma_trans_t trans, 
    magma_int_t* n, magma_int_t* k,
    magmaDoubleComplex alpha,
    magmaDoubleComplex const * const * dA_array, magma_int_t* ldda,
    magmaDoubleComplex const * const * dB_array, magma_int_t* lddb,
    magmaDoubleComplex beta,
    magmaDoubleComplex **dC_array, magma_int_t* lddc, 
    magma_int_t max_n, magma_int_t max_k, 
    magma_int_t batchCount, magma_queue_t queue )
{
    magmablas_zsyrkherk_vbatched<1>(uplo, trans, n, k, alpha, dA_array, ldda, dB_array, lddb, beta, dC_array, lddc, max_n, max_k, batchCount, queue );
}


/******************************************************************************/
extern "C" void
magmablas_zsyrk_internal_vbatched(
    magma_uplo_t uplo, magma_trans_t trans, 
    magma_int_t* n, magma_int_t* k,
    magmaDoubleComplex alpha,
    magmaDoubleComplex const * const * dA_array, magma_int_t* ldda,
    magmaDoubleComplex const * const * dB_array, magma_int_t* lddb,
    magmaDoubleComplex beta,
    magmaDoubleComplex **dC_array, magma_int_t* lddc, 
    magma_int_t max_n, magma_int_t max_k, 
    magma_int_t batchCount, magma_queue_t queue )
{
    magmablas_zsyrkherk_vbatched<0>(uplo, trans, n, k, alpha, dA_array, ldda, dB_array, lddb, beta, dC_array, lddc, max_n, max_k, batchCount, queue );
}


/******************************************************************************/
extern "C" void 
magmablas_zsyrk_vbatched_max_nocheck(
    magma_uplo_t uplo, magma_trans_t trans, 
    magma_int_t* n, magma_int_t* k,
    magmaDoubleComplex alpha,
    magmaDoubleComplex const * const * dA_array, magma_int_t* ldda,
    magmaDoubleComplex beta,
    magmaDoubleComplex **dC_array, magma_int_t* lddc, 
    magma_int_t batchCount, 
    magma_int_t max_n, magma_int_t max_k, magma_queue_t queue )
{
    magmablas_zsyrk_internal_vbatched(uplo, trans, n, k, alpha, dA_array, ldda, dA_array, ldda, beta, dC_array, lddc, max_n, max_k, batchCount, queue );
}


/******************************************************************************/
extern "C" void 
magmablas_zherk_vbatched_max_nocheck(
    magma_uplo_t uplo, magma_trans_t trans, 
    magma_int_t* n, magma_int_t* k,
    double alpha,
    magmaDoubleComplex const * const * dA_array, magma_int_t* ldda,
    double beta,
    magmaDoubleComplex **dC_array, magma_int_t* lddc, 
    magma_int_t batchCount, 
    magma_int_t max_n, magma_int_t max_k, magma_queue_t queue )
{
    magmablas_zherk_internal_vbatched(uplo, trans, n, k, MAGMA_Z_MAKE(alpha, 0.), dA_array, ldda, dA_array, ldda, MAGMA_Z_MAKE(beta, 0.), dC_array, lddc, max_n, max_k, batchCount, queue );
}
