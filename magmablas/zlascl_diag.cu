#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c
*/
#include "magma_internal.h"

#define MB 64
#define NB 160


// each thread block does one NB x n block row of A.
// each thread does one row, starting from left edge and moving right to diagonal.
__global__ void
zlascl_diag_lower(
    int m, int n,
    const magmaDoubleComplex* D, int ldd,
    magmaDoubleComplex*       A, int lda)
{
    int ind_x = blockIdx.x * MB + threadIdx.x;
    int ind_y = blockIdx.y * NB;

    A += ind_x;
    if (ind_x < m) {
        for (int j=ind_y; j < min(ind_y+NB, n); j++ ) {
            A[j*lda] = MAGMA_Z_DIV( A[j*lda], D[j + j*ldd] );
        }
    }
}


// each thread block does one NB x n block row of A.
// each thread does one row, starting from right edge and moving left to diagonal.
__global__ void
zlascl_diag_upper(
    int m, int n,
    const magmaDoubleComplex* D, int ldd,
    magmaDoubleComplex*       A, int lda)
{
    int ind_x = blockIdx.x * MB + threadIdx.x;
    int ind_y = blockIdx.y * NB;

    A += ind_x;
    if (ind_x < m) {
        for (int j=ind_y; j < min(ind_y+NB, n); j++ ) {
            A[j*lda] = MAGMA_Z_DIV( A[j*lda], D[ind_x + ind_x*ldd] );
        }
    }
}


/***************************************************************************//**
    Purpose
    -------
    ZLASCL_DIAG scales the M by N complex matrix A by the real diagonal matrix dD.
    TYPE specifies that A may be upper triangular or lower triangular.

    Arguments
    ---------
    @param[in]
    type    magma_type_t
            TYPE indices the storage type of the input matrix A.
            = MagmaLower:  lower triangular matrix.
            = MagmaUpper:  upper triangular matrix.
            Other formats that LAPACK supports, MAGMA does not currently support.

    @param[in]
    m       INTEGER
            The number of rows of the matrix A.  M >= 0.

    @param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.

    @param[in]
    dD      DOUBLE PRECISION vector, dimension (LDDD,M)
            The matrix storing the scaling factor on its diagonal.

    @param[in]
    lddd    INTEGER
            The leading dimension of the array D.

    @param[in,out]
    dA      COMPLEX*16 array, dimension (LDDA,N)
            The matrix to be scaled by dD.  See TYPE for the
            storage type.

    @param[in]
    ldda    INTEGER
            The leading dimension of the array A.  LDDA >= max(1,M).

    @param[out]
    info    INTEGER
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_lascl_diag
*******************************************************************************/
extern "C" void
magmablas_zlascl_diag(
    magma_type_t type, magma_int_t m, magma_int_t n,
    magmaDoubleComplex_const_ptr dD, magma_int_t lddd,
    magmaDoubleComplex_ptr       dA, magma_int_t ldda,
    magma_queue_t queue,
    magma_int_t *info )
{
    *info = 0;
    if ( type != MagmaLower && type != MagmaUpper )
        *info = -1;
    else if ( m < 0 )
        *info = -2;
    else if ( n < 0 )
        *info = -3;
    else if ( lddd < max(1,m) )
        *info = -5;
    else if ( ldda < max(1,m) )
        *info = -7;
    
    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return;  //info;
    }
    
    dim3 threads( MB );
    dim3 grid( magma_ceildiv( m, MB ), magma_ceildiv( n, NB ) );
    
    if (type == MagmaLower) {
        zlascl_diag_lower
            <<< grid, threads, 0, queue->hip_stream() >>>
            (m, n, dD, lddd, dA, ldda);
    }
    else if (type == MagmaUpper) {
        zlascl_diag_upper
            <<< grid, threads, 0, queue->hip_stream() >>>
            (m, n, dD, lddd, dA, ldda);
    }
}
