#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c
*/
#include "common_magma.h"

#define NB 64


// each thread block does one NB x n block row of A.
// each thread does one row, starting from left edge and moving right to diagonal.
__global__ void
zlascl_diag_lower(int m, int n, magmaDoubleComplex const* D, int ldd, 
                                magmaDoubleComplex*       A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    A += ind;
    if (ind < m) {
        for (int j=0; j < n; j++ )
            A[j*lda] /= D[j + j*ldd];
    }
}


// each thread block does one NB x n block row of A.
// each thread does one row, starting from right edge and moving left to diagonal.
__global__ void
zlascl_diag_upper(int m, int n, magmaDoubleComplex const* D, int ldd, 
                                magmaDoubleComplex*       A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    A += ind;
    if (ind < m) {
        for (int j=0; j < n; j++ )
            A[j*lda] /= D[ind + ind*ldd];
    }
}


/**
    Purpose
    -------
    ZLASCL_DIAG scales the M by N complex matrix A by the real diagonal matrix dD.
    TYPE specifies that A may be full, upper triangular, lower triangular.

    Arguments
    ---------
    @param[in]
    type    magma_type_t
            TYPE indices the storage type of the input matrix A.
            = MagmaFull:   full matrix.
            = MagmaLower:  lower triangular matrix.
            = MagmaUpper:  upper triangular matrix.
            Other formats that LAPACK supports, MAGMA does not currently support.

    @param[in]
    m       INTEGER
            The number of rows of the matrix A.  M >= 0.

    @param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.

    @param[in]
    dD      DOUBLE PRECISION vector, dimension (LDDD,M)
            The matrix storing the scaling factor on its diagonal. 

    @param[in]
    lddd    INTEGER
            The leading dimension of the array D.  

    @param[in,out]
    dA      COMPLEX*16 array, dimension (LDDA,N)
            The matrix to be scaled by dD.  See TYPE for the
            storage type.

    @param[in]
    ldda    INTEGER
            The leading dimension of the array A.  LDDA >= max(1,M).

    @param[out]
    info    INTEGER
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_zaux2
    ********************************************************************/
extern "C" void
magmablas_zlascl_diag_q(
    magma_type_t type, magma_int_t m, magma_int_t n,
    magmaDoubleComplex_const_ptr dD, magma_int_t lddd, 
    magmaDoubleComplex_ptr       dA, magma_int_t ldda, 
    magma_int_t *info, magma_queue_t queue )
{
    *info = 0;
    if ( type != MagmaLower && type != MagmaUpper && type != MagmaFull )
        *info = -1;
    else if ( m < 0 )
        *info = -2;
    else if ( n < 0 )
        *info = -3;
    //else if ( ldda < max(1,m) )
    //    *info = -5;
    
    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return;  //info;
    }
    
    dim3 grid( magma_ceildiv( m, NB ) );
    dim3 threads( NB );
    
    if (type == MagmaLower) {
        zlascl_diag_lower <<< grid, threads, 0, queue >>> (m, n, dD, lddd, dA, ldda);
    }
    else if (type == MagmaUpper) {
        zlascl_diag_upper <<< grid, threads, 0, queue >>> (m, n, dD, lddd, dA, ldda);
    }
}


/**
    @see magmablas_zlascl2_q
    @ingroup magma_zaux2
    ********************************************************************/
extern "C" void
magmablas_zlascl_diag(
    magma_type_t type, magma_int_t m, magma_int_t n,
    magmaDoubleComplex_const_ptr dD, magma_int_t lddd, 
    magmaDoubleComplex_ptr       dA, magma_int_t ldda, 
    magma_int_t *info )
{
    magmablas_zlascl_diag_q( type, m, n, dD, lddd, dA, ldda, info, magma_stream );
}
