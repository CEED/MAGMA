#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010
*/
#include "common_magma.h"
#include "commonblas_d.h"

static __device__ void daxpy(double a,double *b, double *c) {
	c[0] += a * b[0];
	c[1] += a * b[1];
	c[2] += a * b[2];
	c[3] += a * b[3];
	c[4] += a * b[4];
	c[5] += a * b[5];
	c[6] += a * b[6];
	c[7] += a * b[7];
	c[8] += a * b[8];
	c[9] += a * b[9];
	c[10] += a * b[10];
	c[11] += a * b[11];
	c[12] += a * b[12];
	c[13] += a * b[13];
	c[14] += a * b[14];
	c[15] += a * b[15];
}

extern "C" __global__ void 
dgemm_kernel_N_N_64_16_16_16_4_special(double *C, const double *A, const double *B,
                                       int m, int n, int k, 
                                       int lda, int ldb, int ldc, 
                                       double alpha, double beta) 
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose:
    ========
    This routine computes
       C = alpha* A*B  + beta * C

    B is put into shared memory
    Parameters Used:
        blk_M=64 blk_N=16 blk_K=16 nthd_x=16 nthd_y=4

    This kernel is for matrices devisible by the corresponding
    blocking sizes.
    ===============================================================  */

	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	const int ibx = blockIdx.x * 64;
	const int iby = blockIdx.y *16;

	const int idt = ty * 16 + tx;

	B+=tx+__mul24(iby+ty,ldb);
	A += ibx + idt;
	C += ibx +idt +__mul24( iby,ldc);

	const double *Bend = B + k;


	double Cb[16] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
	m = 2*lda ; 
	n = 3*lda ;

	do {
		//double Ab[4] = {A[0], A[lda], A[2*lda], A[3*lda]};
		double Ab[4] = {A[0], A[lda], A[m], A[n]};
		__shared__ double Bb[16][17];
		Bb[tx][ty+0] = B[0];
		Bb[tx][ty+4] = B[4*ldb];
		Bb[tx][ty+8] = B[8*ldb];
		Bb[tx][ty+12] = B[12*ldb];

		__syncthreads();

		A += 4 * lda;
		daxpy(Ab[0], &Bb[0][0], Cb); Ab[0] = A[0];
		daxpy(Ab[1], &Bb[1][0], Cb); Ab[1] = A[lda];
		daxpy(Ab[2], &Bb[2][0], Cb); Ab[2] = A[m];
		daxpy(Ab[3], &Bb[3][0], Cb); Ab[3] = A[n];

		A += 4 * lda;
		daxpy(Ab[0], &Bb[4][0], Cb); Ab[0] = A[0];
		daxpy(Ab[1], &Bb[5][0], Cb); Ab[1] = A[lda];
		daxpy(Ab[2], &Bb[6][0], Cb); Ab[2] = A[m];
		daxpy(Ab[3], &Bb[7][0], Cb); Ab[3] = A[n];

		A += 4 * lda;
		daxpy(Ab[0], &Bb[8][0], Cb); Ab[0] = A[0];
		daxpy(Ab[1], &Bb[9][0], Cb); Ab[1] = A[lda];
		daxpy(Ab[2], &Bb[10][0], Cb); Ab[2] = A[m];
		daxpy(Ab[3], &Bb[11][0], Cb); Ab[3] = A[n];

		A += 4 * lda;
		daxpy(Ab[0], &Bb[12][0], Cb);
		daxpy(Ab[1], &Bb[13][0], Cb);
		daxpy(Ab[2], &Bb[14][0], Cb);
		daxpy(Ab[3], &Bb[15][0], Cb);

		B += 16;

		__syncthreads();
	} while (B < Bend);

	#pragma unroll 16
	for (int i = 0; i < 16; i++, C += ldc) {
		C[0] =alpha*Cb[i] + beta * C[0];
	}
}

extern "C" void
magmablas_dgemm_kernel_N_N_64_16_16_16_4_special(double *C, 
                                                 const double *A, 
                                                 const double *B, 
                                                 int m, int n, int k,
                                                 int lda, int ldb, int ldc,
                                                 double alpha, double beta)
{
        dim3 threads( 16, 4 );
        dim3 grid(m/64,n/16);
        dgemm_kernel_N_N_64_16_16_16_4_special<<< grid, threads, 0, magma_stream >>>(C, A, B,
                                                                    m, n, k,
                                                                    lda, ldb, ldc, 
                                                                    alpha, beta);
}

