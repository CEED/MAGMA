#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c
       
       @author Azzam Haidar
       @author Tingxing Dong
*/
#include "common_magma.h"
#include "batched_kernel_param.h"

#define BLK_SIZE 256
// SWP_WIDTH is number of threads in a block
// 64 and 256 are better on Kepler; 
extern __shared__ magmaDoubleComplex shared_data[];


//=================================================================================================
static __device__ 
void zlaswp_rowparallel_devfunc(  
                              int n, int width, int height,
                              magmaDoubleComplex *dA, int lda, 
                              magmaDoubleComplex *dout, int ldo,
                              magma_int_t* pivinfo)
{

    //int height = k2- k1;
    //int height = blockDim.x;
    unsigned int tid = threadIdx.x;
    dA   += SWP_WIDTH * blockIdx.x * lda;
    dout += SWP_WIDTH * blockIdx.x * ldo;
    magmaDoubleComplex *sdata = shared_data;

    if(blockIdx.x == gridDim.x -1)
    {
       width = n - blockIdx.x * SWP_WIDTH;
    }

    if(tid < height)
    {
        int mynewroworig = pivinfo[tid]-1; //-1 to get the index in C
        int itsreplacement = pivinfo[mynewroworig] -1 ; //-1 to get the index in C
        #pragma unroll
        for(int i=0; i<width; i++)
        {
          sdata[ tid + i * height ]    = dA[ mynewroworig + i * lda ];
          dA[ mynewroworig + i * lda ] = dA[ itsreplacement + i * lda ];
        }
    }
    __syncthreads();

    if(tid < height)
    {
        // copy back the upper swapped portion of A to dout 
        #pragma unroll
        for(int i=0; i<width; i++)
        {
           dout[tid + i * ldo] = sdata[tid + i * height];
        }
    }
}

//=================================================================================================
// parallel swap the swaped dA(1:nb,i:n) is stored in dout 
//=================================================================================================
__global__ 
void zlaswp_rowparallel_kernel( 
                                int n, int width, int height,
                                magmaDoubleComplex *dinput, int ldi, 
                                magmaDoubleComplex *doutput, int ldo,
                                magma_int_t*  pivinfo)
{

    zlaswp_rowparallel_devfunc(n, width, height, dinput, ldi, doutput, ldo, pivinfo);

}
//=================================================================================================

__global__ 
void zlaswp_rowparallel_kernel_batched(
                                int n, int width, int height,
                                magmaDoubleComplex **input_array, int ldi, 
                                magmaDoubleComplex **output_array, int ldo,
                                magma_int_t** pivinfo_array)
{
    int batchid = blockIdx.z;
    zlaswp_rowparallel_devfunc(n, width, height, input_array[batchid], ldi, output_array[batchid], ldo, pivinfo_array[batchid]);
}


//=================================================================================================

//=================================================================================================
extern "C" void
magma_zlaswp_rowparallel_batched_q( magma_int_t n, 
                       magmaDoubleComplex** input_array, magma_int_t ldi,
                       magmaDoubleComplex** output_array, magma_int_t ldo,
                       magma_int_t k1, magma_int_t k2,
                       magma_int_t **pivinfo_array, 
                       magma_queue_t stream, magma_int_t batchCount )
{

    if(n == 0 ) return ;
    int height = k2-k1;
    if( height  > 1024) 
    {
       printf(" n=%d > 1024, not supported \n", n);

    }

    int blocks =  (n-1)/ SWP_WIDTH + 1;
    dim3  grid(blocks, 1, batchCount);

    if( n < SWP_WIDTH)
    {
        zlaswp_rowparallel_kernel_batched<<<grid, height, sizeof(magmaDoubleComplex) * height * n, stream >>>
                                           ( n, n, height, input_array, ldi, output_array, ldo, pivinfo_array ); 
    }
    else
    {
        zlaswp_rowparallel_kernel_batched<<< grid, height, sizeof(magmaDoubleComplex) * height * SWP_WIDTH , stream >>>
                                            (n, SWP_WIDTH, height, input_array, ldi, output_array, ldo, pivinfo_array ); 
 
    }
}

//=================================================================================================


extern "C" void
magma_zlaswp_rowparallel_batched( magma_int_t n, magmaDoubleComplex** input_array, magma_int_t ldi,
                   magmaDoubleComplex** output_array, magma_int_t ldo,
                   magma_int_t k1, magma_int_t k2,
                   magma_int_t **pivinfo_array, 
                   magma_int_t batchCount )
{
    magma_zlaswp_rowparallel_batched_q(n, input_array, ldi, output_array, ldo, k1, k2, pivinfo_array, magma_stream, batchCount);
}

//=================================================================================================




//=================================================================================================
extern "C" void
magma_zlaswp_rowparallel_q( magma_int_t n, 
                       magmaDoubleComplex* input, magma_int_t ldi,
                       magmaDoubleComplex* output, magma_int_t ldo,
                       magma_int_t k1, magma_int_t k2,
                       magma_int_t *pivinfo, 
                       magma_queue_t stream)
{
    if(n == 0 ) return ;
    int height = k2-k1;
    if( height  > MAX_NTHREADS) 
    {
       printf(" height=%d > %d, magma_zlaswp_rowparallel_q not supported \n", n,MAX_NTHREADS);

    }

    int blocks =  (n-1)/ SWP_WIDTH + 1;
    dim3  grid(blocks, 1, 1);

    if( n < SWP_WIDTH)
    {
        zlaswp_rowparallel_kernel<<<grid, height, sizeof(magmaDoubleComplex) * height * n, stream >>>
                                   ( n, n, height, input, ldi, output, ldo, pivinfo ); 
    }
    else
    {
        zlaswp_rowparallel_kernel<<< grid, height, sizeof(magmaDoubleComplex) * height * SWP_WIDTH , stream >>>
                                    (n, SWP_WIDTH, height, input, ldi, output, ldo, pivinfo ); 
    }
}


//=================================================================================================

extern "C" void
magma_zlaswp_rowparallel( magma_int_t n, magmaDoubleComplex* input, magma_int_t ldi,
                   magmaDoubleComplex* output, magma_int_t ldo,
                   magma_int_t k1, magma_int_t k2,
                   magma_int_t *pivinfo)
{
    magma_zlaswp_rowparallel_q(n, input, ldi, output, ldo, k1, k2, pivinfo, magma_stream);
}

//=================================================================================================





//=================================================================================================
//  serial swap that does swapping one row by one row
//=================================================================================================
__global__ void zlaswp_rowserial_kernel_batched( int n, magmaDoubleComplex **dA_array, int lda, int k1, int k2, magma_int_t** ipiv_array )
{
    magmaDoubleComplex* dA = dA_array[blockIdx.z];
    magma_int_t *d_ipiv = ipiv_array[blockIdx.z];
    
    unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
    
    k1--;
    k2--;

    if( tid < n) {

        magmaDoubleComplex A1;

        for( int i1 = k1; i1 < k2; i1++ ) 
        {
            int i2 = d_ipiv[i1] - 1;  // Fortran index, switch i1 and i2
            if( i2 != i1)
            {
                A1 = dA[i1 + tid * lda];
                dA[i1 + tid * lda] = dA[i2 + tid * lda];
                dA[i2 + tid * lda] = A1;
            }
        }
    }
}

//=================================================================================================
//  serial swap that does swapping one row by one row, similar to LAPACK
//  K1, K2 are in Fortran indexing  
//=================================================================================================
extern "C" void
magma_zlaswp_rowserial_batched_q(magma_int_t n, magmaDoubleComplex** dA_array, magma_int_t lda,
                   magma_int_t k1, magma_int_t k2,
                   magma_int_t **ipiv_array, 
                   magma_queue_t stream, magma_int_t batchCount)
{

    if(n == 0 ) return ;

    int blocks =  (n-1)/ BLK_SIZE + 1;
    dim3  grid(blocks, 1, batchCount);

    zlaswp_rowserial_kernel_batched<<< grid, max(BLK_SIZE, n), 0, stream >>>(
        n, dA_array, lda, k1, k2, ipiv_array); 

}

extern "C" void
magma_zlaswp_rowserial_batched(magma_int_t n, magmaDoubleComplex** dA_array, magma_int_t lda,
                   magma_int_t k1, magma_int_t k2,
                   magma_int_t **ipiv_array, 
                   magma_int_t batchCount)
{
    magma_zlaswp_rowserial_batched_q(n, dA_array, lda, k1, k2, ipiv_array,  magma_stream, batchCount);
}




//=================================================================================================
//  serial swap that does swapping one column by one column
//=================================================================================================
__global__ void zlaswp_columnserial_kernel_batched( int n, magmaDoubleComplex **dA_array, int lda, int k1, int k2, magma_int_t** ipiv_array )
{
    magmaDoubleComplex* dA = dA_array[blockIdx.z];
    magma_int_t *d_ipiv = ipiv_array[blockIdx.z];
    
    unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
    k1--;
    k2--;
    if( k1 < 0 || k2 < 0 ) return;


    if( tid < n) {
        magmaDoubleComplex A1;
        if(k1 <= k2)
        {
            for( int i1 = k1; i1 <= k2; i1++ ) 
            {
                int i2 = d_ipiv[i1] - 1;  // Fortran index, switch i1 and i2
                if( i2 != i1)
                {
                    A1 = dA[i1 * lda + tid];
                    dA[i1 * lda + tid] = dA[i2 * lda + tid];
                    dA[i2 * lda + tid] = A1;
                }
            }
        }else
        {
            for( int i1 = k1; i1 >= k2; i1-- ) 
            {
                int i2 = d_ipiv[i1] - 1;  // Fortran index, switch i1 and i2
                if( i2 != i1)
                {
                    A1 = dA[i1 * lda + tid];
                    dA[i1 * lda + tid] = dA[i2 * lda + tid];
                    dA[i2 * lda + tid] = A1;
                }
            }
        }
    }
}

//=================================================================================================
//  serial swap that does swapping one column by one column
//  K1, K2 are in Fortran indexing  
//=================================================================================================
extern "C" void
magma_zlaswp_columnserial_batched_q(magma_int_t n, magmaDoubleComplex** dA_array, magma_int_t lda,
                   magma_int_t k1, magma_int_t k2,
                   magma_int_t **ipiv_array, 
                   magma_queue_t stream, magma_int_t batchCount)
{

    if(n == 0 ) return ;

    int blocks =  (n-1)/ BLK_SIZE + 1;
    dim3  grid(blocks, 1, batchCount);

    zlaswp_columnserial_kernel_batched<<< grid, min(BLK_SIZE, n), 0, stream >>>(
        n, dA_array, lda, k1, k2, ipiv_array); 

}

extern "C" void
magma_zlaswp_columnserial_batched(magma_int_t n, magmaDoubleComplex** dA_array, magma_int_t lda,
                   magma_int_t k1, magma_int_t k2,
                   magma_int_t **ipiv_array, 
                   magma_int_t batchCount)
{
    magma_zlaswp_columnserial_batched_q(n, dA_array, lda, k1, k2, ipiv_array,  magma_stream, batchCount);
}

