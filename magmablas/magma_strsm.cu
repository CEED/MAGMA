#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 0.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       October 2009
*/

#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "magmablas.h"

#define BLOCK_SIZE 32 

__global__ void
inplace_sgemm_kernel_T(int M, float alpha, float *A, int lda, float *B, int ldb)
{
        int i;
        float myvalue1=0, myvalue2= 0 ;
        float med;
        int tx = threadIdx.x;
        int ty = threadIdx.y;
        const int bx = blockIdx.x;
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE+1];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        A+= bx*32 + __mul24(lda,ty) + tx ;
        B+=      __mul24(ldb,ty) + tx ;


        As[tx][ ty]=A[0];
        As[tx][ty+16]= A[16*lda];
        Bs[tx][ty]= B[0];
        Bs[tx][ty+16]= B[16*ldb];

        __syncthreads();
        med  = As[tx][0];
        float py1 = Bs[ty][0] ;
        float py2 = Bs[ty+16][0] ;
        #pragma unroll
        for (i=0; i<31; i++){
                myvalue1 +=  med*py1;
                py1 = Bs[ty][i+1] ;
                myvalue2 +=  med*py2;
                py2 = Bs[ty+16][i+1] ;
                med  = As[tx][i+1];
        }
        myvalue1 +=  med*py1;
        myvalue2 +=  med*py2;
        A[0] = alpha*myvalue1 ;
        A[lda*16] = alpha*myvalue2;
}

__global__ void
inplace_sgemm_kernel_N(int M, float alpha, float *A, int lda, float *B, int ldb)
{
        int i;
        float myvalue1=0, myvalue2= 0 ;
        float med;
        int tx = threadIdx.x;
        int ty = threadIdx.y;
        const int bx = blockIdx.x;
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE+1];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        A+= bx*32 + __mul24(lda,ty) + tx ;
        B+=      __mul24(ldb,ty) + tx ;


        As[tx][ ty]=A[0];
        As[tx][ty+16]= A[16*lda];
        Bs[ty][tx]= B[0];
        Bs[ty+16][tx]= B[16*ldb];

        __syncthreads();
        med  = As[tx][0];
        float py1 = Bs[ty][0] ;
        float py2 = Bs[ty+16][0] ;
        #pragma unroll
        for (i=0; i<31; i++){
                myvalue1 +=  med*py1;
                py1 = Bs[ty][i+1] ;
                myvalue2 +=  med*py2;
                py2 = Bs[ty+16][i+1] ;
                med  = As[tx][i+1];
        }
        myvalue1 +=  med*py1;
        myvalue2 +=  med*py2;
        A[0] = alpha*myvalue1 ;
        A[lda*16] = alpha*myvalue2;
}

__global__ void
diag_strtri_kernel (char uplo, char diag, float *A, float *d_dinvA, int lda)
{
	int i,j;
	float Ystx=0;
	float *Bw=NULL, *x=NULL, *y=NULL, *Aoff=NULL;
	float *my_d_dinvA;
	int switcher=0;

	// Thread index
	int tx = threadIdx.x;
	int txw;

	// Block index
	int bx = blockIdx.x;
		
	Aoff = A+bx*lda*BLOCK_SIZE+bx*BLOCK_SIZE;
	my_d_dinvA = d_dinvA+bx*BLOCK_SIZE*BLOCK_SIZE;

	__shared__ float Bs[BLOCK_SIZE*BLOCK_SIZE];
	__shared__ float workspace[BLOCK_SIZE]; // workspace used to store the current working column

	// load A
	#pragma unroll
	for (i=0; i<BLOCK_SIZE; i++)
		Bs[i*BLOCK_SIZE+tx] = *(Aoff+i*lda+tx);	// read in the whole square block of my A
												// not the upper or lower diagonal

	// Synchronize to make sure the matrices are loaded
	__syncthreads();

	Bs[tx*BLOCK_SIZE+tx] = ((diag=='u' || diag=='U')?1:(1/Bs[tx*BLOCK_SIZE+tx]));	// solve the diagonals

	if (uplo == 'l' || uplo == 'L')
	{
		/*
		 * the lower case
		 */
		if (tx < BLOCK_SIZE-1)
			Bs[(BLOCK_SIZE-1)*BLOCK_SIZE+tx] = 0;	//zero out the last column, except the diagonal element

		for (i=BLOCK_SIZE-2; i>=0; i--)
		{
			Ystx = 0;
			switcher = (tx>i);
			
			//strmv
			Bw = Bs+(i+1)*BLOCK_SIZE+i+1;
			workspace[tx] = *(Bs+i*BLOCK_SIZE+tx);
			x = workspace+i+1;
			y = Bs+i*BLOCK_SIZE;

			txw = (tx-i-1);

			#pragma unroll
			for (j=0; j<txw+1; j++)
				Ystx += (float)switcher*(*(Bw+j*BLOCK_SIZE+txw)*x[j]);

			//sscal
			switcher = (tx != i); 
			//if (tx !=i ) y[tx]=switcher*Ystx*(-Bs[i*BLOCK_SIZE+i]);
			y[tx] = (float)switcher*Ystx*(-Bs[i*BLOCK_SIZE+i])+(float)(!switcher)*y[tx];

			__syncthreads();
		}

	}
	else
	{
		 /* the upper case */
		for (i=0; i<BLOCK_SIZE; i++)
		{
			Ystx = 0;
			switcher = (float)(tx<i);
			
			//strmv
			workspace[tx] = *(Bs+i*BLOCK_SIZE+tx);
			y = Bs+i*BLOCK_SIZE;

			#pragma unroll
			for (j=tx; j<i; j++)
				Ystx += switcher*(*(Bs+j*BLOCK_SIZE+tx)*workspace[j]);

			//sscal
			switcher = (tx != i); // if (tx !=i ) y[tx]=switcher*Ystx*(-Bs[i*BLOCK_SIZE+i]);
			y[tx] = switcher*Ystx*(-Bs[i*BLOCK_SIZE+i])+!switcher*y[tx];

			__syncthreads();
		}


	}
		
	// write back A
	#pragma unroll
	for (i=0; i<BLOCK_SIZE; i++)
		*(my_d_dinvA+i*BLOCK_SIZE+tx) = Bs[i*BLOCK_SIZE+tx];
}

extern "C" void
inplace_sgemm (char tran, int M, float alpha, float *A, int lda, float *B, int ldb)
{
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE/2);

	if (tran == 'n' || tran == 'N')
		inplace_sgemm_kernel_N<<<M/BLOCK_SIZE,dimBlock>>>(M, alpha, A, lda, B, ldb); 
	else
		inplace_sgemm_kernel_T<<<M/BLOCK_SIZE,dimBlock>>>(M, alpha, A, lda, B, ldb); 
}

/*
 * magmablas_strsmx
 * the expert interface
 */
void magmablas_strsmx ( char side, char uplo, char tran, char diag, int M, int N, float alpha, float* A, int lda, float* b, int ldb, float *d_dinvA)
{
/*  -- MAGMA (version 0.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       October 2009

	   Purpose
	   =======
	   
	   STRSM  solves one of the matrix equations on GPU
	   
	      op( A )*X = alpha*B,   or   X*op( A ) = alpha*B,
	   
	   where alpha is a scalar, X and B are m by n matrices, A is a unit, or
	   non-unit,  upper or lower triangular matrix  and  op( A )  is one  of
	   
	      op( A ) = A   or   op( A ) = A'.
	   
	   The matrix X is overwritten on B.

	   When M or N is not a multiple of blocking size, which is 32 for now, hipblasStrsm will
	   be called instead. There soon will not be this limitation both for arbitrary problem 
	   size and blocking size.
	   
	   Arguments
	   ==========
	   
	   side   - CHARACTER*1.
	            On entry, side specifies whether op( A ) appears on the left
	            or right of X as follows:
	   
	               side = 'L' or 'l'   op( A )*X = alpha*B.
	   
	               side = 'R' or 'r'   X*op( A ) = alpha*B.
	   
	            Unchanged on exit.
	   
	   uplo   - CHARACTER*1.
	            On entry, uplo specifies whether the matrix A is an upper or
	            lower triangular matrix as follows:
	   
	               uplo = 'U' or 'u'   A is an upper triangular matrix.
	   
	               uplo = 'L' or 'l'   A is a lower triangular matrix.
	   
	            Unchanged on exit.
	   
	   tran - CHARACTER*1.
	            On entry, tran specifies the form of op( A ) to be used in
	            the matrix multiplication as follows:
	   
	               tran = 'N' or 'n'   op( A ) = A.
	   
	               tran = 'T' or 't'   op( A ) = A'.
	   
	               tran = 'C' or 'c'   op( A ) = A'.
	   
	            Unchanged on exit.
	   
	   diag   - CHARACTER*1.
	            On entry, diag specifies whether or not A is unit triangular
	            as follows:
	   
	               diag = 'U' or 'u'   A is assumed to be unit triangular.
	   
	               diag = 'N' or 'n'   A is not assumed to be unit
	                                   triangular.
	   
	            Unchanged on exit.
	   
	   m      - INTEGER.
	            On entry, m specifies the number of rows of B. m must be at
	            least zero.
	            Unchanged on exit.
	   
	    n      - INTEGER.
	             On entry, n specifies the number of columns of B.  n must be
	             at least zero.
	             Unchanged on exit.
	   
	    alpha  - REAL.
	             On entry,  alpha specifies the scalar  alpha. When  alpha is
	             zero then  A is not referenced and  B need not be set before
	             entry.
	             Unchanged on exit.
	   
	    A      - REAL             array of DIMENSION ( lda, k ), where k is m
	             when  side = 'L' or 'l'  and is  n  when  side = 'R' or 'r'.
	             Before entry  with  uplo = 'U' or 'u',  the  leading  k by k
	             upper triangular part of the array  A must contain the upper
	             triangular matrix  and the strictly lower triangular part of
	             A is not referenced.
	             Before entry  with  uplo = 'L' or 'l',  the  leading  k by k
	             lower triangular part of the array  A must contain the lower
	             triangular matrix  and the strictly upper triangular part of
	             A is not referenced.
	             Note that when  diag = 'U' or 'u',  the diagonal elements of
	             A  are not referenced either,  but are assumed to be  unity.
	             Unchanged on exit.
	   
	    lda    - INTEGER.
	             On entry, lda specifies the first dimension of A as declared
	             in the calling (sub) program.  When  side = 'L' or 'l'  then
	             lda  must be at least  max( 1, m ),  when  side = 'R' or 'r'
	             then lda must be at least max( 1, n ).
	             Unchanged on exit.
	   
	    b      - REAL             array of DIMENSION ( ldb, n ).
	             Before entry,  the leading  m by n part of the array  B must
	             contain  the  right-hand  side  matrix  B,  and  on exit  is
	             overwritten by the solution matrix  X.
	   
	    ldb    - INTEGER.
	             On entry, ldb specifies the first dimension of B as declared
	             in  the  calling  (sub)  program.   ldb  must  be  at  least
	             max( 1, m ).
	             Unchanged on exit.

		d_dinvA  REAL array of DIMENSION (BLOCKSIZE, M) when side='L', 
				 (BLOCKSIZE, N) when side='R'. On exit this space is filled
			     with the inverse of blocks on the diagonal, each inverse is
				 of size BLOCKSIZE x BLOCKSIZE, and the leading dimension of
				 d_dinvA is BLOCKSIZE;
	   
	   
	    Level 3 Blas routine.
		*
    ===================================================================== */

	int i, nblocks;
	
	/* quick return on wrong size */
	if (M<=0 || N<=0 || d_dinvA == NULL)
		return;

	/* 
	 * call hipblasStrsm when size of the problem is not a multiple of blocksize which is 32
	 * subject to change soon
	 */

	if ((M%BLOCK_SIZE)!=0 || (N>1 && (N%BLOCK_SIZE)!=0))
	{
		hipblasStrsm (side, uplo, tran, diag, M, N, alpha, A, lda, b, ldb);
		return;
	}

	if (side == 'l' || side == 'L')
	{
		/* inverse the diagonals
		 * Allocate device memory for the inversed diagonal blocks, size=m*BLOCK_SIZE 
		 */
		nblocks = M/BLOCK_SIZE;
		diag_strtri_kernel<<<nblocks, BLOCK_SIZE>>>(uplo, diag, A, d_dinvA, lda);

		if (tran == 'N' || tran == 'n')
		/* the non-transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				if (N == 1)
					magmablas_sgemv32 ('N', BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, b);
				else
					hipblasSgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, ldb, 0, b, ldb);  

				if (BLOCK_SIZE>=M)
					return;

				hipblasSgemm ('N', 'N', M-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+BLOCK_SIZE, lda, b, ldb, alpha, b+BLOCK_SIZE, ldb);

				/* the rest blocks */
				for (i=BLOCK_SIZE; i<M; i+=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_sgemv32 ('N', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						hipblasSgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  

					if (i+BLOCK_SIZE>=M)
						break;

					hipblasSgemm ('N', 'N', M-i-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+i*lda+i+BLOCK_SIZE, lda, b+i, ldb, 1.0, b+i+BLOCK_SIZE, ldb);
				}
			}
			else
			{
			/* the upper case */

				/* handle the first block seperately with alpha */
				i = M-BLOCK_SIZE;
				if (N == 1)
					magmablas_sgemv32 ('N', BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
				else
					hipblasSgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0.0, b+i, ldb); 
					
				if (i-BLOCK_SIZE<0)
					return;

				hipblasSgemm ('N', 'N', i, N, BLOCK_SIZE, -1.0, A+i*lda, lda, b+i, ldb, alpha, b, ldb);

				/* the rest blocks */
				for (i=M-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_sgemv32 ('N', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						hipblasSgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0.0, b+i, ldb); 

					if (i-BLOCK_SIZE<0)
						break;

					hipblasSgemm ('N', 'N', i, N, BLOCK_SIZE, -1.0, A+i*lda, lda, b+i, ldb, 1.0, b, ldb);
				}
			}
		}
		else
		/* the transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				i=M-BLOCK_SIZE; 
				if (N == 1)
					magmablas_sgemv32 ('T', BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
				else
					hipblasSgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  

				if (i-BLOCK_SIZE<0)
					return;

				hipblasSgemm ('T', 'N', i, N, BLOCK_SIZE, -1.0, A+i, lda, b+i, ldb, alpha, b, ldb);

				/* the rest blocks */
				for (i=M-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_sgemv32 ('T', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						hipblasSgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  

					if (i-BLOCK_SIZE<0)
						break;

					hipblasSgemm ('T', 'N', i, N, BLOCK_SIZE, -1.0, A+i, lda, b+i, ldb, 1.0, b, ldb);
				}
			}
			else
			{
			/* the upper case */
					
				/* handle the first block seperately with alpha */
				if (N == 1)
					magmablas_sgemv32 ('T', BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, b);
				else
					hipblasSgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, ldb, 0, b, ldb);  

				if (BLOCK_SIZE>=M)
					return;

				hipblasSgemm ('T', 'N', M-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+(BLOCK_SIZE)*lda, lda, b, ldb, alpha, b+BLOCK_SIZE, ldb);

				/* the rest blocks */
				for (i=BLOCK_SIZE; i<M; i+=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_sgemv32 ('T', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						hipblasSgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  
					
					if (i+BLOCK_SIZE>=M)
						break;

					hipblasSgemm ('T', 'N', M-i-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+(i+BLOCK_SIZE)*lda+i, lda, b+i, ldb, 1.0, b+i+BLOCK_SIZE, ldb);
				}
			}
		}
	}
	else
	{	// side=R

		/* inverse the diagonals
		 * Allocate device memory for the inversed diagonal blocks, size=N*BLOCK_SIZE 
		 */
		nblocks = N/BLOCK_SIZE;
		diag_strtri_kernel<<<nblocks, BLOCK_SIZE>>>(uplo, diag, A, d_dinvA, lda);
		
		if (tran == 'N' || tran == 'n')
		/* the non-transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				i=N-BLOCK_SIZE;
				inplace_sgemm ('N', M, alpha, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

				if (i-BLOCK_SIZE<0)
					return;

				hipblasSgemm ('N', 'N', M, i, BLOCK_SIZE, -1.0, b+ldb*i, ldb, A+i, lda, alpha, b, ldb);

				/* the rest blocks */
				for (i=N-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					inplace_sgemm ('N', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);
					
					if (i-BLOCK_SIZE<0)
						break;

					hipblasSgemm ('N', 'N', M, i, BLOCK_SIZE, -1.0, b+ldb*i, ldb, A+i, lda, 1.0, b, ldb);
				}
			}
			else
			{
			/* the upper case */
				
				/* handle the first block seperately with alpha */
				inplace_sgemm ('N', M, alpha, b, ldb, d_dinvA, BLOCK_SIZE);

				if (BLOCK_SIZE>=N)
					return;

				hipblasSgemm ('N', 'N', M, N-BLOCK_SIZE, BLOCK_SIZE, -1.0, b, ldb, A+(BLOCK_SIZE)*lda, lda, alpha, b+(BLOCK_SIZE)*ldb, ldb);
				
				
				/* the rest blocks */
				for (i=BLOCK_SIZE; i<N; i+=BLOCK_SIZE)
				{
					inplace_sgemm ('N', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

					if (i+BLOCK_SIZE>=N)
						break;

					hipblasSgemm ('N', 'N', M, N-i-BLOCK_SIZE, BLOCK_SIZE, -1.0, b+i*ldb, ldb, A+(i+BLOCK_SIZE)*lda+i, lda, 1.0, b+(i+BLOCK_SIZE)*ldb, ldb);
				}
			}
		}
		else
		/* the transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				inplace_sgemm ('T', M, alpha, b, ldb, d_dinvA, BLOCK_SIZE);

				if (BLOCK_SIZE>=N)
					return;

				hipblasSgemm ('N', 'T', M, N-BLOCK_SIZE, BLOCK_SIZE, -1.0, b, ldb, A+BLOCK_SIZE, lda, alpha, b+(BLOCK_SIZE)*ldb, ldb);

				/* the rest blocks */
				for (i=BLOCK_SIZE; i<N; i+=BLOCK_SIZE)
				{
					inplace_sgemm ('T', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

					if (i+BLOCK_SIZE>=N)
						break;

					hipblasSgemm ('N', 'T', M, N-i-BLOCK_SIZE, BLOCK_SIZE, -1.0, b+ldb*i, ldb, A+i*lda+BLOCK_SIZE+i, lda, 1.0, b+(i+BLOCK_SIZE)*ldb, ldb);
				}
			}
			else
			{
			/* the upper case */
				
				/* handle the first block seperately with alpha */
				i=N-BLOCK_SIZE;
				inplace_sgemm ('T', M, alpha, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

				if (i-BLOCK_SIZE<0)
					return;

				hipblasSgemm ('N', 'T', M, i, BLOCK_SIZE, -1.0, b+i*ldb, ldb, A+i*lda, lda, alpha, b, ldb);
				
				/* the rest blocks */
				for (i=N-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					inplace_sgemm ('T', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

					if (i-BLOCK_SIZE<0)
						break;

					hipblasSgemm ('N', 'T', M, i, BLOCK_SIZE, -1.0, b+i*ldb, ldb, A+i*lda, lda, 1.0, b, ldb);
				}
			}
		}
	}
}

/*
 * magmablas_strsm
 */
extern "C"
void magmablas_strsm ( char side, char uplo, char tran, char diag, int M, int N, float alpha, float* A, int lda, float* b, int ldb)
{
/*  -- MAGMA (version 0.2) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       October 2009

	   Purpose
	   =======
	   
	   STRSM  solves one of the matrix equations on GPU
	   
	      op( A )*X = alpha*B,   or   X*op( A ) = alpha*B,
	   
	   where alpha is a scalar, X and B are m by n matrices, A is a unit, or
	   non-unit,  upper or lower triangular matrix  and  op( A )  is one  of
	   
	      op( A ) = A   or   op( A ) = A'.
	   
	   The matrix X is overwritten on B.
	   
	   When M or N is not a multiple of blocking size, which is 32 for now, hipblasStrsm will
	   be called instead. There soon will not be this limitation both for arbitrary problem 
	   size and blocking size.
	   
	   
	   Arguments
	   ==========
	   
	   side   - CHARACTER*1.
	            On entry, side specifies whether op( A ) appears on the left
	            or right of X as follows:
	   
	               side = 'L' or 'l'   op( A )*X = alpha*B.
	   
	               side = 'R' or 'r'   X*op( A ) = alpha*B.
	   
	            Unchanged on exit.
	   
	   uplo   - CHARACTER*1.
	            On entry, uplo specifies whether the matrix A is an upper or
	            lower triangular matrix as follows:
	   
	               uplo = 'U' or 'u'   A is an upper triangular matrix.
	   
	               uplo = 'L' or 'l'   A is a lower triangular matrix.
	   
	            Unchanged on exit.
	   
	   tran - CHARACTER*1.
	            On entry, tran specifies the form of op( A ) to be used in
	            the matrix multiplication as follows:
	   
	               tran = 'N' or 'n'   op( A ) = A.
	   
	               tran = 'T' or 't'   op( A ) = A'.
	   
	               tran = 'C' or 'c'   op( A ) = A'.
	   
	            Unchanged on exit.
	   
	   diag   - CHARACTER*1.
	            On entry, diag specifies whether or not A is unit triangular
	            as follows:
	   
	               diag = 'U' or 'u'   A is assumed to be unit triangular.
	   
	               diag = 'N' or 'n'   A is not assumed to be unit
	                                   triangular.
	   
	            Unchanged on exit.
	   
	   m      - INTEGER.
	            On entry, m specifies the number of rows of B. m must be at
	            least zero.
	            Unchanged on exit.
	   
	    n      - INTEGER.
	             On entry, n specifies the number of columns of B.  n must be
	             at least zero.
	             Unchanged on exit.
	   
	    alpha  - REAL.
	             On entry,  alpha specifies the scalar  alpha. When  alpha is
	             zero then  A is not referenced and  B need not be set before
	             entry.
	             Unchanged on exit.
	   
	    A      - REAL             array of DIMENSION ( lda, k ), where k is m
	             when  side = 'L' or 'l'  and is  n  when  side = 'R' or 'r'.
	             Before entry  with  uplo = 'U' or 'u',  the  leading  k by k
	             upper triangular part of the array  A must contain the upper
	             triangular matrix  and the strictly lower triangular part of
	             A is not referenced.
	             Before entry  with  uplo = 'L' or 'l',  the  leading  k by k
	             lower triangular part of the array  A must contain the lower
	             triangular matrix  and the strictly upper triangular part of
	             A is not referenced.
	             Note that when  diag = 'U' or 'u',  the diagonal elements of
	             A  are not referenced either,  but are assumed to be  unity.
	             Unchanged on exit.
	   
	    lda    - INTEGER.
	             On entry, lda specifies the first dimension of A as declared
	             in the calling (sub) program.  When  side = 'L' or 'l'  then
	             lda  must be at least  max( 1, m ),  when  side = 'R' or 'r'
	             then lda must be at least max( 1, n ).
	             Unchanged on exit.
	   
	    b      - REAL             array of DIMENSION ( ldb, n ).
	             Before entry,  the leading  m by n part of the array  B must
	             contain  the  right-hand  side  matrix  B,  and  on exit  is
	             overwritten by the solution matrix  X.
	   
	    ldb    - INTEGER.
	             On entry, ldb specifies the first dimension of B as declared
	             in  the  calling  (sub)  program.   ldb  must  be  at  least
	             max( 1, m ).
	             Unchanged on exit.
	   
	   
	    Level 3 Blas routine.
		*
    ===================================================================== */

	int i, nblocks;
	float *d_dinvA;

	/* quick return on wrong size */
	if (M<=0 || N<=0)
		return;

	/* 
	 * call hipblasStrsm when size of the problem is not a multiple of blocksize which is 32
	 * subject to change soon
	 */
	if ((M%BLOCK_SIZE)!=0 || (N>1 && (N%BLOCK_SIZE)!=0))
	{
		hipblasStrsm (side, uplo, tran, diag, M, N, alpha, A, lda, b, ldb);
		return;
	}

	if (side == 'l' || side == 'L')
	{
		/* inverse the diagonals
		 * Allocate device memory for the inversed diagonal blocks, size=m*BLOCK_SIZE 
		 */
		hipMalloc((void**)&d_dinvA, BLOCK_SIZE*M*sizeof(float));
		nblocks = M/BLOCK_SIZE;

		diag_strtri_kernel<<<nblocks, BLOCK_SIZE>>>(uplo, diag, A, d_dinvA, lda);

		if (tran == 'N' || tran == 'n')
		/* the non-transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				if (N == 1)
					magmablas_sgemv32 ('N', BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, b);
				else
					hipblasSgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, ldb, 0, b, ldb);  

				if (BLOCK_SIZE>=M)
				{
					hipFree(d_dinvA);
					return;
				}

				hipblasSgemm ('N', 'N', M-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+BLOCK_SIZE, lda, b, ldb, alpha, b+BLOCK_SIZE, ldb);

				/* the rest blocks */
				for (i=BLOCK_SIZE; i<M; i+=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_sgemv32 ('N', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						hipblasSgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  

					if (i+BLOCK_SIZE>=M)
						break;

					hipblasSgemm ('N', 'N', M-i-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+i*lda+i+BLOCK_SIZE, lda, b+i, ldb, 1.0, b+i+BLOCK_SIZE, ldb);
				}
			}
			else
			{
			/* the upper case */

				/* handle the first block seperately with alpha */
				i = M-BLOCK_SIZE;
				if (N == 1)
					magmablas_sgemv32 ('N', BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
				else
					hipblasSgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0.0, b+i, ldb); 
					
				if (i-BLOCK_SIZE<0)
				{
					hipFree(d_dinvA);
					return;
				}

				hipblasSgemm ('N', 'N', i, N, BLOCK_SIZE, -1.0, A+i*lda, lda, b+i, ldb, alpha, b, ldb);

				/* the rest blocks */
				for (i=M-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_sgemv32 ('N', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						hipblasSgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0.0, b+i, ldb); 

					if (i-BLOCK_SIZE<0)
						break;

					hipblasSgemm ('N', 'N', i, N, BLOCK_SIZE, -1.0, A+i*lda, lda, b+i, ldb, 1.0, b, ldb);
				}
			}
		}
		else
		/* the transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				i=M-BLOCK_SIZE; 
				if (N == 1)
					magmablas_sgemv32 ('T', BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
				else
					hipblasSgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  

				if (i-BLOCK_SIZE<0)
				{
					hipFree(d_dinvA);
					return;
				}

				hipblasSgemm ('T', 'N', i, N, BLOCK_SIZE, -1.0, A+i, lda, b+i, ldb, alpha, b, ldb);

				/* the rest blocks */
				for (i=M-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_sgemv32 ('T', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						hipblasSgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  

					if (i-BLOCK_SIZE<0)
						break;

					hipblasSgemm ('T', 'N', i, N, BLOCK_SIZE, -1.0, A+i, lda, b+i, ldb, 1.0, b, ldb);
				}
			}
			else
			{
			/* the upper case */
					
				/* handle the first block seperately with alpha */
				if (N == 1)
					magmablas_sgemv32 ('T', BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, b);
				else
					hipblasSgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, ldb, 0, b, ldb);  

				if (BLOCK_SIZE>=M)
				{
					hipFree(d_dinvA);
					return;
				}

				hipblasSgemm ('T', 'N', M-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+(BLOCK_SIZE)*lda, lda, b, ldb, alpha, b+BLOCK_SIZE, ldb);

				/* the rest blocks */
				for (i=BLOCK_SIZE; i<M; i+=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_sgemv32 ('T', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						hipblasSgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  
					
					if (i+BLOCK_SIZE>=M)
						break;

					hipblasSgemm ('T', 'N', M-i-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+(i+BLOCK_SIZE)*lda+i, lda, b+i, ldb, 1.0, b+i+BLOCK_SIZE, ldb);
				}
			}
		}
	}
	else
	{	// side=R

		/* inverse the diagonals
		 * Allocate device memory for the inversed diagonal blocks, size=N*BLOCK_SIZE 
		 */
		hipMalloc((void**)&d_dinvA, BLOCK_SIZE*N*sizeof(float));
		nblocks = N/BLOCK_SIZE;
		diag_strtri_kernel<<<nblocks, BLOCK_SIZE>>>(uplo, diag, A, d_dinvA, lda);
		
		if (tran == 'N' || tran == 'n')
		/* the non-transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				i=N-BLOCK_SIZE;
				inplace_sgemm ('N', M, alpha, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

				if (i-BLOCK_SIZE<0)
				{
					hipFree(d_dinvA);
					return;
				}

				hipblasSgemm ('N', 'N', M, i, BLOCK_SIZE, -1.0, b+ldb*i, ldb, A+i, lda, alpha, b, ldb);

				/* the rest blocks */
				for (i=N-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					inplace_sgemm ('N', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);
					
					if (i-BLOCK_SIZE<0)
						break;

					hipblasSgemm ('N', 'N', M, i, BLOCK_SIZE, -1.0, b+ldb*i, ldb, A+i, lda, 1.0, b, ldb);
				}
			}
			else
			{
			/* the upper case */
				
				/* handle the first block seperately with alpha */
				inplace_sgemm ('N', M, alpha, b, ldb, d_dinvA, BLOCK_SIZE);

				if (BLOCK_SIZE>=N)
				{
					hipFree(d_dinvA);
					return;
				}

				hipblasSgemm ('N', 'N', M, N-BLOCK_SIZE, BLOCK_SIZE, -1.0, b, ldb, A+(BLOCK_SIZE)*lda, lda, alpha, b+(BLOCK_SIZE)*ldb, ldb);
				
				
				/* the rest blocks */
				for (i=BLOCK_SIZE; i<N; i+=BLOCK_SIZE)
				{
					inplace_sgemm ('N', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

					if (i+BLOCK_SIZE>=N)
						break;

					hipblasSgemm ('N', 'N', M, N-i-BLOCK_SIZE, BLOCK_SIZE, -1.0, b+i*ldb, ldb, A+(i+BLOCK_SIZE)*lda+i, lda, 1.0, b+(i+BLOCK_SIZE)*ldb, ldb);
				}
			}
		}
		else
		/* the transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				inplace_sgemm ('T', M, alpha, b, ldb, d_dinvA, BLOCK_SIZE);

				if (BLOCK_SIZE>=N)
				{
					hipFree(d_dinvA);
					return;
				}

				hipblasSgemm ('N', 'T', M, N-BLOCK_SIZE, BLOCK_SIZE, -1.0, b, ldb, A+BLOCK_SIZE, lda, alpha, b+(BLOCK_SIZE)*ldb, ldb);

				/* the rest blocks */
				for (i=BLOCK_SIZE; i<N; i+=BLOCK_SIZE)
				{
					inplace_sgemm ('T', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

					if (i+BLOCK_SIZE>=N)
						break;

					hipblasSgemm ('N', 'T', M, N-i-BLOCK_SIZE, BLOCK_SIZE, -1.0, b+ldb*i, ldb, A+i*lda+BLOCK_SIZE+i, lda, 1.0, b+(i+BLOCK_SIZE)*ldb, ldb);
				}
			}
			else
			{
			/* the upper case */
				
				/* handle the first block seperately with alpha */
				i=N-BLOCK_SIZE;
				inplace_sgemm ('T', M, alpha, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

				if (i-BLOCK_SIZE<0)
				{
					hipFree(d_dinvA);
					return;
				}

				hipblasSgemm ('N', 'T', M, i, BLOCK_SIZE, -1.0, b+i*ldb, ldb, A+i*lda, lda, alpha, b, ldb);
				
				/* the rest blocks */
				for (i=N-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					inplace_sgemm ('T', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

					if (i-BLOCK_SIZE<0)
						break;

					hipblasSgemm ('N', 'T', M, i, BLOCK_SIZE, -1.0, b+i*ldb, ldb, A+i*lda, lda, 1.0, b, ldb);
				}
			}
		}
	}
		
	hipFree(d_dinvA);
}

