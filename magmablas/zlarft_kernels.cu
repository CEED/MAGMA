#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal z -> s d c
       @author Azzam Haidar
*/

#include "common_magma.h"
#include "magma_templates.h"
#define zgemv_bs 32
#define BLOCK_SIZE 512

#define use_gemm_larft

extern __shared__ magmaDoubleComplex shared_data[];


//===================================================================================================
static __device__
void zlarft_gemvcolwise_device( int m, magmaDoubleComplex *v, magmaDoubleComplex *tau,
                         magmaDoubleComplex *c, int ldc, magmaDoubleComplex *T, int ldt, int step )
{

    const int thblk =  blockIdx.x;
    if (thblk > step)
        return;
    /* if blockIdx.x<step step performs the z = V(tx:n,tx)' * V(tx:n,1:tx-1) used for computing T:*/

    if ( !MAGMA_Z_EQUAL(*tau, MAGMA_Z_ZERO) ) {
        if(thblk<step){    
            const int tx = threadIdx.x;
            magmaDoubleComplex *dc = c + blockIdx.x * ldc;
           
            __shared__ magmaDoubleComplex sum[ BLOCK_SIZE ];
            magmaDoubleComplex tmp;
           
            /* perform  {T_i}^H := V(:,i)' * V(:,1:i-1)  */
            if (tx==0)
                tmp = dc[0]; //since V[0] should be one
            else
                tmp = MAGMA_Z_ZERO;
            for( int j = tx+1; j < m; j += BLOCK_SIZE ){
                tmp +=  MAGMA_Z_CNJG( v[j] ) * dc[j];
            }
            sum[tx] = tmp;
            magma_sum_reduce< BLOCK_SIZE >( tx, sum );
            #if defined (use_gemm_larft)
            *(T+thblk) = MAGMA_Z_CNJG(sum[0]);
            #else
            tmp = - MAGMA_Z_CNJG(*tau) * sum[0]; 
            *(T+thblk) = MAGMA_Z_CNJG(tmp); // T = - tau(tx) * V(tx:n,1:tx-1)' * V(tx:n,tx) = tmp'
            //*(T+thblk) = - MAGMA_Z_CNJG(sum[0]) * (*tau); // T = - tau(tx) * V(tx:n,1:tx-1)' * V(tx:n,tx) = tmp'
            #endif
        }
        else{
            #if defined (use_gemm_larft)
            *(T+thblk) = MAGMA_Z_ONE;
            #else
            *(T+thblk) = *tau;
            #endif
        }
    }// in case tau is zero put the corresponding column of T to zero
    else 
    {
        *(T+thblk) = MAGMA_Z_ZERO;
    }
}
//===================================================================================================
__global__
void zlarft_gemvcolwise_kernel( int m, magmaDoubleComplex *v, int ldv, magmaDoubleComplex *tau,
                          magmaDoubleComplex *T, int ldt, int step )
{
    zlarft_gemvcolwise_device(m, v+step+step*ldv, tau+step, v+step, ldv, T+step*ldt, ldt, step);
}
//===================================================================================================
__global__
void zlarft_gemvcolwise_kernel_batched( int m, magmaDoubleComplex **v_array, int ldv, magmaDoubleComplex **tau_array,
                          magmaDoubleComplex **T_array, int ldt, int step )
{
    int batchid = blockIdx.z;
    zlarft_gemvcolwise_device(m, v_array[batchid]+step+step*ldv, tau_array[batchid]+step, v_array[batchid]+step, ldv, T_array[batchid]+step*ldt, ldt, step);
}
//===================================================================================================
extern "C" 
void magmablas_zlarft_gemvcolwise(
    magma_int_t m,  magma_int_t step,
    magmaDoubleComplex *v, magma_int_t ldv, 
    magmaDoubleComplex *T,  magma_int_t ldt,
    magmaDoubleComplex *tau)
{
    dim3 grid( step+1, 1, 1 );
    dim3 threads( BLOCK_SIZE );
    zlarft_gemvcolwise_kernel<<< grid, threads, 0, magma_stream >>>( m, v, ldv, tau, T, ldt, step);

}
//===================================================================================================
extern "C" 
void magmablas_zlarft_gemvcolwise_batched(
    magma_int_t m,  magma_int_t step,
    magmaDoubleComplex **v_array, magma_int_t ldv, 
    magmaDoubleComplex **T_array,  magma_int_t ldt,
    magmaDoubleComplex **tau_array, magma_int_t batchCount, magma_queue_t queue )
{
    dim3 grid( step+1, 1, batchCount );
    dim3 threads( BLOCK_SIZE );
    zlarft_gemvcolwise_kernel_batched<<< grid, threads, 0, queue >>>( m, v_array, ldv, tau_array, T_array, ldt, step);

}
//===================================================================================================




//===================================================================================================
// zgemv(y=alpha*A*x) interface: T/W=tau*v*x, 
static __device__ void
zlarft_gemvrowwise_device(
    int m, int i,
    magmaDoubleComplex *tau, 
    magmaDoubleComplex *v_ptr, int ldv, 
    magmaDoubleComplex *x_ptr, int incx,
    magmaDoubleComplex *T_ptr, int ldt,
    magmaDoubleComplex *W, magmaDoubleComplex* sdata)
{
    int tx = threadIdx.x; 
    int ty = threadIdx.y; 


    if(tx ==0 && ty == 0)
    {
        T_ptr[0] = *tau;
    } 

    if(i <= 0) return;
    
    magmaDoubleComplex res = MAGMA_Z_ZERO;

    v_ptr += ldv * ty;
            

   
    if(tx < zgemv_bs)
    {
        for(int s=tx; s<m; s+= zgemv_bs)
        {
            res += MAGMA_Z_CNJG (v_ptr[s]) * x_ptr[s*incx];
        }
    
        sdata[ty * zgemv_bs + tx] = res;
    }
    __syncthreads();

    magma_sum_reduce<zgemv_bs>(tx, &(sdata[ty*zgemv_bs+0]));

    #if defined (use_gemm_larft)
    if(tx == 0)
    {
            W[ty] = -sdata[ty * zgemv_bs + 0];
    } 
    #else
    if(tx == 0)
    {
            W[ty] = -sdata[ty * zgemv_bs + 0] * (*tau);
    }
    #endif 
}




//T(1:i-1,i) := - tau(i) * V(i:n,1:i-1)' * V(i:n,i)
//T(i,i) = tau(i)
//===================================================================================================
 __global__ void
zlarft_gemvrowwise_kernel(
    int m, int i, 
    magmaDoubleComplex *tau, 
    magmaDoubleComplex *v, int ldv, 
    magmaDoubleComplex *T, int ldt)
{

    magmaDoubleComplex *W =  T +i*ldt;

    magmaDoubleComplex *sdata = (magmaDoubleComplex*)shared_data;

    zlarft_gemvrowwise_device(m, i, tau+i, v+i, ldv,  v+i+i*ldv, 1,  
                           T+i+i*ldt , ldt, W, sdata);
}

//===================================================================================================
__global__ void
zlarft_gemvrowwise_kernel_batched(
    int m, int i,
    magmaDoubleComplex **tau_array, 
    magmaDoubleComplex **v_array, int ldv, 
    magmaDoubleComplex **T_array, int ldt)
{

    int batchid = blockIdx.z;

    magmaDoubleComplex *W =  T_array[batchid] +i*ldt;

    magmaDoubleComplex *sdata = (magmaDoubleComplex*)shared_data;

    zlarft_gemvrowwise_device(m, i, tau_array[batchid]+i, v_array[batchid]+i, ldv,  v_array[batchid]+i+i*ldv, 1,  
                           T_array[batchid] +i+i*ldt , ldt, W, sdata);
}

//===================================================================================================
extern "C"
void magmablas_zlarft_gemvrowwise(
    magma_int_t m, magma_int_t i, 
    magmaDoubleComplex *tau, 
    magmaDoubleComplex *v, magma_int_t ldv, 
    magmaDoubleComplex *T, magma_int_t ldt,
    magmaDoubleComplex *W)
{

    dim3 grid(1);


    dim3 threads(zgemv_bs, max(i,1), 1);


    zlarft_gemvrowwise_kernel <<< grid, threads, sizeof(magmaDoubleComplex)*zgemv_bs*(i+1), magma_stream>>>(m, i, tau, v, ldv, T, ldt);
}
//===================================================================================================
extern "C"
void magmablas_zlarft_gemvrowwise_batched(
    magma_int_t m, magma_int_t i, 
    magmaDoubleComplex **tau_array, 
    magmaDoubleComplex **v_array, magma_int_t ldv, 
    magmaDoubleComplex **T_array, magma_int_t ldt,
    magma_int_t batchCount, magma_queue_t queue)
{

    dim3 grid(1, 1, batchCount);
    dim3 threads(zgemv_bs, max(i,1), 1);

    /*  zgemvrowwise used a bigger shared memory and has more data reuse and performs better
    */
    zlarft_gemvrowwise_kernel_batched <<< grid, threads, sizeof(magmaDoubleComplex)*zgemv_bs*(i+1), queue>>>(m, i,  tau_array, v_array, ldv, T_array, ldt);
}
//===================================================================================================
   


//===================================================================================================
/*
   loop_inside
*/
static __device__ void
zlarft_gemv_loop_inside_device(
    int n, int k, 
    magmaDoubleComplex *tau, 
    magmaDoubleComplex *v, int ldv, 
    magmaDoubleComplex *T, int ldt)
{
    int tx = threadIdx.x; 
    int ty = threadIdx.y; 
    
    int incx = 1;
    magmaDoubleComplex *sdata = (magmaDoubleComplex*)shared_data;

    magmaDoubleComplex res;

    // write the first elment
    if(tx ==0 && ty == 0)
    {
        T[0] = tau[0];
    } 
 
    for(int i=1; i < k; i++)
    {

        int m = n-i; 

        magmaDoubleComplex *v_ptr = v;

        v_ptr += i;

        magmaDoubleComplex *x_ptr = v_ptr + i * ldv;
            
        res = MAGMA_Z_ZERO;
            
        if(tx < zgemv_bs && ty < i)
        {
            v_ptr += ldv * ty;

            for(int s=tx; s<m; s+= zgemv_bs)
            {
                res += MAGMA_Z_CNJG (v_ptr[s]) * x_ptr[s*incx];
            }
    
            sdata[ty * zgemv_bs + tx] = res;
        }
        __syncthreads();

        magma_sum_reduce<zgemv_bs>(tx, &(sdata[ty*zgemv_bs+0]));
        

       __syncthreads();
       #if defined (use_gemm_larft)
       if(tx < i && ty == 0)
       {
            T[i* ldt + tx] = sdata[tx * zgemv_bs + 0];  
       } 
       // not needed since it is overwritten in trmv
       /*
       if(tx == i && ty == 0)
       {
           T[i * ldt + i] = tau[i];
       }
       */
       #else
       if(tx < i && ty == 0)
       {
           T[i* ldt + tx] = -sdata[tx * zgemv_bs + 0] * (tau[i]);  
       } 
      
       if(tx == i && ty == 0)
       {
           T[i * ldt + i] = tau[i];
       }
       #endif
     
       v_ptr -= i;

    }// end of loop k
}
//===================================================================================================
__global__ void
zlarft_gemv_loop_inside_kernel(
    int n, int k, 
    magmaDoubleComplex *tau, 
    magmaDoubleComplex *v, int ldv, 
    magmaDoubleComplex *T, int ldt)
{
    zlarft_gemv_loop_inside_device(n, k, tau, v, ldv, T, ldt);
}
//===================================================================================================
__global__ void
zlarft_gemv_loop_inside_kernel_batched(
    int n, int k, 
    magmaDoubleComplex **tau_array, 
    magmaDoubleComplex **v_array, int ldv, 
    magmaDoubleComplex **T_array, int ldt)
{
    int batchid = blockIdx.z;
    zlarft_gemv_loop_inside_device(n, k, tau_array[batchid], v_array[batchid], ldv, T_array[batchid], ldt);
}
//===================================================================================================
//===================================================================================================
//===================================================================================================
extern "C"
void magmablas_zlarft_gemv_loop_inside(
    magma_int_t n, magma_int_t k, 
    magmaDoubleComplex *tau, 
    magmaDoubleComplex *v, magma_int_t ldv, 
    magmaDoubleComplex *T, magma_int_t ldt)
{

    dim3 grid(1);
    dim3 threads(zgemv_bs, max(k,1), 1);
    zlarft_gemv_loop_inside_kernel<<<grid, threads, sizeof(magmaDoubleComplex) * (zgemv_bs*(k+1)), magma_stream>>>(n, k, tau, v, ldv, T, ldt); 
}
//===================================================================================================
extern "C"
void magmablas_zlarft_gemv_loop_inside_batched(
    magma_int_t n, magma_int_t k, 
    magmaDoubleComplex **tau_array, 
    magmaDoubleComplex **v_array, magma_int_t ldv, 
    magmaDoubleComplex **T_array, magma_int_t ldt, magma_int_t batchCount, magma_queue_t queue)
{

    dim3 grid(1, 1, batchCount);
    dim3 threads(zgemv_bs, max(k,1), 1);
    zlarft_gemv_loop_inside_kernel_batched<<<grid, threads, sizeof(magmaDoubleComplex) * (zgemv_bs*(k+1)), queue>>>(n, k, tau_array, v_array, ldv, T_array, ldt); 
}
//===================================================================================================





//===================================================================================================
static  __device__ void 
zlarft_ztrmv_sm32x32_device(
    int n, int k, magmaDoubleComplex *tau,
    magmaDoubleComplex *Tin, int ldtin,  magmaDoubleComplex *Tout, int ldtout )
{
    int tx = threadIdx.x; 
    magmaDoubleComplex *sdata = (magmaDoubleComplex*)shared_data;
    magmaDoubleComplex res;

    // this routine apply a sequence of trmv to update k column of the triangular
    // T starting at n-k to n where T is of size n by n and where the first n-k 
    // columns of T are supposed updated previously.
    // So the routine load all of T nxn to the shared memory 
    // and apply the sequence of trmv.
    // to update a certain column i, threads go in horizontal fashion where
    // every thread read one row and do it gemv(dot) to generate 
    // one element of the column of T then move to the next column

    // read T into shared
    for(int s=0; s<n-k; s++)
    {
        sdata[tx + s*n] = Tin[tx + s * ldtin];
    }
    
#if defined(use_gemm_larft)
    for(int s=n-k; s<n; s++)
    {
        if(tx == s)
            sdata[tx + s*n] = tau[s];
        else
            sdata[tx + s*n] = -tau[s] * Tin[tx + s * ldtin];
    }
#else
    for(int s=n-k; s<n; s++)
    {
        sdata[tx + s*n] = Tin[tx + s * ldtin];
    }
#endif

    // perform trmv
    for(int i=n-k; i < n; i++)
    {
       __syncthreads();  
       res = MAGMA_Z_ZERO;
       if(tx < i)
       {
           for(int j=tx; j<i; j++)
           {
               res += sdata[tx + j * n] * sdata[j+ i * n];      
           }
       }       
       __syncthreads();  
       if(tx < i)
       {
           sdata[tx + i * n] = res;
       }
    } 

    __syncthreads();  
    // write back the updated block of k column of T
    for(int s=n-k; s<n; s++)
    {
       Tout[tx + s * ldtout] = sdata[tx + s*n];
    }

}
//===================================================================================================
__global__ void 
zlarft_ztrmv_sm32x32_kernel(
    int n, int k, magmaDoubleComplex *tau,
    magmaDoubleComplex *Tin, int ldtin,  magmaDoubleComplex *Tout, int ldtout )
{
    zlarft_ztrmv_sm32x32_device( n, k, tau, Tin, ldtin, Tout, ldtout);
}
//===================================================================================================
__global__ void 
zlarft_ztrmv_sm32x32_kernel_batched(
    int n, int k, magmaDoubleComplex **tau_array,
    magmaDoubleComplex **Tin_array, int ldtin,  magmaDoubleComplex **Tout_array, int ldtout )
{
    int batchId = blockIdx.z;
    zlarft_ztrmv_sm32x32_device( n, k, tau_array[batchId], Tin_array[batchId], ldtin, Tout_array[batchId], ldtout);
}
//===================================================================================================
//===================================================================================================
extern "C"
void magmablas_zlarft_ztrmv_sm32x32(
    magma_int_t m, magma_int_t n, 
    magmaDoubleComplex *tau, 
    magmaDoubleComplex *Tin, magma_int_t ldtin, 
    magmaDoubleComplex *Tout, magma_int_t ldtout)
{

    dim3 grid(1);
    dim3 threads(max(m,1), 1, 1);
    zlarft_ztrmv_sm32x32_kernel <<< grid, threads, sizeof(magmaDoubleComplex)*(m*m), magma_stream >>> (m, n,  tau, Tin, ldtin, Tout, ldtout);
}
//===================================================================================================
extern "C"
void magmablas_zlarft_ztrmv_sm32x32_batched(
    magma_int_t m, magma_int_t n, 
    magmaDoubleComplex **tau_array, 
    magmaDoubleComplex **Tin_array, magma_int_t ldtin, 
    magmaDoubleComplex **Tout_array, magma_int_t ldtout,
    magma_int_t batchCount, magma_queue_t queue)
{

    dim3 grid(1, 1, batchCount);
    dim3 threads(max(m,1), 1, 1);
    zlarft_ztrmv_sm32x32_kernel_batched <<< grid, threads, sizeof(magmaDoubleComplex)*(m*m), queue >>> (m, n,  tau_array, Tin_array, ldtin, Tout_array, ldtout);
}
//===================================================================================================




//===================================================================================================
//===================================================================================================
static __device__ void 
zlarft_recztrmv_sm32x32_device(
    int m, int n, magmaDoubleComplex *tau,
    magmaDoubleComplex *Trec, int ldtrec, magmaDoubleComplex *Ttri, int ldttri)
{
    int tx = threadIdx.x; 
    magmaDoubleComplex *sdata = (magmaDoubleComplex*)shared_data;
    magmaDoubleComplex res;

    // to update a certain column i, threads go in horizontal fashion where
    // every thread read one row and do it gemv(dot) to generate 
    // one element of the column of T then move to the next column

    // read T into shared
    for(int s=0; s<n; s++)
    {
        sdata[tx + s*n] = Trec[tx + s * ldtrec];
    }
    __syncthreads();  
    
    // perform sequence of n-1 gemv
    for(int i=0; i < n; i++)
    {
       res = MAGMA_Z_ZERO;
       for(int j=0; j<i; j++)
       {
           res += sdata[tx + j * n] * Ttri[j+ i * ldttri];      
       }
       __syncthreads();   // a enlever
       sdata[tx + i * n] = -tau[i] * (sdata[tx + i * n] + res);
       __syncthreads();  
    } 

    // write back the updated block of k column of T  multiplying by -tau
    for(int s=0; s<n; s++)
    {
       Trec[tx + s * ldtrec] = sdata[tx + s*n];
    }

}

//===================================================================================================
__global__ void 
zlarft_recztrmv_sm32x32_kernel(
    int m, int n, magmaDoubleComplex *tau,
    magmaDoubleComplex *Trec, int ldtrec, magmaDoubleComplex *Ttri, int ldttri)
{
    zlarft_recztrmv_sm32x32_device(m, n, tau, Trec, ldtrec, Ttri, ldttri);
}
//===================================================================================================
__global__ void 
zlarft_recztrmv_sm32x32_kernel_batched(
    int m, int n, magmaDoubleComplex **tau_array,
    magmaDoubleComplex **Trec_array, int ldtrec, magmaDoubleComplex **Ttri_array, int ldttri)
{
    int batchId = blockIdx.z;
    zlarft_recztrmv_sm32x32_device(m, n, tau_array[batchId], Trec_array[batchId], ldtrec, Ttri_array[batchId], ldttri);
}
//===================================================================================================
extern "C"
void magmablas_zlarft_recztrmv_sm32x32(
    magma_int_t m, magma_int_t n, 
    magmaDoubleComplex *tau, 
    magmaDoubleComplex *Trec, magma_int_t ldtrec, 
    magmaDoubleComplex *Ttri, magma_int_t ldttri)
{

    dim3 grid(1);
    dim3 threads(max(m,1), 1, 1);
    zlarft_recztrmv_sm32x32_kernel <<< grid, threads, sizeof(magmaDoubleComplex)*(m*n), magma_stream >>> (m, n,  tau, Trec, ldtrec, Ttri, ldttri);
}
//===================================================================================================
extern "C"
void magmablas_zlarft_recztrmv_sm32x32_batched(
    magma_int_t m, magma_int_t n, 
    magmaDoubleComplex **tau_array, 
    magmaDoubleComplex **Trec_array, magma_int_t ldtrec, 
    magmaDoubleComplex **Ttri_array, magma_int_t ldttri,
    magma_int_t batchCount, magma_queue_t queue)
{

    dim3 grid(1, 1, batchCount);
    dim3 threads(max(m,1), 1, 1);
    zlarft_recztrmv_sm32x32_kernel_batched <<< grid, threads, sizeof(magmaDoubleComplex)*(m*n), queue >>> (m, n,  tau_array, Trec_array, ldtrec, Ttri_array, ldttri);
}
//===================================================================================================


