#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date
       
       @precisions normal s

       @author Tingxing Dong
       @author Azzam Haidar

*/
#include "common_magma.h"
#include "commonblas_z.h"
#include "magma_templates.h"

#define PRECISION_z

#include "gemv_template_kernel_batched.cuh"
#include "gemv_config/gemvn_param.h"
#include "gemv_config/gemvt_param.h"
#define version(s,v) s ## _V_ ## v




/**
    Purpose
    -------
    ZGEMV performs one of the matrix-vector operations
    
        y := alpha*A*x    + beta*y,   or
        y := alpha*A**T*x + beta*y,   or
        y := alpha*A**H*x + beta*y,
    
    where alpha and beta are scalars, x and y are vectors and A is an
    m by n matrix.

    Arguments
    ----------
    @param[in]
    trans   magma_trans_t
            On entry, TRANS specifies the operation to be performed as
            follows:
      -     = MagmaNoTrans:    y := alpha*A  *x + beta*y
      -     = MagmaTrans:      y := alpha*A^T*x + beta*y
      -     = MagmaConjTrans:  y := alpha*A^H*x + beta*y

    @param[in]
    m       INTEGER
            On entry, m specifies the number of rows of the matrix A.

    @param[in]
    n       INTEGER
            On entry, n specifies the number of columns of the matrix A
 
    @param[in]
    alpha   COMPLEX_16
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    dA      COMPLEX_16 array of dimension ( LDDA, n ) on the GPU.
   
    @param[in]
    ldda    INTEGER
            LDDA specifies the leading dimension of A.

    @param[in]
    dx      COMPLEX_16 array of dimension
            n if trans == MagmaNoTrans
            m if trans == MagmaTrans or MagmaConjTrans
     
    @param[in]
    incx    Specifies the increment for the elements of X.
            INCX must not be zero.
  
    @param[in]
    beta    DOUBLE REAL
            On entry, BETA specifies the scalar beta. When BETA is
            supplied as zero then Y need not be set on input.

    @param[out]
    dy      DOUBLE PRECISION array of dimension
            m if trans == MagmaNoTrans
            n if trans == MagmaTrans or MagmaConjTrans

    @param[in]
    incy    Specifies the increment for the elements of Y.
            INCY must not be zero.

    @ingroup magma_dblas2
    ********************************************************************/

extern "C" void
magmablas_zgemv_batched(
    magma_trans_t trans, magma_int_t m, magma_int_t n, 
    magmaDoubleComplex alpha,
    magmaDoubleComplex_ptr dA_array[], magma_int_t ldda, 
    magmaDoubleComplex_ptr dx_array[], magma_int_t incx,
    magmaDoubleComplex beta,
    magmaDoubleComplex_ptr dy_array[], magma_int_t incy, 
    magma_int_t batchCount, magma_queue_t queue)
{
    magma_int_t info = 0;
    if ( trans != MagmaNoTrans && trans != MagmaTrans && trans != MagmaConjTrans )
        info = -1;
    else if ( m < 0 )
        info = -2;
    else if ( n < 0 )
        info = -3;
    else if ( ldda < m )
        info = -6;
    else if ( incx == 0 )
        info = -8;
    else if ( incy == 0 )
        info = -11;
    
    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }    
/*
    gemvn_template_batched<magmaDoubleComplex, version(N, 70)>             
                 ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );

     return;
*/
    if ( trans == MagmaNoTrans ) {                                                   
        if(max(m, n) <= 96){ // small size                         
            if(m <n){ // Fat matrix
                if ( m<=8) 
                {    
                    gemvn_template_batched<magmaDoubleComplex, version(N, 72)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );

                }
                else if( m<=32) 
                {
                    gemvn_template_batched<magmaDoubleComplex, version(N, 100)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else if( m<=64)            
                {
                    gemvn_template_batched<magmaDoubleComplex, version(N, 121)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else
                {
                    gemvn_template_batched<magmaDoubleComplex, version(N, 132)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }  
            }else{   // Tall or square matrix
                if ( n<=16) 
                {    
                    gemvn_template_batched<magmaDoubleComplex, version(N, 129)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else if( n<=64)            
                {
                    gemvn_template_batched<magmaDoubleComplex, version(N, 131)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else
                {
                    gemvn_template_batched<magmaDoubleComplex, version(N, 132)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }  
            }
        }
        else{ // big size
            if(m < n){ // Fat matrix
                if(m <= 16)
                {
                    gemvn_template_batched<magmaDoubleComplex, version(N, 72)>              
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else if(m <= 32)
                {
                    gemvn_template_batched<magmaDoubleComplex, version(N, 100)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else if(m <= 64)
                {
                    gemvn_template_batched<magmaDoubleComplex, version(N, 116)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else
                {
                    gemvn_template_batched<magmaDoubleComplex, version(N, 133)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
            }
            else{ // Tall or square matrix
                if(m <= 256)
                {
                    gemvn_template_batched<magmaDoubleComplex, version(N, 137)>             
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
                else
                {
                    gemvn_template_batched<magmaDoubleComplex, version(N, 140)>               
                        ( m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, batchCount, queue );
                }
            }
        }// big size        
     }else {

        magma_int_t CONJA = -1;

        if ( trans == MagmaConjTrans ) {
            CONJA = 1;
        }else if( trans == MagmaTrans ) {
            CONJA = 0;
        }
        else{
            return ;
        }      
                                                
        if(max(m, n) <= 96){// small size                 
            if(n <= 16)
            {
                gemvc_template_batched<magmaDoubleComplex, version(T, 42)>             
                        (m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, CONJA, batchCount, queue );
            }
            else
            {
                gemvc_template_batched<magmaDoubleComplex, version(T, 46)>             
                        (m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, CONJA, batchCount, queue );
            }  
        }
        else{ // big size
            if(m <= n) // Fat or square matrix
            {    
                if(m <=64)
                {
                    gemvc_template_batched<magmaDoubleComplex, version(T, 47)>             
                        (m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, CONJA, batchCount, queue );
                }
                else
                {
                    gemvc_template_batched<magmaDoubleComplex, version(T, 46)>             
                        (m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, CONJA, batchCount, queue );
                }

            }                           
            else// (m > n) Tall matrix
            {
            
                if(n <= 8)
                {
                    gemvc_template_batched<magmaDoubleComplex, version(T, 130)>             
                        (m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, CONJA, batchCount, queue );
                }
                else
                {
                    gemvc_template_batched<magmaDoubleComplex, version(T, 46)>             
                        (m, n, alpha, dA_array, ldda, dx_array, incx, beta, dy_array, incy, CONJA, batchCount, queue );
                }
            }
        }        
     }                   

}
///////////////////////////////////////////////////////////////////////////////////////////////////
