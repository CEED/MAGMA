/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal d

       @author Jakub Kurzak
       @author Stan Tomov
       @author Mark Gates
       @author Azzam Haidar
       @author Ahmad Abdelfattah
       
*/
#include "magma_internal.h"
#define PRECISION_d
#include "gemm_template_kernel_vbatched.cuh"

#include "gemm_config/dgemm_param_nn.h"
#include "gemm_config/dgemm_param_nt.h"
#include "gemm_config/dgemm_param_tn.h"
#include "gemm_config/dgemm_param_tt.h"

#define version(s,v) s ## _V_ ## v

#ifdef __cplusplus
extern "C" 
#endif
void 
magmablas_dgemm_vbatched_core(
    magma_trans_t transA, magma_trans_t transB, 
    magma_int_t* m, magma_int_t* n, magma_int_t* k,
    double alpha,
    double const * const * dA_array, magma_int_t* ldda,
    double const * const * dB_array, magma_int_t* lddb,
    double beta,
    double **dC_array, magma_int_t* lddc, 
    magma_int_t max_m, magma_int_t max_n, magma_int_t max_k, 
    magma_int_t roffA, magma_int_t coffA, magma_int_t roffB, magma_int_t coffB, magma_int_t roffC, magma_int_t coffC, 
    magma_int_t spec_m, magma_int_t spec_n, magma_int_t spec_k, 
    magma_int_t batchCount, magma_queue_t queue )
{
    if(max_m <=0 || max_n <= 0 || max_k <= 0) return;
    
    magma_int_t shape = 0;
    if      (transA == MagmaNoTrans   && transB == MagmaNoTrans)   {shape = 0;} // nn
    else if (transA == MagmaNoTrans   && transB == MagmaTrans)     {shape = 1;} // nt
    else if (transA == MagmaNoTrans   && transB == MagmaConjTrans) {shape = 2;} // nc
    else if (transA == MagmaTrans     && transB == MagmaNoTrans)   {shape = 3;} // tn
    else if (transA == MagmaTrans     && transB == MagmaTrans)     {shape = 4;} // tt
    else if (transA == MagmaTrans     && transB == MagmaConjTrans) {shape = 5;} // tc
    else if (transA == MagmaConjTrans && transB == MagmaNoTrans)   {shape = 6;} // cn
    else if (transA == MagmaConjTrans && transB == MagmaTrans)     {shape = 7;} // ct
    else if (transA == MagmaConjTrans && transB == MagmaConjTrans) {shape = 8;} // cc
    
    switch(shape)
    {
        case 0: // nn
            {
                if(max_k < 32)
                {
                    if(max_k==8 && max_n==24)
                    gemm_template_vbatched_nn<double, version(NN,32), 0, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                    else if (max_n<32)
                    gemm_template_vbatched_nn<double, version(NN,49), 0, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                    else
                    gemm_template_vbatched_nn<double, version(NN,111), 0, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                }
                else
                {
                    if(max_m < 80)
                    {
                        gemm_template_vbatched_nn<double, version(NN,93), 0, 0>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                    }
                    else
                    {
                        gemm_template_vbatched_nn<double, version(NN,111), 0, 0>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                    }
                }
            }
            break;
        case 1: // nt
            {
                if(max_k < 128)
                {
                    gemm_template_vbatched_nt<double, version(NT,160), 0, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                }
                else
                {
                    if(max_m < 256)
                    {
                        gemm_template_vbatched_nt<double, version(NT,160), 0, 0>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                    }
                    else
                    {
                        gemm_template_vbatched_nt<double, version(NT,190), 0, 0>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                    }
                }
            }
            break;
        case 2: // nc
            {
                if(max_k < 128)
                {
                    gemm_template_vbatched_nt<double, version(NT,160), 0, 1>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                }
                else
                {
                    if(max_m < 256)
                    {
                        gemm_template_vbatched_nt<double, version(NT,160), 0, 1>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                    }
                    else
                    {
                        gemm_template_vbatched_nt<double, version(NT,190), 0, 1>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                    }
                }
            }
            break;
        case 3: // tn
            {
                if(max_k < 64)
                {
                    gemm_template_vbatched_tn<double, version(TN,207), 0, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                }
                else
                {
                    if(max_m < 256)
                    {
                        gemm_template_vbatched_tn<double, version(TN,207), 0, 0>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                    }
                    else
                    {
                        gemm_template_vbatched_tn<double, version(TN,209), 0, 0>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                    }
                }
            }
            break;
        case 6: // cn
            {
                if(max_k < 64)
                {
                    gemm_template_vbatched_tn<double, version(TN,207), 1, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                }
                else
                {
                    if(max_m < 256)
                    {
                        gemm_template_vbatched_tn<double, version(TN,207), 1, 0>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                    }
                    else
                    {
                        gemm_template_vbatched_tn<double, version(TN,209), 1, 0>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                    }
                }
            }
            break;
        case 4: // tt
            {
                if(max_k < 128)
                {
                    gemm_template_vbatched_tt<double, version(TT,81), 0, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                }
                else
                {
                    if(max_m < 256)
                    {
                        gemm_template_vbatched_tt<double, version(TT,81), 0, 0>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                    }
                    else
                    {
                        gemm_template_vbatched_tt<double, version(TT,85), 0, 0>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                    }
                }
            }
            break;
        case 5: // tc
            {
                if(max_k < 128)
                {
                    gemm_template_vbatched_tt<double, version(TT,81), 0, 1>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                }
                else
                {
                    if(max_m < 256)
                    {
                        gemm_template_vbatched_tt<double, version(TT,81), 0, 1>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                    }
                    else
                    {
                        gemm_template_vbatched_tt<double, version(TT,85), 0, 1>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                    }
                }
            }
            break;
        case 7: // ct
            {
                if(max_k < 128)
                {
                    gemm_template_vbatched_tt<double, version(TT,81), 1, 0>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                }
                else
                {
                    if(max_m < 256)
                    {
                        gemm_template_vbatched_tt<double, version(TT,81), 1, 0>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                    }
                    else
                    {
                        gemm_template_vbatched_tt<double, version(TT,85), 1, 0>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                    }
                }
            }
            break;
        case 8: // cc
            {
                if(max_k < 128)
                {
                    gemm_template_vbatched_tt<double, version(TT,81), 1, 1>
                    (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                }
                else
                {
                    if(max_m < 256)
                    {
                        gemm_template_vbatched_tt<double, version(TT,81), 1, 1>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                    }
                    else
                    {
                        gemm_template_vbatched_tt<double, version(TT,85), 1, 1>
                        (m, n, k, dA_array, ldda, dB_array, lddb, dC_array, lddc, alpha, beta, max_m, max_n, roffA, coffA, roffB, coffB, roffC, coffC, spec_m, spec_n, spec_k, batchCount, queue);
                    }
                }
            }
            break;
        default:; // propose something
    }
}
