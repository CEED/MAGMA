#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010
*/
#include "common_magma.h"

#define num_threads 64
#define sgemv_bs 64
#define magmablas_sgemv_tesla magmablas_sgemv
#define magmablas_sgemvt_tesla magmablas_sgemvt

__global__ void 
sgemv_kernel(magma_int_t n, magma_int_t m, magma_int_t n1, float* A, magma_int_t lda, float *x, float *y)
{
  magma_int_t ind = blockIdx.x*num_threads + threadIdx.x;

  A += ind;
  x += threadIdx.x;

  float res = 0.f;

  __shared__ float buff[sgemv_bs];
  for(magma_int_t i=0; i<n1; i += sgemv_bs ){
    __syncthreads();
    buff[threadIdx.x]  = x[i];

    __syncthreads();
    #pragma unroll
    for(magma_int_t j=0; j < sgemv_bs ; j++){
       res+=A[0]*buff[j];
       A+=lda;
    }
  }
  __syncthreads();

  if (m>n1){
     buff[threadIdx.x]  = x[n1];

     __syncthreads();
     for(magma_int_t j=0; j<(m-n1); j++){
         res += A[0]*buff[j];
         A+=lda;
     }
  }

  if (ind<n)
     y[ind] = res;
}

__global__ void
sgemv_kernel2(magma_int_t n, magma_int_t m, magma_int_t n1, float* A, magma_int_t lda, 
              float *x, magma_int_t incx, float *y)
{
  magma_int_t ind = blockIdx.x*num_threads + threadIdx.x;

  A += ind;
  x += threadIdx.x * incx;

  float res = 0.f;

  __shared__ float buff[sgemv_bs];
  for(magma_int_t i=0; i<n1; i += sgemv_bs ){
    __syncthreads();
    buff[threadIdx.x]  = x[i*incx];

    __syncthreads();
    #pragma unroll
    for(magma_int_t j=0; j < sgemv_bs ; j++){
       res+=A[0]*buff[j];
       A+=lda;
    }
  }
  __syncthreads();

  if (m>n1){
     buff[threadIdx.x]  = x[n1*incx];

     __syncthreads();
     for(magma_int_t j=0; j<(m-n1); j++){
         res += A[0]*buff[j];
         A+=lda;
     }
  }

  if (ind<n)
     y[ind] = res;
}

extern "C" void
magmablas_sgemvt_tesla(magma_int_t m, magma_int_t n, float alpha, float *A, magma_int_t lda,
                       float *x, float *z);


extern "C" void
magmablas_sgemv_tesla(char trans,
                      magma_int_t m, magma_int_t n,
                      float alpha, 
                      float *A, magma_int_t lda, 
                      float *x, magma_int_t incx,
                      float beta,
                      float *z, magma_int_t incz)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======
    This routine computes:
    1) z =       A   x    if trans == 'N' or 'n', alpha == 1, beta == 0, 
                          and incx == incz == 1 (using magmablas code)
    2) z = alpha A^t x    if trans == 'T' or 't', beta == 0,
                          and incx == incz == 1 (using magmablas code)
    3) z = alpha A^trans x + beta z
                          otherwise, using CUBLAS.

   Arguments
   ==========
    TRANS  - CHARACTER*1
             On entry, TRANS specifies the operation to be performed as
             follows:
               TRANS = 'N' or 'n'   z := alpha*A *x + beta*z
               TRANS = 'T' or 't'   z := alpha*A'*x + beta*z

    M      - (input) INTEGER
             On entry, M specifies the number of rows of the matrix A.

    N      - (input) INTEGER
             On entry, N specifies the number of columns of the matrix A
 
    ALPHA  - REAL
             On entry, ALPHA specifies the scalar alpha.
             Unchanged on exit.

    A      - (input) SINGLE PRECISION array of dimension ( LDA, n ) on the GPU.
   
    LDA    - (input) INTEGER
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension 
             n if trans == 'n'
             m if trans == 't'
     
    INCX   - (input) Specifies the increment for the elements of X.
             INCX must not be zero. Unchanged on exit.
  
    BETA   - REAL
             On entry, BETA specifies the scalar beta. When BETA is
             supplied as zero then Y need not be set on input.
             Unchanged on exit.

    Z      - (output) SINGLE PRECISION array of	dimension 
             m if trans == 'n'
             n if trans == 't' 

    INCZ  - (input) Specifies the increment for the elements of Z.
            INCZ must not be zero. Unchanged on exit.
    ===================================================================== */

    if (incx == 1 && incz == 1 && beta == 0.)
       if (trans == 'n' || trans == 'N')
          if (alpha == 1.)
            {
              magma_int_t blocks;
              if (m % num_threads==0)
                 blocks = m/num_threads;
              else
                 blocks = m/num_threads + 1;

              dim3 grid(blocks, 1, 1);
              dim3 threads(num_threads, 1, 1);
 
              sgemv_kernel<<< grid, threads, 0, magma_stream >>>(m, n, 
                                              (n/sgemv_bs)*sgemv_bs, 
                                              A, lda, x, z);
           }
         else
           hipblasSgemv(trans, m, n, alpha, A, lda,
                       x, incx, beta, z, incz);
       else
         magmablas_sgemvt_tesla(m, n, alpha, A, lda, x, z);
    else
      hipblasSgemv(trans, m, n, alpha, A, lda,
                  x, incx, beta, z, incz);   
}

extern "C" void
magmablas_sgemv2(magma_int_t n, magma_int_t m, float *A, magma_int_t lda, float *x, magma_int_t incx, float *z)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======

    This routine computes z = A x on the GPU.
    This version has INCX as an argument. 

    N      - (input) INTEGER.
             On entry, N specifies the number of rows of the matrix A.

    M      - (input) INTEGER.
             On entry, M specifies the number of columns of the matrix A

    A      - (input) SINGLE PRECISION array of dimension ( LDA, m ) on the GPU.

    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension n.

    INCX   - (input) Specifies the increment for the elements of X. 
             INCX must not be zero. Unchanged on exit.

    Z      - (output) SINGLE PRECISION array of dimension m.
             On exit Z = A X.

    ===================================================================== */

    magma_int_t blocks;
    if (n % num_threads==0)
        blocks = n/num_threads;
    else
        blocks = n/num_threads + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(num_threads, 1, 1);

    sgemv_kernel2<<< grid, threads, 0, magma_stream >>>(n, m, (m / sgemv_bs)*sgemv_bs,
                                     A, lda, x, incx, z);
}

__global__ void 
sgemvt_kernel1(magma_int_t n, magma_int_t m, float alpha, magma_int_t n1, float* A, magma_int_t lda,
              float *x, float *y)
{
  const magma_int_t inx = threadIdx.x;
  const magma_int_t iny = threadIdx.y;

  magma_int_t ind  = iny + __mul24(blockIdx.x,32);
  ind = inx + __mul24(ind,lda);
  magma_int_t ind2 = inx + __mul24(iny,32);

  A += ind;
  x += ind2;

  float res = 0.f;

  __shared__ float buff[sgemv_bs];
  __shared__ float la[32][33];

  for(magma_int_t i=0; i<n1; i += sgemv_bs ){
      buff[ind2]  = x[i];
      #pragma unroll
      for(magma_int_t j=0; j<16; j++)
         la[iny+__mul24(j,2)][inx] = A[j*__mul24(2,lda)];

      __syncthreads();
      #pragma unroll
      for(magma_int_t j=0; j < 16; j++)
        res += la[inx][iny*16+j]*buff[j+iny*16];

      A += 32;

      //===============================================
      #pragma unroll
      for(magma_int_t j=0; j<16; j++)
         la[iny+__mul24(j,2)][inx] = A[j*__mul24(2,lda)];

      __syncthreads();

      #pragma unroll
      for(magma_int_t j=0; j < 16; j++)
        res += la[inx][iny*16+j]*buff[j+32+iny*16];
      A += 32;
    }

    if (n>n1){
      if (ind2>=(n-n1))
         buff[ind2]=0.;
      else
         buff[ind2]  = x[n1];

      #pragma unroll
      for(magma_int_t j=0; j<16; j++)
         la[iny+__mul24(j,2)][inx] = A[j*__mul24(2,lda)];

     __syncthreads();

     if (n-n1>16){
        #pragma unroll
        for(magma_int_t j=0; j < 16; j++)
           res += la[inx][iny*16+j]*buff[j+iny*16];

        A += 32;
        #pragma unroll
        for(magma_int_t j=0; j<16; j++)
          la[iny+__mul24(j,2)][inx] = A[j*__mul24(2,lda)];

        __syncthreads();

        #pragma unroll
        for(magma_int_t j=0; j < 16; j++)
           res += la[inx][iny*16+j]*buff[j+32+iny*16];
     }
     else {
        #pragma unroll
        for(magma_int_t j=0; j < 16; j++)
          res += la[inx][iny*16+j]*buff[j+iny*16];
     }
  }
  ind = inx + __mul24(blockIdx.x,32);

  la[inx][iny]= res;
  if (ind<m){
     res = la[inx][0] + la[inx][1];
     y[ind] = alpha*res;
  }
}

__global__ void 
sgemvt_kernel2(magma_int_t n, magma_int_t m, float alpha,
               magma_int_t n1, float* A, magma_int_t lda, float *x, float *y)
{
  const magma_int_t inx = threadIdx.x;
  const magma_int_t iny = threadIdx.y;

  magma_int_t ind  = iny + __mul24(blockIdx.x,16);
  ind = inx + __mul24(ind,lda);
  magma_int_t ind2 = inx + __mul24(iny,16);
  if (ind2>31)
     ind2-=32;

  A += ind;
  x += ind2;

  float res = 0.f;

  __shared__ float buff[32];
  __shared__ float la[16][17];

  for(magma_int_t i=0; i<n1; i += 32 ){
     buff[ind2]  = x[i];
     #pragma unroll
     for(magma_int_t j=0; j<4; j++)
        la[iny+__mul24(j,4)][inx] = A[j*__mul24(4,lda)];

     __syncthreads();
     #pragma unroll
     for(magma_int_t j=0; j < 4; j++)
       res += la[inx][iny*4+j]*buff[j+iny*4];

     A += 16;

     __syncthreads();
     //===========================================
     #pragma unroll
     for(magma_int_t j=0; j<4; j++)
         la[iny+__mul24(j,4)][inx] = A[j*__mul24(4,lda)];

     __syncthreads();

     #pragma unroll
     for(magma_int_t j=0; j < 4; j++)
        res += la[inx][iny*4+j]*buff[j+16+iny*4];
     A += 16;
  }

  __syncthreads(); // 1
  if (n>n1){
     if (ind2>=(n-n1))
	buff[ind2]=0.;
     else
        buff[ind2]  = x[n1];

     __syncthreads();
     #pragma unroll
     for(magma_int_t j=0; j<4; j++)
         if (inx>=(n-n1))
            la[iny+__mul24(j,4)][inx] =  0.f;
         else
            la[iny+__mul24(j,4)][inx] = A[j*__mul24(4,lda)];

     __syncthreads();
     if (n-n1>4){
        #pragma unroll
        for(magma_int_t j=0; j < 4; j++){
           ind =  j+iny*4;
           res += la[inx][ind]*buff[ind];
        }
	A += 16;
        __syncthreads();
	#pragma unroll
	for(magma_int_t j=0; j<4; j++)
          if (inx+16>=(n-n1))
             la[iny+__mul24(j,4)][inx] = 0.f;
          else
             la[iny+__mul24(j,4)][inx] = A[j*__mul24(4,lda)];

        __syncthreads();

        #pragma unroll
	for(magma_int_t j=0; j < 4; j++){
           ind = j+4*iny;
           res += la[inx][ind]*buff[16+ind];
        }
     }
     else {
	#pragma unroll
        for(magma_int_t j=0; j < 4; j++){
          ind = j+iny*4;
          res += la[inx][ind]*buff[ind];
        }
     }
  }

  __syncthreads();
  ind = inx + __mul24(blockIdx.x,16);
  la[inx][iny]= res;
  __syncthreads();
  if (ind<m && iny==0){
     res = la[inx][0] + la[inx][1] + la[inx][2] + la[inx][3];
     y[ind] = alpha*res;
  }
}

extern "C" void
magmablas_sgemvt1_tesla(magma_int_t m, magma_int_t n, float alpha, float *A, magma_int_t lda,
                        float *x, float *z)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======

    This routine computes z = alpha A^t x on the GPU. 
    Recommended for large M and N.

    M      - (input) INTEGER.
             On entry, M specifies the number of rows of the matrix A.

    N      - (input) INTEGER.
             On entry, N specifies the number of columns of the matrix A

    A      - (input) SINGLE PRECISION array of dimension ( LDA, N ) on the GPU.

    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension m.

    Z      - (output) SINGLE PRECISION array of dimension n.
             On exit Z = alpha A^t X.

    ===================================================================== */
    magma_int_t blocks;

    if (n % 32==0)
        blocks = n/32;
    else
        blocks = n/32 + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(32, 2, 1);

    sgemvt_kernel1<<< grid, threads, 0, magma_stream >>>(m, n, alpha, (m / sgemv_bs)*sgemv_bs,
                                      A, lda, x, z);
}

extern "C" void
magmablas_sgemvt2_tesla(magma_int_t m, magma_int_t n, float alpha, float *A, magma_int_t lda,
                        float *x, float *z)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======

    This routine computes z = alpha A^t x on the GPU. Used in least squares 
    solver for N small (e.g. = BS, a block size of order 64, 128, etc).

    M      - (input) INTEGER.
             On entry, M specifies the number of rows of the matrix A.

    N      - (input) INTEGER.
             On entry, N specifies the number of columns of the matrix A

    A      - (input) SINGLE PRECISION array of dimension ( LDA, n ) on the GPU.

    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension m.

    Z      - (output) SINGLE PRECISION array of dimension n.
             On exit Z = alpha A^t X.

    ===================================================================== */

    magma_int_t blocks;

    if (n % 16==0)
        blocks = n/16;
    else
        blocks = n/16 + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(16, 4, 1);

    sgemvt_kernel2<<< grid, threads, 0, magma_stream >>>(m, n, alpha, (m / 32)*32,
                                      A, lda, x, z);
}

extern "C" void
magmablas_sgemvt_tesla(magma_int_t m, magma_int_t n, float alpha, float *A, magma_int_t lda, 
                       float *x, float *z)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======

    This routine computes z = alpha A^t x on the GPU.

    M      - (input) INTEGER.
             On entry, M specifies the number of rows of the matrix A.

    N      - (input) INTEGER.
             On entry, N specifies the number of columns of the matrix A

    A      - (input) SINGLE PRECISION array of dimension ( LDA, n ) on the GPU.

    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension m.

    Z      - (output) SINGLE PRECISION array of dimension n.
             On exit Z = alpha A^t X.

    ===================================================================== */

    if (n<=128)
      magmablas_sgemvt2_tesla(m, n, alpha, A, lda, x, z);
    else
      magmablas_sgemvt1_tesla(m, n, alpha, A, lda, x, z);
}

#undef num_threads
#undef sgemv_bs
