#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010
*/

#include "hipblas.h"
#include "magma.h"

#define num_threads 64
#define sgemv_bs 64
#define magmablas_sgemv_tesla magmablas_sgemv
#define magmablas_sgemvt_tesla magmablas_sgemvt

__global__ void 
sgemv_kernel(int n, int m, int n1, float* A, int lda, float *x, float *y)
{
  int ind = blockIdx.x*num_threads + threadIdx.x;

  A += ind;
  x += threadIdx.x;

  float res = 0.f;

  __shared__ float buff[sgemv_bs];
  for(int i=0; i<n1; i += sgemv_bs ){
    __syncthreads();
    buff[threadIdx.x]  = x[i];

    __syncthreads();
    #pragma unroll
    for(int j=0; j < sgemv_bs ; j++){
       res+=A[0]*buff[j];
       A+=lda;
    }
  }
  __syncthreads();

  if (m>n1){
     buff[threadIdx.x]  = x[n1];

     __syncthreads();
     for(int j=0; j<(m-n1); j++){
         res += A[0]*buff[j];
         A+=lda;
     }
  }

  if (ind<n)
     y[ind] = res;
}

__global__ void
sgemv_kernel2(int n, int m, int n1, float* A, int lda, 
              float *x, int incx, float *y)
{
  int ind = blockIdx.x*num_threads + threadIdx.x;

  A += ind;
  x += threadIdx.x * incx;

  float res = 0.f;

  __shared__ float buff[sgemv_bs];
  for(int i=0; i<n1; i += sgemv_bs ){
    __syncthreads();
    buff[threadIdx.x]  = x[i*incx];

    __syncthreads();
    #pragma unroll
    for(int j=0; j < sgemv_bs ; j++){
       res+=A[0]*buff[j];
       A+=lda;
    }
  }
  __syncthreads();

  if (m>n1){
     buff[threadIdx.x]  = x[n1*incx];

     __syncthreads();
     for(int j=0; j<(m-n1); j++){
         res += A[0]*buff[j];
         A+=lda;
     }
  }

  if (ind<n)
     y[ind] = res;
}

extern "C" void
magmablas_sgemv_tesla(int m, int n, float *A, int lda, float *x, float *z)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======

    This routine computes z = A x on the GPU.

    M      - (input) INTEGER.
             On entry, N specifies the number of rows of the matrix A.

    N      - (input) INTEGER.
             On entry, M specifies the number of columns of the matrix A

    A      - (input) SINGLE PRECISION array of dimension ( LDA, n ) on the GPU.
   
    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension n.
     
    Z      - (output) SINGLE PRECISION array of	dimension m. 
             On exit Z = A X.

    ===================================================================== */

    int blocks;
    if (m % num_threads==0)
        blocks = m/num_threads;
    else
        blocks = m/num_threads + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(num_threads, 1, 1);
 
    sgemv_kernel<<<grid, threads>>>(m, n, (n / sgemv_bs)*sgemv_bs, 
                                    A, lda, x, z);
}

extern "C" void
magmablas_sgemv2(int n, int m, float *A, int lda, float *x, int incx, float *z)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======

    This routine computes z = A x on the GPU.
    This version has INCX as an argument. 

    N      - (input) INTEGER.
             On entry, N specifies the number of rows of the matrix A.

    M      - (input) INTEGER.
             On entry, M specifies the number of columns of the matrix A

    A      - (input) SINGLE PRECISION array of dimension ( LDA, m ) on the GPU.

    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension n.

    INCX   - (input) Specifies the increment for the elements of X. 
             INCX must not be zero. Unchanged on exit.

    Z      - (output) SINGLE PRECISION array of dimension m.
             On exit Z = A X.

    ===================================================================== */

    int blocks;
    if (n % num_threads==0)
        blocks = n/num_threads;
    else
        blocks = n/num_threads + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(num_threads, 1, 1);

    sgemv_kernel2<<<grid, threads>>>(n, m, (m / sgemv_bs)*sgemv_bs,
                                     A, lda, x, incx, z);
}

__global__ void 
sgemvt_kernel1(int n, int m, float alpha, int n1, float* A, int lda,
              float *x, float *y)
{
  const int inx = threadIdx.x;
  const int iny = threadIdx.y;

  int ind  = iny + __mul24(blockIdx.x,32);
  ind = inx + __mul24(ind,lda);
  int ind2 = inx + __mul24(iny,32);

  A += ind;
  x += ind2;

  float res = 0.f;

  __shared__ float buff[sgemv_bs];
  __shared__ float la[32][33];

  for(int i=0; i<n1; i += sgemv_bs ){
      buff[ind2]  = x[i];
      #pragma unroll
      for(int j=0; j<16; j++)
         la[iny+__mul24(j,2)][inx] = A[j*__mul24(2,lda)];

      __syncthreads();
      #pragma unroll
      for(int j=0; j < 16; j++)
        res += la[inx][iny*16+j]*buff[j+iny*16];

      A += 32;

      //===============================================
      #pragma unroll
      for(int j=0; j<16; j++)
         la[iny+__mul24(j,2)][inx] = A[j*__mul24(2,lda)];

      __syncthreads();

      #pragma unroll
      for(int j=0; j < 16; j++)
        res += la[inx][iny*16+j]*buff[j+32+iny*16];
      A += 32;
    }

    if (n>n1){
      if (ind2>=(n-n1))
         buff[ind2]=0.;
      else
         buff[ind2]  = x[n1];

      #pragma unroll
      for(int j=0; j<16; j++)
         la[iny+__mul24(j,2)][inx] = A[j*__mul24(2,lda)];

     __syncthreads();

     if (n-n1>16){
        #pragma unroll
        for(int j=0; j < 16; j++)
           res += la[inx][iny*16+j]*buff[j+iny*16];

        A += 32;
        #pragma unroll
        for(int j=0; j<16; j++)
          la[iny+__mul24(j,2)][inx] = A[j*__mul24(2,lda)];

        __syncthreads();

        #pragma unroll
        for(int j=0; j < 16; j++)
           res += la[inx][iny*16+j]*buff[j+32+iny*16];
     }
     else {
        #pragma unroll
        for(int j=0; j < 16; j++)
          res += la[inx][iny*16+j]*buff[j+iny*16];
     }
  }
  ind = inx + __mul24(blockIdx.x,32);

  la[inx][iny]= res;
  if (ind<m){
     res = la[inx][0] + la[inx][1];
     y[ind] = alpha*res;
  }
}

__global__ void 
sgemvt_kernel2(int n, int m, float alpha,
               int n1, float* A, int lda, float *x, float *y)
{
  const int inx = threadIdx.x;
  const int iny = threadIdx.y;

  int ind  = iny + __mul24(blockIdx.x,16);
  ind = inx + __mul24(ind,lda);
  int ind2 = inx + __mul24(iny,16);
  if (ind2>31)
     ind2-=32;

  A += ind;
  x += ind2;

  float res = 0.f;

  __shared__ float buff[32];
  __shared__ float la[16][17];

  for(int i=0; i<n1; i += 32 ){
     buff[ind2]  = x[i];
     #pragma unroll
     for(int j=0; j<4; j++)
        la[iny+__mul24(j,4)][inx] = A[j*__mul24(4,lda)];

     __syncthreads();
     #pragma unroll
     for(int j=0; j < 4; j++)
       res += la[inx][iny*4+j]*buff[j+iny*4];

     A += 16;

     __syncthreads();
     //===========================================
     #pragma unroll
     for(int j=0; j<4; j++)
         la[iny+__mul24(j,4)][inx] = A[j*__mul24(4,lda)];

     __syncthreads();

     #pragma unroll
     for(int j=0; j < 4; j++)
        res += la[inx][iny*4+j]*buff[j+16+iny*4];
     A += 16;
  }

  __syncthreads(); // 1
  if (n>n1){
     if (ind2>=(n-n1))
	buff[ind2]=0.;
     else
        buff[ind2]  = x[n1];

     __syncthreads();
     #pragma unroll
     for(int j=0; j<4; j++)
         if (inx>=(n-n1))
            la[iny+__mul24(j,4)][inx] =  0.f;
         else
            la[iny+__mul24(j,4)][inx] = A[j*__mul24(4,lda)];

     __syncthreads();
     if (n-n1>4){
        #pragma unroll
        for(int j=0; j < 4; j++){
           ind =  j+iny*4;
           res += la[inx][ind]*buff[ind];
        }
	A += 16;
        __syncthreads();
	#pragma unroll
	for(int j=0; j<4; j++)
          if (inx+16>=(n-n1))
             la[iny+__mul24(j,4)][inx] = 0.f;
          else
             la[iny+__mul24(j,4)][inx] = A[j*__mul24(4,lda)];

        __syncthreads();

        #pragma unroll
	for(int j=0; j < 4; j++){
           ind = j+4*iny;
           res += la[inx][ind]*buff[16+ind];
        }
     }
     else {
	#pragma unroll
        for(int j=0; j < 4; j++){
          ind = j+iny*4;
          res += la[inx][ind]*buff[ind];
        }
     }
  }

  __syncthreads();
  ind = inx + __mul24(blockIdx.x,16);
  la[inx][iny]= res;
  __syncthreads();
  if (ind<m && iny==0){
     res = la[inx][0] + la[inx][1] + la[inx][2] + la[inx][3];
     y[ind] = alpha*res;
  }
}

extern "C" void
magmablas_sgemvt1_tesla(int m, int n, float alpha, float *A, int lda,
                        float *x, float *z)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======

    This routine computes z = alpha A^t x on the GPU. 
    Recommended for large M and N.

    M      - (input) INTEGER.
             On entry, M specifies the number of rows of the matrix A.

    N      - (input) INTEGER.
             On entry, N specifies the number of columns of the matrix A

    A      - (input) SINGLE PRECISION array of dimension ( LDA, N ) on the GPU.

    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension m.

    Z      - (output) SINGLE PRECISION array of dimension n.
             On exit Z = alpha A^t X.

    ===================================================================== */
    int blocks;

    if (n % 32==0)
        blocks = n/32;
    else
        blocks = n/32 + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(32, 2, 1);

    sgemvt_kernel1<<<grid, threads>>>(m, n, alpha, (m / sgemv_bs)*sgemv_bs,
                                      A, lda, x, z);
}

extern "C" void
magmablas_sgemvt2_tesla(int m, int n, float alpha, float *A, int lda,
                        float *x, float *z)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======

    This routine computes z = alpha A^t x on the GPU. Used in least squares 
    solver for N small (e.g. = BS, a block size of order 64, 128, etc).

    M      - (input) INTEGER.
             On entry, M specifies the number of rows of the matrix A.

    N      - (input) INTEGER.
             On entry, N specifies the number of columns of the matrix A

    A      - (input) SINGLE PRECISION array of dimension ( LDA, n ) on the GPU.

    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension m.

    Z      - (output) SINGLE PRECISION array of dimension n.
             On exit Z = alpha A^t X.

    ===================================================================== */

    int blocks;

    if (n % 16==0)
        blocks = n/16;
    else
        blocks = n/16 + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(16, 4, 1);

    sgemvt_kernel2<<<grid, threads>>>(m, n, alpha, (m / 32)*32,
                                      A, lda, x, z);
}

extern "C" void
magmablas_sgemvt_tesla(int m, int n, float alpha, float *A, int lda, 
                       float *x, float *z)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======

    This routine computes z = alpha A^t x on the GPU.

    M      - (input) INTEGER.
             On entry, M specifies the number of rows of the matrix A.

    N      - (input) INTEGER.
             On entry, N specifies the number of columns of the matrix A

    A      - (input) SINGLE PRECISION array of dimension ( LDA, n ) on the GPU.

    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension m.

    Z      - (output) SINGLE PRECISION array of dimension n.
             On exit Z = alpha A^t X.

    ===================================================================== */

    if (n<=128)
      magmablas_sgemvt2_tesla(m, n, alpha, A, lda, x, z);
    else
      magmablas_sgemvt1_tesla(m, n, alpha, A, lda, x, z);
}

#undef num_threads
#undef sgemv_bs
