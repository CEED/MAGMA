
#include <hip/hip_runtime.h>
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010
*/

__global__ void stranspose_32( float *B, int ldb, float *A, int lda )
{	
	__shared__ float a[32][33];
	
	int inx = threadIdx.x;
	int iny = threadIdx.y;
	int ibx = blockIdx.x*32;
	int iby = blockIdx.y*32;
	
	A += ibx + inx + __mul24( iby + iny, lda );
	B += iby + inx + __mul24( ibx + iny, ldb );
	
	a[iny+0][inx] = A[0*lda];
	a[iny+8][inx] = A[8*lda];
	a[iny+16][inx] = A[16*lda];
	a[iny+24][inx] = A[24*lda];
	
	__syncthreads();
	
	B[0*ldb] = a[inx][iny+0];
	B[8*ldb] = a[inx][iny+8];
	B[16*ldb] = a[inx][iny+16];
	B[24*ldb] = a[inx][iny+24];
} 

//
//	m, n - dimensions in the source matrix
//             This version works when m and n are divisible by 32.
//
extern "C" void 
magmablas_stranspose(float *odata, int ldo, 
                     float *idata, int ldi, 
                     int m, int n )
{
	//assert( (m%32) == 0 && (n%32) == 0, "misaligned transpose" );
	dim3 threads( 32, 8, 1 );
	dim3 grid( m/32, n/32, 1 );
	stranspose_32<<< grid, threads >>>( odata, ldo, idata, ldi );
}
