#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 0.2) --
	Univ. of Tennessee, Knoxville
	Univ. of California, Berkeley
	Univ. of Colorado, Denver
	November 2009
*/

#include "hipblas.h"
#include "magma.h"


__global__ void 
sgemvT32_kernel(int m, float alpha, float* A, int lda, float *x, float *y)
{
/*  -- MAGMA (version 0.2) --

    Purpose
    =======

    This routine computes y = alpha A^T x where A is single precision 
    array of dimension (32, M).
*/

    const int inx = threadIdx.x;
    const int iny = threadIdx.y;

    int ind  = iny + __mul24(blockIdx.x,32);
    ind = inx + __mul24(ind,lda);
    int ind2 = inx + __mul24(iny,32);

    A += ind;
    x += inx;

    float res = 0.f;

    __shared__ float buff[64];
    __shared__ float la[32][33];

    buff[ind2]  = x[0];

    #pragma unroll
    for(int j=0; j<16; j++)
      la[iny+__mul24(2,j)][inx] = A[j*__mul24(2,lda)];

    __syncthreads();

    // multiply with the sub-matrix
    #pragma unroll
    for(int j=0; j <16; j++)
      res += la[inx][j+iny*16]*buff[j+iny*16];

    ind = inx + __mul24(blockIdx.x,32);
    la[inx][iny]= res;

    __syncthreads();

    if (ind<m){
       res = la[inx][0] + la[inx][1];
       y[ind] = alpha*res;
    }
}

__global__ void 
dgemvT32_kernel(int m, double alpha, double* A, int lda, double *x, double *y)
{
/*  -- MAGMA (version 0.2) --

    Purpose
    =======

    This routine computes y = alpha A^T x where A is double precision
    array of dimension (32, M).
*/
 
    const int inx = threadIdx.x;
    const int iny = threadIdx.y;

    int ind  = iny + __mul24(blockIdx.x,32);
    ind = inx + __mul24(ind,lda);
    int ind2 = inx + __mul24(iny,32);

    A += ind;
    x += inx;

    double res = 0.f;

    __shared__ double buff[64];
    __shared__ double la[32][33];

    buff[ind2]  = x[0];
    #pragma unroll
    for(int j=0; j<16; j++)
      la[iny+__mul24(2, j)][inx] = A[j*__mul24(2,lda)];

    __syncthreads();

    #pragma unroll
    for(int j=0; j < 16; j++)
      res += la[inx][j+iny*16]*buff[j+iny*16];

    ind = inx + __mul24(blockIdx.x,32);
    la[inx][iny]= res;

    __syncthreads();

    if (ind<m){
      res = la[inx][0] + la[inx][1];
      y[ind] = alpha*res;
    }
}

__global__ void 
sgemv32_kernel(int n, float alpha, float* A, int lda, float *x, float *y)
{
/*  -- MAGMA (version 0.2) --

    Purpose
    =======

    This routine computes y = alpha A x where A is single precision
    array of dimension (N, 32).
*/

    int ind = blockIdx.x*32 + threadIdx.x;

    A += ind;
    x += threadIdx.x;

    float res = 0.f;

    __shared__ float buff[32];
    buff[threadIdx.x]  = x[0];

    __syncthreads();
    #pragma unroll
    for(int j=0; j < n; j++){
       res+=A[0]*buff[j];
       A+=lda;
    }

    if (ind<n)
      y[ind] = alpha*res;
}


__global__ void
dgemv32_kernel(int n, double alpha, double* A, int lda, double *x, double *y)
{
/*  -- MAGMA (version 0.2) --

    Purpose
    =======

    This routine computes y = alpha A x where A is double precision
    array of dimension (N, 32).
*/

    int ind = blockIdx.x*32 + threadIdx.x;

    A += ind;
    x += threadIdx.x;

    double res = 0.f;

    __shared__ double buff[32];
    buff[threadIdx.x]  = x[0];

    __syncthreads();
    #pragma unroll
    for(int j=0; j < n; j++){
       res+=A[0]*buff[j];
       A+=lda;
    }

    if (ind<n)
      y[ind] = alpha*res;
}


void magmablas_sgemv32(char tran, int n, float alpha, 
                       float *A, int lda, float *x, float *y)
{
/*  -- MAGMA (version 0.2) --

    Purpose
    =======

    This routine computes 
       y = alpha A^T x           for tran = 'T' / 't' or
       y = alpha A x 
    where A is single precision array of dimension (32, N) for 
    tran = 'T' / 't', or of dimension (N, 32) otherwise.
*/

    int blocks;
    if (n % 32 == 0)
      blocks = n/32;
    else
      blocks = n/32 + 1;
    dim3 grid(blocks, 1, 1);

    if (tran == 'T' || tran == 't'){
      dim3 threads(32, 2, 1);
      sgemvT32_kernel<<<grid, threads>>>(n, alpha, A, lda, x, y);
    }
    else 
    {
      dim3 threads(32, 1, 1);
      sgemv32_kernel<<<grid, threads>>>(n, alpha, A, lda, x, y);
    }
}


void magmablas_dgemv32(char tran, int n, double alpha, double *A, int lda,
		       double *x, double *y)
{
/*  -- MAGMA (version 0.2) --

    Purpose
    =======

    This routine computes
       y = alpha A^T x 	      	 for tran = 'T' / 't' or
      	y = alpha A x
    where A is double precision array of dimension (32, N) for
    tran = 'T' / 't', or of dimension (N, 32) otherwise.
*/

    int blocks;
    if (n % 32==0)
      blocks = n/32;
    else
      blocks = n/32 + 1;
    dim3 grid(blocks, 1, 1);

    if (tran == 'T' || tran == 't'){
      dim3 threads(32, 2, 1);
      dgemvT32_kernel<<<grid, threads>>>(n, alpha, A, lda, x, y);
    }
    else
    {
      dim3 threads(32, 1, 1);
      dgemv32_kernel<<<grid, threads>>>(n, alpha, A, lda, x, y);
    }
}
