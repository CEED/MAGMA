#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

       @precisions normal z -> s d c

*/
#include "common_magma.h"
#define PRECISION_z
#include "commonblas.h"

//
//	m, n - dimensions in the output (ha) matrix.
//             This routine copies the dat matrix from the GPU
//             to ha on the CPU. In addition, the output matrix
//             is transposed. The routine uses a buffer of size
//             2*lddb*nb pointed to by dB (lddb > m) on the GPU. 
//             Note that lda >= m and lddat >= n.
//
extern "C" void 
magmablas_zdtoht(hipDoubleComplex *dat, int ldda,
                 hipDoubleComplex  *ha, int lda,
                 hipDoubleComplex  *dB, int lddb,
                 int m, int n , int nb)
{
    int i = 0, j = 0, ib;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || ldda < n || lddb < m){
        printf("Wrong arguments in zdtoht.\n");
	return;
    }

    static hipStream_t stream[2];
    hipStreamCreate(&stream[0]);
    hipStreamCreate(&stream[1]);

    for(i=0; i<n; i+=nb){
       /* Move data from GPU to CPU using two buffers; first transpose the data on the GPU */
       ib   = min(n-i, nb);
       hipStreamSynchronize(stream[j%2]);
       magmablas_ztranspose2( dB + (j%2)*nb*lddb, lddb, dat+i, ldda, ib, m);
       hipMemcpy2DAsync(ha+i*lda, lda*sizeof(hipDoubleComplex),
                         dB + (j%2) * nb * lddb, lddb*sizeof(hipDoubleComplex),
                         sizeof(hipDoubleComplex)*m, ib, 
                         hipMemcpyDeviceToHost, stream[j%2]);
       j++;
    }

    hipStreamDestroy( stream[0] );
    hipStreamDestroy( stream[1] );
}

