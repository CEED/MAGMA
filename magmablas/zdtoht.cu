#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

       @precisions normal z -> s d c

*/
#include "common_magma.h"
#define PRECISION_z
#include "commonblas.h"

//
//	m, n - dimensions in the output (ha) matrix.
//             This routine copies the dat matrix from the GPU
//             to ha on the CPU. In addition, the output matrix
//             is transposed. The routine uses a buffer of size
//             2*lddb*nb pointed to by dB (lddb > m) on the GPU. 
//             Note that lda >= m and lddat >= n.
//
extern "C" void 
magmablas_zdtoht(hipDoubleComplex *dat, int ldda,
                 hipDoubleComplex  *ha, int lda,
                 hipDoubleComplex  *dB, int lddb,
                 int m, int n , int nb)
{
    int i = 0, j = 0, ib;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || ldda < n || lddb < m){
        printf("Wrong arguments in zdtoht.\n");
	return;
    }

    static hipStream_t stream[2];
    hipStreamCreate(&stream[0]);
    hipStreamCreate(&stream[1]);

    for(i=0; i<n; i+=nb){
       /* Move data from GPU to CPU using 2 buffers; 1st transpose the data on the GPU */
       ib   = min(n-i, nb);
       hipStreamSynchronize(stream[j%2]);
       magmablas_ztranspose2( dB + (j%2)*nb*lddb, lddb, dat+i, ldda, ib, m);
       hipMemcpy2DAsync(ha+i*lda, lda*sizeof(hipDoubleComplex),
                         dB + (j%2) * nb * lddb, lddb*sizeof(hipDoubleComplex),
                         sizeof(hipDoubleComplex)*m, ib, 
                         hipMemcpyDeviceToHost, stream[j%2]);
       j++;
    }

    hipStreamDestroy( stream[0] );
    hipStreamDestroy( stream[1] );
}

//===========================================================================
//  This version is similar to the above but for multiGPUs. The distribution
//  is 1D block cyclic. The input arrays are pointers for the corresponding
//  GPUs. The streams are passed as argument, in contrast to the single GPU
//  routine.
//===========================================================================
extern "C" void
magmablas_zdtoht2(int num_gpus, hipStream_t stream[][2],
                  hipDoubleComplex **dat, int *ldda,
                  hipDoubleComplex  *ha, int lda,
                  hipDoubleComplex  **dB, int lddb,
                  int m, int n , int nb)
{
    int i = 0, j[4] = {0, 0, 0, 0}, ib, k;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || lddb < m){
        printf("Wrong arguments in zdtoht.\n");
        return;
    }

    for(i=0; i<n; i+=nb){
       /* Move data from GPU to CPU using 2 buffers; 1st transpose the data on the GPU */
       k = (i/nb)%num_gpus;
       ib   = min(n-i, nb);
       hipSetDevice(k);

       hipStreamSynchronize(stream[k][j[k]%2]);
       magmablas_ztranspose2( dB[k] + (j[k]%2)*nb*lddb, lddb, 
                              dat[k]+i/(nb*num_gpus)*nb, ldda[k], ib, m);
       hipMemcpy2DAsync(ha+i*lda, lda*sizeof(hipDoubleComplex),
                         dB[k] + (j[k]%2) * nb * lddb, lddb*sizeof(hipDoubleComplex),
                         sizeof(hipDoubleComplex)*m, ib,
                         hipMemcpyDeviceToHost,
                         stream[k][j[k]%2]);
       j[k]++;
    }
}

