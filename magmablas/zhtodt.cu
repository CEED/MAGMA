#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

       @precisions normal z -> s d c

*/
#include "common_magma.h"
#define PRECISION_z
#include "commonblas.h"

//
//	m, n - dimensions in the source (input) matrix.
//             This routine copies the ha matrix from the CPU
//             to dat on the GPU. In addition, the output matrix
//             is transposed. The routine uses a buffer of size
//             2*lddb*nb pointed to by dB (lddb > m). 
//             Note that lda >= m and lddat >= n.
//
extern "C" void 
magmablas_zhtodt(hipDoubleComplex  *ha, int lda, 
                 hipDoubleComplex *dat, int ldda,
                 hipDoubleComplex  *dB, int lddb,
                 int m, int n , int nb)
{
    int i = 0, j = 0, ib;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || ldda < n || lddb < m){
        printf("Wrong arguments in zhotodt.\n");
	return;
    }

    static hipStream_t stream[2];
    hipStreamCreate(&stream[0]);
    hipStreamCreate(&stream[1]);
   
    /* Move data from CPU to GPU in the first panel in the dB buffer */
    ib   = min(n-i, nb);
    hipMemcpy2DAsync(dB + (j%2) * nb * lddb, lddb*sizeof(hipDoubleComplex),
                      ha + i*lda, lda*sizeof(hipDoubleComplex),
                      sizeof(hipDoubleComplex)*m, ib,
                      hipMemcpyHostToDevice, stream[j%2]);
    j++;

    for(i=nb; i<n; i+=nb){
       /* Move data from CPU to GPU in the second panel in the dB buffer */
       ib   = min(n-i, nb);
       hipMemcpy2DAsync(dB + (j%2) * nb * lddb, lddb*sizeof(hipDoubleComplex),
                         ha+i*lda, lda*sizeof(hipDoubleComplex),
                         sizeof(hipDoubleComplex)*m, ib, 
                         hipMemcpyHostToDevice, stream[j%2]);
       j++;
  
       /* Make sure that the previous panel (i.e., j%2) has arrived 
          and transpose it directly into the dat matrix                  */
       hipStreamSynchronize(stream[j%2]);
       magmablas_ztranspose2( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, nb);
    }

    /* Transpose the last part of the matrix.                            */
    j++;
    hipStreamSynchronize(stream[j%2]);
    magmablas_ztranspose2( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, ib);

    hipStreamDestroy( stream[0] );
    hipStreamDestroy( stream[1] );
}

