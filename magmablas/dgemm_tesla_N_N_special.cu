#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal d -> s
*/
#include "common_magma.h"
#include "commonblas_d.h"

/*
 * daxpy computes c += alpha*b, where b and c are 16-element vectors.
 */
static __device__ void daxpy(
    double alpha,
    const double* __restrict__ b,
    double*       __restrict__ c )
{
    c[0]  += alpha * b[0];
    c[1]  += alpha * b[1];
    c[2]  += alpha * b[2];
    c[3]  += alpha * b[3];
    c[4]  += alpha * b[4];
    c[5]  += alpha * b[5];
    c[6]  += alpha * b[6];
    c[7]  += alpha * b[7];
    c[8]  += alpha * b[8];
    c[9]  += alpha * b[9];
    c[10] += alpha * b[10];
    c[11] += alpha * b[11];
    c[12] += alpha * b[12];
    c[13] += alpha * b[13];
    c[14] += alpha * b[14];
    c[15] += alpha * b[15];
}


__global__ void
dgemm_kernel_N_N_64_16_16_16_4_special(
    double*       __restrict__ C,
    const double* __restrict__ A,
    const double* __restrict__ B,
    int m, int n, int k,
    int lda, int ldb, int ldc,
    double alpha, double beta )
{
/*  -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

    Purpose:
    ========
    This routine computes
        C = alpha * A*B + beta * C

    B is put into shared memory
    Parameters Used:
        blk_M=64 blk_N=16 blk_K=16 nthd_x=16 nthd_y=4

    This kernel is for matrices divisible by the corresponding
    blocking sizes.
    ===============================================================  */

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    const int ibx = blockIdx.x * 64;
    const int iby = blockIdx.y * 16;

    const int idt = ty * 16 + tx;

    B += tx + __mul24(iby+ty, ldb);
    A += ibx + idt;
    C += ibx + idt + __mul24(iby, ldc);

    const double *Bend = B + k;

    double Cb[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
    m = 2*lda;
    n = 3*lda;

    do {
        //double Ab[4] = {A[0], A[lda], A[2*lda], A[3*lda]};
        double Ab[4] = {A[0], A[lda], A[m], A[n]};
        __shared__ double Bb[16][17];
        Bb[tx][ty+0 ] = B[0];
        Bb[tx][ty+4 ] = B[4*ldb];
        Bb[tx][ty+8 ] = B[8*ldb];
        Bb[tx][ty+12] = B[12*ldb];

        __syncthreads();

        A += 4 * lda;
        daxpy( Ab[0], &Bb[0][0], Cb );  Ab[0] = A[0];
        daxpy( Ab[1], &Bb[1][0], Cb );  Ab[1] = A[lda];
        daxpy( Ab[2], &Bb[2][0], Cb );  Ab[2] = A[m];
        daxpy( Ab[3], &Bb[3][0], Cb );  Ab[3] = A[n];

        A += 4 * lda;
        daxpy( Ab[0], &Bb[4][0], Cb );  Ab[0] = A[0];
        daxpy( Ab[1], &Bb[5][0], Cb );  Ab[1] = A[lda];
        daxpy( Ab[2], &Bb[6][0], Cb );  Ab[2] = A[m];
        daxpy( Ab[3], &Bb[7][0], Cb );  Ab[3] = A[n];

        A += 4 * lda;
        daxpy( Ab[0], &Bb[8][0],  Cb );  Ab[0] = A[0];
        daxpy( Ab[1], &Bb[9][0],  Cb );  Ab[1] = A[lda];
        daxpy( Ab[2], &Bb[10][0], Cb );  Ab[2] = A[m];
        daxpy( Ab[3], &Bb[11][0], Cb );  Ab[3] = A[n];

        A += 4 * lda;
        daxpy( Ab[0], &Bb[12][0], Cb );
        daxpy( Ab[1], &Bb[13][0], Cb );
        daxpy( Ab[2], &Bb[14][0], Cb );
        daxpy( Ab[3], &Bb[15][0], Cb );

        B += 16;

        __syncthreads();
    } while (B < Bend);

    #pragma unroll 16
    for(int i=0; i < 16; i++) {
        C[0] = alpha * Cb[i] + beta * C[0];
        C += ldc;
    }
}


extern "C" void
magmablas_dgemm_N_N_64_16_16_16_4_special(
    double *C, const double *A, const double *B,
    magma_int_t m, magma_int_t n, magma_int_t k,
    magma_int_t lda, magma_int_t ldb, magma_int_t ldc,
    double alpha, double beta )
{
    dim3 threads( 16, 4 );
    dim3 grid( m/64, n/16 );
    dgemm_kernel_N_N_64_16_16_16_4_special<<< grid, threads, 0, magma_stream >>>
        ( C, A, B, m, n, k, lda, ldb, ldc, alpha, beta );
}
