#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011
*/
#include "common_magma.h"
#include "commonblas_d.h"

__global__ void  
dgemm_kernel_ab_0(double *C, const double *A, const double *B,
                  int m, int n, int k, 
                  int lda, int ldb, int ldc, 
                  double alpha, double beta)
{
        const int tx = threadIdx.x;
        const int ty = threadIdx.y;

        int ibx = blockIdx.x * 64;
        int iby = blockIdx.y *16;

        const int idt = ty * 16 + tx;


        C += ibx +idt +__mul24(iby,ldc);

        ibx = ibx+idt - m  ;
        
        if( (iby+16)>=n) { 
                lda = n-iby;
        }
        else    {
                lda = 16;
        }
        if( ibx >= 0 )
                lda = 0 ;
        else lda = lda ;

        switch(lda){
                case 16:
                        C[ 0    ] =0;
                        C[ 1*ldc] =0;
                        C[ 2*ldc] =0;
                        C[ 3*ldc] =0;
                        C[ 4*ldc] =0;
                        C[ 5*ldc] =0;
                        C[ 6*ldc] =0;
                        C[ 7*ldc] =0;
                        C[ 8*ldc] =0;
                        C[ 9*ldc] =0;
                        C[10*ldc] =0;
                        C[11*ldc] =0;
                        C[12*ldc] =0;
                        C[13*ldc] =0;
                        C[14*ldc] =0;
                        C[15*ldc] =0;
                        break;
                case 0:
                        break;
                case 15:
                        C[ 0    ] =0;
                        C[ 1*ldc] =0;
                        C[ 2*ldc] =0;
                        C[ 3*ldc] =0;
                        C[ 4*ldc] =0;
                        C[ 5*ldc] =0;
                        C[ 6*ldc] =0;
                        C[ 7*ldc] =0;
                        C[ 8*ldc] =0;
                        C[ 9*ldc] =0;
                        C[10*ldc] =0;
                        C[11*ldc] =0;
                        C[12*ldc] =0;
                        C[13*ldc] =0;
                        C[14*ldc] =0;
                        break;
                case 14:
                        C[ 0    ] =0;
                        C[ 1*ldc] =0;
                        C[ 2*ldc] =0;
                        C[ 3*ldc] =0;
                        C[ 4*ldc] =0;
                        C[ 5*ldc] =0;
                        C[ 6*ldc] =0;
                        C[ 7*ldc] =0;
                        C[ 8*ldc] =0;
                        C[ 9*ldc] =0;
                        C[10*ldc] =0;
                        C[11*ldc] =0;
                        C[12*ldc] =0;
                        C[13*ldc] =0;
                        break;
                case 13:
                        C[ 0    ] =0;
                        C[ 1*ldc] =0;
                        C[ 2*ldc] =0;
                        C[ 3*ldc] =0;
                        C[ 4*ldc] =0;
                        C[ 5*ldc] =0;
                        C[ 6*ldc] =0;
                        C[ 7*ldc] =0;
                        C[ 8*ldc] =0;
                        C[ 9*ldc] =0;
                        C[10*ldc] =0;
                        C[11*ldc] =0;
                        C[12*ldc] =0;
                        break;
                case 12:
                        C[ 0    ] =0;
                        C[ 1*ldc] =0;
                        C[ 2*ldc] =0;
                        C[ 3*ldc] =0;
                        C[ 4*ldc] =0;
                        C[ 5*ldc] =0;
                        C[ 6*ldc] =0;
                        C[ 7*ldc] =0;
                        C[ 8*ldc] =0;
                        C[ 9*ldc] =0;
                        C[10*ldc] =0;
                        C[11*ldc] =0;
                        break;
                case 11:
                        C[ 0    ] =0;
                        C[ 1*ldc] =0;
                        C[ 2*ldc] =0;
                        C[ 3*ldc] =0;
                        C[ 4*ldc] =0;
                        C[ 5*ldc] =0;
                        C[ 6*ldc] =0;
                        C[ 7*ldc] =0;
                        C[ 8*ldc] =0;
                        C[ 9*ldc] =0;
                        C[10*ldc] =0;
                        break;
                case 10:
                        C[ 0    ] =0;
                        C[ 1*ldc] =0;
                        C[ 2*ldc] =0;
                        C[ 3*ldc] =0;
                        C[ 4*ldc] =0;
                        C[ 5*ldc] =0;
                        C[ 6*ldc] =0;
                        C[ 7*ldc] =0;
                        C[ 8*ldc] =0;
                        C[ 9*ldc] =0;
                        break;
                case 9:
                        C[ 0    ] =0;
                        C[ 1*ldc] =0;
                        C[ 2*ldc] =0;
                        C[ 3*ldc] =0;
                        C[ 4*ldc] =0;
                        C[ 5*ldc] =0;
                        C[ 6*ldc] =0;
                        C[ 7*ldc] =0;
                        C[ 8*ldc] =0;
                        break;
                case 8:
                        C[ 0    ] =0;
                        C[ 1*ldc] =0;
                        C[ 2*ldc] =0;
                        C[ 3*ldc] =0;
                        C[ 4*ldc] =0;
                        C[ 5*ldc] =0;
                        C[ 6*ldc] =0;
                        C[ 7*ldc] =0;
                        break;
                case 7:
                        C[ 0    ] =0;
                        C[ 1*ldc] =0;
                        C[ 2*ldc] =0;
                        C[ 3*ldc] =0;
                        C[ 4*ldc] =0;
                        C[ 5*ldc] =0;
                        C[ 6*ldc] =0;
                        break;
                case 6:
                        C[ 0    ] =0;
                        C[ 1*ldc] =0;
                        C[ 2*ldc] =0;
                        C[ 3*ldc] =0;
                        C[ 4*ldc] =0;
                        C[ 5*ldc] =0;
                        break;
                case 5:
                        C[ 0    ] =0;
                        C[ 1*ldc] =0;
                        C[ 2*ldc] =0;
                        C[ 3*ldc] =0;
                        C[ 4*ldc] =0;
                        break;
                case 4:
                        C[ 0    ] =0;
                        C[ 1*ldc] =0;
                        C[ 2*ldc] =0;
                        C[ 3*ldc] =0;
                        break;
                case 3:
                        C[ 0    ] =0;
                        C[ 1*ldc] =0;
                        C[ 2*ldc] =0;
                        break;
                case 2:
                        C[ 0    ] =0;
                        C[ 1*ldc] =0;
                        break;
                case 1:
                        C[ 0    ] =0;
                        break;
        }
}

extern "C" void
magmablas_dgemm_kernel_ab_0(double *C, const double *A, const double *B,
                            magma_int_t m, magma_int_t n, magma_int_t k, 
                            magma_int_t lda, magma_int_t ldb, magma_int_t ldc, 
                            double alpha, double beta)
{
        dim3 threads( 16, 4 );
        dim3 grid(m/64+(m%64!=0),n/16+(n%16!=0));
        dgemm_kernel_ab_0<<< grid, threads, 0, magma_stream >>>(C, A, B, 
                                               m, n, k, 
                                               lda, ldb, ldc,
                                               alpha, beta);
}
