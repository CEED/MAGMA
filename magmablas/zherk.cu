#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 0.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
	   Univ. of Colorado, Denver
       June 2009
*/

#include "hipblas.h"
#include "magma.h"

extern "C" void
magmablas_zherk(char uplo, char trans, int n, int k, double alpha, 
            	double2 *A, int lda, double beta, double2 *C, int ldc){
/*  -- MAGMA (version 0.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       June 2009

    Purpose   
    =======   

    ZHERK  performs one of the hermitian rank k operations   
       C := alpha*A*conjg( A' ) + beta*C,   
    or   
       C := alpha*conjg( A' )*A + beta*C,   
    where  alpha and beta  are  real scalars,  C is an  n by n  hermitian 
    matrix and  A  is an  n by k  matrix in the  first case and a  k by n 
    matrix in the second case.   

    Parameters   
    ==========   

    UPLO   - CHARACTER
             On  entry,   UPLO  specifies  whether  the  upper  or  lower 
             triangular  part  of the  array  C  is to be  referenced  as 
             follows:   
                UPLO = 'U' or 'u'   Only the  upper triangular part of  C 
                                    is to be referenced.   
                UPLO = 'L' or 'l'   Only the  lower triangular part of  C 
                                    is to be referenced.   

             Unchanged on exit.   

    TRANS  - CHARACTER   
             On entry,  TRANS  specifies the operation to be performed as 
             follows:   
                TRANS = 'N' or 'n'   C := alpha*A*conjg( A' ) + beta*C.   
                TRANS = 'C' or 'c'   C := alpha*conjg( A' )*A + beta*C.   

             Unchanged on exit.   

    N      - INTEGER.   
             On entry,  N specifies the order of the matrix C.  N must be 
             at least zero.   
             Unchanged on exit.   

    K      - INTEGER.   
             On entry with  TRANS = 'N' or 'n',  K  specifies  the number 
             of  columns   of  the   matrix   A,   and  on   entry   with 
             TRANS = 'C' or 'c',  K  specifies  the number of rows of the 
             matrix A.  K must be at least zero.   
             Unchanged on exit.   

    ALPHA  - DOUBLE.   
             On entry, ALPHA specifies the scalar alpha.   
             Unchanged on exit.   

    A      - DOUBLE COMPLEX array of DIMENSION ( LDA, ka ), where ka is 
             k  when  TRANS = 'N' or 'n',  and is  n  otherwise.   
             Before entry with  TRANS = 'N' or 'n',  the  leading  n by k 
             part of the array  A  must contain the matrix  A,  otherwise 
             the leading  k by n  part of the array  A  must contain  the 
             matrix A.   
             Unchanged on exit.   

    LDA    - INTEGER.   
             On entry, LDA specifies the first dimension of A as declared 
             in  the  calling  (sub)  program.   When  TRANS = 'N' or 'n' 
             then  LDA must be at least  max( 1, n ), otherwise  LDA must 
             be at least  max( 1, k ).   
             Unchanged on exit.   

    BETA   - DOUBLE.   
             On entry, BETA specifies the scalar beta.   
             Unchanged on exit.   

    C      - DOUBLE COMPLEX array of DIMENSION ( LDC, n ).   
             Before entry  with  UPLO = 'U' or 'u',  the leading  n by n 
             upper triangular part of the array C must contain the upper 
             triangular part  of the  hermitian matrix  and the strictly 
             lower triangular part of C is not referenced.  On exit, the 
             upper triangular part of the array  C is overwritten by the 
             upper triangular part of the updated matrix.   
             Before entry  with  UPLO = 'L' or 'l',  the leading  n by n 
             lower triangular part of the array C must contain the lower 
             triangular part  of the  hermitian matrix  and the strictly 
             upper triangular part of C is not referenced.  On exit, the 
             lower triangular part of the array  C is overwritten by the 
             lower triangular part of the updated matrix.   
             Note that the imaginary parts of the diagonal elements need 
             not be set,  they are assumed to be zero,  and on exit they 
             are set to zero.   

    LDC    - INTEGER.   
             On entry, LDC specifies the first dimension of C as declared 
             in  the  calling  (sub)  program.   LDC  must  be  at  least 
             max( 1, n ).   
             Unchanged on exit.   

    Level 3 Blas routine.   

    ===================================================================== */

    int ka, ldamin;
    if (trans == 'N' || trans == 'n')
       ka = k, ldamin = n;
    else
       ka = n, ldamin = k;


    double2 *a = (double2*)malloc(ka*ldamin * sizeof(double2));
    double2 *c = (double2*)malloc(n*n * sizeof(double2));

    hipblasGetMatrix(ldamin, ka, sizeof(double2), A, lda, a, ldamin);
    hipblasGetMatrix(n, n, sizeof(double2), C, ldc, c, n);

    if (ldamin>0)
    zherk_(&uplo, &trans, &n, &k, &alpha, a, &ldamin, &beta, c, &n);

    hipblasSetMatrix(n, n, sizeof(double2), c, n, C, ldc);

    free(a);
    free(c);
}

