/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal z -> z
       
*/
#include "common_magma.h"

#define magmablas_zgemv_tesla magmablas_zgemv

extern "C" void
magmablas_zgemv_tesla(char trans, int m, int n, 
                      hipDoubleComplex alpha, hipDoubleComplex *A, int lda, 
                                             hipDoubleComplex *x, int incx, 
                      hipDoubleComplex beta,  hipDoubleComplex *y, int incy) 
{
    hipblasZgemv(trans, m, n, alpha, A, lda, x, incx, beta, y, incy);
}
