/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal z -> z
       
*/
#include "common_magma.h"

#define magmablas_zgemv_tesla magmablas_zgemv

extern "C" void
magmablas_zgemv_tesla(char trans, magma_int_t m, magma_int_t n, 
                      hipDoubleComplex alpha, const hipDoubleComplex *A, magma_int_t lda, 
                                             const hipDoubleComplex *x, magma_int_t incx, 
                      hipDoubleComplex beta,  hipDoubleComplex       *y, magma_int_t incy) 
{
    hipblasZgemv(trans, m, n, alpha, A, lda, x, incx, beta, y, incy);
}
