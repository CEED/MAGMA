#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

       @precisions normal z -> z
       
*/
#include "common_magma.h"

#define magmablas_zgemv_tesla magmablas_zgemv

extern "C" void
magmablas_zgemv_tesla(char trans, int m, int n, double2 alpha, double2 *A, int lda, double2 *x, int incx, double2 beta, double2 *y, int incy) 
{
    hipblasZgemv(trans, m, n, alpha, A, lda, x, incx, beta, y, incy);
}
