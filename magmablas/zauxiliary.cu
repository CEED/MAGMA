#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

       @precisions normal z -> s d c

*/

#include <hipblas.h>
#include "magma.h"

/* ////////////////////////////////////////////////////////////////////////////
   -- This is an auxiliary routine called from zgehrd.  The routine is called
      in 16 blocks, 32 thread per block and initializes to zero the 1st 
      32x32 block of A.
*/

__global__ void zset_to_zero(hipDoubleComplex *A, int lda){
    int ind = blockIdx.x*lda + threadIdx.x;
    
    A += ind;
    A[0] = MAGMA_Z_ZERO;
//   A[16*lda] = 0.;
}

__global__ void zset_nbxnb_to_zero(int nb, hipDoubleComplex *A, int lda){
   int ind = blockIdx.x*lda + threadIdx.x, i, j;

   A += ind;
   for(i=0; i<nb; i+=32){
     for(j=0; j<nb; j+=32)
         A[j] = MAGMA_Z_ZERO;
     A += 32*lda;
   }
}

void zzero_32x32_block(hipDoubleComplex *A, int lda)
{
  // zset_to_zero<<<16, 32>>>(A, lda);
  zset_to_zero<<<32, 32>>>(A, lda);
}

void zzero_nbxnb_block(int nb, hipDoubleComplex *A, int lda)
{
  zset_nbxnb_to_zero<<<32, 32>>>(nb, A, lda);
}

/* ////////////////////////////////////////////////////////////////////////////
   -- GPU kernel for initializing a matrix by 0
*/
#define zlaset_threads 64

__global__ void zlaset(int m, int n, hipDoubleComplex *A, int lda){
   int ibx = blockIdx.x * zlaset_threads;
   int iby = blockIdx.y * 32;

   int ind = ibx + threadIdx.x;

   A += ind + __mul24(iby, lda);

   #pragma unroll
   for(int i=0; i<32; i++)
     if (iby+i < n && ind < m)
        A[i*lda] = MAGMA_Z_ZERO;
}

/* ////////////////////////////////////////////////////////////////////////////
   -- Set the m x n matrix pointed by A to 0 on the GPU.
*/
extern "C" void
magmablas_zlaset(magma_int_t m, magma_int_t n, 
                 hipDoubleComplex *A, magma_int_t lda)
{
   dim3 threads(zlaset_threads, 1, 1);
   dim3 grid(m/zlaset_threads+(m % zlaset_threads != 0), n/32+(n%32!=0));

   zlaset<<< grid, threads >>> (m, n, A, lda);
}

