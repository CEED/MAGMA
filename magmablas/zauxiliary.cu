#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

       @precisions normal z -> s d c

*/

#include <hipblas.h>
#include "magma.h"
#include "magma_lapack.h"

/* ////////////////////////////////////////////////////////////////////////////
   -- This is an auxiliary routine called from zgehrd.  The routine is called
      in 16 blocks, 32 thread per block and initializes to zero the 1st 
      32x32 block of A.
*/

__global__ void zset_to_zero(hipDoubleComplex *A, int lda){
    int ind = blockIdx.x*lda + threadIdx.x;
    
    A += ind;
    A[0] = MAGMA_Z_ZERO;
//   A[16*lda] = 0.;
}

__global__ void zset_nbxnb_to_zero(int nb, hipDoubleComplex *A, int lda){
   int ind = blockIdx.x*lda + threadIdx.x, i, j;

   A += ind;
   for(i=0; i<nb; i+=32){
     for(j=0; j<nb; j+=32)
         A[j] = MAGMA_Z_ZERO;
     A += 32*lda;
   }
}

void zzero_32x32_block(hipDoubleComplex *A, int lda)
{
  // zset_to_zero<<<16, 32>>>(A, lda);
  zset_to_zero<<<32, 32>>>(A, lda);
}

void zzero_nbxnb_block(int nb, hipDoubleComplex *A, int lda)
{
  zset_nbxnb_to_zero<<<32, 32>>>(nb, A, lda);
}

/* ////////////////////////////////////////////////////////////////////////////
   -- GPU kernel for initializing a matrix by 0
*/
#define zlaset_threads 64

__global__ void zlaset(int m, int n, hipDoubleComplex *A, int lda){
   int ibx = blockIdx.x * zlaset_threads;
   int iby = blockIdx.y * 32;

   int ind = ibx + threadIdx.x;

   A += ind + __mul24(iby, lda);

   #pragma unroll
   for(int i=0; i<32; i++)
     if (iby+i < n && ind < m)
        A[i*lda] = MAGMA_Z_ZERO;
}

/* ////////////////////////////////////////////////////////////////////////////
   -- Set the m x n matrix pointed by A to 0 on the GPU.
*/
extern "C" void
magmablas_zlaset(magma_int_t m, magma_int_t n, 
                 hipDoubleComplex *A, magma_int_t lda)
{
   dim3 threads(zlaset_threads, 1, 1);
   dim3 grid(m/zlaset_threads+(m % zlaset_threads != 0), n/32+(n%32!=0));

   zlaset<<< grid, threads >>> (m, n, A, lda);
}

/* ////////////////////////////////////////////////////////////////////////////
   -- Given two matrices, 'a' on the CPU and 'da' on the GPU, this function
      returns the Frobenious norm of the difference of the two matrices.
      The function is used for debugging.
*/
double cpu_gpu_zdiff(int M, int N, double2 * a, int lda, double2 *da, int ldda)
{
  int one = 1, j;
  double2 mone = MAGMA_Z_NEG_ONE;
  double  work[1];
  double2 *ha = (double2*)malloc( M * N * sizeof(double2));
  double res;

  hipblasGetMatrix(M, N, sizeof(double2), da, ldda, ha, M);
  for(j=0; j<N; j++)
    blasf77_zaxpy(&M, &mone, a+j*lda, &one, ha+j*M, &one);
  res = lapackf77_zlange("f", &M, &N, ha, &M, work);

  free(ha);
  return res;
}

