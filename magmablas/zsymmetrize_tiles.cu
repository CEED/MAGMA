#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c
       @author Mark Gates
*/
#include "common_magma.h"
#include <assert.h>

#define NB 64

/*
    Symmetrizes ntile tiles at a time, e.g., all diagonal tiles of a matrix.
    Grid is ntile x ceil(m/NB).
    Each tile is m x m, and is divided into block rows, each NB x m.
    Each block has NB threads.
    Each thread copies one row, iterating across all columns below diagonal.
    The bottom block of rows may be partially outside the matrix;
    if so, rows outside the matrix (i >= m) are disabled.
*/
__global__ void
zsymmetrize_tiles_lower( int m, magmaDoubleComplex *dA, int ldda, int mstride, int nstride )
{
    // shift dA to tile's top-left corner
    dA += blockIdx.x*(mstride + nstride*ldda);
    
    // dA iterates across row i and dAT iterates down column i.
    int i = blockIdx.y*NB + threadIdx.x;
    magmaDoubleComplex *dAT = dA;
    if ( i < m ) {
        dA  += i;
        dAT += i*ldda;
        magmaDoubleComplex *dAend = dA + i*ldda;
        while( dA < dAend ) {
            *dAT = hipConj(*dA);  // upper := lower
            dA  += ldda;
            dAT += 1;
        }
    }
}


// only difference with _lower version is direction dA=dAT instead of dAT=dA.
__global__ void
zsymmetrize_tiles_upper( int m, magmaDoubleComplex *dA, int ldda, int mstride, int nstride )
{
    // shift dA to tile's top-left corner
    dA += blockIdx.x*(mstride + nstride*ldda);
    
    // dA iterates across row i and dAT iterates down column i.
    int i = blockIdx.y*NB + threadIdx.x;
    magmaDoubleComplex *dAT = dA;
    if ( i < m ) {
        dA  += i;
        dAT += i*ldda;
        magmaDoubleComplex *dAend = dA + i*ldda;
        while( dA < dAend ) {
            *dA  = hipConj(*dAT);  // lower := upper
            dA  += ldda;
            dAT += 1;
        }
    }
}


/**
    Purpose
    -------
    
    ZSYMMETRIZE_TILES copies lower triangle to upper triangle, or vice-versa,
    to make some blocks of dA into general representations of a symmetric block.
    This processes NTILE blocks, typically the diagonal blocks.
    Each block is offset by mstride rows and nstride columns from the previous block.
    
    Arguments
    ---------
    
    @param[in]
    uplo    magma_uplo_t
            Specifies the part of the matrix dA that is valid on input.
      -     = MagmaUpper:      Upper triangular part
      -     = MagmaLower:      Lower triangular part
    
    @param[in]
    m       INTEGER
            The number of rows & columns of each square block of dA.  M >= 0.
    
    @param[in,out]
    dA      COMPLEX DOUBLE PRECISION array, dimension (LDDA,N)
            The matrix dA. N = m + nstride*(ntile-1).
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDDA >= max(1, m + mstride*(ntile-1)).
    
    @param[in]
    ntile   INTEGER
            Number of blocks to symmetrize.
    
    @param[in]
    mstride INTEGER
            Row offset from start of one block to start of next block.
    
    @param[in]
    nstride INTEGER
            Column offset from start of one block to start of next block.

    @ingroup magma_zaux2
    ********************************************************************/
extern "C" void
magmablas_zsymmetrize_tiles( magma_uplo_t uplo, magma_int_t m, magmaDoubleComplex *dA, magma_int_t ldda,
                             magma_int_t ntile, magma_int_t mstride, magma_int_t nstride )
{
    if ( m == 0 || ntile == 0 )
        return;
    
    assert( m >= 0 );
    assert( ldda >= m );
    assert( ldda >= (ntile - 1)*mstride + m );
    assert( ntile >= 0 );
    assert( mstride >= 0 );
    assert( nstride >= 0 );
    assert( mstride >= m || nstride >= m );  // prevent tile overlap
    
    dim3 threads( NB );
    dim3 grid( ntile, (m + NB - 1)/NB );
    
    //printf( "m %d, grid %d x %d, threads %d\n", m, grid.x, grid.y, threads.x );
    if ( uplo == MagmaUpper ) {
        zsymmetrize_tiles_upper<<< grid, threads, 0, magma_stream >>>( m, dA, ldda, mstride, nstride );
    }
    else if ( uplo == MagmaLower ) {
        zsymmetrize_tiles_lower<<< grid, threads, 0, magma_stream >>>( m, dA, ldda, mstride, nstride );
    }
    else {
        printf( "uplo has illegal value\n" );
        exit(1);
    }
}
