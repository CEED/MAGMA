#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010
*/

#include "stdio.h"
#include "hipblas.h"
#include "magma.h"
#include "constant.h"

#define num_threads 128
#define cgemv_bs 32
#define threadSize 128

#define MAGMA_Z_SET2REAL(v, t) v.x = (t); v.y = 0.0
#define magmablas_cgemv_fermi magmablas_cgemv 



inline __host__ __device__ float2 make_float2(float s)
{
	return make_float2(s, s);
}
inline __host__ __device__ float2 make_float2(int2 a)
{
	return make_float2(float(a.x), float(a.y));
}

// negate
inline __host__ __device__ float2 operator-(float2 &a)
{
	return make_float2(-a.x, -a.y);
}
// addition
inline __host__ __device__ float2 operator+(float2 a, float2 b)
{
	return make_float2(a.x + b.x, a.y + b.y);
}
inline __host__ __device__ void operator+=(float2 &a, float2 b)
{
	a.x += b.x; a.y += b.y;
}

// subtract
inline __host__ __device__ float2 operator-(float2 a, float2 b)
{
	return make_float2(a.x - b.x, a.y - b.y);
}
inline __host__ __device__ void operator-=(float2 &a, float2 b)
{
	a.x -= b.x; a.y -= b.y;
}

// multiply
inline __host__ __device__ float2 operator*(float2 a, float2 b)
{
    return make_float2(a.x * b.x - a.y * b.y, a.y * b.x + a.x * b.y);
}
inline __host__ __device__ float2 operator*(float2 a, float s)
{
	return make_float2(a.x * s, a.y * s);
}
inline __host__ __device__ float2 operator*(float s, float2 a)
{
	return make_float2(a.x * s, a.y * s);
}
inline __host__ __device__ void operator*=(float2 &a, float s)
{
	a.x *= s; a.y *= s;
}

inline __host__ __device__ float2 conjugate(float2 a)
{
   float2 b;
   b.x = a.x;
   b.y = 0.0f-a.y;
   return b;
}





__global__ void 
cgemvn_kernel1_fermi(int n, int m, int n1, float2 alpha, float2* A, int lda, float2 *x, float2 *y)
{
  int ind = blockIdx.x*num_threads + threadIdx.x;

  A += ind;

  float2 res;
  MAGMA_Z_SET2REAL(res, 0.0f);

  for(int i=0; i<n1; i += cgemv_bs ){

    #pragma unroll
    for(int j=0; j < cgemv_bs ; j++){
       res += A[0] * x[j];
       A   += lda;
    }
	x += cgemv_bs;
  }

  if (m>n1){

     for(int j=0; j<(m-n1); j++){
         res += A[0] * x[j];
         A   += lda;
     }
  }

  if (ind<n)
     y[ind] = alpha * res;

}

__global__ void 
cgemvn_kernel2_fermi(int n, int m, int n1, float2 alpha,  float2* A, int lda, float2 *x, float2 *y)
{
  int ind = blockIdx.x*num_threads + threadIdx.x;

  A += ind;
  x += threadIdx.x;

  float2 res;
  MAGMA_Z_SET2REAL(res, 0.0f);

  __shared__ float2 buff[num_threads];
  for(int i=0; i<n1; i += num_threads ){
    __syncthreads();
    buff[threadIdx.x]  = x[i];

    __syncthreads();
    #pragma unroll
    for(int j=0; j < num_threads ; j++){
       res+=A[0]*buff[j];
       A+=lda;
    }
  }
  __syncthreads();

  if (m>n1){
     buff[threadIdx.x]  = x[n1];

     __syncthreads();
     for(int j=0; j<(m-n1); j++){
         res += A[0]*buff[j];
         A+=lda;
     }
  }

  if (ind<n)
     y[ind] = alpha * res;
}

extern "C" void
magmablas_cgemvn_fermi(int n, int m, float2 alpha, float2 *A, int lda, float2 *x, float2 *y)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======

    This routine computes Y = alpha A x on the GPU.

    N      - (input) INTEGER.
             On entry, N specifies the number of rows of the matrix A.

    M      - (input) INTEGER.
             On entry, M specifies the number of columns of the matrix A

    A      - (input) SINGLE PRECISION array of dimension ( LDA, m ) on the GPU.
   
    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension m.
     
    Y      - (output) SINGLE PRECISION array of	dimension m. 
             On exit Y = alpha A X.

    ===================================================================== */

    int blocks;
    if (n % num_threads==0)
        blocks = n/num_threads;
    else
        blocks = n/num_threads + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(num_threads, 1, 1);
//    if(n<=8500) 
		cgemvn_kernel1_fermi<<<grid, threads>>>(n, m, (m / cgemv_bs)*cgemv_bs, 
			                           alpha, A, lda, x, y);
 /*   else 
		cgemvn_kernel2_fermi<<<grid, threads>>>(n, m, (m / num_threads)*num_threads, 
			                           alpha, A, lda, x, y);
  */
}



__global__ void 
cgemvt_kernel1_fermi(int m, int n, float2 alpha, int n1, float2* A, int lda,
              float2 *x, float2 *y)
{
	unsigned int tx = threadIdx.x;

	__shared__ float2 sdata[threadSize];
	

	float2 res;
    MAGMA_Z_SET2REAL(res, 0.0f);
	float2 zero;
    MAGMA_Z_SET2REAL(zero, 0.0f);
     
	for(int i=0; i<n1; i+= threadSize)
	{
		res += A[tx + i + lda * blockIdx.y] * x[tx + i];
	}

	
	if(m > n1)
	{
		if( tx + n1 <  m )
		{
			res  += A[tx + n1 + lda *blockIdx.y] * x[tx + n1];
		}
		else 
		{
			res  += zero;
		}
	}	

    sdata[tx] = res;
	__syncthreads();
    
    /*
	if(tx < 128) 
	{
		sdata[tx] += sdata[tx + 128];
	}
    __syncthreads();
	*/

	if(tx < 64) 
	{
		sdata[tx] += sdata[tx + 64];
	}
    __syncthreads();

	if(tx < 32) 
	{
		sdata[tx] += sdata[tx + 32];
	}

    if(tx == 0)
	{
		for(int i=1;i<32;i++)
		{
			sdata[tx] += sdata[tx + i];
		}
	}

    if( tx == 0 ) 
	{
		y[blockIdx.y] = sdata[0]; 		

		if (blockIdx.y < n)
		{
			y[blockIdx.y] = y[blockIdx.y] * alpha;
		}
	}
}


__global__ void 
cgemvt_kernel2_fermi(int m, int n, float2 alpha,
               int n1, float2* A, int lda, float2 *x, float2 *y)
{
  const int inx = threadIdx.x;
  const int iny = threadIdx.y;

  int ind  = iny + blockIdx.x * 16;
  ind = inx + ind * lda;
  int ind2 = inx + iny * 16;
  if (ind2>31)
     ind2-=32;

  A += ind;
  x += ind2;

  float2 res;
  MAGMA_Z_SET2REAL(res, 0.0f);
  float2 zero;
  MAGMA_Z_SET2REAL(zero, 0.0f);

  __shared__ float2 buff[32];
  __shared__ float2 la[16][17];

  for(int i=0; i<n1; i += 32 ){
     buff[ind2]  = x[i];
     #pragma unroll
     for(int j=0; j<4; j++)
        la[iny + j * 4][inx] = A[j* 4 * lda];

     __syncthreads();
     #pragma unroll
     for(int j=0; j < 4; j++)
       res += la[inx][iny*4+j]*buff[j+iny*4];

     A += 16;

     __syncthreads();
     //===========================================
     #pragma unroll
     for(int j=0; j<4; j++)
         la[iny+ j * 4][inx] = A[j* 4 * lda];

     __syncthreads();

     #pragma unroll
     for(int j=0; j < 4; j++)
        res += la[inx][iny*4+j]*buff[j+16+iny*4];
     A += 16;
  }

  __syncthreads(); // 1
  if (n>n1){
     if (ind2>=(n-n1))
	buff[ind2]=zero;
     else
        buff[ind2]  = x[n1];

     __syncthreads();
     #pragma unroll
     for(int j=0; j<4; j++)
         if (inx>=(n-n1))
            la[iny + j * 4][inx] =  zero;
         else
            la[iny + j * 4][inx] = A[j* 4 * lda];

     __syncthreads();
     if (n-n1>4){
        #pragma unroll
        for(int j=0; j < 4; j++){
           ind =  j+iny*4;
           res += la[inx][ind]*buff[ind];
        }
	A += 16;
        __syncthreads();
	#pragma unroll
	for(int j=0; j<4; j++)
          if (inx+16>=(n-n1))
             la[iny+ j * 4][inx] = zero;
          else
             la[iny+ j * 4][inx] = A[j* 4* lda];

        __syncthreads();

        #pragma unroll
	for(int j=0; j < 4; j++){
           ind = j+4*iny;
           res += la[inx][ind]*buff[16+ind];
        }
     }
     else {
	#pragma unroll
        for(int j=0; j < 4; j++){
          ind = j+iny*4;
          res += la[inx][ind]*buff[ind];
        }
     }
  }

  __syncthreads();
  ind = inx + blockIdx.x * 16;
  la[inx][iny]= res;
  __syncthreads();
  if (ind<n && iny==0){
     res = la[inx][0] + la[inx][1] + la[inx][2] + la[inx][3];
     y[ind] = alpha*res;
  }
}

extern "C" void
magmablas_cgemvt1_fermi(int m, int n, float2 alpha, float2 *A, int lda,
                  float2 *x, float2 *y)
{


    dim3 grid    ( 1,  n,  1);
    dim3 threads ( threadSize,   1,  1);

    cgemvt_kernel1_fermi<<<grid, threads>>>( m, n, alpha, ( m / threadSize) * threadSize,
                                       A, lda, x, y);

									  
}

extern "C" void
magmablas_cgemvt2_fermi(int m, int n, float2 alpha, float2 *A, int lda,
                  float2 *x, float2 *y)
{

    int blocks;

    if (n % 16==0)
        blocks = n/16;
    else
        blocks = n/16 + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(16, 4, 1);

    cgemvt_kernel2_fermi<<<grid, threads>>>(m, n, alpha, (m / 32)*32,
                                      A, lda, x, y);
}

extern "C" void
magmablas_cgemvt_fermi(int m, int n, float2 alpha, float2 *A, int lda, 
                 float2 *x, float2 *y)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======

    This routine computes y = alpha *  A^t *  x on the GPU.

    M      - (input) INTEGER.
             On entry, M specifies the number of rows of the matrix A.

    N      - (input) INTEGER.
             On entry, N specifies the number of columns of the matrix A

    A      - (input) SINGLE PRECISION array of dimension ( LDA, n ) on the GPU.

    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension m.

    Y      - (output) SINGLE PRECISION array of dimension n.
             On exit Y = alpha A^t X.

    ===================================================================== */

    if (n<=128)
      magmablas_cgemvt2_fermi(m, n, alpha, A, lda, x, y);
    else
      magmablas_cgemvt1_fermi(m, n, alpha, A, lda, x, y);
    

}


extern "C" void
magmablas_cgemv_fermi(char flag, int m, int n, float2 alpha, float2 *A, int lda, float2 *x, float2 *y) 
{

	if (flag == 'N' || flag == 'n')
	{
		magmablas_cgemvn_fermi(m,  n, alpha, A, lda, x, y);
	}
	else if(flag == 'T' || flag == 't')
	{
		magmablas_cgemvt_fermi(m,  n, alpha, A, lda, x, y);
	}
	else 
	{
		printf("Not Available\n");
	}
}


#undef num_threads
#undef cgemv_bs
#undef threadSize 
