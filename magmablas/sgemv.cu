#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 0.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       June 2009
*/

#include "hipblas.h"
#include "magma.h"

#define num_threads 64
#define sgemv_bs 64

__global__ void 
sgemv_kernel(int n, int m, int n1, float* A, int lda, float *x, float *y)
{
  int ind = blockIdx.x*num_threads + threadIdx.x;

  A += ind;
  x += threadIdx.x;

  float res = 0.f;

  __shared__ float buff[sgemv_bs];
  for(int i=0; i<n1; i += sgemv_bs ){
    __syncthreads();
    buff[threadIdx.x]  = x[i];

    __syncthreads();
    #pragma unroll
    for(int j=0; j < sgemv_bs ; j++){
       res+=A[0]*buff[j];
       A+=lda;
    }
  }
  __syncthreads();

  if (m>n1){
     buff[threadIdx.x]  = x[n1];

     __syncthreads();
     for(int j=0; j<(m-n1); j++){
         res += A[0]*buff[j];
         A+=lda;
     }
  }

  if (ind<n)
     y[ind] = res;
}

extern "C" void
magmablas_sgemv(int n, int m, float *A, int lda, float *x, float *z)
{
/*  -- MAGMA (version 0.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       June 2009

    Purpose
    =======

    This routine computes z = A x on the GPU.

    N      - (input) INTEGER.
             On entry, N specifies the number of rows of the matrix A.

    M      - (input) INTEGER.
             On entry, M specifies the number of columns of the matrix A

    A      - (input) SINGLE PRECISION array of dimension ( LDA, m ) on the GPU.
   
    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension m.
     
    Z      - (output) SINGLE PRECISION array of	dimension m. 
             On exit Z = A X.

    ===================================================================== */

    int blocks;
    if (n % num_threads==0)
        blocks = n/num_threads;
    else
        blocks = n/num_threads + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(num_threads, 1, 1);
 
    sgemv_kernel<<<grid, threads>>>(n, m, (m / sgemv_bs)*sgemv_bs, 
                                    A, lda, x, z);
}

#undef num_threads
#undef sgemv_bs
