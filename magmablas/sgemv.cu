#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 0.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       June 2009
*/

#include "hipblas.h"
#include "magma.h"

#define num_threads 64
#define sgemv_bs 64

__global__ void 
sgemv_kernel(int n, int m, int n1, float* A, int lda, float *x, float *y)
{
  int ind = blockIdx.x*num_threads + threadIdx.x;

  A += ind;
  x += threadIdx.x;

  float res = 0.f;

  __shared__ float buff[sgemv_bs];
  for(int i=0; i<n1; i += sgemv_bs ){
    __syncthreads();
    buff[threadIdx.x]  = x[i];

    __syncthreads();
    #pragma unroll
    for(int j=0; j < sgemv_bs ; j++){
       res+=A[0]*buff[j];
       A+=lda;
    }
  }
  __syncthreads();

  if (m>n1){
     buff[threadIdx.x]  = x[n1];

     __syncthreads();
     for(int j=0; j<(m-n1); j++){
         res += A[0]*buff[j];
         A+=lda;
     }
  }

  if (ind<n)
     y[ind] = res;
}

extern "C" void
magmablas_sgemv(int n, int m, float *A, int lda, float *x, float *z)
{
/*  -- MAGMA (version 0.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       June 2009

    Purpose
    =======

    This routine computes z = A x on the GPU.

    N      - (input) INTEGER.
             On entry, N specifies the number of rows of the matrix A.

    M      - (input) INTEGER.
             On entry, M specifies the number of columns of the matrix A

    A      - (input) SINGLE PRECISION array of dimension ( LDA, m ) on the GPU.
   
    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension m.
     
    Z      - (output) SINGLE PRECISION array of	dimension m. 
             On exit Z = A X.

    ===================================================================== */

    int blocks;
    if (n % num_threads==0)
        blocks = n/num_threads;
    else
        blocks = n/num_threads + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(num_threads, 1, 1);
 
    sgemv_kernel<<<grid, threads>>>(n, m, (m / sgemv_bs)*sgemv_bs, 
                                    A, lda, x, z);
}


__global__ void 
sgemvt_kernel(int n, int m, float alpha, int n1, float* A, int lda,
              float *x, float *y)
{
  const int inx = threadIdx.x;
  const int iny = threadIdx.y;

  int ind  = iny + __mul24(blockIdx.x,32);
  ind = inx + __mul24(ind,lda);
  int ind2 = inx + __mul24(iny,32);

  A += ind;
  x += ind2;

  float res = 0.f;

  __shared__ float buff[sgemv_bs];
  __shared__ float la[32][33];

  for(int i=0; i<n1; i += sgemv_bs ){
      buff[ind2]  = x[i];
      #pragma unroll
      for(int j=0; j<16; j++)
         la[iny+__mul24(j,2)][inx] = A[j*__mul24(2,lda)];

      __syncthreads();
      #pragma unroll
      for(int j=0; j < 16; j++)
        res += la[inx][iny*16+j]*buff[j+iny*16];

      A += 32;

      //===============================================
      #pragma unroll
      for(int j=0; j<16; j++)
         la[iny+__mul24(j,2)][inx] = A[j*__mul24(2,lda)];

      __syncthreads();

      #pragma unroll
      for(int j=0; j < 16; j++)
        res += la[inx][iny*16+j]*buff[j+32+iny*16];
      A += 32;
    }

    if (n>n1){
      if (ind2>=(n-n1))
         buff[ind2]=0.;
      else
         buff[ind2]  = x[n1];

      #pragma unroll
      for(int j=0; j<16; j++)
         la[iny+__mul24(j,2)][inx] = A[j*__mul24(2,lda)];

     __syncthreads();

     if (n-n1>16){
        #pragma unroll
        for(int j=0; j < 16; j++)
           res += la[inx][iny*16+j]*buff[j+iny*16];

        A += 32;
        #pragma unroll
        for(int j=0; j<16; j++)
          la[iny+__mul24(j,2)][inx] = A[j*__mul24(2,lda)];

        __syncthreads();

        #pragma unroll
        for(int j=0; j < 16; j++)
           res += la[inx][iny*16+j]*buff[j+32+iny*16];
     }
     else {
        #pragma unroll
        for(int j=0; j < 16; j++)
          res += la[inx][iny*16+j]*buff[j+iny*16];
     }
  }
  ind = inx + __mul24(blockIdx.x,32);

  la[inx][iny]= res;
  if (ind<n){
     res = la[inx][0] + la[inx][1];
     y[ind] = alpha*res;
  }
}


extern "C" void
magmablas_sgemvt(int n, int m, float alpha, float *A, int lda, 
                 float *x, float *z)
{
/*  -- MAGMA (version 0.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       June 2009

    Purpose
    =======

    This routine computes z = alpha A^t x on the GPU.

    N      - (input) INTEGER.
             On entry, N specifies the number of rows of the matrix A.

    M      - (input) INTEGER.
             On entry, M specifies the number of columns of the matrix A

    A      - (input) SINGLE PRECISION array of dimension ( LDA, m ) on the GPU.

    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension n.

    Z      - (output) SINGLE PRECISION array of dimension n.
             On exit Z = alpha A^t X.

    ===================================================================== */

    int blocks;
    if (m % 32==0)
        blocks = m/32;
    else
        blocks = m/32 + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(32, 2, 1);

    sgemvt_kernel<<<grid, threads>>>(n, m, alpha, (n / sgemv_bs)*sgemv_bs,
                                     A, lda, x, z);
}

#undef num_threads
#undef sgemv_bs
