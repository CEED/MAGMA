#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date
       
       @author Mark Gates
       @author Tingxing Dong
       @author Azzam Haidar

       @precisions normal z -> s d c
*/
#include "common_magma.h"
#include "commonblas_z.h"
#include "magma_templates.h"

#define PRECISION_z

#include "gemv_template_device.cuh"

#include "gemv_config/gemvn_param.h"
#include "gemv_config/gemvt_param.h"

#define version(s,v) s ## _V_ ## v


template<const int BLK_X, const int BLK_Y, const int TILE_SIZE>
__global__ void
zgemvn_template_kernel_fermi(
    int m, int n, magmaDoubleComplex alpha,
    const magmaDoubleComplex * __restrict__ A, int lda,
    const magmaDoubleComplex * __restrict__ x, int incx, magmaDoubleComplex beta,
    magmaDoubleComplex       *y, int incy)
{
#if (__CUDA_ARCH__ >= 200)

    gemvn_template_device<magmaDoubleComplex, BLK_X, BLK_Y, TILE_SIZE>(m, n, alpha, A, lda, x, incx, beta, y, incy);
#endif /* (__CUDA_ARCH__ >= 200) */
}


template<const int BLK_X, const int BLK_Y, const int TILE_SIZE>
void
zgemvn_template_fermi(
    magma_int_t m, magma_int_t n, magmaDoubleComplex alpha,
    const magmaDoubleComplex * __restrict__ A, magma_int_t lda,
    const magmaDoubleComplex * __restrict__ x, magma_int_t incx, magmaDoubleComplex beta,
    magmaDoubleComplex       *y, magma_int_t incy)
{

    dim3 grid( magma_ceildiv(m, TILE_SIZE) );
    dim3 threads( BLK_X, BLK_Y, 1 );

    zgemvn_template_kernel_fermi<BLK_X, BLK_Y, TILE_SIZE><<< grid, threads, 0, magma_stream >>>(m, n, alpha, A, lda, x, incx, beta, y, incy);

}



//////////////////////////////////////////////////////////////////////////////////////////

template<const int BLK_X, const int BLK_Y, const int TILE_SIZE, int CONJA>
__global__ void
zgemvc_template_kernel_fermi(
    int m, int n, magmaDoubleComplex alpha,
    const magmaDoubleComplex * __restrict__ A, int lda,
    const magmaDoubleComplex * __restrict__ x, int incx, magmaDoubleComplex beta,
    magmaDoubleComplex       *y, int incy)
{
#if (__CUDA_ARCH__ >= 200)
    gemvc_template_device<magmaDoubleComplex, BLK_X, BLK_Y, TILE_SIZE, CONJA>(m, n, alpha, A, lda, x, incx, beta, y, incy);
#endif /* (__CUDA_ARCH__ >= 200) */
}


template<const int BLK_X, const int BLK_Y, const int TILE_SIZE>
void
zgemvc_template_fermi(
    magma_int_t m, magma_int_t n, magmaDoubleComplex alpha,
    const magmaDoubleComplex * __restrict__ A, magma_int_t lda,
    const magmaDoubleComplex * __restrict__ x, magma_int_t incx, magmaDoubleComplex beta,
    magmaDoubleComplex       *y, magma_int_t incy, magma_int_t CONJA)
{

    dim3 grid    ( 1,  magma_ceildiv(n, TILE_SIZE),  1 );
    dim3 threads ( BLK_X, BLK_Y, 1 );

    if(CONJA == 1)
    {
        zgemvc_template_kernel_fermi<BLK_X, BLK_Y, TILE_SIZE, 1><<< grid, threads, 0, magma_stream >>>(m, n, alpha, A, lda, x, incx, beta, y, incy);
    }
    else
    {
        zgemvc_template_kernel_fermi<BLK_X, BLK_Y, TILE_SIZE, 0><<< grid, threads, 0, magma_stream >>>(m, n, alpha, A, lda, x, incx, beta, y, incy);
    }

}



//////////////////////////////////////////////////////////////////////////////////////////

/**
    Purpose
    -------
    ZGEMV performs one of the matrix-vector operations
    
        y := alpha*A*x    + beta*y,   or
        y := alpha*A**T*x + beta*y,   or
        y := alpha*A**H*x + beta*y,
    
    where alpha and beta are scalars, x and y are vectors and A is an
    m by n matrix.

    Arguments
    ----------
    @param[in]
    trans   magma_trans_t
            On entry, TRANS specifies the operation to be performed as
            follows:
      -     = MagmaNoTrans:    y := alpha*A  *x + beta*y
      -     = MagmaTrans:      y := alpha*A^T*x + beta*y
      -     = MagmaConjTrans:  y := alpha*A^H*x + beta*y

    @param[in]
    m       INTEGER
            On entry, m specifies the number of rows of the matrix A.

    @param[in]
    n       INTEGER
            On entry, n specifies the number of columns of the matrix A
 
    @param[in]
    alpha   COMPLEX_16
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    dA      COMPLEX_16 array of dimension ( LDDA, n ) on the GPU.
   
    @param[in]
    ldda    INTEGER
            LDDA specifies the leading dimension of A.

    @param[in]
    dx      COMPLEX_16 array of dimension
            n if trans == MagmaNoTrans
            m if trans == MagmaTrans or MagmaConjTrans
     
    @param[in]
    incx    Specifies the increment for the elements of X.
            INCX must not be zero.
  
    @param[in]
    beta    DOUBLE REAL
            On entry, BETA specifies the scalar beta. When BETA is
            supplied as zero then Y need not be set on input.

    @param[out]
    dy      DOUBLE PRECISION array of dimension
            m if trans == MagmaNoTrans
            n if trans == MagmaTrans or MagmaConjTrans

    @param[in]
    incy    Specifies the increment for the elements of Y.
            INCY must not be zero.

    @ingroup magma_dblas2
    ********************************************************************/
extern "C" void
magmablas_zgemv(
    magma_trans_t trans, magma_int_t m, magma_int_t n, magmaDoubleComplex alpha,
    magmaDoubleComplex_const_ptr dA, magma_int_t ldda,
    magmaDoubleComplex_const_ptr dx, magma_int_t incx,
    magmaDoubleComplex beta,
    magmaDoubleComplex_ptr dy, magma_int_t incy)
{
    magma_int_t info = 0;
    if ( trans != MagmaNoTrans && trans != MagmaTrans && trans != MagmaConjTrans )
        info = -1;
    else if ( m < 0 )
        info = -2;
    else if ( n < 0 )
        info = -3;
    else if ( ldda < m )
        info = -6;
    else if ( incx == 0 )
        info = -8;
    else if ( incy == 0 )
        info = -11;
    
    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }

    magma_int_t arch = magma_getdevice_arch();
    if ( arch < 200  ) {
        // --------------------
        // call CUDA ARCH 1.x version
        // magmablas for [sd] precisions, cublas for [zc] precisions.
        #if defined(PRECISION_z) || defined(PRECISION_c)
        magma_zgemv( trans, m, n, alpha, dA, ldda, dx, incx, beta, dy, incy );
        #else
        magmablas_zgemv_tesla( trans, m, n, alpha, dA, ldda, dx, incx, beta, dy, incy );
        #endif
        return;
    }
    
    
    // --------------------
    // CUDA ARCH 2.x (Fermi) version
    if ( trans == MagmaNoTrans ) {
        if(m <= 256){
            zgemvn_template_fermi<version(N, 137)>
                ( m, n, alpha, dA, ldda, dx, incx, beta, dy, incy );
        }
        else{
            zgemvn_template_fermi<version(N, 140)>
                ( m, n, alpha, dA, ldda, dx, incx, beta, dy, incy );
        }

    }
    else{

        magma_int_t CONJA = -1;

        if ( trans == MagmaConjTrans ) {
            CONJA = 1;
        }else if( trans == MagmaTrans ) {
            CONJA = 0;
        }
        else{
            return ;
        }       

        zgemvc_template_fermi<version(T, 189)>
            ( m, n, alpha, dA, ldda, dx, incx, beta, dy, incy, CONJA );

    }

}



