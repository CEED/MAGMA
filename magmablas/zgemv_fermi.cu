#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal z -> c
*/
#include "common_magma.h"
#include "commonblas_z.h"

#define PRECISION_z

#define num_threads 128
#define gemv_bs      32
#define threadSize  128

__global__ void
zgemvn_kernel1_fermi(
    int m, int n, int n1, magmaDoubleComplex alpha,
    const magmaDoubleComplex * __restrict__ A, int lda,
    const magmaDoubleComplex * __restrict__ x, magmaDoubleComplex beta,
    magmaDoubleComplex       * __restrict__ y)
{
#if (__CUDA_ARCH__ >= 200)
    int ind = blockIdx.x*num_threads + threadIdx.x;
    
    A += ind;
    
    magmaDoubleComplex res = MAGMA_Z_ZERO;
    
    for( int i=0; i < n1; i += gemv_bs ) {
        #pragma unroll
        for(int j=0; j < gemv_bs; j++) {
            res += A[0] * x[j];
            A   += lda;
        }
        x += gemv_bs;
    }
    
    if ( n > n1 ) {
        for(int j=0; j < (n-n1); j++) {
            res += A[0] * x[j];
            A   += lda;
        }
    }
    
    if ( ind < m )
        y[ind] = alpha * res + beta * y[ind];
#endif /* (__CUDA_ARCH__ >= 200) */
}

__global__ void
zgemvn_kernel2_fermi(
    int m, int n, int n1, magmaDoubleComplex alpha,
    const magmaDoubleComplex * __restrict__ A, int lda,
    const magmaDoubleComplex * __restrict__ x, magmaDoubleComplex beta,
    magmaDoubleComplex       * __restrict__ y)
{
#if (__CUDA_ARCH__ >= 200)
    int ind = blockIdx.x*num_threads + threadIdx.x;
    
    A += ind;
    x += threadIdx.x;
    
    magmaDoubleComplex res = MAGMA_Z_ZERO;
    
    __shared__ magmaDoubleComplex buff[num_threads];
    for( int i=0; i < n1; i += num_threads ) {
        __syncthreads();
        buff[threadIdx.x] = x[i];
        
        __syncthreads();
        #pragma unroll
        for(int j=0; j < num_threads; j++) {
            res += A[0]*buff[j];
            A += lda;
        }
    }
    __syncthreads();
    
    if ( n > n1 ) {
        buff[threadIdx.x] = x[n1];
        
        __syncthreads();
        for(int j=0; j<(n-n1); j++) {
            res += A[0]*buff[j];
            A += lda;
        }
    }
    
    if ( ind < m )
        y[ind] = alpha * res + beta * y[ind];
#endif /* (__CUDA_ARCH__ >= 200) */
}

extern "C" void
magmablas_zgemvn_fermi(
    magma_int_t m, magma_int_t n, magmaDoubleComplex alpha,
    const magmaDoubleComplex *A, magma_int_t lda,
    const magmaDoubleComplex *x, magmaDoubleComplex beta,
    magmaDoubleComplex       *y)
{
/*  -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

    Purpose
    =======

    This routine computes Y = alpha A x on the GPU.

    M       (input) INTEGER.
            On entry, M specifies the number of rows of the matrix A.

    N       (input) INTEGER.
            On entry, N specifies the number of columns of the matrix A

    A       (input) COMPLEX*16 array of dimension ( LDA, n ) on the GPU.
   
    LDA     (input) INTEGER.
            LDA specifies the leading dimension of A.

    X       (input) COMPLEX*16 array of dimension n.
     
    Y       (output) COMPLEX*16 array of dimension n.
            On exit Y = alpha A X.

    ===================================================================== */

    magma_int_t blocks = (m - 1)/num_threads + 1;
    dim3 grid(blocks, 1, 1);
    dim3 threads(num_threads, 1, 1);
    /*
    if ( m <= 8500 )
        zgemvn_kernel1_fermi<<< grid, threads, 0, magma_stream >>>
            (m, n, (n / gemv_bs)*gemv_bs, alpha, A, lda, x, y);
    else
    */
        zgemvn_kernel2_fermi<<< grid, threads, 0, magma_stream >>>
            (m, n, (n / num_threads)*num_threads, alpha, A, lda, x, beta, y);
}

__global__ void
zgemvt_kernel_fermi(
    int m, int n, magmaDoubleComplex alpha, int n1,
    const magmaDoubleComplex * __restrict__ A, int lda,
    const magmaDoubleComplex * __restrict__ x, magmaDoubleComplex beta,
    magmaDoubleComplex       * __restrict__ y)
{
#if (__CUDA_ARCH__ >= 200)
    unsigned int tx = threadIdx.x;

    __shared__ magmaDoubleComplex sdata[threadSize];

    magmaDoubleComplex res = MAGMA_Z_ZERO;
    magmaDoubleComplex c_zero = MAGMA_Z_ZERO;
 
    for(int i=0; i < n1; i += threadSize) {
        res += A[tx + i + lda * blockIdx.y] * x[tx + i];
    }

    if ( m > n1 ) {
        if ( tx + n1 < m ) {
            res += A[tx + n1 + lda*blockIdx.y] * x[tx + n1];
        }
        else {
            res += c_zero;
        }
    }

    sdata[tx] = res;
    __syncthreads();

    for(int s=blockDim.x/2; s > 32; s /= 2) {
        if ( tx < s ) {
            sdata[tx] += sdata[tx+s];
        }
        __syncthreads();
    }

    if ( tx < 32 ) {
        sdata[tx] += sdata[tx + 32];
    }

    if ( tx == 0 ) {
        for(int i=1; i < 32; i++) {
            sdata[tx] += sdata[tx + i];
        }
    }

    if ( tx == 0 ) {
        if ( blockIdx.y < n ) {
            y[blockIdx.y] = sdata[0] * alpha + beta * y[blockIdx.y];
        }
    }
#endif /* (__CUDA_ARCH__ >= 200) */
}

extern "C" void
magmablas_zgemvt_fermi(
    magma_int_t m, magma_int_t n, magmaDoubleComplex alpha,
    const magmaDoubleComplex *A, magma_int_t lda,
    const magmaDoubleComplex *x, magmaDoubleComplex beta,
    magmaDoubleComplex       *y)
{
/*  -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

    Purpose
    =======

    This routine computes y = alpha * A^T * x on the GPU.

    M       (input) INTEGER.
            On entry, M specifies the number of rows of the matrix A.

    N       (input) INTEGER.
            On entry, N specifies the number of columns of the matrix A

    A       (input) COMPLEX*16 array of dimension ( LDA, n ) on the GPU.

    LDA     (input) INTEGER.
            LDA specifies the leading dimension of A.

    X       (input) COMPLEX*16 array of dimension m.

    Y       (output) COMPLEX*16 array of dimension n.
            On exit Y = alpha A^T X.

    ===================================================================== */

    dim3 grid    ( 1, n, 1 );
    dim3 threads ( threadSize, 1, 1 );
    zgemvt_kernel_fermi<<< grid, threads, 0, magma_stream >>>
        (m, n, alpha, (m / threadSize) * threadSize, A, lda, x, beta, y );
}

__global__ void
zgemvc_kernel_fermi(
    int m, int n, magmaDoubleComplex alpha, int n1,
    const magmaDoubleComplex * __restrict__ A, int lda,
    const magmaDoubleComplex * __restrict__ x, magmaDoubleComplex beta,
    magmaDoubleComplex       * __restrict__ y)
{
#if (__CUDA_ARCH__ >= 200)
    unsigned int tx = threadIdx.x;

    __shared__ magmaDoubleComplex sdata[threadSize];

    magmaDoubleComplex res = MAGMA_Z_ZERO;
    magmaDoubleComplex c_zero = MAGMA_Z_ZERO;
 
    for(int i=0; i < n1; i += threadSize) {
        res += hipConj(A[tx + i + lda * blockIdx.y]) * x[tx + i];
    }

    if ( m > n1 ) {
        if ( tx + n1 < m ) {
            res += hipConj(A[tx + n1 + lda*blockIdx.y]) * x[tx + n1];
        }
        else {
            res += c_zero;
        }
    }

    sdata[tx] = res;
    __syncthreads();

    /*
    if ( tx < 128 ) {
        sdata[tx] += sdata[tx + 128];
    }
    __syncthreads();
    */

    if ( tx < 64 ) {
        sdata[tx] += sdata[tx + 64];
    }
    __syncthreads();

    if ( tx < 32 ) {
        sdata[tx] += sdata[tx + 32];
    }

    if ( tx == 0 ) {
        for(int i=1; i < 32; i++) {
            sdata[tx] += sdata[tx + i];
        }
    }

    if ( tx == 0 ) {
        if ( blockIdx.y < n ) {
            y[blockIdx.y] = sdata[0] * alpha + beta * y[blockIdx.y];
        }
    }
#endif /* (__CUDA_ARCH__ >= 200) */
}

extern "C" void
magmablas_zgemvc_fermi(
    magma_int_t m, magma_int_t n, magmaDoubleComplex alpha,
    const magmaDoubleComplex *A, magma_int_t lda,
    const magmaDoubleComplex *x,
    magmaDoubleComplex beta,
    magmaDoubleComplex *y)
{
/*  -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

    Purpose
    =======

    This routine computes y = alpha * A^H * x on the GPU.

    M       (input) INTEGER.
            On entry, M specifies the number of rows of the matrix A.

    N       (input) INTEGER.
            On entry, N specifies the number of columns of the matrix A

    A       (input) COMPLEX*16 array of dimension ( LDA, n ) on the GPU.

    LDA     (input) INTEGER.
            LDA specifies the leading dimension of A.

    X       (input) COMPLEX*16 array of dimension m.

    Y       (output) COMPLEX*16 array of dimension n.
            On exit Y = alpha A^H X.

    ===================================================================== */

    dim3 grid    ( 1, n, 1 );
    dim3 threads ( threadSize, 1, 1 );
    zgemvc_kernel_fermi<<< grid, threads, 0, magma_stream >>>
        (m, n, alpha, (m / threadSize) * threadSize, A, lda, x, beta, y);
}

extern "C" void
magmablas_zgemv(
    char trans, magma_int_t m, magma_int_t n, magmaDoubleComplex alpha,
    const magmaDoubleComplex *A, magma_int_t lda,
    const magmaDoubleComplex *x, magma_int_t incx,
    magmaDoubleComplex beta,
    magmaDoubleComplex *y, magma_int_t incy)
{
    magma_int_t arch = magma_getdevice_arch();
    if ( arch < 200  ) {
        // --------------------
        // call CUDA ARCH 1.x version
        // magmablas for [sd] precisions, cublas for [zc] precisions.
        #if defined(PRECISION_z) || defined(PRECISION_c)
        hipblasZgemv( trans, m, n, alpha, A, lda, x, incx, beta, y, incy );
        #else
        magmablas_zgemv_tesla( trans, m, n, alpha, A, lda, x, incx, beta, y, incy );
        #endif
        return;
    }
    
    // --------------------
    // CUDA ARCH 2.x (Fermi) version
    if ( incx == 1 && incy == 1 ) {
        if ( trans == 'N' || trans == 'n' ) {
            if ( m < 7000 ) {
                hipblasZgemv(trans, m, n, alpha, A, lda, x, incx, beta, y, incy);
            }
            else {
                magmablas_zgemvn_fermi(m, n, alpha, A, lda, x, beta, y);
            }
        }
        else if ( trans == 'T' || trans == 't' ) {
            magmablas_zgemvt_fermi(m, n, alpha, A, lda, x, beta, y);
        }
        else if ( trans == 'C' || trans == 'c' ) {
            magmablas_zgemvc_fermi(m, n, alpha, A, lda, x, beta, y);
        }
        else {
            fprintf( stderr, "trans = %c is invalid\n", trans );
        }
    }
    else {
        hipblasZgemv(trans, m, n, alpha, A, lda, x, incx, beta, y, incy);
    }
}

#undef num_threads
#undef gemv_bs
#undef threadSize
