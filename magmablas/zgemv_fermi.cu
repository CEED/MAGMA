#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010
*/

#include "stdio.h"
#include "hipblas.h"
#include "magma.h"
#include "constant.h"

#define num_threads 128
#define zgemv_bs 32
#define threadSize 128

#define MAGMA_Z_SET2REAL(v, t) v.x = (t); v.y = 0.0
#define magmablas_zgemv_fermi magmablas_zgemv


inline __host__ __device__ double2 make_double2(double s)
{
	return make_double2(s, s);
}
inline __host__ __device__ double2 make_double2(int2 a)
{
	return make_double2(double(a.x), double(a.y));
}

// negate
inline __host__ __device__ double2 operator-(double2 &a)
{
	return make_double2(-a.x, -a.y);
}
// addition
inline __host__ __device__ double2 operator+(double2 a, double2 b)
{
	return make_double2(a.x + b.x, a.y + b.y);
}
inline __host__ __device__ void operator+=(double2 &a, double2 b)
{
	a.x += b.x; a.y += b.y;
}

// subtract
inline __host__ __device__ double2 operator-(double2 a, double2 b)
{
	return make_double2(a.x - b.x, a.y - b.y);
}
inline __host__ __device__ void operator-=(double2 &a, double2 b)
{
	a.x -= b.x; a.y -= b.y;
}

// multiply
inline __host__ __device__ double2 operator*(double2 a, double2 b)
{
    return make_double2(a.x * b.x - a.y * b.y, a.y * b.x + a.x * b.y);
}
inline __host__ __device__ double2 operator*(double2 a, double s)
{
	return make_double2(a.x * s, a.y * s);
}
inline __host__ __device__ double2 operator*(double s, double2 a)
{
	return make_double2(a.x * s, a.y * s);
}
inline __host__ __device__ void operator*=(double2 &a, double s)
{
	a.x *= s; a.y *= s;
}

inline __host__ __device__ double2 conjugate(double2 a)
{
   double2 b;
   b.x = a.x;
   b.y = 0.0f-a.y;
   return b;
}





__global__ void 
zgemvn_kernel1_fermi(int n, int m, int n1, double2 alpha, double2* A, int lda, double2 *x, double2 *y)
{
  int ind = blockIdx.x*num_threads + threadIdx.x;

  A += ind;

  double2 res;
  MAGMA_Z_SET2REAL(res, 0.0f);

  for(int i=0; i<n1; i += zgemv_bs ){

    #pragma unroll
    for(int j=0; j < zgemv_bs ; j++){
       res += A[0] * x[j];
       A   += lda;
    }
	x += zgemv_bs;
  }

  if (m>n1){

     for(int j=0; j<(m-n1); j++){
         res += A[0] * x[j];
         A   += lda;
     }
  }

  if (ind<n)
     y[ind] = alpha * res;

}

__global__ void 
zgemvn_kernel2_fermi(int n, int m, int n1, double2 alpha,  double2* A, int lda, double2 *x, double2 *y)
{
  int ind = blockIdx.x*num_threads + threadIdx.x;

  A += ind;
  x += threadIdx.x;

  double2 res;
  MAGMA_Z_SET2REAL(res, 0.0f);

  __shared__ double2 buff[num_threads];
  for(int i=0; i<n1; i += num_threads ){
    __syncthreads();
    buff[threadIdx.x]  = x[i];

    __syncthreads();
    #pragma unroll
    for(int j=0; j < num_threads ; j++){
       res+=A[0]*buff[j];
       A+=lda;
    }
  }
  __syncthreads();

  if (m>n1){
     buff[threadIdx.x]  = x[n1];

     __syncthreads();
     for(int j=0; j<(m-n1); j++){
         res += A[0]*buff[j];
         A+=lda;
     }
  }

  if (ind<n)
     y[ind] = alpha * res;
}

extern "C" void
magmablas_zgemvn_fermi(int n, int m, double2 alpha, double2 *A, int lda, double2 *x, double2 *y)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======

    This routine computes Y = alpha A x on the GPU.

    N      - (input) INTEGER.
             On entry, N specifies the number of rows of the matrix A.

    M      - (input) INTEGER.
             On entry, M specifies the number of columns of the matrix A

    A      - (input) SINGLE PRECISION array of dimension ( LDA, m ) on the GPU.
   
    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension m.
     
    Y      - (output) SINGLE PRECISION array of	dimension m. 
             On exit Y = alpha A X.

    ===================================================================== */

    int blocks;
    if (n % num_threads==0)
        blocks = n/num_threads;
    else
        blocks = n/num_threads + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(num_threads, 1, 1);
/*    if(n<=8500) 
		zgemvn_kernel1_fermi<<<grid, threads>>>(n, m, (m / zgemv_bs)*zgemv_bs, 
			                           alpha, A, lda, x, y);
    else */
		zgemvn_kernel2_fermi<<<grid, threads>>>(n, m, (m / num_threads)*num_threads, 
			                           alpha, A, lda, x, y);
}



__global__ void 
zgemvt_kernel1_fermi(int m, int n, double2 alpha, int n1, double2* A, int lda,
              double2 *x, double2 *y)
{
	unsigned int tx = threadIdx.x;

	__shared__ double2 sdata[threadSize];
	

	double2 res;
    MAGMA_Z_SET2REAL(res, 0.0f);
	double2 zero;
    MAGMA_Z_SET2REAL(zero, 0.0f);
     
	for(int i=0; i<n1; i+= threadSize)
	{
		res += A[tx + i + lda * blockIdx.y] * x[tx + i];
	}

	
	if(m > n1)
	{
		if( tx + n1 <  m )
		{
			res  += A[tx + n1 + lda *blockIdx.y] * x[tx + n1];
		}
		else 
		{
			res  += zero;
		}
	}	

    sdata[tx] = res;
	__syncthreads();
    
    /*
	if(tx < 128) 
	{
		sdata[tx] += sdata[tx + 128];
	}
    __syncthreads();
	*/

	if(tx < 64) 
	{
		sdata[tx] += sdata[tx + 64];
	}
    __syncthreads();

	if(tx < 32) 
	{
		sdata[tx] += sdata[tx + 32];
	}

    if(tx == 0)
	{
		for(int i=1;i<32;i++)
		{
			sdata[tx] += sdata[tx + i];
		}
	}

    if( tx == 0 ) 
	{
		y[blockIdx.y] = sdata[0]; 		

		if (blockIdx.y < n)
		{
			y[blockIdx.y] = y[blockIdx.y] * alpha;
		}
	}
}


__global__ void 
zgemvt_kernel2_fermi(int m, int n, double2 alpha,
               int n1, double2* A, int lda, double2 *x, double2 *y)
{
  const int inx = threadIdx.x;
  const int iny = threadIdx.y;

  int ind  = iny + blockIdx.x * 16;
  ind = inx + ind * lda;
  int ind2 = inx + iny * 16;
  if (ind2>31)
     ind2-=32;

  A += ind;
  x += ind2;

  double2 res;
  MAGMA_Z_SET2REAL(res, 0.0f);
  double2 zero;
  MAGMA_Z_SET2REAL(zero, 0.0f);

  __shared__ double2 buff[32];
  __shared__ double2 la[16][17];

  for(int i=0; i<n1; i += 32 ){
     buff[ind2]  = x[i];
     #pragma unroll
     for(int j=0; j<4; j++)
        la[iny + j * 4][inx] = A[j* 4 * lda];

     __syncthreads();
     #pragma unroll
     for(int j=0; j < 4; j++)
       res += la[inx][iny*4+j]*buff[j+iny*4];

     A += 16;

     __syncthreads();
     //===========================================
     #pragma unroll
     for(int j=0; j<4; j++)
         la[iny+ j * 4][inx] = A[j* 4 * lda];

     __syncthreads();

     #pragma unroll
     for(int j=0; j < 4; j++)
        res += la[inx][iny*4+j]*buff[j+16+iny*4];
     A += 16;
  }

  __syncthreads(); // 1
  if (n>n1){
     if (ind2>=(n-n1))
	buff[ind2]=zero;
     else
        buff[ind2]  = x[n1];

     __syncthreads();
     #pragma unroll
     for(int j=0; j<4; j++)
         if (inx>=(n-n1))
            la[iny + j * 4][inx] =  zero;
         else
            la[iny + j * 4][inx] = A[j* 4 * lda];

     __syncthreads();
     if (n-n1>4){
        #pragma unroll
        for(int j=0; j < 4; j++){
           ind =  j+iny*4;
           res += la[inx][ind]*buff[ind];
        }
	A += 16;
        __syncthreads();
	#pragma unroll
	for(int j=0; j<4; j++)
          if (inx+16>=(n-n1))
             la[iny+ j * 4][inx] = zero;
          else
             la[iny+ j * 4][inx] = A[j* 4* lda];

        __syncthreads();

        #pragma unroll
	for(int j=0; j < 4; j++){
           ind = j+4*iny;
           res += la[inx][ind]*buff[16+ind];
        }
     }
     else {
	#pragma unroll
        for(int j=0; j < 4; j++){
          ind = j+iny*4;
          res += la[inx][ind]*buff[ind];
        }
     }
  }

  __syncthreads();
  ind = inx + blockIdx.x * 16;
  la[inx][iny]= res;
  __syncthreads();
  if (ind<n && iny==0){
     res = la[inx][0] + la[inx][1] + la[inx][2] + la[inx][3];
     y[ind] = alpha*res;
  }
}

extern "C" void
magmablas_zgemvt1_fermi(int m, int n, double2 alpha, double2 *A, int lda,
                  double2 *x, double2 *y)
{


    dim3 grid    ( 1,  n,  1);
    dim3 threads ( threadSize,   1,  1);

    zgemvt_kernel1_fermi<<<grid, threads>>>( m, n, alpha, ( m / threadSize) * threadSize,
                                       A, lda, x, y);

									  
}

extern "C" void
magmablas_zgemvt2_fermi(int m, int n, double2 alpha, double2 *A, int lda,
                  double2 *x, double2 *y)
{

    int blocks;

    if (n % 16==0)
        blocks = n/16;
    else
        blocks = n/16 + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(16, 4, 1);

    zgemvt_kernel2_fermi<<<grid, threads>>>(m, n, alpha, (m / 32)*32,
                                      A, lda, x, y);
}

extern "C" void
magmablas_zgemvt_fermi(int m, int n, double2 alpha, double2 *A, int lda, 
                 double2 *x, double2 *y)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======

    This routine computes y = alpha *  A^t *  x on the GPU.

    M      - (input) INTEGER.
             On entry, M specifies the number of rows of the matrix A.

    N      - (input) INTEGER.
             On entry, N specifies the number of columns of the matrix A

    A      - (input) SINGLE PRECISION array of dimension ( LDA, n ) on the GPU.

    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension m.

    Y      - (output) SINGLE PRECISION array of dimension n.
             On exit Y = alpha A^t X.

    ===================================================================== */

    if (n<=128)
      magmablas_zgemvt2_fermi(m, n, alpha, A, lda, x, y);
    else
      magmablas_zgemvt1_fermi(m, n, alpha, A, lda, x, y);
    

}


extern "C" void
magmablas_zgemv_fermi(char flag, int m, int n, double2 alpha, double2 *A, int lda, double2 *x, double2 *y) 
{

	if (flag == 'N' || flag == 'n')
	{
		magmablas_zgemvn_fermi(m,  n, alpha, A, lda, x, y);
	}
	else if(flag == 'T' || flag == 't')
	{
		magmablas_zgemvt_fermi(m,  n, alpha, A, lda, x, y);
	}
	else 
	{
		printf("Not Available\n");
	}
}


#undef num_threads
#undef zgemv_bs
#undef threadSize 
