#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c
*/
#include "common_magma.h"
#include "commonblas_z.h"

#define PRECISION_z

#define num_threads 128
#define gemv_bs      32
#define threadSize  128

__global__ void
zgemvn_kernel1_fermi(
    int m, int n, int n1, magmaDoubleComplex alpha,
    const magmaDoubleComplex * __restrict__ A, int lda,
    const magmaDoubleComplex * __restrict__ x, magmaDoubleComplex beta,
    magmaDoubleComplex       * __restrict__ y)
{
#if (__CUDA_ARCH__ >= 200)
    int ind = blockIdx.x*num_threads + threadIdx.x;
    
    A += ind;
    
    magmaDoubleComplex res = MAGMA_Z_ZERO;
    
    for( int i=0; i < n1; i += gemv_bs ) {
        #pragma unroll
        for(int j=0; j < gemv_bs; j++) {
            res += A[0] * x[j];
            A   += lda;
        }
        x += gemv_bs;
    }
    
    if ( n > n1 ) {
        for(int j=0; j < (n-n1); j++) {
            res += A[0] * x[j];
            A   += lda;
        }
    }
    
    if ( ind < m )
        y[ind] = alpha * res + beta * y[ind];
#endif /* (__CUDA_ARCH__ >= 200) */
}

__global__ void
zgemvn_kernel2_fermi(
    int m, int n, int n1, magmaDoubleComplex alpha,
    const magmaDoubleComplex * __restrict__ A, int lda,
    const magmaDoubleComplex * __restrict__ x, magmaDoubleComplex beta,
    magmaDoubleComplex       * __restrict__ y)
{
#if (__CUDA_ARCH__ >= 200)
    int ind = blockIdx.x*num_threads + threadIdx.x;
    
    A += ind;
    x += threadIdx.x;
    
    magmaDoubleComplex res = MAGMA_Z_ZERO;
    
    __shared__ magmaDoubleComplex buff[num_threads];
    for( int i=0; i < n1; i += num_threads ) {
        __syncthreads();
        buff[threadIdx.x] = x[i];
        
        __syncthreads();
        #pragma unroll
        for(int j=0; j < num_threads; j++) {
            res += A[0]*buff[j];
            A += lda;
        }
    }
    __syncthreads();
    
    if ( n > n1 ) {
        buff[threadIdx.x] = x[n1];
        
        __syncthreads();
        for(int j=0; j<(n-n1); j++) {
            res += A[0]*buff[j];
            A += lda;
        }
    }
    
    if ( ind < m )
        y[ind] = alpha * res + beta * y[ind];
#endif /* (__CUDA_ARCH__ >= 200) */
}

/**
    Purpose
    -------

    This routine computes Y = alpha A x + beta y, on the GPU.

    @param[in]
    m       INTEGER.
            On entry, M specifies the number of rows of the matrix A.

    @param[in]
    n       INTEGER.
            On entry, N specifies the number of columns of the matrix A

    @param[in]
    alpha   COMPLEX*16.
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    A       COMPLEX*16 array of dimension ( LDA, n ) on the GPU.
   
    @param[in]
    lda     INTEGER.
            LDA specifies the leading dimension of A.

    @param[in]
    x       COMPLEX*16 array of dimension n.

    @param[in]
    beta    DOUBLE PRECISION.
            On entry, BETA specifies the scalar beta.

    @param[out]
    y       COMPLEX*16 array of dimension n.
            On exit Y = alpha A X + beta Y.

    @ingroup magma_zblas2
    ********************************************************************/
extern "C" void
magmablas_zgemvn_fermi(
    magma_int_t m, magma_int_t n, magmaDoubleComplex alpha,
    const magmaDoubleComplex *A, magma_int_t lda,
    const magmaDoubleComplex *x, magmaDoubleComplex beta,
    magmaDoubleComplex       *y)
{
    magma_int_t blocks = (m - 1)/num_threads + 1;
    dim3 grid(blocks, 1, 1);
    dim3 threads(num_threads, 1, 1);
    /*
    if ( m <= 8500 )
        zgemvn_kernel1_fermi<<< grid, threads, 0, magma_stream >>>
            (m, n, (n / gemv_bs)*gemv_bs, alpha, A, lda, x, y);
    else
    */
        zgemvn_kernel2_fermi<<< grid, threads, 0, magma_stream >>>
            (m, n, (n / num_threads)*num_threads, alpha, A, lda, x, beta, y);
}

__global__ void
zgemvt_kernel_fermi(
    int m, int n, magmaDoubleComplex alpha, int n1,
    const magmaDoubleComplex * __restrict__ A, int lda,
    const magmaDoubleComplex * __restrict__ x, magmaDoubleComplex beta,
    magmaDoubleComplex       * __restrict__ y)
{
#if (__CUDA_ARCH__ >= 200)
    unsigned int tx = threadIdx.x;

    __shared__ magmaDoubleComplex sdata[threadSize];

    magmaDoubleComplex res = MAGMA_Z_ZERO;
    magmaDoubleComplex c_zero = MAGMA_Z_ZERO;
 
    for(int i=0; i < n1; i += threadSize) {
        res += A[tx + i + lda * blockIdx.y] * x[tx + i];
    }

    if ( m > n1 ) {
        if ( tx + n1 < m ) {
            res += A[tx + n1 + lda*blockIdx.y] * x[tx + n1];
        }
        else {
            res += c_zero;
        }
    }

    sdata[tx] = res;
    __syncthreads();

    for(int s=blockDim.x/2; s > 32; s /= 2) {
        if ( tx < s ) {
            sdata[tx] += sdata[tx+s];
        }
        __syncthreads();
    }

    if ( tx < 32 ) {
        sdata[tx] += sdata[tx + 32];
    }

    if ( tx == 0 ) {
        for(int i=1; i < 32; i++) {
            sdata[tx] += sdata[tx + i];
        }
    }

    if ( tx == 0 ) {
        if ( blockIdx.y < n ) {
            y[blockIdx.y] = sdata[0] * alpha + beta * y[blockIdx.y];
        }
    }
#endif /* (__CUDA_ARCH__ >= 200) */
}


/**
    Purpose
    -------

    This routine computes y = alpha * A^T * x + beta*y, on the GPU.

    @param[in]
    m       INTEGER.
            On entry, M specifies the number of rows of the matrix A.

    @param[in]
    n       INTEGER.
            On entry, N specifies the number of columns of the matrix A

    @param[in]
    alpha   COMPLEX*16.
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    A       COMPLEX*16 array of dimension ( LDA, n ) on the GPU.

    @param[in]
    lda     INTEGER.
            LDA specifies the leading dimension of A.

    @param[in]
    x       COMPLEX*16 array of dimension m.

    @param[in]
    beta    COMPLEX*16.
            On entry, BETA specifies the scalar beta.

    @param[out]
    y       COMPLEX*16 array of dimension n.
            On exit Y = alpha A^T X + beta Y.

    @ingroup magma_zblas2
    ********************************************************************/
extern "C" void
magmablas_zgemvt_fermi(
    magma_int_t m, magma_int_t n, magmaDoubleComplex alpha,
    const magmaDoubleComplex *A, magma_int_t lda,
    const magmaDoubleComplex *x, magmaDoubleComplex beta,
    magmaDoubleComplex       *y)
{
    dim3 grid    ( 1, n, 1 );
    dim3 threads ( threadSize, 1, 1 );
    zgemvt_kernel_fermi<<< grid, threads, 0, magma_stream >>>
        (m, n, alpha, (m / threadSize) * threadSize, A, lda, x, beta, y );
}

__global__ void
zgemvc_kernel_fermi(
    int m, int n, magmaDoubleComplex alpha, int n1,
    const magmaDoubleComplex * __restrict__ A, int lda,
    const magmaDoubleComplex * __restrict__ x, magmaDoubleComplex beta,
    magmaDoubleComplex       * __restrict__ y)
{
#if (__CUDA_ARCH__ >= 200)
    unsigned int tx = threadIdx.x;

    __shared__ magmaDoubleComplex sdata[threadSize];

    magmaDoubleComplex res = MAGMA_Z_ZERO;
    magmaDoubleComplex c_zero = MAGMA_Z_ZERO;
 
    for(int i=0; i < n1; i += threadSize) {
        res += hipConj(A[tx + i + lda * blockIdx.y]) * x[tx + i];
    }

    if ( m > n1 ) {
        if ( tx + n1 < m ) {
            res += hipConj(A[tx + n1 + lda*blockIdx.y]) * x[tx + n1];
        }
        else {
            res += c_zero;
        }
    }

    sdata[tx] = res;
    __syncthreads();

    /*
    if ( tx < 128 ) {
        sdata[tx] += sdata[tx + 128];
    }
    __syncthreads();
    */

    if ( tx < 64 ) {
        sdata[tx] += sdata[tx + 64];
    }
    __syncthreads();

    if ( tx < 32 ) {
        sdata[tx] += sdata[tx + 32];
    }

    if ( tx == 0 ) {
        for(int i=1; i < 32; i++) {
            sdata[tx] += sdata[tx + i];
        }
    }

    if ( tx == 0 ) {
        if ( blockIdx.y < n ) {
            y[blockIdx.y] = sdata[0] * alpha + beta * y[blockIdx.y];
        }
    }
#endif /* (__CUDA_ARCH__ >= 200) */
}


/**
    Purpose
    -------

    This routine computes y = alpha * A^H * x + beta*y, on the GPU.

    @param[in]
    m       INTEGER.
            On entry, M specifies the number of rows of the matrix A.

    @param[in]
    n       INTEGER.
            On entry, N specifies the number of columns of the matrix A

    @param[in]
    alpha   COMPLEX*16.
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    A       COMPLEX*16 array of dimension ( LDA, n ) on the GPU.

    @param[in]
    lda     INTEGER.
            LDA specifies the leading dimension of A.

    @param[in]
    x       COMPLEX*16 array of dimension m.

    @param[in]
    beta    COMPLEX*16.
            On entry, BETA specifies the scalar beta.

    @param[out]
    y       COMPLEX*16 array of dimension n.
            On exit Y = alpha A^H X + beta y.

    @ingroup magma_zblas2
    ********************************************************************/
extern "C" void
magmablas_zgemvc_fermi(
    magma_int_t m, magma_int_t n, magmaDoubleComplex alpha,
    const magmaDoubleComplex *A, magma_int_t lda,
    const magmaDoubleComplex *x,
    magmaDoubleComplex beta,
    magmaDoubleComplex *y)
{
    dim3 grid    ( 1, n, 1 );
    dim3 threads ( threadSize, 1, 1 );
    zgemvc_kernel_fermi<<< grid, threads, 0, magma_stream >>>
        (m, n, alpha, (m / threadSize) * threadSize, A, lda, x, beta, y);
}

extern "C" void
magmablas_zgemv(
    magma_trans_t trans, magma_int_t m, magma_int_t n, magmaDoubleComplex alpha,
    const magmaDoubleComplex *A, magma_int_t lda,
    const magmaDoubleComplex *x, magma_int_t incx,
    magmaDoubleComplex beta,
    magmaDoubleComplex *y, magma_int_t incy)
{
    magma_int_t arch = magma_getdevice_arch();
    if ( arch < 200  ) {
        // --------------------
        // call CUDA ARCH 1.x version
        // magmablas for [sd] precisions, cublas for [zc] precisions.
        #if defined(PRECISION_z) || defined(PRECISION_c)
        magma_zgemv( trans, m, n, alpha, A, lda, x, incx, beta, y, incy );
        #else
        magmablas_zgemv_tesla( trans, m, n, alpha, A, lda, x, incx, beta, y, incy );
        #endif
        return;
    }
    
    // --------------------
    // CUDA ARCH 2.x (Fermi) version
    if ( incx == 1 && incy == 1 ) {
        if ( trans == MagmaNoTrans ) {
            if ( m < 7000 ) {
                magma_zgemv( trans, m, n, alpha, A, lda, x, incx, beta, y, incy);
            }
            else {
                magmablas_zgemvn_fermi(m, n, alpha, A, lda, x, beta, y);
            }
        }
        else if ( trans == MagmaTrans ) {
            magmablas_zgemvt_fermi(m, n, alpha, A, lda, x, beta, y);
        }
        else if ( trans == MagmaConjTrans ) {
            magmablas_zgemvc_fermi(m, n, alpha, A, lda, x, beta, y);
        }
        else {
            fprintf( stderr, "trans = %c is invalid\n", trans );
        }
    }
    else {
        magma_zgemv( trans, m, n, alpha, A, lda, x, incx, beta, y, incy);
    }
}

#undef num_threads
#undef gemv_bs
#undef threadSize
