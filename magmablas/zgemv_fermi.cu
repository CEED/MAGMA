#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date
       
       @author Mark Gates
       @author Tingxing Dong
       @author Azzam Haidar

       @precisions normal z -> s d c
*/
#include "magma_internal.h"
#include "commonblas_z.h"
#include "magma_templates.h"

#define PRECISION_z

#include "gemv_template_device.cuh"

#include "gemv_config/gemvn_param.h"
#include "gemv_config/gemvt_param.h"

#define version(s,v) s ## _V_ ## v


/******************************************************************************/
// NoTrans kernel
template<const int DIM_X, const int DIM_Y, const int TILE_SIZE>
__global__ void
zgemvn_template_kernel_fermi(
    int m, int n, magmaDoubleComplex alpha,
    const magmaDoubleComplex * __restrict__ A, int lda,
    const magmaDoubleComplex * __restrict__ x, int incx, magmaDoubleComplex beta,
    magmaDoubleComplex       * __restrict__ y, int incy)
{
#if (__CUDA_ARCH__ >= 200)
    gemvn_template_device<magmaDoubleComplex, DIM_X, DIM_Y, TILE_SIZE>
        (m, n, alpha, A, lda, x, incx, beta, y, incy);
#endif /* (__CUDA_ARCH__ >= 200) */
}


/******************************************************************************/
// Trans/ConjTans kernel
template<const int DIM_X, const int DIM_Y, const int TILE_SIZE, magma_trans_t trans>
__global__ void
zgemvc_template_kernel_fermi(
    int m, int n, magmaDoubleComplex alpha,
    const magmaDoubleComplex * __restrict__ A, int lda,
    const magmaDoubleComplex * __restrict__ x, int incx, magmaDoubleComplex beta,
    magmaDoubleComplex       * __restrict__ y, int incy)
{
#if (__CUDA_ARCH__ >= 200)
    gemvc_template_device< magmaDoubleComplex, DIM_X, DIM_Y, TILE_SIZE, trans >
        (m, n, alpha, A, lda, x, incx, beta, y, incy);
#endif /* (__CUDA_ARCH__ >= 200) */
}


/******************************************************************************/
// NoTrans CPU driver
template<const int DIM_X, const int DIM_Y, const int TILE_SIZE>
void
zgemvn_template_fermi(
    magma_int_t m, magma_int_t n, magmaDoubleComplex alpha,
    const magmaDoubleComplex * __restrict__ A, magma_int_t lda,
    const magmaDoubleComplex * __restrict__ x, magma_int_t incx, magmaDoubleComplex beta,
    magmaDoubleComplex       * __restrict__ y, magma_int_t incy,
    magma_queue_t queue)
{
    dim3 grid( magma_ceildiv(m, TILE_SIZE), 1 );
    dim3 threads( DIM_X, DIM_Y );

    zgemvn_template_kernel_fermi<DIM_X, DIM_Y, TILE_SIZE>
        <<< grid, threads, 0, queue->hip_stream() >>>
        (m, n, alpha, A, lda, x, incx, beta, y, incy);
}


/******************************************************************************/
// Trans/ConjTans CPU driver
template<const int DIM_X, const int DIM_Y, const int TILE_SIZE>
void
zgemvc_template_fermi(
    magma_trans_t trans, magma_int_t m, magma_int_t n, magmaDoubleComplex alpha,
    const magmaDoubleComplex * __restrict__ A, magma_int_t lda,
    const magmaDoubleComplex * __restrict__ x, magma_int_t incx, magmaDoubleComplex beta,
    magmaDoubleComplex       * __restrict__ y, magma_int_t incy,
    magma_queue_t queue)
{
    dim3 grid    ( magma_ceildiv(n, TILE_SIZE), 1 );
    dim3 threads ( DIM_X, DIM_Y );
    
    if (trans == MagmaConjTrans) {
        zgemvc_template_kernel_fermi< DIM_X, DIM_Y, TILE_SIZE, MagmaConjTrans >
            <<< grid, threads, 0, queue->hip_stream() >>>
            (m, n, alpha, A, lda, x, incx, beta, y, incy);
    }
    else {
        zgemvc_template_kernel_fermi< DIM_X, DIM_Y, TILE_SIZE, MagmaTrans >
            <<< grid, threads, 0, queue->hip_stream() >>>
            (m, n, alpha, A, lda, x, incx, beta, y, incy);
    }
}


/***************************************************************************//**
    Purpose
    -------
    ZGEMV performs one of the matrix-vector operations
    
        y := alpha*A*x    + beta*y,   or
        y := alpha*A**T*x + beta*y,   or
        y := alpha*A**H*x + beta*y,
    
    where alpha and beta are scalars, x and y are vectors and A is an
    m by n matrix.

    Arguments
    ----------
    @param[in]
    trans   magma_trans_t
            On entry, TRANS specifies the operation to be performed as
            follows:
      -     = MagmaNoTrans:    y := alpha*A  *x + beta*y
      -     = MagmaTrans:      y := alpha*A^T*x + beta*y
      -     = MagmaConjTrans:  y := alpha*A^H*x + beta*y

    @param[in]
    m       INTEGER
            On entry, m specifies the number of rows of the matrix A.

    @param[in]
    n       INTEGER
            On entry, n specifies the number of columns of the matrix A
 
    @param[in]
    alpha   COMPLEX_16
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    dA      COMPLEX_16 array of dimension ( LDDA, n ) on the GPU.
   
    @param[in]
    ldda    INTEGER
            LDDA specifies the leading dimension of A.

    @param[in]
    dx      COMPLEX_16 array of dimension
            n if trans == MagmaNoTrans
            m if trans == MagmaTrans or MagmaConjTrans
     
    @param[in]
    incx    Specifies the increment for the elements of X.
            INCX must not be zero.
  
    @param[in]
    beta    COMPLEX_16
            On entry, BETA specifies the scalar beta. When BETA is
            supplied as zero then Y need not be set on input.

    @param[out]
    dy      COMPLEX_16 array of dimension
            m if trans == MagmaNoTrans
            n if trans == MagmaTrans or MagmaConjTrans

    @param[in]
    incy    Specifies the increment for the elements of Y.
            INCY must not be zero.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_gemv
*******************************************************************************/
extern "C" void
magmablas_zgemv(
    magma_trans_t trans, magma_int_t m, magma_int_t n, 
    magmaDoubleComplex alpha,
    magmaDoubleComplex_const_ptr dA, magma_int_t ldda,
    magmaDoubleComplex_const_ptr dx, magma_int_t incx,
    magmaDoubleComplex beta,
    magmaDoubleComplex_ptr dy, magma_int_t incy, 
    magma_queue_t queue)
{
    magma_int_t info = 0;
    if ( trans != MagmaNoTrans && trans != MagmaTrans && trans != MagmaConjTrans )
        info = -1;
    else if ( m < 0 )
        info = -2;
    else if ( n < 0 )
        info = -3;
    else if ( ldda < m )
        info = -6;
    else if ( incx == 0 )
        info = -8;
    else if ( incy == 0 )
        info = -11;
    
    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }

    // --------------------
    // CUDA ARCH 2.x (Fermi) version
    if ( trans == MagmaNoTrans ) {
        if (m <= 256) {
            zgemvn_template_fermi<version(N, 137)>
                ( m, n, alpha, dA, ldda, dx, incx, beta, dy, incy, queue );
        }
        else {
            zgemvn_template_fermi<version(N, 140)>
                ( m, n, alpha, dA, ldda, dx, incx, beta, dy, incy, queue );
        }
    }
    else {
        zgemvc_template_fermi<version(T, 189)>
            ( trans, m, n, alpha, dA, ldda, dx, incx, beta, dy, incy, queue );
    }
}
