#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal z -> s d c
       @author Ichitaro Yamazaki
*/
#include "common_magma.h"
#define PRECISION_z
#include "commonblas.h"

//
//    m, n - dimensions in the output (ha) matrix.
//             This routine copies the dat matrix from the GPU
//             to ha on the CPU. In addition, the output matrix
//             is transposed. The routine uses a buffer of size
//             2*lddb*nb pointed to by dB (lddb > m) on the GPU. 
//             Note that lda >= m and lddat >= n.
//
extern "C" void 
magmablas_zgetmatrix_transpose_mgpu(
                  magma_int_t ngpus, magma_queue_t stream[][2],
                  magmaDoubleComplex **dat, magma_int_t ldda,
                  magmaDoubleComplex   *ha, magma_int_t lda,
                  magmaDoubleComplex  **db, magma_int_t lddb,
                  magma_int_t m, magma_int_t n, magma_int_t nb)
{
#define   A(j)     (ha  + (j)*lda)
#define  dB(d, j)  (db[(d)]  + (j)*nb*lddb)
#define  dAT(d, j) (dat[(d)] + (j)*nb)
    int nstreams = 2, j, j_local, d, id, ib;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || ngpus*ldda < n || lddb < m){
        printf( "Wrong arguments in magmablas_zgetmatrix_transpose_mgpu (%d<%d), (%d*%d<%d), or (%d<%d).\n",
                lda, m, ngpus, ldda, n, lddb, m );
        return;
    }
    
    /* Move data from GPU to CPU using two buffers; first transpose the data on the GPU */
    for(j=0; j<n; j+=nb){
       d       = (j/nb)%ngpus;
       j_local = (j/nb)/ngpus;
       id      = j_local%nstreams;
       magma_setdevice(d);

       ib = min(n-j, nb);
       magmablasSetKernelStream(stream[d][id]);
       magmablas_ztranspose2(  dB(d, id),      lddb, 
                              dAT(d, j_local), ldda, 
                              ib, m);
       magma_zgetmatrix_async( m, ib,
                               dB(d, id), lddb,
                               A(j),      lda, 
                               stream[d][id] );
    }
}



