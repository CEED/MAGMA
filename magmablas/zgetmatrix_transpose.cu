#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal z -> s d c

*/
#include "common_magma.h"
#define PRECISION_z
#include "commonblas.h"

extern "C" void
magmablas_ztranspose2s(hipDoubleComplex *odata, int ldo,
                       hipDoubleComplex *idata, int ldi,
                       int m, int n, hipStream_t *stream );


//
//      m, n - dimensions in the output (ha) matrix.
//             This routine copies the dat matrix from the GPU
//             to ha on the CPU. In addition, the output matrix
//             is transposed. The routine uses a buffer of size
//             2*lddb*nb pointed to by dB (lddb > m) on the GPU. 
//             Note that lda >= m and lddat >= n.
//
extern "C" void 
magmablas_zgetmatrix_transpose( int m, int n,
                                hipDoubleComplex *dat, int ldda,
                                hipDoubleComplex  *ha, int lda,
                                hipDoubleComplex  *dB, int lddb, int nb )
{
    int i = 0, j = 0, ib;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || ldda < n || lddb < m){
        printf("Wrong arguments in zdtoht.\n");
        return;
    }

    static hipStream_t stream[2];
    hipStreamCreate(&stream[0]);
    hipStreamCreate(&stream[1]);

    for(i=0; i<n; i+=nb){
       /* Move data from GPU to CPU using 2 buffers; 1st transpose the data on the GPU */
       ib   = min(n-i, nb);

       //magmablas_ztranspose2 ( dB + (j%2)*nb*lddb, lddb, dat+i, ldda, ib, m);
       magmablas_ztranspose2s( dB + (j%2)*nb*lddb, lddb, dat+i, ldda, ib, m, &stream[j%2]);
       magma_zgetmatrix_async( m, ib,
                               dB + (j%2) * nb * lddb, lddb,
                               ha+i*lda,               lda, stream[j%2] );
       j++;
    }

    hipStreamDestroy( stream[0] );
    hipStreamDestroy( stream[1] );
}

//===========================================================================
//  This version is similar to the above but for multiGPUs. The distribution
//  is 1D block cyclic. The input arrays are pointers for the corresponding
//  GPUs. The streams are passed as argument, in contrast to the single GPU
//  routine.
//===========================================================================
extern "C" void
magmablas_zgetmatrix_transpose2( int m, int n,
                                 hipDoubleComplex **dat, int *ldda,
                                 hipDoubleComplex  *ha,  int  lda,
                                 hipDoubleComplex **dB,  int  lddb, int nb,
                                 int num_gpus, hipStream_t stream[][2] )
{
    int i = 0, j[4] = {0, 0, 0, 0}, ib, k;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || lddb < m){
        printf("Wrong arguments in zdtoht.\n");
        return;
    }

    for(i=0; i<n; i+=nb){
       /* Move data from GPU to CPU using 2 buffers; 1st transpose the data on the GPU */
       k = (i/nb)%num_gpus;
       ib   = min(n-i, nb);
       hipSetDevice(k);

       //hipStreamSynchronize(stream[k][j[k]%2]);
       //magmablas_ztranspose2( dB[k] + (j[k]%2)*nb*lddb, lddb, 
       //                       dat[k]+i/(nb*num_gpus)*nb, ldda[k], ib, m);
       magmablas_ztranspose2s(dB[k] + (j[k]%2)*nb*lddb, lddb,
                              dat[k]+i/(nb*num_gpus)*nb, ldda[k], 
                              ib, m, &stream[k][j[k]%2]);
       magma_zgetmatrix_async( m, ib,
                               dB[k] + (j[k]%2) * nb * lddb, lddb,
                               ha+i*lda,                     lda, stream[k][j[k]%2] );
       j[k]++;
    }
}

