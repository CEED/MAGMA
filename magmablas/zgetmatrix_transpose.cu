#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal z -> s d c

*/
#include "common_magma.h"
#define PRECISION_z
#include "commonblas.h"

extern "C" void
magmablas_ztranspose2s(hipDoubleComplex *odata, magma_int_t ldo,
                       hipDoubleComplex *idata, magma_int_t ldi,
                       magma_int_t m, magma_int_t n, hipStream_t *stream );


//
//      m, n - dimensions in the output (ha) matrix.
//             This routine copies the dat matrix from the GPU
//             to ha on the CPU. In addition, the output matrix
//             is transposed. The routine uses a buffer of size
//             2*lddb*nb pointed to by dB (lddb > m) on the GPU. 
//             Note that lda >= m and lddat >= n.
//
extern "C" void 
magmablas_zgetmatrix_transpose( magma_int_t m, magma_int_t n,
                                hipDoubleComplex *dat, magma_int_t ldda,
                                hipDoubleComplex  *ha, magma_int_t lda,
                                hipDoubleComplex  *dB, magma_int_t lddb, magma_int_t nb )
{
    magma_int_t i = 0, j = 0, ib;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || ldda < n || lddb < m){
        printf("Wrong arguments in zdtoht.\n");
        return;
    }

    hipStream_t stream[2];
    magma_queue_create( &stream[0] );
    magma_queue_create( &stream[1] );

    for(i=0; i<n; i+=nb){
       /* Move data from GPU to CPU using 2 buffers; 1st transpose the data on the GPU */
       ib   = min(n-i, nb);

       //magmablas_ztranspose2 ( dB + (j%2)*nb*lddb, lddb, dat+i, ldda, ib, m);
       magmablas_ztranspose2s( dB + (j%2)*nb*lddb, lddb, dat+i, ldda, ib, m, &stream[j%2]);
       magma_zgetmatrix_async( m, ib,
                               dB + (j%2) * nb * lddb, lddb,
                               ha+i*lda,               lda, stream[j%2] );
       j++;
    }

    magma_queue_destroy( stream[0] );
    magma_queue_destroy( stream[1] );
}

//===========================================================================
//  This version is similar to the above but for multiGPUs. The distribution
//  is 1D block cyclic. The input arrays are pointers for the corresponding
//  GPUs. The streams are passed as argument, in contrast to the single GPU
//  routine.
//  NOTE: see magmablas_zgetmatrix_transpose_mgpu.
//===========================================================================
extern "C" void
magmablas_zgetmatrix_transpose2( magma_int_t m, magma_int_t n,
                                 hipDoubleComplex **dat, magma_int_t *ldda,
                                 hipDoubleComplex  *ha,  magma_int_t  lda,
                                 hipDoubleComplex **dB,  magma_int_t  lddb, magma_int_t nb,
                                 magma_int_t num_gpus, hipStream_t stream[][2] )
{
    magma_int_t i = 0, j[4] = {0, 0, 0, 0}, ib, k;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || lddb < m){
        printf("Wrong arguments in zdtoht.\n");
        return;
    }

    for(i=0; i<n; i+=nb){
       /* Move data from GPU to CPU using 2 buffers; 1st transpose the data on the GPU */
       k = (i/nb)%num_gpus;
       ib   = min(n-i, nb);
       magma_setdevice(k);

       //magma_queue_sync( stream[k][j[k]%2] );
       //magmablas_ztranspose2( dB[k] + (j[k]%2)*nb*lddb, lddb, 
       //                       dat[k]+i/(nb*num_gpus)*nb, ldda[k], ib, m);
       magmablas_ztranspose2s(dB[k] + (j[k]%2)*nb*lddb, lddb,
                              dat[k]+i/(nb*num_gpus)*nb, ldda[k], 
                              ib, m, &stream[k][j[k]%2]);
       magma_zgetmatrix_async( m, ib,
                               dB[k] + (j[k]%2) * nb * lddb, lddb,
                               ha+i*lda,                     lda, stream[k][j[k]%2] );
       j[k]++;
    }
}

