#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c
       @author Mark Gates
*/
#include "common_magma.h"

#define NB 64

/* =====================================================================
    Batches zlacpy of multiple arrays;
    y-dimension of grid is different arrays,
    x-dimension of grid is blocks for each array.
    Matrix is m x n, and is divided into block rows, each NB x n.
    Each CUDA block has NB threads to handle one block row.
    Each thread adds one row, iterating across all columns.
    The bottom block of rows may be partially outside the matrix;
    if so, rows outside the matrix (i >= m) are disabled.
    
    TODO. Block in both directions, for large matrices.
    E.g., each block does 64x64 tile, instead of 64xN tile.
*/
__global__ void
zgeadd_batched_kernel(
    int m, int n,
    magmaDoubleComplex alpha,
    const magmaDoubleComplex * const *dAarray, int ldda,
    magmaDoubleComplex              **dBarray, int lddb )
{
    // dA and dB iterate across row i
    const magmaDoubleComplex *dA = dAarray[ blockIdx.y ];
    magmaDoubleComplex       *dB = dBarray[ blockIdx.y ];
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if ( i < m ) {
        dA += i;
        dB += i;
        const magmaDoubleComplex *dAend = dA + n*ldda;
        while( dA < dAend ) {
            *dB = alpha*(*dA) + (*dB);
            dA += ldda;
            dB += lddb;
        }
    }
}


/* ===================================================================== */
/**
    Purpose
    -------
    ZGEADD adds two sets of matrices, dAarray[i] = alpha*dAarray[i] + dBarray[i],
    for i = 0, ..., batchCount-1.
    
    Arguments
    ---------
    
    @param[in]
    m       INTEGER
            The number of rows of each matrix dAarray[i].  M >= 0.
    
    @param[in]
    n       INTEGER
            The number of columns of each matrix dAarray[i].  N >= 0.
    
    @param[in]
    alpha   COMPLEX_16
            The scalar alpha.
            
    @param[in]
    dAarray array on GPU, dimension(batchCount), of pointers to arrays,
            with each array a COMPLEX_16 array, dimension (LDDA,N)
            The m by n matrices dAarray[i].
    
    @param[in]
    ldda    INTEGER
            The leading dimension of each array dAarray[i].  LDDA >= max(1,M).
            
    @param[in,out]
    dBarray array on GPU, dimension(batchCount), of pointers to arrays,
            with each array a COMPLEX_16 array, dimension (LDDB,N)
            The m by n matrices dBarray[i].
    
    @param[in]
    lddb    INTEGER
            The leading dimension of each array dBarray[i].  LDDB >= max(1,M).
    
    @param[in]
    batchCount INTEGER
            The number of matrices to add; length of dAarray and dBarray.
            batchCount >= 0.
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_zaux2
    ********************************************************************/
extern "C" void
magmablas_zgeadd_batched_q(
    magma_int_t m, magma_int_t n,
    magmaDoubleComplex alpha,
    magmaDoubleComplex_const_ptr  const dAarray[], magma_int_t ldda,
    magmaDoubleComplex_ptr              dBarray[], magma_int_t lddb,
    magma_int_t batchCount,
    magma_queue_t queue )
{
    magma_int_t info = 0;
    if ( m < 0 )
        info = -1;
    else if ( n < 0 )
        info = -2;
    else if ( ldda < max(1,m))
        info = -5;
    else if ( lddb < max(1,m))
        info = -7;
    else if ( batchCount < 0 )
        info = -8;
    
    if ( info != 0 ) {
        magma_xerbla( __func__, -(info) );
        return;
    }
    
    if ( m == 0 || n == 0 || batchCount == 0 )
        return;
    
    dim3 threads( NB );
    dim3 grid( (m + NB - 1)/NB, batchCount );
        
    zgeadd_batched_kernel<<< grid, threads, 0, queue >>>(
        m, n, alpha, dAarray, ldda, dBarray, lddb );
}


/**
    @see magmablas_zgeadd_batched_q
    @ingroup magma_zaux2
    ********************************************************************/
extern "C" void
magmablas_zgeadd_batched(
    magma_int_t m, magma_int_t n,
    magmaDoubleComplex alpha,
    magmaDoubleComplex_const_ptr  const dAarray[], magma_int_t ldda,
    magmaDoubleComplex_ptr              dBarray[], magma_int_t lddb,
    magma_int_t batchCount )
{
    magmablas_zgeadd_batched_q(
        m, n, alpha, dAarray, ldda, dBarray, lddb, batchCount, magma_stream );
}
