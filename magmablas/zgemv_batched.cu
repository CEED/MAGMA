#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c
*/

#include <stdio.h>
#include "common_magma.h"


#define zgemv_bs 32

extern __shared__ magmaDoubleComplex shared_data[];


__global__ void
kernel_zgemvn_batched(
    int m, int n, magmaDoubleComplex alpha,
    magmaDoubleComplex **A_array, int lda,
    magmaDoubleComplex **x_array, int incx,
    magmaDoubleComplex beta, magmaDoubleComplex  **y_array, int incy)
{

    magmaDoubleComplex *A = A_array[blockIdx.x];
    magmaDoubleComplex *x = x_array[blockIdx.x];
    magmaDoubleComplex *y = y_array[blockIdx.x];

    int tx = threadIdx.x;

    magmaDoubleComplex res = MAGMA_Z_ZERO;

    magmaDoubleComplex *buff = (magmaDoubleComplex*)shared_data;

    if(tx < n)
    {
        buff[tx] = x[tx*incx];
    }
    __syncthreads();
   
    
    if(tx < m )
    {
        for(int j=0; j < n ; j++)
        {
            res += A[tx]*buff[j];
            A += lda;
        }
  
        y[tx*incy] = alpha * res + y[tx*incy] * beta;
    }

}

/*
    Matrix Non-transpose Vector Multiplication
    y := alpha*A*x + beta*y,
*/
extern "C"
void magmablas_zgemvn_batched(
    int m, int n, 
    magmaDoubleComplex alpha, magmaDoubleComplex **A_array, int lda, 
    magmaDoubleComplex **x_array,  int incx,
    magmaDoubleComplex beta, magmaDoubleComplex **y_array,  int incy, 
    int batchCount)
{

    if( m > 512 || n > 512)
    {
        fprintf( stderr, "m=%d, n=%d, zgemv_batched nontranspose assume row && column lower than %d. Plz call magmablas_zgemv instead", m, n, 512);
        return ;
    }

    dim3 grid(batchCount, 1, 1);
    dim3 threads(max(m,n), 1, 1);
   
    kernel_zgemvn_batched<<< grid, threads, n * sizeof(magmaDoubleComplex) >>>( m, n, alpha,  A_array, lda, x_array, incx,  
                                                                         beta, y_array, incy);
}



__global__ void
kernel_zgemvt_batched(
    int m, int n, int m1, magmaDoubleComplex alpha,
    magmaDoubleComplex **A_array, int lda,
    magmaDoubleComplex **x_array, int incx,
    magmaDoubleComplex beta, magmaDoubleComplex  **y_array, int incy)
{
  

    magmaDoubleComplex *A_ptr = A_array[blockIdx.x];
    magmaDoubleComplex *x_ptr = x_array[blockIdx.x];
    magmaDoubleComplex *y_ptr = y_array[blockIdx.x];

    int tx = threadIdx.x;
    
    magmaDoubleComplex res = MAGMA_Z_ZERO;

    if(tx<m)
    {  
        A_ptr += lda * blockIdx.y + tx;
        x_ptr += tx * incx;
    }
        
    __shared__ magmaDoubleComplex sdata[zgemv_bs];

    for(int i=0; i<m1; i+= zgemv_bs)
    {
        res += A_ptr[i] * x_ptr[i*incx];
    }

    if(m > m1)
    {
        if( tx + m1 <  m )
        {
            res  += A_ptr[m1] * x_ptr[m1*incx];
        }
        else
        {
            res  = res;
        }
    }

    sdata[tx] = res;
    __syncthreads();

    for(int s=blockDim.x/2; s>32;s>>=1)
    {
        if(tx<s)
        {
            sdata[tx] += sdata[tx+s];
        } 
        __syncthreads();
    }

    if(zgemv_bs > 32)
    {  
        if(tx<32)
        {
            sdata[tx] += sdata[tx+32];
        }
    }

    if(tx == 0)
    {
        for(int i=1;i<32;i++)
        {
            sdata[tx] += sdata[tx + i];
        }
        
        y_ptr[blockIdx.y * incy] = sdata[0] * alpha + beta * y_ptr[blockIdx.y*incy];
               
    }
}

/*
    Matrix Transpose Vector Multiplication
    y := alpha*A'*x + beta*y,
*/

extern "C"
void magmablas_zgemvt_batched(
    int m, int n, 
    magmaDoubleComplex alpha, magmaDoubleComplex **A_array, int lda, 
    magmaDoubleComplex **x_array,  int incx,
    magmaDoubleComplex beta, magmaDoubleComplex **y_array,  int incy, 
    int batchCount)
{

    dim3 grid(batchCount, n, 1);
    dim3 threads(zgemv_bs, 1, 1);

    int m1 = (m / zgemv_bs) * zgemv_bs;

    kernel_zgemvt_batched <<< grid, threads >>>(m, n, m1, alpha,  A_array, lda, x_array, incx, beta, y_array, incy);

}
   

#if defined(PRECISION_z) || defined (PRECISION_c)


__global__ void
kernel_zgemvc_batched(
    int m, int n, int m1, magmaDoubleComplex alpha,
    magmaDoubleComplex **A_array, int lda,
    magmaDoubleComplex **x_array, int incx,
    magmaDoubleComplex beta, magmaDoubleComplex  **y_array, int incy)
{
  

    magmaDoubleComplex *A_ptr = A_array[blockIdx.x];
    magmaDoubleComplex *x_ptr = x_array[blockIdx.x];
    magmaDoubleComplex *y_ptr = y_array[blockIdx.x];

    int tx = threadIdx.x;
    
    magmaDoubleComplex res = MAGMA_Z_ZERO;

    if(tx<m)
    {
        A_ptr += lda * blockIdx.y + tx;
        x_ptr += tx * incx;
    }
        
    __shared__ magmaDoubleComplex sdata[zgemv_bs];

    for(int i=0; i<m1; i+= zgemv_bs)
    {
        res += MAGMA_Z_CNJG (A_ptr[i]) * x_ptr[i*incx];
    }

    if(m > m1)
    {
        if( tx + m1 <  m )
        {
            res  += MAGMA_Z_CNJG(A_ptr[m1]) * x_ptr[m1*incx];
        }
        else
        {
            res  = res;
        }
    }

    sdata[tx] = res;
    __syncthreads();

    for(int s=blockDim.x/2; s>32;s>>=1)
    {
        if(tx<s)
        {
            sdata[tx] += sdata[tx+s];
        } 
        __syncthreads();
    }

    if(zgemv_bs > 32)
    {  
        if(tx<32)
        {
            sdata[tx] += sdata[tx+32];
        }
    }

    if(tx == 0)
    {
        for(int i=1;i<32;i++)
        {
            sdata[tx] += sdata[tx + i];
        }
        
        y_ptr[blockIdx.y * incy] = sdata[0] * alpha + beta * y_ptr[blockIdx.y*incy];
               
    }
}

/*
    Matrix Conjugate Transpose Vector Multiplication
    y := alpha*conjg(A')*x + beta*y,
*/

extern "C"
void magmablas_zgemvc_batched(
    int m, int n, 
    magmaDoubleComplex alpha, magmaDoubleComplex **A_array, int lda, 
    magmaDoubleComplex **x_array,  int incx,
    magmaDoubleComplex beta, magmaDoubleComplex **y_array,  int incy, 
    int batchCount)
{

    dim3 grid(batchCount, n, 1);
    dim3 threads(zgemv_bs, 1, 1);

    int m1 = (m / zgemv_bs) * zgemv_bs;

    kernel_zgemvc_batched <<< grid, threads >>>(m, n, m1, alpha,  A_array, lda, x_array, incx, beta, y_array, incy);
}
   
#endif // defined(PRECISION_z) || defined (PRECISION_c)


 
/**
    Purpose
    -------

    This routine computes Y = alpha opt(A) x + beta y, on the GPU, where
    A = A_array[i],x = x_array[i] and y = y_array[i], i=[0,batchCount-1].
    This is a batched version.

    @param[in]
    trans  CHARACTER*1.
           On entry, TRANS specifies the form of op( A ) to be used in
           the matrix multiplication as follows:
           = 'N':  op( A ) = A.
           = 'T':  op( A ) = A**T.
           = 'C':  op( A ) = A**H.

    @param[in]
    m       INTEGER.
            On entry, M specifies the number of rows of the matrix opt(A).

    @param[in]
    n       INTEGER.
            On entry, N specifies the number of columns of the matrix opt(A)

    @param[in]
    alpha   COMPLEX*16.
            On entry, ALPHA specifies the scalar alpha.

    @param[in]
    A_array A = A_array[i] 
            A: COMPLEX*16 array of dimension ( LDA, n ) on the GPU.
   
    @param[in]
    lda     INTEGER.
            LDA specifies the leading dimension of A.

    @param[in]
    x_array x = x_array[i]
            x: COMPLEX*16 array of dimension n.

    @param[in]
    beta    DOUBLE PRECISION.
            On entry, BETA specifies the scalar beta.

    @param[out]
    y_array y = y_array[i]:       
            y: COMPLEX*16 array of dimension n.
            On exit y = alpha opt(A) x + beta y.

    @param[in]
    batchCount INTEGER
            number of pointers contained in A_array, x_array and y_array.

    @ingroup magma_zblas2
    *******************************************************************   */

extern "C"
void magmablas_zgemv_batched(
    magma_trans_t trans, magma_int_t m, magma_int_t n, 
    magmaDoubleComplex alpha, magmaDoubleComplex **A_array, magma_int_t lda, 
    magmaDoubleComplex **x_array,  magma_int_t incx,
    magmaDoubleComplex beta, magmaDoubleComplex **y_array,  magma_int_t incy, 
    magma_int_t batchCount)
{       
    magma_int_t info = 0;
    if ( trans != MagmaNoTrans && trans != MagmaTrans && trans != MagmaConjTrans )
        info = -1;
    else if ( m < 0 )
        info = -2;
    else if ( n < 0 )
        info = -3;
    else if ( lda < m )
        info = -6;
    else if ( incx == 0 )
        info = -8;
    else if ( incy == 0 )
        info = -11;

    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }

    if ( trans == MagmaNoTrans ) {

        magmablas_zgemvn_batched(m, n, alpha, A_array, lda, x_array, incx, beta, y_array, incy, batchCount);
            
    }
    else if ( trans == MagmaTrans ) {
        magmablas_zgemvt_batched(m, n, alpha, A_array, lda, x_array, incx, beta, y_array, incy, batchCount);
    }
    else if ( trans == MagmaConjTrans ) {
#if defined(PRECISION_z) || defined (PRECISION_c)
        magmablas_zgemvc_batched(m, n, alpha, A_array, lda, x_array, incx, beta, y_array, incy, batchCount);
#else
        magmablas_zgemvt_batched(m, n, alpha, A_array, lda, x_array, incx, beta, y_array, incy, batchCount);
#endif
    }
    else {
        fprintf( stderr, "trans = %c is invalid\n", lapacke_trans_const(trans) );
    }
}

#undef zgemv_bs 


