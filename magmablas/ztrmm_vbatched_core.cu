/*
    -- MAGMA (version 2.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c

       @author Ahmad Abdelfattah
       
*/
#include "magma_internal.h"

#define PRECISION_z
#if defined(PRECISION_z) || defined(PRECIISION_c)
#define BATRMM_NB    (16)
#else
#define BATRMM_NB    (32)
#endif
#include "trmm_template_kernel_vbatched.cuh"

magmaDoubleComplex* magma_zptrb[2];
magmaDoubleComplex* magma_zptra[2];
magmaDoubleComplex* magma_zA;
magmaDoubleComplex* magma_zB;

magma_int_t magma_get_ztrmm_vbatched_nb(magma_int_t n)
{
    if      ( n > 2048 ) return 2048;
    else if ( n > 1024 ) return 1024;
    else if ( n >  512 ) return 512;
    else if ( n >  256 ) return 256;
    else if ( n >  128 ) return 128;
    else if ( n >   64 ) return  64;
    else if ( n >   32 ) return  32;
    else if ( n >   16 ) return  16;
    else if ( n >    8 ) return   8;
    else if ( n >    4 ) return   4;
    else if ( n >    2 ) return   2;
    else return 1;
}
///////////////////////////////////////////////////////////////////////////////////////////////////
void
magmablas_ztrmm_small_vbatched(
        magma_side_t side, magma_uplo_t uplo, magma_trans_t transA, magma_diag_t diag, 
        magma_int_t* m, magma_int_t* n, 
        magmaDoubleComplex alpha, 
        magmaDoubleComplex **dA_array, magma_int_t* ldda,
        magmaDoubleComplex **dB_array, magma_int_t* lddb, 
        magma_int_t max_m, magma_int_t max_n, 
        magma_int_t roffA, magma_int_t coffA, magma_int_t roffB, magma_int_t coffB, 
        magma_int_t spec_m, magma_int_t spec_n, 
        magma_int_t batchCount, magma_queue_t queue )
{
    magma_int_t shape = 0;
    if      (side == MagmaLeft  && transA == MagmaNoTrans   ) { shape = 0; } // left  - NoTrans   (lNx)
    else if (side == MagmaLeft  && transA == MagmaTrans     ) { shape = 1; } // left  - Trans     (lTx)
    else if (side == MagmaLeft  && transA == MagmaConjTrans ) { shape = 2; } // left  - ConjTrans (lCx)
    else if (side == MagmaRight && transA == MagmaNoTrans   ) { shape = 3; } // right - NoTrans   (rNx)
    else if (side == MagmaRight && transA == MagmaTrans     ) { shape = 4; } // right - Trans     (rTx)
    else if (side == MagmaRight && transA == MagmaConjTrans ) { shape = 5; } // right - ConjTrans (rCx)
    
    switch(shape)
    {
        case 0: // lNx
            trmm_template_vbatched_lNx<magmaDoubleComplex, BATRMM_NB>
            (uplo, diag, m, n, alpha, dA_array, ldda, dB_array, lddb, max_m, max_n, roffA, coffA, roffB, coffB, spec_m, spec_n, batchCount, queue);
            break;
        case 1: // lTx
            trmm_template_vbatched_lTx<magmaDoubleComplex, BATRMM_NB, 0>
            (uplo, diag, m, n, alpha, dA_array, ldda, dB_array, lddb, max_m, max_n, roffA, coffA, roffB, coffB, spec_m, spec_n, batchCount, queue);
            break;
        case 2: // lCx
            trmm_template_vbatched_lTx<magmaDoubleComplex, BATRMM_NB, 1>
            (uplo, diag, m, n, alpha, dA_array, ldda, dB_array, lddb, max_m, max_n, roffA, coffA, roffB, coffB, spec_m, spec_n, batchCount, queue);
            break;
        case 3: // rNx
            trmm_template_vbatched_rNx<magmaDoubleComplex, BATRMM_NB>
            (uplo, diag, m, n, alpha, dA_array, ldda, dB_array, lddb, max_m, max_n, roffA, coffA, roffB, coffB, spec_m, spec_n, batchCount, queue);
            break;
        case 4: // rTx
            trmm_template_vbatched_rTx<magmaDoubleComplex, BATRMM_NB, 0>
            (uplo, diag, m, n, alpha, dA_array, ldda, dB_array, lddb, max_m, max_n, roffA, coffA, roffB, coffB, spec_m, spec_n, batchCount, queue);
            break;
        case 5: // rCx
            trmm_template_vbatched_rTx<magmaDoubleComplex, BATRMM_NB, 1>
            (uplo, diag, m, n, alpha, dA_array, ldda, dB_array, lddb, max_m, max_n, roffA, coffA, roffB, coffB, spec_m, spec_n, batchCount, queue);
            break;
        default:; // propose something
    }
}
///////////////////////////////////////////////////////////////////////////////////////////////////
extern "C" void 
magmablas_ztrmm_vbatched_core(
        magma_side_t side, magma_uplo_t uplo, magma_trans_t transA, magma_diag_t diag, 
        magma_int_t* m, magma_int_t* n, 
        magmaDoubleComplex alpha, 
        magmaDoubleComplex **dA_array, magma_int_t* ldda,
        magmaDoubleComplex **dB_array, magma_int_t* lddb, 
        magma_int_t max_m, magma_int_t max_n, 
        magma_int_t roffA, magma_int_t coffA, magma_int_t roffB, magma_int_t coffB, 
        magma_int_t spec_m, magma_int_t spec_n, 
        magma_int_t batchCount, magma_queue_t queue )
{
    const magmaDoubleComplex c_one = MAGMA_Z_ONE; 
    
    magma_int_t max_nrowA = (side == MagmaLeft ? max_m : max_n);
    // stopping condition
    if(max_nrowA <= BATRMM_NB){
        magmablas_ztrmm_small_vbatched( side, uplo, transA, diag, m, n, alpha, dA_array, ldda, dB_array, lddb, max_m, max_n, roffA, coffA, roffB, coffB, spec_m, spec_n, batchCount, queue );
        magma_queue_sync( queue );
        return;
    }
    
    magma_int_t shape = 0;
    if      (side == MagmaLeft   && transA == MagmaNoTrans  && uplo == MagmaLower) { shape = 0; } // lNL
    else if (side == MagmaLeft   && transA == MagmaNoTrans  && uplo == MagmaUpper) { shape = 1; } // lNU
    else if (side == MagmaLeft   && transA != MagmaNoTrans  && uplo == MagmaLower) { shape = 2; } // lTL | lCL
    else if (side == MagmaLeft   && transA != MagmaNoTrans  && uplo == MagmaUpper) { shape = 3; } // lTU | lCU
    else if (side == MagmaRight  && transA == MagmaNoTrans  && uplo == MagmaLower) { shape = 4; } // rNL
    else if (side == MagmaRight  && transA == MagmaNoTrans  && uplo == MagmaUpper) { shape = 5; } // rNU
    else if (side == MagmaRight  && transA != MagmaNoTrans  && uplo == MagmaLower) { shape = 6; } // rTL | rCL
    else if (side == MagmaRight  && transA != MagmaNoTrans  && uplo == MagmaUpper) { shape = 7; } // rTU | rCU
    
    // at this point we can say that max_nrowA > BATRMM_NB
    switch(shape)
    {
        case 0: // lNl
            {
                const int m1 = magma_get_ztrmm_vbatched_nb(max_m); 
                const int m2 = max_m - m1;

                magmablas_ztrmm_vbatched_core( 
                        side, uplo, transA, diag, 
                        m, n, alpha, 
                        dA_array, ldda, 
                        dB_array, lddb, 
                        m2, max_n, 
                        roffA+m1, coffA+m1, roffB+m1, coffB, 
                        m2, 0, 
                        batchCount, queue );
                
                magmablas_zgemm_vbatched_core( 
                        MagmaNoTrans, MagmaNoTrans, 
                        m, n, m, 
                        alpha, dA_array, ldda, 
                               dB_array, lddb, 
                        c_one, dB_array, lddb, 
                        m2, max_n, m1, 
                        roffA+m1, coffA, roffB, coffB, roffB+m1, coffB, 
                        m2, 0, m1, 
                        batchCount, queue );
                
                magmablas_ztrmm_vbatched_core( 
                        side, uplo, transA, diag, 
                        m, n, alpha, 
                        dA_array, ldda, 
                        dB_array, lddb, 
                        m1, max_n, 
                        roffA, coffA, roffB, coffB, 
                        m1, 0, 
                        batchCount, queue );
            }
            break;
        case 1: // lNU
            {
                const int m2 = magma_get_ztrmm_vbatched_nb(max_m); 
                const int m1 = max_m - m2;
                
                magmablas_ztrmm_vbatched_core( 
                        side, uplo, transA, diag, 
                        m, n, alpha, 
                        dA_array, ldda, 
                        dB_array, lddb, 
                        m1, max_n, 
                        roffA, coffA, roffB, coffB, 
                        m1, 0, 
                        batchCount, queue );
                        
                magmablas_zgemm_vbatched_core( 
                        MagmaNoTrans, MagmaNoTrans, 
                        m, n, m, 
                        alpha, dA_array, ldda, 
                               dB_array, lddb, 
                        c_one, dB_array, lddb, 
                        m1, max_n, m2, 
                        roffA, coffA+m1, roffB+m1, coffB, roffB, coffB, 
                        m1, 0, m2, 
                        batchCount, queue );
                        
                magmablas_ztrmm_vbatched_core( 
                        side, uplo, transA, diag, 
                        m, n, alpha, 
                        dA_array, ldda, 
                        dB_array, lddb, 
                        m2, max_n, 
                        roffA+m1, coffA+m1, roffB+m1, coffB, 
                        m2, 0, 
                        batchCount, queue );
            }
            break;  
        case 2: // lTL || lCL
            {
                const int m2 = magma_get_ztrmm_vbatched_nb(max_m); 
                const int m1 = max_m - m2;

                magmablas_ztrmm_vbatched_core( 
                        side, uplo, transA, diag, 
                        m, n, alpha, 
                        dA_array, ldda, 
                        dB_array, lddb, 
                        m1, max_n, 
                        roffA, coffA, roffB, coffB, 
                        m1, 0, 
                        batchCount, queue );

                magmablas_zgemm_vbatched_core( 
                        transA, MagmaNoTrans, 
                        m, n, m, 
                        alpha, dA_array, ldda, 
                               dB_array, lddb, 
                        c_one, dB_array, lddb, 
                        m1, max_n, m2, 
                        roffA+m1, coffA, roffB+m1, coffB, roffB, coffB, 
                        m1, 0, m2, 
                        batchCount, queue );

                magmablas_ztrmm_vbatched_core( 
                        side, uplo, transA, diag, 
                        m, n, alpha, 
                        dA_array, ldda, 
                        dB_array, lddb, 
                        m2, max_n, 
                        roffA+m1, coffA+m1, roffB+m1, coffB, 
                        m2, 0, 
                        batchCount, queue );
            }
            break;
        case 3: // lTU | lCU
            {
                const int m1 = magma_get_ztrmm_vbatched_nb(max_m); 
                const int m2 = max_m - m1;

                magmablas_ztrmm_vbatched_core( 
                        side, uplo, transA, diag, 
                        m, n, alpha, 
                        dA_array, ldda, 
                        dB_array, lddb, 
                        m2, max_n, 
                        roffA+m1, coffA+m1, roffB+m1, coffB, 
                        m2, 0, 
                        batchCount, queue );

                magmablas_zgemm_vbatched_core( 
                        transA, MagmaNoTrans, 
                        m, n, m, 
                        alpha, dA_array, ldda, 
                               dB_array, lddb, 
                        c_one, dB_array, lddb, 
                        m2, max_n, m1, 
                        roffA, coffA+m1, roffB, coffB, roffB+m1, coffB, 
                        m2, 0, m1, 
                        batchCount, queue );

                magmablas_ztrmm_vbatched_core( 
                        side, uplo, transA, diag, 
                        m, n, alpha, 
                        dA_array, ldda, 
                        dB_array, lddb, 
                        m1, max_n, 
                        roffA, coffA, roffB, coffB, 
                        m1, 0, 
                        batchCount, queue );
            }
            break;
        case 4: // rNL
             {
                const int n2 = magma_get_ztrmm_vbatched_nb(max_n); 
                const int n1 = max_n - n2;

                magmablas_ztrmm_vbatched_core( 
                        side, uplo, transA, diag, 
                        m, n, alpha, 
                        dA_array, ldda, 
                        dB_array, lddb, 
                        max_m, n1, 
                        roffA, coffA, roffB, coffB, 
                        0, n1, 
                        batchCount, queue );

                magmablas_zgemm_vbatched_core( 
                        MagmaNoTrans, transA, 
                        m, n, n, 
                        alpha, dB_array, lddb, 
                               dA_array, ldda, 
                        c_one, dB_array, lddb, 
                        max_m, n1, n2, 
                        roffB, coffB+n1, roffA+n1, coffA, roffB, coffB, 
                        0, n1, n2, 
                        batchCount, queue );

                magmablas_ztrmm_vbatched_core( 
                        side, uplo, transA, diag, 
                        m, n, alpha, 
                        dA_array, ldda, 
                        dB_array, lddb, 
                        max_m, n2, 
                        roffA+n1, coffA+n1, roffB, coffB+n1, 
                        0, n2, 
                        batchCount, queue );
            }
            break;
        case 5: // rNU
            {
                const int n1 = magma_get_ztrmm_vbatched_nb(max_n); 
                const int n2 = max_n - n1;

                magmablas_ztrmm_vbatched_core( 
                        side, uplo, transA, diag, 
                        m, n, alpha, 
                        dA_array, ldda, 
                        dB_array, lddb, 
                        max_m, n2, 
                        roffA+n1, coffA+n1, roffB, coffB+n1, 
                        0, n2, 
                        batchCount, queue );

                magmablas_zgemm_vbatched_core( 
                        MagmaNoTrans, transA, 
                        m, n, n, 
                        alpha, dB_array, lddb, 
                               dA_array, ldda, 
                        c_one, dB_array, lddb, 
                        max_m, n2, n1, 
                        roffB, coffB, roffA, coffA+n1, roffB, coffB+n1, 
                        0, n2, n1, 
                        batchCount, queue );

                magmablas_ztrmm_vbatched_core( 
                        side, uplo, transA, diag, 
                        m, n, alpha, 
                        dA_array, ldda, 
                        dB_array, lddb,
                        max_m, n1,  
                        roffA, coffA, roffB, coffB, 
                        0, n1, 
                        batchCount, queue );
            }
            break;
        case 6: // rTL | rCL
            {
                const int n1 = magma_get_ztrmm_vbatched_nb(max_n); 
                const int n2 = max_n - n1;

                magmablas_ztrmm_vbatched_core( 
                        side, uplo, transA, diag, 
                        m, n, alpha, 
                        dA_array, ldda, 
                        dB_array, lddb, 
                        max_m, n2, 
                        roffA+n1, coffA+n1, roffB, coffB+n1, 
                        0, n2, 
                        batchCount, queue );

                magmablas_zgemm_vbatched_core( 
                        MagmaNoTrans, transA, 
                        m, n, n, 
                        alpha, dB_array, lddb, 
                               dA_array, ldda, 
                        c_one, dB_array, lddb, 
                        max_m, n2, n1, 
                        roffB, coffB, roffA+n1, coffA, roffB, coffB+n1, 
                        0, n2, n1, 
                        batchCount, queue );

                magmablas_ztrmm_vbatched_core( 
                        side, uplo, transA, diag, 
                        m, n, alpha, 
                        dA_array, ldda, 
                        dB_array, lddb, 
                        max_m, n1, 
                        roffA, coffA, roffB, coffB, 
                        0, n1, 
                        batchCount, queue );
            }
            break;
        case 7: // rTU | rCU
            {
                const int n2 = magma_get_ztrmm_vbatched_nb(max_n); 
                const int n1 = max_n - n2;

                magmablas_ztrmm_vbatched_core( 
                        side, uplo, transA, diag, 
                        m, n, alpha, 
                        dA_array, ldda, 
                        dB_array, lddb, 
                        max_m, n1, 
                        roffA, coffA, roffB, coffB, 
                        0, n1, 
                        batchCount, queue );

                magmablas_zgemm_vbatched_core( 
                        MagmaNoTrans, transA, 
                        m, n, n, 
                        alpha, dB_array, lddb, 
                               dA_array, ldda, 
                        c_one, dB_array, lddb, 
                        max_m, n1, n2, 
                        roffB, coffB+n1, roffA, coffA+n1, roffB, coffB, 
                        0, n1, n2, 
                        batchCount, queue );

                magmablas_ztrmm_vbatched_core( 
                        side, uplo, transA, diag, 
                        m, n, alpha, 
                        dA_array, ldda, 
                        dB_array, lddb, 
                        max_m, n2, 
                        roffA+n1, coffA+n1, roffB, coffB+n1, 
                        0, n2, 
                        batchCount, queue );
            }
            break;
        default:; // propose something
    }
}
///////////////////////////////////////////////////////////////////////////////////////////////////
