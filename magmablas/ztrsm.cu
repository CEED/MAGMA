#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

       @author Peng Du
       @author Tingxing Dong
*/
#include "common_magma.h"

#define BLOCK_SIZE 16 // inner blocking size, <=32
#define NB 128        // outer blocking size, >BLOCK_SIZE

__global__ void
ztrsm_copy_kernel (int M, int N, magmaDoubleComplex *b, int ldb, magmaDoubleComplex *d_x, int ldx)
{
    int by = blockIdx.y;
    int gx = blockIdx.x*blockDim.x + threadIdx.x;
    if (gx < M)
        b[by*ldb+gx] = d_x[by*ldx+gx];
}


#define MAX_THREAD_PER_BLOCK 512
#define WARP_SIZE 32


#define ztrsm_copy() \
    do { \
        dim3 dimBlock( (M >= MAX_THREAD_PER_BLOCK) ? MAX_THREAD_PER_BLOCK : (WARP_SIZE*((M/WARP_SIZE)+(M % WARP_SIZE != 0))), 1 ); \
        dim3 dimGrid( (M - 1)/dimBlock.x + 1, N ); \
        ztrsm_copy_kernel<<< dimGrid, dimBlock, 0, magma_stream >>>(M, N, b, ldb, d_x, M); \
        magma_device_sync(); \
    } while(0)


/*
 * magmablas_ztrsm
 */

extern "C"
void diag_ztrtri (magma_int_t M, magma_uplo_t uplo, magma_diag_t diag, const magmaDoubleComplex *A, magmaDoubleComplex *d_dinvA, magma_int_t lda);

extern "C"
void magmablas_ztrsm(
    magma_side_t side, magma_uplo_t uplo, magma_trans_t transA, magma_diag_t diag, magma_int_t M, magma_int_t N,
    magmaDoubleComplex alpha,
    const magmaDoubleComplex* A, magma_int_t lda,
    magmaDoubleComplex* b, magma_int_t ldb )
{
/*  -- MAGMA (version 1.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

    Purpose
    =======

    ztrsm solves one of the matrix equations on gpu

        op( A )*x = alpha*b,   or   x*op( A ) = alpha*b,

    where alpha is a scalar, x and b are m by n matrices, A is a unit, or
    non-unit, upper or lower triangular matrix and op( A ) is one of

        op( A ) = A   or   op( A ) = A^T.

    The matrix X is overwritten on B.


    Arguments
    ==========

    side    (input) CHARACTER*1.
            On entry, side specifies whether op( A ) appears on the left
            or right of X as follows:

                side = 'L' or 'l'   op( A )*X = alpha*B.

                side = 'R' or 'r'   X*op( A ) = alpha*B.

    uplo    (input) CHARACTER*1.
            On entry, uplo specifies whether the matrix A is an upper or
            lower triangular matrix as follows:

                uplo = 'U' or 'u'   A is an upper triangular matrix.

                uplo = 'L' or 'l'   A is a lower triangular matrix.

    transA  (input) CHARACTER*1.
            On entry, transA specifies the form of op( A ) to be used in
            the matrix multiplication as follows:

                transA = 'N' or 'n'   op( A ) = A.

                transA = 'T' or 't'   op( A ) = A^T.

                transA = 'C' or 'c'   op( A ) = A^T.

    diag    (input) CHARACTER*1.
            On entry, diag specifies whether or not A is unit triangular
            as follows:

                diag = 'U' or 'u'   A is assumed to be unit triangular.

                diag = 'N' or 'n'   A is not assumed to be unit triangular.

    m       (input) INTEGER.
            On entry, m specifies the number of rows of B. m must be at
            least zero.

    n       (input) INTEGER.
            On entry, n specifies the number of columns of B. n must be
            at least zero.

    alpha   (input) COMPLEX.
            On entry, alpha specifies the scalar alpha. When alpha is
            zero then A is not referenced and B need not be set before
            entry.

    A       (input) COMPLEX array of DIMENSION ( lda, k ), where k is m
            when side = 'L' or 'l' and is n when side = 'R' or 'r'.
            Before entry with uplo = 'U' or 'u', the leading k by k
            upper triangular part of the array A must contain the upper
            triangular matrix and the strictly lower triangular part of
            A is not referenced.
            Before entry with uplo = 'L' or 'l', the leading k by k
            lower triangular part of the array A must contain the lower
            triangular matrix and the strictly upper triangular part of
            A is not referenced.
            Note that when diag = 'U' or 'u', the diagonal elements of
            A are not referenced either, but are assumed to be unity.

    lda     (input) INTEGER.
            On entry, lda specifies the first dimension of A as declared
            in the calling (sub) program. When side = 'L' or 'l' then
            lda must be at least max( 1, m ), when side = 'R' or 'r'
            then lda must be at least max( 1, n ).

    b       (input,output) COMPLEX array of DIMENSION ( ldb, n ).
            Before entry, the leading m by n part of the array B must
            contain the right-hand side matrix B, and on exit is
            overwritten by the solution matrix X.

    ldb     (input) INTEGER.
            On entry, ldb specifies the first dimension of B as declared
            in the calling (sub) program. ldb must be at least
            max( 1, m ).

    Level 3 Blas routine.
    ===================================================================== */

    int i;
    magmaDoubleComplex *d_dinvA, *d_x;

    /* quick return on wrong size */
    if (M <= 0 || N <= 0)
        return;
    
    char Notrans = 'N';
    char Trans = 'T';
    char Conjtrans = 'C';
    magmaDoubleComplex neg_one = MAGMA_Z_NEG_ONE;
    magmaDoubleComplex one = MAGMA_Z_ONE;
    magmaDoubleComplex zero = MAGMA_Z_ZERO;

    if (side == MagmaLeft) {
        // side=L
        /* invert the diagonals
         * Allocate device memory for the inverted diagonal blocks, size=m*NB
         */
        magma_zmalloc( &d_dinvA, NB*((M/NB)+(M % NB != 0))*NB );
        magma_zmalloc( &d_x,     N*M );

        hipMemset(d_x,     0, N*M*sizeof(magmaDoubleComplex));
        hipMemset(d_dinvA, 0, NB*((M/NB)+(M % NB != 0))*NB*sizeof(magmaDoubleComplex));
        diag_ztrtri (M, uplo, diag, A, d_dinvA, lda);

        if (transA == MagmaNoTrans) {
            /* the non-transpose case */
            if (uplo == MagmaLower) {

                /* the lower case */
                /* handle the first block seperately with alpha */
                int MM = min (NB, M);
                hipblasZgemm(Notrans, Notrans, MM, N, MM, alpha, d_dinvA, NB, b, ldb, zero, d_x, M);

                if (NB >= M) {
                    ztrsm_copy();
                    magma_free( d_dinvA );
                    magma_free( d_x );
                    return;
                }

                hipblasZgemm(Notrans, Notrans, M-NB, N, NB, neg_one, A+NB, lda, d_x, M, alpha, b+NB, ldb);

                /* the rest blocks */
                for( i=NB; i < M; i += NB ) {
                    MM = min (M-i, NB);
                    hipblasZgemm(Notrans, Notrans, MM, N, MM, one, d_dinvA+i*NB, NB, b+i, ldb, zero, d_x+i, M);

                    if (i+NB >= M)
                        break;

                    hipblasZgemm(Notrans, Notrans, M-i-NB, N, NB, neg_one, A+i*lda+i+NB, lda, d_x+i, M, one, b+i+NB, ldb);
                }
            }
            else {
                /* the upper case */
                /* handle the first block seperately with alpha */
                int MM = (M % NB == 0) ? NB : (M % NB);
                i = M-MM;
                hipblasZgemm(Notrans, Notrans, MM, N, MM, alpha, d_dinvA+i*NB, NB, b+i, ldb, zero, d_x+i, M);

                if (i-NB < 0) {
                    ztrsm_copy();
                    magma_free( d_dinvA );
                    magma_free( d_x );
                    return;
                }

                hipblasZgemm(Notrans, Notrans, i, N, MM, neg_one, A+i*lda, lda, d_x+i, M, alpha, b, ldb);

                /* the rest blocks */
                for( i=M-MM-NB; i >= 0; i -= NB ) {
                    hipblasZgemm(Notrans, Notrans, NB, N, NB, one, d_dinvA+i*NB, NB, b+i, ldb, zero, d_x+i, M);

                    if (i-NB < 0)
                        break;

                    hipblasZgemm(Notrans, Notrans, i, N, NB, neg_one, A+i*lda, lda, d_x+i, M, one, b, ldb);
                }
            }
        }
        else if( transA == MagmaTrans) {
            /* the transpose case */
            if (uplo == MagmaLower) {
                /* the lower case */
                /* handle the first block seperately with alpha */
                int MM = (M % NB == 0) ? NB : (M % NB);
                i = M-MM;
                hipblasZgemm(Trans, Notrans, MM, N, MM, alpha, d_dinvA+i*NB, NB, b+i, ldb, zero, d_x+i, M);

                if (i-NB < 0) {
                    ztrsm_copy();
                    magma_free( d_dinvA );
                    magma_free( d_x );
                    return;
                }

                hipblasZgemm(Trans, Notrans, i, N, MM, neg_one, A+i, lda, d_x+i, M, alpha, b, ldb);

                /* the rest blocks */
                for( i=M-MM-NB; i >= 0; i -= NB ) {
                    hipblasZgemm(Trans, Notrans, NB, N, NB, one, d_dinvA+i*NB, NB, b+i, ldb, zero, d_x+i, M);

                    if (i-NB < 0)
                        break;

                    hipblasZgemm(Trans, Notrans, i, N, NB, neg_one, A+i, lda, d_x+i, M, one, b, ldb);
                }
            }
            else {
                /* the upper case */
                /* handle the first block seperately with alpha */
                int MM = min (NB, M);
                hipblasZgemm(Trans, Notrans, MM, N, MM, alpha, d_dinvA, NB, b, ldb, zero, d_x, M);

                if (NB >= M) {
                    ztrsm_copy();
                    magma_free( d_dinvA );
                    magma_free( d_x );
                    return;
                }

                hipblasZgemm(Trans, Notrans, M-NB, N, NB, neg_one, A+(NB)*lda, lda, d_x, M, alpha, b+NB, ldb);

                /* the rest blocks */
                for( i=NB; i < M; i += NB ) {
                    MM = min (M-i, NB);
                    hipblasZgemm(Trans, Notrans, MM, N, MM, one, d_dinvA+i*NB, NB, b+i, ldb, zero, d_x+i, M);

                    if (i+NB >= M)
                        break;

                    hipblasZgemm(Trans, Notrans, M-i-NB, N, NB, neg_one, A+(i+NB)*lda+i, lda, d_x+i, M, one, b+i+NB, ldb);
                }
            }
        }
        else{
            /* the conj transpose case */
            if (uplo == MagmaLower) {
                /* the lower case */
                /* handle the first block seperately with alpha */
                int MM = (M % NB == 0) ? NB : (M % NB);
                i = M-MM;
                hipblasZgemm(Conjtrans, Notrans, MM, N, MM, alpha, d_dinvA+i*NB, NB, b+i, ldb, zero, d_x+i, M);

                if (i-NB < 0) {
                    ztrsm_copy();
                    magma_free( d_dinvA );
                    magma_free( d_x );
                    return;
                }

                hipblasZgemm(Conjtrans, Notrans, i, N, MM, neg_one, A+i, lda, d_x+i, M, alpha, b, ldb);

                /* the rest blocks */
                for( i=M-MM-NB; i >= 0; i -= NB ) {
                    hipblasZgemm(Conjtrans, Notrans, NB, N, NB, one, d_dinvA+i*NB, NB, b+i, ldb, zero, d_x+i, M);

                    if (i-NB < 0)
                        break;

                    hipblasZgemm(Conjtrans, Notrans, i, N, NB, neg_one, A+i, lda, d_x+i, M, one, b, ldb);
                }
            }
            else {
                /* the upper case */
                /* handle the first block seperately with alpha */
                int MM = min (NB, M);
                hipblasZgemm(Conjtrans, Notrans, MM, N, MM, alpha, d_dinvA, NB, b, ldb, zero, d_x, M);

                if (NB >= M) {
                    ztrsm_copy();
                    magma_free( d_dinvA );
                    magma_free( d_x );
                    return;
                }

                hipblasZgemm(Conjtrans, Notrans, M-NB, N, NB, neg_one, A+(NB)*lda, lda, d_x, M, alpha, b+NB, ldb);

                /* the rest blocks */
                for( i=NB; i < M; i += NB ) {
                    MM = min (M-i, NB);
                    hipblasZgemm(Conjtrans, Notrans, MM, N, MM, one, d_dinvA+i*NB, NB, b+i, ldb, zero, d_x+i, M);

                    if (i+NB >= M)
                        break;

                    hipblasZgemm(Conjtrans, Notrans, M-i-NB, N, NB, neg_one, A+(i+NB)*lda+i, lda, d_x+i, M, one, b+i+NB, ldb);
                }
            }
        }
    }
    else {
        // side=R
        /* invert the diagonals
         * Allocate device memory for the inverted diagonal blocks, size=N*BLOCK_SIZE
         */
        magma_zmalloc( &d_dinvA, NB*((N/NB) + (N % NB != 0))*NB );
        magma_zmalloc( &d_x,     N*M );
        hipMemset(d_x,     0, N*M*sizeof(magmaDoubleComplex));
        hipMemset(d_dinvA, 0, NB*((N/NB)+(N % NB != 0))*NB*sizeof(magmaDoubleComplex));
        diag_ztrtri (N, uplo, diag, A, d_dinvA, lda);

        if (transA == MagmaNoTrans) {
            /* the non-transpose case */
            if (uplo == MagmaLower) {
                /* the lower case */
                /* handle the first block seperately with alpha */
                int NN = (N % NB == 0) ? NB : (N % NB);
                i = N-NN;
                hipblasZgemm(Notrans, Notrans, M, NN, NN, alpha, b+ldb*i, ldb, d_dinvA+i*NB, NB, zero, d_x+i*M, M);

                if (i-NB < 0) {
                    ztrsm_copy();
                    magma_free( d_x );
                    magma_free( d_dinvA );
                    return;
                }

                hipblasZgemm(Notrans, Notrans, M, i, NN, neg_one, d_x+i*M, M, A+i, lda, alpha, b, ldb);

                /* the rest blocks */
                for( i=N-NN-NB; i >= 0; i -= NB ) {
                    hipblasZgemm(Notrans, Notrans, M, NB, NB, one, b+ldb*i, ldb, d_dinvA+i*NB, NB, zero, d_x+i*M, M);

                    if (i-NB < 0)
                        break;

                    hipblasZgemm(Notrans, Notrans, M, i, NB, neg_one, d_x+i*M, M, A+i, lda, one, b, ldb);
                }
            }
            else {
                /* the upper case */
                /* handle the first block seperately with alpha */
                int NN = min(NB, N);
                hipblasZgemm(Notrans, Notrans, M, NN, NN, alpha, b, ldb, d_dinvA, NB, zero, d_x, M);

                if (NB >= N) {
                    ztrsm_copy();
                    magma_free( d_x );
                    magma_free( d_dinvA );
                    return;
                }

                hipblasZgemm(Notrans, Notrans, M, N-NB, NB, neg_one, d_x, M, A+NB*lda, lda, alpha, b+NB*ldb, ldb);

                /* the rest blocks */
                for( i=NB; i < N; i += NB ) {
                    NN = min(NB, N-i);
                    hipblasZgemm(Notrans, Notrans, M, NN, NN, one, b+ldb*i, ldb, d_dinvA+i*NB, NB, zero, d_x+i*M, M);

                    if (i+NB >= N)
                        break;

                    hipblasZgemm(Notrans, Notrans, M, N-i-NB, NB, neg_one, d_x+i*M, M,   A+(i+NB)*lda+i, lda, one, b+(i+NB)*ldb, ldb);
                }
            }
        }
        else if (transA == MagmaTrans) {
            /* the transpose case */
            if (uplo == MagmaLower) {
                /* the lower case */
                /* handle the first block seperately with alpha */
                int NN = min(NB, N);
                hipblasZgemm(Notrans, Trans, M, NN, NN, alpha, b, ldb, d_dinvA, NB, zero, d_x, M);

                if (NB >= N) {
                    ztrsm_copy();
                    magma_free( d_x );
                    magma_free( d_dinvA );
                    return;
                }

                hipblasZgemm(Notrans, Trans, M, N-NB, NB, neg_one, d_x, M, A+NB, lda, alpha, b+NB*ldb, ldb);

                /* the rest blocks */
                for( i=NB; i < N; i += NB ) {
                    NN = min(NB, N-i);
                    hipblasZgemm(Notrans, Trans, M, NN, NN, one, b+ldb*i, ldb, d_dinvA+i*NB, NB, zero, d_x+i*M, M);

                    if (i+NB >= N)
                        break;

                    hipblasZgemm(Notrans, Trans, M, N-i-NB, NB, neg_one, d_x+i*M, M,   A+i*lda+NB+i, lda, one, b+(i+NB)*ldb, ldb);
                }
            }
            else {
                /* the upper case */
                /* handle the first block seperately with alpha */
                int NN = (N % NB == 0) ? NB : (N % NB);
                i = N-NN;
                hipblasZgemm(Notrans, Trans, M, NN, NN, alpha, b+ldb*i, ldb, d_dinvA+i*NB, NB, zero, d_x+i*M, M);

                if (i-NB < 0) {
                    ztrsm_copy();
                    magma_free( d_x );
                    magma_free( d_dinvA );
                    return;
                }

                hipblasZgemm(Notrans, Trans, M, i, NN, neg_one, d_x+i*M, M, A+i*lda, lda, alpha, b, ldb);

                /* the rest blocks */
                for( i=N-NN-NB; i >= 0; i -= NB ) {
                    hipblasZgemm(Notrans, Trans, M, NB, NB, one, b+ldb*i, ldb, d_dinvA+i*NB, NB, zero, d_x+i*M, M);

                    if (i-NB < 0)
                        break;

                    hipblasZgemm(Notrans, Trans, M, i, NB, neg_one, d_x+i*M, M, A+i*lda, lda, one, b, ldb);
                }
            }
        }
        else{
            /* the Conj transpose case */
            if (uplo == MagmaLower) {
                /* the lower case */
                /* handle the first block seperately with alpha */
                int NN = min(NB, N);
                hipblasZgemm(Notrans, Conjtrans, M, NN, NN, alpha, b, ldb, d_dinvA, NB, zero, d_x, M);

                if (NB >= N) {
                    ztrsm_copy();
                    magma_free( d_x );
                    magma_free( d_dinvA );
                    return;
                }

                hipblasZgemm(Notrans, Conjtrans, M, N-NB, NB, neg_one, d_x, M, A+NB, lda, alpha, b+NB*ldb, ldb);

                /* the rest blocks */
                for( i=NB; i < N; i += NB ) {
                    NN = min(NB, N-i);
                    hipblasZgemm(Notrans, Conjtrans, M, NN, NN, one, b+ldb*i, ldb, d_dinvA+i*NB, NB, zero, d_x+i*M, M);

                    if (i+NB >= N)
                        break;

                    hipblasZgemm(Notrans, Conjtrans, M, N-i-NB, NB, neg_one, d_x+i*M, M,   
                                                A+i*lda+NB+i, lda, one, b+(i+NB)*ldb, ldb);
                }
            }
            else {
                /* the upper case */
                /* handle the first block seperately with alpha */
                int NN = (N % NB == 0) ? NB : (N % NB);
                i = N-NN;
                hipblasZgemm(Notrans, Conjtrans, M, NN, NN, alpha, b+ldb*i, ldb, d_dinvA+i*NB, NB, zero, d_x+i*M, M);

                if (i-NB < 0) {
                    ztrsm_copy();
                    magma_free( d_x );
                    magma_free( d_dinvA );
                    return;
                }

                hipblasZgemm(Notrans, Conjtrans, M, i, NN, neg_one, d_x+i*M, M, A+i*lda, lda, alpha, b, ldb);

                /* the rest blocks */
                for( i=N-NN-NB; i >= 0; i -= NB ) {
                    hipblasZgemm(Notrans, Conjtrans, M, NB, NB, one, b+ldb*i, ldb, d_dinvA+i*NB, NB, zero, d_x+i*M, M);

                    if (i-NB < 0)
                        break;

                    hipblasZgemm(Notrans, Conjtrans, M, i, NB, neg_one, d_x+i*M, M, A+i*lda, lda, one, b, ldb);
                }
            }
        }

    }

    ztrsm_copy();
    magma_free( d_dinvA );
    magma_free( d_x );
}
