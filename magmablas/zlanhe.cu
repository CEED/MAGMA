#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal z -> s d c

*/
#include "common_magma.h"

#define BLOCK_SIZE 32

//#define num_threads 64
#define dgemv_bs 32

#define zlanhe_bs 64

#define PRECISION_z
#if (!defined(PRECISION_z)) || (GPUSHMEM >= 200)

__global__ void
l_zlanhe_special (int n, hipDoubleComplex* A, int lda,  double *y){
  int tx = threadIdx.x ; 
  int ty = threadIdx.y ; 
  int ind = blockIdx.x*  dgemv_bs + tx ;
  double res = 0.;

  __shared__ hipDoubleComplex la[dgemv_bs][dgemv_bs+1];
          
  A += ind;
  A+= ty * lda  ;  
  int break_d  =   blockIdx.x* dgemv_bs ;

  for(int  i=0; i<break_d; i += dgemv_bs ){
    #pragma unroll 8 
    for(int j=0; j < dgemv_bs ; j+=4){
        la[tx][ty+j] = A[j*lda] ;
    }
    __syncthreads();

    #pragma unroll 8 
    for(int j=0; j < 8 ; j++){
       res+=hipCabs( la[tx][j+ty*8]) ;
    }
    A+=lda* dgemv_bs ;
    __syncthreads(); 
  }

 
  #pragma unroll 8
  for(int j =0; j<dgemv_bs; j+=4)
         la[ty+j][tx] = A[ j * lda];


  A+= dgemv_bs ;
  __syncthreads();
  #pragma unroll 8
  for(int  i=ty*8; i<(1+ty)* dgemv_bs/4 ; i++){
         if ( i < tx )   {
                la[tx][i] = la[i][tx] ; 
         }
         else 
                la[tx][i] = la[tx][i]  ;
  
  }
  __syncthreads();
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4 ; j++){
     res+=hipCabs(la[tx][j+ty*8]);
    }
  break_d  += dgemv_bs ; 
  __syncthreads();

  for(int i=break_d; i<n; i += dgemv_bs ){
   #pragma unroll 8
    for(int j=0; j<dgemv_bs; j+=4)
       la[ty+j][tx] = A[ j * lda];
    A+= dgemv_bs ;
      __syncthreads();
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4;j++){
       res+= hipCabs(la[tx][j+ty*8]);
    }
      __syncthreads();
  }


  la[tx][ty] = MAGMA_Z_MAKE( res, 0. );
   __syncthreads();
   if( ty == 0 ) {
     res = res 
       + MAGMA_Z_REAL( la[tx][1] ) 
       + MAGMA_Z_REAL( la[tx][2] )
       + MAGMA_Z_REAL( la[tx][3] );
     y[ind] = res;
   }

}

__global__ void
l_zlanhe_generic(int n, hipDoubleComplex* A, int lda,  double *y , int m_full_block , 
                 int m_mod_32)
{ 
  int tx = threadIdx.x ; 
  int ty = threadIdx.y ; 

  int ind = blockIdx.x*  dgemv_bs + tx ;
  
  double res = 0.;

  __shared__ hipDoubleComplex la   [dgemv_bs][dgemv_bs+1];

  if( blockIdx.x == m_full_block ) {
  /************************************************************************
   -- Last block --
   -- We will do something unusual here 
   -- For sufficiently large matrix the overhead will be very low
  *************************************************************************/
       if  ( tx < m_mod_32 ){
                A+= ( blockIdx.x * dgemv_bs + tx ) ;
       }                  
       else{
                A+= ( blockIdx.x * dgemv_bs + m_mod_32 -1) ; 
       }
       A+= ty * lda  ;  
       int break_d  =   blockIdx.x* dgemv_bs ;

          /*----------------------------
                Go Right
          -------------------------------*/

          for(int  i=0; i<break_d; i += dgemv_bs ){
            #pragma unroll 8 
            for(int j=0; j < dgemv_bs ; j+=4){
                la[tx][ty+j] = A[j*lda] ;
            }
            __syncthreads();

            #pragma unroll 8 
            for(int j=0; j < 8 ; j++){
               res+=hipCabs( la[tx][j+ty*8]);
            }
            A+=lda* dgemv_bs ;
            __syncthreads(); 
          }
          /*
           we don't need to make zero, as those computation will be discarded. 
          */
          if( ty==0  ) {
                /*--------------------------------------------
                        he will compute the triangular parts
                        others will be waiting with values. 
                -----------------------------------------------*/
                int j ;
                int count = 1 ; 
                if( tx < m_mod_32 ) 
                        count = tx ; 
                else
                        count = m_mod_32 ;
                for(j =0;j<=count;j++){
                        res+= hipCabs( A[j*lda]) ;
                }
                A+=(tx)*lda;
                count = 1 ; 
                for(;j<m_mod_32;j++){
                        res+=hipCabs( A[count]) ;
                        count++;
                }
          }
          else{
          }
          __syncthreads(); 
          la[tx][ty]= MAGMA_Z_MAKE( res, 0. ) ;
          __syncthreads();
         /*--------------------------------------------------------
         The leader accumulates all the results from his peer. 
         ----------------------------------------------------------*/
         if( ty == 0 ) {
           res = res 
             + MAGMA_Z_REAL( la[tx][1] ) 
             + MAGMA_Z_REAL( la[tx][2] )
             + MAGMA_Z_REAL( la[tx][3] );
           if( tx < m_mod_32)
             y[ind] = res;
         }
         
  }

  else{ 
  /***************************************
    -----------------------------------
  -- All the blocks but the last one --
  ****************************************
  -------------------------------------*/
  A += ind;
  A+= ty * lda  ;  
  int break_d  =   blockIdx.x* dgemv_bs ;

  /*----------------------------
        Go Right
  -------------------------------*/
  for(int  i=0; i<break_d; i += dgemv_bs ){
    #pragma unroll 8 
    for(int j=0; j < dgemv_bs ; j+=4){
        la[tx][ty+j] = A[j*lda] ;
    }
    __syncthreads();

    #pragma unroll 8 
    for(int j=0; j < 8 ; j++){
       res+=hipCabs(la[tx][j+ty*8]);
    }
    A+=lda* dgemv_bs ;
    __syncthreads(); 
  }

 
  /*------------------------------------
        Diagonal 
        Copy + Transpose lower triangle
  --------------------------------------*/
  #pragma unroll 8
  for(int j =0; j<dgemv_bs; j+=4)
         la[ty+j][tx] = A[ j * lda];


  A+= dgemv_bs ;
  __syncthreads();
  /*--------------------------------------------
        Mirror Upper Triangle to Lower triangle
  ---------------------------------------------*/
  #pragma unroll 8
  for(int  i=ty*8; i<(1+ty)* dgemv_bs/4 ; i++){
         if ( i < tx )   {
                la[tx][i] = la[i][tx] ; 
         }
         else 
                la[tx][i] = la[tx][i]  ;
  
  }
  __syncthreads();
  /*--------------------------------
        Do diagonal Computation
  -----------------------------------*/
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4 ; j++){
     res+= hipCabs(la[tx][j+ty*8]);
    }
  break_d  += dgemv_bs ; 
  __syncthreads();


  n -= m_mod_32 ;  // @ 
  /*-----------------------------
        Go Down 
  -------------------------------*/
  for(int i=break_d; i<n; i += dgemv_bs ){
   #pragma unroll 8
    for(int j=0; j<dgemv_bs; j+=4)
       la[ty+j][tx] = A[ j * lda];
    A+= dgemv_bs ;
      __syncthreads();
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4;j++){
       res+=hipCabs(la[tx][j+ty*8]);
    }
      __syncthreads();
  }

  
  /*---------------------------------------------
        doing m_mod_32 stuffs here.
        Symmetric is giving us benefit .. true
  -----------------------------------------------*/
    A-=tx;
    if( tx < m_mod_32){
        A+=tx;
    }
    else{
        A+=(m_mod_32-1); /* Same as above*/
    }

   #pragma unroll 8
    for(int j=0; j<dgemv_bs; j+=4){
       if( tx < m_mod_32 ) 
         la[ty+j][tx] = MAGMA_Z_MUL( MAGMA_Z_ONE,  A[ j * lda] );
       else
         la[ty+j][tx] = MAGMA_Z_MUL( MAGMA_Z_ZERO, A[ j * lda] );
       
    }
    __syncthreads();

    /*----------------------------------------
        What about doing some Zeroing here?
        instead of zeroing before?
    -----------------------------------------*/        
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4;j++){
       res+=hipCabs(la[tx][j+ty*8]);
    }
    __syncthreads();
   

    la[tx][ty]= MAGMA_Z_MAKE( res, 0. );
   __syncthreads();
   /*--------------------------------------------------------
        The leader accumulates all the results from his peer. 
   ----------------------------------------------------------*/
   if( ty == 0 ) {
     res = res 
       + MAGMA_Z_REAL( la[tx][1] ) 
       + MAGMA_Z_REAL( la[tx][2] )
       + MAGMA_Z_REAL( la[tx][3] );
     y[ind] = res;
   }

  }

}

__global__ void
u_zlanhe_generic (int n, hipDoubleComplex* A, int lda, double *y , int m_full_block , int m_mod_32){

  
  int tx = threadIdx.x ; 
  int ty = threadIdx.y ; 

  int ind = blockIdx.x*  dgemv_bs + tx ;
  
  double res = 0.;


  __shared__ hipDoubleComplex la   [dgemv_bs][dgemv_bs+1];
  int blockIdxx =  blockIdx.x ;

  if( blockIdx.x == m_full_block ) {

  /************************************************************************
   -- Last block --
   -- We will do something unusual here 
   -- For sufficiently large matrix the overhead will be very low
  *************************************************************************/

  ind =  tx ;
  A+= lda*(n-1) ; 


       if  ( tx < m_mod_32 ){
                A+= (  tx ) ;
       }                  
       else{
                A+= (  m_mod_32 -1) ; 
       }
       A-= ty * lda  ;  
       int break_d  =   (blockIdx.x)* dgemv_bs ;

          /*----------------------------
                Go Right
          -------------------------------*/

          for(int  i=0; i<break_d; i += dgemv_bs ){
            #pragma unroll 8 
            for(int j=0; j < dgemv_bs ; j+=4){
                la[tx][ty+j] = A[-j*lda] ;
            }
            __syncthreads();

            #pragma unroll 8 
            for(int j=0; j < 8 ; j++){
               res+=hipCabs(la[tx][j+ty*8]);
            }
            A-=lda* dgemv_bs ;
            __syncthreads(); 
          }
          /*
           we don't need to make zero, as those computation will be discarded. 
          */
          if( ty==0  ) {
                /*--------------------------------------------
                        he will compute the triangular parts
                        others will be waiting with values. 
                -----------------------------------------------*/
                int j ;
                int count = 1 ; 
                if( tx < m_mod_32 ) 
                        count =m_mod_32- tx ; 
                else
                        count = m_mod_32 ;
                for(j =0;j<count;j++){
                        res+= hipCabs( A[-j*lda] );
                }
                A-=(count-1)*lda;
                count = 1 ; 
                for(;j<m_mod_32;j++){
                        res+= hipCabs( A[-count] );
                        count++;
                }
          }
          else{
          }
          __syncthreads(); 
          la[tx][ty]= MAGMA_Z_MAKE( res, 0. );
          __syncthreads();
         /*--------------------------------------------------------
         The leader accumulates all the results from his peer. 
         ----------------------------------------------------------*/
         if( ty == 0 ) {
           res = res 
             + MAGMA_Z_REAL( la[tx][1] ) 
             + MAGMA_Z_REAL( la[tx][2] )
             + MAGMA_Z_REAL( la[tx][3] );
           if( tx < m_mod_32)
             y[ind] = res;
         }
         
  }

  else{ 
  /***************************************
    -----------------------------------
  -- All the blocks but the last one --
  -- By the way this code can be optimized more. 
  ****************************************
  -------------------------------------*/
  ind = blockIdx.x *  dgemv_bs + tx + m_mod_32 ;
  hipDoubleComplex *A1 = A ; 
  A+= lda*(n-1)  ; 

  A += ind;
  A-= ty * lda  ;  

  int break_d  = (n / dgemv_bs -   blockIdxx-1 )* dgemv_bs ;
  /*----------------------------
        Go Left
  -------------------------------*/
  for(int  i=0; i<break_d; i += dgemv_bs ){
    #pragma unroll 8 
    for(int j=0; j < dgemv_bs ; j+=4){
        la[tx][ty+j] = A[-j*lda] ;
    }
    __syncthreads();

    #pragma unroll 8 
    for(int j=0; j < 8 ; j++){
       res+=hipCabs( la[tx][j+ty*8]);
    }
    A-=lda* dgemv_bs ;
    __syncthreads(); 
  }

 
  /*------------------------------------
        Diagonal 
        Copy + Transpose lower triangle
  --------------------------------------*/
  #pragma unroll 8
  for(int j =0; j<dgemv_bs; j+=4){
         la[tx][31-ty-j] = A[ -j * lda];
  }

  A-= dgemv_bs ;
  __syncthreads();
  /*--------------------------------------------
        Mirror Upper Triangle to Lower triangle
  ---------------------------------------------*/
  #pragma unroll 8
  for(int  i=ty*8; i<(1+ty)* dgemv_bs/4 ; i++){
         if ( i <tx ){
                la[tx][i] = la[i][tx]; 
         }
         else{ 
                la[tx][i] = la[tx][i]  ;
         }
  }
  __syncthreads();
  /*--------------------------------
        Do diagonal Computation
  -----------------------------------*/
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4 ; j++){
     res+=hipCabs(  la[tx][j+ty*8] ) ;
    }
  break_d  += dgemv_bs ; 
  __syncthreads();


  n -= m_mod_32 ;  // @ 
  /*-----------------------------
        Go Up 
  -------------------------------*/
  int i ;
  for( i=break_d; i<n; i+= dgemv_bs ){
   #pragma unroll 8
    for(int j=0; j<dgemv_bs; j+=4){
       la[ty+j][tx] = A[- j * lda];
    }
    A-= dgemv_bs ;
      __syncthreads();
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4;j++){
       res+=hipCabs ( la[31-tx][j+ty*8] );
    }
      __syncthreads();
  }
  /*---------------------------------------------
        doing m_mod_32 stuffs here.
        Symmetric is giving us benefit .. true
        Do the other way please......
  -----------------------------------------------*/
   A1 = A1 + m_mod_32 * lda + tx *lda ;  
   if( ty == 0  ) {
        for( int j = 0 ;  j < m_mod_32 ; j++){
                res+=  hipCabs (  A1[ j + lda * (blockIdx.x) * 32 ] ) ;
        }
   }
    __syncthreads();

    la[tx][ty]= MAGMA_Z_MAKE( res, 0);
   __syncthreads();
   /*--------------------------------------------------------
        The leader accumulates all the results from his peer. 
   ----------------------------------------------------------*/
   if( ty == 0 ) {
     res = res 
       + MAGMA_Z_REAL( la[tx][1] ) 
       + MAGMA_Z_REAL( la[tx][2] )
       + MAGMA_Z_REAL( la[tx][3] );
     y[ind] =  res;
   }
  }
}

__global__ void
u_zlanhe_special (int n, hipDoubleComplex* A, int lda, double *y ){
  int tx = threadIdx.x ; 
  int ty = threadIdx.y ; 
  int ind = blockIdx.x*  dgemv_bs + tx ;
  double res = 0.;

  /*
        Reverse Computation ... 
                - Left 
                - Triangle 
                - Up 
  */

  A+= lda*(n-1) ; 
  __shared__ hipDoubleComplex la   [dgemv_bs][dgemv_bs+1];

  A += ind;
  A-= ty * lda  ;  
  int break_d  = (n / dgemv_bs -   blockIdx.x-1 )* dgemv_bs ;

  for(int  i=0; i<break_d; i += dgemv_bs ){
    #pragma unroll 8 
    for(int j=0; j < dgemv_bs ; j+=4){
        la[tx][ty+j] = A[-j*lda] ;
    }
    __syncthreads();

    #pragma unroll 8 
    for(int j=0; j < 8 ; j++){
       res+=hipCabs(la[tx][j+ty*8]);
    }
    A-=lda* dgemv_bs ;
    __syncthreads(); 
  }

  #pragma unroll 8
  for(int j =0; j<dgemv_bs; j+=4)
         la[tx][31-ty-j] = A[ -j * lda];
  /*
        Look at the indexing changes
  */

  A-= dgemv_bs ;
  __syncthreads();
  #pragma unroll 8
  for(int  i=ty*8; i<(1+ty)* dgemv_bs/4 ; i++){
         if ( i <tx ){
                la[tx][i] = la[i][tx]; 
         }
         else{ 
                la[tx][i] = la[tx][i]  ;
         }
  
  }
  __syncthreads();
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4 ; j++){
     res+= hipCabs(la[tx][j+ty*8]);
    }

  break_d  += dgemv_bs ; 
  __syncthreads();

  for(int i=break_d; i<n; i+= dgemv_bs ){
   #pragma unroll 8
    for(int j=0; j<dgemv_bs; j+=4)
       la[ty+j][tx] = A[ -j * lda];

    A-= dgemv_bs ;
      __syncthreads();
    #pragma unroll 8
    for(int j=0; j < dgemv_bs/4;j++){
       res+=hipCabs( la[31-tx][j+ty*8]);
    }
      __syncthreads();
  }

  la[tx][ty]= MAGMA_Z_MAKE( res, 0. );

   __syncthreads();
   if( ty == 0 ) {
     res = res 
       + MAGMA_Z_REAL( la[tx][1] ) 
       + MAGMA_Z_REAL( la[tx][2] )
       + MAGMA_Z_REAL( la[tx][3] );
     y[ind] =   res;
   }
}


extern "C" void mzlanhe (char uplo , int m ,  hipDoubleComplex *A , int lda ,  double *Y  )
{
/*
Note:
        The UPLO = 'U' Version can be optimized more.
        side is not needed........................... 
*/
    int blocks;
    if (m % dgemv_bs==0)
        blocks = m/ dgemv_bs;
    else
        blocks = m/ dgemv_bs + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(32, 4, 1);

    if( m % dgemv_bs == 0 ) {
            if( uplo == 'L' || uplo == 'l'){        
                    l_zlanhe_special <<< grid, threads, 0, magma_stream >>> (m, A, lda, Y);
            }
            else{
                    u_zlanhe_special <<< grid, threads, 0, magma_stream >>> (m, A, lda,  Y);
            } 
                
    } 
    else{        
            int  m_full_block = (m - m % 32 ) /32 ; 
            int  m_mod_32 = m%32 ;  
            if( uplo == 'L' || uplo == 'l'){
                    l_zlanhe_generic <<< grid, threads, 0, magma_stream >>> (m, A, lda, Y , m_full_block , m_mod_32);
            }        
            else{
                    u_zlanhe_generic <<< grid, threads, 0, magma_stream >>> (m, A, lda, Y , m_full_block , m_mod_32);
            }        
    }
}

#endif /* (!defined(PRECISION_z)) || (GPUSHMEM >= 200) */

__global__ void
l_zlanhe_max (int m, hipDoubleComplex* A, int lda,  double *y){
    int tx  = threadIdx.x ;
    int ind =  blockIdx.x * zlanhe_bs + tx ;
    double res = 0., res1;

    int break_d = blockIdx.x* zlanhe_bs;

    if (ind < m)
    {
       A += ind;

       for(int i=0; i<break_d; i += zlanhe_bs ){
          #pragma unroll 8
          for(int j=0; j< zlanhe_bs; j++){
             res1 = hipCabs(A[j*lda]);
             res = fmax(res,res1);
          }
    
          A += lda*zlanhe_bs;
       }   
  
     
       for(int j=0; j<=tx; j++){
          res1 = hipCabs(A[j*lda]);
          res = fmax(res,res1);
       }

       y[ind] = res;
    }
}

__global__ void
u_zlanhe_max (int m, hipDoubleComplex* A, int lda,  double *y){
    int ind =  blockIdx.x * zlanhe_bs + threadIdx.x ;
    double res = 0.;

    A += ind;
    if (ind < m){
      for(int j=m-1; j>= ind; j--)
         res = fmax(res, hipCabs(A[j*lda]));
      
      y[ind] = res;
    }
}


extern "C" void zlanhe_max (char uplo, int m, hipDoubleComplex *A , int lda , double *y){
    int blocks;
    if (m % zlanhe_bs==0)
        blocks = m/ zlanhe_bs;
    else
        blocks = m/ zlanhe_bs + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(zlanhe_bs, 1, 1);

    if( uplo == 'L' || uplo == 'l'){
      l_zlanhe_max <<< grid, threads, 0, magma_stream >>> (m, A, lda, y);
    }
    else{
      u_zlanhe_max <<< grid, threads, 0, magma_stream >>> (m, A, lda, y);
    }
}
 
extern "C" double 
magmablas_zlanhe(char norm, char uplo, magma_int_t n, 
                 hipDoubleComplex *A, magma_int_t lda, double *WORK )
{
        if (norm == 'I' || norm =='i')  
            {
#if (GPUSHMEM >= 200)
                mzlanhe ( uplo , n , A , lda , WORK);
                int val = hipblasIdamax(n,WORK,1);
                double retVal[1];
                hipblasGetMatrix( 1, 1, sizeof( double ), WORK+val-1, 1, retVal, 1 ) ;
                return retVal[0];
#else
                printf("Only normM is available. Exit.\n");
                exit(1);
#endif
            }
        else if (norm == 'M' || norm =='m')
            {  
                zlanhe_max ( uplo , n , A , lda , WORK);
                int val = hipblasIdamax(n,WORK,1);
                double retVal[1];
                hipblasGetMatrix( 1, 1, sizeof( double ), WORK+val-1, 1, retVal, 1 ) ;
                return retVal[0];
            }
        else
            {
                printf("Only normI and normM are available. Exit.\n");
                exit(1);
            }
}


