#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal z -> s d c

*/
#include "common_magma.h"
#define PRECISION_z
#include "commonblas.h"

extern "C" void
magmablas_ztranspose2s(hipDoubleComplex *odata, int ldo,
                       hipDoubleComplex *idata, int ldi,
                       int m, int n, hipStream_t *stream );


//
//      m, n - dimensions in the source (input) matrix.
//             This routine copies the ha matrix from the CPU
//             to dat on the GPU. In addition, the output matrix
//             is transposed. The routine uses a buffer of size
//             2*lddb*nb pointed to by dB (lddb > m) on the GPU. 
//             Note that lda >= m and lddat >= n.
//
extern "C" void 
magmablas_zsetmatrix_transpose( int m, int n,
                                hipDoubleComplex  *ha, int lda, 
                                hipDoubleComplex *dat, int ldda,
                                hipDoubleComplex  *dB, int lddb, int nb )
{
    int i = 0, j = 0, ib;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || ldda < n || lddb < m){
        printf("Wrong arguments in zhtodt.\n");
        return;
    }

    static hipStream_t stream[2];
    magma_queue_create( &stream[0] );
    magma_queue_create( &stream[1] );
   
    /* Move data from CPU to GPU in the first panel in the dB buffer */
    ib   = min(n-i, nb);
    magma_zsetmatrix_async( m, ib,
                            ha + i*lda,             lda,
                            dB + (j%2) * nb * lddb, lddb, stream[j%2] );
    j++;

    for(i=nb; i<n; i+=nb){
       /* Move data from CPU to GPU in the second panel in the dB buffer */
       ib   = min(n-i, nb);
       magma_zsetmatrix_async( m, ib,
                               ha+i*lda,               lda,
                               dB + (j%2) * nb * lddb, lddb, stream[j%2] );
       j++;
  
       /* Note that the previous panel (i.e., j%2) comes through the stream
          for the kernel so there is no need to synchronize.             */
       // magmablas_ztranspose2( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, nb);
       magmablas_ztranspose2s( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, nb, &stream[j%2]);
    }

    /* Transpose the last part of the matrix.                            */
    j++;
    // magmablas_ztranspose2( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, ib);
    magmablas_ztranspose2s( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, ib, &stream[j%2]);

    magma_queue_destroy( stream[0] );
    magma_queue_destroy( stream[1] );
}

//===========================================================================
//  This version is similar to the above but for multiGPUs. The distribution
//  is 1D block cyclic. The input arrays are pointers for the corresponding 
//  GPUs. The streams are passed as argument, in contrast to the single GPU
//  routine.
//===========================================================================
extern "C" void 
magmablas_zsetmatrix_transpose2( int m, int n,
                                 hipDoubleComplex  *ha,  int  lda, 
                                 hipDoubleComplex **dat, int *ldda,
                                 hipDoubleComplex **dB,  int  lddb, int nb,
                                 int num_gpus, hipStream_t stream[][2] )
{
    int i = 0, j[4] = {0, 0, 0, 0}, ib, k = 0;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || lddb < m){
        printf("Wrong arguments in zhtodt2.\n");
        return;
    }

    if (n<num_gpus*nb){
       for(i=0; i<n; i+=nb){
          k = (i/nb)%num_gpus;
          magma_setdevice(k);

          ib = min(n-i, nb);
          magma_zsetmatrix_async( m, ib,
                                  ha+i*lda, lda,
                                  dB[k],    lddb, stream[k][0] );
       }
       for(i=0; i<n; i+=nb){
          k = (i/nb)%num_gpus;
          magma_setdevice(k);

          ib = min(n-i, nb);
          //magma_queue_sync( stream[k][0] );
          //magmablas_ztranspose2( dat[k]+ i/(nb*num_gpus)*nb, ldda[k],
          //                       dB[k], lddb, m, ib);
          magmablas_ztranspose2s( dat[k]+ i/(nb*num_gpus)*nb, ldda[k],
                                 dB[k], lddb, m, ib, &stream[k][0]);
       }
    } 
    else
    {
      for(i=0; i<(n + num_gpus*nb); i+=nb){
         k = (i/nb)%num_gpus;
         magma_setdevice(k);

         if (i<n){
            /* Move data from CPU to GPU in the second panel in the dB buffer */
            ib = min(n-i, nb);
            magma_zsetmatrix_async( m, ib,
                                    ha+i*lda,                 lda,
                                    dB[k] + (j[k]%2)*nb*lddb, lddb, stream[k][j[k]%2] );
         }
         j[k]++;
  
         if (i> (num_gpus-1)*nb){
            /* Make sure that the previous panel (i.e., j[k]%2) has arrived 
               and transpose it directly into the dat matrix                  */
            //magma_queue_sync( stream[k][ j[k]%2 ] );
            ib = min(n - i + num_gpus*nb, nb);
            //magmablas_ztranspose2( dat[k]+ i/(nb*num_gpus)*nb -nb, ldda[k],
            //                       dB[k] +(j[k]%2)*nb*lddb, lddb, m, ib);
            magmablas_ztranspose2s( dat[k]+ i/(nb*num_gpus)*nb -nb, ldda[k],
                                   dB[k] +(j[k]%2)*nb*lddb, lddb, m, ib, &stream[k][j[k]%2]);

         }
      }
    }
}
