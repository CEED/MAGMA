#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c

*/
#include "common_magma.h"
#define PRECISION_z
#include "commonblas.h"


//
//      m, n - dimensions in the source (input) matrix.
//             This routine copies the ha matrix from the CPU
//             to dat on the GPU. In addition, the output matrix
//             is transposed. The routine uses a buffer of size
//             2*lddb*nb pointed to by dB (lddb > m) on the GPU. 
//             Note that lda >= m and lddat >= n.
//
extern "C" void 
magmablas_zsetmatrix_transpose( magma_int_t m, magma_int_t n,
                                const magmaDoubleComplex  *ha, magma_int_t lda, 
                                magmaDoubleComplex       *dat, magma_int_t ldda,
                                magmaDoubleComplex        *dB, magma_int_t lddb, magma_int_t nb )
{
    magma_int_t i = 0, j = 0, ib;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || ldda < n || lddb < m){
        printf("Wrong arguments in %s.\n", __func__);
        return;
    }

    magma_queue_t stream[2];
    magma_queue_create( &stream[0] );
    magma_queue_create( &stream[1] );
   
    /* Move data from CPU to GPU in the first panel in the dB buffer */
    ib   = min(n-i, nb);
    magma_zsetmatrix_async( m, ib,
                            ha + i*lda,             lda,
                            dB + (j%2) * nb * lddb, lddb, stream[j%2] );
    j++;

    for(i=nb; i<n; i+=nb){
       /* Move data from CPU to GPU in the second panel in the dB buffer */
       ib   = min(n-i, nb);
       magma_zsetmatrix_async( m, ib,
                               ha+i*lda,               lda,
                               dB + (j%2) * nb * lddb, lddb, stream[j%2] );
       j++;
  
       /* Note that the previous panel (i.e., j%2) comes through the stream
          for the kernel so there is no need to synchronize.             */
       // magmablas_ztranspose2( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, nb);
       magmablas_ztranspose2s( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, nb, stream[j%2]);
    }

    /* Transpose the last part of the matrix.                            */
    j++;
    // magmablas_ztranspose2( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, ib);
    magmablas_ztranspose2s( dat+i-nb, ldda, dB + (j%2)*nb*lddb, lddb, m, ib, stream[j%2]);

    magma_queue_destroy( stream[0] );
    magma_queue_destroy( stream[1] );
}
