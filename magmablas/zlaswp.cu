#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal z -> s d c
       
       @author Mathieu Faverge
       @author Ichitaro Yamazaki
       @author Mark Gates
*/
#include "common_magma.h"

// MAX_PIVOTS is maximum number of pivots to apply in each kernel launch
// NTHREADS is number of threads in a block
#define MAX_PIVOTS 32
#define NTHREADS   64

typedef struct {
    hipDoubleComplex *dAT;
    int n, lda, j0, npivots;
    int ipiv[MAX_PIVOTS];
} zlaswp_params_t;


// Matrix A is stored row-wise in dAT.
// Divide matrix A into block-columns of NTHREADS columns each.
// Each GPU block processes one block-column of A.
// Each thread goes down a column of A,
// swapping rows according to pivots stored in params.
__global__ void zlaswp_kernel( zlaswp_params_t params )
{
    unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if( tid < params.n ) {
        int lda = params.lda;
        hipDoubleComplex *dAT = params.dAT + tid + params.j0*lda;
        hipDoubleComplex *A1  = dAT;
        
        for( int i1 = 0; i1 < params.npivots; ++i1 ) {
            int i2 = params.ipiv[i1];
            hipDoubleComplex *A2 = dAT + i2*lda;
            hipDoubleComplex temp = *A1;
            *A1 = *A2;
            *A2 = temp;
            A1 += lda;  // A1 = dA + i1*ldx
        }
    }
}


// Launch zlaswp kernel with ceil( n / NTHREADS ) blocks of NTHREADS threads each.
extern "C" void zlaswp( zlaswp_params_t &params )
{
    int blocks = (params.n + NTHREADS - 1) / NTHREADS;
    zlaswp_kernel<<< blocks, NTHREADS, 0, magma_stream >>>( params );
}


// Swap rows of A, stored row-wise.
// This version updates each entry of ipiv by adding ind.
// It is used in zgetrf, zgetrf_gpu, zgetrf_mgpu, zgetrf_ooc.
extern "C" void
magmablas_zpermute_long2( magma_int_t n, hipDoubleComplex *dAT, magma_int_t lda,
                          magma_int_t *ipiv, magma_int_t nb, magma_int_t ind )
{
    for( int k = 0; k < nb; k += MAX_PIVOTS ) {
        int npivots = min( MAX_PIVOTS, nb-k );
        // fields are:             dAT  n  lda  j0       npivots
        zlaswp_params_t params = { dAT, n, lda, ind + k, npivots };
        for( int j = 0; j < npivots; ++j ) {
            params.ipiv[j] = ipiv[ind + k + j] - k - 1;
            ipiv[ind + k + j] += ind;
        }
        zlaswp( params );
    }
}


// Swap rows of A, stored row-wise.
// This version assumes ind has already been added to ipiv.
// It is used in zgetrf_mgpu, zgetrf_ooc.
extern "C" void
magmablas_zpermute_long3( hipDoubleComplex *dAT, magma_int_t lda,
                          const magma_int_t *ipiv, magma_int_t nb, magma_int_t ind )
{
    for( int k = 0; k < nb; k += MAX_PIVOTS ) {
        int npivots = min( MAX_PIVOTS, nb-k );
        // fields are:             dAT  n    lda  j0       npivots
        zlaswp_params_t params = { dAT, lda, lda, ind + k, npivots };
        for( int j = 0; j < MAX_PIVOTS; ++j ) {
            params.ipiv[j] = ipiv[ind + k + j] - k - 1 - ind;
        }
        zlaswp( params );
    }
}


// Swap rows of A, stored row-wise.
// This interface is identical to LAPACK's laswp interface.
// It is used in zgessm, zgetrf_incpiv.
extern "C" void
magmablas_zlaswp( magma_int_t n, hipDoubleComplex *dAT, magma_int_t lda,
                  magma_int_t i1, magma_int_t i2,
                  const magma_int_t *ipiv, magma_int_t inci )
{
    for( int k = i1-1; k < i2; k += MAX_PIVOTS ) {
        int npivots = min( MAX_PIVOTS, i2-k );
        // fields are:             dAT        n  lda  j0 npivots
        zlaswp_params_t params = { dAT+k*lda, n, lda, 0, npivots };
        for( int j = 0; j < npivots; ++j ) {
            params.ipiv[j] = ipiv[(k+j)*inci] - k - 1;
        }
        zlaswp( params );
    }
}


// ------------------------------------------------------------
// Extended version has stride in both directions (ldx, ldy)
// to handle both row-wise and column-wise storage.

typedef struct {
    hipDoubleComplex *dA;
    int n, ldx, ldy, j0, npivots;
    int ipiv[MAX_PIVOTS];
} zlaswpx_params_t;


// Matrix A is stored row-wise in dA.
// Divide matrix A into block-columns of NTHREADS columns each.
// Each GPU block processes one block-column of A.
// Each thread goes down a column of A,
// swapping rows according to pivots stored in params.
__global__ void zlaswpx_kernel( zlaswpx_params_t params )
{
    unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if( tid < params.n ) {
        int ldx = params.ldx;
        hipDoubleComplex *dA = params.dA + tid*params.ldy + params.j0*ldx;
        hipDoubleComplex *A1  = dA;
        
        for( int i1 = 0; i1 < params.npivots; ++i1 ) {
            int i2 = params.ipiv[i1];
            hipDoubleComplex *A2 = dA + i2*ldx;
            hipDoubleComplex temp = *A1;
            *A1 = *A2;
            *A2 = temp;
            A1 += ldx;  // A1 = dA + i1*ldx
        }
    }
}


// Launch zlaswpx kernel with ceil( n / NTHREADS ) blocks of NTHREADS threads each.
extern "C" void zlaswpx( zlaswpx_params_t &params )
{
    int blocks = (params.n + NTHREADS - 1) / NTHREADS;
    zlaswpx_kernel<<< blocks, NTHREADS, 0, magma_stream >>>( params );
}


// Swap rows of A.
// For A stored row-wise,    set ldx=lda and ldy=1.
// For A stored column-wise, set ldx=1   and ldy=lda.
// Otherwise, this interface is identical to LAPACK's laswp interface.
extern "C" void
magmablas_zlaswpx( magma_int_t n, hipDoubleComplex *dA, magma_int_t ldx, magma_int_t ldy,
                   magma_int_t i1, magma_int_t i2,
                   const magma_int_t *ipiv, magma_int_t inci )
{
    for( int k = i1-1; k < i2; k += MAX_PIVOTS ) {
        int npivots = min( MAX_PIVOTS, i2-k );
        // fields are:              dA        n  ldx  ldy  j0 npivots
        zlaswpx_params_t params = { dA+k*ldx, n, ldx, ldy, 0, npivots };
        for( int j = 0; j < npivots; ++j ) {
            params.ipiv[j] = ipiv[(k+j)*inci] - k - 1;
        }
        zlaswpx( params );
    }
}


// ------------------------------------------------------------
// This version takes d_ipiv on the GPU. Thus it does not pass pivots
// as an argument using a structure, avoiding all the argument size
// limitations of CUDA and OpenCL. It also needs just one kernel launch
// with all the pivots, instead of multiple kernel launches with small
// batches of pivots. On Fermi, it is faster than magmablas_zlaswp
// (including copying pivots to the GPU).

__global__ void zlaswp2_kernel( int n, hipDoubleComplex *dAT, int lda, int npivots, const magma_int_t* d_ipiv )
{
    unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
    if( tid < n ) {
        dAT += tid;
        hipDoubleComplex *A1  = dAT;
        
        for( int i1 = 0; i1 < npivots; ++i1 ) {
            int i2 = d_ipiv[i1] - 1;  // Fortran index
            hipDoubleComplex *A2 = dAT + i2*lda;
            hipDoubleComplex temp = *A1;
            *A1 = *A2;
            *A2 = temp;
            A1 += lda;  // A1 = dA + i1*ldx
        }
    }
}

// Swap rows of A, stored row-wise.
// d_ipiv is vector of pivots stored on the GPU,
// unlike magmablas_zlaswp where ipiv is stored on the CPU.
// This interface is identical to LAPACK's laswp interface.
extern "C" void
magmablas_zlaswp2( magma_int_t n, hipDoubleComplex* dAT, magma_int_t lda,
                   magma_int_t i1, magma_int_t i2,
                   const magma_int_t *d_ipiv )
{
    int blocks = (n + NTHREADS - 1) / NTHREADS;
    zlaswp2_kernel<<< blocks, NTHREADS, 0, magma_stream >>>(
        n, dAT + (i1-1)*lda, lda, i2-(i1-1), d_ipiv );
}
