#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c

       @author Ichitaro Yamazaki
*/
#include "common_magma.h"

#define NB 64
#define A(i,j) (A[(i) + (j)*lda])
#define W(i,j) (W[(i) + (j)*ldw])


// each thread block does one NB x n block row of A.
// each thread does one row, starting from left edge and moving right to diagonal.
__global__ void
zlascl_2x2_lower(
    int m,
    const magmaDoubleComplex* W, int ldw,
    magmaDoubleComplex* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    magmaDoubleComplex D21 = W( 1, 0 );
    magmaDoubleComplex D11 = MAGMA_Z_DIV( W( 1, 1 ), D21 );
    magmaDoubleComplex D22 = MAGMA_Z_DIV( W( 0, 0 ), MAGMA_Z_CNJG( D21 ) );
    double T = 1.0 / ( MAGMA_Z_REAL( D11*D22 ) - 1.0 );
    D21 = MAGMA_Z_DIV( MAGMA_Z_MAKE(T,0.0), D21 );

    if (ind < m) {
        A( ind, 0 ) = MAGMA_Z_CNJG( D21 )*( D11*W( 2+ind, 0 )-W( 2+ind, 1 ) );
        A( ind, 1 ) = D21*( D22*W( 2+ind, 1 )-W( 2+ind, 0 ) );
    }
}


// each thread block does one NB x n block row of A.
// each thread does one row, starting from right edge and moving left to diagonal.
__global__ void
zlascl_2x2_upper(
    int m,
    const magmaDoubleComplex *W, int ldw,
    magmaDoubleComplex* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    magmaDoubleComplex D21 = W( m, 1 );
    magmaDoubleComplex D11 = MAGMA_Z_DIV( W( m+1, 1 ), MAGMA_Z_CNJG( D21 ) );
    magmaDoubleComplex D22 = MAGMA_Z_DIV( W( m, 0 ), D21 );
    double T = 1.0 / ( MAGMA_Z_REAL( D11*D22 ) - 1.0 );
    D21 = MAGMA_Z_DIV( MAGMA_Z_MAKE(T,0.0), D21 );

    if (ind < m) {
        A( ind, 0 ) = D21*( D11*W( ind, 0 )-W( ind, 1 ) );
        A( ind, 1 ) = MAGMA_Z_CNJG( D21 )*( D22*W( ind, 1 )-W( ind, 0 ) );
    }
}


/**
    Purpose
    -------
    ZLASCL_2x2 scales the M by M complex matrix A by the 2-by-2 pivot.
    TYPE specifies that A may be upper or lower triangular.

    Arguments
    ---------
    @param[in]
    type    magma_type_t
            TYPE indices the storage type of the input matrix A.
            = MagmaLower:  lower triangular matrix.
            = MagmaUpper:  upper triangular matrix.
            Other formats that LAPACK supports, MAGMA does not currently support.

    @param[in]
    m       INTEGER
            The number of rows of the matrix A.  M >= 0.

    @param[in]
    dW      DOUBLE PRECISION vector, dimension (2*lddw)
            The matrix containing the 2-by-2 pivot.

    @param[in]
    lddw    INTEGER
            The leading dimension of the array W.  LDDA >= max(1,M).

    @param[in,out]
    dA      COMPLEX*16 array, dimension (LDDA,N)
            The matrix to be scaled by dW.  See TYPE for the
            storage type.

    @param[in]
    ldda    INTEGER
            The leading dimension of the array A.  LDDA >= max(1,M).

    @param[out]
    info    INTEGER
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value.

    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_zaux2
    ********************************************************************/
extern "C" void
magmablas_zlascl_2x2_q(
    magma_type_t type, magma_int_t m,
    magmaDoubleComplex_const_ptr dW, magma_int_t lddw,
    magmaDoubleComplex_ptr       dA, magma_int_t ldda,
    magma_queue_t queue,
    magma_int_t *info )
{
    *info = 0;
    if ( type != MagmaLower && type != MagmaUpper )
        *info = -1;
    else if ( m < 0 )
        *info = -2;
    else if ( ldda < max(1,m) )
        *info = -4;
    
    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return;  //info;
    }
    
    dim3 threads( NB );
    dim3 grid( magma_ceildiv( m, NB ) );
    
    if (type == MagmaLower) {
        zlascl_2x2_lower <<< grid, threads, 0, queue >>> (m, dW, lddw, dA, ldda);
    }
    else {
        zlascl_2x2_upper <<< grid, threads, 0, queue >>> (m, dW, lddw, dA, ldda);
    }
}


/**
    @see magmablas_zlascl2_q
    @ingroup magma_zaux2
    ********************************************************************/
extern "C" void
magmablas_zlascl_2x2(
    magma_type_t type, magma_int_t m,
    magmaDoubleComplex_const_ptr dW, magma_int_t lddw,
    magmaDoubleComplex_ptr       dA, magma_int_t ldda,
    magma_int_t *info )
{
    magmablas_zlascl_2x2_q( type, m, dW, lddw, dA, ldda, magma_stream, info );
}
