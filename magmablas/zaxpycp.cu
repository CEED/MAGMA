#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c

*/
#include "common_magma.h"

#define NB 64

// adds   x += r  --and--
// copies r = b
// each thread does one index, x[i] and r[i]
__global__ void
zaxpycp_kernel(
    int m,
    magmaDoubleComplex *r,
    magmaDoubleComplex *x,
    const magmaDoubleComplex *b)
{
    const int i = threadIdx.x + blockIdx.x*NB;
    if ( i < m ) {
        x[i] = MAGMA_Z_ADD( x[i], r[i] );
        r[i] = b[i];
    }
}


// ----------------------------------------------------------------------
// adds   x += r  --and--
// copies r = b
extern "C" void
magmablas_zaxpycp_q(
    magma_int_t m,
    magmaDoubleComplex_ptr r,
    magmaDoubleComplex_ptr x,
    magmaDoubleComplex_const_ptr b,
    magma_queue_t queue )
{
    dim3 threads( NB );
    dim3 grid( magma_ceildiv( m, NB ) );
    zaxpycp_kernel <<< grid, threads, 0, queue->hip_stream() >>> ( m, r, x, b );
}

extern "C" void
magmablas_zaxpycp(
    magma_int_t m,
    magmaDoubleComplex_ptr r,
    magmaDoubleComplex_ptr x,
    magmaDoubleComplex_const_ptr b)
{
    magmablas_zaxpycp_q( m, r, x, b, magmablasGetQueue() );
}
