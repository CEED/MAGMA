#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal d -> s
*/
#include "common_magma.h"
#include "commonblas_d.h"

/*
 * daxpy computes c += alpha*b, where b and c are 16-element vectors.
 */
static __device__ void daxpy(
    double alpha,
    const double* __restrict__ b,
    double*       __restrict__ c )
{
    c[0]  += alpha * b[0];
    c[1]  += alpha * b[1];
    c[2]  += alpha * b[2];
    c[3]  += alpha * b[3];
    c[4]  += alpha * b[4];
    c[5]  += alpha * b[5];
    c[6]  += alpha * b[6];
    c[7]  += alpha * b[7];
    c[8]  += alpha * b[8];
    c[9]  += alpha * b[9];
    c[10] += alpha * b[10];
    c[11] += alpha * b[11];
    c[12] += alpha * b[12];
    c[13] += alpha * b[13];
    c[14] += alpha * b[14];
    c[15] += alpha * b[15];
}


/**
    Purpose:
    --------
    This routine computes
        C = alpha * A*B^T + beta * C

    B is put into shared memory
    Parameters Used:
        blk_M=64 blk_N=16 blk_K=4 nthd_x=16 nthd_y=4

    This code should run for any matrix size.

    @ingroup magma_dblas3
    ********************************************************************/
__global__ void
dgemm_kernel_N_T_64_16_4_16_4(
    double*       __restrict__ C,
    const double* __restrict__ A,
    const double* __restrict__ B,
    int m, int n, int k,
    int lda, int ldb, int ldc,
    double alpha, double beta )
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    const int ibx = blockIdx.x * 64;
    const int iby = blockIdx.y * 16;
    
    const int idt = ty * 16 + tx;

    if ( iby + tx >= n )
        B += iby + 0;
    else
        B += iby + tx;
    /*
        Taking care of boundary cases where K < 4.
    */
    if ( ty >= k )
        B += __mul24( 0, ldb );
    else
        B += __mul24( ty, ldb );
    
    if ( ibx + idt >= m )
        A += ibx + 0;
    else
        A += ibx + idt;

    int s2=lda, s3=2*lda, s4=3*lda;

    switch (k) {
        case 1: s2=0;    s3=0;      s4=0;  break;
        case 2: s2=lda;  s3=0;      s4=0;  break;
        case 3: s2=lda;  s3=2*lda;  s4=0;  break;
    }
    
    C += ibx + idt + __mul24( iby, ldc );

    double Ap[4] = { A[0], A[s2], A[s3], A[s4] };

    double b = B[0];

    const double *Bend = B + ldb*(k - k % 4);

    B += 4*ldb;
    A += 4*lda;

    __shared__ double Bb[4][16];

    double Cb[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

    if ( k > 7 ) {
        do {
            double Ab[4] = {Ap[0], Ap[1], Ap[2], Ap[3]};

            Bb[ty][tx]=b;

            __syncthreads();

            Ap[0] = A[0];
            Ap[1] = A[s2];
            Ap[2] = A[s3];
            Ap[3] = A[s4];

            b=B[0];

            daxpy( Ab[0], &Bb[0][0], Cb );
            daxpy( Ab[1], &Bb[1][0], Cb );
            daxpy( Ab[2], &Bb[2][0], Cb );
            daxpy( Ab[3], &Bb[3][0], Cb );

            A += 4*lda;
            B += 4*ldb;

            __syncthreads();
        } while (B < Bend);
    }

    if ( k > 3 ) {
        Bb[ty][tx]=b;
        int k1 = k - k % 4;

        if ( (k1+ty) >= k )
            B -= 4*ldb;
        else
            B -= 0*ldb;

        if ( (k1+0) >= k ) {s2=0;    s3=0*lda;  s4=0;  A -= 4*lda; } else
        if ( (k1+1) >= k ) {s2=0;    s3=0*lda;  s4=0;  A -= 0*lda; } else
        if ( (k1+2) >= k ) {s2=lda;  s3=0*lda;  s4=0;  A -= 0*lda; } else
        if ( (k1+3) >= k ) {s2=lda;  s3=2*lda;  s4=0;  A -= 0*lda; }
                
        __syncthreads();

        b=B[0];

        daxpy( Ap[0], &Bb[0][0], Cb );  Ap[0] = A[0];
        daxpy( Ap[1], &Bb[1][0], Cb );  Ap[1] = A[s2];
        daxpy( Ap[2], &Bb[2][0], Cb );  Ap[2] = A[s3];
        daxpy( Ap[3], &Bb[3][0], Cb );  Ap[3] = A[s4];
    }

    k = k % 4;

    if ( k != 0 ) {
        __syncthreads();

        Bb[ty][tx]=b;

        __syncthreads();

        for(int i=0; i < k; i++) {
            daxpy( Ap[i], &Bb[i][0], Cb );
        }
    }

    if ( (iby+16)>=n) {
        lda = n-iby;
    }
    else{
        lda = 16;
    }

    if ( (ibx+idt) >= m )
        lda = 0;
    else
        lda = lda;

    switch(lda) {
        case 16:
                C[ 0    ] = alpha * Cb[0]  + beta * C[ 0    ];
                C[ 1*ldc] = alpha * Cb[1]  + beta * C[ 1*ldc];
                C[ 2*ldc] = alpha * Cb[2]  + beta * C[ 2*ldc];
                C[ 3*ldc] = alpha * Cb[3]  + beta * C[ 3*ldc];
                C[ 4*ldc] = alpha * Cb[4]  + beta * C[ 4*ldc];
                C[ 5*ldc] = alpha * Cb[5]  + beta * C[ 5*ldc];
                C[ 6*ldc] = alpha * Cb[6]  + beta * C[ 6*ldc];
                C[ 7*ldc] = alpha * Cb[7]  + beta * C[ 7*ldc];
                C[ 8*ldc] = alpha * Cb[8]  + beta * C[ 8*ldc];
                C[ 9*ldc] = alpha * Cb[9]  + beta * C[ 9*ldc];
                C[10*ldc] = alpha * Cb[10] + beta * C[10*ldc];
                C[11*ldc] = alpha * Cb[11] + beta * C[11*ldc];
                C[12*ldc] = alpha * Cb[12] + beta * C[12*ldc];
                C[13*ldc] = alpha * Cb[13] + beta * C[13*ldc];
                C[14*ldc] = alpha * Cb[14] + beta * C[14*ldc];
                C[15*ldc] = alpha * Cb[15] + beta * C[15*ldc];
                break;
        case 15:
                C[ 0    ] = alpha * Cb[0]  + beta * C[ 0    ];
                C[ 1*ldc] = alpha * Cb[1]  + beta * C[ 1*ldc];
                C[ 2*ldc] = alpha * Cb[2]  + beta * C[ 2*ldc];
                C[ 3*ldc] = alpha * Cb[3]  + beta * C[ 3*ldc];
                C[ 4*ldc] = alpha * Cb[4]  + beta * C[ 4*ldc];
                C[ 5*ldc] = alpha * Cb[5]  + beta * C[ 5*ldc];
                C[ 6*ldc] = alpha * Cb[6]  + beta * C[ 6*ldc];
                C[ 7*ldc] = alpha * Cb[7]  + beta * C[ 7*ldc];
                C[ 8*ldc] = alpha * Cb[8]  + beta * C[ 8*ldc];
                C[ 9*ldc] = alpha * Cb[9]  + beta * C[ 9*ldc];
                C[10*ldc] = alpha * Cb[10] + beta * C[10*ldc];
                C[11*ldc] = alpha * Cb[11] + beta * C[11*ldc];
                C[12*ldc] = alpha * Cb[12] + beta * C[12*ldc];
                C[13*ldc] = alpha * Cb[13] + beta * C[13*ldc];
                C[14*ldc] = alpha * Cb[14] + beta * C[14*ldc];
                break;
        case 14:
                C[ 0    ] = alpha * Cb[0]  + beta * C[ 0    ];
                C[ 1*ldc] = alpha * Cb[1]  + beta * C[ 1*ldc];
                C[ 2*ldc] = alpha * Cb[2]  + beta * C[ 2*ldc];
                C[ 3*ldc] = alpha * Cb[3]  + beta * C[ 3*ldc];
                C[ 4*ldc] = alpha * Cb[4]  + beta * C[ 4*ldc];
                C[ 5*ldc] = alpha * Cb[5]  + beta * C[ 5*ldc];
                C[ 6*ldc] = alpha * Cb[6]  + beta * C[ 6*ldc];
                C[ 7*ldc] = alpha * Cb[7]  + beta * C[ 7*ldc];
                C[ 8*ldc] = alpha * Cb[8]  + beta * C[ 8*ldc];
                C[ 9*ldc] = alpha * Cb[9]  + beta * C[ 9*ldc];
                C[10*ldc] = alpha * Cb[10] + beta * C[10*ldc];
                C[11*ldc] = alpha * Cb[11] + beta * C[11*ldc];
                C[12*ldc] = alpha * Cb[12] + beta * C[12*ldc];
                C[13*ldc] = alpha * Cb[13] + beta * C[13*ldc];
                break;
        case 13:
                C[ 0    ] = alpha * Cb[0]  + beta * C[ 0    ];
                C[ 1*ldc] = alpha * Cb[1]  + beta * C[ 1*ldc];
                C[ 2*ldc] = alpha * Cb[2]  + beta * C[ 2*ldc];
                C[ 3*ldc] = alpha * Cb[3]  + beta * C[ 3*ldc];
                C[ 4*ldc] = alpha * Cb[4]  + beta * C[ 4*ldc];
                C[ 5*ldc] = alpha * Cb[5]  + beta * C[ 5*ldc];
                C[ 6*ldc] = alpha * Cb[6]  + beta * C[ 6*ldc];
                C[ 7*ldc] = alpha * Cb[7]  + beta * C[ 7*ldc];
                C[ 8*ldc] = alpha * Cb[8]  + beta * C[ 8*ldc];
                C[ 9*ldc] = alpha * Cb[9]  + beta * C[ 9*ldc];
                C[10*ldc] = alpha * Cb[10] + beta * C[10*ldc];
                C[11*ldc] = alpha * Cb[11] + beta * C[11*ldc];
                C[12*ldc] = alpha * Cb[12] + beta * C[12*ldc];
                break;
        case 12:
                C[ 0    ] = alpha * Cb[0]  + beta * C[ 0    ];
                C[ 1*ldc] = alpha * Cb[1]  + beta * C[ 1*ldc];
                C[ 2*ldc] = alpha * Cb[2]  + beta * C[ 2*ldc];
                C[ 3*ldc] = alpha * Cb[3]  + beta * C[ 3*ldc];
                C[ 4*ldc] = alpha * Cb[4]  + beta * C[ 4*ldc];
                C[ 5*ldc] = alpha * Cb[5]  + beta * C[ 5*ldc];
                C[ 6*ldc] = alpha * Cb[6]  + beta * C[ 6*ldc];
                C[ 7*ldc] = alpha * Cb[7]  + beta * C[ 7*ldc];
                C[ 8*ldc] = alpha * Cb[8]  + beta * C[ 8*ldc];
                C[ 9*ldc] = alpha * Cb[9]  + beta * C[ 9*ldc];
                C[10*ldc] = alpha * Cb[10] + beta * C[10*ldc];
                C[11*ldc] = alpha * Cb[11] + beta * C[11*ldc];
                break;
        case 11:
                C[ 0    ] = alpha * Cb[0]  + beta * C[ 0    ];
                C[ 1*ldc] = alpha * Cb[1]  + beta * C[ 1*ldc];
                C[ 2*ldc] = alpha * Cb[2]  + beta * C[ 2*ldc];
                C[ 3*ldc] = alpha * Cb[3]  + beta * C[ 3*ldc];
                C[ 4*ldc] = alpha * Cb[4]  + beta * C[ 4*ldc];
                C[ 5*ldc] = alpha * Cb[5]  + beta * C[ 5*ldc];
                C[ 6*ldc] = alpha * Cb[6]  + beta * C[ 6*ldc];
                C[ 7*ldc] = alpha * Cb[7]  + beta * C[ 7*ldc];
                C[ 8*ldc] = alpha * Cb[8]  + beta * C[ 8*ldc];
                C[ 9*ldc] = alpha * Cb[9]  + beta * C[ 9*ldc];
                C[10*ldc] = alpha * Cb[10] + beta * C[10*ldc];
                break;
        case 10:
                C[0    ] = alpha * Cb[0] + beta * C[0    ];
                C[1*ldc] = alpha * Cb[1] + beta * C[1*ldc];
                C[2*ldc] = alpha * Cb[2] + beta * C[2*ldc];
                C[3*ldc] = alpha * Cb[3] + beta * C[3*ldc];
                C[4*ldc] = alpha * Cb[4] + beta * C[4*ldc];
                C[5*ldc] = alpha * Cb[5] + beta * C[5*ldc];
                C[6*ldc] = alpha * Cb[6] + beta * C[6*ldc];
                C[7*ldc] = alpha * Cb[7] + beta * C[7*ldc];
                C[8*ldc] = alpha * Cb[8] + beta * C[8*ldc];
                C[9*ldc] = alpha * Cb[9] + beta * C[9*ldc];
                break;
        case 9:
                C[0    ] = alpha * Cb[0] + beta * C[0    ];
                C[1*ldc] = alpha * Cb[1] + beta * C[1*ldc];
                C[2*ldc] = alpha * Cb[2] + beta * C[2*ldc];
                C[3*ldc] = alpha * Cb[3] + beta * C[3*ldc];
                C[4*ldc] = alpha * Cb[4] + beta * C[4*ldc];
                C[5*ldc] = alpha * Cb[5] + beta * C[5*ldc];
                C[6*ldc] = alpha * Cb[6] + beta * C[6*ldc];
                C[7*ldc] = alpha * Cb[7] + beta * C[7*ldc];
                C[8*ldc] = alpha * Cb[8] + beta * C[8*ldc];
                break;
        case 8:
                C[0    ] = alpha * Cb[0] + beta * C[0    ];
                C[1*ldc] = alpha * Cb[1] + beta * C[1*ldc];
                C[2*ldc] = alpha * Cb[2] + beta * C[2*ldc];
                C[3*ldc] = alpha * Cb[3] + beta * C[3*ldc];
                C[4*ldc] = alpha * Cb[4] + beta * C[4*ldc];
                C[5*ldc] = alpha * Cb[5] + beta * C[5*ldc];
                C[6*ldc] = alpha * Cb[6] + beta * C[6*ldc];
                C[7*ldc] = alpha * Cb[7] + beta * C[7*ldc];
                break;
        case 7:
                C[0    ] = alpha * Cb[0] + beta * C[0    ];
                C[1*ldc] = alpha * Cb[1] + beta * C[1*ldc];
                C[2*ldc] = alpha * Cb[2] + beta * C[2*ldc];
                C[3*ldc] = alpha * Cb[3] + beta * C[3*ldc];
                C[4*ldc] = alpha * Cb[4] + beta * C[4*ldc];
                C[5*ldc] = alpha * Cb[5] + beta * C[5*ldc];
                C[6*ldc] = alpha * Cb[6] + beta * C[6*ldc];
                break;
        case 6:
                C[0    ] = alpha * Cb[0] + beta * C[0    ];
                C[1*ldc] = alpha * Cb[1] + beta * C[1*ldc];
                C[2*ldc] = alpha * Cb[2] + beta * C[2*ldc];
                C[3*ldc] = alpha * Cb[3] + beta * C[3*ldc];
                C[4*ldc] = alpha * Cb[4] + beta * C[4*ldc];
                C[5*ldc] = alpha * Cb[5] + beta * C[5*ldc];
                break;
        case 5:
                C[0    ] = alpha * Cb[0] + beta * C[0    ];
                C[1*ldc] = alpha * Cb[1] + beta * C[1*ldc];
                C[2*ldc] = alpha * Cb[2] + beta * C[2*ldc];
                C[3*ldc] = alpha * Cb[3] + beta * C[3*ldc];
                C[4*ldc] = alpha * Cb[4] + beta * C[4*ldc];
                break;
        case 4:
                C[0    ] = alpha * Cb[0] + beta * C[0    ];
                C[1*ldc] = alpha * Cb[1] + beta * C[1*ldc];
                C[2*ldc] = alpha * Cb[2] + beta * C[2*ldc];
                C[3*ldc] = alpha * Cb[3] + beta * C[3*ldc];
                break;
        case 3:
                C[0    ] = alpha * Cb[0] + beta * C[0    ];
                C[1*ldc] = alpha * Cb[1] + beta * C[1*ldc];
                C[2*ldc] = alpha * Cb[2] + beta * C[2*ldc];
                break;
        case 2:
                C[0    ] = alpha * Cb[0] + beta * C[0    ];
                C[1*ldc] = alpha * Cb[1] + beta * C[1*ldc];
                break;
        case 1:
                C[0    ] = alpha * Cb[0] + beta * C[0    ];
                break;
        case 0:
                break;
    }
}


extern "C" void
magmablas_dgemm_N_T_64_16_4_16_4(
    double *C, const double *A, const double *B,
    magma_int_t m, magma_int_t n, magma_int_t k,
    magma_int_t lda, magma_int_t ldb, magma_int_t ldc,
    double alpha, double beta )
{
    dim3 threads( 16, 4 );
    dim3 grid( (m - 1)/64 + 1, (n - 1)/16 + 1 );
    dgemm_kernel_N_T_64_16_4_16_4<<< grid, threads, 0, magma_stream >>>
        ( C, A, B, m, n, k, lda, ldb, ldc, alpha, beta );
}
