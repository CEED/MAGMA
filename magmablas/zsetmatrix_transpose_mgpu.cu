#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c
       @author Ichitaro Yamazaki
*/
#include "common_magma.h"
#define PRECISION_z
#include "commonblas.h"

//
//    m, n - dimensions in the source (input) matrix.
//             This routine copies the ha matrix from the CPU
//             to dat on the GPU. In addition, the output matrix
//             is transposed. The routine uses a buffer of size
//             2*lddb*nb pointed to by dB (lddb > m) on the GPU. 
//             Note that lda >= m and lddat >= n.
//
extern "C" void 
magmablas_zsetmatrix_transpose_mgpu(
                  magma_int_t ngpus, magma_queue_t stream[][2],
                  const magmaDoubleComplex *ha,  magma_int_t lda, 
                  magmaDoubleComplex       *dat[], magma_int_t ldda, 
                  magmaDoubleComplex       *db[],  magma_int_t lddb,
                  magma_int_t m, magma_int_t n, magma_int_t nb)
{
#define   A(j)    (ha       + (j)*lda)
#define  dB(d, j) (db[(d)]  + (j)*nb*lddb)
#define dAT(d, j) (dat[(d)] + (j)*nb)
    magma_int_t nstreams = 2, d, j, j_local, id, ib;

    /* Quick return */
    if ( (m == 0) || (n == 0) )
        return;

    if (lda < m || ngpus*ldda < n || lddb < m){
        printf( "Wrong arguments in magmablas_zsetmatrix_transpose_mgpu (%d<%d), (%d*%d<%d), or (%d<%d).\n",
                (int) lda, (int) m, (int) ngpus, (int) ldda, (int) n, (int) lddb, (int) m );
        return;
    }
    
    /* Move data from CPU to GPU by block columns and transpose it */
    for(j=0; j<n; j+=nb){
       d       = (j/nb)%ngpus;
       j_local = (j/nb)/ngpus;
       id      = j_local%nstreams;
       magma_setdevice(d);

       ib = min(n-j, nb);
       magma_zsetmatrix_async( m, ib,
                               A(j),      lda,
                               dB(d, id), lddb, 
                               stream[d][id] );

       magmablas_ztranspose_stream( m, ib, dB(d,id), lddb, dAT(d,j_local), ldda, stream[d][id] );
    }
}
