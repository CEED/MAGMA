#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c

*/
#include "common_magma.h"


/**
    Purpose
    -------
    ZGEQR2 computes a QR factorization of a complex m by n matrix A:
    A = Q * R.

    Arguments
    ---------
    @param[in]
    m       INTEGER
            The number of rows of the matrix A.  M >= 0.

    @param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.

    @param[in,out]
    A       COMPLEX*16 array, dimension (LDA,N)
            On entry, the m by n matrix A.
            On exit, the elements on and above the diagonal of the array
            contain the min(m,n) by n upper trapezoidal matrix R (R is
            upper triangular if m >= n); the elements below the diagonal,
            with the array TAU, represent the unitary matrix Q as a
            product of elementary reflectors (see Further Details).

    @param[in]
    lda     INTEGER
            The leading dimension of the array A.  LDA >= max(1,M).

    @param[out]
    tau     COMPLEX*16 array, dimension (min(M,N))
            The scalar factors of the elementary reflectors (see Further
            Details).

    @param
    work    (workspace) DOUBLE_PRECISION array, dimension (N)

    @param[out]
    info    INTEGER
      -     = 0: successful exit
      -     < 0: if INFO = -i, the i-th argument had an illegal value

    Further Details
    ---------------
    The matrix Q is represented as a product of elementary reflectors

       Q = H(1) H(2) . . . H(k), where k = min(m,n).

    Each H(i) has the form

       H(i) = I - tau * v * v'

    where tau is a complex scalar, and v is a complex vector with
    v(1:i-1) = 0 and v(i) = 1; v(i+1:m) is stored on exit in A(i+1:m,i),
    and tau in TAU(i).

    @ingroup magma_zgeqrf_aux
    ********************************************************************/
extern "C" magma_int_t
magma_zgeqr2_gpu(
    magma_int_t m, magma_int_t n,
    magmaDoubleComplex *dA, magma_int_t ldda,
    magmaDoubleComplex *dtau, double *dwork,
    magma_int_t *info)
{
    #define  da_ref(a_1,a_2) ( dA+(a_2)*(ldda) + (a_1))
    
    magma_int_t i, k;

    *info = 0;
    if (m < 0) {
        *info = -1;
    } else if (n < 0) {
        *info = -2;
    } else if (ldda < max(1,m)) {
        *info = -4;
    }
    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return *info;
    }

    /* Compute the norms of the trailing columns */
    k = min(m,n);

    magmaDoubleComplex *Aks;
    magma_zmalloc( &Aks, k );    

    for (i = 0; i < k; ++i) {

        /*  Generate elementary reflector H(i) to annihilate A(i+1:m,i) */
        magma_zlarfg_gpu(m-i, da_ref(i, i), da_ref(min(i+1,m), i), dtau+i, dwork, &Aks[i]);

        if (n-i-1>0)
           /* Apply H(i)' to A(i:m,i+1:n) from the left */
           magma_zlarf_gpu(m-i, n-i-1, da_ref(i, i), dtau+i, da_ref(i, i+1), ldda,
                           dwork);
    }

    if (n >= m)
       magma_zcopymatrix( 1, k-1, Aks, 1, da_ref(0, 0), ldda+1 );
    else
       magma_zcopymatrix( 1, k, Aks, 1, da_ref(0, 0), ldda+1 );

    magma_free(Aks);
    return *info;
} /* magma_zgeqr2 */
