#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c


       @author Mark Gates
*/
#include "common_magma.h"

#define NB 64


// each thread block does one NB x n block row of A.
// each thread does one row, starting from left edge and moving right.
__global__ void
zlascl_full(
    int m, int n, double mul,
    magmaDoubleComplex* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    A += ind;
    if (ind < m) {
        for (int j=0; j < n; j++ )
            A[j*lda] *= mul;
    }
}


// each thread block does one NB x n block row of A.
// each thread does one row, starting from left edge and moving right to diagonal.
__global__ void
zlascl_lower(
    int m, int n, double mul,
    magmaDoubleComplex* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    int break_d = (ind < n) ? ind : n-1;

    A += ind;
    if (ind < m) {
        for (int j=0; j <= break_d; j++ )
            A[j*lda] *= mul;
    }
}


// each thread block does one NB x n block row of A.
// each thread does one row, starting from right edge and moving left to diagonal.
__global__ void
zlascl_upper(
    int m, int n, double mul,
    magmaDoubleComplex* A, int lda)
{
    int ind = blockIdx.x * NB + threadIdx.x;

    A += ind;
    if (ind < m) {
        for (int j=n-1; j >= ind; j--)
            A[j*lda] *= mul;
    }
}


/**
    Purpose
    -------
    ZLASCL multiplies the M by N complex matrix A by the real scalar
    CTO/CFROM.  This is done without over/underflow as long as the final
    result CTO*A(I,J)/CFROM does not over/underflow. TYPE specifies that
    A may be full, upper triangular, lower triangular.

    Arguments
    ---------
    @param[in]
    type    magma_type_t
            TYPE indices the storage type of the input matrix A.
            = MagmaFull:   full matrix.
            = MagmaLower:  lower triangular matrix.
            = MagmaUpper:  upper triangular matrix.
            Other formats that LAPACK supports, MAGMA does not currently support.

    @param[in]
    kl      INTEGER
            Unused, for LAPACK compatability.

    @param[in]
    ku      KU is INTEGER
            Unused, for LAPACK compatability.

    @param[in]
    cfrom   DOUBLE PRECISION

    @param[in]
    cto     DOUBLE PRECISION
    \n
            The matrix A is multiplied by CTO/CFROM. A(I,J) is computed
            without over/underflow if the final result CTO*A(I,J)/CFROM
            can be represented without over/underflow.
            CFROM must be nonzero. CFROM and CTO must not be NAN.

    @param[in]
    m       INTEGER
            The number of rows of the matrix A.  M >= 0.

    @param[in]
    n       INTEGER
            The number of columns of the matrix A.  N >= 0.

    @param[in,out]
    dA      COMPLEX*16 array, dimension (LDDA,N)
            The matrix to be multiplied by CTO/CFROM.  See TYPE for the
            storage type.

    @param[in]
    ldda    INTEGER
            The leading dimension of the array A.  LDDA >= max(1,M).

    @param[out]
    info    INTEGER
      -     = 0:  successful exit
      -     < 0:  if INFO = -i, the i-th argument had an illegal value.
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_zaux2
    ********************************************************************/
extern "C" void
magmablas_zlascl_q(
    magma_type_t type, magma_int_t kl, magma_int_t ku,
    double cfrom, double cto,
    magma_int_t m, magma_int_t n,
    magmaDoubleComplex_ptr dA, magma_int_t ldda,
    magma_queue_t queue,
    magma_int_t *info )
{
    *info = 0;
    if ( type != MagmaLower && type != MagmaUpper && type != MagmaFull )
        *info = -1;
    else if ( cfrom == 0 || isnan(cfrom) )
        *info = -4;
    else if ( isnan(cto) )
        *info = -5;
    else if ( m < 0 )
        *info = -6;
    else if ( n < 0 )
        *info = -3;
    else if ( ldda < max(1,m) )
        *info = -7;
    
    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return;  //info;
    }
    
    dim3 threads( NB );
    dim3 grid( magma_ceildiv( m, NB ) );
    
    double smlnum, bignum, cfromc, ctoc, cto1, cfrom1, mul;
    magma_int_t done = false;
    
    // Uses over/underflow procedure from LAPACK zlascl
    // Get machine parameters
    smlnum = lapackf77_dlamch("s");
    bignum = 1 / smlnum;
    
    cfromc = cfrom;
    ctoc   = cto;
    int cnt = 0;
    while( ! done ) {
        cfrom1 = cfromc*smlnum;
        if ( cfrom1 == cfromc ) {
            // cfromc is an inf.  Multiply by a correctly signed zero for
            // finite ctoc, or a nan if ctoc is infinite.
            mul  = ctoc / cfromc;
            done = true;
            cto1 = ctoc;
        }
        else {
            cto1 = ctoc / bignum;
            if ( cto1 == ctoc ) {
                // ctoc is either 0 or an inf.  In both cases, ctoc itself
                // serves as the correct multiplication factor.
                mul  = ctoc;
                done = true;
                cfromc = 1;
            }
            else if ( fabs(cfrom1) > fabs(ctoc) && ctoc != 0 ) {
                mul  = smlnum;
                done = false;
                cfromc = cfrom1;
            }
            else if ( fabs(cto1) > fabs(cfromc) ) {
                mul  = bignum;
                done = false;
                ctoc = cto1;
            }
            else {
                mul  = ctoc / cfromc;
                done = true;
            }
        }
        
        if (type == MagmaLower) {
            zlascl_lower <<< grid, threads, 0, queue->hip_stream() >>> (m, n, mul, dA, ldda);
        }
        else if (type == MagmaUpper) {
            zlascl_upper <<< grid, threads, 0, queue->hip_stream() >>> (m, n, mul, dA, ldda);
        }
        else if (type == MagmaFull) {
            zlascl_full  <<< grid, threads, 0, queue->hip_stream() >>> (m, n, mul, dA, ldda);
        }
     
        cnt += 1;
    }
}


/**
    @see magmablas_zlascl_q
    @ingroup magma_zaux2
    ********************************************************************/
extern "C" void
magmablas_zlascl(
    magma_type_t type, magma_int_t kl, magma_int_t ku,
    double cfrom, double cto,
    magma_int_t m, magma_int_t n,
    magmaDoubleComplex_ptr dA, magma_int_t ldda,
    magma_int_t *info )
{
    magmablas_zlascl_q( type, kl, ku, cfrom, cto, m, n, dA, ldda, magmablasGetQueue(), info );
}
