#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c

*/
#include "common_magma.h"

#define zlascl_bs 64


__global__ void
l_zlascl (int m, int n, double mul, magmaDoubleComplex* A, int lda){
    int ind =  blockIdx.x * zlascl_bs + threadIdx.x ;

    int break_d = (ind < n)? ind: n-1;

    A += ind;
    if (ind < m)
       for(int j=0; j<=break_d; j++ )
           A[j*lda] *= mul;
}

__global__ void
u_zlascl (int m, int n, double mul, magmaDoubleComplex* A, int lda){
    int ind =  blockIdx.x * zlascl_bs + threadIdx.x ;

    A += ind;
    if (ind < m)
      for(int j=n-1; j>= ind; j--)
         A[j*lda] *= mul;
}


extern "C" void
magmablas_zlascl(magma_type_t type, magma_int_t kl, magma_int_t ku, 
                 double cfrom, double cto,
                 magma_int_t m, magma_int_t n, 
                 magmaDoubleComplex *A, magma_int_t lda, magma_int_t *info )
{
    int blocks;
    if (m % zlascl_bs==0)
        blocks = m/ zlascl_bs;
    else
        blocks = m/ zlascl_bs + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(zlascl_bs, 1, 1);

    /* To do : implment the accuracy procedure */
    double mul = cto / cfrom;

    if (type == MagmaLower)  
       l_zlascl <<< grid, threads, 0, magma_stream >>> (m, n, mul, A, lda);
    else if (type == MagmaUpper)
       u_zlascl <<< grid, threads, 0, magma_stream >>> (m, n, mul, A, lda);  
    else {
       printf("Only type L and U are available in zlascl. Exit.\n");
       exit(1);
    }
}
