#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal z -> s d c

*/
#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 512
#else
   #define BLOCK_SIZE 768
#endif

extern "C" void
magma_zlarfbx_gpu(int m, int k, hipDoubleComplex *V, int ldv,
                  hipDoubleComplex *dT, int ldt, hipDoubleComplex *c,
                  hipDoubleComplex *dwork);

extern "C" void
magma_zlarfgtx_gpu(int n, hipDoubleComplex *dx0, hipDoubleComplex *dx,
                   hipDoubleComplex *dtau, double *dxnorm,
                   hipDoubleComplex *dA, int it,
                   hipDoubleComplex *V, int ldv, hipDoubleComplex *T, int ldt,
                   hipDoubleComplex *dwork);

extern "C" void
magmablas_dznrm2_adjust(int k, double *xnorm, hipDoubleComplex *c);
    
extern "C" void
magmablas_zgemm_reduce(magma_int_t m, magma_int_t n, magma_int_t k,
                       hipDoubleComplex alpha, const hipDoubleComplex *d_A, magma_int_t lda,
                       const hipDoubleComplex *d_B, magma_int_t ldb,
                       hipDoubleComplex beta,        hipDoubleComplex *d_C, magma_int_t ldc );

__global__ void 
magma_ztrmv_kernel2(const hipDoubleComplex *T, int ldt,
                    hipDoubleComplex *v, hipDoubleComplex *y, hipDoubleComplex *tau);

__global__ void 
magma_zgemv_kernel3(int m, const hipDoubleComplex * __restrict__ V, int ldv,
                    hipDoubleComplex *c, hipDoubleComplex *dwork,
                    hipDoubleComplex *tau);


extern "C" magma_int_t
magma_zlarfb2_gpu( magma_int_t m, magma_int_t n, magma_int_t k,
                   const hipDoubleComplex *dV,    magma_int_t ldv,
                   const hipDoubleComplex *dT,    magma_int_t ldt,
                   hipDoubleComplex *dC,          magma_int_t ldc,
                   hipDoubleComplex *dwork,       magma_int_t ldwork )
{
    hipDoubleComplex c_zero    = MAGMA_Z_ZERO;
    hipDoubleComplex c_one     = MAGMA_Z_ONE;
    hipDoubleComplex c_neg_one = MAGMA_Z_NEG_ONE;

    if (m <= 0 || n <= 0)
        return MAGMA_SUCCESS;

    // W = C^H V
    // magma_zgemm( MagmaConjTrans, MagmaNoTrans,
    magmablas_zgemm_reduce(
                           n, k, m,
                           c_one,  dC,    ldc,
                           dV,    ldv,
                           c_zero, dwork, ldwork);

    // W = W T^H = C^H V T^H
    magma_ztrmm( MagmaRight, MagmaUpper, MagmaNoTrans, MagmaNonUnit,
                 n, k,
                 c_one, dT,    ldt,
                 dwork, ldwork);

    // C = C - V W^H = C - V T V^H C = (I - V T V^H) C = H C
    magma_zgemm( MagmaNoTrans, MagmaConjTrans,
                 m, n, k,
                 c_neg_one, dV,    ldv,
                 dwork, ldwork,
                 c_one,     dC,    ldc);

    return MAGMA_SUCCESS;
}

//////////////////////////////////////////////////////////////////////////////

__global__ void
magma_zgemv_kernel1(int m, const hipDoubleComplex * __restrict__ V, int ldv,
                    const hipDoubleComplex * __restrict__ c,
                    hipDoubleComplex *dwork);
__global__ void
magma_zgemv_kernel2(int m, int n, const hipDoubleComplex * __restrict__ V, int ldv,
                    const hipDoubleComplex * __restrict__ x, hipDoubleComplex *c);
__global__ void 
magma_ztrmv_tkernel(hipDoubleComplex *T, int ldt, hipDoubleComplex *v,
                                    hipDoubleComplex *y);
__global__ void
magma_dznrm2_adjust_kernel(double *xnorm, hipDoubleComplex *c);

extern "C" void
magma_zlarfgx_gpu(int n, hipDoubleComplex *dx0, hipDoubleComplex *dx,
                  hipDoubleComplex *dtau, double *dxnorm,
                  hipDoubleComplex *dA, int it);

extern "C" magma_int_t
magma_zgeqr2x4_gpu(magma_int_t *m, magma_int_t *n, hipDoubleComplex *dA, 
                   magma_int_t *ldda, hipDoubleComplex *dtau,
                   hipDoubleComplex *dT, hipDoubleComplex *ddA,
                   double *dwork, magma_int_t *info, hipStream_t stream)
{
/*  -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

    Purpose   
    =======   
    ZGEQR2 computes a QR factorization of a complex m by n matrix A:   
    A = Q * R.

    This expert routine requires two more arguments than the standard 
    zgeqr2, namely, dT and ddA, explained below. The storage for A is 
    also not as in the LAPACK's zgeqr2 routine (see below). 

    The first is used to output the triangular 
    n x n factor T of the block reflector used in the factorization. 
    The second holds the diagonal nxn blocks of A, i.e., the diagonal
    submatrices of R. This routine implements the left looking QR.

    This version adds internal blocking.

    Arguments   
    =========   
    M       (input) INTEGER   
            The number of rows of the matrix A.  M >= 0.   

    N       (input) INTEGER   
            The number of columns of the matrix A.  N >= 0.   

    A       (input/output) COMPLEX_16 array, dimension (LDA,N)   
            On entry, the m by n matrix A.   
            On exit, the unitary matrix Q as a
            product of elementary reflectors (see Further Details).

            the elements on and above the diagonal of the array   
            contain the min(m,n) by n upper trapezoidal matrix R (R is   
            upper triangular if m >= n); the elements below the diagonal,   
            with the array TAU, represent the unitary matrix Q as a   
            product of elementary reflectors (see Further Details).   

    LDA     (input) INTEGER   
            The leading dimension of the array A.  LDA >= max(1,M).   

    TAU     (output) COMPLEX_16 array, dimension (min(M,N))   
            The scalar factors of the elementary reflectors (see Further   
            Details).   

    dT      (output) COMPLEX_16 array, dimension N x N.
            Stores the triangular N x N factor T of the block reflector 
            used in the factorization. The lower triangular part is 0.

    ddA     (output) COMPLEX_16 array, dimension N x N.
            Stores the elements of the upper N x N diagonal block of A.
            LAPACK stores this array in A. There are 0s below the diagonal.

    RWORK   (workspace) DOUBLE_PRECISION array, dimension (3 N)

    INFO    (output) INTEGER   
            = 0: successful exit   
            < 0: if INFO = -i, the i-th argument had an illegal value   

    Further Details   
    ===============   
    The matrix Q is represented as a product of elementary reflectors   

       Q = H(1) H(2) . . . H(k), where k = min(m,n).   

    Each H(i) has the form   

       H(i) = I - tau * v * v'   

    where tau is a complex scalar, and v is a complex vector with   
    v(1:i-1) = 0 and v(i) = 1; v(i+1:m) is stored on exit in A(i+1:m,i),   
    and tau in TAU(i).   
    =====================================================================    */

    #define da_ref(a_1,a_2) ( dA+(a_2)*(*ldda) + (a_1))
    #define dt_ref(a_1,a_2) ( dT+(a_2)*(k) + (a_1))
    #define BS 32

    static magma_int_t i, k;

    double *dnorm = (double *)dwork;
    hipDoubleComplex *work = (hipDoubleComplex *)(dwork+2*(*n));

    magma_queue_t cstream;
    magmablasGetKernelStream(&cstream);
    magmablasSetKernelStream(stream);

    *info = 0;
    if (*m < 0) {
        *info = -1;
    } else if (*n < 0) {
        *info = -2;
    } else if (*ldda < max(1,*m)) {
        *info = -4;
    }
    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return *info;
    }

    /* Compute the norms of the trailing columns */
    k = min(*m,*n);
    magmablas_dznrm2(*m, k, da_ref(0,0), *ldda, dnorm);

    for (int b=0; b < k; b += BS) {
        for (i = b; i < min(k, b+BS); ++i) {

            /*   Apply H' to A(:,i) from the left                           */    
            if ( i-b > 0){
                magma_zgemv_kernel3<<< i-1, BLOCK_SIZE, 0, magma_stream >>>( *m-i+1, da_ref(i-1,0), *ldda,
                                                    da_ref(i-1, i-1), work, dtau+i-1);
                magma_ztrmv_kernel2<<< i-1, i-1, 0, magma_stream >>>( dt_ref(0,0), k, work,
                                                    dt_ref(0,i-1), dtau+i-1);

                /* dwork = V' c                   */
                magma_zgemv_kernel1<<< i-b, BLOCK_SIZE, 0, magma_stream >>>(*m-b, da_ref(b, b), 
                             *ldda, da_ref(b,i), work);

                /* dwork = T' work                */
                magma_ztrmv_tkernel<<< i-b, i-b, 0, magma_stream >>>(dt_ref(b,b), k, work, work+i-b);

                /* c = c - V work                 */
                dim3  blocks3( (*m-b + BLOCK_SIZE-1) / BLOCK_SIZE );
                dim3 threads3( BLOCK_SIZE );
                magma_zgemv_kernel2<<< blocks3, threads3, 0, magma_stream >>>(*m-b, i-b, da_ref(b,b), *ldda, 
                                   work+i-b, da_ref(b, i));
            }

            /*   Adjust the dnorm[i] to hold the norm of A(i:m,i)           */ 
            if ( i > 0 )
                magma_dznrm2_adjust_kernel<<< 1, i, 0, magma_stream >>> (dnorm+i, da_ref(0, i));
            
            /*  Generate elementary reflector H(i) to annihilate A(i+1:m,i) 
                1. 1 is not yet put on the diagonal of A
                2. Elements above the diagonal are copied in ddA and
                   the ones in A are set to zero                                         
                3. update T                                                 */
            magma_zlarfgx_gpu(*m-i, da_ref(i, i), da_ref(min(i+1,*m),i), dtau+i, 
                              dnorm+i, ddA + i + i*(*n), i);

            if (i==0){
              hipDoubleComplex tt = MAGMA_Z_ONE;
              magmablas_zlacpy(MagmaUpperLower, 1, 1, dtau, 1, dt_ref(0,0), 1);
              magma_zsetmatrix(1,1, &tt,1, da_ref(i, i),1);
            }
/*
            else
             {
                // Compute the i-th column of T.
                //   Set da_ref(i, i) = 1.                                    
                magma_zgemv_kernel3<<< i, BLOCK_SIZE, 0, magma_stream >>>( *m-i, da_ref(i,0), *ldda, 
                                          da_ref(i, i), work, dtau+i);
                magma_ztrmv_kernel2<<< i, i, 0, magma_stream          >>>( dt_ref(0,0), k, work, 
                                                          dt_ref(0,i), dtau+i);
              }
*/

        }
        magma_zgemv_kernel3<<< i-1, BLOCK_SIZE, 0, magma_stream >>>( *m-i+1, da_ref(i-1,0), *ldda,
                                                    da_ref(i-1, i-1), work, dtau+i-1);
        magma_ztrmv_kernel2<<< i-1, i-1, 0, magma_stream >>>( dt_ref(0,0), k, work,
                                                    dt_ref(0,i-1), dtau+i-1);

        
        /* Apply the transformations to the trailing matrix. */
        //magma_zlarfb2_gpu( MagmaLeft, MagmaConjTrans, MagmaForward, MagmaColumnwise,
        magma_zlarfb2_gpu(
                           *m-b, k-i, BS,
                           da_ref(b, b), *ldda, dT+b+b*k, k,
                           da_ref(b, i), *ldda, work, k-i);
    }

    magmablasSetKernelStream(cstream);

    return *info;
} /* magma_zgeqr2 */
