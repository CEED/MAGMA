#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c

*/
#include "common_magma.h"

//#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 512
//#else
//   #define BLOCK_SIZE 768
//#endif

__global__ void 
magma_ztrmv_kernel2(const magmaDoubleComplex *T, int ldt,
                    magmaDoubleComplex *v, magmaDoubleComplex *y, magmaDoubleComplex *tau);

__global__ void 
magma_zgemv_kernel3(int m, const magmaDoubleComplex * __restrict__ V, int ldv,
                    magmaDoubleComplex *c, magmaDoubleComplex *dwork,
                    magmaDoubleComplex *tau);


//////////////////////////////////////////////////////////////////////////////

__global__ void
magma_zgemv_kernel1(int m, const magmaDoubleComplex * __restrict__ V, int ldv,
                    const magmaDoubleComplex * __restrict__ c,
                    magmaDoubleComplex *dwork);
__global__ void
magma_zgemv_kernel2(int m, int n, const magmaDoubleComplex * __restrict__ V, int ldv,
                    const magmaDoubleComplex * __restrict__ x, magmaDoubleComplex *c);
__global__ void 
magma_ztrmv_tkernel(magmaDoubleComplex *T, int ldt, magmaDoubleComplex *v,
                                    magmaDoubleComplex *y);
__global__ void
magma_dznrm2_adjust_kernel(double *xnorm, magmaDoubleComplex *c);

extern "C" magma_int_t
magma_zgeqr2x4_gpu(magma_int_t *m, magma_int_t *n, magmaDoubleComplex *dA, 
                   magma_int_t *ldda, magmaDoubleComplex *dtau,
                   magmaDoubleComplex *dT, magmaDoubleComplex *ddA,
                   double *dwork, magma_int_t *info, magma_queue_t stream)
{
/*  -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

    Purpose   
    =======   
    ZGEQR2 computes a QR factorization of a complex m by n matrix A:   
    A = Q * R.

    This expert routine requires two more arguments than the standard 
    zgeqr2, namely, dT and ddA, explained below. The storage for A is 
    also not as in the LAPACK's zgeqr2 routine (see below). 

    The first is used to output the triangular 
    n x n factor T of the block reflector used in the factorization. 
    The second holds the diagonal nxn blocks of A, i.e., the diagonal
    submatrices of R. This routine implements the left looking QR.

    This version adds internal blocking.

    Arguments   
    =========   
    M       (input) INTEGER   
            The number of rows of the matrix A.  M >= 0.   

    N       (input) INTEGER   
            The number of columns of the matrix A.  N >= 0.   

    A       (input/output) COMPLEX_16 array, dimension (LDA,N)   
            On entry, the m by n matrix A.   
            On exit, the unitary matrix Q as a
            product of elementary reflectors (see Further Details).

            the elements on and above the diagonal of the array   
            contain the min(m,n) by n upper trapezoidal matrix R (R is   
            upper triangular if m >= n); the elements below the diagonal,   
            with the array TAU, represent the unitary matrix Q as a   
            product of elementary reflectors (see Further Details).   

    LDA     (input) INTEGER   
            The leading dimension of the array A.  LDA >= max(1,M).   

    TAU     (output) COMPLEX_16 array, dimension (min(M,N))   
            The scalar factors of the elementary reflectors (see Further   
            Details).   

    dT      (output) COMPLEX_16 array, dimension N x N.
            Stores the triangular N x N factor T of the block reflector 
            used in the factorization. The lower triangular part is 0.

    ddA     (output) COMPLEX_16 array, dimension N x N.
            Stores the elements of the upper N x N diagonal block of A.
            LAPACK stores this array in A. There are 0s below the diagonal.

    RWORK   (workspace) DOUBLE_PRECISION array, dimension (3 N)

    INFO    (output) INTEGER   
            = 0: successful exit   
            < 0: if INFO = -i, the i-th argument had an illegal value   

    Further Details   
    ===============   
    The matrix Q is represented as a product of elementary reflectors   

       Q = H(1) H(2) . . . H(k), where k = min(m,n).   

    Each H(i) has the form   

       H(i) = I - tau * v * v'   

    where tau is a complex scalar, and v is a complex vector with   
    v(1:i-1) = 0 and v(i) = 1; v(i+1:m) is stored on exit in A(i+1:m,i),   
    and tau in TAU(i).   
    =====================================================================    */

    #define da_ref(a_1,a_2) ( dA+(a_2)*(*ldda) + (a_1))
    #define dt_ref(a_1,a_2) ( dT+(a_2)*(k) + (a_1))
    #define BS 32

    magma_int_t i, k;

    double *dnorm = (double *)dwork;
    magmaDoubleComplex *work = (magmaDoubleComplex *)(dwork+2*(*n));

    magma_queue_t cstream;
    magmablasGetKernelStream(&cstream);
    magmablasSetKernelStream(stream);

    *info = 0;
    if (*m < 0) {
        *info = -1;
    } else if (*n < 0) {
        *info = -2;
    } else if (*ldda < max(1,*m)) {
        *info = -4;
    }
    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return *info;
    }

    /* Compute the norms of the trailing columns */
    k = min(*m,*n);
    magmablas_dznrm2_cols(*m, k, da_ref(0,0), *ldda, dnorm);

    for (magma_int_t b=0; b < k; b += BS) {
        for (i = b; i < min(k, b+BS); ++i) {

            /*   Apply H' to A(:,i) from the left                           */    
            if ( i-b > 0){
                magma_zgemv_kernel3<<< i-1, BLOCK_SIZE, 0, magma_stream >>>( *m-i+1, da_ref(i-1,0), *ldda,
                                                    da_ref(i-1, i-1), work, dtau+i-1);
                magma_ztrmv_kernel2<<< i-1, i-1, 0, magma_stream >>>( dt_ref(0,0), k, work,
                                                    dt_ref(0,i-1), dtau+i-1);

                /* dwork = V' c                   */
                magma_zgemv_kernel1<<< i-b, BLOCK_SIZE, 0, magma_stream >>>(*m-b, da_ref(b, b), 
                             *ldda, da_ref(b,i), work);

                /* dwork = T' work                */
                magma_ztrmv_tkernel<<< i-b, i-b, 0, magma_stream >>>(dt_ref(b,b), k, work, work+i-b);

                /* c = c - V work                 */
                dim3  blocks3( (*m-b + BLOCK_SIZE-1) / BLOCK_SIZE );
                dim3 threads3( BLOCK_SIZE );
                magma_zgemv_kernel2<<< blocks3, threads3, 0, magma_stream >>>(*m-b, i-b, da_ref(b,b), *ldda, 
                                   work+i-b, da_ref(b, i));
            }

            /*   Adjust the dnorm[i] to hold the norm of A(i:m,i)           */ 
            if ( i > 0 )
                magma_dznrm2_adjust_kernel<<< 1, i, 0, magma_stream >>> (dnorm+i, da_ref(0, i));
            
            /*  Generate elementary reflector H(i) to annihilate A(i+1:m,i) 
                1. 1 is not yet put on the diagonal of A
                2. Elements above the diagonal are copied in ddA and
                   the ones in A are set to zero                                         
                3. update T                                                 */
            magma_zlarfgx_gpu(*m-i, da_ref(i, i), da_ref(min(i+1,*m),i), dtau+i, 
                              dnorm+i, ddA + i + i*(*n), i);

            if (i==0){
              magmaDoubleComplex tt = MAGMA_Z_ONE;
              magmablas_zlacpy(MagmaUpperLower, 1, 1, dtau, 1, dt_ref(0,0), 1);
              magma_zsetmatrix(1,1, &tt,1, da_ref(i, i),1);
            }
/*
            else
             {
                // Compute the i-th column of T.
                //   Set da_ref(i, i) = 1.                                    
                magma_zgemv_kernel3<<< i, BLOCK_SIZE, 0, magma_stream >>>( *m-i, da_ref(i,0), *ldda, 
                                          da_ref(i, i), work, dtau+i);
                magma_ztrmv_kernel2<<< i, i, 0, magma_stream          >>>( dt_ref(0,0), k, work, 
                                                          dt_ref(0,i), dtau+i);
              }
*/

        }
        magma_zgemv_kernel3<<< i-1, BLOCK_SIZE, 0, magma_stream >>>( *m-i+1, da_ref(i-1,0), *ldda,
                                                    da_ref(i-1, i-1), work, dtau+i-1);
        magma_ztrmv_kernel2<<< i-1, i-1, 0, magma_stream >>>( dt_ref(0,0), k, work,
                                                    dt_ref(0,i-1), dtau+i-1);

        
        /* Apply the transformations to the trailing matrix. */
        //magma_zlarfb2_gpu( MagmaLeft, MagmaConjTrans, MagmaForward, MagmaColumnwise,
        magma_zlarfb2_gpu(
                           *m-b, k-i, BS,
                           da_ref(b, b), *ldda, dT+b+b*k, k,
                           da_ref(b, i), *ldda, work, k-i);
    }

    magmablasSetKernelStream(cstream);

    return *info;
} /* magma_zgeqr2 */
