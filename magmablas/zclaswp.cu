#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions mixed zc -> ds

*/
#include "common_magma.h"

#define NB 64

// TODO check precision, as in zlag2c?

__global__ void
zclaswp_kernel(int n, magmaDoubleComplex *A, int lda, magmaFloatComplex *SA, int m, const magma_int_t *ipiv)
{
    int ind = blockIdx.x*NB + threadIdx.x;
    int newind;
    magmaFloatComplex res;
    
    if (ind < m) {
        SA   += ind;
        ipiv += ind;
        
        newind = ipiv[0];
        
        for (int i=0; i < n; i++) {
            res = MAGMA_C_MAKE( (float)hipCreal(A[newind+i*lda]),
                                (float)hipCimag(A[newind+i*lda]) );
            SA[i*lda] = res; 
        }
    }
}

__global__ void
zclaswp_inv_kernel(int n, magmaDoubleComplex *A, int lda, magmaFloatComplex *SA, int m, const magma_int_t *ipiv)
{
    int ind = blockIdx.x*NB + threadIdx.x;
    int newind;
    magmaDoubleComplex res;

    if (ind < m) {
        A    += ind;
        ipiv += ind;

        newind = ipiv[0];

        for (int i=0; i < n; i++) {
            res = MAGMA_Z_MAKE( (double)hipCrealf(SA[newind+i*lda]),
                                (double)hipCimagf(SA[newind+i*lda]) );
            A[i*lda] = res;
        }
    }
}


/**
    Purpose
    -------
    Row i of  A is cast to single precision in row ipiv[i] of SA (incx > 0), or
    row i of SA is cast to double precision in row ipiv[i] of  A (incx < 0),
    for 0 <= i < M.

    @param[in]
    n       INTEGER.
            On entry, N specifies the number of columns of the matrix A.

    @param[in,out]
    A       DOUBLE PRECISION array on the GPU, dimension (LDA,N)
            On entry, the M-by-N matrix to which the row interchanges will be applied.
            TODO update docs

    @param[in]
    lda     INTEGER.
            LDA specifies the leading dimension of A.

    @param[in,out]
    SA      REAL array on the GPU, dimension (LDA,N)
            On exit, the single precision, permuted matrix.
            TODO update docs
        
    @param[in]
    m       The number of rows to be interchanged.

    @param[in]
    ipiv    INTEGER array on the GPU, dimension (M)
            The vector of pivot indices. Row i of A is cast to single 
            precision in row ipiv[i] of SA, for 0 <= i < m. 

    @param[in]
    incx    INTEGER
            If INCX is negative, the pivots are applied in reverse order,
            otherwise in straight-forward order.
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_zaux2
    ********************************************************************/
extern "C" void
magmablas_zclaswp_q(
    magma_int_t n,
    magmaDoubleComplex_ptr A, magma_int_t lda,
    magmaFloatComplex_ptr SA, magma_int_t m,
    const magma_int_t *ipiv, magma_int_t incx,
    magma_queue_t queue )
{
    int blocks = magma_ceildiv( m, NB );
    dim3 grid( blocks );
    dim3 threads( NB );

    if (incx >= 0)
        zclaswp_kernel<<< grid, threads, 0, queue >>>(n, A, lda, SA, m, ipiv);
    else
        zclaswp_inv_kernel<<< grid, threads, 0, queue >>>(n, A, lda, SA, m, ipiv);
}


/**
    @see magmablas_zclaswp_q
    @ingroup magma_zaux2
    ********************************************************************/
extern "C" void
magmablas_zclaswp(
    magma_int_t n,
    magmaDoubleComplex_ptr A, magma_int_t lda,
    magmaFloatComplex_ptr SA, magma_int_t m,
    const magma_int_t *ipiv, magma_int_t incx )
{
    magmablas_zclaswp_q( n, A, lda, SA, m, ipiv, incx, magma_stream );
}
