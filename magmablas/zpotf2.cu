#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date
       
       @precisions normal z -> s d c
*/
#include "common_magma.h"

#define PRECISION_z

#define zdotc_max_bs 512  // 512 is max threads for 1.x cards

void zpotf2_zdscal(magma_int_t n, magmaDoubleComplex *x, magma_int_t incx);
void zpotf2_zdotc(magma_int_t n, magmaDoubleComplex *x, magma_int_t incx);

#if defined(PRECISION_z) || defined(PRECISION_c)
void magmablas_zlacgv(magma_int_t n, magmaDoubleComplex *x, magma_int_t incx);
#endif

/**
    Purpose
    -------

    zpotf2 computes the Cholesky factorization of a real symmetric
    positive definite matrix A.

    The factorization has the form
        A = U**H * U,  if UPLO = MagmaUpper, or
        A = L  * L**H, if UPLO = MagmaLower,
    where U is an upper triangular matrix and L is lower triangular.

    This is the unblocked version of the algorithm, calling Level 2 BLAS.

    Arguments
    ---------

    @param[in]
    uplo    magma_uplo_t
            Specifies whether the upper or lower triangular part of the
            symmetric matrix A is stored.
      -     = MagmaUpper:  Upper triangular
      -     = MagmaLower:  Lower triangular

    @param[in]
    n       INTEGER
            The order of the matrix A.  N >= 0 and N <= 512.

    @param[in,out]
    dA      COMPLEX_16 array, dimension (LDDA,N)
            On entry, the symmetric matrix A.  If UPLO = MagmaUpper, the leading
            n by n upper triangular part of A contains the upper
            triangular part of the matrix A, and the strictly lower
            triangular part of A is not referenced.  If UPLO = MagmaLower, the
            leading n by n lower triangular part of A contains the lower
            triangular part of the matrix A, and the strictly upper
            triangular part of A is not referenced.
    \n
            On exit, if INFO = 0, the factor U or L from the Cholesky
            factorization A = U**H * U  or A = L * L**H.

    @param[in]
    ldda    INTEGER
            The leading dimension of the array A.  LDDA >= max(1,N).

    @param[out]
    info    INTEGER
      -     = 0: successful exit
      -     < 0: if INFO = -k, the k-th argument had an illegal value
      -     > 0: if INFO = k, the leading minor of order k is not
                 positive definite, and the factorization could not be
                 completed.

    @ingroup magma_zposv_aux
    ********************************************************************/
extern "C" magma_int_t
magma_zpotf2_gpu(
    magma_uplo_t uplo, magma_int_t n,
    magmaDoubleComplex_ptr dA, magma_int_t ldda,
    magma_int_t *info )
{
#define dA(i_, j_)  (dA + (i_) + (j_)*ldda)

    magma_int_t j;

    *info = 0;
    if ( uplo != MagmaUpper && uplo != MagmaLower) {
        *info = -1;
    } else if (n < 0 || n > zdotc_max_bs) {
        *info = -2;
    } else if (ldda < max(1,n)) {
        *info = -4;
    }

    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return *info;
    }

    // Quick return if possible
    if (n == 0) {
        return *info;
    }

    magmaDoubleComplex alpha = MAGMA_Z_NEG_ONE;
    magmaDoubleComplex beta  = MAGMA_Z_ONE;

    if (uplo == MagmaUpper) {
        for (j = 0; j < n; j++) {
            zpotf2_zdotc(j, dA(0,j), 1); // including zdotc product and update a(j,j)
            if (j < n) {
                #if defined(PRECISION_z) || defined(PRECISION_c)
                magmablas_zlacgv(j, dA(0, j), 1);
                #endif
                magma_zgemv( MagmaTrans, j, n-j-1,
                             alpha, dA(0, j+1), ldda,
                                    dA(0, j),   1,
                             beta,  dA(j, j+1), ldda);

                #if defined(PRECISION_z) || defined(PRECISION_c)
                magmablas_zlacgv(j, dA(0, j), 1);
                #endif
                zpotf2_zdscal(n-j, dA(j,j), ldda);
            }
        }
    }
    else {
        for (j = 0; j < n; j++) {
            zpotf2_zdotc(j, dA(j,0), ldda); // including zdotc product and update a(j,j)
            if (j < n) {
                #if defined(PRECISION_z) || defined(PRECISION_c)
                magmablas_zlacgv(j, dA(j, 0), ldda);
                #endif
                magma_zgemv( MagmaNoTrans, n-j-1, j,
                             alpha, dA(j+1, 0), ldda,
                                    dA(j,0),    ldda,
                             beta,  dA(j+1, j), 1 );

                #if defined(PRECISION_z) || defined(PRECISION_c)
                magmablas_zlacgv(j, dA(j, 0), ldda);
                #endif
                zpotf2_zdscal(n-j, dA(j,j), 1);
            }
        }
    }

    return *info;
}

#define zdscal_bs  32
#define zdotc_bs  512
#define zlacgv_bs 512

// dynamically allocated shared memory, set to size number of threads when the kernel is launched.
// See CUDA Guide B.2.3
extern __shared__ double shared_data[];

__global__ void kernel_zdotc(int n, magmaDoubleComplex *x, int incx, int threadSize)
{
    int tx = threadIdx.x;

    double *sdata = shared_data;

    magmaDoubleComplex res = MAGMA_Z_ZERO;

    if (tx < n) {
        res = x[tx*incx];
    }

    sdata[tx] = MAGMA_Z_REAL(res * MAGMA_Z_CNJG(res));

    __syncthreads();

    for (int s = blockDim.x/2; s > 32; s >>= 1 ) {
        if (tx < s) {
            sdata[tx] += sdata[tx+s];
        }
        __syncthreads();
    }

    if (tx < 32) {
        volatile double* smem = sdata;
        smem[tx] += smem[tx+32];
        smem[tx] += smem[tx+16];
        smem[tx] += smem[tx+8];
        smem[tx] += smem[tx+4];
        smem[tx] += smem[tx+2];
        smem[tx] += smem[tx+1];
    }

    if (tx == 0) {
        double xreal = MAGMA_Z_REAL(x[n*incx]);
        x[n*incx] = MAGMA_Z_MAKE( sqrt(xreal - sdata[0]), 0 );
    }
}

void zpotf2_zdotc(magma_int_t n, magmaDoubleComplex *x, magma_int_t incx)
{
    /*
    Specialized Zdotc
    1) performs zdotc sum = x[0:n-1]*conj(x[0:n-1])
    2) updates x[n] = sqrt(x[n]-sum);

    */
    if (n > zdotc_max_bs) {
        fprintf( stderr, "n = %d > %d is not supported in zpotf2_zdotc\n", (int) n, (int) zdotc_max_bs);
        return;
    }
    int threadSize;

    if (n <= 1024 && n > 512) {
        threadSize = 1024;
    }
    else if (n <= 512 && n > 256 ) {
        threadSize = 512;
    }
    else if (n <= 256 && n > 128) {
        threadSize = 256;
    }
    else if (n <= 128 && n > 64) {
        threadSize = 128;
    }
    else {
        threadSize = 64;
    }

    kernel_zdotc<<< 1, threadSize, threadSize * sizeof(double), magma_stream>>> (n, x, incx, threadSize);
}

__global__ void kernel_zdscal(int n, magmaDoubleComplex *x, int incx)
{
    int id = blockIdx.x * zdscal_bs + threadIdx.x;

    __shared__ magmaDoubleComplex factor;

    if (threadIdx.x == 0) {
        factor = MAGMA_Z_MAKE(1.0/MAGMA_Z_REAL(x[0]), 0.0);
    }

    __syncthreads();

    if ( id < n && id > 0) {
        x[id*incx] = x[id*incx] * factor;
    }
}


void zpotf2_zdscal(magma_int_t n, magmaDoubleComplex *x, magma_int_t incx)
{
    /*
    Specialized Zdscal perform x[1:n-1]/x[0]
    */
    dim3 threads(zdscal_bs, 1, 1);
    int num_blocks = magma_ceildiv( n, zdscal_bs );
    dim3 grid(num_blocks,1);
    kernel_zdscal<<< grid, threads, 0, magma_stream >>> (n, x, incx);
}


#if defined(PRECISION_z) || defined(PRECISION_c)

__global__ void kernel_zlacgv(int n, magmaDoubleComplex *x, int incx)
{
    int id = blockIdx.x * zlacgv_bs + threadIdx.x;

    if ( id < n ) {
        x[id*incx] = MAGMA_Z_CNJG(x[id*incx]);
    }
}


/**
    Purpose
    -------

    ZLACGV conjugates a complex vector of length N.

    Arguments
    ---------

    @param[in]
    n       INTEGER
            The length of the vector X.  N >= 0.

    @param[in,out]
    x       COMPLEX*16 array, dimension (1+(N-1)*abs(INCX))
            On entry, the vector of length N to be conjugated.
            On exit, X is overwritten with conjg(X).

    @param[in]
    incx    INTEGER
            The spacing between successive elements of X.

    @ingroup magma_zaux1
    ********************************************************************/
void magmablas_zlacgv(magma_int_t n, magmaDoubleComplex *x, magma_int_t incx)
{
    dim3 threads(zlacgv_bs, 1, 1);
    int num_blocks = magma_ceildiv( n, zlacgv_bs );
    dim3 grid(num_blocks,1);
    kernel_zlacgv<<< grid, threads, 0, magma_stream >>> (n, x, incx);
}

#endif // defined(PRECISION_z) || defined(PRECISION_c)
