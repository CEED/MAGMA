#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c


       @author Adrien REMY
*/
#include "common_magma.h"


#define block_height  32
#define block_width  4
#define block_length 256
#define NB 64

__global__ void 
magmablas_zelementary_multiplication(
    magma_int_t n,
    magmaDoubleComplex *dA, magma_int_t ldda, 
    magmaDoubleComplex *du, 
    magmaDoubleComplex *dv)
{    
    magma_int_t idx, idy;

    idx = blockIdx.x * blockDim.x + threadIdx.x;
    idy = blockIdx.y * blockDim.y + threadIdx.y;

    if ((idx < n/2)&&(idy < n/2)){

        dA += idx + idy * ldda;

        magmaDoubleComplex a00, a10, a01, a11, b1, b2, b3, b4;
        __shared__ magmaDoubleComplex u1[block_height], u2[block_height], v1[block_width], v2[block_width];

        du += idx;
        dv += idy;

        u1[threadIdx.x]=du[0];
        u2[threadIdx.x]=du[n/2];
        v1[threadIdx.y]=dv[0];
        v2[threadIdx.y]=dv[n/2];

        __syncthreads();

        a00 = dA[0];
        a01 = dA[ldda*n/2];
        a10 = dA[n/2];
        a11 = dA[ldda*n/2+n/2];

        b1 = a00 + a01;
        b2 = a10 + a11;
        b3 = a00 - a01;
        b4 = a10 - a11;

        dA[0] = u1[threadIdx.x] * v1[threadIdx.y] * (b1 + b2);
        dA[ldda*n/2] = u1[threadIdx.x] * v2[threadIdx.y] * (b3 + b4);
        dA[n/2] = u2[threadIdx.x] * v1[threadIdx.y] * (b1 - b2);
        dA[ldda*n/2+n/2] = u2[threadIdx.x] * v2[threadIdx.y] *(b3 - b4);
    }
}


__global__ void 
magmablas_zapply_vector(
    magma_int_t n,
    magmaDoubleComplex *du, magmaDoubleComplex *db)
{
    magma_int_t idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n/2){

        du += idx;
        db += idx;

        magmaDoubleComplex a1,a2;

        a1 = du[0]*db[0];
        a2 = du[n/2]*db[n/2];

        db[0] = a1 + a2;
        db[n/2] = a1 -a2;
    }
}


__global__ void 
magmablas_zapply_transpose_vector(
    magma_int_t n,
    magmaDoubleComplex *du,magmaDoubleComplex *db )
{
    magma_int_t idx;

    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n/2){

        du += idx;
        db += idx;

        magmaDoubleComplex a1,a2;

        a1 = db[0] + db[n/2];
        a2 = db[0] - db[n/2];

        db[0] = du[0]*a1;
        db[n/2] = du[n/2]*a2;
    }
}


/**
    Purpose
    -------
    ZPRBT_MVT compute B = UTB to randomize B
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of values of db.  n >= 0.

    @param[in]
    du     COMPLEX_16 array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in,out]
    db     COMPLEX_16 array, dimension (n)
            The n vector db computed by ZGESV_NOPIV_GPU
            On exit db = du*db
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.
    ********************************************************************/
extern "C" void
magmablas_zprbt_mtv_q(
    magma_int_t n, 
    magmaDoubleComplex *du, magmaDoubleComplex *db,
    magma_queue_t queue)
{
    /*

     */
    magma_int_t threads = block_length;
    magma_int_t grid = n/(4*block_length) + ((n%(4*block_length))!=0);

    magmablas_zapply_transpose_vector<<< grid, threads, 0, queue >>>(n/2, du+n, db);
    magmablas_zapply_transpose_vector<<< grid, threads, 0, queue >>>(n/2, du+n+n/2, db+n/2);

    threads = block_length;
    grid = n/(2*block_length) + ((n%(2*block_length))!=0);
    magmablas_zapply_transpose_vector<<< grid, threads, 0, queue >>>(n, du, db);
}


/**
    @see magmablas_zprbt_mtv_q
    ********************************************************************/
extern "C" void
magmablas_zprbt_mtv(
    magma_int_t n, 
    magmaDoubleComplex *du, magmaDoubleComplex *db)
{
    magmablas_zprbt_mtv_q(n, du, db, magma_stream);
}



/**
    Purpose
    -------
    ZPRBT_MV compute B = VB to obtain the non randomized solution
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of values of db.  n >= 0.
    
    @param[in,out]
    db      COMPLEX_16 array, dimension (n)
            The n vector db computed by ZGESV_NOPIV_GPU
            On exit db = dv*db
    
    @param[in]
    dv      COMPLEX_16 array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.
    ********************************************************************/
extern "C" void
magmablas_zprbt_mv_q(
    magma_int_t n, 
    magmaDoubleComplex *dv, magmaDoubleComplex *db,
    magma_queue_t queue)
{

    magma_int_t threads = block_length;
    magma_int_t grid = n/(2*block_length) + ((n%(2*block_length))!=0);

    magmablas_zapply_vector<<< grid, threads, 0, queue >>>(n, dv, db);


    threads = block_length;
    grid = n/(4*block_length) + ((n%(4*block_length))!=0);

    magmablas_zapply_vector<<< grid, threads, 0, queue >>>(n/2, dv+n, db);
    magmablas_zapply_vector<<< grid, threads, 0, queue >>>(n/2, dv+n+n/2, db+n/2);


}



/**
    @see magmablas_zprbt_mtv_q
    ********************************************************************/
extern "C" void
magmablas_zprbt_mv(
    magma_int_t n, 
    magmaDoubleComplex *dv, magmaDoubleComplex *db)
{
    magmablas_zprbt_mv_q(n, dv, db, magma_stream);
}


/**
    Purpose
    -------
    ZPRBT randomize a square general matrix using partial randomized transformation
    
    Arguments
    ---------
    @param[in]
    n       INTEGER
            The number of columns and rows of the matrix dA.  n >= 0.
    
    @param[in,out]
    dA      COMPLEX_16 array, dimension (n,ldda)
            The n-by-n matrix dA
            On exit dA = duT*dA*d_V
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDA >= max(1,n).
    
    @param[in]
    du      COMPLEX_16 array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix U
    
    @param[in]
    dv      COMPLEX_16 array, dimension (n,2)
            The 2*n vector representing the random butterfly matrix V
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    ********************************************************************/
extern "C" void 
magmablas_zprbt_q(
    magma_int_t n, 
    magmaDoubleComplex *dA, magma_int_t ldda, 
    magmaDoubleComplex *du, magmaDoubleComplex *dv,
    magma_queue_t queue)
{
    du += ldda;
    dv += ldda;

    dim3 threads(block_height, block_width);
    dim3 grid(n/(4*block_height) + ((n%(4*block_height))!=0), 
            n/(4*block_width)  + ((n%(4*block_width))!=0));

    magmablas_zelementary_multiplication<<< grid, threads, 0, queue >>>(n/2, dA, ldda, du, dv);
    magmablas_zelementary_multiplication<<< grid, threads, 0, queue >>>(n/2, dA+ldda*n/2, ldda, du, dv+n/2);
    magmablas_zelementary_multiplication<<< grid, threads, 0, queue >>>(n/2, dA+n/2, ldda, du+n/2, dv);
    magmablas_zelementary_multiplication<<< grid, threads, 0, queue >>>(n/2, dA+ldda*n/2+n/2, ldda, du+n/2, dv+n/2);

    dim3 threads2(block_height, block_width);
    dim3 grid2(n/(2*block_height) + ((n%(2*block_height))!=0), 
            n/(2*block_width)  + ((n%(2*block_width))!=0));
    magmablas_zelementary_multiplication<<< grid2, threads2, 0, queue >>>(n, dA, ldda, du-ldda, dv-ldda);
}


/**
    @see magmablas_zprbt_q
    ********************************************************************/
extern "C" void 
magmablas_zprbt(
    magma_int_t n, 
    magmaDoubleComplex *dA, magma_int_t ldda, 
    magmaDoubleComplex *du, magmaDoubleComplex *dv)
{
    magmablas_zprbt_q(n, dA, ldda, du, dv, magma_stream);
}



// adds   x += r  --and--
// copies r = b
// each thread does one index, x[i] and r[i]
__global__ void
zaxpycp2_kernel(
    int m, magmaDoubleComplex *r, magmaDoubleComplex *x,
    const magmaDoubleComplex *b)
{
    const int i = threadIdx.x + blockIdx.x*NB;
    if ( i < m ) {
        x[i] = MAGMA_Z_ADD( x[i], r[i] );
        r[i] = b[i];
    }
}


// ----------------------------------------------------------------------
// adds   x += r  --and--
// copies r = b
extern "C" void
magmablas_zaxpycp2_q(
    magma_int_t m, magmaDoubleComplex *r, magmaDoubleComplex *x,
    const magmaDoubleComplex *b,
    magma_queue_t queue )
{
    dim3 threads( NB );
    dim3 grid( (m + NB - 1)/NB );
    zaxpycp2_kernel <<< grid, threads, 0, queue >>> ( m, r, x, b );
}


extern "C" void
magmablas_zaxpycp2(
    magma_int_t m, magmaDoubleComplex *r, magmaDoubleComplex *x,
    const magmaDoubleComplex *b)
{
    magmablas_zaxpycp2_q( m, r, x, b, magma_stream );
}
