#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c
       @author Mark Gates
*/
#include "common_magma.h"

#define BLK_X 64
#define BLK_Y 32

/*
    Divides matrix into ceil( m/BLK_X ) x ceil( n/BLK_Y ) blocks.
    Each block has BLK_X threads.
    Each thread loops across one row, updating BLK_Y entries.

    Code similar to zlaset.
*/
__global__
void zgeadd_full(
    int m, int n,
    magmaDoubleComplex alpha,
    const magmaDoubleComplex *dA, int ldda,
    magmaDoubleComplex       *dB, int lddb )
{
    int ind = blockIdx.x*BLK_X + threadIdx.x;
    int iby = blockIdx.y*BLK_Y;
    /* check if full block-column */
    bool full = (iby + BLK_Y <= n);
    /* do only rows inside matrix */
    if ( ind < m ) {
        dA += ind + iby*ldda;
        dB += ind + iby*lddb;
        if ( full ) {
            // full block-column
            #pragma unroll
            for( int j=0; j < BLK_Y; ++j ) {
                dB[j*lddb] = alpha*dA[j*ldda] + dB[j*lddb];
            }
        }
        else {
            // partial block-column
            for( int j=0; j < BLK_Y && iby+j < n; ++j ) {
                dB[j*lddb] = alpha*dA[j*ldda] + dB[j*lddb];
            }
        }
    }
}


/**
    Purpose
    -------
    ZGEADD adds two matrices, dB = alpha*dA + dB.
    
    Arguments
    ---------
    
    @param[in]
    m       INTEGER
            The number of rows of the matrix dA.  M >= 0.
    
    @param[in]
    n       INTEGER
            The number of columns of the matrix dA.  N >= 0.
    
    @param[in]
    alpha   COMPLEX_16
            The scalar alpha.
            
    @param[in]
    dA      COMPLEX_16 array, dimension (LDDA,N)
            The m by n matrix dA.
    
    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDDA >= max(1,M).
            
    @param[in,out]
    dB      COMPLEX_16 array, dimension (LDDB,N)
            The m by n matrix dB.
    
    @param[in]
    lddb    INTEGER
            The leading dimension of the array dB.  LDDB >= max(1,M).
    
    @param[in]
    queue   magma_queue_t
            Queue to execute in.

    @ingroup magma_zaux2
    ********************************************************************/
extern "C" void
magmablas_zgeadd_q(
    magma_int_t m, magma_int_t n,
    magmaDoubleComplex alpha,
    magmaDoubleComplex_const_ptr dA, magma_int_t ldda,
    magmaDoubleComplex_ptr       dB, magma_int_t lddb,
    magma_queue_t queue )
{
    magma_int_t info = 0;
    if ( m < 0 )
        info = -1;
    else if ( n < 0 )
        info = -2;
    else if ( ldda < max(1,m))
        info = -5;
    else if ( lddb < max(1,m))
        info = -7;
    
    if ( info != 0 ) {
        magma_xerbla( __func__, -(info) );
        return;
    }
    
    if ( m == 0 || n == 0 )
        return;
    
    dim3 threads( BLK_X, 1 );
    dim3 grid( magma_ceildiv( m, BLK_X ), magma_ceildiv( n, BLK_Y ) );
    
    zgeadd_full<<< grid, threads, 0, queue->hip_stream() >>>
        ( m, n, alpha, dA, ldda, dB, lddb );
}


/**
    @see magmablas_zgeadd_q
    @ingroup magma_zaux2
    ********************************************************************/
extern "C" void
magmablas_zgeadd(
    magma_int_t m, magma_int_t n,
    magmaDoubleComplex alpha,
    magmaDoubleComplex_const_ptr dA, magma_int_t ldda,
    magmaDoubleComplex_ptr       dB, magma_int_t lddb )
{
    magmablas_zgeadd_q( m, n, alpha, dA, ldda, dB, lddb, magmablasGetQueue() );
}
