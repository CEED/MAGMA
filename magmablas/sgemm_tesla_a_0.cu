#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011
*/
#include "common_magma.h"
#include "commonblas_s.h"

__global__ void  
sgemm_kernel_a_0(float *C, const float *A, const float *B, 
                 int m, int n, int k, 
                 int lda, int ldb, int ldc,
                 float alpha, float beta)
{
        const int tx = threadIdx.x;
        const int ty = threadIdx.y;

        int ibx = blockIdx.x * 64;
        int iby = blockIdx.y *16;

        const int idt = ty * 16 + tx;


        C += ibx +idt +__mul24(iby,ldc);

        ibx = ibx+idt - m  ;
        
        if( (iby+16)>=n) { 
                lda = n-iby;
        }
        else    {
                lda = 16;
        }
        if( ibx >= 0 )
                lda = 0 ;
        else lda = lda ;

        switch(lda){
                case 16:
                        C[ 0    ] *=beta;
                        C[ 1*ldc] *=beta;
                        C[ 2*ldc] *=beta;
                        C[ 3*ldc] *=beta;
                        C[ 4*ldc] *=beta;
                        C[ 5*ldc] *=beta;
                        C[ 6*ldc] *=beta;
                        C[ 7*ldc] *=beta;
                        C[ 8*ldc] *=beta;
                        C[ 9*ldc] *=beta;
                        C[10*ldc] *=beta;
                        C[11*ldc] *=beta;
                        C[12*ldc] *=beta;
                        C[13*ldc] *=beta;
                        C[14*ldc] *=beta;;
                        C[15*ldc] *=beta;
                        break;
                case 0:
                        break;
                case 15:
                        C[ 0    ] *=beta;
                        C[ 1*ldc] *=beta;
                        C[ 2*ldc] *=beta;
                        C[ 3*ldc] *=beta;
                        C[ 4*ldc] *=beta;
                        C[ 5*ldc] *=beta;
                        C[ 6*ldc] *=beta;
                        C[ 7*ldc] *=beta;
                        C[ 8*ldc] *=beta;
                        C[ 9*ldc] *=beta;
                        C[10*ldc] *=beta;
                        C[11*ldc] *=beta;
                        C[12*ldc] *=beta;
                        C[13*ldc] *=beta;
                        C[14*ldc] *=beta;
                        break;
                case 14:
                        C[ 0    ] *=beta;
                        C[ 1*ldc] *=beta;
                        C[ 2*ldc] *=beta;
                        C[ 3*ldc] *=beta;
                        C[ 4*ldc] *=beta;
                        C[ 5*ldc] *=beta;
                        C[ 6*ldc] *=beta;
                        C[ 7*ldc] *=beta;
                        C[ 8*ldc] *=beta;
                        C[ 9*ldc] *=beta;
                        C[10*ldc] *=beta;
                        C[11*ldc] *=beta;
                        C[12*ldc] *=beta;
                        C[13*ldc] *=beta;
                        break;
                case 13:
                        C[ 0    ] *=beta;
                        C[ 1*ldc] *=beta;
                        C[ 2*ldc] *=beta;
                        C[ 3*ldc] *=beta;
                        C[ 4*ldc] *=beta;
                        C[ 5*ldc] *=beta;
                        C[ 6*ldc] *=beta;
                        C[ 7*ldc] *=beta;
                        C[ 8*ldc] *=beta;
                        C[ 9*ldc] *=beta;
                        C[10*ldc] *=beta;
                        C[11*ldc] *=beta;
                        C[12*ldc] *=beta;
                        break;
                case 12:
                        C[ 0    ] *=beta;
                        C[ 1*ldc] *=beta;
                        C[ 2*ldc] *=beta;
                        C[ 3*ldc] *=beta;
                        C[ 4*ldc] *=beta;
                        C[ 5*ldc] *=beta;
                        C[ 6*ldc] *=beta;
                        C[ 7*ldc] *=beta;
                        C[ 8*ldc] *=beta;
                        C[ 9*ldc] *=beta;
                        C[10*ldc] *=beta;
                        C[11*ldc] *=beta;
                        break;
                case 11:
                        C[ 0    ] *=beta;
                        C[ 1*ldc] *=beta;
                        C[ 2*ldc] *=beta;
                        C[ 3*ldc] *=beta;
                        C[ 4*ldc] *=beta;
                        C[ 5*ldc] *=beta;
                        C[ 6*ldc] *=beta;
                        C[ 7*ldc] *=beta;
                        C[ 8*ldc] *=beta;
                        C[ 9*ldc] *=beta;
                        C[10*ldc] *=beta;
                        break;
                case 10:
                        C[ 0    ] *=beta;
                        C[ 1*ldc] *=beta;
                        C[ 2*ldc] *=beta;
                        C[ 3*ldc] *=beta;
                        C[ 4*ldc] *=beta;
                        C[ 5*ldc] *=beta;
                        C[ 6*ldc] *=beta;
                        C[ 7*ldc] *=beta;
                        C[ 8*ldc] *=beta;
                        C[ 9*ldc] *=beta;
                        break;
                case 9:
                        C[ 0    ] *=beta;
                        C[ 1*ldc] *=beta;
                        C[ 2*ldc] *=beta;
                        C[ 3*ldc] *=beta;
                        C[ 4*ldc] *=beta;
                        C[ 5*ldc] *=beta;
                        C[ 6*ldc] *=beta;
                        C[ 7*ldc] *=beta;
                        C[ 8*ldc] *=beta;
                        break;
                case 8:
                        C[ 0    ] *=beta;
                        C[ 1*ldc] *=beta;
                        C[ 2*ldc] *=beta;
                        C[ 3*ldc] *=beta;
                        C[ 4*ldc] *=beta;
                        C[ 5*ldc] *=beta;
                        C[ 6*ldc] *=beta;
                        C[ 7*ldc] *=beta;
                        break;
                case 7:
                        C[ 0    ] *=beta;
                        C[ 1*ldc] *=beta;
                        C[ 2*ldc] *=beta;
                        C[ 3*ldc] *=beta;
                        C[ 4*ldc] *=beta;
                        C[ 5*ldc] *=beta;
                        C[ 6*ldc] *=beta;
                        break;
                case 6:
                        C[ 0    ] *=beta;
                        C[ 1*ldc] *=beta;
                        C[ 2*ldc] *=beta;
                        C[ 3*ldc] *=beta;
                        C[ 4*ldc] *=beta;
                        C[ 5*ldc] *=beta;
                        break;
                case 5:
                        C[ 0    ] *=beta;
                        C[ 1*ldc] *=beta;
                        C[ 2*ldc] *=beta;
                        C[ 3*ldc] *=beta;
                        C[ 4*ldc] *=beta;
                        break;
                case 4:
                        C[ 0    ] *=beta;
                        C[ 1*ldc] *=beta;
                        C[ 2*ldc] *=beta;
                        C[ 3*ldc] *=beta;
                        break;
                case 3:
                        C[ 0    ] *=beta;
                        C[ 1*ldc] *=beta;
                        C[ 2*ldc] *=beta;
                        break;
                case 2:
                        C[ 0    ] *=beta;
                        C[ 1*ldc] *=beta;
                        break;
                case 1:
                        C[ 0    ] *=beta;
                        break;
        }
}


extern "C" void
magmablas_sgemm_kernel_a_0(float *C, const float *A, const float *B, 
                           magma_int_t m, magma_int_t n, magma_int_t k, 
                           magma_int_t lda, magma_int_t ldb, magma_int_t ldc,
                           float alpha, float beta)
{
        dim3 threads( 16, 4 );
        dim3 grid(m/64+(m%64!=0),n/16+(n%16!=0));
        sgemm_kernel_a_0<<< grid, threads, 0, magma_stream >>> (C, A, B, m, n, k, 
                                               lda, ldb, ldc,
                                               alpha, beta ) ;
}
