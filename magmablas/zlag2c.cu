#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

       @precisions mixed zc -> ds

*/
#include <stdio.h>
#include <hipblas.h>
#include "magma.h"
#include "magmablas.h"

#define PRECISION_z
#define blksize 64

__device__ int flag = 0; 

static __global__ void 
magmaint_zlag2c( magma_int_t M, magma_int_t N, 
                  const hipDoubleComplex *A, int lda, 
                  hipFloatComplex *SA,       int ldsa, 
                  double RMAX ) 
{
    const hipDoubleComplex *Aend = A + lda*N;
    hipDoubleComplex tmp;
    double mRMAX = - RMAX;
    int    mym   = blockIdx.x * blksize + threadIdx.x;

    if ( mym < M ){
        A += mym;
        SA+= mym; 
        
        tmp = *A;
        for ( ; A < Aend; )
        {
            A  += lda;
            if( (hipCreal(tmp) < mRMAX) || (hipCreal(tmp) > RMAX)
#if defined(PRECISION_z) || defined(PRECISION_c)
                || (hipCimag(tmp) < mRMAX) || (hipCimag(tmp) > RMAX) 
#endif
                )
            {
                flag = 1; 
            }
            *SA = hipComplexDoubleToFloat( tmp );
            tmp = *A;
            SA += ldsa;
        }
    }
}


extern "C" void 
magmablas_zlag2c( int M, int N , 
                  const hipDoubleComplex *A, int lda, 
                  hipFloatComplex *SA,       int ldsa, 
                  magma_int_t *info ) 
{    
/*
  Note
  ====
	- We have to provide INFO at the end that zlag2c isn't doable now. 
	- Transfer a single value TO/FROM CPU/GPU
	- SLAMCH that's needed is called from underlying BLAS
	- Only used in iterative refinement
	- Do we want to provide this in the release?
  Purpose
  =======

  ZLAG2C converts a DOUBLE PRECISION matrix, SA, to a SINGLE
  PRECISION matrix, A.

  RMAX is the overflow for the SINGLE PRECISION arithmetic
  ZLAG2C checks that all the entries of A are between -RMAX and
  RMAX. If not the convertion is aborted and a flag is raised.

  This is an auxiliary routine so there is no argument checking.

  Arguments
  =========

  M       (input) INTEGER
          The number of lines of the matrix A.  M >= 0.

  N       (input) INTEGER
          The number of columns of the matrix A.  N >= 0.

  A       (input) DOUBLE PRECISION array, dimension (LDA,N)
          On entry, the M-by-N coefficient matrix A.

  LDA     (input) INTEGER
          The leading dimension of the array A.  LDA >= max(1,M).

  SA      (output) REAL array, dimension (LDSA,N)
          On exit, if INFO=0, the M-by-N coefficient matrix SA; if
          INFO>0, the content of SA is unspecified.

  LDSA    (input) INTEGER
          The leading dimension of the array SA.  LDSA >= max(1,M).

  INFO    (output) INTEGER
          = 0:  successful exit.
          = 1:  an entry of the matrix A is greater than the SINGLE
                PRECISION overflow threshold, in this case, the content
                of SA in exit is unspecified.

  ===========================================================================  */

    double RMAX = (double)lapackf77_slamch("O");

    dim3 threads( blksize, 1, 1 );
    dim3 grid( (M+blksize-1)/blksize, 1, 1);
    flag = 0;
    magmaint_zlag2c<<< grid, threads >>>( M, N, A, lda, SA, ldsa, RMAX ) ; 
    *info = flag;
}
