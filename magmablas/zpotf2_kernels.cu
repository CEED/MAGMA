#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.4) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2013
       
       @author Azzam Haidar
       @author Tingxing Dong

       @precisions normal z -> s d c
*/

#include "common_magma.h"
#include "batched_kernel_param.h"
#include "magma_templates.h"

#define PRECISION_z


#define A(i, j)  (A + (i) + (j)*lda)   // A(i, j) means at i row, j column


// dynamically allocated shared memory, set to size number of threads when the kernel is launched.
// See CUDA Guide B.2.3
extern __shared__ magmaDoubleComplex shared_data[];


// dynamically allocated shared memory, set to size number of threads when the kernel is launched.
// See CUDA Guide B.2.3
extern __shared__ double dble_shared_data[];

/////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void zdotc_kernel_batched(int n, magmaDoubleComplex **x_array, int incx, int offset, magma_int_t *info_array, int gbstep)
{
    int tx = threadIdx.x;

    magmaDoubleComplex *x = x_array[blockIdx.z]+offset;

    double *sdata = dble_shared_data;

    magmaDoubleComplex res = MAGMA_Z_ZERO;

    if (tx < n) {
       res = x[tx*incx];
    }

    sdata[tx] = MAGMA_Z_REAL(res * MAGMA_Z_CNJG(res));

    __syncthreads();

    for(int s = blockDim.x/2; s > 32; s >>= 1 ) {
        if (tx < s) {
            sdata[tx] += sdata[tx+s];
        }
        __syncthreads();
    }

    if (tx < 32) {
        volatile double* smem = sdata;
        smem[tx] += smem[tx+32];
        smem[tx] += smem[tx+16];
        smem[tx] += smem[tx+8];
        smem[tx] += smem[tx+4];
        smem[tx] += smem[tx+2];
        smem[tx] += smem[tx+1];
    }

    if (tx == 0) {
        double xreal = MAGMA_Z_REAL(x[n*incx]);        
        //MAGMA_Z_SET2REAL(x[n*incx], sqrt(xreal - sdata[0]));
        x[n*incx] = MAGMA_Z_MAKE(sqrt(xreal - sdata[0]), 0);
        if(x[n*incx] == MAGMA_Z_ZERO){
            info_array[blockIdx.z] = offset + gbstep + 1;
        }
    }
}


void magma_zpotf2_zdotc_batched(magma_int_t n, magmaDoubleComplex **x_array, magma_int_t incx, magma_int_t offset, magma_int_t *info_array, magma_int_t gbstep, magma_int_t batchCount)
{
/*
    Specialized Zdotc
    1) performs zdotc sum = x[0:n-1]*conj(x[0:n-1])
    2) updates x[n] = sqrt(x[n]-sum);

*/
    if (n > MAX_NTHREADS) {
        printf("n = %d > %d is not supported in zpotf2_zdotc\n", (int) n, (int) MAX_NTHREADS);
        
    }
    int threadSize;

    if (n <= 1024 && n > 512) {
        threadSize = 1024;
    }
    else if (n <= 512 && n > 256 ) {
        threadSize = 512;
    }
    else if (n <= 256 && n > 128) {
        threadSize = 256;
    }
    else if (n <= 128 && n > 64) {
        threadSize = 128;
    }
    else {
        threadSize = 64;
    }

    
    dim3 grid(1, 1, batchCount);
    zdotc_kernel_batched<<< grid, threadSize, 
                  threadSize * sizeof(double), magma_stream>>> (n, x_array, incx, offset, info_array, gbstep);
}

/////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void zdscal_kernel_batched(int n, magmaDoubleComplex **x_array, int incx, int offset, magma_int_t *info_array)
{
    // checkinfo to avoid computation of the singular matrix
    if(info_array[blockIdx.z] != 0 ) return;

    int id = threadIdx.x;
    magmaDoubleComplex *x = x_array[blockIdx.z]+offset;

    __shared__ magmaDoubleComplex factor;

    if (threadIdx.x == 0) {
        factor = MAGMA_Z_MAKE(1.0/MAGMA_Z_REAL(x[0]), 0.0);
    }

    __syncthreads();

    if ( id < n && id >0) {
        x[id*incx] = x[id*incx] * factor;
        //printf("x=%f", x[id*incx]);
    }
}


void magma_zpotf2_zdscal_batched(magma_int_t n, magmaDoubleComplex **x_array, magma_int_t incx, magma_int_t offset, magma_int_t *info_array, magma_int_t batchCount)
{
/*
    Specialized Zdscal perform x[1:n-1]/x[0]

*/
    dim3 grid(1, 1, batchCount);
    dim3 threads(n, 1, 1); 

    zdscal_kernel_batched<<< grid, threads, 0, magma_stream >>> (n, x_array, incx, offset, info_array);
}

/////////////////////////////////////////////////////////////////////////////////////////////////


#if defined(PRECISION_z) || defined(PRECISION_c)

__global__ void zlacgv_kernel_batched(int n, magmaDoubleComplex **x_array, int incx, int offset)
{
    int id = threadIdx.x;

    magmaDoubleComplex *x = x_array[blockIdx.z]+offset;

    if ( id < n ) {
        x[id*incx] = MAGMA_Z_CNJG(x[id*incx]);
    }
}

void magma_zlacgv_batched(magma_int_t n, magmaDoubleComplex **x_array, magma_int_t incx, int offset, int batchCount)
{
/*
    Purpose
    =======

    ZLACGV conjugates a complex vector of length N.

    Arguments
    =========

    N       (input) INTEGER
            The length of the vector X.  N >= 0.

    X       (input/output) COMPLEX*16 array, dimension
                           (1+(N-1)*abs(INCX))
            On entry, the vector of length N to be conjugated.
            On exit, X is overwritten with conjg(X).

    INCX    (input) INTEGER
            The spacing between successive elements of X.

    ===================================================================== */

    dim3 grid(1, 1, batchCount);
    dim3 threads(n, 1, 1);
   
    zlacgv_kernel_batched<<< grid, threads, 0, magma_stream >>> (n, x_array, incx, offset);
}

#endif // defined(PRECISION_z) || defined(PRECISION_c)



/////////////////////////////////////////////////////////////////////////////////////////////////
static __device__ void zpotf2_device(int m, int n, 
                              magmaDoubleComplex *A, int lda, 
                              magmaDoubleComplex alpha, 
                              magmaDoubleComplex beta, magma_int_t *info, int gbstep)
{
/*
    Each thread block load entire A into shared memory
    factorize it and copy back. n must be small enough to fit shared memory.
    n is checked by a macro POTF2_TILE_SIZE before the kernel. 
*/
    // checkinfo to avoid computation of the singular matrix
    if(*info != 0 ) return;

    int tx = threadIdx.x;
    magmaDoubleComplex *sdata_A = shared_data;
    __shared__ magmaDoubleComplex factor;
    __shared__ double sum[POTF2_TILE_SIZE];

    // load A into sdata_A
    if(tx < m)
    {
        for(int i=0; i<n; i++)
        {  
             sdata_A[tx + i * m] =  A[tx + i * lda];
        }
    }
    __syncthreads();

    for(int iter=0; iter<n; iter++)
    {
        double res = MAGMA_D_ZERO;
        magmaDoubleComplex res1 = MAGMA_Z_ZERO;

        //1) performs zdotc sum = A[iter, 0:iter-1]*conj(A[iter, 0:iter-1])
        //2) updates A[iter,iter] = sqrt(A[iter,iter]-sum);
        if(tx<iter)
        {
            res = MAGMA_Z_REAL (sdata_A[iter + tx * m] * MAGMA_Z_CNJG(sdata_A[iter + tx * m]));         
            sum[tx] = res;
        }
        else
        {
            sum[tx] = 0.0;
        }
        __syncthreads();
        magma_sum_reduce<POTF2_TILE_SIZE>(tx, sum);//tried on K40: if m=32 n=32 the overall zpotf2_device routine time is 60ms n=16 time=25 n=8 time=20ms 
        //magma_sum_reduce_n(iter, tx, sum); //tried on K40: if m=32 n=32 the time went from 61ms to 70ms when switching to reduce_n. n=16 time=28.
        //magma_sum_reduce_inlined(iter, tx, sum); //tried on K40: similar to magma_sum_reduce<POTF2_TILE_SIZE>(tx, sum);
        
        if (tx == 0) {
              double xreal = MAGMA_Z_REAL(sdata_A[iter + iter * m]);        
              sdata_A[iter + iter * m] = MAGMA_Z_MAKE(sqrt(xreal - sum[0]), 0);
              if(sdata_A[iter + iter * m] == MAGMA_Z_ZERO){
                  *info = iter + gbstep + 1;
              }
        }
        __syncthreads();
        if(sdata_A[iter + iter * m] == MAGMA_Z_ZERO) return;
        __syncthreads();

        //zlacgv conjugates a complex vector of length iter. //TODO
        #if defined(PRECISION_z) || defined(PRECISION_c)
        if(tx < iter)
        {
             sdata_A[iter + tx * m] = MAGMA_Z_CNJG(sdata_A[iter + tx * m]);
        }
        __syncthreads();  
        #endif
  
        // zgemv  
        // Compute elements iter:n-1 of column iter = A(iter:n,0:iter-1) * A(iter-1,0:iter-1) (row).
        if(tx < m && tx > iter)
        {
            for(int j=0; j < iter; j++)
            {
                res1 += sdata_A[tx + j * m]  *  sdata_A[iter + j * m]; // TODO move the zlacgv conj to be done automatically here implicitly.
            }   
            sdata_A [tx + iter * m] = alpha * res1 + sdata_A [tx + iter * m] * beta;   
        }
        __syncthreads();  

        //zlacgv conjugates a complex vector of length iter.
        #if defined(PRECISION_z) || defined(PRECISION_c)
        if(tx < iter)
        {
             sdata_A[iter + tx * m] = MAGMA_Z_CNJG(sdata_A[iter + tx * m]);
        }
        __syncthreads();  
        #endif

        // zdscal perform A[iter:n-1, iter]/A[iter,iter];
        if (tx == 0) {
            factor = MAGMA_Z_MAKE(1.0/MAGMA_Z_REAL(sdata_A[iter + iter * m]), 0.0);
        }
        __syncthreads();

        if ( tx < m && tx > iter) {
            sdata_A[ tx + iter * m ]  *= factor;
        }
        __syncthreads();
    }// end of iter

    //copy sdata_A to A
    if(tx < m)
    {
        for(int i=0; i<n; i++)
        {  
             A[tx + i * lda] = sdata_A[tx + i * m];
        }
    }
}

/////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void zpotf2_kernel_batched(int m, int n, 
                              magmaDoubleComplex **dA_array, int lda, 
                              magmaDoubleComplex alpha, 
                              magmaDoubleComplex beta, 
                              magma_int_t *info_array, int gbstep)
{
/*
    Each thread block load entire dA_array[blockIdx.z] into shared memory
    factorize it and copy back. n must be small enough to fit shared memory.
    n is checked by a macro POTF2_TILE_SIZE before the kernel. 
*/
    int batchid = blockIdx.z;
    zpotf2_device(m, n, dA_array[batchid], lda, alpha, beta, &(info_array[batchid]), gbstep);
}
/////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void zpotf2_kernel(int m, int n, 
                              magmaDoubleComplex *dA, int lda, 
                              magmaDoubleComplex alpha, 
                              magmaDoubleComplex beta,
                              magma_int_t *info)
{
    zpotf2_device(m, n, dA, lda, alpha, beta, info, 0);
}
/////////////////////////////////////////////////////////////////////////////////////////////////
/**
    Purpose
    -------

    zpotf2 computes the Cholesky factorization of a real symmetric
    positive definite matrix A.

    The factorization has the form
        A = U**H * U,  if UPLO = MagmaUpper, or
        A = L  * L**H, if UPLO = MagmaLower,
    where U is an upper triangular matrix and L is lower triangular.

    This is the unblocked version of the algorithm, calling Level 2 BLAS.

    Arguments
    ---------

    @param[in]
    uplo    magma_uplo_t
            Specifies whether the upper or lower triangular part of the
            symmetric matrix A is stored.
      -     = MagmaUpper:  Upper triangular
      -     = MagmaLower:  Lower triangular

    @param[in]
    n       INTEGER
            The order of the matrix A.  N >= 0 and N <= 512.

    @param[in,out]
    dA      COMPLEX_16 array, dimension (LDDA,N)
            On entry, the symmetric matrix A.  If UPLO = MagmaUpper, the leading
            n by n upper triangular part of A contains the upper
            triangular part of the matrix A, and the strictly lower
            triangular part of A is not referenced.  If UPLO = MagmaLower, the
            leading n by n lower triangular part of A contains the lower
            triangular part of the matrix A, and the strictly upper
            triangular part of A is not referenced.
    \n
            On exit, if INFO = 0, the factor U or L from the Cholesky
            factorization A = U**H * U  or A = L * L**H.

    @param[in]
    ldda    INTEGER
            The leading dimension of the array A.  LDDA >= max(1,N).

    @param[out]
    info    INTEGER
      -     = 0: successful exit
      -     < 0: if INFO = -k, the k-th argument had an illegal value
      -     > 0: if INFO = k, the leading minor of order k is not
                 positive definite, and the factorization could not be
                 completed.

    @ingroup magma_zposv_aux
    ********************************************************************/
extern "C" magma_int_t
magma_zpotf2_tile_batched(
    magma_uplo_t uplo, magma_int_t m, magma_int_t n,
    magmaDoubleComplex **dA_array, magma_int_t lda,
    magma_int_t *info_array, magma_int_t gbstep, magma_int_t batchCount)
{

    magma_int_t arginfo = 0;
    
    if ( uplo != MagmaUpper && uplo != MagmaLower) {
        arginfo = -1;
    } else if (m < 0 || n < 0 || m > POTF2_TILE_SIZE || n > POTF2_TILE_SIZE) {
        arginfo = -2;
    } else if (lda < max(1,m)) {
        arginfo = -4;
    } else if (m < n) {
        arginfo = -10;
    }
    if (uplo == MagmaUpper) {
        printf("Upper side is unavailable \n");
        arginfo = -1;
    }

    if (arginfo != 0) {
        magma_xerbla( __func__, -(arginfo) );
        return arginfo;
    }
    
    // Quick return if possible
    if (m == 0 || n == 0) {
        return arginfo;
    }

    magmaDoubleComplex alpha = MAGMA_Z_NEG_ONE;
    magmaDoubleComplex beta  = MAGMA_Z_ONE;

    dim3 dimGrid(1, 1, batchCount);
    dim3 threads(POTF2_TILE_SIZE, 1);
    int shared_mem_size = sizeof(magmaDoubleComplex)*m*n; // + sizeof(double)*(POTF2_TILE_SIZE+1);

    zpotf2_kernel_batched<<<dimGrid, threads, shared_mem_size >>>(m, n, dA_array, lda, alpha, beta, info_array, gbstep);

    return arginfo;
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////
extern "C" magma_int_t
magma_zpotf2_tile(
    magma_uplo_t uplo, magma_int_t m, magma_int_t n,
    magmaDoubleComplex *dA, magma_int_t lda,
    magma_int_t *info)
{

    *info = 0;
    if ( uplo != MagmaUpper && uplo != MagmaLower) {
        *info = -1;
    } else if (m < 0 || n < 0 || m > POTF2_TILE_SIZE) {
        *info = -2;
    } else if (lda < max(1,m)) {
        *info = -4;
    } else if (m < n) {
        *info = -10;
    }
    if (uplo == MagmaUpper) {
        printf("Upper side is unavailable \n");
        *info = -1;
    }


    if (*info != 0) {
        magma_xerbla( __func__, -(*info) );
        return *info;
    }

    // Quick return if possible
    if (m == 0 || n == 0) {
        return *info;
    }

    magmaDoubleComplex alpha = MAGMA_Z_NEG_ONE;
    magmaDoubleComplex beta  = MAGMA_Z_ONE;

    dim3 dimGrid(1);
    dim3 threads(POTF2_TILE_SIZE, 1);
    int shared_mem_size = sizeof(magmaDoubleComplex)*m*n; // + sizeof(double)*(POTF2_TILE_SIZE+1);

    zpotf2_kernel<<<dimGrid, threads, shared_mem_size >>>(m, n, dA, lda, alpha, beta, info);

    return *info;
}

