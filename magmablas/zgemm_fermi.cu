#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c

       @author Jakub Kurzak
       @author Stan Tomov
       @author Mark Gates

       [zcds]gemm_fermi.cu          defines the CPU driver.
       [zcds]gemm_fermi_kernels.h   defines the block sizes for each precision.
       gemm_stencil_defs.h          defines types and functions for precision-independent code.
       gemm_stencil.cu              defines the GPU kernel. It gets included
                                    multiple times, once for each transpose version.
*/
#include "common_magma.h"
#include "commonblas_z.h"
#include <assert.h>

#define PRECISION_z

///////////////////////////////////////////////////////////////////////////////////////////////////

#include "zgemm_fermi_kernels.h"

///////////////////////////////////////////////////////////////////////////////////////////////////

/**
    Purpose
    -------
    ZGEMM performs one of the matrix-matrix operations
    
        C = alpha*op( A )*op( B ) + beta*C,
    
    where op( X ) is one of
    
        op( X ) = X   or   op( X ) = X**T   or   op( X ) = X**H,
    
    alpha and beta are scalars, and A, B and C are matrices, with
    op( A ) an m by k matrix, op( B ) a k by n matrix and C an m by n matrix.
    
    Parameters
    ----------
    @param[in]
    TRANSA  CHARACTER*1.
            On entry, TRANSA specifies the form of op( A ) to be used in
            the matrix multiplication as follows:
      -     = 'N':  op( A ) = A.
      -     = 'T':  op( A ) = A**T.
      -     = 'C':  op( A ) = A**H.
    
    @param[in]
    TRANSB  CHARACTER*1.
            On entry, TRANSB specifies the form of op( B ) to be used in
            the matrix multiplication as follows:
      -     = 'N':  op( B ) = B.
      -     = 'T':  op( B ) = B**T.
      -     = 'C':  op( B ) = B**H.
    
    @param[in]
    m       INTEGER.
            On entry,  M  specifies  the number  of rows  of the  matrix
            op( d_A )  and of the  matrix d_C.  M  must  be at least  zero.
    
    @param[in]
    n       INTEGER.
            On entry,  N  specifies the number  of columns of the matrix
            op( d_B ) and the number of columns of the matrix d_C. N must be
            at least zero.
    
    @param[in]
    k       INTEGER.
            On entry,  K  specifies  the number of columns of the matrix
            op( d_A ) and the number of rows of the matrix op( d_B ). K must
            be at least  zero.
    
    @param[in]
    alpha   COMPLEX_16
            On entry, ALPHA specifies the scalar alpha.
    
    @param[in]
    d_A     COMPLEX_16 array of DIMENSION ( LDA, ka ), where ka is
            k  when  TRANSA = MagmaNoTrans,  and is  m  otherwise.
            Before entry with  TRANSA = MagmaNoTrans,  the leading  m by k
            part of the array d_A must contain the matrix d_A, otherwise
            the leading  k by m  part of the array d_A must contain  the
            matrix d_A.
    
    @param[in]
    lda     INTEGER.
            On entry, LDA specifies the first dimension of A as declared
            in the calling (sub) program. When  TRANSA = MagmaNoTrans then
            LDA must be at least  max( 1, m ), otherwise  LDA must be at
            least  max( 1, k ).
    
    @param[in]
    d_B     COMPLEX_16 array of DIMENSION ( LDB, kb ), where kb is
            n  when  TRANSB = MagmaNoTrans,  and is  k  otherwise.
            Before entry with  TRANSB = MagmaNoTrans,  the leading  k by n
            part of the array d_B must contain the matrix d_B, otherwise
            the leading  n by k  part of the array d_B must contain  the
            matrix d_B.
    
    @param[in]
    ldb     INTEGER.
            On entry, LDB specifies the first dimension of d_B as declared
            in the calling (sub) program. When  TRANSB = MagmaNoTrans then
            LDB must be at least  max( 1, k ), otherwise  LDB must be at
            least  max( 1, n ).
    
    @param[in]
    beta    COMPLEX_16.
            On entry,  BETA  specifies the scalar  beta.  When  BETA  is
            supplied as zero then d_C need not be set on input.
    
    @param[in,out]
    d_C     COMPLEX_16 array of DIMENSION ( LDC, n ).
            Before entry, the leading  m by n  part of the array  d_C must
            contain the matrix  d_C,  except when  beta  is zero, in which
            case d_C need not be set on entry.
            On exit, the array  d_C  is overwritten by the  m by n  matrix
            ( alpha*op( d_A )*op( d_B ) + beta*d_C ).
    
    @param[in]
    ldc     INTEGER.
            On entry, LDC specifies the first dimension of d_C as declared
            in  the  calling  (sub)  program.   LDC  must  be  at  least
            max( 1, m ).

    @ingroup magma_zblas3
    ********************************************************************/
extern "C" void
magmablas_zgemm(
    magma_trans_t TRANSA, magma_trans_t TRANSB, magma_int_t m, magma_int_t n, magma_int_t k,
    magmaDoubleComplex alpha,
    const magmaDoubleComplex *d_A, magma_int_t lda,
    const magmaDoubleComplex *d_B, magma_int_t ldb,
    magmaDoubleComplex beta,
    magmaDoubleComplex *d_C, magma_int_t ldc )
{
    magma_int_t info = 0;
    if      ( TRANSA != MagmaNoTrans && TRANSA != MagmaTrans && TRANSA != MagmaConjTrans )
        info = -1;
    else if ( TRANSB != MagmaNoTrans && TRANSB != MagmaTrans && TRANSB != MagmaConjTrans )
        info = -2;
    else if ( m < 0 )
        info = -3;
    else if ( n < 0 )
        info = -4;
    else if ( k < 0 )
        info = -5;
    else if ( TRANSA == MagmaNoTrans ? lda < m : lda < k )
        info = -8;
    else if ( TRANSB == MagmaNoTrans ? ldb < k : lda < n )
        info = -10;
    else if ( ldc < m )
        info = -13;
    
    if (info != 0) {
        magma_xerbla( __func__, -(info) );
        return;  //info;
    }
    
    magma_int_t arch = magma_getdevice_arch();
    if ( arch < 200  ) {
        // --------------------
        // call CUDA ARCH 1.x version
        // magmablas for [sd] precisions, cublas for [zc] precisions.
        #if defined(PRECISION_z) || defined(PRECISION_c)
        magma_zgemm(
            TRANSA, TRANSB,
            m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc );
        #else
        magmablas_zgemm_tesla(
            TRANSA, TRANSB, m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc );
        #endif
        return;
    }
    
    // --------------------
    // CUDA ARCH 2.x (Fermi) version
    if ( m <= 0 || n <= 0 || k <= 0 )
        return;
    
    size_t offsetA = 0;
    size_t offsetB = 0;

    int TransA = 2, TransB = 2;
    if      ( TRANSA == MagmaTrans )
        TransA = 1;
    else if ( TRANSA == MagmaNoTrans )
        TransA = 0;
                    
    if      ( TRANSB == MagmaTrans )
        TransB = 1;
    else if ( TRANSB == MagmaNoTrans )
        TransB = 0;

    size_t sizeA = (size_t) lda * (size_t) (!TransA ? k : m);
    size_t sizeB = (size_t) ldb * (size_t) (!TransB ? n : k);

    size_t CUBLAS_MAX_1DBUF_SIZE = ((1 << 27) - 512);
    if ( sizeA >= CUBLAS_MAX_1DBUF_SIZE ||
         sizeB >= CUBLAS_MAX_1DBUF_SIZE )
    {
        magma_zgemm( TRANSA, TRANSB, m, n, k, alpha,
                     d_A, lda, d_B, ldb,
                     beta, d_C, ldc );
        return;
    }

    #ifdef TEXTURE_1D
        // Set textures parameters
        tex_ref_A.normalized = false;
        tex_ref_A.filterMode = hipFilterModePoint;
        tex_ref_A.addressMode[0] = hipAddressModeClamp;

        tex_ref_B.normalized = false;
        tex_ref_B.filterMode = hipFilterModePoint;
        tex_ref_B.addressMode[0] = hipAddressModeClamp;

        // Bind A and B to texture references
        hipError_t err;
        err = hipBindTexture(&offsetA, tex_ref_A, d_A, sizeA*sizeof(magmaDoubleComplex));
        if ( err != hipSuccess ) {
            fprintf( stderr, "cannot bind A to texture: %s (%d)\n", hipGetErrorString(err), err );
            return;
        }
        err = hipBindTexture(&offsetB, tex_ref_B, d_B, sizeB*sizeof(magmaDoubleComplex));
        if ( err != hipSuccess ) {
            fprintf( stderr, "cannot bind B to texture: %s (%d)\n", hipGetErrorString(err), err );
            hipUnbindTexture( tex_ref_A );
            return;
        }
    #endif

    // Set up grids
    dim3 dimBlock(DIM_X, DIM_Y);

    offsetA = offsetA/sizeof(d_A[0]);
    offsetB = offsetB/sizeof(d_B[0]);
 
    if ( TransA == 0 && TransB == 0 ) {
        dim3 dimGrid( (m - 1)/BLK_M_nn + 1,
                      (n - 1)/BLK_N_nn + 1 );
        zgemm_kernel_fermi_nn<<< dimGrid, dimBlock, 0, magma_stream >>>(
            m, n, k, d_A, lda, d_B, ldb, d_C, ldc, alpha, beta,
            (int)offsetA, (int)offsetB );
    }
    else if ( TransA == 0 && TransB == 1 ) {
        dim3 dimGrid( (m - 1)/BLK_M_nt + 1,
                      (n - 1)/BLK_N_nt + 1 );
        zgemm_kernel_fermi_nt<<< dimGrid, dimBlock, 0, magma_stream >>>(
            m, n, k, d_A, lda, d_B, ldb, d_C, ldc, alpha, beta,
            (int)offsetA, (int)offsetB );
    }
    else if ( TransA == 0 && TransB == 2 ) {
        dim3 dimGrid( (m - 1)/BLK_M_nc + 1,
                      (n - 1)/BLK_N_nc + 1 );
        zgemm_kernel_fermi_nc<<< dimGrid, dimBlock, 0, magma_stream >>>(
            m, n, k, d_A, lda, d_B, ldb, d_C, ldc, alpha, beta,
            (int)offsetA, (int)offsetB );
    }
    else if ( TransA == 1 && TransB == 0 ) {
        dim3 dimGrid( (m - 1)/BLK_M_tn + 1,
                      (n - 1)/BLK_N_tn + 1 );
        zgemm_kernel_fermi_tn<<< dimGrid, dimBlock, 0, magma_stream >>>(
            m, n, k, d_A, lda, d_B, ldb, d_C, ldc, alpha, beta,
            (int)offsetA, (int)offsetB );
    }
    else if ( TransA == 1 && TransB == 1 ) {
        dim3 dimGrid( (m - 1)/BLK_M_tt + 1,
                      (n - 1)/BLK_N_tt + 1 );
        zgemm_kernel_fermi_tt<<< dimGrid, dimBlock, 0, magma_stream >>>(
            m, n, k, d_A, lda, d_B, ldb, d_C, ldc, alpha, beta,
            (int)offsetA, (int)offsetB );
    }
    else if ( TransA == 1 && TransB == 2 ) {
        dim3 dimGrid( (m - 1)/BLK_M_tc + 1,
                      (n - 1)/BLK_N_tc + 1 );
        zgemm_kernel_fermi_tc<<< dimGrid, dimBlock, 0, magma_stream >>>(
            m, n, k, d_A, lda, d_B, ldb, d_C, ldc, alpha, beta,
            (int)offsetA, (int)offsetB );
    }
    else if ( TransA == 2 && TransB == 0 ) {
        dim3 dimGrid( (m - 1)/BLK_M_cn + 1,
                      (n - 1)/BLK_N_cn + 1 );
        zgemm_kernel_fermi_cn<<< dimGrid, dimBlock, 0, magma_stream >>>(
            m, n, k, d_A, lda, d_B, ldb, d_C, ldc, alpha, beta,
            (int)offsetA, (int)offsetB );
    }
    else if ( TransA == 2 && TransB == 1 ) {
        dim3 dimGrid( (m - 1)/BLK_M_ct + 1,
                      (n - 1)/BLK_N_ct + 1 );
        zgemm_kernel_fermi_ct<<< dimGrid, dimBlock, 0, magma_stream >>>(
            m, n, k, d_A, lda, d_B, ldb, d_C, ldc, alpha, beta,
            (int)offsetA, (int)offsetB );
    }
    else if ( TransA == 2 && TransB == 2 ) {
        dim3 dimGrid( (m - 1)/BLK_M_cc + 1,
                      (n - 1)/BLK_N_cc + 1 );
        zgemm_kernel_fermi_cc<<< dimGrid, dimBlock, 0, magma_stream >>>(
            m, n, k, d_A, lda, d_B, ldb, d_C, ldc, alpha, beta,
            (int)offsetA, (int)offsetB );
    }

    hipUnbindTexture( tex_ref_A );
    hipUnbindTexture( tex_ref_B );
}

///////////////////////////////////////////////////////////////////////////////////////////////////
