#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010
*/

#include "hipblas.h"
#include "magma.h"
#define magmablas_dgemv_tesla magmablas_dgemv
#define magmablas_dgemvt_tesla magmablas_dgemvt

#define num_threads 64
#define dgemv_bs 64

__global__ void 
dgemv_kernel(int n, int m, int n1, double* A, int lda, double *x, double *y)
{
  int ind = blockIdx.x*num_threads + threadIdx.x;

  A += ind;
  x += threadIdx.x;

  double res = 0.f;

  __shared__ double buff[dgemv_bs];
  for(int i=0; i<n1; i += dgemv_bs ){
    __syncthreads();
    buff[threadIdx.x]  = x[i];

    __syncthreads();
    #pragma unroll
    for(int j=0; j < dgemv_bs ; j++){
       res+=A[0]*buff[j];
       A+=lda;
    }
  }
  __syncthreads();

  if (m>n1){
     buff[threadIdx.x]  = x[n1];

     __syncthreads();
     for(int j=0; j<(m-n1); j++){
         res += A[0]*buff[j];
         A+=lda;
     }
  }

  if (ind<n)
     y[ind] = res;
}

extern "C" void
magmablas_dgemv_tesla(int n, int m, double *A, int lda, double *x, double *z)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======

    This routine computes z = A x on the GPU.

    N      - (input) INTEGER.
             On entry, N specifies the number of rows of the matrix A.

    M      - (input) INTEGER.
             On entry, M specifies the number of columns of the matrix A

    A      - (input) DOUBLE PRECISION array of dimension ( LDA, m ) on the GPU.
   
    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) DOUBLE PRECISION array of dimension n.
     
    Z      - (output) DOUBLE PRECISION array of	dimension m. 
             On exit Z = A X.

    ===================================================================== */

    int blocks;
    if (n % num_threads==0)
        blocks = n/num_threads;
    else
        blocks = n/num_threads + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(num_threads, 1, 1);
 
    dgemv_kernel<<<grid, threads>>>(n, m, (m / dgemv_bs)*dgemv_bs, 
                                    A, lda, x, z);
}

__global__ void
dgemvt_kernel1(int n, int m, double alpha, int n1, double* A, int lda,
              double *x, double *y)
{
  const int inx = threadIdx.x;
  const int iny = threadIdx.y;

  int ind  = iny + __mul24(blockIdx.x,32);
  ind = inx + __mul24(ind,lda);
  int ind2 = inx + __mul24(iny,32);

  A += ind;
  x += ind2;

  double res = 0.f;

  __shared__ double buff[dgemv_bs];
  __shared__ double la[32][33];

  for(int i=0; i<n1; i += dgemv_bs ){
      buff[ind2]  = x[i];
      #pragma unroll
      for(int j=0; j<16; j++)
         la[iny+__mul24(j,2)][inx] = A[j*__mul24(2,lda)];

      __syncthreads();
      #pragma unroll
      for(int j=0; j < 16; j++)
        res += la[inx][iny*16+j]*buff[j+iny*16];

      A += 32;

      //===============================================
      #pragma unroll
      for(int j=0; j<16; j++)
         la[iny+__mul24(j,2)][inx] = A[j*__mul24(2,lda)];

      __syncthreads();

      #pragma unroll
      for(int j=0; j < 16; j++)
        res += la[inx][iny*16+j]*buff[j+32+iny*16];
      A += 32;
    }

    if (n>n1){
      if (ind2>=(n-n1))
         buff[ind2]=0.;
      else
         buff[ind2]  = x[n1];

      #pragma unroll
      for(int j=0; j<16; j++)
         la[iny+__mul24(j,2)][inx] = A[j*__mul24(2,lda)];

     __syncthreads();

     if (n-n1>16){
	#pragma unroll
        for(int j=0; j < 16; j++)
           res += la[inx][iny*16+j]*buff[j+iny*16];

        A += 32;
        #pragma unroll
        for(int j=0; j<16; j++)
          la[iny+__mul24(j,2)][inx] = A[j*__mul24(2,lda)];

	__syncthreads();

        #pragma unroll
        for(int j=0; j < 16; j++)
           res += la[inx][iny*16+j]*buff[j+32+iny*16];
     }
     else {
        #pragma unroll
        for(int j=0; j < 16; j++)
          res += la[inx][iny*16+j]*buff[j+iny*16];
     }
  }
  ind = inx + __mul24(blockIdx.x,32);

  la[inx][iny]= res;
  if (ind<m){
     res = la[inx][0] + la[inx][1];
     y[ind] = alpha*res;
  }
}

__global__ void
dgemvt_kernel2(int n, int m, double alpha,
		int n1, double* A, int lda, double *x, double *y)
{
  const int inx = threadIdx.x;
  const int iny = threadIdx.y;

  int ind  = iny + __mul24(blockIdx.x,16);
  ind = inx + __mul24(ind,lda);
  int ind2 = inx + __mul24(iny,16);
  if (ind2>31)
     ind2-=32;

  A += ind;
  x += ind2;
  if (ind2>31)
     ind2-=32;

  double res = 0.f;

  __shared__ double buff[32];
  __shared__ double la[16][17];

  for(int i=0; i<n1; i += 32 ){
     buff[ind2]  = x[i];
     #pragma unroll
     for(int j=0; j<4; j++)
        la[iny+__mul24(j,4)][inx] = A[j*__mul24(4,lda)];

     __syncthreads();
     #pragma unroll
     for(int j=0; j < 4; j++)
       res += la[inx][iny*4+j]*buff[j+iny*4];

     A += 16;
    __syncthreads();
     //===========================================
     #pragma unroll
     for(int j=0; j<4; j++)
         la[iny+__mul24(j,4)][inx] = A[j*__mul24(4,lda)];

     __syncthreads();

     #pragma unroll
     for(int j=0; j < 4; j++)
        res += la[inx][iny*4+j]*buff[j+16+iny*4];
     A += 16;
  }

  if (n>n1){
     if (ind2>=(n-n1))
        buff[ind2]=0.;
     else
        buff[ind2]  = x[n1];

     __syncthreads();
     #pragma unroll
     for(int j=0; j<4; j++)
         la[iny+__mul24(j,4)][inx] = A[j*__mul24(4,lda)];

     __syncthreads();
     if (n-n1>4){
        #pragma unroll
	for(int j=0; j < 4; j++)
           res += la[inx][iny*4+j]*buff[j+iny*4];

        A += 16;
        __syncthreads();
        #pragma unroll
          for(int j=0; j<4; j++)
            la[iny+__mul24(j,4)][inx] = A[j*__mul24(4,lda)];

        __syncthreads();

        #pragma unroll
        for(int j=0; j < 4; j++)
           res += la[inx][iny*4+j]*buff[j+16+iny*4];
     }
     else {
        #pragma unroll
        for(int j=0; j < 4; j++)
          res += la[inx][iny*4+j]*buff[j+iny*4];
     }
  }

  __syncthreads();
  ind = inx + __mul24(blockIdx.x,16);
  la[inx][iny]= res;
  __syncthreads();
  if (ind<m){
     res = la[inx][0] + la[inx][1] + la[inx][2] + la[inx][3];
     y[ind] = alpha*res;
  }
}

extern "C" void
magmablas_dgemvt1_tesla(int m, int n, double alpha, double *A, int lda,
                        double *x, double *z)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======

    This routine computes z = alpha A^t x on the GPU.
    Recommended for large M and N.

    M      - (input) INTEGER.
             On entry, M specifies the number of rows of the matrix A.

    N      - (input) INTEGER.
             On entry, N specifies the number of columns of the matrix A

    A      - (input) DOUBLE PRECISION array of dimension ( LDA, n ) on the GPU.

    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) DOUBLE PRECISION array of dimension m.

    Z      - (output) DOUBLE PRECISION array of dimension n.
             On exit Z = alpha A^t X.

    ===================================================================== */
    int blocks;

    if (n % 32==0)
        blocks = n/32;
    else
        blocks = n/32 + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(32, 2, 1);
    dgemvt_kernel1<<<grid, threads>>>(m, n, alpha, (m / dgemv_bs)*dgemv_bs,
                                      A, lda, x, z);
}

extern "C" void
magmablas_dgemvt2_tesla(int m, int n, double alpha, double *A, int lda,
                        double *x, double *z)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======

    This routine computes z = alpha A^t x on the GPU. Used in least squares
    solver for N small (e.g. = BS, a block size of order 64, 128, etc).

    M      - (input) INTEGER.
             On entry, M specifies the number of rows of the matrix A.

    N      - (input) INTEGER.
             On entry, N specifies the number of columns of the matrix A

    A      - (input) DOUBLE PRECISION array of dimension ( LDA, n ) on the GPU.

    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) DOUBLE PRECISION array of dimension m.

    Z      - (output) DOUBLE PRECISION array of dimension n.
             On exit Z = alpha A^t X.

    ===================================================================== */

    int blocks;

    if (n % 16==0)
        blocks = n/16;
    else
        blocks = n/16 + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(16, 4, 1);

    dgemvt_kernel2<<<grid, threads>>>(m, n, alpha, (m / 32)*32,
                                      A, lda, x, z);
}

extern "C" void
magmablas_dgemvt_tesla(int m, int n, double alpha, double *A, int lda,
                       double *x, double *z)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======

    This routine computes z = alpha A^t x on the GPU.

    M      - (input) INTEGER.
             On entry, M specifies the number of rows of the matrix A.

    N      - (input) INTEGER.
             On entry, N specifies the number of columns of the matrix A

    A      - (input) SINGLE PRECISION array of dimension ( LDA, n ) on the GPU.

    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension m.

    Z      - (output) SINGLE PRECISION array of dimension n.
             On exit Z = alpha A^t X.

    ===================================================================== */

    if (n<=128)
      magmablas_dgemvt2_tesla(m, n, alpha, A, lda, x, z);
    else
      magmablas_dgemvt1_tesla(m, n, alpha, A, lda, x, z);
}

#undef num_threads
#undef dgemv_bs
