#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010
*/
#include "common_magma.h"

#define magmablas_dgemv_tesla magmablas_dgemv
#define magmablas_dgemvt_tesla magmablas_dgemvt

#define num_threads 64
#define dgemv_bs 64

__global__ void 
dgemv_kernel(magma_int_t n, magma_int_t m, magma_int_t n1, double* A, magma_int_t lda, double *x, double *y)
{
  magma_int_t ind = blockIdx.x*num_threads + threadIdx.x;

  A += ind;
  x += threadIdx.x;

  double res = 0.f;

  __shared__ double buff[dgemv_bs];
  for(magma_int_t i=0; i<n1; i += dgemv_bs ){
    __syncthreads();
    buff[threadIdx.x]  = x[i];

    __syncthreads();
    #pragma unroll
    for(magma_int_t j=0; j < dgemv_bs ; j++){
       res+=A[0]*buff[j];
       A+=lda;
    }
  }
  __syncthreads();

  if (m>n1){
     buff[threadIdx.x]  = x[n1];

     __syncthreads();
     for(magma_int_t j=0; j<(m-n1); j++){
         res += A[0]*buff[j];
         A+=lda;
     }
  }

  if (ind<n)
     y[ind] = res;
}

extern "C" void
magmablas_dgemvt_tesla(magma_int_t m, magma_int_t n, double alpha, double *A, magma_int_t lda,
                       double *x, double *z);

extern "C" void
magmablas_dgemv_tesla(char trans,
                      magma_int_t m, magma_int_t n,
                      double alpha, 
                      double *A, magma_int_t lda, 
                      double *x, magma_int_t incx,
                      double beta,
                      double *z, magma_int_t incz)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======
    This routine computes:
    1) z =       A   x    if trans == 'N' or 'n', alpha == 1, beta == 0,
                          and incx == incz == 1 (using magmablas code)
    2) z = alpha A^t x    if trans == 'T' or 't', beta == 0,
                          and incx == incz == 1 (using magmablas code)
    3) z = alpha A^trans x + beta z
                          otherwise, using CUBLAS.

   Arguments
   ==========
    TRANS  - CHARACTER*1
             On entry, TRANS specifies the operation to be performed as
             follows:
               TRANS = 'N' or 'n'   z := alpha*A *x + beta*z
               TRANS = 'T' or 't'   z := alpha*A'*x + beta*z

    M      - (input) INTEGER
             On entry, M specifies the number of rows of the matrix A.

    N      - (input) INTEGER.
             On entry, N specifies the number of columns of the matrix A

    ALPHA  - DOUBLE PRECISION
             On entry, ALPHA specifies the scalar alpha.
             Unchanged on exit.

    A      - (input) DOUBLE PRECISION array of dimension ( LDA, n ) on the GPU.
   
    LDA    - (input) INTEGER
             LDA specifies the leading dimension of A.

    X      - (input) DOUBLE PRECISION array of dimension 
             n if trans == 'n'
             m if trans == 't'
      
    INCX   - (input) Specifies the increment for the elements of X.
             INCX must not be zero. Unchanged on exit.

    BETA   - DOUBLE PRECISION
             On entry, BETA specifies the scalar beta. When BETA is
             supplied as zero then Y need not be set on input.
             Unchanged on exit

    Z      - (output) DOUBLE PRECISION array of        dimension 
             m if trans == 'n'
             n if trans == 't' 
             
    INCZ  - (input) Specifies the increment for the elements of Z.
            INCZ must not be zero. Unchanged on exit.
    ===================================================================== */

    if (incx == 1 && incz == 1 && beta == 0.)
       if (trans == 'n' || trans == 'N')
          if (alpha == 1.)
            {
              magma_int_t blocks;
              if (m % num_threads==0)
                 blocks = m/num_threads;
              else
                 blocks = m/num_threads + 1;

              dim3 grid(blocks, 1, 1);
              dim3 threads(num_threads, 1, 1);
 
              dgemv_kernel<<< grid, threads, 0, magma_stream >>>(m, n, 
                                              (n/dgemv_bs)*dgemv_bs, 
                                              A, lda, x, z);
          }
         else
           hipblasDgemv(trans, m, n, alpha, A, lda,
                       x, incx, beta, z, incz);
       else
         magmablas_dgemvt_tesla(m, n, alpha, A, lda, x, z);
    else
      hipblasDgemv(trans, m, n, alpha, A, lda,
                  x, incx, beta, z, incz);

}

__global__ void
dgemvt_kernel1(magma_int_t n, magma_int_t m, double alpha, magma_int_t n1, double* A, magma_int_t lda,
              double *x, double *y)
{
  const magma_int_t inx = threadIdx.x;
  const magma_int_t iny = threadIdx.y;

  magma_int_t ind  = iny + __mul24(blockIdx.x,32);
  ind = inx + __mul24(ind,lda);
  magma_int_t ind2 = inx + __mul24(iny,32);

  A += ind;
  x += ind2;

  double res = 0.f;

  __shared__ double buff[dgemv_bs];
  __shared__ double la[32][33];

  for(magma_int_t i=0; i<n1; i += dgemv_bs ){
      buff[ind2]  = x[i];
      #pragma unroll
      for(magma_int_t j=0; j<16; j++)
         la[iny+__mul24(j,2)][inx] = A[j*__mul24(2,lda)];

      __syncthreads();
      #pragma unroll
      for(magma_int_t j=0; j < 16; j++)
        res += la[inx][iny*16+j]*buff[j+iny*16];

      A += 32;

      //===============================================
      #pragma unroll
      for(magma_int_t j=0; j<16; j++)
         la[iny+__mul24(j,2)][inx] = A[j*__mul24(2,lda)];

      __syncthreads();

      #pragma unroll
      for(magma_int_t j=0; j < 16; j++)
        res += la[inx][iny*16+j]*buff[j+32+iny*16];
      A += 32;
    }

    if (n>n1){
      if (ind2>=(n-n1))
         buff[ind2]=0.;
      else
         buff[ind2]  = x[n1];

      #pragma unroll
      for(magma_int_t j=0; j<16; j++)
         la[iny+__mul24(j,2)][inx] = A[j*__mul24(2,lda)];

     __syncthreads();

     if (n-n1>16){
        #pragma unroll
        for(magma_int_t j=0; j < 16; j++)
           res += la[inx][iny*16+j]*buff[j+iny*16];

        A += 32;
        #pragma unroll
        for(magma_int_t j=0; j<16; j++)
          la[iny+__mul24(j,2)][inx] = A[j*__mul24(2,lda)];

        __syncthreads();

        #pragma unroll
        for(magma_int_t j=0; j < 16; j++)
           res += la[inx][iny*16+j]*buff[j+32+iny*16];
     }
     else {
        #pragma unroll
        for(magma_int_t j=0; j < 16; j++)
          res += la[inx][iny*16+j]*buff[j+iny*16];
     }
  }
  ind = inx + __mul24(blockIdx.x,32);

  la[inx][iny]= res;
  if (ind<m){
     res = la[inx][0] + la[inx][1];
     y[ind] = alpha*res;
  }
}

__global__ void
dgemvt_kernel2(magma_int_t n, magma_int_t m, double alpha,
                magma_int_t n1, double* A, magma_int_t lda, double *x, double *y)
{
  const magma_int_t inx = threadIdx.x;
  const magma_int_t iny = threadIdx.y;

  magma_int_t ind  = iny + __mul24(blockIdx.x,16);
  ind = inx + __mul24(ind,lda);
  magma_int_t ind2 = inx + __mul24(iny,16);
  if (ind2>31)
     ind2-=32;

  A += ind;
  x += ind2;
  if (ind2>31)
     ind2-=32;

  double res = 0.f;

  __shared__ double buff[32];
  __shared__ double la[16][17];

  for(magma_int_t i=0; i<n1; i += 32 ){
     buff[ind2]  = x[i];
     #pragma unroll
     for(magma_int_t j=0; j<4; j++)
        la[iny+__mul24(j,4)][inx] = A[j*__mul24(4,lda)];

     __syncthreads();
     #pragma unroll
     for(magma_int_t j=0; j < 4; j++)
       res += la[inx][iny*4+j]*buff[j+iny*4];

     A += 16;
    __syncthreads();
     //===========================================
     #pragma unroll
     for(magma_int_t j=0; j<4; j++)
         la[iny+__mul24(j,4)][inx] = A[j*__mul24(4,lda)];

     __syncthreads();

     #pragma unroll
     for(magma_int_t j=0; j < 4; j++)
        res += la[inx][iny*4+j]*buff[j+16+iny*4];
     A += 16;
  }

  if (n>n1){
     if (ind2>=(n-n1))
        buff[ind2]=0.;
     else
        buff[ind2]  = x[n1];

     __syncthreads();
     #pragma unroll
     for(magma_int_t j=0; j<4; j++)
         la[iny+__mul24(j,4)][inx] = A[j*__mul24(4,lda)];

     __syncthreads();
     if (n-n1>4){
        #pragma unroll
        for(magma_int_t j=0; j < 4; j++)
           res += la[inx][iny*4+j]*buff[j+iny*4];

        A += 16;
        __syncthreads();
        #pragma unroll
          for(magma_int_t j=0; j<4; j++)
            la[iny+__mul24(j,4)][inx] = A[j*__mul24(4,lda)];

        __syncthreads();

        #pragma unroll
        for(magma_int_t j=0; j < 4; j++)
           res += la[inx][iny*4+j]*buff[j+16+iny*4];
     }
     else {
        #pragma unroll
        for(magma_int_t j=0; j < 4; j++)
          res += la[inx][iny*4+j]*buff[j+iny*4];
     }
  }

  __syncthreads();
  ind = inx + __mul24(blockIdx.x,16);
  la[inx][iny]= res;
  __syncthreads();
  if (ind<m){
     res = la[inx][0] + la[inx][1] + la[inx][2] + la[inx][3];
     y[ind] = alpha*res;
  }
}

extern "C" void
magmablas_dgemvt1_tesla(magma_int_t m, magma_int_t n, double alpha, double *A, magma_int_t lda,
                        double *x, double *z)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======

    This routine computes z = alpha A^t x on the GPU.
    Recommended for large M and N.

    M      - (input) INTEGER.
             On entry, M specifies the number of rows of the matrix A.

    N      - (input) INTEGER.
             On entry, N specifies the number of columns of the matrix A

    A      - (input) DOUBLE PRECISION array of dimension ( LDA, n ) on the GPU.

    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) DOUBLE PRECISION array of dimension m.

    Z      - (output) DOUBLE PRECISION array of dimension n.
             On exit Z = alpha A^t X.
    ===================================================================== */

    magma_int_t blocks;

    if (n % 32==0)
        blocks = n/32;
    else
        blocks = n/32 + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(32, 2, 1);
    dgemvt_kernel1<<< grid, threads, 0, magma_stream >>>(m, n, alpha, (m / dgemv_bs)*dgemv_bs,
                                      A, lda, x, z);
}

extern "C" void
magmablas_dgemvt2_tesla(magma_int_t m, magma_int_t n, double alpha, double *A, magma_int_t lda,
                        double *x, double *z)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======
    This routine computes z = alpha A^t x on the GPU. Used in least squares
    solver for N small (e.g. = BS, a block size of order 64, 128, etc).

    M      - (input) INTEGER.
             On entry, M specifies the number of rows of the matrix A.

    N      - (input) INTEGER.
             On entry, N specifies the number of columns of the matrix A

    A      - (input) DOUBLE PRECISION array of dimension ( LDA, n ) on the GPU.

    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) DOUBLE PRECISION array of dimension m.

    Z      - (output) DOUBLE PRECISION array of dimension n.
             On exit Z = alpha A^t X.
    ===================================================================== */

    magma_int_t blocks;

    if (n % 16==0)
        blocks = n/16;
    else
        blocks = n/16 + 1;

    dim3 grid(blocks, 1, 1);
    dim3 threads(16, 4, 1);

    dgemvt_kernel2<<< grid, threads, 0, magma_stream >>>(m, n, alpha, (m / 32)*32,
                                      A, lda, x, z);
}

extern "C" void
magmablas_dgemvt_tesla(magma_int_t m, magma_int_t n, double alpha, double *A, magma_int_t lda,
                       double *x, double *z)
{
/*  -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

    Purpose
    =======
    This routine computes z = alpha A^t x on the GPU.

    M      - (input) INTEGER.
             On entry, M specifies the number of rows of the matrix A.

    N      - (input) INTEGER.
             On entry, N specifies the number of columns of the matrix A

    A      - (input) SINGLE PRECISION array of dimension (LDA, n) on the GPU.

    LDA    - (input) INTEGER.
             LDA specifies the leading dimension of A.

    X      - (input) SINGLE PRECISION array of dimension m.

    Z      - (output) SINGLE PRECISION array of dimension n.
             On exit Z = alpha A^t X.
    ===================================================================== */

    if (n<=128)
      magmablas_dgemvt2_tesla(m, n, alpha, A, lda, x, z);
    else
      magmablas_dgemvt1_tesla(m, n, alpha, A, lda, x, z);
}

#undef num_threads
#undef dgemv_bs
