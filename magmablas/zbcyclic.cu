#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

       @precisions normal z -> s d c

*/
#include "common_magma.h"
#define PRECISION_z
#include "commonblas.h"

//===========================================================================
//  Set a matrix from CPU to multi-GPUs is 1D block cyclic distribution. 
//  The da arrays are pointers to the matrix data for the corresponding GPUs. 
//===========================================================================
extern "C" void 
magmablas_zsetmatrix_1D_bcyclic( int m, int n,
                                 hipDoubleComplex  *ha, int lda, 
                                 hipDoubleComplex  *da[], int ldda, 
                                 int num_gpus, int nb )
{
    int i, k, nk, cdevice;

    hipGetDevice(&cdevice);

    for(i=0; i<n; i+=nb){
       k = (i/nb)%num_gpus;
       hipSetDevice(k);
         
       nk = min(nb, n-i);
       //hipblasSetMatrix( m, nk, sizeof(hipDoubleComplex), ha+i*lda, lda,
       //                 da[k]+i/(nb*num_gpus)*nb*ldda, ldda);
       hipMemcpy2DAsync(da[k]+i/(nb*num_gpus)*nb*ldda, ldda*sizeof(hipDoubleComplex),
                         ha + i*lda, lda*sizeof(hipDoubleComplex),
                         sizeof(hipDoubleComplex)*m, nk,
                         hipMemcpyHostToDevice, NULL);
    }

    hipSetDevice(cdevice);
}


//===========================================================================
//  Get a matrix with 1D block cyclic distribution on multiGPUs to the CPU.
//  The da arrays are pointers to the matrix data for the corresponding GPUs.
//===========================================================================
extern "C" void
magmablas_zgetmatrix_1D_bcyclic( int m, int n,
                                 hipDoubleComplex  *da[], int ldda,
                                 hipDoubleComplex  *ha, int lda,
                                 int num_gpus, int nb )
{
    int i, k, nk, cdevice;

    hipGetDevice(&cdevice);

    for(i=0; i<n; i+=nb){
       k = (i/nb)%num_gpus;
       hipSetDevice(k);

       nk = min(nb, n-i);
       //hipblasGetMatrix( m, nk, sizeof(hipDoubleComplex),
       //                 da[k]+i/(nb*num_gpus)*nb*ldda, ldda,
       //                 ha+i*lda, lda);
       hipMemcpy2DAsync(ha + i*lda, lda*sizeof(hipDoubleComplex),
                         da[k]+i/(nb*num_gpus)*nb*ldda, ldda*sizeof(hipDoubleComplex),
                         sizeof(hipDoubleComplex)*m, nk,
                         hipMemcpyDeviceToHost, NULL);
    }
        
    hipSetDevice(cdevice);
}

