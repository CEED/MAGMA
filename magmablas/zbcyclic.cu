#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @author Stan Tomov
       @precisions normal z -> s d c
*/
#include "common_magma.h"
#define PRECISION_z
#include "commonblas.h"

//===========================================================================
//  Set a matrix from CPU to multi-GPUs is 1D block cyclic distribution.
//  The dA arrays are pointers to the matrix data for the corresponding GPUs.
//===========================================================================
extern "C" void
magmablas_zsetmatrix_1D_bcyclic( int m, int n,
                                 hipDoubleComplex  *hA,   int lda,
                                 hipDoubleComplex  *dA[], int ldda,
                                 int num_gpus, int nb )
{
    int i, d, nk, cdevice;

    magma_getdevice( &cdevice );

    for( i = 0; i < n; i += nb ) {
        d = (i/nb) % num_gpus;
        magma_setdevice( d );
        nk = min(nb, n-i);
        magma_zsetmatrix_async( m, nk,
                                hA + i*lda, lda,
                                dA[d] + i/(nb*num_gpus)*nb*ldda, ldda, NULL );
    }

    magma_setdevice( cdevice );
}


//===========================================================================
//  Get a matrix with 1D block cyclic distribution on multiGPUs to the CPU.
//  The dA arrays are pointers to the matrix data for the corresponding GPUs.
//===========================================================================
extern "C" void
magmablas_zgetmatrix_1D_bcyclic( int m, int n,
                                 hipDoubleComplex  *dA[], int ldda,
                                 hipDoubleComplex  *hA,   int lda,
                                 int num_gpus, int nb )
{
    int i, d, nk, cdevice;

    magma_getdevice( &cdevice );

    for( i = 0; i < n; i += nb ) {
        d = (i/nb) % num_gpus;
        magma_setdevice( d );
        nk = min(nb, n-i);
        magma_zgetmatrix_async( m, nk,
                                dA[d] + i/(nb*num_gpus)*nb*ldda, ldda,
                                hA + i*lda, lda, NULL );
    }

    magma_setdevice( cdevice );
}
