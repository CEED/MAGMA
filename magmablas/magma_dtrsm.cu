/*
    -- MAGMA (version 0.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       October 2009
*/

#include "hipblas.h"
#include "hip/hip_runtime.h"

#define BLOCK_SIZE 32 

__global__ void
dgemv32_kernel(int n, double alpha, double* A, int lda, double *x, double *y)
{
/*  -- MAGMA (version 0.1) --

    Purpose
    =======

    This routine computes y = alpha A x where A is double precision
    array of dimension (N, 32).
*/

    int ind = blockIdx.x*32 + threadIdx.x;

    A += ind;
    x += threadIdx.x;

    double res = 0.f;

    __shared__ double buff[32];
    buff[threadIdx.x]  = x[0];

    __syncthreads();
    #pragma unroll
    for(int j=0; j < 32; j++){
       res+=A[0]*buff[j];
       A+=lda;
    }

    if (ind<n)
      y[ind] = alpha*res;
}

__global__ void 
dgemvT32_kernel(int m, double alpha, double* A, int lda, double *x, double *y)
{
/*  -- MAGMA (version 0.1) --

    Purpose
    =======

    This routine computes y = alpha A^T x where A is double precision
    array of dimension (32, M).
*/
 
    const int inx = threadIdx.x;
    const int iny = threadIdx.y;

    int ind  = iny + __mul24(blockIdx.x,32);
    ind = inx + __mul24(ind,lda);
    int ind2 = inx + __mul24(iny,32);

    A += ind;
    x += inx;

    double res = 0.f;

    __shared__ double buff[64];
    __shared__ double la[32][33];

    buff[ind2]  = x[0];
    #pragma unroll
    for(int j=0; j<16; j++)
      la[iny+__mul24(2, j)][inx] = A[j*__mul24(2,lda)];

    __syncthreads();

    #pragma unroll
    for(int j=0; j < 16; j++)
      res += la[inx][j+iny*16]*buff[j+iny*16];

    ind = inx + __mul24(blockIdx.x,32);
    la[inx][iny]= res;

    __syncthreads();

    if (ind<m){
      res = la[inx][0] + la[inx][1];
      y[ind] = alpha*res;
    }
}

void magmablas_dgemv32(char tran, int n, double alpha, double *A, int lda,
		       double *x, double *y)
{
/*  -- MAGMA (version 0.1) --

    Purpose
    =======

    This routine computes
       y = alpha A^T x 	      	 for tran = 'T' / 't' or
      	y = alpha A x
    where A is double precision array of dimension (32, N) for
    tran = 'T' / 't', or of dimension (N, 32) otherwise.
*/

    int blocks;
    if (n % 32==0)
      blocks = n/32;
    else
      blocks = n/32 + 1;
    dim3 grid(blocks, 1, 1);

    if (tran == 'T' || tran == 't'){
      dim3 threads(32, 2, 1);
      dgemvT32_kernel<<<grid, threads>>>(n, alpha, A, lda, x, y);
    }
    else
    {
      dim3 threads(32, 1, 1);
      dgemv32_kernel<<<grid, threads>>>(n, alpha, A, lda, x, y);
    }
}

__global__ void
inplace_dgemm_kernel_T(int M, double alpha, double *A, int lda, double *B, int ldb)
{
        int i;
        double myvalue1=0, myvalue2= 0 ;
        double med;
        int tx = threadIdx.x;
        int ty = threadIdx.y;
        const int bx = blockIdx.x;
        __shared__ double As[BLOCK_SIZE][BLOCK_SIZE+1];
        __shared__ double Bs[BLOCK_SIZE/2][BLOCK_SIZE];

        A+= bx*32 + __mul24(lda,ty) + tx ;
        B+=      __mul24(ldb,tx) + ty ;

        As[tx][ty]=A[0];
        As[tx][ty+16]= A[16*lda];
        Bs[ty][tx]= B[0];
        __syncthreads();

        med  = As[tx][0];
        double py1 = Bs[ty][0] ;
        #pragma unroll
        for (i=0; i<31; i++){
                myvalue1 +=  med*py1;
                py1 = Bs[ty][i+1] ;
                med = As[tx][i+1];
        }
        myvalue1 += med*py1;

        Bs[ty][tx]= B[16];
        __syncthreads();

        med  = As[tx][0];
        double py2 = Bs[ty][0] ;
        #pragma unroll
        for (i=0; i<31; i++){
                myvalue2 +=  med*py2;
                py2 = Bs[ty][i+1] ;
                med = As[tx][i+1];
        }
        myvalue2 += med*py2;
        
		A[0] = alpha*myvalue1 ;
        A[lda*16] = alpha*myvalue2;
}

__global__ void
inplace_dgemm_kernel_N(int M, double alpha, double *A, int lda, double *B, int ldb)
{
        int i;
        double myvalue1=0, myvalue2= 0 ;
        double med;
        int tx = threadIdx.x;
        int ty = threadIdx.y;
        const int bx = blockIdx.x;
        __shared__ double As[BLOCK_SIZE][BLOCK_SIZE+1];
        __shared__ double Bs[BLOCK_SIZE/2][BLOCK_SIZE];

        A+= bx*32 + __mul24(lda,ty) + tx ;
        B+=      __mul24(ldb,ty) + tx ;

        As[tx][ ty]=A[0];
        As[tx][ty+16]= A[16*lda];
        Bs[ty][tx]= B[0];

        __syncthreads();
        med  = As[tx][0];
        double py1 = Bs[ty][0] ;

        #pragma unroll
        for (i=0; i<31; i++){
                myvalue1 +=  med*py1;
                py1 = Bs[ty][i+1] ;
                med  = As[tx][i+1];
        }
        myvalue1 +=  med*py1;

        Bs[ty][tx]= B[16*ldb];
        __syncthreads();

        med  = As[tx][0];
        double py2 = Bs[ty][0] ;
        #pragma unroll
        for (i=0; i<31; i++){
                myvalue2 +=  med*py2;
                py2 = Bs[ty][i+1] ;
                med  = As[tx][i+1];
        }
        myvalue2 +=  med*py2;

        A[0] = alpha*myvalue1 ;
        A[lda*16] = alpha*myvalue2;
}

__global__ void
diag_dtrtri_kernel (char uplo, char diag, double *A, double *d_dinvA, int lda)
{
	int i,j;
	double Ystx=0;
	double *Bw=NULL, *x=NULL, *y=NULL, *Aoff=NULL;
	double *my_d_dinvA;
	int switcher=0;

	// Thread index
	int tx = threadIdx.x;
	int txw;

	// Block index
	int bx = blockIdx.x;
		
	Aoff = A+bx*lda*BLOCK_SIZE+bx*BLOCK_SIZE;
	my_d_dinvA = d_dinvA+bx*BLOCK_SIZE*BLOCK_SIZE;

	__shared__ double Bs[BLOCK_SIZE*BLOCK_SIZE];
	__shared__ double workspace[BLOCK_SIZE]; // workspace used to store the current working column

	// load A
	#pragma unroll
	for (i=0; i<BLOCK_SIZE; i++)
		Bs[i*BLOCK_SIZE+tx] = *(Aoff+i*lda+tx);	// read in the whole square block of my A
												// not the upper or lower diagonal

	// Synchronize to make sure the matrices are loaded
	__syncthreads();

	Bs[tx*BLOCK_SIZE+tx] = ((diag=='u' || diag=='U')?1:(1/Bs[tx*BLOCK_SIZE+tx]));	// solve the diagonals

	if (uplo == 'l' || uplo == 'L')
	{
		/*
		 * the lower case
		 */
		if (tx < BLOCK_SIZE-1)
			Bs[(BLOCK_SIZE-1)*BLOCK_SIZE+tx] = 0;	//zero out the last column, except the diagonal element

		for (i=BLOCK_SIZE-2; i>=0; i--)
		{
			Ystx = 0;
			switcher = (tx>i);
			
			//dtrmv
			Bw = Bs+(i+1)*BLOCK_SIZE+i+1;
			workspace[tx] = *(Bs+i*BLOCK_SIZE+tx);
			x = workspace+i+1;
			y = Bs+i*BLOCK_SIZE;

			txw = (tx-i-1);

			#pragma unroll
			for (j=0; j<txw+1; j++)
				Ystx += (double)switcher*(*(Bw+j*BLOCK_SIZE+txw)*x[j]);

			//sscal
			switcher = (tx != i); 
			//if (tx !=i ) y[tx]=switcher*Ystx*(-Bs[i*BLOCK_SIZE+i]);
			y[tx] = (double)switcher*Ystx*(-Bs[i*BLOCK_SIZE+i])+(double)(!switcher)*y[tx];

			__syncthreads();
		}

	}
	else
	{
		 /* the upper case */
		for (i=0; i<BLOCK_SIZE; i++)
		{
			Ystx = 0;
			switcher = (double)(tx<i);
			
			//dtrmv
			workspace[tx] = *(Bs+i*BLOCK_SIZE+tx);
			y = Bs+i*BLOCK_SIZE;

			#pragma unroll
			for (j=tx; j<i; j++)
				Ystx += switcher*(*(Bs+j*BLOCK_SIZE+tx)*workspace[j]);

			//sscal
			switcher = (tx != i); // if (tx !=i ) y[tx]=switcher*Ystx*(-Bs[i*BLOCK_SIZE+i]);
			y[tx] = switcher*Ystx*(-Bs[i*BLOCK_SIZE+i])+!switcher*y[tx];

			__syncthreads();
		}


	}
		
	// write back A
	#pragma unroll
	for (i=0; i<BLOCK_SIZE; i++)
		*(my_d_dinvA+i*BLOCK_SIZE+tx) = Bs[i*BLOCK_SIZE+tx];
}

extern "C" void
inplace_dgemm (char tran, int M, double alpha, double *A, int lda, double *B, int ldb)
{
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE/2);

	if (tran == 'n' || tran == 'N')
		inplace_dgemm_kernel_N<<<M/BLOCK_SIZE,dimBlock>>>(M, alpha, A, lda, B, ldb); 
	else
		inplace_dgemm_kernel_T<<<M/BLOCK_SIZE,dimBlock>>>(M, alpha, A, lda, B, ldb); 
}

/*
 * magmablas_dtrsmx
 * the expert interface
 */

void magmablas_dtrsmx (char side, char uplo, char tran, char diag, int M, int N, double alpha, double* A, int lda, double* b, int ldb, double * d_dinvA)
{
/*  -- MAGMA (version 0.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       October 2009

	   Purpose
	   =======
	   
	   DTRSM  solves one of the matrix equations on GPU
	   
	      op( A )*X = alpha*B,   or   X*op( A ) = alpha*B,
	   
	   where alpha is a scalar, X and B are m by n matrices, A is a unit, or
	   non-unit,  upper or lower triangular matrix  and  op( A )  is one  of
	   
	      op( A ) = A   or   op( A ) = A'.
	   
	   The matrix X is overwritten on B.
	   
	   When M or N is not a multiple of blocking size, which is 32 for now, cublasDtrsm will
	   be called instead. There soon will not be this limitation both for arbitrary problem 
	   size and blocking size.
	   
	   
	   Arguments
	   ==========
	   
	   side   - CHARACTER*1.
	            On entry, side specifies whether op( A ) appears on the left
	            or right of X as follows:
	   
	               side = 'L' or 'l'   op( A )*X = alpha*B.
	   
	               side = 'R' or 'r'   X*op( A ) = alpha*B.
	   
	            Unchanged on exit.
	   
	   uplo   - CHARACTER*1.
	            On entry, uplo specifies whether the matrix A is an upper or
	            lower triangular matrix as follows:
	   
	               uplo = 'U' or 'u'   A is an upper triangular matrix.
	   
	               uplo = 'L' or 'l'   A is a lower triangular matrix.
	   
	            Unchanged on exit.
	   
	   tran - CHARACTER*1.
	            On entry, tran specifies the form of op( A ) to be used in
	            the matrix multiplication as follows:
	   
	               tran = 'N' or 'n'   op( A ) = A.
	   
	               tran = 'T' or 't'   op( A ) = A'.
	   
	               tran = 'C' or 'c'   op( A ) = A'.
	   
	            Unchanged on exit.
	   
	   diag   - CHARACTER*1.
	            On entry, diag specifies whether or not A is unit triangular
	            as follows:
	   
	               diag = 'U' or 'u'   A is assumed to be unit triangular.
	   
	               diag = 'N' or 'n'   A is not assumed to be unit
	                                   triangular.
	   
	            Unchanged on exit.
	   
	   m      - INTEGER.
	            On entry, m specifies the number of rows of B. m must be at
	            least zero.
	            Unchanged on exit.
	   
	    n      - INTEGER.
	             On entry, n specifies the number of columns of B.  n must be
	             at least zero.
	             Unchanged on exit.
	   
	    alpha  - DOUBLE PRECISION.
	             On entry,  alpha specifies the scalar  alpha. When  alpha is
	             zero then  A is not referenced and  B need not be set before
	             entry.
	             Unchanged on exit.
	   
	    A      - DOUBLE PRECISION             array of DIMENSION ( lda, k ), where k is m
	             when  side = 'L' or 'l'  and is  n  when  side = 'R' or 'r'.
	             Before entry  with  uplo = 'U' or 'u',  the  leading  k by k
	             upper triangular part of the array  A must contain the upper
	             triangular matrix  and the dtrictly lower triangular part of
	             A is not referenced.
	             Before entry  with  uplo = 'L' or 'l',  the  leading  k by k
	             lower triangular part of the array  A must contain the lower
	             triangular matrix  and the dtrictly upper triangular part of
	             A is not referenced.
	             Note that when  diag = 'U' or 'u',  the diagonal elements of
	             A  are not referenced either,  but are assumed to be  unity.
	             Unchanged on exit.
	   
	    lda    - INTEGER.
	             On entry, lda specifies the first dimension of A as declared
	             in the calling (sub) program.  When  side = 'L' or 'l'  then
	             lda  must be at least  max( 1, m ),  when  side = 'R' or 'r'
	             then lda must be at least max( 1, n ).
	             Unchanged on exit.
	   
	    b      - DOUBLE PRECISION array of DIMENSION ( ldb, n ).
	             Before entry,  the leading  m by n part of the array  B must
	             contain  the  right-hand  side  matrix  B,  and  on exit  is
	             overwritten by the solution matrix  X.
	   
	    ldb    - INTEGER.
	             On entry, ldb specifies the first dimension of B as declared
	             in  the  calling  (sub)  program.   ldb  must  be  at  least
	             max( 1, m ).
	             Unchanged on exit.

		d_dinvA -DOUBLE PRECISION array of DIMENSION (BLOCKSIZE, M) when side='L', 
				 (BLOCKSIZE, N) when side='R'. On exit this space is filled
			     with the inverse of blocks on the diagonal, each inverse is
				 of size BLOCKSIZE x BLOCKSIZE, and the leading dimension of
				 d_dinvA is BLOCKSIZE;
	   
	   
	    Level 3 Blas routine.
		*
    ===================================================================== */

	int i, nblocks;

	/* 
	 * call cublasDtrsm when size of the problem is not a multiple of blocksize which is 32
	 * subject to change soon
	 */
	if ((M%BLOCK_SIZE) != 0 || (N%BLOCK_SIZE) != 0 )
	{
		cublasDtrsm (side, uplo, tran, diag, M, N, alpha, A, lda, b, ldb);
		return;
	}

	if (d_dinvA == NULL)
		return;

	if (side == 'l' || side == 'L')
	{
		/* inverse the diagonals
		 * Allocate device memory for the inversed diagonal blocks, size=m*BLOCK_SIZE 
		 */
		nblocks = M/BLOCK_SIZE;
		diag_dtrtri_kernel<<<nblocks, BLOCK_SIZE>>>(uplo, diag, A, d_dinvA, lda);

		if (tran == 'N' || tran == 'n')
		/* the non-transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				if (N == 1)
					magmablas_dgemv32 ('N', BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, b);
				else
					cublasDgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, ldb, 0, b, ldb);  

				if (BLOCK_SIZE>=M)
					return;

				cublasDgemm ('N', 'N', M-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+BLOCK_SIZE, lda, b, ldb, alpha, b+BLOCK_SIZE, ldb);

				/* the rest blocks */
				for (i=BLOCK_SIZE; i<M; i+=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_dgemv32 ('N', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						cublasDgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  

					if (i+BLOCK_SIZE>=M)
						break;

					cublasDgemm ('N', 'N', M-i-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+i*lda+i+BLOCK_SIZE, lda, b+i, ldb, 1.0, b+i+BLOCK_SIZE, ldb);
				}
			}
			else
			{
			/* the upper case */

				/* handle the first block seperately with alpha */
				i = M-BLOCK_SIZE;
				if (N == 1)
					magmablas_dgemv32 ('N', BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
				else
					cublasDgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0.0, b+i, ldb); 
					
				if (i-BLOCK_SIZE<0)
					return;

				cublasDgemm ('N', 'N', i, N, BLOCK_SIZE, -1.0, A+i*lda, lda, b+i, ldb, alpha, b, ldb);

				/* the rest blocks */
				for (i=M-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_dgemv32 ('N', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						cublasDgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0.0, b+i, ldb); 

					if (i-BLOCK_SIZE<0)
						break;

					cublasDgemm ('N', 'N', i, N, BLOCK_SIZE, -1.0, A+i*lda, lda, b+i, ldb, 1.0, b, ldb);
				}
			}
		}
		else
		/* the transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				i=M-BLOCK_SIZE; 
				if (N == 1)
					magmablas_dgemv32 ('T', BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
				else
					cublasDgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  

				if (i-BLOCK_SIZE<0)
					return;

				cublasDgemm ('T', 'N', i, N, BLOCK_SIZE, -1.0, A+i, lda, b+i, ldb, alpha, b, ldb);

				/* the rest blocks */
				for (i=M-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_dgemv32 ('T', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						cublasDgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  

					if (i-BLOCK_SIZE<0)
						break;

					cublasDgemm ('T', 'N', i, N, BLOCK_SIZE, -1.0, A+i, lda, b+i, ldb, 1.0, b, ldb);
				}
			}
			else
			{
			/* the upper case */
					
				/* handle the first block seperately with alpha */
				if (N == 1)
					magmablas_dgemv32 ('T', BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, b);
				else
					cublasDgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, ldb, 0, b, ldb);  

				if (BLOCK_SIZE>=M)
					return;

				cublasDgemm ('T', 'N', M-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+(BLOCK_SIZE)*lda, lda, b, ldb, alpha, b+BLOCK_SIZE, ldb);

				/* the rest blocks */
				for (i=BLOCK_SIZE; i<M; i+=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_dgemv32 ('T', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						cublasDgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  
					
					if (i+BLOCK_SIZE>=M)
						break;

					cublasDgemm ('T', 'N', M-i-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+(i+BLOCK_SIZE)*lda+i, lda, b+i, ldb, 1.0, b+i+BLOCK_SIZE, ldb);
				}
			}
		}
	}
	else
	{	// side=R

		/* inverse the diagonals
		 * Allocate device memory for the inversed diagonal blocks, size=N*BLOCK_SIZE 
		 */
		nblocks = N/BLOCK_SIZE;
		diag_dtrtri_kernel<<<nblocks, BLOCK_SIZE>>>(uplo, diag, A, d_dinvA, lda);
		
		if (tran == 'N' || tran == 'n')
		/* the non-transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				i=N-BLOCK_SIZE;
				inplace_dgemm ('N', M, alpha, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

				if (i-BLOCK_SIZE<0)
					return;

				cublasDgemm ('N', 'N', M, i, BLOCK_SIZE, -1.0, b+ldb*i, ldb, A+i, lda, alpha, b, ldb);

				/* the rest blocks */
				for (i=N-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					inplace_dgemm ('N', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);
					
					if (i-BLOCK_SIZE<0)
						break;

					cublasDgemm ('N', 'N', M, i, BLOCK_SIZE, -1.0, b+ldb*i, ldb, A+i, lda, 1.0, b, ldb);
				}
			}
			else
			{
			/* the upper case */
				
				/* handle the first block seperately with alpha */
				inplace_dgemm ('N', M, alpha, b, ldb, d_dinvA, BLOCK_SIZE);

				if (BLOCK_SIZE>=N)
					return;

				cublasDgemm ('N', 'N', M, N-BLOCK_SIZE, BLOCK_SIZE, -1.0, b, ldb, A+(BLOCK_SIZE)*lda, lda, alpha, b+(BLOCK_SIZE)*ldb, ldb);
				
				
				/* the rest blocks */
				for (i=BLOCK_SIZE; i<N; i+=BLOCK_SIZE)
				{
					inplace_dgemm ('N', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

					if (i+BLOCK_SIZE>=N)
						break;

					cublasDgemm ('N', 'N', M, N-i-BLOCK_SIZE, BLOCK_SIZE, -1.0, b+i*ldb, ldb, A+(i+BLOCK_SIZE)*lda+i, lda, 1.0, b+(i+BLOCK_SIZE)*ldb, ldb);
				}
			}
		}
		else
		/* the transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				inplace_dgemm ('T', M, alpha, b, ldb, d_dinvA, BLOCK_SIZE);

				if (BLOCK_SIZE>=N)
					return;

				cublasDgemm ('N', 'T', M, N-BLOCK_SIZE, BLOCK_SIZE, -1.0, b, ldb, A+BLOCK_SIZE, lda, alpha, b+(BLOCK_SIZE)*ldb, ldb);

				/* the rest blocks */
				for (i=BLOCK_SIZE; i<N; i+=BLOCK_SIZE)
				{
					inplace_dgemm ('T', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

					if (i+BLOCK_SIZE>=N)
						break;

					cublasDgemm ('N', 'T', M, N-i-BLOCK_SIZE, BLOCK_SIZE, -1.0, b+ldb*i, ldb, A+i*lda+BLOCK_SIZE+i, lda, 1.0, b+(i+BLOCK_SIZE)*ldb, ldb);
				}
			}
			else
			{
			/* the upper case */
				
				/* handle the first block seperately with alpha */
				i=N-BLOCK_SIZE;
				inplace_dgemm ('T', M, alpha, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

				if (i-BLOCK_SIZE<0)
					return;

				cublasDgemm ('N', 'T', M, i, BLOCK_SIZE, -1.0, b+i*ldb, ldb, A+i*lda, lda, alpha, b, ldb);
				
				/* the rest blocks */
				for (i=N-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					inplace_dgemm ('T', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

					if (i-BLOCK_SIZE<0)
						break;

					cublasDgemm ('N', 'T', M, i, BLOCK_SIZE, -1.0, b+i*ldb, ldb, A+i*lda, lda, 1.0, b, ldb);
				}
			}
		}
	}
}

/*
 * magmablas_dtrsm
 */

extern "C"
void magmablas_dtrsm (char side, char uplo, char tran, char diag, int M, int N, double alpha, double* A, int lda, double* b, int ldb)
{
/*  -- MAGMA (version 0.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       October 2009

	   Purpose
	   =======
	   
	   DTRSM  solves one of the matrix equations on GPU
	   
	      op( A )*X = alpha*B,   or   X*op( A ) = alpha*B,
	   
	   where alpha is a scalar, X and B are m by n matrices, A is a unit, or
	   non-unit,  upper or lower triangular matrix  and  op( A )  is one  of
	   
	      op( A ) = A   or   op( A ) = A'.
	   
	   The matrix X is overwritten on B.
	   
	   When M or N is not a multiple of blocking size, which is 32 for now, cublasDtrsm will
	   be called instead. There soon will not be this limitation both for arbitrary problem 
	   size and blocking size.
	   
	   Arguments
	   ==========
	   
	   side   - CHARACTER*1.
	            On entry, side specifies whether op( A ) appears on the left
	            or right of X as follows:
	   
	               side = 'L' or 'l'   op( A )*X = alpha*B.
	   
	               side = 'R' or 'r'   X*op( A ) = alpha*B.
	   
	            Unchanged on exit.
	   
	   uplo   - CHARACTER*1.
	            On entry, uplo specifies whether the matrix A is an upper or
	            lower triangular matrix as follows:
	   
	               uplo = 'U' or 'u'   A is an upper triangular matrix.
	   
	               uplo = 'L' or 'l'   A is a lower triangular matrix.
	   
	            Unchanged on exit.
	   
	   tran - CHARACTER*1.
	            On entry, tran specifies the form of op( A ) to be used in
	            the matrix multiplication as follows:
	   
	               tran = 'N' or 'n'   op( A ) = A.
	   
	               tran = 'T' or 't'   op( A ) = A'.
	   
	               tran = 'C' or 'c'   op( A ) = A'.
	   
	            Unchanged on exit.
	   
	   diag   - CHARACTER*1.
	            On entry, diag specifies whether or not A is unit triangular
	            as follows:
	   
	               diag = 'U' or 'u'   A is assumed to be unit triangular.
	   
	               diag = 'N' or 'n'   A is not assumed to be unit
	                                   triangular.
	   
	            Unchanged on exit.
	   
	   m      - INTEGER.
	            On entry, m specifies the number of rows of B. m must be at
	            least zero.
	            Unchanged on exit.
	   
	    n      - INTEGER.
	             On entry, n specifies the number of columns of B.  n must be
	             at least zero.
	             Unchanged on exit.
	   
	    alpha  - DOUBLE PRECISION.
	             On entry,  alpha specifies the scalar  alpha. When  alpha is
	             zero then  A is not referenced and  B need not be set before
	             entry.
	             Unchanged on exit.
	   
	    A      - DOUBLE PRECISION             array of DIMENSION ( lda, k ), where k is m
	             when  side = 'L' or 'l'  and is  n  when  side = 'R' or 'r'.
	             Before entry  with  uplo = 'U' or 'u',  the  leading  k by k
	             upper triangular part of the array  A must contain the upper
	             triangular matrix  and the dtrictly lower triangular part of
	             A is not referenced.
	             Before entry  with  uplo = 'L' or 'l',  the  leading  k by k
	             lower triangular part of the array  A must contain the lower
	             triangular matrix  and the dtrictly upper triangular part of
	             A is not referenced.
	             Note that when  diag = 'U' or 'u',  the diagonal elements of
	             A  are not referenced either,  but are assumed to be  unity.
	             Unchanged on exit.
	   
	    lda    - INTEGER.
	             On entry, lda specifies the first dimension of A as declared
	             in the calling (sub) program.  When  side = 'L' or 'l'  then
	             lda  must be at least  max( 1, m ),  when  side = 'R' or 'r'
	             then lda must be at least max( 1, n ).
	             Unchanged on exit.
	   
	    b      - DOUBLE PRECISION             array of DIMENSION ( ldb, n ).
	             Before entry,  the leading  m by n part of the array  B must
	             contain  the  right-hand  side  matrix  B,  and  on exit  is
	             overwritten by the solution matrix  X.
	   
	    ldb    - INTEGER.
	             On entry, ldb specifies the first dimension of B as declared
	             in  the  calling  (sub)  program.   ldb  must  be  at  least
	             max( 1, m ).
	             Unchanged on exit.
	   
	   
	    Level 3 Blas routine.
		*
    ===================================================================== */

	int i, nblocks;
	double *d_dinvA;
	
	/* 
	 * call cublasDtrsm when size of the problem is not a multiple of blocksize which is 32
	 * subject to change soon
	 */
	if ((M%BLOCK_SIZE) != 0 || (N%BLOCK_SIZE) != 0 )
	{
		cublasDtrsm (side, uplo, tran, diag, M, N, alpha, A, lda, b, ldb);
		return;
	}

	if (side == 'l' || side == 'L')
	{
		/* inverse the diagonals
		 * Allocate device memory for the inversed diagonal blocks, size=m*BLOCK_SIZE 
		 */
		hipMalloc((void**)&d_dinvA, BLOCK_SIZE*M*sizeof(double));
		nblocks = M/BLOCK_SIZE;

		diag_dtrtri_kernel<<<nblocks, BLOCK_SIZE>>>(uplo, diag, A, d_dinvA, lda);

		if (tran == 'N' || tran == 'n')
		/* the non-transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				if (N == 1)
					magmablas_dgemv32 ('N', BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, b);
				else
					cublasDgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, ldb, 0, b, ldb);  

				if (BLOCK_SIZE>=M)
				{
					hipFree(d_dinvA);
					return;
				}

				cublasDgemm ('N', 'N', M-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+BLOCK_SIZE, lda, b, ldb, alpha, b+BLOCK_SIZE, ldb);

				/* the rest blocks */
				for (i=BLOCK_SIZE; i<M; i+=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_dgemv32 ('N', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						cublasDgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  

					if (i+BLOCK_SIZE>=M)
						break;

					cublasDgemm ('N', 'N', M-i-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+i*lda+i+BLOCK_SIZE, lda, b+i, ldb, 1.0, b+i+BLOCK_SIZE, ldb);
				}
			}
			else
			{
			/* the upper case */

				/* handle the first block seperately with alpha */
				i = M-BLOCK_SIZE;
				if (N == 1)
					magmablas_dgemv32 ('N', BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
				else
					cublasDgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0.0, b+i, ldb); 
					
				if (i-BLOCK_SIZE<0)
				{
					hipFree(d_dinvA);
					return;
				}

				cublasDgemm ('N', 'N', i, N, BLOCK_SIZE, -1.0, A+i*lda, lda, b+i, ldb, alpha, b, ldb);

				/* the rest blocks */
				for (i=M-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_dgemv32 ('N', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						cublasDgemm ('N', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0.0, b+i, ldb); 

					if (i-BLOCK_SIZE<0)
						break;

					cublasDgemm ('N', 'N', i, N, BLOCK_SIZE, -1.0, A+i*lda, lda, b+i, ldb, 1.0, b, ldb);
				}
			}
		}
		else
		/* the transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				i=M-BLOCK_SIZE; 
				if (N == 1)
					magmablas_dgemv32 ('T', BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
				else
					cublasDgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  

				if (i-BLOCK_SIZE<0)
				{
					hipFree(d_dinvA);
					return;
				}

				cublasDgemm ('T', 'N', i, N, BLOCK_SIZE, -1.0, A+i, lda, b+i, ldb, alpha, b, ldb);

				/* the rest blocks */
				for (i=M-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_dgemv32 ('T', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						cublasDgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  

					if (i-BLOCK_SIZE<0)
						break;

					cublasDgemm ('T', 'N', i, N, BLOCK_SIZE, -1.0, A+i, lda, b+i, ldb, 1.0, b, ldb);
				}
			}
			else
			{
			/* the upper case */
					
				/* handle the first block seperately with alpha */
				if (N == 1)
					magmablas_dgemv32 ('T', BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, b);
				else
					cublasDgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, alpha, d_dinvA, BLOCK_SIZE, b, ldb, 0, b, ldb);  

				if (BLOCK_SIZE>=M)
				{
					hipFree(d_dinvA);
					return;
				}

				cublasDgemm ('T', 'N', M-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+(BLOCK_SIZE)*lda, lda, b, ldb, alpha, b+BLOCK_SIZE, ldb);

				/* the rest blocks */
				for (i=BLOCK_SIZE; i<M; i+=BLOCK_SIZE)
				{
					if (N == 1)
						magmablas_dgemv32 ('T', BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, b+i);
					else
						cublasDgemm ('T', 'N', BLOCK_SIZE, N, BLOCK_SIZE, 1.0, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE, b+i, ldb, 0, b+i, ldb);  
					
					if (i+BLOCK_SIZE>=M)
						break;

					cublasDgemm ('T', 'N', M-i-BLOCK_SIZE, N, BLOCK_SIZE, -1.0, A+(i+BLOCK_SIZE)*lda+i, lda, b+i, ldb, 1.0, b+i+BLOCK_SIZE, ldb);
				}
			}
		}
	}
	else
	{	// side=R

		/* inverse the diagonals
		 * Allocate device memory for the inversed diagonal blocks, size=N*BLOCK_SIZE 
		 */
		hipMalloc((void**)&d_dinvA, BLOCK_SIZE*N*sizeof(double));
		nblocks = N/BLOCK_SIZE;
		diag_dtrtri_kernel<<<nblocks, BLOCK_SIZE>>>(uplo, diag, A, d_dinvA, lda);
		
		if (tran == 'N' || tran == 'n')
		/* the non-transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				i=N-BLOCK_SIZE;
				inplace_dgemm ('N', M, alpha, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

				if (i-BLOCK_SIZE<0)
				{
					hipFree(d_dinvA);
					return;
				}

				cublasDgemm ('N', 'N', M, i, BLOCK_SIZE, -1.0, b+ldb*i, ldb, A+i, lda, alpha, b, ldb);

				/* the rest blocks */
				for (i=N-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					inplace_dgemm ('N', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);
					
					if (i-BLOCK_SIZE<0)
						break;

					cublasDgemm ('N', 'N', M, i, BLOCK_SIZE, -1.0, b+ldb*i, ldb, A+i, lda, 1.0, b, ldb);
				}
			}
			else
			{
			/* the upper case */
				
				/* handle the first block seperately with alpha */
				inplace_dgemm ('N', M, alpha, b, ldb, d_dinvA, BLOCK_SIZE);

				if (BLOCK_SIZE>=N)
				{
					hipFree(d_dinvA);
					return;
				}

				cublasDgemm ('N', 'N', M, N-BLOCK_SIZE, BLOCK_SIZE, -1.0, b, ldb, A+(BLOCK_SIZE)*lda, lda, alpha, b+(BLOCK_SIZE)*ldb, ldb);
				
				
				/* the rest blocks */
				for (i=BLOCK_SIZE; i<N; i+=BLOCK_SIZE)
				{
					inplace_dgemm ('N', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

					if (i+BLOCK_SIZE>=N)
						break;

					cublasDgemm ('N', 'N', M, N-i-BLOCK_SIZE, BLOCK_SIZE, -1.0, b+i*ldb, ldb, A+(i+BLOCK_SIZE)*lda+i, lda, 1.0, b+(i+BLOCK_SIZE)*ldb, ldb);
				}
			}
		}
		else
		/* the transpose case */
		{
			if (uplo == 'L' || uplo == 'l')
			{
			/* the lower case */
				
				/* handle the first block seperately with alpha */
				inplace_dgemm ('T', M, alpha, b, ldb, d_dinvA, BLOCK_SIZE);

				if (BLOCK_SIZE>=N)
				{
					hipFree(d_dinvA);
					return;
				}

				cublasDgemm ('N', 'T', M, N-BLOCK_SIZE, BLOCK_SIZE, -1.0, b, ldb, A+BLOCK_SIZE, lda, alpha, b+(BLOCK_SIZE)*ldb, ldb);

				/* the rest blocks */
				for (i=BLOCK_SIZE; i<N; i+=BLOCK_SIZE)
				{
					inplace_dgemm ('T', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

					if (i+BLOCK_SIZE>=N)
						break;

					cublasDgemm ('N', 'T', M, N-i-BLOCK_SIZE, BLOCK_SIZE, -1.0, b+ldb*i, ldb, A+i*lda+BLOCK_SIZE+i, lda, 1.0, b+(i+BLOCK_SIZE)*ldb, ldb);
				}
			}
			else
			{
			/* the upper case */
				
				/* handle the first block seperately with alpha */
				i=N-BLOCK_SIZE;
				inplace_dgemm ('T', M, alpha, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

				if (i-BLOCK_SIZE<0)
				{
					hipFree(d_dinvA);
					return;
				}

				cublasDgemm ('N', 'T', M, i, BLOCK_SIZE, -1.0, b+i*ldb, ldb, A+i*lda, lda, alpha, b, ldb);
				
				/* the rest blocks */
				for (i=N-2*BLOCK_SIZE; i>=0; i-=BLOCK_SIZE)
				{
					inplace_dgemm ('T', M, 1.0, b+ldb*i, ldb, d_dinvA+i*BLOCK_SIZE, BLOCK_SIZE);

					if (i-BLOCK_SIZE<0)
						break;

					cublasDgemm ('N', 'T', M, i, BLOCK_SIZE, -1.0, b+i*ldb, ldb, A+i*lda, lda, 1.0, b, ldb);
				}
			}
		}
	}

	hipFree(d_dinvA);
}

