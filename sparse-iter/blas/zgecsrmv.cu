#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal z -> c d s

*/
#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 128
#else
   #define BLOCK_SIZE 512
#endif



__global__ void 
zgecsrmv_kernel( int m, 
                 magmaDoubleComplex alpha, 
                 magmaDoubleComplex *d_val, 
                 int *d_rowptr, 
                 int *d_colind,
                 magmaDoubleComplex *d_x,
                 magmaDoubleComplex beta, 
                 magmaDoubleComplex *d_y)
{
  int index = blockIdx.x*blockDim.x+threadIdx.x;
  int j;

  if(index<m){
    magmaDoubleComplex tmp = MAGMA_Z_ZERO;
    for( j=d_rowptr[index]; j<d_rowptr[index+1]; j++ ){
      tmp += d_val[j] * d_x[d_colind[j]];
    }
    d_y[index] = alpha * tmp + beta * d_y[index];
  }
}


extern "C" magma_int_t
magma_zgecsrmv(char transA,
               magma_int_t m, magma_int_t n,
               magmaDoubleComplex alpha,
               magmaDoubleComplex *d_val,
               magma_int_t *d_rowptr,
               magma_int_t *d_colind,
               magmaDoubleComplex *d_x,
               magmaDoubleComplex beta,
               magmaDoubleComplex *d_y)
{
/*  -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

    Purpose
    =======
    
    This routine computes y = alpha *  A^t *  x + beta * y on the GPU.
    
    Arguments
    =========

    magma_int_t m                   number of rows in A
    magma_int_t n                   number of columns in A 
    magmaDoubleComplex alpha        scalar multiplier
    magmaDoubleComplex *d_val       array containing values of A in CSR
    magma_int_t *d_rowptr           rowpointer of A in CSR
    magma_int_t *d_colind           columnindices of A in CSR
    magmaDoubleComplex *d_x         input vector x
    magmaDoubleComplex beta         scalar multiplier
    magmaDoubleComplex *d_y         input/output vector y

    =====================================================================    */

   dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);

   zgecsrmv_kernel<<< grid, BLOCK_SIZE, 0, magma_stream >>>(m, alpha,
                                                            d_val, d_rowptr, d_colind,
                                                            d_x, beta, d_y);

   return MAGMA_SUCCESS;
}



