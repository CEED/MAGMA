#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/
#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 256
#else
   #define BLOCK_SIZE 256
#endif


// CSR-SpMV kernel
__global__ void 
zgecsrmv_kernel( int num_rows, int num_cols, 
                 magmaDoubleComplex alpha, 
                 magmaDoubleComplex *d_val, 
                 magma_index_t *d_rowptr, 
                 magma_index_t *d_colind,
                 magmaDoubleComplex *d_x,
                 magmaDoubleComplex beta, 
                 magmaDoubleComplex *d_y){

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        magmaDoubleComplex dot = MAGMA_Z_ZERO;
        int start = d_rowptr[ row ];
        int end = d_rowptr[ row+1 ];
        for( j=start; j<end; j++)
            dot += d_val[ j ] * d_x[ d_colind[j] ];
        d_y[ row ] =  dot *alpha + beta * d_y[ row ];
    }
}

// shifted CSR-SpMV kernel
__global__ void 
zgecsrmv_kernel_shift( int num_rows, int num_cols, 
                       magmaDoubleComplex alpha, 
                       magmaDoubleComplex lambda, 
                       magmaDoubleComplex *d_val, 
                       magma_index_t *d_rowptr, 
                       magma_index_t *d_colind,
                       magmaDoubleComplex *d_x,
                       magmaDoubleComplex beta, 
                       int offset,
                       int blocksize,
                       magma_index_t *add_rows,
                       magmaDoubleComplex *d_y){

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        magmaDoubleComplex dot = MAGMA_Z_ZERO;
        int start = d_rowptr[ row ];
        int end = d_rowptr[ row+1 ];
        for( j=start; j<end; j++)
            dot += d_val[ j ] * d_x[ d_colind[j] ];
        if( row<blocksize )
            d_y[ row ] = dot * alpha - lambda 
                        * d_x[ offset+row ] + beta * d_y [ row ];
        else
            d_y[ row ] = dot * alpha - lambda 
                        * d_x[ add_rows[row-blocksize] ] + beta * d_y [ row ];   
    }
}


/**
    Purpose
    -------
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    The input format is CSR (val, row, col).
    
    Arguments
    ---------
    
    @param
    transA      magma_trans_t
                transposition parameter for A
                
    @param
    m           magma_int_t
                number of rows in A

    @param
    n           magma_int_t
                number of columns in A 

    @param
    alpha       magmaDoubleComplex
                scalar multiplier

    @param
    d_val       magmaDoubleComplex*
                array containing values of A in CSR

    @param
    d_rowptr    magma_int_t*
                rowpointer of A in CSR

    @param
    d_colind    magma_int_t*
                columnindices of A in CSR

    @param
    d_x         magmaDoubleComplex*
                input vector x

    @param
    beta        magmaDoubleComplex
                scalar multiplier

    @param
    d_y         magmaDoubleComplex*
                input/output vector y


    @ingroup magmasparse_z
    ********************************************************************/

extern "C" magma_int_t
magma_zgecsrmv(     magma_trans_t transA,
                    magma_int_t m, magma_int_t n,
                    magmaDoubleComplex alpha,
                    magmaDoubleComplex *d_val,
                    magma_index_t *d_rowptr,
                    magma_index_t *d_colind,
                    magmaDoubleComplex *d_x,
                    magmaDoubleComplex beta,
                    magmaDoubleComplex *d_y ){

    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);

    zgecsrmv_kernel<<< grid, BLOCK_SIZE, 0, magma_stream >>>
                    (m, n, alpha, d_val, d_rowptr, d_colind, d_x, beta, d_y);

    return MAGMA_SUCCESS;
}



/**
    Purpose
    -------
    
    This routine computes y = alpha * ( A -lambda I ) * x + beta * y on the GPU.
    It is a shifted version of the CSR-SpMV.
    
    Arguments
    ---------
    
    @param
    transA      magma_trans_t
                transposition parameter for A

    @param
    m           magma_int_t
                number of rows in A

    @param
    n           magma_int_t
                number of columns in A 

    @param
    alpha       magmaDoubleComplex
                scalar multiplier

    @param
    lambda      magmaDoubleComplex
                scalar multiplier

    @param
    d_val       magmaDoubleComplex*
                array containing values of A in CSR

    @param
    d_rowptr    magma_int_t*
                rowpointer of A in CSR

    @param
    d_colind    magma_int_t*
                columnindices of A in CSR

    @param
    d_x         magmaDoubleComplex*
                input vector x

    @param
    beta        magmaDoubleComplex
                scalar multiplier

    @param
    offset      magma_int_t 
                in case not the main diagonal is scaled
                
    @param
    blocksize   magma_int_t 
                in case of processing multiple vectors  
                
    @param
    add_rows    magma_int_t*
                in case the matrixpowerskernel is used
                
    @param
    d_y         magmaDoubleComplex*
                output vector y  

    @ingroup magmasparse_z
    ********************************************************************/

extern "C" magma_int_t
magma_zgecsrmv_shift( magma_trans_t transA,
                      magma_int_t m, magma_int_t n,
                      magmaDoubleComplex alpha,
                      magmaDoubleComplex lambda,
                      magmaDoubleComplex *d_val,
                      magma_index_t *d_rowptr,
                      magma_index_t *d_colind,
                      magmaDoubleComplex *d_x,
                      magmaDoubleComplex beta,
                      int offset,
                      int blocksize,
                      magma_index_t *add_rows,
                      magmaDoubleComplex *d_y ){

    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);

    zgecsrmv_kernel_shift<<< grid, BLOCK_SIZE, 0, magma_stream >>>
                         (m, n, alpha, lambda, d_val, d_rowptr, d_colind, d_x, 
                                    beta, offset, blocksize, add_rows, d_y);

    return MAGMA_SUCCESS;
}



