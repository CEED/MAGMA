#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/
#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 256
#else
   #define BLOCK_SIZE 256
#endif


// CSR-SpMV kernel
__global__ void 
zgecsrmv_kernel( int num_rows, int num_cols, 
                 magmaDoubleComplex alpha, 
                 magmaDoubleComplex *d_val, 
                 int *d_rowptr, 
                 int *d_colind,
                 magmaDoubleComplex *d_x,
                 magmaDoubleComplex beta, 
                 magmaDoubleComplex *d_y){

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        magmaDoubleComplex dot = MAGMA_Z_ZERO;
        int start = d_rowptr[ row ];
        int end = d_rowptr[ row+1 ];
        for( j=start; j<end; j++)
            dot += d_val[ j ] * d_x[ d_colind[j] ];
        d_y[ row ] =  dot *alpha + beta * d_y[ row ];
    }
}

// shifted CSR-SpMV kernel
__global__ void 
zgecsrmv_kernel_shift( int num_rows, int num_cols, 
                       magmaDoubleComplex alpha, 
                       magmaDoubleComplex lambda, 
                       magmaDoubleComplex *d_val, 
                       int *d_rowptr, 
                       int *d_colind,
                       magmaDoubleComplex *d_x,
                       magmaDoubleComplex beta, 
                       int offset,
                       int blocksize,
                       int *add_rows,
                       magmaDoubleComplex *d_y){

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        magmaDoubleComplex dot = MAGMA_Z_ZERO;
        int start = d_rowptr[ row ];
        int end = d_rowptr[ row+1 ];
        for( j=start; j<end; j++)
            dot += d_val[ j ] * d_x[ d_colind[j] ];
        if( row<blocksize )
            d_y[ row ] = dot * alpha - lambda 
                        * d_x[ offset+row ] + beta * d_y [ row ];
        else
            d_y[ row ] = dot * alpha - lambda 
                        * d_x[ add_rows[row-blocksize] ] + beta * d_y [ row ];   
    }
}


/*  -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

    Purpose
    =======
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    The input format is CSR (val, row, col).
    
    Arguments
    =========

    magma_int_t m                   number of rows in A
    magma_int_t n                   number of columns in A 
    magmaDoubleComplex alpha        scalar multiplier
    magmaDoubleComplex *d_val       array containing values of A in CSR
    magma_int_t *d_rowptr           rowpointer of A in CSR
    magma_int_t *d_colind           columnindices of A in CSR
    magmaDoubleComplex *d_x         input vector x
    magmaDoubleComplex beta         scalar multiplier
    magmaDoubleComplex *d_y         input/output vector y

    ======================================================================    */

extern "C" magma_int_t
magma_zgecsrmv(     magma_trans_t transA,
                    magma_int_t m, magma_int_t n,
                    magmaDoubleComplex alpha,
                    magmaDoubleComplex *d_val,
                    magma_int_t *d_rowptr,
                    magma_int_t *d_colind,
                    magmaDoubleComplex *d_x,
                    magmaDoubleComplex beta,
                    magmaDoubleComplex *d_y ){

    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);

    zgecsrmv_kernel<<< grid, BLOCK_SIZE, 0, magma_stream >>>
                    (m, n, alpha, d_val, d_rowptr, d_colind, d_x, beta, d_y);

    return MAGMA_SUCCESS;
}



/*  -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

    Purpose
    =======
    
    This routine computes y = alpha * ( A -lambda I ) * x + beta * y on the GPU.
    It is a shifted version of the CSR-SpMV.
    
    Arguments
    =========

    magma_int_t m                   number of rows in A
    magma_int_t n                   number of columns in A 
    magmaDoubleComplex alpha        scalar multiplier
    magmaDoubleComplex alpha        scalar multiplier
    magmaDoubleComplex *d_val       array containing values of A in CSR
    magma_int_t *d_rowptr           rowpointer of A in CSR
    magma_int_t *d_colind           columnindices of A in CSR
    magmaDoubleComplex *d_x         input vector x
    magmaDoubleComplex beta         scalar multiplier
    magmaDoubleComplex *d_y         input/output vector y

    ======================================================================    */

extern "C" magma_int_t
magma_zgecsrmv_shift( magma_trans_t transA,
                      magma_int_t m, magma_int_t n,
                      magmaDoubleComplex alpha,
                      magmaDoubleComplex lambda,
                      magmaDoubleComplex *d_val,
                      magma_int_t *d_rowptr,
                      magma_int_t *d_colind,
                      magmaDoubleComplex *d_x,
                      magmaDoubleComplex beta,
                      int offset,
                      int blocksize,
                      int *add_rows,
                      magmaDoubleComplex *d_y ){

    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);

    zgecsrmv_kernel_shift<<< grid, BLOCK_SIZE, 0, magma_stream >>>
                         (m, n, alpha, lambda, d_val, d_rowptr, d_colind, d_x, 
                                    beta, offset, blocksize, add_rows, d_y);

    return MAGMA_SUCCESS;
}



