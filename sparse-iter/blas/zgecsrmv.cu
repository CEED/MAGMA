#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/
#include "common_magma.h"

#define BLOCK_SIZE 256


// CSR-SpMV kernel
__global__ void 
zgecsrmv_kernel( 
    int num_rows, 
    int num_cols, 
    magmaDoubleComplex alpha, 
    magmaDoubleComplex * dval, 
    magma_index_t * drowptr, 
    magma_index_t * dcolind,
    magmaDoubleComplex * dx,
    magmaDoubleComplex beta, 
    magmaDoubleComplex * dy)
{
    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        magmaDoubleComplex dot = MAGMA_Z_ZERO;
        int start = drowptr[ row ];
        int end = drowptr[ row+1 ];
        for( j=start; j<end; j++)
            dot += dval[ j ] * dx[ dcolind[j] ];
        dy[ row ] =  dot *alpha + beta * dy[ row ];
    }
}

// shifted CSR-SpMV kernel
__global__ void 
zgecsrmv_kernel_shift( 
    int num_rows, 
    int num_cols, 
    magmaDoubleComplex alpha, 
    magmaDoubleComplex lambda, 
    magmaDoubleComplex * dval, 
    magma_index_t * drowptr, 
    magma_index_t * dcolind,
    magmaDoubleComplex * dx,
    magmaDoubleComplex beta, 
    int offset,
    int blocksize,
    magma_index_t * addrows,
    magmaDoubleComplex * dy)
{
    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;

    if(row<num_rows){
        magmaDoubleComplex dot = MAGMA_Z_ZERO;
        int start = drowptr[ row ];
        int end = drowptr[ row+1 ];
        for( j=start; j<end; j++)
            dot += dval[ j ] * dx[ dcolind[j] ];
        if( row<blocksize )
            dy[ row ] = dot * alpha - lambda 
                        * dx[ offset+row ] + beta * dy [ row ];
        else
            dy[ row ] = dot * alpha - lambda 
                        * dx[ addrows[row-blocksize] ] + beta * dy [ row ];   
    }
}


/**
    Purpose
    -------
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    The input format is CSR (val, row, col).
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_trans_t
                transposition parameter for A
                
    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 

    @param[in]
    alpha       magmaDoubleComplex
                scalar multiplier

    @param[in]
    dval        magmaDoubleComplex_ptr
                array containing values of A in CSR

    @param[in]
    drowptr     magmaIndex_ptr
                rowpointer of A in CSR

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in CSR

    @param[in]
    dx          magmaDoubleComplex_ptr
                input vector x

    @param[in]
    beta        magmaDoubleComplex
                scalar multiplier

    @param[out]
    dy          magmaDoubleComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zblas
    ********************************************************************/

extern "C" magma_int_t
magma_zgecsrmv(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    magmaDoubleComplex alpha,
    magmaDoubleComplex_ptr dval,
    magmaIndex_ptr drowptr,
    magmaIndex_ptr dcolind,
    magmaDoubleComplex_ptr dx,
    magmaDoubleComplex beta,
    magmaDoubleComplex_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( m, BLOCK_SIZE ) );
    magma_int_t threads = BLOCK_SIZE;
    zgecsrmv_kernel<<< grid, threads, 0, queue->hip_stream() >>>
                    (m, n, alpha, dval, drowptr, dcolind, dx, beta, dy);

    return MAGMA_SUCCESS;
}



/**
    Purpose
    -------
    
    This routine computes y = alpha * ( A -lambda I ) * x + beta * y on the GPU.
    It is a shifted version of the CSR-SpMV.
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_trans_t
                transposition parameter for A

    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 

    @param[in]
    alpha       magmaDoubleComplex
                scalar multiplier

    @param[in]
    lambda      magmaDoubleComplex
                scalar multiplier

    @param[in]
    dval        magmaDoubleComplex_ptr
                array containing values of A in CSR

    @param[in]
    drowptr     magmaIndex_ptr
                rowpointer of A in CSR

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in CSR

    @param[in]
    dx          magmaDoubleComplex_ptr
                input vector x

    @param[in]
    beta        magmaDoubleComplex
                scalar multiplier

    @param[in]
    offset      magma_int_t 
                in case not the main diagonal is scaled
                
    @param[in]
    blocksize   magma_int_t 
                in case of processing multiple vectors  
                
    @param[in]
    addrows     magmaIndex_ptr
                in case the matrixpowerskernel is used
                
    @param[out]
    dy          magmaDoubleComplex_ptr
                output vector y  
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zblas
    ********************************************************************/

extern "C" magma_int_t
magma_zgecsrmv_shift(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    magmaDoubleComplex alpha,
    magmaDoubleComplex lambda,
    magmaDoubleComplex_ptr dval,
    magmaIndex_ptr drowptr,
    magmaIndex_ptr dcolind,
    magmaDoubleComplex_ptr dx,
    magmaDoubleComplex beta,
    int offset,
    int blocksize,
    magma_index_t * addrows,
    magmaDoubleComplex_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( m, BLOCK_SIZE ) );
    magma_int_t threads = BLOCK_SIZE;
    zgecsrmv_kernel_shift<<< grid, threads, 0, queue->hip_stream() >>>
                         (m, n, alpha, lambda, dval, drowptr, dcolind, dx, 
                                    beta, offset, blocksize, addrows, dy);

    return MAGMA_SUCCESS;
}
