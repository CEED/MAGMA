#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/

#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 128
#else
   #define BLOCK_SIZE 512
#endif


#define  blockinfo(i,j)  blockinfo[(i)*c_blocks   + (j)]
#define  val(i,j) val+((blockinfo(i,j)-1)*size_b*size_b)



// every thread initializes one entry
__global__ void 
zbcsrblockinfo5_kernel( 
    magma_int_t num_blocks,
    magmaDoubleComplex * address,
    magmaDoubleComplex **AII )
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if( i < num_blocks ){
        *AII[ i ] = *address;
        if(i==0)
        printf("address: %d\n", address);
    }
}



/**
    Purpose
    -------
    
    For a Block-CSR ILU factorization, this routine copies the filled blocks
    from the original matrix A and initializes the blocks that will later be 
    filled in the factorization process with zeros.
    
    Arguments
    ---------


    @param[in]
    lustep      magma_int_t
                lustep

    @param[in]
    num_blocks  magma_int_t
                number of nonzero blocks

    @param[in]
    c_blocks    magma_int_t
                number of column-blocks
                
    @param[in]
    size_b      magma_int_t
                blocksize
                
    @param[in]
    blockinfo   magma_int_t*
                block filled? location?

    @param[in]
    val         magmaDoubleComplex*
                pointers to the nonzero blocks in A

    @param[in]
    AII         magmaDoubleComplex**
                pointers to the respective nonzero blocks in B

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_zbcsrblockinfo5(
    magma_int_t lustep,
    magma_int_t num_blocks, 
    magma_int_t c_blocks, 
    magma_int_t size_b,
    magma_index_t *blockinfo,
    magmaDoubleComplex_ptr val,
    magmaDoubleComplex_ptr *AII,
    magma_queue_t queue )
{
    dim3 dimBlock( BLOCK_SIZE, 1, 1 );

        int dimgrid = magma_ceildiv( num_blocks, BLOCK_SIZE );
        dim3 dimGrid( dimgrid, 1, 1 );


        printf("dim grid: %d x %d", dimgrid, BLOCK_SIZE);
        magmaDoubleComplex **hAII;
        magma_malloc((void **)&hAII, num_blocks*sizeof(magmaDoubleComplex*));

        for(int i=0; i<num_blocks; i++) {
           hAII[i] = val(lustep,lustep);
        }
        magma_setvector( num_blocks, sizeof(magmaDoubleComplex*), 
                                                            hAII, 1, AII, 1 );
/*
    magma_setvector( 1, sizeof(magmaDoubleComplex*), address, 1, daddress, 1 );
    zbcsrblockinfo5_kernel<<<dimGrid,dimBlock, 0, queue >>>
                        ( num_blocks, daddress, AII );

*/
        return MAGMA_SUCCESS;
}



