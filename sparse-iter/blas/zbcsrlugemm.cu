#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>  // include before magma.h

#include "magma.h"


#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 128
#else
   #define BLOCK_SIZE 512
#endif

#define PRECISION_z

#define  Ablockinfo(i,j)  Ablockinfo[(i)*c_blocks   + (j)]
#define  Bblockinfo(i,j)  Bblockinfo[(i)*c_blocks   + (j)]
#define A(i,j) ((Ablockinfo(i,j)-1)*size_b*size_b)
#define B(i,j) ((Bblockinfo(i,j)-1)*size_b*size_b)

//============================================================

#define ldb m
#define lda m
#define ldc m


#define fetch_x_A(i) (((i)<m*m)?Aval[i]:0)
#define fetch_x_B(i) (((i)<m*m)?B[i]:0)


// every multiprocessor handles one BCSR-block
__global__ void 
zbcsr_gemm_kernel32( 
                  int m,
                  int n,
                  int kblocks,   
                  double **Avals, 
                  double **Bval,
                  double **Cval)
{
#if (__CUDA_ARCH__ >= 200)

#if defined(PRECISION_d)
    const  int tx = threadIdx.x;
    const  int ty = threadIdx.y;
  
    const int idt = ty * 64 + tx;

    const int tx2 = idt%16;
    const int ty2 = idt/16;

    double xxB[4];
    double *B;

    int trackA = __mul24( ty2, lda) + tx2 ;
    double *Aval = Avals[blockIdx.z];

    __shared__ double Abs[64][65];
    __shared__ double  Bb[16][65];


    for(int j=ty2; j<64; j+=16){
        for(int y=tx2; y<64; y+=16){
           Abs[y][j] = fetch_x_A(trackA + y-tx2) ;
            }
        trackA += __mul24( 16, m);
    }

    for(int k=0; k<kblocks; k++){
        B = Bval[k];
        int trackB = tx2+ __mul24( ty2 * 16, ldb );

        // Prefetch part of B
          #pragma unroll
          for(int y=0; y<4; y++){
                 Bb[tx2][ty2*4+y] = fetch_x_B( trackB + y * ldb) ;
          }
        __syncthreads();    // this is necessary!!!

        double Axs[4];
        double Bxp[4];
        double Cb[16] = {0,0,0,0, 0,0,0,0, 0,0,0,0, 0,0,0,0};

        int k1;
        for(k1=0; k1<m-16; k1+=16)
        {
                trackB += 16;

                #pragma unroll
                for( int y=0; y<4; y++)
                        xxB[y] = fetch_x_B( trackB + y*ldb);
                #pragma unroll
                for( int j1=0;j1<16;j1++)
                {
                        #pragma unroll
                        for( int y=0; y<4; y++){
                                Axs[y] =  Abs[tx2+y*16][j1+k1] ;
                        }

                        #pragma unroll
                        for( int y=0; y<4; y++){
                                Bxp[y]= Bb[j1][ty2+y*16];
                        }

                        #pragma unroll
                        for( int x=0; x<4; x++)
                        {
                                #pragma unroll
                                for( int y=0; y<4; y++)
                                {
                                        Cb[x*4+y]  += Axs[x]*Bxp[y];
                                }
                        }

                }
                #pragma unroll
                for(int y=0; y<4; y++)
                        Bb[tx2][ty2*4 + y] = xxB[y];

                __syncthreads();     // this is necessary!!!
        }
        // Prepare where to write the result
        double *C = Cval[blockIdx.z * kblocks + k];
        C += tx2 + __mul24 (ty2 ,ldc);

        #pragma unroll
        for(int j1=0;j1<16;j1++)
        {

                #pragma unroll
                for( int y=0; y<4; y++)
                        Axs[y] =  Abs[tx2 + y*16][j1+k1] ;

                #pragma unroll
                for( int y=0; y<4; y++)
                        Bxp[y]= Bb[j1][ty2 + y*16];

                #pragma unroll
                for( int x=0; x<4; x++)
                {
                        #pragma unroll
                        for( int y=0;y<4; y++)
                        {
                                Cb[x*4 + y]  += Axs[x]*Bxp[y];
                        }
                }
        }   
        int gy = ty2;
        #pragma unroll
        for( int y=0;y<4;y++, gy+=16)
        {
                int gx = tx2;
        #pragma unroll
                for(int x=0;x<4;x++, gx+=16)
                {
                        if (gx < m && gy < n){
                              C[x*16] -= Cb[y+x*4];
                       }
                }
                C += ldc*16;
        }
      }
#endif

#endif
}

// every multiprocessor handles one BCSR-block
__global__ void 
zbcsr_gemm_kernel64( 
                  int m,
                  int n,
                  int kblocks,   
                  double **Avals, 
                  double **Bval,
                  double **Cval)
{
#if (__CUDA_ARCH__ >= 200)

#if defined(PRECISION_d)
    const  int tx = threadIdx.x;
    const  int ty = threadIdx.y;
  
    const int idt = ty * 64 + tx;

    const int tx2 = idt%16;
    const int ty2 = idt/16;

    double xxB[4];

    double *B;

    int trackA = __mul24( ty2, lda) + tx2 ;
    double *Aval = Avals[blockIdx.z];

    __shared__ double Abs[64][65];
    __shared__ double  Bb[16][65];


    for(int j=ty2; j<64; j+=16){
        for(int y=tx2; y<64; y+=16){
           Abs[y][j] = fetch_x_A(trackA + y-tx2) ;
            }
        trackA += __mul24( 16, m);
    }


    for(int k=0; k<kblocks; k++){

        B = Bval[k];
        int trackB = tx2+ __mul24( ty2 * 4, ldb );

        // Prefetch part of B
          #pragma unroll
          for(int y=0; y<4; y++){
                 Bb[tx2][ty2*4+y] = fetch_x_B( trackB + y * ldb) ;
          }

        __syncthreads();    // this is necessary!!!

        double Axs[4];
        double Bxp[4];

        double Cb[16] = {0,0,0,0, 0,0,0,0, 0,0,0,0, 0,0,0,0};

        int k1;
        for(k1=0; k1<m-16; k1+=16)
        {
                trackB += 16;

                #pragma unroll
                for( int y=0; y<4; y++)
                        xxB[y] = fetch_x_B( trackB + y*ldb);

                #pragma unroll
                for( int j1=0;j1<16;j1++)
                {
                        #pragma unroll
                        for( int y=0; y<4; y++){
                                Axs[y] =  Abs[tx2+y*16][j1+k1] ;
                        }

                        #pragma unroll
                        for( int y=0; y<4; y++){
                                Bxp[y]= Bb[j1][ty2+y*16];
                        }

                        #pragma unroll
                        for( int x=0; x<4; x++)
                        {
                                #pragma unroll
                                for( int y=0; y<4; y++)
                                {
                                        Cb[x*4+y]  += Axs[x]*Bxp[y];
                                }
                        }

                }

                __syncthreads();
                #pragma unroll
                for(int y=0; y<4; y++)
                        Bb[tx2][ty2*4 + y] = xxB[y];

                __syncthreads();     // this is necessary!!!

        }
        // Prepare where to write the result
        double *C = Cval[blockIdx.z * kblocks + k];
        C += tx2 + __mul24 (ty2 ,ldc);

        #pragma unroll
        for(int j1=0;j1<16;j1++)
        {

                #pragma unroll
                for( int y=0; y<4; y++)
                        Axs[y] =  Abs[tx2 + y*16][j1+k1] ;

                #pragma unroll
                for( int y=0; y<4; y++)
                        Bxp[y]= Bb[j1][ty2 + y*16];

                #pragma unroll
                for( int x=0; x<4; x++)
                {
                        #pragma unroll
                        for( int y=0;y<4; y++)
                        {
                                Cb[x*4 + y]  += Axs[x]*Bxp[y];
                        }
                }
        }   

        int gy = ty2;
        #pragma unroll
        for( int y=0;y<4;y++, gy+=16)
        {
                int gx = tx2;
        #pragma unroll
                for(int x=0;x<4;x++, gx+=16)
                {
                        if (gx < m && gy < n){
                              C[x*16] -= Cb[y+x*4];
                       }
                }

                C += ldc*16;
        }

      }
#endif

#endif
}





/**
    Purpose
    -------
    
    For a Block-CSR ILU factorization, this routine updates all blocks in
    the trailing matrix.
    
    Arguments
    ---------

    @param
    size_b      magma_int_t
                blocksize in BCSR

    @param
    num_brows   magma_int_t
                number of block rows

    @param
    kblocks     magma_int_t
                number of blocks in row

    @param
    dA          magmaDoubleComplex**
                input blocks of matrix A
                
    @param
    dB          magmaDoubleComplex**
                input blocks of matrix B
                
    @param
    dC          magmaDoubleComplex**
                output blocks of matrix C

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_zbcsrluegemm( magma_int_t size_b, 
                    magma_int_t num_brows,
                    magma_int_t kblocks,
                    magmaDoubleComplex **dA,  
                    magmaDoubleComplex **dB,  
                    magmaDoubleComplex **dC ){

#if defined(PRECISION_d)

    magma_int_t arch = magma_getdevice_arch();

    if ( arch < 200  ) {
        printf("error: magma_zbcsrluegemm needs a CUDA architecture"
               " with at least 48K shared memory (Fermi +).\n"
               "Please run zbcsrlu.cpp using CUBLAS batched.\n");
    
    }
    else {

    dim3 threads( 64, 4 );

    dim3 grid(1, 1, num_brows);
    zbcsr_gemm_kernel64<<< grid, threads, 0, magma_stream >>>( 
                  size_b, size_b, kblocks, dA, dB, dC );

    }

#else
    printf("error: currently only supported for double precision.\n"
           "Please run zbcsrlu.cpp using CUBLAS batched.\n");
#endif

    return MAGMA_SUCCESS;
}



