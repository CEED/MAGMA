#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/
#include "magmasparse_internal.h"

#define BLOCK_SIZE 512


__global__ void 
zmgecsrmv_kernel( 
    int num_rows, 
    int num_cols, 
    int num_vecs,
    magmaDoubleComplex alpha, 
    magmaDoubleComplex * dval, 
    magma_index_t * drowptr, 
    magma_index_t * dcolind,
    magmaDoubleComplex * dx,
    magmaDoubleComplex beta, 
    magmaDoubleComplex * dy)
{
    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;
    extern __shared__ magmaDoubleComplex dot[];

    if( row<num_rows ){
        for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x+ i*blockDim.x ] = MAGMA_Z_MAKE(0.0, 0.0);
        int start = drowptr[ row ];
        int end = drowptr[ row+1 ];
        for( j=start; j<end; j++ ){
            int col = dcolind [ j ];
            magmaDoubleComplex val = dval[ j ];
            for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x + i*blockDim.x ] += 
                                    val * dx[ col + i*num_cols ];
        }
        for( int i=0; i<num_vecs; i++ )
            dy[ row +i*num_cols ] = alpha * dot[ threadIdx.x + i*blockDim.x ] 
                                             + beta * dy[ row + i*num_cols ];
    }
}



/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is CSR. 
    
    Arguments
    ---------
    
    @param[in]
    transA      magma_trans_t
                transposition parameter for A

    @param[in]
    m           magma_int_t
                number of rows in A

    @param[in]
    n           magma_int_t
                number of columns in A 
                
    @param[in]
    num_vecs    mama_int_t
                number of vectors
    @param[in]
    alpha       magmaDoubleComplex
                scalar multiplier

    @param[in]
    dval        magmaDoubleComplex_ptr
                array containing values of A in CSR

    @param[in]
    drowptr     magmaIndex_ptr
                rowpointer of A in CSR

    @param[in]
    dcolind     magmaIndex_ptr
                columnindices of A in CSR

    @param[in]
    dx          magmaDoubleComplex_ptr
                input vector x

    @param[in]
    beta        magmaDoubleComplex
                scalar multiplier

    @param[out]
    dy          magmaDoubleComplex_ptr
                input/output vector y

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zblas
    ********************************************************************/

extern "C" magma_int_t
magma_zmgecsrmv(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    magma_int_t num_vecs, 
    magmaDoubleComplex alpha,
    magmaDoubleComplex_ptr dval,
    magmaIndex_ptr drowptr,
    magmaIndex_ptr dcolind,
    magmaDoubleComplex_ptr dx,
    magmaDoubleComplex beta,
    magmaDoubleComplex_ptr dy,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( m, BLOCK_SIZE ), 1, 1);
    magma_int_t threads = BLOCK_SIZE;
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                    * sizeof( magmaDoubleComplex ); // num_vecs vectors 
    zmgecsrmv_kernel<<< grid, threads, MEM_SIZE, queue->hip_stream()>>>
            (m, n, num_vecs, alpha, dval, drowptr, dcolind, dx, beta, dy);

   return MAGMA_SUCCESS;
}
