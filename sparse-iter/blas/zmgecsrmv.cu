#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/
#include "common_magma.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 128
#else
   #define BLOCK_SIZE 512
#endif



__global__ void 
zmgecsrmv_kernel( int num_rows, int num_cols, 
                  int num_vecs,
                  magmaDoubleComplex alpha, 
                  magmaDoubleComplex *d_val, 
                  magma_index_t *d_rowptr, 
                  magma_index_t *d_colind,
                  magmaDoubleComplex *d_x,
                  magmaDoubleComplex beta, 
                  magmaDoubleComplex *d_y){

    int row = blockIdx.x*blockDim.x+threadIdx.x;
    int j;
    extern __shared__ magmaDoubleComplex dot[];

    if( row<num_rows ){
        for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x+ i*blockDim.x ] = MAGMA_Z_MAKE(0.0, 0.0);
        int start = d_rowptr[ row ] ;
        int end = d_rowptr[ row+1 ];
        for( j=start; j<end; j++ ){
            int col = d_colind [ j ];
            magmaDoubleComplex val = d_val[ j ];
            for( int i=0; i<num_vecs; i++ )
                dot[ threadIdx.x + i*blockDim.x ] += 
                                    val * d_x[ col + i*num_cols ];
        }
        for( int i=0; i<num_vecs; i++ )
            d_y[ row +i*num_cols ] = alpha * dot[ threadIdx.x + i*blockDim.x ] 
                                             + beta * d_y[ row + i*num_cols ];
    }
}



/**
    Purpose
    -------
    
    This routine computes Y = alpha *  A *  X + beta * Y for X and Y sets of 
    num_vec vectors on the GPU. Input format is CSR. 
    
    Arguments
    ---------
    
    @param
    transA      magma_trans_t
                transposition parameter for A

    @param
    m           magma_int_t
                number of rows in A

    @param
    n           magma_int_t
                number of columns in A 
                
    @param
    num_vecs    mama_int_t
                number of vectors
    @param
    alpha       magmaDoubleComplex
                scalar multiplier

    @param
    d_val       magmaDoubleComplex*
                array containing values of A in CSR

    @param
    d_rowptr    magma_int_t*
                rowpointer of A in CSR

    @param
    d_colind    magma_int_t*
                columnindices of A in CSR

    @param
    d_x         magmaDoubleComplex*
                input vector x

    @param
    beta        magmaDoubleComplex
                scalar multiplier

    @param
    d_y         magmaDoubleComplex*
                input/output vector y


    @ingroup magmasparse_zblas
    ********************************************************************/

extern "C" magma_int_t
magma_zmgecsrmv(    magma_trans_t transA,
                    magma_int_t m, magma_int_t n,
                    magma_int_t num_vecs, 
                    magmaDoubleComplex alpha,
                    magmaDoubleComplex *d_val,
                    magma_index_t *d_rowptr,
                    magma_index_t *d_colind,
                    magmaDoubleComplex *d_x,
                    magmaDoubleComplex beta,
                    magmaDoubleComplex *d_y ){

    dim3 grid( (m+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    unsigned int MEM_SIZE =  num_vecs* BLOCK_SIZE 
                    * sizeof( magmaDoubleComplex ); // num_vecs vectors 
    zmgecsrmv_kernel<<< grid, BLOCK_SIZE, MEM_SIZE >>>
            (m, n, num_vecs, alpha, d_val, d_rowptr, d_colind, d_x, beta, d_y);

   return MAGMA_SUCCESS;
}



