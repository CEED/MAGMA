#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s
       @author Hartwig Anzt

*/
#include "common_magma.h"

#define BLOCK_SIZE 512

#define PRECISION_z


// These routines merge multiple kernels from zmergebicgstab into one
// The difference to zmergedbicgstab2 is that the SpMV is not merged into the
// kernes. This results in higher flexibility at the price of lower performance.

/* -------------------------------------------------------------------------- */

__global__ void 
magma_zbicgmerge1_kernel(  
                    int n, 
                    magmaDoubleComplex *skp,
                    magmaDoubleComplex *v, 
                    magmaDoubleComplex *r, 
                    magmaDoubleComplex *p ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    magmaDoubleComplex beta=skp[1];
    magmaDoubleComplex omega=skp[2];
    if( i<n ){
        p[i] =  r[i] + beta * ( p[i] - omega * v[i] );

    }

}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    p = beta*p
    p = p-omega*beta*v
    p = p+r
    
    -> p = r + beta * ( p - omega * v ) 

    Arguments
    ---------

    @param
    n           int
                dimension n

    @param
    skp         magmaDoubleComplex*
                set of scalar parameters

    @param
    v           magmaDoubleComplex*
                input v

    @param
    r           magmaDoubleComplex*
                input r

    @param
    p           magmaDoubleComplex*
                input/output p


    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" int
magma_zbicgmerge1(  int n, 
                    magmaDoubleComplex *skp,
                    magmaDoubleComplex *v, 
                    magmaDoubleComplex *r, 
                    magmaDoubleComplex *p ){

    
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( (n+BLOCK_SIZE-1)/BLOCK_SIZE );
    magma_zbicgmerge1_kernel<<<Gs, Bs, 0>>>( n, skp, v, r, p );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

__global__ void 
magma_zbicgmerge2_kernel(  
                    int n, 
                    magmaDoubleComplex *skp, 
                    magmaDoubleComplex *r,
                    magmaDoubleComplex *v, 
                    magmaDoubleComplex *s ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    magmaDoubleComplex alpha=skp[0];
    if( i<n ){
        s[i] =  r[i] - alpha * v[i] ;
    }

}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    s=r
    s=s-alpha*v
        
    -> s = r - alpha * v

    Arguments
    ---------

    @param
    n           int
                dimension n

    @param
    skp         magmaDoubleComplex*
                set of scalar parameters

    @param
    r           magmaDoubleComplex*
                input r

    @param
    v           magmaDoubleComplex*
                input v

    @param
    s           magmaDoubleComplex*
                input/output s


    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" int
magma_zbicgmerge2(  int n, 
                    magmaDoubleComplex *skp, 
                    magmaDoubleComplex *r,
                    magmaDoubleComplex *v, 
                    magmaDoubleComplex *s ){

    
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( (n+BLOCK_SIZE-1)/BLOCK_SIZE );

    magma_zbicgmerge2_kernel<<<Gs, Bs, 0>>>( n, skp, r, v, s );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

__global__ void 
magma_zbicgmerge3_kernel(  
                    int n, 
                    magmaDoubleComplex *skp, 
                    magmaDoubleComplex *p,
                    magmaDoubleComplex *se,
                    magmaDoubleComplex *t,
                    magmaDoubleComplex *x, 
                    magmaDoubleComplex *r ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    magmaDoubleComplex alpha=skp[0];
    magmaDoubleComplex omega=skp[2];
    if( i<n ){
        magmaDoubleComplex s;
        s = se[i];
        x[i] = x[i] + alpha * p[i] + omega * s;
        r[i] = s - omega * t[i];
    }

}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    x=x+alpha*p
    x=x+omega*s
    r=s
    r=r-omega*t
        
    -> x = x + alpha * p + omega * s
    -> r = s - omega * t

    Arguments
    ---------

    @param
    n           int
                dimension n

    @param
    skp         magmaDoubleComplex*
                set of scalar parameters

    @param
    p           magmaDoubleComplex*
                input p

    @param
    s           magmaDoubleComplex*
                input s

    @param
    t           magmaDoubleComplex*
                input t

    @param
    x           magmaDoubleComplex*
                input/output x

    @param
    r           magmaDoubleComplex*
                input/output r


    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" int
magma_zbicgmerge3(  int n, 
                    magmaDoubleComplex *skp,
                    magmaDoubleComplex *p,
                    magmaDoubleComplex *s,
                    magmaDoubleComplex *t,
                    magmaDoubleComplex *x, 
                    magmaDoubleComplex *r ){

    
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( (n+BLOCK_SIZE-1)/BLOCK_SIZE );
    magma_zbicgmerge3_kernel<<<Gs, Bs, 0>>>( n, skp, p, s, t, x, r );

   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

__global__ void 
magma_zbicgmerge4_kernel_1(  
                    magmaDoubleComplex *skp ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        magmaDoubleComplex tmp = skp[0];
        skp[0] = skp[4]/tmp;
    }
}

__global__ void 
magma_zbicgmerge4_kernel_2(  
                    magmaDoubleComplex *skp ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        skp[2] = skp[6]/skp[7];
        skp[3] = skp[4];
    }
}

__global__ void 
magma_zbicgmerge4_kernel_3(  
                    magmaDoubleComplex *skp ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        magmaDoubleComplex tmp1 = skp[4]/skp[3];
        magmaDoubleComplex tmp2 = skp[0] / skp[2];
        skp[1] =  tmp1*tmp2;
        //skp[1] =  skp[4]/skp[3] * skp[0] / skp[2];

    }
}

/**
    Purpose
    -------

    Performs some parameter operations for the BiCGSTAB with scalars on GPU.

    Arguments
    ---------

    @param
    type        int
                kernel type

    @param
    skp         magmaDoubleComplex*
                vector with parameters


    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" int
magma_zbicgmerge4(  int type, 
                    magmaDoubleComplex *skp ){

    dim3 Bs( 1 );
    dim3 Gs( 1 );
    if( type == 1 )
        magma_zbicgmerge4_kernel_1<<<Gs, Bs, 0>>>( skp );
    else if( type == 2 )
        magma_zbicgmerge4_kernel_2<<<Gs, Bs, 0>>>( skp );
    else if( type == 3 )
        magma_zbicgmerge4_kernel_3<<<Gs, Bs, 0>>>( skp );
    else
        printf("error: no kernel called\n");

   return MAGMA_SUCCESS;
}

