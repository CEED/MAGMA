#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s
       @author Hartwig Anzt

*/
#include "common_magma.h"

#define BLOCK_SIZE 512

#define PRECISION_z


// These routines merge multiple kernels from bicgstab into one.

/* -------------------------------------------------------------------------- */

__global__ void
magma_zbicgstab_1_kernel(  
    int num_rows, 
    int num_cols, 
    magmaDoubleComplex beta,
    magmaDoubleComplex omega,
    magmaDoubleComplex *r, 
    magmaDoubleComplex *v,
    magmaDoubleComplex *p )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            p[ i+j*num_rows ] = r[ i+j*num_rows ] + 
                beta * ( p[ i+j*num_rows ] - omega * v[ i+j*num_rows ] );
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    p = r + beta * ( p - omega * v )
    
    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    beta        magmaDoubleComplex
                scalar
                
    @param[in]
    omega       magmaDoubleComplex
                scalar
                
    @param[in]
    r           magmaDoubleComplex_ptr 
                vector
                
    @param[in]
    v           magmaDoubleComplex_ptr 
                vector
                
    @param[in,out]
    p           magmaDoubleComplex_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_zbicgstab_1(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaDoubleComplex beta,
    magmaDoubleComplex omega,
    magmaDoubleComplex_ptr r, 
    magmaDoubleComplex_ptr v,
    magmaDoubleComplex_ptr p,
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_zbicgstab_1_kernel<<<Gs, Bs, 0, queue>>>( num_rows, num_cols, beta, omega,
                     r, v, p );

   return MAGMA_SUCCESS;
}





__global__ void
magma_zbicgstab_2_kernel(  
    int num_rows,
    int num_cols,
    magmaDoubleComplex alpha,
    magmaDoubleComplex_ptr r,
    magmaDoubleComplex_ptr v,
    magmaDoubleComplex_ptr s )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            s[ i+j*num_rows ] = r[ i+j*num_rows ] - alpha * v[ i+j*num_rows ];
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    s = r - alpha v

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    alpha       magmaDoubleComplex
                scalar
                
    @param[in]
    r           magmaDoubleComplex_ptr 
                vector
                
    @param[in]
    v           magmaDoubleComplex_ptr 
                vector

    @param[in,out]
    s           magmaDoubleComplex_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_zbicgstab_2(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaDoubleComplex alpha,
    magmaDoubleComplex_ptr r,
    magmaDoubleComplex_ptr v,
    magmaDoubleComplex_ptr s, 
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_zbicgstab_2_kernel<<<Gs, Bs, 0, queue>>>( num_rows, num_cols, alpha, r, v, s );

   return MAGMA_SUCCESS;
}





__global__ void
magma_zbicgstab_3_kernel(  
    int num_rows,
    int num_cols,
    magmaDoubleComplex alpha,
    magmaDoubleComplex omega,
    magmaDoubleComplex *p,
    magmaDoubleComplex *s,
    magmaDoubleComplex *t,
    magmaDoubleComplex *x,
    magmaDoubleComplex *r )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            magmaDoubleComplex tmp = s[ i+j*num_rows ];
            x[ i+j*num_rows ] = x[ i+j*num_rows ] 
                        + alpha * p[ i+j*num_rows ] + omega * tmp;
            r[ i+j*num_rows ] = tmp - omega * t[ i+j*num_rows ];
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    x = x + alpha * p + omega * s
    r = s - omega * t

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    alpha       magmaDoubleComplex
                scalar
                
    @param[in]
    omega       magmaDoubleComplex
                scalar
                
    @param[in]
    p           magmaDoubleComplex_ptr 
                vector
                    
    @param[in]
    s           magmaDoubleComplex_ptr 
                vector
                    
    @param[in]
    t           magmaDoubleComplex_ptr 
                vector

    @param[in,out]
    x           magmaDoubleComplex_ptr 
                vector
                
    @param[in,out]
    r           magmaDoubleComplex_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_zbicgstab_3(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaDoubleComplex alpha,
    magmaDoubleComplex omega,
    magmaDoubleComplex_ptr p,
    magmaDoubleComplex_ptr s,
    magmaDoubleComplex_ptr t,
    magmaDoubleComplex_ptr x,
    magmaDoubleComplex_ptr r,
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_zbicgstab_3_kernel<<<Gs, Bs, 0, queue>>>( num_rows, num_cols, alpha, omega, p, s, t, x, r );

   return MAGMA_SUCCESS;
}



