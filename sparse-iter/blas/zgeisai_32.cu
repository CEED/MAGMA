#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/
#include "common_magmasparse.h"

#define PRECISION_z
#define COMPLEX
#define BLOCKSIZE 32
#define WARP_SIZE 32
#define WRP 32
#define WRQ 4


#include <hip/hip_runtime.h>  // for CUDA_VERSION

#if (CUDA_VERSION <= 6000) // this won't work, just to have something...
// CUDA 6.5 adds Double precision version; here's an implementation for CUDA 6.0 and earlier.
// from https://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/
__device__ inline
real_Double_t __shfl(real_Double_t var, unsigned int srcLane, int width=32) {
  int2 a = *reinterpret_cast<int2*>(&var);
  a.x = __shfl(a.x, srcLane, width);
  a.y = __shfl(a.y, srcLane, width);
  return *reinterpret_cast<double*>(&a);
}
#endif
         
                                                                                                


__device__
void ztrsv_lower_32kernel_general(magmaDoubleComplex *dA, magmaDoubleComplex *dB, int *sizes)
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    
    magmaDoubleComplex rB[ 2 ];
    magmaDoubleComplex rA[ 2 ];

    int n;
    int k;
    int N = sizes[j];

    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;
    

    // Read B to regs.
    #pragma unroll
    for (n = 0; n < 2; n++)
        rB[n] = dB[n*WARP_SIZE+idn];
    

    // Triangular solve in regs.
    #pragma unroll
    for (k = 0; k < N; k++)
    {
        #pragma unroll
        for (n = 0; n < 2; n++)
            rA[n] = dA[k*WARP_SIZE+n*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB[k/WARP_SIZE] /= rA[k/WARP_SIZE];
        
        magmaDoubleComplex top = __shfl(rB[k/WARP_SIZE], k%WARP_SIZE);

        #pragma unroll
        for (n = 0; n < 2; n++)
          if (n*WARP_SIZE+idn > k)
            rB[n] -= (top*rA[n]);
    }
    // Drop B to dev mem.
    #pragma unroll
    for (n = 0; n < 2; n++)
        if (n*WARP_SIZE+idn < N)
            dB[n*WARP_SIZE+idn] = rB[n];
        
#endif  
}


__device__
void ztrsv_upper_32kernel_general(magmaDoubleComplex *dA, magmaDoubleComplex *dB, int *sizes)
{
#ifdef REAL
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int idn = threadIdx.x;
    
    magmaDoubleComplex rB[ 2 ];
    magmaDoubleComplex rA[ 2 ];

    int n;
    int N = sizes[j];

    dA += (j)*WARP_SIZE*WARP_SIZE;
    dB += (j)*WARP_SIZE;
    

    // Read B to regs.
    #pragma unroll
    for (n = 0; n < 2; n++)
        rB[n] = dB[n*WARP_SIZE+idn];
    

    // Triangular solve in regs.
    #pragma unroll
    for (int k = N-1; k > -1; k--)
    {
        #pragma unroll
        for (n = 0; n < 2; n++)
            rA[n] = dA[k*WARP_SIZE+n*WARP_SIZE+idn];
        if (k%WARP_SIZE == idn)
            rB[k/WARP_SIZE] /= rA[k/WARP_SIZE];
        
        magmaDoubleComplex top = __shfl(rB[k/WARP_SIZE], k%WARP_SIZE);

        #pragma unroll
        for (n = 0; n < 2; n++)
          if (n*WARP_SIZE+idn < k)
            rB[n] -= (top*rA[n]);
    }
    // Drop B to dev mem.
    #pragma unroll
    for (n = 0; n < 2; n++)
        if (n*WARP_SIZE+idn < N)
            dB[n*WARP_SIZE+idn] = rB[n];
        
#endif  
}



__device__                                                                                          
void ztrsv_lower_32kernel_1(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 1; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_2(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 2; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_3(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 3; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_4(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 4; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_5(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 5; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_6(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 6; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_7(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 7; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_8(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 8; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_9(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 9; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_10(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 10; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_11(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 11; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_12(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 12; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_13(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 13; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_14(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 14; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_15(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 15; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_16(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 16; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_17(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 17; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_18(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 18; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_19(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 19; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_20(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 20; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_21(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 21; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_22(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 22; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_23(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 23; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_24(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 24; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_25(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 25; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_26(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 26; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_27(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 27; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_28(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 28; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_29(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 29; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_30(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 30; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_31(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 31; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_lower_32kernel_32(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 32; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( idn > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                




                                                                                                
__global__                                                                                              
void ztrsv_lower_32kernel_switch(magmaDoubleComplex *dA, magmaDoubleComplex *dB, int *sizes, int num_rows )
{                                                                                                       
                                                                                                        
                                                                                                        
                                                                                                        
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                        
    if( j < num_rows ){                                                                                 
    int N = sizes[j];                                                                                   
    switch( N ) {                                                                                       
        case  1:                                                                                       
            ztrsv_lower_32kernel_1( dA, dB ); break;                                                     
        case  2:                                                                                       
            ztrsv_lower_32kernel_2( dA, dB ); break;                                                     
        case  3:                                                                                       
            ztrsv_lower_32kernel_3( dA, dB ); break;                                                     
        case  4:                                                                                       
            ztrsv_lower_32kernel_4( dA, dB ); break;                                                     
        case  5:                                                                                       
            ztrsv_lower_32kernel_5( dA, dB ); break;                                                     
        case  6:                                                                                       
            ztrsv_lower_32kernel_6( dA, dB ); break;                                                     
        case  7:                                                                                       
            ztrsv_lower_32kernel_7( dA, dB ); break;                                                     
        case  8:                                                                                       
            ztrsv_lower_32kernel_8( dA, dB ); break;                                                     
        case  9:                                                                                       
            ztrsv_lower_32kernel_9( dA, dB ); break;                                                     
        case  10:                                                                                       
            ztrsv_lower_32kernel_10( dA, dB ); break;                                                     
        case  11:                                                                                       
            ztrsv_lower_32kernel_11( dA, dB ); break;                                                     
        case  12:                                                                                       
            ztrsv_lower_32kernel_12( dA, dB ); break;                                                     
        case  13:                                                                                       
            ztrsv_lower_32kernel_13( dA, dB ); break;                                                     
        case  14:                                                                                       
            ztrsv_lower_32kernel_14( dA, dB ); break;                                                     
        case  15:                                                                                       
            ztrsv_lower_32kernel_15( dA, dB ); break;                                                     
        case  16:                                                                                       
            ztrsv_lower_32kernel_16( dA, dB ); break;                                                     
        case  17:                                                                                       
            ztrsv_lower_32kernel_17( dA, dB ); break;                                                     
        case  18:                                                                                       
            ztrsv_lower_32kernel_18( dA, dB ); break;                                                     
        case  19:                                                                                       
            ztrsv_lower_32kernel_19( dA, dB ); break;                                                     
        case  20:                                                                                       
            ztrsv_lower_32kernel_20( dA, dB ); break;                                                     
        case  21:                                                                                       
            ztrsv_lower_32kernel_21( dA, dB ); break;                                                     
        case  22:                                                                                       
            ztrsv_lower_32kernel_22( dA, dB ); break;                                                     
        case  23:                                                                                       
            ztrsv_lower_32kernel_23( dA, dB ); break;                                                     
        case  24:                                                                                       
            ztrsv_lower_32kernel_24( dA, dB ); break;                                                     
        case  25:                                                                                       
            ztrsv_lower_32kernel_25( dA, dB ); break;                                                     
        case  26:                                                                                       
            ztrsv_lower_32kernel_26( dA, dB ); break;                                                     
        case  27:                                                                                       
            ztrsv_lower_32kernel_27( dA, dB ); break;                                                     
        case  28:                                                                                       
            ztrsv_lower_32kernel_28( dA, dB ); break;                                                     
        case  29:                                                                                       
            ztrsv_lower_32kernel_29( dA, dB ); break;                                                     
        case  30:                                                                                       
            ztrsv_lower_32kernel_30( dA, dB ); break;                                                     
        case  31:                                                                                       
            ztrsv_lower_32kernel_31( dA, dB ); break;                                                     
        case  32:                                                                                       
            ztrsv_lower_32kernel_32( dA, dB ); break;                                                     
        default:                                                                                        
            ztrsv_lower_32kernel_general( dA, dB, sizes );break;                                          
    }                                                                                                   
    }                                                                                                   
}                                                                                                       
__device__                                                                                          
void ztrsv_upper_32kernel_1(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 1-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_2(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 2-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_3(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 3-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_4(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 4-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_5(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 5-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_6(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 6-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_7(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 7-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_8(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 8-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_9(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 9-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_10(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 10-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_11(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 11-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_12(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 12-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_13(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 13-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_14(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 14-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_15(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 15-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_16(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 16-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_17(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 17-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_18(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 18-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_19(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 19-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_20(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 20-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_21(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 21-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_22(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 22-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_23(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 23-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_24(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 24-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_25(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 25-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_26(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 26-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_27(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 27-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_28(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 28-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_29(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 29-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_30(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 30-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_31(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 31-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                
__device__                                                                                          
void ztrsv_upper_32kernel_32(magmaDoubleComplex *dA, magmaDoubleComplex *dB )                         
{                                                                                                   
#ifdef REAL                                                                                         
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                    
    int idn = threadIdx.x;                                                                          
    magmaDoubleComplex rB;                                                                          
    magmaDoubleComplex rA;                                                                          
    dA += (j)*WARP_SIZE*WARP_SIZE;                                                                  
    dB += (j)*WARP_SIZE;                                                                            
                                                                                                    
    // Read B to regs.                                                                              
    rB = dB[idn];                                                                                   
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 32-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+idn];                                                                   
        if (k%WARP_SIZE == idn)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( idn < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev mem.                                                                           
    dB[idn] = rB;                                                                                   
#endif                                                                                              
}                                                                                                   


                                                                                                




                                                                                                
__global__                                                                                              
void ztrsv_upper_32kernel_switch(magmaDoubleComplex *dA, magmaDoubleComplex *dB, int *sizes, int num_rows )
{                                                                                                       
                                                                                                        
                                                                                                        
                                                                                                        
    int j = blockIdx.y * gridDim.x + blockIdx.x;                                                        
    if( j < num_rows ){                                                                                 
    int N = sizes[j];                                                                                   
    switch( N ) {                                                                                       
        case  1:                                                                                       
            ztrsv_upper_32kernel_1( dA, dB ); break;                                                     
        case  2:                                                                                       
            ztrsv_upper_32kernel_2( dA, dB ); break;                                                     
        case  3:                                                                                       
            ztrsv_upper_32kernel_3( dA, dB ); break;                                                     
        case  4:                                                                                       
            ztrsv_upper_32kernel_4( dA, dB ); break;                                                     
        case  5:                                                                                       
            ztrsv_upper_32kernel_5( dA, dB ); break;                                                     
        case  6:                                                                                       
            ztrsv_upper_32kernel_6( dA, dB ); break;                                                     
        case  7:                                                                                       
            ztrsv_upper_32kernel_7( dA, dB ); break;                                                     
        case  8:                                                                                       
            ztrsv_upper_32kernel_8( dA, dB ); break;                                                     
        case  9:                                                                                       
            ztrsv_upper_32kernel_9( dA, dB ); break;                                                     
        case  10:                                                                                       
            ztrsv_upper_32kernel_10( dA, dB ); break;                                                     
        case  11:                                                                                       
            ztrsv_upper_32kernel_11( dA, dB ); break;                                                     
        case  12:                                                                                       
            ztrsv_upper_32kernel_12( dA, dB ); break;                                                     
        case  13:                                                                                       
            ztrsv_upper_32kernel_13( dA, dB ); break;                                                     
        case  14:                                                                                       
            ztrsv_upper_32kernel_14( dA, dB ); break;                                                     
        case  15:                                                                                       
            ztrsv_upper_32kernel_15( dA, dB ); break;                                                     
        case  16:                                                                                       
            ztrsv_upper_32kernel_16( dA, dB ); break;                                                     
        case  17:                                                                                       
            ztrsv_upper_32kernel_17( dA, dB ); break;                                                     
        case  18:                                                                                       
            ztrsv_upper_32kernel_18( dA, dB ); break;                                                     
        case  19:                                                                                       
            ztrsv_upper_32kernel_19( dA, dB ); break;                                                     
        case  20:                                                                                       
            ztrsv_upper_32kernel_20( dA, dB ); break;                                                     
        case  21:                                                                                       
            ztrsv_upper_32kernel_21( dA, dB ); break;                                                     
        case  22:                                                                                       
            ztrsv_upper_32kernel_22( dA, dB ); break;                                                     
        case  23:                                                                                       
            ztrsv_upper_32kernel_23( dA, dB ); break;                                                     
        case  24:                                                                                       
            ztrsv_upper_32kernel_24( dA, dB ); break;                                                     
        case  25:                                                                                       
            ztrsv_upper_32kernel_25( dA, dB ); break;                                                     
        case  26:                                                                                       
            ztrsv_upper_32kernel_26( dA, dB ); break;                                                     
        case  27:                                                                                       
            ztrsv_upper_32kernel_27( dA, dB ); break;                                                     
        case  28:                                                                                       
            ztrsv_upper_32kernel_28( dA, dB ); break;                                                     
        case  29:                                                                                       
            ztrsv_upper_32kernel_29( dA, dB ); break;                                                     
        case  30:                                                                                       
            ztrsv_upper_32kernel_30( dA, dB ); break;                                                     
        case  31:                                                                                       
            ztrsv_upper_32kernel_31( dA, dB ); break;                                                     
        case  32:                                                                                       
            ztrsv_upper_32kernel_32( dA, dB ); break;                                                     
        default:                                                                                        
            ztrsv_upper_32kernel_general( dA, dB, sizes );break;                                          
    }                                                                                                   
    }                                                                                                   
}       




// initialize arrays with zero
__global__ void
magma_zgpumemzero_32kernel(  
    magmaDoubleComplex * d, 
    int n, 
    int dim_x,
    int dim_y )
{
    int i = blockIdx.y * gridDim.x + blockIdx.x;
    int idx = threadIdx.x;
    
    if( i >= n ){
       return;
    }
    if( idx >= dim_x ){
       return;
    }
    
    for( int j=0; j<dim_y; j++)
        d[ i*dim_x*dim_y + j*dim_y + idx ] = MAGMA_Z_MAKE( 0.0, 0.0 );
}

__global__ void 
magma_zlocations_lower_32kernel(    
    magma_int_t n, 
    magma_index_t *row,
    magma_index_t *col,
    magmaDoubleComplex *val,
    magma_index_t *sizes,
    magma_index_t *locations,
    magmaDoubleComplex *trisystems,
    magmaDoubleComplex *rhs )
{
    int i = threadIdx.x;
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    
    if( j >= n ){
        return;
    }
    int start = row[j];
    int end = row[j+1];
    int count = end-start;
    if( i == 0 ){
        sizes[j] = count;
        rhs[ j*WARP_SIZE ] = MAGMA_Z_ONE;
    }
    
    if ( i<count ){
        locations[ j*WARP_SIZE + i ] = col[ row[j]+i ];
    }
}// kernel 


__global__ void 
magma_zlocations_trunc_lower_32kernel(    
    magma_int_t n, 
    magma_index_t *row,
    magma_index_t *col,
    magmaDoubleComplex *val,
    magma_index_t *sizes,
    magma_index_t *locations,
    magmaDoubleComplex *trisystems,
    magmaDoubleComplex *rhs )
{
    int i = threadIdx.x;
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    
    if( j >= n ){
        return;
    }
    int start = row[j];
    int end = row[j+1];
    int count = end-start;
    
    // normal case
    if( count <= BLOCKSIZE ){ // normal case
        if( i == 0 ){
            sizes[j] = count;
            rhs[ j*WARP_SIZE ] = MAGMA_Z_ONE;
        }
        if ( i<count ){
            locations[ j*WARP_SIZE + i ] = col[ row[j]+i ];
        }
    } else { // truncate in this row to the blocksize, 
             // take only the 32 elements close to the main diagonal into account
          count = BLOCKSIZE;
       if( i == 0 ){
            sizes[j] = count;
            rhs[ j*WARP_SIZE ] = MAGMA_Z_ONE;
        }
        
        locations[ j*WARP_SIZE + i ] = col[ row[j+1]-BLOCKSIZE+i ];
        
    }
}// kernel 



__global__ void 
magma_zlocations_upper_32kernel(    
    magma_int_t n, 
    magma_index_t *row,
    magma_index_t *col,
    magmaDoubleComplex *val,
    magma_index_t *sizes,
    magma_index_t *locations,
    magmaDoubleComplex *trisystems,
    magmaDoubleComplex *rhs )
{
    int i = threadIdx.x;
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    
    if( j >= n ){
        return;
    }
    int start = row[j];
    int end = row[j+1];
    int count = end-start;
    if( i == 0 ){
        sizes[j] = count;
        rhs[ j*WARP_SIZE+count-1 ] = MAGMA_Z_ONE;
    }
    
    if ( i<count ){
        locations[ j*WARP_SIZE + i ] = col[ row[j]+i ];
    }
}// kernel 

__global__ void 
magma_zlocations_trunc_upper_32kernel(    
    magma_int_t n, 
    magma_index_t *row,
    magma_index_t *col,
    magmaDoubleComplex *val,
    magma_index_t *sizes,
    magma_index_t *locations,
    magmaDoubleComplex *trisystems,
    magmaDoubleComplex *rhs )
{
    int i = threadIdx.x;
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    
    if( j >= n ){
        return;
    }
    int start = row[j];
    int end = row[j+1];
    int count = end-start;
    
    // normal case
    if( count <= BLOCKSIZE ){ // normal case
        if( i == 0 ){
            sizes[j] = count;
            rhs[ j*WARP_SIZE+count-1 ] = MAGMA_Z_ONE;
        }
        if ( i<count ){
            locations[ j*WARP_SIZE + i ] = col[ row[j]+i ];
        }
    } else { // truncate in this row to the blocksize, 
             // take only the 32 elements close to the main diagonal into account
          count = BLOCKSIZE;
       if( i == 0 ){
            sizes[j] = count;
            rhs[ j*WARP_SIZE+count-1 ] = MAGMA_Z_ONE;
        }
        
        locations[ j*WARP_SIZE + i ] = col[ row[j]+i ];
        
    }
}// kernel 

__global__ void 
magma_zfilltrisystems_32kernel(    
    magma_int_t offset, 
    magma_int_t limit, 
    magma_index_t *row,
    magma_index_t *col,
    magmaDoubleComplex *val,
    magma_index_t *sizes,
    magma_index_t *locations,
    magmaDoubleComplex *trisystems,
    magmaDoubleComplex *rhs )
{
    int i = (blockDim.x * blockIdx.x + threadIdx.x)+offset;
    int ii = (blockDim.x * blockIdx.x + threadIdx.x);

    if ( ii>=limit ){
        return;
    }
    //if ( i<offset ){
    //    return;
    //}
    
    for( int j=0; j<sizes[ i ]; j++ ){// no need for first
        int k = row[ locations[ j+i*WARP_SIZE ] ];
        int l = i*WARP_SIZE;
        int idx = 0;
        while( k < row[ locations[ j+i*WARP_SIZE ]+1 ] && l < (i+1)*WARP_SIZE ){ // stop once this column is done
            if( locations[ l ] == col[k] ){ //match
                // int loc = i*WARP_SIZE*WARP_SIZE + j*WARP_SIZE + idx;
                trisystems[ ii*WARP_SIZE*WARP_SIZE + j*WARP_SIZE + idx ] 
                                                        = val[ k ];
                k++;
                l++;
                idx++;
            } else if( col[k] < locations[ l ] ){// need to check next element
                k++;
            } else { // element does not exist, i.e. l < LC.col[k]
                // printf("increment l\n");
                l++; // check next elment in the sparsity pattern
                idx++; // leave this element equal zero
            }
        } 
        
    }
}// kernel 







__global__ void 
magma_zbackinsert_32kernel(    
    magma_int_t n, 
    magma_index_t *row,
    magma_index_t *col,
    magmaDoubleComplex *val,
    magma_index_t *sizes,
    magmaDoubleComplex *rhs )
{
    int i = threadIdx.x;
    int j = blockIdx.y * gridDim.x + blockIdx.x;
    int end = sizes[j];
    if( j >= n ){
        return;
    }
    
    if ( i>=end ){
        return;
    }
    
    val[row[j]+i] = rhs[j*WARP_SIZE+i];
}// kernel 





// try to do everything in shared memory and registers!
//one thread block per row of A

__global__ void 
magma_zlowertrisystems_32kernel_s(    
    magma_int_t n, 
    magma_index_t *Arow,
    magma_index_t *Acol,
    magmaDoubleComplex *Aval,
    magma_index_t *Mrow,
    magma_index_t *Mcol,
    magmaDoubleComplex *Mval,
    magma_index_t *sizes,
    magma_index_t *locations )
{
    int row = blockIdx.y * gridDim.x + blockIdx.x;
    int tid = threadIdx.x;
    magmaDoubleComplex rB;    // registers for trsv                                                                      
    magmaDoubleComplex rA;  
    
    __shared__ magmaDoubleComplex dA[32*32];

#ifdef REAL   

    // only if within this chunk
    if ( row>=n ){
        return;
    }
    
    // only if within the size
    int size = sizes[ row ];
    if( tid >= size ){
        return;    
    }
    
    // set dA to 0
    for( int j=0; j<32; j++ ){
        dA[ j*32 + tid ] = MAGMA_Z_ZERO;
    }
    /*
    // for debuggging: let thred 0 do everything
    if(tid==0){
    // first: generate the triangular systems
    for( int j=0; j<size; j++ ){// no need for first
        int k = Arow[ locations[ j+row*WARP_SIZE ] ];
        int l = row*WARP_SIZE;
        int idx = 0;
        while( k < Arow[ locations[ j+row*WARP_SIZE ]+1 ] && l < (row+1)*WARP_SIZE ){ // stop once this column is done
            if( locations[ l ] == Acol[k] ){ //match
                // int loc = i*WARP_SIZE*WARP_SIZE + j*WARP_SIZE + idx;
                dA[ j*32 + idx ] = Aval[ k ];
                k++;
                l++;
                idx++;
            } else if( Acol[k] < locations[ l ] ){// need to check next element
                k++;
            } else { // element does not exist, i.e. l < LC.col[k]
                l++; // check next elment in the sparsity pattern
                idx++; // leave this element equal zero
            }
        } 
    }
    }
    __syncthreads();
    */
    int k = Arow[ locations[ tid+row*WARP_SIZE ] ];
    int l = row*WARP_SIZE;
    int idx = 0;
    while( k < Arow[ locations[ tid+row*WARP_SIZE ]+1 ] && l < (row+1)*WARP_SIZE ){ // stop once this column is done
        if( locations[ l ] == Acol[k] ){ //match
            // int loc = i*WARP_SIZE*WARP_SIZE + j*WARP_SIZE + idx;
            dA[ tid*32 + idx ] = Aval[ k ];
            k++;
            l++;
            idx++;
        } else if( Acol[k] < locations[ l ] ){// need to check next element
            k++;
        } else { // element does not exist, i.e. l < LC.col[k]
            l++; // check next elment in the sparsity pattern
            idx++; // leave this element equal zero
        }
    } 
    
    
    // second: solve the triangular systems - in registers
    
    // Read B to regs.                                                                              
    rB = (tid == 0) ? MAGMA_Z_ONE : MAGMA_Z_ZERO;  
    
        // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 32; k++)                                                                    
    {                                                                                               
        rA = dA[k*WARP_SIZE+tid];                                                                   
        if (k%WARP_SIZE == tid)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                           
        if ( tid > k)                                                                               
            rB -= (top*rA);                                                                         
    }                                                                                               
    // Drop B to dev memory - in ISAI preconditioner M                                                                           
    Mval[ Mrow[row] + tid ] = rB;  
    
#endif    
    
}// kernel 


__global__ void 
magma_zuppertrisystems_32kernel_s(    
    magma_int_t n, 
    magma_index_t *Arow,
    magma_index_t *Acol,
    magmaDoubleComplex *Aval,
    magma_index_t *Mrow,
    magma_index_t *Mcol,
    magmaDoubleComplex *Mval,
    magma_index_t *sizes,
    magma_index_t *locations )
{
    int row = blockIdx.y * gridDim.x + blockIdx.x;
    int tid = threadIdx.x;
    magmaDoubleComplex rB;    // registers for trsv                                                                      
    magmaDoubleComplex rA;  
    
    __shared__ magmaDoubleComplex dA[32*32];

#ifdef REAL   

    // only if within this chunk
    if ( row>=n ){
        return;
    }
    
    // only if within the size
    int size = sizes[ row ];
    if( tid >= size ){
        return;    
    }
    
    // set dA to 0
    for( int j=0; j<32; j++ ){
        dA[ j*32 + tid ] = MAGMA_Z_ZERO;
    }
    /*
    // for debuggging: let thred 0 do everything
    if(tid==0){
    // first: generate the triangular systems
    for( int j=0; j<size; j++ ){// no need for first
        int k = Arow[ locations[ j+row*WARP_SIZE ] ];
        int l = row*WARP_SIZE;
        int idx = 0;
        while( k < Arow[ locations[ j+row*WARP_SIZE ]+1 ] && l < (row+1)*WARP_SIZE ){ // stop once this column is done
            if( locations[ l ] == Acol[k] ){ //match
                // int loc = i*WARP_SIZE*WARP_SIZE + j*WARP_SIZE + idx;
                dA[ j*32 + idx ] = Aval[ k ];
                k++;
                l++;
                idx++;
            } else if( Acol[k] < locations[ l ] ){// need to check next element
                k++;
            } else { // element does not exist, i.e. l < LC.col[k]
                l++; // check next elment in the sparsity pattern
                idx++; // leave this element equal zero
            }
        } 
    }
    }
    __syncthreads();
    */
    int k = Arow[ locations[ tid+row*WARP_SIZE ] ];
    int l = row*WARP_SIZE;
    int idx = 0;
    while( k < Arow[ locations[ tid+row*WARP_SIZE ]+1 ] && l < (row+1)*WARP_SIZE ){ // stop once this column is done
        if( locations[ l ] == Acol[k] ){ //match
            // int loc = i*WARP_SIZE*WARP_SIZE + j*WARP_SIZE + idx;
            dA[ tid*32 + idx ] = Aval[ k ];
            k++;
            l++;
            idx++;
        } else if( Acol[k] < locations[ l ] ){// need to check next element
            k++;
        } else { // element does not exist, i.e. l < LC.col[k]
            l++; // check next elment in the sparsity pattern
            idx++; // leave this element equal zero
        }
    } 
    
    
    // second: solve the triangular systems - in registers
    
    // Read B to regs.                                                                              
    rB = (tid == size-1) ? MAGMA_Z_ONE : MAGMA_Z_ZERO;  
    
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 32-1; k >-1; k--)                                                                  
    {                                                                                               
        rA = dA[k*WARP_SIZE+tid];                                                                   
        if (k%WARP_SIZE == tid)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%WARP_SIZE);                                       
        if ( tid < k)                                                                               
            rB -= (bottom*rA);                                                                      
    }                                                                                               
    // Drop B to dev memory - in ISAI preconditioner M                                                                           
    Mval[ Mrow[row] + tid ] = rB;  
    
#endif    
    
}// kernel 




__global__ void 
magma_zlowertrisystems_32kernel(    
    magma_int_t n, 
    magma_index_t *Arow,
    magma_index_t *Acol,
    magmaDoubleComplex *Aval,
    magma_index_t *Mrow,
    magma_index_t *Mcol,
    magmaDoubleComplex *Mval,
    magma_index_t *sizes,
    magma_index_t *locations )
{
    int row = blockIdx.y * gridDim.x + blockIdx.x;
    int tid = threadIdx.x;
    magmaDoubleComplex rB;    // registers for trsv                                                                      
    magmaDoubleComplex rA;  
    
    magmaDoubleComplex dA[32];

#ifdef REAL   

    // only if within this chunk
    if ( row>=n ){
        return;
    }
    
    // only if within the size
    int size = sizes[ row ];
    if( tid >= size ){
        return;    
    }
    
    // set dA to 0
    for( int j=0; j<32; j++ ){
        dA[ j ] = MAGMA_Z_ZERO;
    }
    
    // for debuggging: let thred 0 do everything
    //if(tid==0){
    {
    // first: generate the triangular systems
    #pragma unroll
    for( int j=0; j<size; j++ ){// no need for first
        int k = Arow[ locations[ j+row*WARP_SIZE ] ];
        int l = row*WARP_SIZE;
        int idx = 0;
        while( k < Arow[ locations[ j+row*WARP_SIZE ]+1 ] && l < (row+1)*WARP_SIZE ){ // stop once this column is done
            if( locations[ l ] == Acol[k] ){ //match
                if( tid == idx ){
                    dA[ j ] = Aval[ k ];    
                }
                //__syncthreads();
                // int loc = i*WARP_SIZE*WARP_SIZE + j*WARP_SIZE + idx;
                k++;
                l++;
                idx++;
            } else if( Acol[k] < locations[ l ] ){// need to check next element
                k++;
            } else { // element does not exist, i.e. l < LC.col[k]
                l++; // check next elment in the sparsity pattern
                idx++; // leave this element equal zero
            }
        } 
    }
    }
    // not sure whether we need this here....
    //__syncthreads();
    
    
    // second: solve the triangular systems - in registers
    
    // Read B to regs.                                                                              
    rB = (tid == 0) ? MAGMA_Z_ONE : MAGMA_Z_ZERO;  
    
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 0; k < 32; k++)                                                               
    {                                                                                               
        rA = dA[ k ];                                                                   
        if (k%WARP_SIZE == tid)                                                                     
            rB /= rA;                                                                               
        magmaDoubleComplex top = __shfl(rB, k%WARP_SIZE);                                         
        if ( tid > k)                                                                               
            rB -= (top*rA);                                                                    
    }                                                                                               
    // Drop B to dev memory - in ISAI preconditioner M                                                                           
    Mval[ Mrow[row] + tid ] = rB;  
    
#endif    
    
}// kernel 


__global__ void 
magma_zuppertrisystems_32kernel(    
    magma_int_t n, 
    const magma_index_t * __restrict__ Arow,
    const magma_index_t * __restrict__ Acol,
    const magmaDoubleComplex * __restrict__ Aval,
    magma_index_t *Mrow,
    magma_index_t *Mcol,
    magmaDoubleComplex *Mval )
{
    int row = blockIdx.y * gridDim.x + blockIdx.x;
    int tid = threadIdx.x;
    
#ifdef REAL  
    magmaDoubleComplex rB;    // registers for trsv                                                                      
    magmaDoubleComplex rA[32];

    // only if within this chunk
    if ( row>=n ){
        return;
    }
    
    // only if within the size
    int mstart = Mrow[ row ];
    int mlim = Mrow[ row+1 ];
    int size = mlim - mstart;
    if( tid >= size ){
        return;    
    }
    
    // set rA to 0
    for( int j=0; j<32; j++ ){
        rA[ j ] = MAGMA_Z_ZERO;
    }
    
    // generate the triangular systems
    #pragma unroll
    for( int j=0; j<size; j++ ){// no need for first
        int t = Mcol[ mstart + j ];
        int k = Arow[ t ];
        int l = mstart; 
        int idx = 0;
        while( k < Arow[ t+1 ] && l < mlim ){ // stop once this column is done
            int mcol =  Mcol[ l ];
            int acol = Acol[k];
            if( mcol == acol ){ //match
                if( tid == idx ){
                    rA[ j ] = Aval[ k ];    
                }
                k++;
                l++;
                idx++;
            } else if( acol < mcol ){// need to check next element
                k++;
            } else { // element does not exist, i.e. l < LC.col[k]
                l++; // check next elment in the sparsity pattern
                idx++; // leave this element equal zero
            }
        } 
    }
    
    // second: solve the triangular systems - in registers
    // we know how RHS looks like                                                                         
    rB = (tid == size-1) ? MAGMA_Z_ONE : MAGMA_Z_ZERO;  
    
    // Triangular solve in regs.                                                                    
    #pragma unroll                                                                                  
    for (int k = 32-1; k >-1; k--)                                                                  
    {                                                                                               
        if (k%32 == tid)                                                                     
            rB /= rA[k];                                                                               
        magmaDoubleComplex bottom = __shfl(rB, k%32);                                       
        if ( tid < k)                                                                               
            rB -= (bottom*rA[k]);                                                                      
    }                                                                                               
    // Drop B to dev memory - in ISAI preconditioner M                                                                           
    Mval[ mstart + tid ] = rB;  
    
#endif    
    
}// kernel 

/**
    Purpose
    -------
    This routine is designet to combine all kernels into one.

    Arguments
    ---------
    

    @param[in]
    uplotype    magma_uplo_t
                lower or upper triangular
                
    @param[in]
    transtype   magma_trans_t
                possibility for transposed matrix
                
    @param[in]
    diagtype    magma_diag_t
                unit diagonal or not
                
    @param[in]
    L           magma_z_matrix
                triangular factor for which the ISAI matrix is computed.
                Col-Major CSR storage.
                
    @param[in,out]
    M           magma_z_matrix*
                SPAI preconditioner CSR col-major
                
    @param[out]
    sizes       magma_int_t*
                Number of Elements that are replaced.
                
    @param[out]
    locations   magma_int_t*
                Array indicating the locations.
                
    @param[out]
    trisystems  magmaDoubleComplex*
                trisystems
                
    @param[out]
    rhs         magmaDoubleComplex*
                right-hand sides

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zaux
    ********************************************************************/

extern "C" magma_int_t
magma_zisaigenerator_32_gpu(
    magma_uplo_t uplotype,
    magma_trans_t transtype,
    magma_diag_t diagtype,
    magma_z_matrix L,
    magma_z_matrix *M,
    magma_index_t *sizes,
    magma_index_t *locations,
    magmaDoubleComplex *trisystems,
    magmaDoubleComplex *rhs,    
    magma_queue_t queue )
{
    magma_int_t info = 0;  
    
    hipDeviceSetCacheConfig( hipFuncCachePreferL1 );
    
    
    // routine 1
    int r1bs1 = WARP_SIZE;
    int r1bs2 = 1;
    int r1dg1 = min( int( sqrt( double( M->num_rows ))), 65535 );
    int r1dg2 = min(magma_ceildiv( M->num_rows, r1dg1 ), 65535);
    int r1dg3 = magma_ceildiv( M->num_rows, r1dg1*r1dg2 );
    
    dim3 r1block( r1bs1, r1bs2, 1 );
    dim3 r1grid( r1dg1, r1dg2, r1dg3 );
    
    int r2bs1 = WARP_SIZE;
    int r2bs2 = 1;
    int r2dg1 = magma_ceildiv( L.num_rows, r2bs1 );
    int r2dg2 = 1;
    int r2dg3 = 1;
    dim3 r2block( r2bs1, r2bs2, 1 );
    dim3 r2grid( r2dg1, r2dg2, r2dg3 );
    
    int r3bs1 = WARP_SIZE;
    int r3bs2 = 1;
    int r3dg1 = magma_ceildiv( 32000, r2bs1 );
    int r3dg2 = 1;
    int r3dg3 = 1;
    dim3 r3block( r3bs1, r3bs2, 1 );
    dim3 r3grid( r3dg1, r3dg2, r3dg3 );
    
    int recursive = magma_ceildiv( M->num_rows, 32000 );
    
    magma_zgpumemzero_32kernel<<< r1grid, r1block, 0, queue->hip_stream() >>>(  
            rhs, L.num_rows, WARP_SIZE, 1);
    
    
    if( uplotype == MagmaLower ){
        magma_zlocations_lower_32kernel<<< r1grid, r1block, 0, queue->hip_stream() >>>(    
                        M->num_rows, 
                        M->drow,
                        M->dcol,
                        M->dval,
                        sizes,
                        locations,
                        trisystems,
                        rhs );
    } else {
        magma_zlocations_upper_32kernel<<< r1grid, r1block, 0, queue->hip_stream() >>>(    
                        M->num_rows, 
                        M->drow,
                        M->dcol,
                        M->dval,
                        sizes,
                        locations,
                        trisystems,
                        rhs );
    }
    /*
    if( uplotype == MagmaLower ){printf("in here lower\n");
    magma_zlowertrisystems_32kernel<<< r1grid, r1block, 0, queue->hip_stream() >>>(    
            L.num_rows, 
            L.drow,
            L.dcol,
            L.dval,
            M->drow,
            M->dcol,
            M->dval,
            sizes,
            locations );
    } else { printf("in here upper\n");
            magma_zuppertrisystems_32kernel<<< r1grid, r1block, 0, queue->hip_stream() >>>(    
            L.num_rows, 
            L.drow,
            L.dcol,
            L.dval,
            M->drow,
            M->dcol,
            M->dval ); 
    }
        
*/

    // chunk it recursively into batches of 3200
    for( int z=0; z<recursive; z++ ){
        
        int limit = min(32000, L.num_rows-32000*z);
        
        magma_zgpumemzero_32kernel<<< r1grid, r1block, 0, queue->hip_stream() >>>(  
            trisystems, limit, WARP_SIZE, WARP_SIZE );
        
        magma_zfilltrisystems_32kernel<<< r3grid, r3block, 0, queue->hip_stream() >>>(    
                            32000*z,
                            limit, 
                            L.drow,
                            L.dcol,
                            L.dval,
                            sizes,
                            locations,
                            trisystems,
                            rhs );
        
        
        // routine 2
        if( uplotype == MagmaLower ){ 
            ztrsv_lower_32kernel_switch<<< r1grid, r1block, 0, queue->hip_stream() >>>(
                    trisystems,
                    rhs+32000*32*z,
                    sizes+32000*z,
                    limit );
        } else {
            ztrsv_upper_32kernel_switch<<< r1grid, r1block, 0, queue->hip_stream() >>>(
                    trisystems,
                    rhs+32000*32*z,
                    sizes+32000*z,
                    limit );
        }
    }
    
    // routine 3
    magma_zbackinsert_32kernel<<< r1grid, r1block, 0, queue->hip_stream() >>>(    
            M->num_rows, 
            M->drow,
            M->dcol,
            M->dval,
            sizes,
            rhs );

    
    return info;
}

