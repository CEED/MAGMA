#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s
       @author Hartwig Anzt

*/
#include "common_magma.h"
#include "magmasparse.h"

#if (GPUSHMEM < 200)
   #define BLOCK_SIZE 128
#else
   #define BLOCK_SIZE 512
#endif



__global__ void 
zvjacobisetup_gpu(  int num_rows, 
                    int num_vecs,
                    magmaDoubleComplex *b, 
                    magmaDoubleComplex *d, 
                    magmaDoubleComplex *c,
                    magmaDoubleComplex *x){

    int row = blockDim.x * blockIdx.x + threadIdx.x ;

    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++ ){
            c[row+i*num_rows] = b[row+i*num_rows] / d[row];
            x[row+i*num_rows] = c[row+i*num_rows];
        }
    }
}





/**
    Purpose
    -------

    Prepares the Jacobi Iteration according to
       x^(k+1) = D^(-1) * b - D^(-1) * (L+U) * x^k
       x^(k+1) =      c     -       M        * x^k.

    Returns the vector c. It calls a GPU kernel

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                number of rows
                
    @param[in]
    b           magma_z_vector
                RHS b

    @param[in]
    d           magma_z_vector
                vector with diagonal entries

    @param[out]
    c           magma_z_vector*
                c = D^(-1) * b

    @param[out]
    x           magma_z_vector*
                iteration vector
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_zjacobisetup_vector_gpu(
    int num_rows, 
    magma_z_vector b, 
    magma_z_vector d, 
    magma_z_vector c,
    magma_z_vector *x,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    int num_vecs = b.num_rows / num_rows;
    magma_int_t threads = BLOCK_SIZE;
    zvjacobisetup_gpu<<< grid, threads, 0 >>>
                ( num_rows, num_vecs, b.dval, d.dval, c.dval, x->val );

    return MAGMA_SUCCESS;
}






__global__ void 
zjacobidiagscal_kernel(  int num_rows,
                         int num_vecs, 
                    magmaDoubleComplex *b, 
                    magmaDoubleComplex *d, 
                    magmaDoubleComplex *c){

    int row = blockDim.x * blockIdx.x + threadIdx.x ;

    if(row < num_rows ){
        for( int i=0; i<num_vecs; i++)
            c[row+i*num_rows] = b[row+i*num_rows] * d[row];
    }
}





/**
    Purpose
    -------

    Prepares the Jacobi Iteration according to
       x^(k+1) = D^(-1) * b - D^(-1) * (L+U) * x^k
       x^(k+1) =      c     -       M        * x^k.

    Returns the vector c. It calls a GPU kernel

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                number of rows
                
    @param[in]
    b           magma_z_vector
                RHS b

    @param[in]
    d           magma_z_vector
                vector with diagonal entries

    @param[out]
    c           magma_z_vector*
                c = D^(-1) * b
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_z
    ********************************************************************/

extern "C" magma_int_t
magma_zjacobi_diagscal(
    int num_rows, 
    magma_z_vector d, 
    magma_z_vector b, 
    magma_z_vector *c,
    magma_queue_t queue )
{
    dim3 grid( magma_ceildiv( num_rows, BLOCK_SIZE ));
    int num_vecs = b.num_rows*b.num_cols/num_rows;
    magma_int_t threads = BLOCK_SIZE;
    zjacobidiagscal_kernel<<< grid, threads, 0 >>>( num_rows, num_vecs, b.dval, d.dval, c->val );

    return MAGMA_SUCCESS;
}



