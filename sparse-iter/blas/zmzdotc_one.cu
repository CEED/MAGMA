#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s
       @author Hartwig Anzt

*/
#include "common_magma.h"

#define BLOCK_SIZE 256

#define PRECISION_z


// initialize arrays with zero



// dot product for multiple vectors
__global__ void
magma_zmzdotc_one_kernel_1( 
    int Gs,
    int n, 
    magmaDoubleComplex * v0,
    magmaDoubleComplex * w0,
    magmaDoubleComplex * vtmp)
{
    extern __shared__ magmaDoubleComplex temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;

    // 1 vectors v(i)/w(i)
    
    temp[ Idx ]                 = ( i < n ) ?
                v0[ i ] * w0[ i ] : MAGMA_Z_ZERO;
    
    __syncthreads();
    if ( Idx < 128 ){
            temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ){
            temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
                temp[ Idx ] += temp[ Idx + 32 ];
                __syncthreads();
                temp[ Idx ] += temp[ Idx + 16 ];
                __syncthreads();
                temp[ Idx ] += temp[ Idx + 8 ];
                __syncthreads();
                temp[ Idx ] += temp[ Idx + 4 ];
                __syncthreads();
                temp[ Idx ] += temp[ Idx + 2 ];
                __syncthreads();
                temp[ Idx ] += temp[ Idx + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
                temp2[ Idx ] += temp2[ Idx + 32 ];
                temp2[ Idx ] += temp2[ Idx + 16 ];
                temp2[ Idx ] += temp2[ Idx + 8 ];
                temp2[ Idx ] += temp2[ Idx + 4 ];
                temp2[ Idx ] += temp2[ Idx + 2 ];
                temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
                temp2[ Idx ] += temp2[ Idx + 32 ];
                temp2[ Idx ] += temp2[ Idx + 16 ];
                temp2[ Idx ] += temp2[ Idx + 8 ];
                temp2[ Idx ] += temp2[ Idx + 4 ];
                temp2[ Idx ] += temp2[ Idx + 2 ];
                temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    if ( Idx == 0 ){
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}



// block reduction for 1 vectors
__global__ void
magma_zmzdotc_one_kernel_2( 
    int Gs,
    int n, 
    magmaDoubleComplex * vtmp,
    magmaDoubleComplex * vtmp2 )
{
    extern __shared__ magmaDoubleComplex temp[];    
    int Idx = threadIdx.x;
    int blockSize = 128;
    int gridSize = blockSize  * 2 * gridDim.x; 

        int i = blockIdx.x * ( blockSize * 2 ) + Idx;   
        temp[Idx] = MAGMA_Z_ZERO;
        while (i < Gs ) {
            temp[ Idx  ] += vtmp[ i ]; 
            temp[ Idx  ] += 
                ( i + (blockSize) < Gs ) ? vtmp[ i + (blockSize) ] 
                                                : MAGMA_Z_ZERO;
            i += gridSize;
        }
    __syncthreads();
    if ( Idx < 64 ){
            temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
                temp[ Idx ] += temp[ Idx + 32 ];
                __syncthreads();
                temp[ Idx ] += temp[ Idx + 16 ];
                __syncthreads();
                temp[ Idx ] += temp[ Idx + 8 ];
                __syncthreads();
                temp[ Idx ] += temp[ Idx + 4 ];
                __syncthreads();
                temp[ Idx ] += temp[ Idx + 2 ];
                __syncthreads();
                temp[ Idx ] += temp[ Idx + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
                temp2[ Idx ] += temp2[ Idx + 32 ];
                temp2[ Idx ] += temp2[ Idx + 16 ];
                temp2[ Idx ] += temp2[ Idx + 8 ];
                temp2[ Idx ] += temp2[ Idx + 4 ];
                temp2[ Idx ] += temp2[ Idx + 2 ];
                temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
                temp2[ Idx ] += temp2[ Idx + 32 ];
                temp2[ Idx ] += temp2[ Idx + 16 ];
                temp2[ Idx ] += temp2[ Idx + 8 ];
                temp2[ Idx ] += temp2[ Idx + 4 ];
                temp2[ Idx ] += temp2[ Idx + 2 ];
                temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    if ( Idx == 0 ){
            vtmp2[ blockIdx.x ] = temp[ 0 ];
    }
}

/**
    Purpose
    -------

    Computes the scalar product of a set of 1 vectors such that

    skp[0] = [ <v_0,w_0> ]

    Returns the vector skp.
    In case there are less dot products required, an easy workaround is
    given by doubling input.

    Arguments
    ---------

    @param[in]
    n           int
                length of v_i and w_i

    @param[in]                             
    v0          magmaDoubleComplex_ptr     
                input vector               

    @param[in]                                         
    w0          magmaDoubleComplex_ptr                 
                input vector                           

    @param[in]
    d1          magmaDoubleComplex_ptr 
                workspace

    @param[in]
    d2          magmaDoubleComplex_ptr 
                workspace

    @param[out]
    skp         magmaDoubleComplex_ptr 
                vector[4] of scalar products [<v_i, w_i>]
                This vector is located on the host

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zblas
    ********************************************************************/

extern "C" magma_int_t
magma_zmzdotc_one(
    int n,  
    magmaDoubleComplex_ptr v0, 
    magmaDoubleComplex_ptr w0,
    magmaDoubleComplex_ptr d1,
    magmaDoubleComplex_ptr d2,
    magmaDoubleComplex_ptr skp,
    magma_queue_t queue )
{
    // set queue for old dense routines
    magma_queue_t orig_queue;
    magmablasGetKernelStream( &orig_queue );

    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( magma_ceildiv( n, local_block_size ) );
    dim3 Gs_next;
    int Ms = (local_block_size) * sizeof( magmaDoubleComplex ); // 1 skp 
    magmaDoubleComplex_ptr aux1 = d1, aux2 = d2;
    int b = 1;        


    magma_zmzdotc_one_kernel_1<<<Gs, Bs, Ms, queue>>>
            ( Gs.x, n, v0, w0, d1 );
   
    while( Gs.x > 1 ) {
        Gs_next.x = ( Gs.x+Bs.x-1 )/ Bs.x;
        if ( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_zmzdotc_one_kernel_2<<< Gs_next.x/2, Bs.x/2, Ms/2, queue >>> 
                    ( Gs.x, n, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if ( b ) { aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }
    
        // copy vectors to host
    magma_zgetvector( 1 , aux1, 1, skp, 1 );
    

   magmablasSetKernelStream( orig_queue );
   return MAGMA_SUCCESS;
}


