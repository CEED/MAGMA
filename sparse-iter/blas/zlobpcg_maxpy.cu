#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/

#include "common_magma.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE  512



__global__ void
magma_zlobpcg_maxpy_kernel( 
    magma_int_t num_rows, 
    magma_int_t num_vecs, 
    magmaDoubleComplex * X, 
    magmaDoubleComplex * Y)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x; // global row index

    if ( row < num_rows ) {
        for( int i=0; i < num_vecs; i++ ) {
            Y[ row + i*num_rows ] += X[ row + i*num_rows ];
        }
    }
}




/**
    Purpose
    -------
    
    This routine computes a axpy for a mxn matrix:
        
        Y = X + Y
        
    It replaces:
            magma_zaxpy(m*n, c_one, Y, 1, X, 1);


        / x1[0] x2[0] x3[0] \
        | x1[1] x2[1] x3[1] |
    X = | x1[2] x2[2] x3[2] | = x1[0] x1[1] x1[2] x1[3] x1[4] x2[0] x2[1] .
        | x1[3] x2[3] x3[3] |
        \ x1[4] x2[4] x3[4] /
    
    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                number of rows

    @param[in]
    num_vecs    magma_int_t
                number of vectors

    @param[in]
    X           magmaDoubleComplex_ptr 
                input vector X

    @param[in,out]
    Y           magmaDoubleComplex_ptr 
                input/output vector Y

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_zlobpcg_maxpy(
    magma_int_t num_rows,
    magma_int_t num_vecs, 
    magmaDoubleComplex_ptr X,
    magmaDoubleComplex_ptr Y,
    magma_queue_t queue )
{
    // every thread handles one row

    magma_int_t block_size = BLOCK_SIZE;
     magma_int_t threads = BLOCK_SIZE;
    dim3 block( block_size );
    dim3 grid( magma_ceildiv( num_rows, block_size ) );

    magma_zlobpcg_maxpy_kernel<<< grid, threads, 0, queue >>>
                                ( num_rows, num_vecs, X, Y );


    return MAGMA_SUCCESS;
}
