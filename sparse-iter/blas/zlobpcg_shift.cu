#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/

#include "common_magma.h"

__global__ void
magma_zlobpcg_shift_kernel( 
    magma_int_t num_rows, 
    magma_int_t num_vecs, 
    magma_int_t shift, 
    magmaDoubleComplex_ptr x )
{

    int idx = threadIdx.x ;     // thread in row
    int row = blockIdx.y * gridDim.x + blockIdx.x; // global block index

    if( row<num_rows){
        magmaDoubleComplex tmp = x[idx];
        __syncthreads();

        if( idx > shift-1 ){
            idx-=shift;
            x[idx] = tmp;
            __syncthreads();
        }

    }
}




/**
    Purpose
    -------
    
    For a Block-LOBPCG, the set of residuals (entries consecutive in memory)  
    shrinks and the vectors are shifted in case shift residuals drop below 
    threshold. The memory layout of x is:

        / x1[0] x2[0] x3[0] \
        | x1[1] x2[1] x3[1] |
    x = | x1[2] x2[2] x3[2] | = x1[0] x2[0] x3[0] x1[1] x2[1] x3[1] x1[2] .
        | x1[3] x2[3] x3[3] |
        \ x1[4] x2[4] x3[4] /
    
    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                number of rows

    @param[in]
    num_vecs    magma_int_t
                number of vectors

    @param[in]
    shift       magma_int_t
                shift number

    @param[in/out]
    x           magmaDoubleComplex_ptr 
                input/output vector x

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zaux
    ********************************************************************/

extern "C" magma_int_t
magma_zlobpcg_shift(
    magma_int_t num_rows,
    magma_int_t num_vecs, 
    magma_int_t shift,
    magmaDoubleComplex_ptr x,
    magma_queue_t queue )
{
    magma_int_t num_threads = num_vecs;
    // every thread handles one row containing the 
    if (  num_threads > 1024 )
        printf("error: too many threads requested.\n");

    int Ms = num_threads * sizeof( magmaDoubleComplex );
    if (  Ms > 1024*8 )
        printf("error: too much shared memory requested.\n");

    dim3 block( num_threads, 1, 1 );

    int dimgrid1 = sqrt(num_rows);
    int dimgrid2 = (num_rows + dimgrid1 -1 ) / dimgrid1;

    dim3 grid( dimgrid1, dimgrid2, 1);

    magma_zlobpcg_shift_kernel<<< grid, block, Ms, queue >>>
            ( num_rows, num_vecs, shift, x );


    return MAGMA_SUCCESS;
}



