#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

       @precisions normal z -> c d s
       @author Hartwig Anzt

*/
#include "common_magma.h"

#define BLOCK_SIZE 256

#define PRECISION_z


// initialize arrays with zero
__global__ void 
magma_zgpumemzero(  magmaDoubleComplex *d, int n, int k ){

   int i = blockIdx.x * blockDim.x + threadIdx.x;

   if( i < n ){
    for( int j=0; j<k; j++)
      d[ i+j*n ] = MAGMA_Z_MAKE( 0.0, 0.0 );
    }
}

// dot product
__global__ void 
magma_zdot_kernel( int Gs,
                        int n, 
                        int k,
                        magmaDoubleComplex *v,
                        magmaDoubleComplex *r,
                        magmaDoubleComplex *vtmp){

    extern __shared__ magmaDoubleComplex temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;

    temp[ Idx ] = ( i < n ) ? v[ i ] * r[ i ] : MAGMA_Z_MAKE( 0.0, 0.0);
    __syncthreads();
    if ( Idx < 128 ){
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ){
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            temp[ Idx ] += temp[ Idx + 32 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 8 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 4 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 2 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 1 ];__syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_f)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    if ( Idx == 0 ){
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}

// dot product for multiple vectors
__global__ void 
magma_zblockdot_kernel( int Gs,
                        int n, 
                        int k,
                        magmaDoubleComplex *v,
                        magmaDoubleComplex *r,
                        magmaDoubleComplex *vtmp){

    extern __shared__ magmaDoubleComplex temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int j;

    // k vectors v(i)
    if (i<n){
        for( j=0; j<k; j++)
            temp[Idx+j*blockDim.x] = v[i+j*n] * r[i];
    }
    else{
        for( j=0; j<k; j++)
            temp[Idx+j*blockDim.x] =MAGMA_Z_MAKE( 0.0, 0.0);
    }
    __syncthreads();
    if ( Idx < 128 ){
        for( j=0; j<k; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 128 ];
        }
    }
    __syncthreads();
    if ( Idx < 64 ){
        for( j=0; j<k; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 64 ];
        }
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            for( j=0; j<k; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 32 ];
                __syncthreads();
            for( j=0; j<k; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 16 ];
                __syncthreads();
            for( j=0; j<k; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 8 ];
                __syncthreads();
            for( j=0; j<k; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 4 ];
                __syncthreads();
            for( j=0; j<k; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 2 ];
                __syncthreads();
            for( j=0; j<k; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
            for( j=0; j<k; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    #if defined(PRECISION_f)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<k; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    if ( Idx == 0 ){
        for( j=0; j<k; j++){
            vtmp[ blockIdx.x+j*n ] = temp[ j*blockDim.x ];
        }
    }
}

// block reduction for multiple vectors
__global__ void 
magma_zblockreduce_kernel( int Gs,
                           int n, 
                           int k,
                           magmaDoubleComplex *vtmp,
                           magmaDoubleComplex *vtmp2 ){

    extern __shared__ magmaDoubleComplex temp[];    
    int Idx = threadIdx.x;
    int i = blockIdx.x * blockDim.x + Idx;  
    int j;
    for( j=0; j<k; j++){
        temp[ Idx+j*blockDim.x ] =  ( i < n ) ? vtmp[ i+j*n ] : MAGMA_Z_MAKE( 0.0, 0.0);
    }
    __syncthreads();
    if ( Idx < 128 ){
        for( j=0; j<k; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 128 ];
        }
    }
    __syncthreads();
    if ( Idx < 64 ){
        for( j=0; j<k; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 64 ];
        }
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            for( j=0; j<k; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 32 ];
                __syncthreads();
            for( j=0; j<k; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 16 ];
                __syncthreads();
            for( j=0; j<k; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 8 ];
                __syncthreads();
            for( j=0; j<k; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 4 ];
                __syncthreads();
            for( j=0; j<k; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 2 ];
                __syncthreads();
            for( j=0; j<k; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
            for( j=0; j<k; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    #if defined(PRECISION_f)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<k; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    if ( Idx == 0 ){
        for( j=0; j<k; j++){
            vtmp2[ blockIdx.x+j*n ] = temp[ j*blockDim.x ];
        }
    }
}

// accelerated reduction for one vector
__global__ void 
magma_zreduce_kernel_fast( int Gs,
                           int n, 
                           int k,
                           magmaDoubleComplex *vtmp,
                           magmaDoubleComplex *vtmp2 ){

    extern __shared__ magmaDoubleComplex temp[];    
    int Idx = threadIdx.x;
    int blockSize = 128;
    int gridSize = blockSize  * 2 * gridDim.x; 
    temp[Idx] = MAGMA_Z_MAKE( 0.0, 0.0);
    int i = blockIdx.x * ( blockSize * 2 ) + Idx;   
    while (i < Gs ) {
        temp[ Idx  ] += vtmp[ i ]; 
        temp[ Idx  ] += ( i + blockSize < Gs ) ? vtmp[ i + blockSize ] : MAGMA_Z_MAKE( 0.0, 0.0); 
        i += gridSize;
    }
    __syncthreads();
    if ( Idx < 64 ){
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            temp[ Idx ] += temp[ Idx + 32 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 8 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 4 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 2 ];__syncthreads();
            temp[ Idx ] += temp[ Idx + 1 ];__syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_f)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    if ( Idx == 0 ){
        vtmp2[ blockIdx.x ] = temp[ 0 ];
    }
}

// accelerated block reduction for multiple vectors
__global__ void 
magma_zblockreduce_kernel_fast( int Gs,
                           int n, 
                           int k,
                           magmaDoubleComplex *vtmp,
                           magmaDoubleComplex *vtmp2 ){

    extern __shared__ magmaDoubleComplex temp[];    
    int Idx = threadIdx.x;
    int blockSize = 128;
    int gridSize = blockSize  * 2 * gridDim.x; 
    int j;

    for( j=0; j<k; j++){
        int i = blockIdx.x * ( blockSize * 2 ) + Idx;   
        temp[Idx+j*(blockSize)] = MAGMA_Z_MAKE( 0.0, 0.0);
        while (i < Gs ) {
            temp[ Idx+j*(blockSize)  ] += vtmp[ i+j*n ]; 
            temp[ Idx+j*(blockSize)  ] += ( i + (blockSize) < Gs ) ? vtmp[ i+j*n + (blockSize) ] : MAGMA_Z_MAKE( 0.0, 0.0); 
            i += gridSize;
        }
    }
    __syncthreads();
    if ( Idx < 64 ){
        for( j=0; j<k; j++){
            temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 64 ];
        }
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            for( j=0; j<k; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 32 ];
                __syncthreads();
            for( j=0; j<k; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 16 ];
                __syncthreads();
            for( j=0; j<k; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 8 ];
                __syncthreads();
            for( j=0; j<k; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 4 ];
                __syncthreads();
            for( j=0; j<k; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 2 ];
                __syncthreads();
            for( j=0; j<k; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
            for( j=0; j<k; j++){
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 32 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 16 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 8 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 4 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 2 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 1 ];
            }
        }
    #endif
    #if defined(PRECISION_f)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<k; j++){
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 32 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 16 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 8 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 4 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 2 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 1 ];
            }
        }
    #endif
    if ( Idx == 0 ){
        for( j=0; j<k; j++){
            vtmp2[ blockIdx.x+j*n ] = temp[ j*(blockSize) ];
        }
    }
}

/*  -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

    Purpose
    =======

    Computes the scalar product of a set of vectors v_i such that

    skp = ( <v_0,r>, <v_1,r>, .. )

    Returns the vector skp.

    Arguments
    =========

    int n                             legth of v_i and r
    int k                             # vectors v_i
    magmaDoubleComplex *v                     v = (v_0 .. v_i.. v_k)
    magmaDoubleComplex *r                     r
    magmaDoubleComplex *skp                   vector[k] of scalar products (<v_i,r>...)

    =====================================================================  */

extern "C" int
magma_zmdotc(       int n, 
                    int k, 
                    magmaDoubleComplex *v, 
                    magmaDoubleComplex *r,
                    magmaDoubleComplex *d1,
                    magmaDoubleComplex *d2,
                    magmaDoubleComplex *skp ){
    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( (n+local_block_size-1)/local_block_size );
    dim3 Gs_next;
    int Ms =  (k)* (local_block_size) * sizeof( magmaDoubleComplex ); // k vectors 
    magmaDoubleComplex *aux1 = d1, *aux2 = d2;
    int b = 1;        

    if(k>1){
        magma_zblockdot_kernel<<<Gs, Bs, Ms>>>( Gs.x, n, k, v, r, d1 );
    }
    else{
        magma_zdot_kernel<<<Gs, Bs, Ms>>>( Gs.x, n, k, v, r, d1 );
    }
/*
    magma_zgpumemzero<<<Gs, Bs, 0>>>( d1, n*k,1 );
    magma_zgpumemzero<<<Gs, Bs, 0>>>( d2, n*k,1 );
    //magmablas_zlaset( MagmaUpperLower, n, k, d1, n );
    //magmablas_zlaset( MagmaUpperLower, n, k, d2, n );
    while( Gs.x > 1 ){
        Gs_next.x = ( Gs.x+Bs.x-1 )/ Bs.x ;
        magma_zblockreduce_kernel<<< Gs_next.x, Bs.x, Ms >>> ( Gs.x, n, k, aux1, aux2 );
        Gs.x = Gs_next.x;
        b = 1 - b;
        if( b ){ aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }
    for( int j=0; j<k; j++){
            hipMemcpy( skp+j, aux1+j*n, sizeof( magmaDoubleComplex ), hipMemcpyDeviceToDevice );
    }
*/
   
    if( k>1){
        while( Gs.x > 1 ){
            Gs_next.x = ( Gs.x+Bs.x-1 )/ Bs.x ;
            if( Gs_next.x == 1 ) Gs_next.x = 2;
            magma_zblockreduce_kernel_fast<<< Gs_next.x/2, Bs.x/2, Ms/2 >>> ( Gs.x, n, k, aux1, aux2 );
            Gs_next.x = Gs_next.x /2;
            Gs.x = Gs_next.x;
            b = 1 - b;
            if( b ){ aux1 = d1; aux2 = d2; }
            else   { aux2 = d1; aux1 = d2; }
        }
    }
    else{
        while( Gs.x > 1 ){
            Gs_next.x = ( Gs.x+Bs.x-1 )/ Bs.x ;
            if( Gs_next.x == 1 ) Gs_next.x = 2;
            magma_zreduce_kernel_fast<<< Gs_next.x/2, Bs.x/2, Ms/2 >>> ( Gs.x, n, k, aux1, aux2 );
            Gs_next.x = Gs_next.x /2;
            Gs.x = Gs_next.x;
            b = 1 - b;
            if( b ){ aux1 = d1; aux2 = d2; }
            else   { aux2 = d1; aux1 = d2; }
        }
    }


    for( int j=0; j<k; j++){
            hipMemcpy( skp+j, aux1+j*n, sizeof( magmaDoubleComplex ), hipMemcpyDeviceToDevice );
    }

    


   return MAGMA_SUCCESS;
}

/*  -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2011

    Purpose
    =======

    This is an extension of the merged dot product above by chunking
    the set of vectors v_i such that the data always fits into cache.
    It is equivalent to a matrix vecor product Vr where V
    contains few rows and many columns. The computation is the same:

    skp = ( <v_0,r>, <v_1,r>, .. )

    Returns the vector skp.

    Arguments
    =========

    int n                             legth of v_i and r
    int k                             # vectors v_i
    magmaDoubleComplex *v                     v = (v_0 .. v_i.. v_k)
    magmaDoubleComplex *r                     r
    magmaDoubleComplex *skp                   vector[k] of scalar products (<v_i,r>...)

    =====================================================================  */

extern "C" int
magma_zgemvmdot(    int n, 
                    int k, 
                    magmaDoubleComplex *v, 
                    magmaDoubleComplex *r,
                    magmaDoubleComplex *d1,
                    magmaDoubleComplex *d2,
                    magmaDoubleComplex *skp ){
     
    int rows_left = k;
    int offset = 0;
    int chunk_size = 4;
    // process in chunks of 10 - has to be adapted to hardware and precision
    while( rows_left > (chunk_size) ){
        magma_zmdotc( n, chunk_size, v+offset*n, r, d1, d2, skp+offset );
        offset = offset + chunk_size;
        rows_left = rows_left-chunk_size;

    }
    // process rest
    magma_zmdotc( n, rows_left, v+offset*n, r, d1, d2, skp+offset ); 


   return MAGMA_SUCCESS;
}



