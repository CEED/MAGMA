#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/

#include "common_magma.h"
#include "../include/magmasparse_z.h"
#include "../../include/magma.h"


#define PRECISION_z



// every row is handled by one threadblock
__global__ void 
magma_zailu_csr_s_kernel(   magma_int_t Lnum_rows, 
                            magma_int_t Lnnz,  
                            magma_index_t *rowidxAL, 
                            magma_index_t *colidxAL,
                            const magmaDoubleComplex * __restrict__ AL, 
                            magma_index_t *rowptrL, 
                            magma_index_t *colidxL, 
                            magmaDoubleComplex *valL, 
                            magma_index_t *rowidxAU, 
                            magma_index_t *colidxAU,
                            const magmaDoubleComplex * __restrict__ AU, 
                            magma_index_t *rowptrU, 
                            magma_index_t *rowidxU, 
                            magmaDoubleComplex *valU ){

    int i, j;
    int k = blockDim.x * blockIdx.x + threadIdx.x;

    magmaDoubleComplex zero = MAGMA_Z_MAKE(0.0, 0.0);
    magmaDoubleComplex s, sp;
    int il, iu, jl, ju;


    if (k < Lnnz)
    {     

        i = (blockIdx.y == 0 ) ? rowidxAL[k] : rowidxAU[k]  ;
        j = (blockIdx.y == 0 ) ? colidxAL[k] : colidxAU[k]  ;

#if (__CUDA_ARCH__ >= 350) && (defined(PRECISION_d) || defined(PRECISION_s))
        s = (blockIdx.y == 0 ) ? __ldg( AL+k ) : __ldg( AU+k );
#else
        s = (blockIdx.y == 0 ) ? AL[k] : AU[k] ;
#endif

        il = rowptrL[i];
        iu = rowptrU[j];

        while (il < rowptrL[i+1] && iu < rowptrU[j+1])
        {
            sp = zero;
            jl = colidxL[il];
            ju = rowidxU[iu];

            // avoid branching
            sp = ( jl == ju ) ? valL[il] * valU[iu] : sp;
            s = ( jl == ju ) ? s-sp : s;
            il = ( jl <= ju ) ? il+1 : il;
            iu = ( jl >= ju ) ? iu+1 : iu;

        }
        // undo the last operation (it must be the last)
        s += sp;
        __syncthreads();
        // modify u entry
        if (blockIdx.y == 0)
            valL[il-1] =  s / valU[rowptrU[j+1]-1];
        else{
            valU[il-1] = s;
        }

    }

}// kernel 













/**
    Purpose
    -------
    
    This routine computes the ILU approximation of a matrix iteratively. 
    The idea is according to Edmond Chow's presentation at SIAM 2014.
    The input format of the matrix is Magma_CSRCOO for the upper and lower 
    triangular parts. Note however, that we flip col and rowidx for the 
    U-part.
    Every component of L and U is handled by one thread. 

    Arguments
    ---------

    @param
    A_L         magma_z_sparse_matrix
                input matrix L

    @param
    A_U         magma_z_sparse_matrix
                input matrix U

    @param
    L           magma_z_sparse_matrix
                input/output matrix L containing the ILU approximation

    @param
    U           magma_z_sparse_matrix
                input/output matrix U containing the ILU approximation

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_zailu_csr_a( magma_z_sparse_matrix A_L,
                   magma_z_sparse_matrix A_U,
                   magma_z_sparse_matrix L,
                   magma_z_sparse_matrix U ){
    
    int blocksize1 = 256;
    int blocksize2 = 1;

    int dimgrid1 = ( A_L.nnz + blocksize1 -1 ) / blocksize1;
    int dimgrid2 = 2;
    int dimgrid3 = 1;

    dim3 grid( dimgrid1, dimgrid2, dimgrid3 );
    dim3 block( blocksize1, blocksize2, 1 );
    magma_zailu_csr_s_kernel<<< grid, block, 0, magma_stream >>>
        ( A_L.num_rows, A_L.nnz, 
          A_L.rowidx, A_L.col, A_L.val, 
          L.row, L.col, L.val, 
          A_U.rowidx, A_U.col, A_U.val, 
          U.row, U.col, U.val );


    return MAGMA_SUCCESS;
}
