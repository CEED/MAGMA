#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/

#include "common_magma.h"
#include "../include/magmasparse_z.h"
#include "../../include/magma.h"


#define PRECISION_z



// every row is handled by one threadblock
__global__ void 
magma_zailu_csr_a_kernel(   magma_int_t num_rows, 
                            magma_int_t nnz,  
                            magma_index_t *rowidxA, 
                            magma_index_t *colidxA,
                            const magmaDoubleComplex * __restrict__ A, 
                            magma_index_t *rowptrL, 
                            magma_index_t *colidxL, 
                            magmaDoubleComplex *valL, 
                            magma_index_t *rowptrU, 
                            magma_index_t *rowidxU, 
                            magmaDoubleComplex *valU ){

    int i, j;
    int k = blockDim.x * blockIdx.x + threadIdx.x;

    magmaDoubleComplex zero = MAGMA_Z_MAKE(0.0, 0.0);
    magmaDoubleComplex s, sp;
    int il, iu, jl, ju;


    if (k < nnz)
    {     

         
        i = rowidxA[k];
        j = colidxA[k];
//for( int l=0; l<1; l++){
#if (__CUDA_ARCH__ >= 350) && (defined(PRECISION_d) || defined(PRECISION_s))
        s =  __ldg( A+k );
#else
        s =  A[k];
#endif

        il = rowptrL[i];
        iu = rowptrU[j];

        while (il < rowptrL[i+1] && iu < rowptrU[j+1])
        {
            sp = zero;
            jl = colidxL[il];
            ju = rowidxU[iu];

            // avoid branching
            sp = ( jl == ju ) ? valL[il] * valU[iu] : sp;
            s = ( jl == ju ) ? s-sp : s;
            il = ( jl <= ju ) ? il+1 : il;
            iu = ( jl >= ju ) ? iu+1 : iu;

        }
        // undo the last operation (it must be the last)
        s += sp;
        __syncthreads();
        // modify u entry
        if ( i>j )
            valL[il-1] =  s / valU[rowptrU[j+1]-1];
        else{
            valU[iu-1] = s;
        }

//    }
    }

}// kernel 



/**
    Purpose
    -------
    
    This routine computes the ILU approximation of a matrix iteratively. 
    The idea is according to Edmond Chow's presentation at SIAM 2014.
    The input format of the matrix is Magma_CSRCOO for the upper and lower 
    triangular parts. Note however, that we flip col and rowidx for the 
    U-part.
    Every component of L and U is handled by one thread. 

    Arguments
    ---------

    @param
    A           magma_z_sparse_matrix
                input matrix A determing initial guess & processing order

    @param
    L           magma_z_sparse_matrix
                input/output matrix L containing the ILU approximation

    @param
    U           magma_z_sparse_matrix
                input/output matrix U containing the ILU approximation

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_zailu_csr_a( magma_z_sparse_matrix A,
                   magma_z_sparse_matrix L,
                   magma_z_sparse_matrix U ){
    
    int blocksize1 = 64;
    int blocksize2 = 1;

    int dimgrid1 = ( A.nnz + blocksize1 -1 ) / blocksize1;
    int dimgrid2 = 1;
    int dimgrid3 = 1;

    dim3 grid( dimgrid1, dimgrid2, dimgrid3 );
    dim3 block( blocksize1, blocksize2, 1 );
    magma_zailu_csr_a_kernel<<< grid, block, 0, magma_stream >>>
        ( A.num_rows, A.nnz, 
          A.rowidx, A.col, A.val, 
          L.row, L.col, L.val, 
          U.row, U.col, U.val );


    return MAGMA_SUCCESS;
}
