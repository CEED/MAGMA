#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s
       @author Hartwig Anzt

*/
#include "common_magma.h"

#define BLOCK_SIZE 512

#define PRECISION_z


// These routines merge multiple kernels from zidr into one.

/* -------------------------------------------------------------------------- */

__global__ void
magma_zidr_smoothing_1_kernel(  
    int num_rows,
    int num_cols,
    magmaDoubleComplex *drs,
    magmaDoubleComplex *dr,
    magmaDoubleComplex *dt )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            dt[ i+j*num_rows ] =  drs[ i+j*num_rows ] - dr[ i+j*num_rows ];
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    dt = drs - dr

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n

    @param[in]
    drs         magmaDoubleComplex_ptr 
                vector

    @param[in]
    dr          magmaDoubleComplex_ptr 
                vector

    @param[in,out]
    dt          magmaDoubleComplex_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_zidr_smoothing_1(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaDoubleComplex_ptr drs,
    magmaDoubleComplex_ptr dr, 
    magmaDoubleComplex_ptr dt, 
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_zidr_smoothing_1_kernel<<<Gs, Bs, 0>>>( num_rows, num_cols, drs, dr, dt);

   return MAGMA_SUCCESS;
}



__global__ void
magma_zidr_smoothing_2_kernel(  
    int num_rows,
    int num_cols,
    magmaDoubleComplex omega,
    magmaDoubleComplex *dx,
    magmaDoubleComplex *dxs )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            dxs[ i+j*num_rows ] = dxs[ i+j*num_rows ] - omega * dxs[ i+j*num_rows ]
                    + omega * dx[ i+j*num_rows ] ;;
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    dxs = dxs - gamma*(dxs-dx)

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    omega       magmaDoubleComplex
                scalar
                
    @param[in]
    dx          magmaDoubleComplex_ptr 
                vector

    @param[in,out]
    dxs         magmaDoubleComplex_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_zidr_smoothing_2(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaDoubleComplex omega,
    magmaDoubleComplex_ptr dx,
    magmaDoubleComplex_ptr dxs, 
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_zidr_smoothing_2_kernel<<<Gs, Bs, 0>>>( num_rows, num_cols, omega, dx, dxs);

   return MAGMA_SUCCESS;
}

