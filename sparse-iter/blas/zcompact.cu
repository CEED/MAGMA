#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c
       @author Stan Tomov
*/
#include "common_magmasparse.h"

#define NB 64

/* =====================================================================
    Matrix is m x n, and is divided into block rows, each NB x n.
    Each CUDA block has NB threads to handle one block row.
    Each thread handles one row, iterating across all columns.
*/
__global__ void
zcompact_kernel(
    int m, int n,
    magmaDoubleComplex *dA, 
    int ldda,
    double *dnorms, 
    double tol,
    magma_int_t *active, 
    magma_int_t *cBlock)
{
    // dA is processed across row i (by the current thread)
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int cBlockSize = 0;
    if ( i < m ) {
        dA += i;
        
        for(int j = 0; j<n; j++){
            if (dnorms[j] > tol && active[j]){
               dA[ldda*cBlockSize] = dA[ldda*j];
               cBlockSize++;
            }
            else if (i==0)
               active[j] = 0;
        }
    }

    if (i==0)
       *cBlock = cBlockSize;
}

__global__ void
zcompactactive_kernel(
    int m, 
    int n,
    magmaDoubleComplex *dA, 
    int ldda,
    magma_int_t *active)
{
    // dA is processed across row i (by the current thread)
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int cBlockSize = 0;
    if ( i < m ) {
        dA += i;

        for(int j = 0; j<n; j++){
            if (active[j]){
               dA[ldda*cBlockSize] = dA[ldda*j];
               cBlockSize++;
            }
        }
    }
}


/* ===================================================================== */
/**
    Purpose
    -------
    ZCOMPACT takes a set of n vectors of size m (in dA) and their norms and
    compacts them into the cBlock size<=n vectors that have norms > tol.
    The active mask array has 1 or 0, showing if a vector remained or not
    in the compacted resulting set of vectors.
    
    Arguments
    ---------
    @param[in]
    m           INTEGER
                The number of rows of the matrix dA.  M >= 0.
    
    @param[in]
    n           INTEGER
                The number of columns of the matrix dA.  N >= 0.
    
    @param[in,out]
    dA          COMPLEX DOUBLE PRECISION array, dimension (LDDA,N)
                The m by n matrix dA.
    
    @param[in]
    ldda        INTEGER
                The leading dimension of the array dA.  LDDA >= max(1,M).
    
    @param[in]
    dnorms      DOUBLE PRECISION array, dimension N
                The norms of the N vectors in dA

    @param[in]
    tol         DOUBLE PRECISON
                The tolerance value used in the criteria to compact or not.

    @param[in,out]
    active      INTEGER array, dimension N
                A mask of 1s and 0s showing if a vector remains or has been removed
            
    @param[in,out]
    cBlock      magmaInt_ptr
                The number of vectors that remain in dA (i.e., with norms > tol).
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_zcompact(
    magma_int_t m, 
    magma_int_t n,
    magmaDoubleComplex_ptr dA, 
    magma_int_t ldda,
    magmaDouble_ptr dnorms, 
    double tol, 
    magmaInt_ptr active,
    magmaInt_ptr cBlock,
    magma_queue_t queue )
{
    magma_int_t info = 0;
    if ( m < 0 )
        info = -1;
    else if ( n < 0 )
        info = -2;
    else if ( ldda < max(1,m))
        info = -4;
    
    if ( info != 0 ) {
        magma_xerbla( __func__, -(info) );
        return info;
    }
    
    if ( m == 0 || n == 0 )
        return info;
    
    dim3 threads( NB );
    dim3 grid( magma_ceildiv( m, NB ) );
    
    zcompact_kernel<<< grid, threads, 0, queue->hip_stream() >>>(
            m, n, dA, ldda, dnorms, tol, active, active+n );

    magma_igetvector( 1, active+n, 1, cBlock, 1 );
    return info;
}


/* ===================================================================== */
/**
    Purpose
    -------
    ZCOMPACTACTIVE takes a set of n vectors of size m (in dA) and an
    array of 1s and 0sindicating which vectors to compact (for 1s) and
    which to disregard (for 0s).

    Arguments
    ---------
    @param[in]
    m       INTEGER
            The number of rows of the matrix dA.  M >= 0.

    @param[in]
    n       INTEGER
            The number of columns of the matrix dA.  N >= 0.

    @param[in,out]
    dA      COMPLEX DOUBLE PRECISION array, dimension (LDDA,N)
            The m by n matrix dA.

    @param[in]
    ldda    INTEGER
            The leading dimension of the array dA.  LDDA >= max(1,M).

    @param[in]
    active  INTEGER array, dimension N
            A mask of 1s and 0s showing if a vector remains or has been removed
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_z
    ********************************************************************/

extern "C" magma_int_t
magma_zcompactActive(
    magma_int_t m, 
    magma_int_t n,
    magmaDoubleComplex_ptr dA, 
    magma_int_t ldda,
    magmaInt_ptr active,
    magma_queue_t queue )
{
    magma_int_t info = 0;
    if ( m < 0 )
        info = -1;
    else if ( n < 0 )
        info = -2;
    else if ( ldda < max(1,m))
        info = -4;

    if ( info != 0 ) {
        magma_xerbla( __func__, -(info) );
        return info;
    }

    if ( m == 0 || n == 0 )
        return info;

    dim3 threads( NB );
    dim3 grid( magma_ceildiv( m, NB ) );

    zcompactactive_kernel<<< grid, threads, 0, queue->hip_stream() >>>(
            m, n, dA, ldda, active);
    return info;
}

/* ===================================================================== */
