#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> s d c
       @author Stan Tomov
*/
#include "common_magma.h"
#include <assert.h>

#define NB 512

/* =====================================================================
    Matrix is m x n, and is divided into block rows, each NB x n.
    Each CUDA block has NB threads to handle one block row.
    Each thread handles one row, iterating across all columns.
*/
__global__ void
zcompact_kernel(
    int m, int n,
    magmaDoubleComplex *dA, int ldda,
    double *dnorms, double tol,
    magma_index_t *active, magma_index_t *cBlock)
{
    // dA is processed across row i (by the current thread)
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int cBlockSize = 0;
    if ( i < m ) {
        dA += i;
        
        for(int j = 0; j<n; j++){
            if (dnorms[j] > tol){
               dA[ldda*cBlockSize] = dA[ldda*j];
               cBlockSize++;

               if (i==0)
                  active[j] = 1;
            }
            else if (i==0)
               active[j] = 0;
        }
    }

    if (i==0)
       *cBlock = cBlockSize;
}

__global__ void
zcompactactive_kernel(
    int m, int n,
    magmaDoubleComplex *dA, int ldda,
    magma_index_t *active)
{
    // dA is processed across row i (by the current thread)
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int cBlockSize = 0;
    if ( i < m ) {
        dA += i;

        for(int j = 0; j<n; j++){
            if (active[j]){
               dA[ldda*cBlockSize] = dA[ldda*j];
               cBlockSize++;
            }
        }
    }
}


/* ===================================================================== */
extern "C" void
magma_zcompact(
    magma_int_t m, magma_int_t n,
    magmaDoubleComplex *dA, magma_int_t ldda,
    double *dnorms, double tol, 
    magma_index_t *active, magma_index_t *cBlock)
{
/*
    Purpose
    =======
    ZCOMPACT takes a set of n vectors of size m (in dA) and their norms and
    compacts them into the cBlock size<=n vectors that have norms > tol.
    The active mask array has 1 or 0, showing if a vector remained or not
    in the compacted resulting set of vectors.
    
    Arguments
    =========    
    M       (input) INTEGER
            The number of rows of the matrix dA.  M >= 0.
    
    N       (input) INTEGER
            The number of columns of the matrix dA.  N >= 0.
    
    dA      (input/output) COMPLEX DOUBLE PRECISION array, dimension (LDDA,N)
            The m by n matrix dA.
    
    LDDA    (input) INTEGER
            The leading dimension of the array dA.  LDDA >= max(1,M).
    
    DNORMS  (input) DOUBLE PRECISION array, dimension N
            The norms of the N vectors in dA

    TOL     (input) DOUBLE PRECISON
            The tolerance value used in the criteria to compact or not.

    ACTIVE  (output) INTEGER array, dimension N
            A mask of 1s and 0s showing if a vector remains or has been removed

    CBLOCK  (output)
            The number of vectors that remain in dA (i.e., with norms > tol).
    =====================================================================   */

    magma_int_t info = 0;
    if ( m < 0 )
        info = -1;
    else if ( n < 0 )
        info = -2;
    else if ( ldda < max(1,m))
        info = -4;
    
    if ( info != 0 ) {
        magma_xerbla( __func__, -(info) );
        return;
    }
    
    if ( m == 0 || n == 0 )
        return;
    
    dim3 threads( NB );
    dim3 grid( (m + NB - 1)/NB );
    
    zcompact_kernel<<< grid, threads, 0, magma_stream >>>(
            m, n, dA, ldda, dnorms, tol, active, cBlock );
}

/* ===================================================================== */

extern "C" void
magma_zcompactActive(
    magma_int_t m, magma_int_t n,
    magmaDoubleComplex *dA, magma_int_t ldda,
    magma_index_t *active)
{
/*
    Purpose
    =======
    ZCOMPACTACTIVE takes a set of n vectors of size m (in dA) and an
    array of 1s and 0sindicating which vectors to compact (for 1s) and
    which to disregard (for 0s).

    Arguments
    =========
    M       (input) INTEGER
            The number of rows of the matrix dA.  M >= 0.

    N       (input) INTEGER
            The number of columns of the matrix dA.  N >= 0.

    dA      (input/output) COMPLEX DOUBLE PRECISION array, dimension (LDDA,N)
            The m by n matrix dA.

    LDDA    (input) INTEGER
            The leading dimension of the array dA.  LDDA >= max(1,M).

    ACTIVE  (input) INTEGER array, dimension N
            A mask of 1s and 0s showing if a vector remains or has been removed
    =====================================================================     */

    magma_int_t info = 0;
    if ( m < 0 )
        info = -1;
    else if ( n < 0 )
        info = -2;
    else if ( ldda < max(1,m))
        info = -4;

    if ( info != 0 ) {
        magma_xerbla( __func__, -(info) );
        return;
    }

    if ( m == 0 || n == 0 )
        return;

    dim3 threads( NB );
    dim3 grid( (m + NB - 1)/NB );

    zcompactactive_kernel<<< grid, threads, 0, magma_stream >>>(
            m, n, dA, ldda, active);
}

/* ===================================================================== */
