#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s
       @author Hartwig Anzt

*/
#include "common_magmasparse.h"

#define BLOCK_SIZE 256

#define PRECISION_z


// These routines merge multiple kernels from zmergebicgstab into one
// This is the code used for the ASHES2014 paper
// "Accelerating Krylov Subspace Solvers on Graphics Processing Units".
// notice that only CSR format is supported so far.


// accelerated reduction for one vector
__global__ void
magma_zreduce_kernel_spmv1(    
    int Gs,
    int n, 
    magmaDoubleComplex * vtmp,
    magmaDoubleComplex * vtmp2 )
{
    extern __shared__ magmaDoubleComplex temp[];    
    int Idx = threadIdx.x;
    int blockSize = 128;
    int gridSize = blockSize  * 2 * gridDim.x; 
    temp[Idx] = MAGMA_Z_MAKE( 0.0, 0.0);
    int i = blockIdx.x * ( blockSize * 2 ) + Idx;   
    while (i < Gs ) {
        temp[ Idx  ] += vtmp[ i ]; 
        temp[ Idx  ] += ( i + blockSize < Gs ) ? vtmp[ i + blockSize ] 
                                                : MAGMA_Z_MAKE( 0.0, 0.0); 
        i += gridSize;
    }
    __syncthreads();
    if ( Idx < 64 ){
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            temp[ Idx ] += temp[ Idx + 32 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 8  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 4  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 2  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 1  ]; __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    if ( Idx == 0 ){
        vtmp2[ blockIdx.x ] = temp[ 0 ];
    }
}


__global__ void
magma_zbicgmerge_spmv1_kernel(  
    int n,
    magmaDoubleComplex * dval, 
    magma_index_t * drowptr, 
    magma_index_t * dcolind,
    magmaDoubleComplex * p,
    magmaDoubleComplex * r,
    magmaDoubleComplex * v,
    magmaDoubleComplex * vtmp)
{
    extern __shared__ magmaDoubleComplex temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int j;

    if( i<n ){
        magmaDoubleComplex dot = MAGMA_Z_ZERO;
        int start = drowptr[ i ];
        int end = drowptr[ i+1 ];
        for( j=start; j<end; j++)
            dot += dval[ j ] * p[ dcolind[j] ];
        v[ i ] =  dot;
    }

    __syncthreads(); 

    temp[ Idx ] = ( i < n ) ? v[ i ] * r[ i ] : MAGMA_Z_MAKE( 0.0, 0.0);
    __syncthreads();
    if ( Idx < 128 ){
        temp[ Idx ] += temp[ Idx + 128 ];
    }
    __syncthreads();
    if ( Idx < 64 ){
        temp[ Idx ] += temp[ Idx + 64 ];
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            temp[ Idx ] += temp[ Idx + 32 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 16 ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 8  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 4  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 2  ]; __syncthreads();
            temp[ Idx ] += temp[ Idx + 1  ]; __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            temp2[ Idx ] += temp2[ Idx + 32 ];
            temp2[ Idx ] += temp2[ Idx + 16 ];
            temp2[ Idx ] += temp2[ Idx + 8 ];
            temp2[ Idx ] += temp2[ Idx + 4 ];
            temp2[ Idx ] += temp2[ Idx + 2 ];
            temp2[ Idx ] += temp2[ Idx + 1 ];
        }
    #endif

    if ( Idx == 0 ){
            vtmp[ blockIdx.x ] = temp[ 0 ];
    }
}

__global__ void
magma_zbicgstab_alphakernel(  
                    magmaDoubleComplex * skp ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        magmaDoubleComplex tmp = skp[0];
        skp[0] = skp[4]/tmp;
    }
}

/**
    Purpose
    -------

    Merges the first SpmV using CSR with the dot product 
    and the computation of alpha

    Arguments
    ---------

    @param[in]
    A           magma_z_matrix
                system matrix

    @param[in]
    d1          magmaDoubleComplex_ptr
                temporary vector

    @param[in]
    d2          magmaDoubleComplex_ptr
                temporary vector

    @param[in]
    dp          magmaDoubleComplex_ptr
                input vector p

    @param[in]
    dr          magmaDoubleComplex_ptr
                input vector r

    @param[in]
    dv          magmaDoubleComplex_ptr
                output vector v

    @param[in,out]
    skp         magmaDoubleComplex_ptr
                array for parameters ( skp[0]=alpha )

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_zbicgmerge_spmv1(
    magma_z_matrix A,
    magmaDoubleComplex_ptr d1,
    magmaDoubleComplex_ptr d2,
    magmaDoubleComplex_ptr dp,
    magmaDoubleComplex_ptr dr,
    magmaDoubleComplex_ptr dv,
    magmaDoubleComplex_ptr skp,
    magma_queue_t queue )
{
    // set queue for old dense routines
    magma_queue_t orig_queue;
    magmablasGetKernelStream( &orig_queue );

    int n = A.num_rows;
    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( magma_ceildiv( n, local_block_size ) );
    dim3 Gs_next;
    int Ms =  local_block_size * sizeof( magmaDoubleComplex ); 
    magmaDoubleComplex_ptr aux1 = d1, aux2 = d2;
    int b = 1;        

    if ( A.storage_type == Magma_CSR)
        magma_zbicgmerge_spmv1_kernel<<< Gs, Bs, Ms, queue->hip_stream()>>>
                    ( n, A.dval, A.drow, A.dcol, dp, dr, dv, d1 );
    else
        printf("error: only CSR format supported.\n");

    while( Gs.x > 1 ) {
        Gs_next.x = magma_ceildiv( Gs.x, Bs.x );
        if ( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_zreduce_kernel_spmv1<<< Gs_next.x/2, Bs.x/2, Ms/2, queue->hip_stream()>>> 
                            ( Gs.x, n, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if ( b ) { aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }


    magma_zcopyvector( 1, aux1, 1, skp, 1 );
    dim3 Bs2( 2 );
    dim3 Gs2( 1 );
    magma_zbicgstab_alphakernel<<< Gs2, Bs2, 0, queue->hip_stream()>>>( skp );

   magmablasSetKernelStream( orig_queue );
   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

// accelerated block reduction for multiple vectors
__global__ void
magma_zreduce_kernel_spmv2( 
    int Gs,
    int n, 
    magmaDoubleComplex * vtmp,
    magmaDoubleComplex * vtmp2 )
{
    extern __shared__ magmaDoubleComplex temp[];    
    int Idx = threadIdx.x;
    int blockSize = 128;
    int gridSize = blockSize  * 2 * gridDim.x; 
    int j;

    for( j=0; j<2; j++){
        int i = blockIdx.x * ( blockSize * 2 ) + Idx;   
        temp[Idx+j*(blockSize)] = MAGMA_Z_MAKE( 0.0, 0.0);
        while (i < Gs ) {
            temp[ Idx+j*(blockSize)  ] += vtmp[ i+j*n ]; 
            temp[ Idx+j*(blockSize)  ] += 
                ( i + (blockSize) < Gs ) ? vtmp[ i+j*n + (blockSize) ] 
                : MAGMA_Z_MAKE( 0.0, 0.0); 
            i += gridSize;
        }
    }
    __syncthreads();
    if ( Idx < 64 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 64 ];
        }
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 32 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 16 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 8 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 4 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 2 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 32 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 16 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 8 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 4 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 2 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 1 ];
            }
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 32 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 16 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 8 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 4 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 2 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 1 ];
            }
        }
    #endif
    if ( Idx == 0 ){
        for( j=0; j<2; j++){
            vtmp2[ blockIdx.x+j*n ] = temp[ j*(blockSize) ];
        }
    }
}

__global__ void
magma_zbicgmerge_spmv2_kernel(  
    int n,
    magmaDoubleComplex * dval, 
    magma_index_t * drowptr, 
    magma_index_t * dcolind,
    magmaDoubleComplex * s,
    magmaDoubleComplex * t,
    magmaDoubleComplex * vtmp )
{
    extern __shared__ magmaDoubleComplex temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int j;

    if( i<n ){
        magmaDoubleComplex dot = MAGMA_Z_ZERO;
        int start = drowptr[ i ];
        int end = drowptr[ i+1 ];
        for( j=start; j<end; j++)
            dot += dval[ j ] * s[ dcolind[j] ];
        t[ i ] =  dot;
    }

    __syncthreads(); 

    // 2 vectors 
    if (i<n){
            magmaDoubleComplex tmp2 = t[i];
            temp[Idx] = s[i] * tmp2;
            temp[Idx+blockDim.x] = tmp2 * tmp2;
    }
    else {
        for( j=0; j<2; j++)
            temp[Idx+j*blockDim.x] = MAGMA_Z_MAKE( 0.0, 0.0);
    }
    __syncthreads();
    if ( Idx < 128 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 128 ];
        }
    }
    __syncthreads();
    if ( Idx < 64 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 64 ];
        }
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 32 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 16 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 8 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 4 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 2 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    if ( Idx == 0 ){
        for( j=0; j<2; j++){
            vtmp[ blockIdx.x+j*n ] = temp[ j*blockDim.x ];
        }
    }
}

__global__ void
magma_zbicgstab_omegakernel(  
                    magmaDoubleComplex * skp ){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        skp[2] = skp[6]/skp[7];
        skp[3] = skp[4];
    }
}

/**
    Purpose
    -------

    Merges the second SpmV using CSR with the dot product 
    and the computation of omega

    Arguments
    ---------

    @param[in]
    A           magma_z_matrix
                input matrix 

    @param[in]
    d1          magmaDoubleComplex_ptr
                temporary vector

    @param[in]
    d2          magmaDoubleComplex_ptr
                temporary vector

    @param[in]
    ds          magmaDoubleComplex_ptr
                input vector s

    @param[in]
    dt          magmaDoubleComplex_ptr
                output vector t

    @param[in,out]
    skp         magmaDoubleComplex_ptr
                array for parameters

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_zbicgmerge_spmv2(
    magma_z_matrix A,
    magmaDoubleComplex_ptr d1,
    magmaDoubleComplex_ptr d2,
    magmaDoubleComplex_ptr ds,
    magmaDoubleComplex_ptr dt,
    magmaDoubleComplex_ptr skp,
    magma_queue_t queue )
{
    // set queue for old dense routines
    magma_queue_t orig_queue;
    magmablasGetKernelStream( &orig_queue );

    int n = A.num_rows;
    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( magma_ceildiv( n, local_block_size ) );
    dim3 Gs_next;
    int Ms =  2*local_block_size * sizeof( magmaDoubleComplex ); 
    magmaDoubleComplex_ptr aux1 = d1, aux2 = d2;
    int b = 1;        
    if ( A.storage_type == Magma_CSR)
        magma_zbicgmerge_spmv2_kernel<<< Gs, Bs, Ms, queue->hip_stream()>>>
                    ( n, A.dval, A.drow, A.dcol, ds, dt, d1 );
    else
        printf("error: only CSR format supported.\n");

    while( Gs.x > 1 ) {
        Gs_next.x = magma_ceildiv( Gs.x, Bs.x );
        if ( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_zreduce_kernel_spmv2<<< Gs_next.x/2, Bs.x/2, Ms/2, queue->hip_stream()>>> 
                    ( Gs.x, n, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if ( b ) { aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }


    magma_zcopyvector( 1, aux1, 1, skp+6, 1 );
    magma_zcopyvector( 1, aux1+n, 1, skp+7, 1 );
    dim3 Bs2( 2 );
    dim3 Gs2( 1 );
    magma_zbicgstab_omegakernel<<< Gs2, Bs2, 0, queue->hip_stream()>>>( skp );

   magmablasSetKernelStream( orig_queue );
   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */

__global__ void
magma_zbicgmerge_xrbeta_kernel(  
    int n, 
    magmaDoubleComplex * rr,
    magmaDoubleComplex * r,
    magmaDoubleComplex * p,
    magmaDoubleComplex * s,
    magmaDoubleComplex * t,
    magmaDoubleComplex * x, 
    magmaDoubleComplex * skp,
    magmaDoubleComplex * vtmp )
{
    extern __shared__ magmaDoubleComplex temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int j;

    magmaDoubleComplex alpha=skp[0];
    magmaDoubleComplex omega=skp[2];

    if( i<n ){
        magmaDoubleComplex sl;
        sl = s[i];
        x[i] = x[i] + alpha * p[i] + omega * sl;
        r[i] = sl - omega * t[i];
    }

    __syncthreads(); 

    // 2 vectors 
    if (i<n){
            magmaDoubleComplex tmp2 = r[i];
            temp[Idx] = rr[i] * tmp2;
            temp[Idx+blockDim.x] = tmp2 * tmp2;
    }
    else {
        for( j=0; j<2; j++)
            temp[Idx+j*blockDim.x] = MAGMA_Z_MAKE( 0.0, 0.0);
    }
    __syncthreads();
    if ( Idx < 128 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 128 ];
        }
    }
    __syncthreads();
    if ( Idx < 64 ){
        for( j=0; j<2; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 64 ];
        }
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 32 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 16 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 8 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 4 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 2 ];
                __syncthreads();
            for( j=0; j<2; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<2; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    if ( Idx == 0 ){
        for( j=0; j<2; j++){
            vtmp[ blockIdx.x+j*n ] = temp[ j*blockDim.x ];
        }
    }
}

__global__ void
magma_zbicgstab_betakernel(  
    magmaDoubleComplex * skp )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i==0 ){
        magmaDoubleComplex tmp1 = skp[4]/skp[3];
        magmaDoubleComplex tmp2 = skp[0] / skp[2];
        skp[1] =  tmp1*tmp2;
    }
}

/**
    Purpose
    -------

    Merges the second SpmV using CSR with the dot product 
    and the computation of omega

    Arguments
    ---------

    @param[in]
    n           int
                dimension n

    @param[in]
    d1          magmaDoubleComplex_ptr
                temporary vector

    @param[in]
    d2          magmaDoubleComplex_ptr
                temporary vector

    @param[in]
    rr          magmaDoubleComplex_ptr
                input vector rr

    @param[in]
    r           magmaDoubleComplex_ptr
                input/output vector r

    @param[in]
    p           magmaDoubleComplex_ptr
                input vector p

    @param[in]
    s           magmaDoubleComplex_ptr
                input vector s

    @param[in]
    t           magmaDoubleComplex_ptr
                input vector t

    @param[out]
    x           magmaDoubleComplex_ptr
                output vector x

    @param[in]
    skp         magmaDoubleComplex_ptr
                array for parameters

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_zbicgmerge_xrbeta(
    magma_int_t n,
    magmaDoubleComplex_ptr d1,
    magmaDoubleComplex_ptr d2,
    magmaDoubleComplex_ptr rr,
    magmaDoubleComplex_ptr r,
    magmaDoubleComplex_ptr p,
    magmaDoubleComplex_ptr s,
    magmaDoubleComplex_ptr t,
    magmaDoubleComplex_ptr x, 
    magmaDoubleComplex_ptr skp,
    magma_queue_t queue )
{
    // set queue for old dense routines
    magma_queue_t orig_queue;
    magmablasGetKernelStream( &orig_queue );

    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( magma_ceildiv( n, local_block_size ) );
    dim3 Gs_next;
    int Ms =  2*local_block_size * sizeof( magmaDoubleComplex ); 
    magmaDoubleComplex_ptr aux1 = d1, aux2 = d2;
    int b = 1;        
    magma_zbicgmerge_xrbeta_kernel<<< Gs, Bs, Ms, queue->hip_stream()>>>
                    ( n, rr, r, p, s, t, x, skp, d1);  

    while( Gs.x > 1 ) {
        Gs_next.x = magma_ceildiv( Gs.x, Bs.x );
        if ( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_zreduce_kernel_spmv2<<< Gs_next.x/2, Bs.x/2, Ms/2, queue->hip_stream()>>> 
                            ( Gs.x, n, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if ( b ) { aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }


    magma_zcopyvector( 1, aux1, 1, skp+4, 1 );
    magma_zcopyvector( 1, aux1+n, 1, skp+5, 1 );
    dim3 Bs2( 2 );
    dim3 Gs2( 1 );
    magma_zbicgstab_betakernel<<< Gs2, Bs2, 0, queue->hip_stream()>>>( skp );

   magmablasSetKernelStream( orig_queue );
   return MAGMA_SUCCESS;
}

/* -------------------------------------------------------------------------- */
