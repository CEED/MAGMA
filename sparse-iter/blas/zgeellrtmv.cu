#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/

#include "common_magma.h"

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
zgeellrtmv_kernel_32( 
    int num_rows, 
    int num_cols,
    magmaDoubleComplex alpha, 
    magmaDoubleComplex * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowlength,
    magmaDoubleComplex * dx,
    magmaDoubleComplex beta, 
    magmaDoubleComplex * dy,
    int T,
    int alignment )
{
    int idx = blockIdx.y * gridDim.x * blockDim.x + 
              blockDim.x * blockIdx.x + threadIdx.x;  // global thread index
    int idb = threadIdx.x;   // local thread index
    int idp = idb%T;  // number of threads assigned to one row
    int i = idx/T;  // row index
    
    extern __shared__ magmaDoubleComplex shared[];

    if (i < num_rows ) {
        magmaDoubleComplex dot = MAGMA_Z_MAKE(0.0, 0.0);
        int max_ = magma_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_; k++ ) {
            // original code in paper (not working for me)
            //magmaDoubleComplex val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            magmaDoubleComplex val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * dx[ col ];
        }
        shared[idb]  = dot;
        if ( idp < 16 ) {
            shared[idb] += shared[idb+16];
            if ( idp < 8 ) shared[idb] += shared[idb+8];
            if ( idp < 4 ) shared[idb] += shared[idb+4];
            if ( idp < 2 ) shared[idb] += shared[idb+2];
            if ( idp == 0 ) {
                dy[i] = (shared[idb]+shared[idb+1])*alpha + beta*dy [i];
            }
        }
    }
}

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
zgeellrtmv_kernel_16( 
    int num_rows, 
    int num_cols,
    magmaDoubleComplex alpha, 
    magmaDoubleComplex * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowlength,
    magmaDoubleComplex * dx,
    magmaDoubleComplex beta, 
    magmaDoubleComplex * dy,
    int T,
    int alignment )
{
    int idx = blockIdx.y * gridDim.x * blockDim.x + 
              blockDim.x * blockIdx.x + threadIdx.x;  // global thread index
    int idb = threadIdx.x;   // local thread index
    int idp = idb%T;  // number of threads assigned to one row
    int i = idx/T;  // row index
    
    extern __shared__ magmaDoubleComplex shared[];

    if (i < num_rows ) {
        magmaDoubleComplex dot = MAGMA_Z_MAKE(0.0, 0.0);
        int max_ = magma_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_; k++ ) {
            // original code in paper (not working for me)
            //magmaDoubleComplex val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            magmaDoubleComplex val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * dx[ col ];
        }
        shared[idb]  = dot;
        if ( idp < 8 ) {
            shared[idb] += shared[idb+8];
            if ( idp < 4 ) shared[idb] += shared[idb+4];
            if ( idp < 2 ) shared[idb] += shared[idb+2];
            if ( idp == 0 ) {
                dy[i] = (shared[idb]+shared[idb+1])*alpha + beta*dy [i];
            }
        }
    }
}

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
zgeellrtmv_kernel_8( 
    int num_rows, 
    int num_cols,
    magmaDoubleComplex alpha, 
    magmaDoubleComplex * dval, 
    magma_index_t * dcolind,
    magma_index_t * drowlength,
    magmaDoubleComplex * dx,
    magmaDoubleComplex beta, 
    magmaDoubleComplex * dy,
    int T,
    int alignment )
{
    int idx = blockIdx.y * gridDim.x * blockDim.x + 
              blockDim.x * blockIdx.x + threadIdx.x;  // global thread index
    int idb = threadIdx.x;   // local thread index
    int idp = idb%T;  // number of threads assigned to one row
    int i = idx/T;  // row index
    
    extern __shared__ magmaDoubleComplex shared[];

    if (i < num_rows ) {
        magmaDoubleComplex dot = MAGMA_Z_MAKE(0.0, 0.0);
        int max_ = magma_ceildiv( drowlength[i], T );  
            // number of elements each thread handles

        for ( int k = 0; k < max_; k++ ) {
            // original code in paper (not working for me)
            //magmaDoubleComplex val = dval[ k*(T*alignment)+(i*T)+idp ];  
            //int col = dcolind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            magmaDoubleComplex val = dval[ k*(T)+(i*alignment)+idp ];
            int col = dcolind [ k*(T)+(i*alignment)+idp ];

            dot += val * dx[ col ];
        }
        shared[idb]  = dot;
        if ( idp < 4 ) {
            shared[idb] += shared[idb+4];
            if ( idp < 2 ) shared[idb] += shared[idb+2];
            if ( idp == 0 ) {
                dy[i] = (shared[idb]+shared[idb+1])*alpha + beta*dy [i];
            }
        }
    }
}



/**
    Purpose
    -------
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    Input format is ELLRT. The ideas are taken from 
    "Improving the performance of the sparse matrix
    vector product with GPUs", (CIT 2010), 
    and modified to provide correct values.

    
    Arguments
    ---------

    @param[in]
    transA      magma_trans_t
                transposition parameter for A
    @param[in]
    m           magma_int_t
                number of rows 

    @param[in]
    n           magma_int_t
                number of columns

    @param[in]
    nnz_per_row magma_int_t
                max number of nonzeros in a row

    @param[in]
    alpha       magmaDoubleComplex
                scalar alpha

    @param[in]
    dval        magmaDoubleComplex_ptr
                val array

    @param[in]
    dcolind     magmaIndex_ptr
                col indices  

    @param[in]
    drowlength  magmaIndex_ptr
                number of elements in each row

    @param[in]
    dx          magmaDoubleComplex_ptr
                input vector x

    @param[in]
    beta        magmaDoubleComplex
                scalar beta

    @param[out]
    dy          magmaDoubleComplex_ptr
                output vector y

    @param[in]
    blocksize   magma_int_t
                threads per block

    @param[in]
    alignment   magma_int_t
                threads assigned to each row

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zblas
    ********************************************************************/

extern "C" magma_int_t
magma_zgeellrtmv(
    magma_trans_t transA,
    magma_int_t m, magma_int_t n,
    magma_int_t nnz_per_row,
    magmaDoubleComplex alpha,
    magmaDoubleComplex_ptr dval,
    magmaIndex_ptr dcolind,
    magmaIndex_ptr drowlength,
    magmaDoubleComplex_ptr dx,
    magmaDoubleComplex beta,
    magmaDoubleComplex_ptr dy,
    magma_int_t alignment,
    magma_int_t blocksize,
    magma_queue_t queue )
{
    int num_blocks = magma_ceildiv( m, blocksize );

    magma_int_t num_threads = alignment*blocksize;
    magma_int_t threads = alignment*blocksize;

    int real_row_length = magma_roundup( nnz_per_row, alignment );

    magma_int_t arch = magma_getdevice_arch();
    if ( arch < 200 && num_threads > 256 )
        printf("error: too much shared memory requested.\n");

    int dimgrid1 = int( sqrt( double( num_blocks )));
    int dimgrid2 = magma_ceildiv( num_blocks, dimgrid1 );
    dim3 grid( dimgrid1, dimgrid2, 1);

    int Ms = alignment * blocksize * sizeof( magmaDoubleComplex );
    // printf("launch kernel: %dx%d %d %d\n", grid.x, grid.y, num_threads, Ms);

    if ( alignment == 32 ) {
        zgeellrtmv_kernel_32<<< grid, threads, Ms, queue >>>
                 ( m, n, alpha, dval, dcolind, drowlength, dx, beta, dy, 
                                                 alignment, real_row_length );
    }
    else if ( alignment == 16 ) {
        zgeellrtmv_kernel_16<<< grid, threads, Ms, queue >>>
                 ( m, n, alpha, dval, dcolind, drowlength, dx, beta, dy, 
                                                 alignment, real_row_length );
    }
    else if ( alignment == 8 ) {
        zgeellrtmv_kernel_8<<< grid, threads, Ms, queue >>>
                 ( m, n, alpha, dval, dcolind, drowlength, dx, beta, dy, 
                                                 alignment, real_row_length );
    }
    else {
        printf("error: alignment %d not supported.\n", int(alignment) );
        return MAGMA_ERR_NOT_SUPPORTED;
    }

    return MAGMA_SUCCESS;
}
