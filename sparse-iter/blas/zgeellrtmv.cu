#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/

#include "common_magma.h"

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
zgeellrtmv_kernel_32( int num_rows, 
                 int num_cols,
                 magmaDoubleComplex alpha, 
                 magmaDoubleComplex *d_val, 
                 magma_index_t *d_colind,
                 magma_index_t *d_rowlength,
                 magmaDoubleComplex *d_x,
                 magmaDoubleComplex beta, 
                 magmaDoubleComplex *d_y,
                 int T,
                 int alignment )
{
int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ magmaDoubleComplex shared[];

    if(i < num_rows ){
        magmaDoubleComplex dot = MAGMA_Z_MAKE(0.0, 0.0);
        int max_ = (d_rowlength[i]+T-1)/T;  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //magmaDoubleComplex val = d_val[ k*(T*alignment)+(i*T)+idp ];  
            //int col = d_colind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            magmaDoubleComplex val = d_val[ k*(T)+(i*alignment)+idp ];
            int col = d_colind [ k*(T)+(i*alignment)+idp ];

            dot += val * d_x[ col ];
        }
        shared[idb]  = dot;
        if( idp < 16 ){
            shared[idb]+=shared[idb+16];
            if( idp < 8 ) shared[idb]+=shared[idb+8];
            if( idp < 4 ) shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                d_y[i] = (shared[idb]+shared[idb+1])*alpha + beta*d_y [i];
            }

        }
    }

}

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
zgeellrtmv_kernel_16( int num_rows, 
                 int num_cols,
                 magmaDoubleComplex alpha, 
                 magmaDoubleComplex *d_val, 
                 magma_index_t *d_colind,
                 magma_index_t *d_rowlength,
                 magmaDoubleComplex *d_x,
                 magmaDoubleComplex beta, 
                 magmaDoubleComplex *d_y,
                 int T,
                 int alignment )
{
int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ magmaDoubleComplex shared[];

    if(i < num_rows ){
        magmaDoubleComplex dot = MAGMA_Z_MAKE(0.0, 0.0);
        int max_ = (d_rowlength[i]+T-1)/T;  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //magmaDoubleComplex val = d_val[ k*(T*alignment)+(i*T)+idp ];  
            //int col = d_colind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            magmaDoubleComplex val = d_val[ k*(T)+(i*alignment)+idp ];
            int col = d_colind [ k*(T)+(i*alignment)+idp ];

            dot += val * d_x[ col ];
        }
        shared[idb]  = dot;
        if( idp < 8 ){
            shared[idb]+=shared[idb+8];
            if( idp < 4 ) shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                d_y[i] = (shared[idb]+shared[idb+1])*alpha + beta*d_y [i];
            }

        }
    }

}

//F. Vázquez, G. Ortega, J.J. Fernández, E.M. Garzón, Almeria University
__global__ void 
zgeellrtmv_kernel_8( int num_rows, 
                 int num_cols,
                 magmaDoubleComplex alpha, 
                 magmaDoubleComplex *d_val, 
                 magma_index_t *d_colind,
                 magma_index_t *d_rowlength,
                 magmaDoubleComplex *d_x,
                 magmaDoubleComplex beta, 
                 magmaDoubleComplex *d_y,
                 int T,
                 int alignment )
{
int idx = blockIdx.y * gridDim.x * blockDim.x + 
          blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
int idb = threadIdx.x ;  // local thread index
int idp = idb%T;  // number of threads assigned to one row
int i = idx/T;  // row index

extern __shared__ magmaDoubleComplex shared[];

    if(i < num_rows ){
        magmaDoubleComplex dot = MAGMA_Z_MAKE(0.0, 0.0);
        int max_ = (d_rowlength[i]+T-1)/T;  
            // number of elements each thread handles

        for ( int k = 0; k < max_ ; k++ ){

            // original code in paper (not working for me)
            //magmaDoubleComplex val = d_val[ k*(T*alignment)+(i*T)+idp ];  
            //int col = d_colind [ k*(T*alignment)+(i*T)+idp ];    

            // new code (working for me)        
            magmaDoubleComplex val = d_val[ k*(T)+(i*alignment)+idp ];
            int col = d_colind [ k*(T)+(i*alignment)+idp ];

            dot += val * d_x[ col ];
        }
        shared[idb]  = dot;
        if( idp < 4 ){
            shared[idb]+=shared[idb+4];
            if( idp < 2 ) shared[idb]+=shared[idb+2];
            if( idp == 0 ) {
                d_y[i] = (shared[idb]+shared[idb+1])*alpha + beta*d_y [i];
            }

        }
    }

}



/**
    Purpose
    -------
    
    This routine computes y = alpha *  A *  x + beta * y on the GPU.
    Input format is ELLRT. The ideas are taken from 
    "Improving the performance of the sparse matrix
    vector product with GPUs", (CIT 2010), 
    and modified to provide correct values.

    
    Arguments
    ---------

    @param
    transA      magma_trans_t
                transposition parameter for A
    @param
    m           magma_int_t
                number of rows 

    @param
    n           magma_int_t
                number of columns

    @param
    nnz_per_row magma_int_t
                max number of nonzeros in a row

    @param
    alpha       magmaDoubleComplex
                scalar alpha

    @param
    d_val       magmaDoubleComplex*
                val array

    @param
    d_colind    magma_int_t*
                col indices  

    @param
    d_rowlength magma_int_t*
                number of elements in each row

    @param
    d_x         magmaDoubleComplex*
                input vector x

    @param
    beta        magmaDoubleComplex
                scalar beta

    @param
    d_y         magmaDoubleComplex*
                output vector y

    @param
    blocksize   magma_int_t
                threads per block

    @param
    alignment   magma_int_t
                threads assigned to each row


    @ingroup magmasparse_zblas
    ********************************************************************/

extern "C" magma_int_t
magma_zgeellrtmv(  magma_trans_t transA,
                   magma_int_t m, magma_int_t n,
                   magma_int_t nnz_per_row,
                   magmaDoubleComplex alpha,
                   magmaDoubleComplex *d_val,
                   magma_index_t *d_colind,
                   magma_index_t *d_rowlength,
                   magmaDoubleComplex *d_x,
                   magmaDoubleComplex beta,
                   magmaDoubleComplex *d_y,
                   magma_int_t alignment,
                   magma_int_t blocksize ){


    int num_blocks = ( (m+blocksize-1)/blocksize);

    int num_threads = alignment*blocksize;

    int real_row_length = ((int)(nnz_per_row+alignment-1)/alignment)
                            *alignment;

    magma_int_t arch = magma_getdevice_arch();
    if ( arch < 200 && num_threads > 256 )
        printf("error: too much shared memory requested.\n");

    int dimgrid1 = sqrt(num_blocks);
    int dimgrid2 = (num_blocks + dimgrid1 -1 ) / dimgrid1;
    dim3 grid( dimgrid1, dimgrid2, 1);

    int Ms = alignment * blocksize * sizeof( magmaDoubleComplex );
    // printf("launch kernel: %dx%d %d %d\n", grid.x, grid.y, num_threads , Ms);

    if( alignment == 32 ){
        zgeellrtmv_kernel_32<<< grid, num_threads , Ms, magma_stream >>>
                 ( m, n, alpha, d_val, d_colind, d_rowlength, d_x, beta, d_y, 
                                                 alignment, real_row_length );
    }
    else if( alignment == 16 ){
        zgeellrtmv_kernel_16<<< grid, num_threads , Ms, magma_stream >>>
                 ( m, n, alpha, d_val, d_colind, d_rowlength, d_x, beta, d_y, 
                                                 alignment, real_row_length );
    }
    else if( alignment == 8 ){
        zgeellrtmv_kernel_8<<< grid, num_threads , Ms, magma_stream >>>
                 ( m, n, alpha, d_val, d_colind, d_rowlength, d_x, beta, d_y, 
                                                 alignment, real_row_length );
    }
    else{
        printf("error: alignment %d not supported.\n", alignment);
        exit(-1);
    }



   return MAGMA_SUCCESS;
}


