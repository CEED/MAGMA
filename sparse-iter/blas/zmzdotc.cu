#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s
       @author Hartwig Anzt

*/
#include "common_magma.h"

#define BLOCK_SIZE 256

#define PRECISION_z


// initialize arrays with zero
__global__ void
magma_zmzdotc_gpumemzero(  
    magmaDoubleComplex * d, 
    int n )
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;

   if( i < n ){
    for( int j=0; j<4; j++)
      d[ i+j*n ] = MAGMA_Z_MAKE( 0.0, 0.0 );
    }
}


// dot product for multiple vectors
__global__ void
magma_zmzdotc_kernel_1( 
    int Gs,
    int n, 
    magmaDoubleComplex * v0,
    magmaDoubleComplex * w0,
    magmaDoubleComplex * v1,
    magmaDoubleComplex * w1,
    magmaDoubleComplex * v2,
    magmaDoubleComplex * w2,
    magmaDoubleComplex * v3,
    magmaDoubleComplex * w3,
    magmaDoubleComplex * vtmp)
{
    extern __shared__ magmaDoubleComplex temp[]; 
    int Idx = threadIdx.x;   
    int i   = blockIdx.x * blockDim.x + Idx;
    int j;

    // 4 vectors v(i)/w(i)
    
    temp[ Idx ]                 = ( i < n ) ?
                v0[ i ] * w0[ i ] : MAGMA_Z_ZERO;
                
    temp[ Idx + blockDim.x ]    = ( i < n ) ?
                v1[ i ] * w1[ i ] : MAGMA_Z_ZERO;
                
    temp[ Idx + 2*blockDim.x ]  = ( i < n ) ?
                v2[ i ] * w2[ i ] : MAGMA_Z_ZERO;
                
    temp[ Idx + 3*blockDim.x ]  = ( i < n ) ?
                v3[ i ] * w3[ i ] : MAGMA_Z_ZERO;
               
    
    __syncthreads();
    if ( Idx < 128 ){
        for( j=0; j<4; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 128 ];
        }
    }
    __syncthreads();
    if ( Idx < 64 ){
        for( j=0; j<4; j++){
            temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 64 ];
        }
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            for( j=0; j<4; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 32 ];
                __syncthreads();
            for( j=0; j<4; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 16 ];
                __syncthreads();
            for( j=0; j<4; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 8 ];
                __syncthreads();
            for( j=0; j<4; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 4 ];
                __syncthreads();
            for( j=0; j<4; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 2 ];
                __syncthreads();
            for( j=0; j<4; j++)
                temp[ Idx+j*blockDim.x ] += temp[ Idx+j*blockDim.x + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
            for( j=0; j<4; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<4; j++){
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 32 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 16 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 8 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 4 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 2 ];
                temp2[ Idx+j*blockDim.x ] += temp2[ Idx+j*blockDim.x + 1 ];
            }
        }
    #endif
    if ( Idx == 0 ){
        for( j=0; j<4; j++){
            vtmp[ blockIdx.x+j*n ] = temp[ j*blockDim.x ];
        }
    }
}



// block reduction for 4 vectors
__global__ void
magma_zmzdotc_kernel_2( 
    int Gs,
    int n, 
    magmaDoubleComplex * vtmp,
    magmaDoubleComplex * vtmp2 )
{
    extern __shared__ magmaDoubleComplex temp[];    
    int Idx = threadIdx.x;
    int blockSize = 128;
    int gridSize = blockSize  * 2 * gridDim.x; 
    int j;

    for( j=0; j<4; j++){
        int i = blockIdx.x * ( blockSize * 2 ) + Idx;   
        temp[Idx+j*(blockSize)] = MAGMA_Z_ZERO;
        while (i < Gs ) {
            temp[ Idx+j*(blockSize)  ] += vtmp[ i+j*n ]; 
            temp[ Idx+j*(blockSize)  ] += 
                ( i + (blockSize) < Gs ) ? vtmp[ i+j*n + (blockSize) ] 
                                                : MAGMA_Z_ZERO;
            i += gridSize;
        }
    }
    __syncthreads();
    if ( Idx < 64 ){
        for( j=0; j<4; j++){
            temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 64 ];
        }
    }
    __syncthreads();
    #if defined(PRECISION_z) || defined(PRECISION_c)
        if( Idx < 32 ){
            for( j=0; j<4; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 32 ];
                __syncthreads();
            for( j=0; j<4; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 16 ];
                __syncthreads();
            for( j=0; j<4; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 8 ];
                __syncthreads();
            for( j=0; j<4; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 4 ];
                __syncthreads();
            for( j=0; j<4; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 2 ];
                __syncthreads();
            for( j=0; j<4; j++)
                temp[ Idx+j*(blockSize) ] += temp[ Idx+j*(blockSize) + 1 ];
                __syncthreads();
        }
    #endif
    #if defined(PRECISION_d)
        if( Idx < 32 ){
            volatile double *temp2 = temp;
            for( j=0; j<4; j++){
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 32 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 16 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 8 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 4 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 2 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 1 ];
            }
        }
    #endif
    #if defined(PRECISION_s)
        if( Idx < 32 ){
            volatile float *temp2 = temp;
            for( j=0; j<4; j++){
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 32 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 16 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 8 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 4 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 2 ];
                temp2[ Idx+j*(blockSize) ] += temp2[ Idx+j*(blockSize) + 1 ];
            }
        }
    #endif
    if ( Idx == 0 ){
        for( j=0; j<4; j++){
            vtmp2[ blockIdx.x+j*n ] = temp[ j*(blockSize) ];
        }
    }
}

/**
    Purpose
    -------

    Computes the scalar product of a set of 4 vectors such that

    skp[0,1,2,3] = [ <v_0,w_0>, <v_1,w_1>, <v_2,w_2>, <v3,w_3> ]

    Returns the vector skp.
    In case there are less dot products required, an easy workaround is
    given by doubling input.

    Arguments
    ---------

    @param[in]
    n           int
                length of v_i and w_i

    @param[in]                             
    v0          magmaDoubleComplex_ptr     
                input vector               

    @param[in]                                         
    w0          magmaDoubleComplex_ptr                 
                input vector                           
                                                       
    @param[in]                                         
    v1          magmaDoubleComplex_ptr                 
                input vector                           
                                                       
    @param[in]                                         
    w1          magmaDoubleComplex_ptr                 
                input vector          

    @param[in]                             
    v2          magmaDoubleComplex_ptr     
                input vector               

    @param[in]                                         
    w2          magmaDoubleComplex_ptr                 
                input vector                           
                                                       
    @param[in]                                         
    v3          magmaDoubleComplex_ptr                 
                input vector                           
                                                       
    @param[in]                                         
    w3          magmaDoubleComplex_ptr                 
                input vector          

    @param[in]
    d1          magmaDoubleComplex_ptr 
                workspace

    @param[in]
    d2          magmaDoubleComplex_ptr 
                workspace

    @param[out]
    skp         magmaDoubleComplex_ptr 
                vector[4] of scalar products [<v_i, w_i>]
                This vector is located on the host

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zblas
    ********************************************************************/

extern "C" magma_int_t
magma_zmzdotc(
    int n,  
    magmaDoubleComplex_ptr v0, 
    magmaDoubleComplex_ptr w0,
    magmaDoubleComplex_ptr v1, 
    magmaDoubleComplex_ptr w1,
    magmaDoubleComplex_ptr v2, 
    magmaDoubleComplex_ptr w2,
    magmaDoubleComplex_ptr v3, 
    magmaDoubleComplex_ptr w3,
    magmaDoubleComplex_ptr d1,
    magmaDoubleComplex_ptr d2,
    magmaDoubleComplex_ptr skp,
    magma_queue_t queue )
{
    // set queue for old dense routines
    magma_queue_t orig_queue;
    magmablasGetKernelStream( &orig_queue );

    int local_block_size=256;
    dim3 Bs( local_block_size );
    dim3 Gs( magma_ceildiv( n, local_block_size ) );
    dim3 Gs_next;
    int Ms = 4 * (local_block_size) * sizeof( magmaDoubleComplex ); // 4 skp 
    magmaDoubleComplex_ptr aux1 = d1, aux2 = d2;
    int b = 1;        


    magma_zmzdotc_kernel_1<<<Gs, Bs, Ms, queue>>>
            ( Gs.x, n, v0, w0, v1, w1, v2, w2, v3, w3, d1 );

/*
    // not necessary to zero GPU mem
    magma_zmzdotc_gpumemzero<<<Gs, Bs, 0>>>( d1, n*k,1 );
    magma_zmzdotc_gpumemzero<<<Gs, Bs, 0>>>( d2, n*k,1 );
    //magmablas_zlaset( MagmaUpperLower, n, k, d1, n );
    //magmablas_zlaset( MagmaUpperLower, n, k, d2, n );
    while( Gs.x > 1 ) {
        Gs_next.x = ( Gs.x+Bs.x-1 )/ Bs.x;
        magma_zblockreduce_kernel<<< Gs_next.x, Bs.x, Ms >>> 
                                        ( Gs.x, n, k, aux1, aux2 );
        Gs.x = Gs_next.x;
        b = 1 - b;
        if ( b ) { aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }
    for( int j=0; j<k; j++) {
            magma_zcopyvector( 1, aux1+j*n, 1, skp+j, 1 );
    }
*/
   
    while( Gs.x > 1 ) {
        Gs_next.x = ( Gs.x+Bs.x-1 )/ Bs.x;
        if ( Gs_next.x == 1 ) Gs_next.x = 2;
        magma_zmzdotc_kernel_2<<< Gs_next.x/2, Bs.x/2, Ms/2, queue >>> 
                    ( Gs.x, n, aux1, aux2 );
        Gs_next.x = Gs_next.x /2;
        Gs.x = Gs_next.x;
        b = 1 - b;
        if ( b ) { aux1 = d1; aux2 = d2; }
        else   { aux2 = d1; aux1 = d2; }
    }

        // copy vectors to host
    magma_zgetvector( 4 , aux1, 1, skp, 1 );
    
    magma_zprint_gpu( 4, 1, aux1, 4 );
    
    printf("the skp is: %d %d %d %d\n", skp[0], skp[1], skp[2], skp[3]);

   magmablasSetKernelStream( orig_queue );
   return MAGMA_SUCCESS;
}


