/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/

#include "common_magma.h"


#define  blockinfo(i,j)  blockinfo[(i)*c_blocks   + (j)]
#define A(i,j) A+((blockinfo(i,j)-1)*size_b*size_b)
#define x(i) x+(i*size_b)


/*  -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

    Purpose
    =======
    
    For a Block-CSR ILU factorization, this routine performs the triangular 
    solves.
    
    Arguments
    =========

    magma_int_t r_blocks            number of blocks
    magma_int_t size_b              blocksize in BCSR
    magma_int_t *ipiv               array containing pivots
    magmaDoubleComplex *x           input/output vector x

    ======================================================================    */

extern "C" magma_int_t
magma_zbcsrtrsv( char uplo,
                 magma_int_t r_blocks,
                 magma_int_t c_blocks,
                 magma_int_t size_b, 
                 magmaDoubleComplex *A,
                 magma_int_t *blockinfo,   
                 magmaDoubleComplex *x ){

    // some useful variables
    magmaDoubleComplex one = MAGMA_Z_MAKE(1.0, 0.0);
    magmaDoubleComplex mone = MAGMA_Z_MAKE(-1.0, 0.0);
    magma_int_t j,k;

    if( uplo==MagmaLower ){ 
        // forward solve
        for( k=0; k<r_blocks; k++){
            // do the forward triangular solve for block M(k,k): L(k,k)y = b
            magma_ztrsv(MagmaLower, MagmaNoTrans, MagmaUnit, size_b, A(k,k), 
                                                             size_b, x(k), 1 );

             // update for all nonzero blocks below M(k,k) 
                    // the respective values of y
            for( j=k+1; j<c_blocks; j++ ){
                if( (blockinfo(j,k)!=0) ){
                    magmablas_zgemv( MagmaNoTrans, size_b, size_b, 
                                     mone, A(j,k), size_b,
                                     x(k), 1, one,  x(j), 1 );

                }
            }
        }
    }
    else if( uplo==MagmaUpper ){
        // backward solve
        for( k=r_blocks-1; k>=0; k--){
            // do the backward triangular solve for block M(k,k): U(k,k)x = y
            magma_ztrsv(MagmaUpper, MagmaNoTrans, MagmaNonUnit, size_b, A(k,k), 
                                                             size_b, x(k), 1 );

            // update for all nonzero blocks above M(k,k) 
                    // the respective values of y
            for( j=k-1; j>=0; j-- ){
                if( (blockinfo(j,k)!=0) ){
                    magmablas_zgemv( MagmaNoTrans, size_b, size_b, 
                                     mone, A(j,k), size_b,
                                     x(k), 1, one,  x(j), 1 );

                }
            }
        }
    }

    return MAGMA_SUCCESS;
}



