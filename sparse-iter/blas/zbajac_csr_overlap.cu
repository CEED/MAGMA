#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/
#include "common_magmasparse.h"

#define PRECISION_z
#define BLOCKSIZE 256

__global__ void magma_zk_testLocking(unsigned int* locks, int n) {
    int id = threadIdx.x % n;
    bool leaveLoop = false;
    while (!leaveLoop) {
        if (atomicExch(&(locks[id]), 1u) == 0u) {
            //critical section
            leaveLoop = true;
            atomicExch(&(locks[id]),0u);
        }
    } 
}

/*
__global__ void
magma_zbajac_csr_o_ls_kernel(int localiters, int n, 
                             int matrices, int overlap, 
                             magma_z_matrix *D, magma_z_matrix *R,
                             const magmaDoubleComplex *  __restrict__ b,                            
                             magmaDoubleComplex * x )
{
   // int inddiag =  blockIdx.x*(blockDim.x - overlap) - overlap;
   // int index   =  blockIdx.x*(blockDim.x - overlap) - overlap + threadIdx.x;
        int inddiag =  blockIdx.x*blockDim.x/2-blockDim.x/2;
    int index   = blockIdx.x*blockDim.x/2+threadIdx.x-blockDim.x/2;
    int i, j, start, end;
    
     __shared__ magmaDoubleComplex local_x[ BLOCKSIZE ];
    
    magmaDoubleComplex zero = MAGMA_Z_MAKE(0.0, 0.0);
    magmaDoubleComplex bl, tmp = zero, v = zero; 
    magmaDoubleComplex *valR, *valD;
    magma_index_t *colR, *rowR, *colD, *rowD;
    
    //valR = R[ (1+blockIdx.x-1)%matrices ].dval;
    //colR = R[ (1+blockIdx.x-1)%matrices ].dcol;
    //rowR = R[ (1+blockIdx.x-1)%matrices ].drow;
    //valD = D[ (1+blockIdx.x-1)%matrices ].dval;
    //colD = D[ (1+blockIdx.x-1)%matrices ].dcol;
    //rowD = D[ (1+blockIdx.x-1)%matrices ].drow;
    
        if( blockIdx.x%2==1 ){
        valR = R[0].dval;
        valD = D[0].dval;
        colR = R[0].dcol;
        rowR = R[0].drow;
        colD = D[0].dcol;
        rowD = D[0].drow;
    }else{
        valR = R[1].dval;
        valD = D[1].dval;
        colR = R[1].dcol;
        rowR = R[1].drow;
        colD = D[1].dcol;
        rowD = D[1].drow;
    }

    if ( index>-1 && index < n ) {
        start = rowR[index];
        end   = rowR[index+1];
printf("bdx:%d idx:%d  start:%d  end:%d\n", blockIdx.x, threadIdx.x, start, end);

#if (__CUDA_ARCH__ >= 350) && (defined(PRECISION_d) || defined(PRECISION_s))
        bl = __ldg( b+index );
#else
        bl = b[index];
#endif


     #pragma unroll
     for( i=start; i<end; i++ )
          v += valR[i] * x[ colR[i] ];

     start = rowD[index];
     end   = rowD[index+1];

     #pragma unroll
     for( i=start; i<end; i++ )
         tmp += valD[i] * x[ colD[i] ];

     v =  bl - v;

     // add more local iterations            

     local_x[threadIdx.x] = x[index] ;//+ ( v - tmp);// / (valD[start]);
   __syncthreads();

     #pragma unroll
     for( j=0; j<localiters-1; j++ )
     {
         tmp = zero;
         #pragma unroll
         for( i=start; i<end; i++ )
             tmp += valD[i] * local_x[ colD[i] - inddiag];
     
         local_x[threadIdx.x] +=  ( v - tmp) / (valD[start]);
     }
     if( threadIdx.x > overlap ) { // RAS
         x[index] = local_x[threadIdx.x];
     }
    }   
}

*/
__global__ void
magma_zbajac_csr_o_ls_kernel(int localiters, int n, 
                             int matrices, int overlap, 
                            magmaDoubleComplex * valD1, 
                            magma_index_t * rowD1, 
                            magma_index_t * colD1, 
                            magmaDoubleComplex * valR1, 
                            magma_index_t * rowR1,
                            magma_index_t * colR1, 
                            magmaDoubleComplex * valD2, 
                            magma_index_t * rowD2, 
                            magma_index_t * colD2, 
                            magmaDoubleComplex * valR2, 
                            magma_index_t * rowR2,
                            magma_index_t * colR2, 
                            const magmaDoubleComplex *  __restrict__ b,                            
                            magmaDoubleComplex * x )
{
    int inddiag =  blockIdx.x*blockDim.x/2-blockDim.x/2;
    int index   = blockIdx.x*blockDim.x/2+threadIdx.x-blockDim.x/2;
    int i, j, start, end;
    //bool leaveLoop = false;
    
    magmaDoubleComplex zero = MAGMA_Z_MAKE(0.0, 0.0);
    magmaDoubleComplex bl, tmp = zero, v = zero; 
    magmaDoubleComplex *valR, *valD;
    magma_index_t *colR, *rowR, *colD, *rowD;
    
    if( blockIdx.x%2==1 ){
        valR = valR1;
        valD = valD1;
        colR = colR1;
        rowR = rowR1;
        colD = colD1;
        rowD = rowD1;
    }else{
        valR = valR2; 
        valD = valD2;
        colR = colR2;
        rowR = rowR2;
        colD = colD2;
        rowD = rowD2;
    }

    if ( index>-1 && index < n ) {
        start = rowR[index];
        end   = rowR[index+1];


#if (__CUDA_ARCH__ >= 350) && (defined(PRECISION_d) || defined(PRECISION_s))
        bl = __ldg( b+index );
#else
        bl = b[index];
#endif


        #pragma unroll
        for( i=start; i<end; i++ )
             v += valR[i] * x[ colR[i] ];

        start = rowD[index];
        end   = rowD[index+1];

        #pragma unroll
        for( i=start; i<end; i++ )
            tmp += valD[i] * x[ colD[i] ];

        v =  bl - v;

        // add more local iterations            
        __shared__ magmaDoubleComplex local_x[ BLOCKSIZE ];
        local_x[threadIdx.x] = x[index] + ( v - tmp) / (valD[start]);
        __syncthreads();

        #pragma unroll
        for( j=0; j<localiters-1; j++ )
        {
            tmp = zero;
            #pragma unroll
            for( i=start; i<end; i++ )
                tmp += valD[i] * local_x[ colD[i] - inddiag];
        
            local_x[threadIdx.x] +=  ( v - tmp) / (valD[start]);
        }
        if( threadIdx.x > overlap ) { // only write back the lower subdomain
            x[index] = local_x[threadIdx.x];
        }
    }   
}





__global__ void
magma_zbajac_csr_o_ls_kernel1(int localiters, int n, 
                             int matrices, int overlap, 
                            magmaDoubleComplex * valD1, 
                            magma_index_t * rowD1, 
                            magma_index_t * colD1, 
                            magmaDoubleComplex * valR1, 
                            magma_index_t * rowR1,
                            magma_index_t * colR1, 
                            const magmaDoubleComplex *  __restrict__ b,                            
                            magmaDoubleComplex * x )
{
    int inddiag =  blockIdx.x*blockDim.x;
    int index   = blockIdx.x*blockDim.x+threadIdx.x;
    int i, j, start, end;
    //bool leaveLoop = false;
    
    magmaDoubleComplex zero = MAGMA_Z_MAKE(0.0, 0.0);
    magmaDoubleComplex bl, tmp = zero, v = zero; 
    magmaDoubleComplex *valR, *valD;
    magma_index_t *colR, *rowR, *colD, *rowD;
    
    valR = valR1;
    valD = valD1;
    colR = colR1;
    rowR = rowR1;
    colD = colD1;
    rowD = rowD1;

    if ( index>-1 && index < n ) {
        start = rowR[index];
        end   = rowR[index+1];


#if (__CUDA_ARCH__ >= 350) && (defined(PRECISION_d) || defined(PRECISION_s))
        bl = __ldg( b+index );
#else
        bl = b[index];
#endif


        #pragma unroll
        for( i=start; i<end; i++ )
             v += valR[i] * x[ colR[i] ];

        start = rowD[index];
        end   = rowD[index+1];

        #pragma unroll
        for( i=start; i<end; i++ )
            tmp += valD[i] * x[ colD[i] ];

        v =  bl - v;

        // add more local iterations            
        __shared__ magmaDoubleComplex local_x[ BLOCKSIZE ];
        local_x[threadIdx.x] = x[index] + ( v - tmp) / (valD[start]);
        __syncthreads();

        #pragma unroll
        for( j=0; j<localiters-1; j++ )
        {
            tmp = zero;
            #pragma unroll
            for( i=start; i<end; i++ )
                tmp += valD[i] * local_x[ colD[i] - inddiag];
        
            local_x[threadIdx.x] +=  ( v - tmp) / (valD[start]);
        }
        if( threadIdx.x > overlap ) { // only write back the lower subdomain
            x[index] = local_x[threadIdx.x];
        }
    }   
}





/**
    Purpose
    -------
    
    This routine is a block-asynchronous Jacobi iteration 
    with directed restricted additive Schwarz overlap (top-down) performing s
    local Jacobi-updates within the block. Input format is two CSR matrices,
    one containing the diagonal blocks, one containing the rest.

    Arguments
    ---------

    @param[in]
    localiters  magma_int_t
                number of local Jacobi-like updates

    @param[in]
    D1          magma_z_matrix
                input matrix with diagonal blocks

    @param[in]
    R1          magma_z_matrix
                input matrix with non-diagonal parts
                
    @param[in]
    D2          magma_z_matrix
                input matrix with diagonal blocks

    @param[in]
    R2          magma_z_matrix
                input matrix with non-diagonal parts

    @param[in]
    b           magma_z_matrix
                RHS

    @param[in]
    x           magma_z_matrix*
                iterate/solution

    
    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_zbajac_csr_overlap(
    magma_int_t localiters,
    magma_int_t matrices,
    magma_int_t overlap,
    magma_z_matrix *D,
    magma_z_matrix *R,
    magma_z_matrix b,
    magma_z_matrix *x,
    magma_queue_t queue )
{
    
    
    int blocksize1 = BLOCKSIZE;
    int blocksize2 = 1;
    int size = D[0].num_rows;
    int min_nnz=100;
    

    
    for(int i=0; i<matrices; i++){
       min_nnz = min(min_nnz, R[i].nnz);   
    }
    
    if( min_nnz > 0 ){
        if( matrices == 1 ){
            int dimgrid1 = magma_ceildiv( size  , blocksize1 );
            int dimgrid2 = 1;
            int dimgrid3 = 1;
            dim3 grid( dimgrid1, dimgrid2, dimgrid3 );
            dim3 block( blocksize1, blocksize2, 1 );
            magma_zbajac_csr_o_ls_kernel1<<< grid, block, 0, queue >>>
            ( localiters, size, matrices, overlap,
            D[0].dval, D[0].drow, D[0].dcol, R[0].dval, R[0].drow, R[0].dcol, 
            b.dval, x->dval );  
        } else if (matrices == 2){
            int dimgrid1 = magma_ceildiv( size * blocksize1/(blocksize1-overlap) , blocksize1 );
            int dimgrid2 = 1;
            int dimgrid3 = 1;
            dim3 grid( dimgrid1, dimgrid2, dimgrid3 );
            dim3 block( blocksize1, blocksize2, 1 );
            magma_zbajac_csr_o_ls_kernel<<< grid, block, 0, queue >>>
                ( localiters, size, matrices, overlap,
                    D[0].dval, D[0].drow, D[0].dcol, R[0].dval, R[0].drow, R[0].dcol, 
                    D[1].dval, D[1].drow, D[1].dcol, R[1].dval, R[1].drow, R[1].dcol,
                    b.dval, x->dval );  
        } else
           printf("error: invalid matrix count.\n");
          // magma_zbajac_csr_o_ls_kernel<<< grid, block, 0, queue >>>
            //    ( localiters, size, matrices, overlap, D, R, b.dval, x->dval );


    }
    else {
            printf("error: all elements in diagonal block.\n");
    }
    return MAGMA_SUCCESS;
}
