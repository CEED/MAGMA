#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/
#include "common_magmasparse.h"

#define PRECISION_z


__global__ void 
magma_ziteric_csr_kernel(    
    magma_int_t n, 
    magma_int_t nnz, 
    magma_index_t *Arowidx, 
    magma_index_t *Acolidx, 
    const magmaDoubleComplex * __restrict__  A_val,
    magma_index_t *rowptr, 
    magma_index_t *colidx, 
    magmaDoubleComplex *val )
{
    int i, j;
    int k = (blockDim.x * blockIdx.x + threadIdx.x); // % nnz;


    magmaDoubleComplex zero = MAGMA_Z_MAKE(0.0, 0.0);
    magmaDoubleComplex s, sp;
    int il, iu, jl, ju;

    if ( k < nnz )
    {     
        i = Arowidx[k];
        j = Acolidx[k];

#if (__CUDA_ARCH__ >= 350) && (defined(PRECISION_d) || defined(PRECISION_s))
        s = __ldg( A_val+k );
#else
        s = A_val[k];
#endif

        il = rowptr[i];
        iu = rowptr[j];

        while (il < rowptr[i+1] && iu < rowptr[j+1])
        {
            sp = zero;
            jl = colidx[il];
            ju = colidx[iu];

            if (jl < ju)
                il++;
            else if (ju < jl)
                iu++;
            else
            {
                // we are going to modify this u entry
                sp = val[il] * val[iu];
                s -= sp;
                il++;
                iu++;
            }
        }
        // undo the last operation (it must be the last)
        s += sp;
        __syncthreads();

        // modify entry
        if (i == j)
            val[il-1] = MAGMA_Z_MAKE(sqrt(abs(MAGMA_Z_REAL(s))), 0.0);
        else
            val[il-1] =  s / val[iu-1];
    }
}// kernel 










/**
    Purpose
    -------
    
    This routine iteratively computes an incomplete Cholesky factorization.
    The idea is according to Edmond Chow's presentation at SIAM 2014.
    This routine was used in the ISC 2015 paper:
    E. Chow et al.: 'Study of an Asynchronous Iterative Algorithm
                     for Computing Incomplete Factorizations on GPUs'
                     
    The input format of the initial guess matrix A is Magma_CSRCOO,
    A_CSR is CSR or CSRCOO format. 

    Arguments
    ---------

    @param[in]
    A           magma_z_matrix
                input matrix A - initial guess (lower triangular)

    @param[in,out]
    A_CSR       magma_z_matrix
                input/output matrix containing the IC approximation
                
    @param[in]
    queue       magma_queue_t
                Queue to execute in.
                
    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" magma_int_t
magma_ziteric_csr( 
    magma_z_matrix A,
    magma_z_matrix A_CSR,
    magma_queue_t queue )
{
    int blocksize1 = 128;
    int blocksize2 = 1;

    int dimgrid1 = magma_ceildiv( A.nnz, blocksize1 );
    int dimgrid2 = 1;
    int dimgrid3 = 1;

    // Runtime API
    // hipFuncCachePreferShared: shared memory is 48 KB
    // hipFuncCachePreferEqual: shared memory is 32 KB
    // hipFuncCachePreferL1: shared memory is 16 KB
    // hipFuncCachePreferNone: no preference
    //hipFuncSetCacheConfig(hipFuncCachePreferShared);

    hipDeviceSetCacheConfig( hipFuncCachePreferL1 );

    dim3 grid( dimgrid1, dimgrid2, dimgrid3 );
    dim3 block( blocksize1, blocksize2, 1 );
    magma_ziteric_csr_kernel<<< grid, block, 0, magma_stream >>>
            ( A.num_rows, A.nnz, 
              A.rowidx, A.col, A.val, 
              A_CSR.row, A_CSR.col,  A_CSR.val );

    return MAGMA_SUCCESS;
}
