#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s
       @author Hartwig Anzt

*/
#include "common_magma.h"

#define BLOCK_SIZE 512

#define PRECISION_z


// These routines merge multiple kernels from tfqmr into one.

/* -------------------------------------------------------------------------- */

__global__ void
magma_ztfqmr_1_kernel(  
    int num_rows, 
    int num_cols, 
    magmaDoubleComplex alpha,
    magmaDoubleComplex sigma,
    magmaDoubleComplex *v, 
    magmaDoubleComplex *Au,
    magmaDoubleComplex *u_m,
    magmaDoubleComplex *pu_m,
    magmaDoubleComplex *u_mp1,
    magmaDoubleComplex *w, 
    magmaDoubleComplex *d,
    magmaDoubleComplex *Ad )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            
            u_mp1[ i+j*num_rows ] = u_m[ i+j*num_rows ] - alpha * v[ i+j*num_rows ];
            w[ i+j*num_rows ] = w[ i+j*num_rows ] - alpha * Au[ i+j*num_rows ];
            d[ i+j*num_rows ] = pu_m[ i+j*num_rows ] + sigma * d[ i+j*num_rows ];
            Ad[ i+j*num_rows ] = Au[ i+j*num_rows ] + sigma * Ad[ i+j*num_rows ];
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    u_mp1 = u_mp1 - alpha*v;
    w = w - alpha*Au;
    d = pu_m + sigma*d;
    Ad = Au + sigma*Ad;
    
    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    alpha       magmaDoubleComplex
                scalar
                
    @param[in]
    sigma       magmaDoubleComplex
                scalar
                
    @param[in]
    v           magmaDoubleComplex_ptr 
                vector
                
    @param[in]
    Au          magmaDoubleComplex_ptr 
                vector
                
    @param[in,out]
    u_m         magmaDoubleComplex_ptr 
                vector
                
    @param[in,out]
    pu_m         magmaDoubleComplex_ptr 
                vector
                
    @param[in,out]
    u_mp1       magmaDoubleComplex_ptr 
                vector

    @param[in,out]
    w           magmaDoubleComplex_ptr 
                vector
                
    @param[in,out]
    d           magmaDoubleComplex_ptr 
                vector
                
    @param[in,out]
    Ad          magmaDoubleComplex_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_ztfqmr_1(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaDoubleComplex alpha,
    magmaDoubleComplex sigma,
    magmaDoubleComplex_ptr v, 
    magmaDoubleComplex_ptr Au,
    magmaDoubleComplex_ptr u_m,
    magmaDoubleComplex_ptr pu_m,
    magmaDoubleComplex_ptr u_mp1,
    magmaDoubleComplex_ptr w, 
    magmaDoubleComplex_ptr d,
    magmaDoubleComplex_ptr Ad,
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_ztfqmr_1_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, alpha, sigma,
                     v, Au, u_m, pu_m, u_mp1, w, d, Ad );

   return MAGMA_SUCCESS;
}





__global__ void
magma_ztfqmr_2_kernel(  
    int num_rows,
    int num_cols,
    magmaDoubleComplex eta,
    magmaDoubleComplex_ptr d,
    magmaDoubleComplex_ptr Ad,
    magmaDoubleComplex_ptr x, 
    magmaDoubleComplex_ptr r )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            x[ i+j*num_rows ] = x[ i+j*num_rows ] + eta * d[ i+j*num_rows ];
            r[ i+j*num_rows ] = r[ i+j*num_rows ] - eta * Ad[ i+j*num_rows ];
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    x = x + eta * d
    r = r - eta * Ad

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    eta         magmaDoubleComplex
                scalar
                
    @param[in]
    d           magmaDoubleComplex_ptr 
                vector
                
    @param[in]
    Ad          magmaDoubleComplex_ptr 
                vector

    @param[in,out]
    x           magmaDoubleComplex_ptr 
                vector
                
    @param[in,out]
    r           magmaDoubleComplex_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_ztfqmr_2(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaDoubleComplex eta,
    magmaDoubleComplex_ptr d,
    magmaDoubleComplex_ptr Ad,
    magmaDoubleComplex_ptr x, 
    magmaDoubleComplex_ptr r, 
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_ztfqmr_2_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, eta, d, Ad, x, r );

   return MAGMA_SUCCESS;
}





__global__ void
magma_ztfqmr_3_kernel(  
    int num_rows,
    int num_cols,
    magmaDoubleComplex beta,
    magmaDoubleComplex *w,
    magmaDoubleComplex *u_m,
    magmaDoubleComplex *u_mp1 )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            u_mp1[ i+j*num_rows ] = w[ i+j*num_rows ] + beta * u_m[ i+j*num_rows ];
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    u_mp1 = w + beta*u_mp1

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    beta        magmaDoubleComplex
                scalar
                
    @param[in]
    w           magmaDoubleComplex_ptr 
                vector
                
    @param[in]
    u_m         magmaDoubleComplex_ptr 
                vector

    @param[in,out]
    u_mp1       magmaDoubleComplex_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_ztfqmr_3(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaDoubleComplex beta,
    magmaDoubleComplex_ptr w,
    magmaDoubleComplex_ptr u_m,
    magmaDoubleComplex_ptr u_mp1, 
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_ztfqmr_3_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, beta, w, u_m, u_mp1 );

   return MAGMA_SUCCESS;
}




__global__ void
magma_ztfqmr_4_kernel(  
    int num_rows,
    int num_cols,
    magmaDoubleComplex beta,
    magmaDoubleComplex *Au_new,
    magmaDoubleComplex *v,
    magmaDoubleComplex *Au )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            magmaDoubleComplex tmp = Au_new[ i+j*num_rows ];
                v[ i+j*num_rows ] = tmp + beta * Au[ i+j*num_rows ] 
                                    + beta * beta * v[ i+j*num_rows ];
                Au[ i+j*num_rows ] = tmp; 
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    v = Au_new + beta*(Au+beta*v);
    Au = Au_new

    Arguments
    ---------

    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    beta        magmaDoubleComplex
                scalar
                
    @param[in]
    Au_new      magmaDoubleComplex_ptr 
                vector

    @param[in,out]
    v           magmaDoubleComplex_ptr 
                vector
                
    @param[in,out]
    Au          magmaDoubleComplex_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_ztfqmr_4(  
    magma_int_t num_rows, 
    magma_int_t num_cols, 
    magmaDoubleComplex beta,
    magmaDoubleComplex_ptr Au_new,
    magmaDoubleComplex_ptr v,
    magmaDoubleComplex_ptr Au, 
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_ztfqmr_4_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, beta, Au_new, v, Au );

   return MAGMA_SUCCESS;
}


__global__ void
magma_ztfqmr_5_kernel(  
    int num_rows,                   
    int num_cols, 
    magmaDoubleComplex alpha,
    magmaDoubleComplex sigma,
    magmaDoubleComplex *v, 
    magmaDoubleComplex *Au,
    magmaDoubleComplex *u_mp1,
    magmaDoubleComplex *w, 
    magmaDoubleComplex *d,
    magmaDoubleComplex *Ad )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i<num_rows ) {
        for( int j=0; j<num_cols; j++ ){
            w[ i+j*num_rows ] = w[ i+j*num_rows ] - alpha * Au[ i+j*num_rows ];
            d[ i+j*num_rows ] = u_mp1[ i+j*num_rows ] + sigma * d[ i+j*num_rows ];
            Ad[ i+j*num_rows ] = Au[ i+j*num_rows ] + sigma * Ad[ i+j*num_rows ];
        }
    }
}

/**
    Purpose
    -------

    Mergels multiple operations into one kernel:

    w = w - alpha*Au;
    d = pu_m + sigma*d;
    Ad = Au + sigma*Ad;
    
    @param[in]
    num_rows    magma_int_t
                dimension m
                
    @param[in]
    num_cols    magma_int_t
                dimension n
                
    @param[in]
    alpha       magmaDoubleComplex
                scalar
                
    @param[in]
    sigma       magmaDoubleComplex
                scalar
                
    @param[in]
    v           magmaDoubleComplex_ptr 
                vector
                
    @param[in]
    Au          magmaDoubleComplex_ptr 
                vector
                
    @param[in,out]
    u_mp1       magmaDoubleComplex_ptr 
                vector

    @param[in,out]
    w           magmaDoubleComplex_ptr 
                vector
                
    @param[in,out]
    d           magmaDoubleComplex_ptr 
                vector
                
    @param[in,out]
    Ad          magmaDoubleComplex_ptr 
                vector

    @param[in]
    queue       magma_queue_t
                Queue to execute in.

    @ingroup magmasparse_zgegpuk
    ********************************************************************/

extern "C" 
magma_int_t
magma_ztfqmr_5(  
    magma_int_t num_rows,               
    magma_int_t num_cols, 
    magmaDoubleComplex alpha,
    magmaDoubleComplex sigma,
    magmaDoubleComplex_ptr v, 
    magmaDoubleComplex_ptr Au,
    magmaDoubleComplex_ptr u_mp1,
    magmaDoubleComplex_ptr w, 
    magmaDoubleComplex_ptr d,
    magmaDoubleComplex_ptr Ad,
    magma_queue_t queue )
{
    dim3 Bs( BLOCK_SIZE );
    dim3 Gs( magma_ceildiv( num_rows, BLOCK_SIZE ) );
    magma_ztfqmr_5_kernel<<< Gs, Bs, 0, queue->hip_stream() >>>( num_rows, num_cols, alpha, sigma,
                     v, Au, u_mp1, w, d, Ad );

   return MAGMA_SUCCESS;
}

