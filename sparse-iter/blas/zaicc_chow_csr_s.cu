#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.1) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       @date

       @precisions normal z -> c d s

*/

#include "common_magma.h"
#include "../include/magmasparse_z.h"
#include "../../include/magma.h"


// includes CUDA
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hipsparse.h>
#include "sm_32_intrinsics.h"

// 512 is maximum number of threads for CUDA capability 1.x
#define BLOCK_SIZE  64


#define PRECISION_z



// every row is handled by one threadblock
__global__ void 
magma_zaic_csr_s_kernel( magma_int_t num_rows, 
                         magma_int_t nnz,
                         const magmaDoubleComplex * __restrict__ A_val, 
                         magmaDoubleComplex *val,
                         magma_index_t *rowptr,
                         magma_index_t *rowidx, 
                         magma_index_t *colidx ){

    int i, j;
    int k = (blockDim.x * blockIdx.x + threadIdx.x);// % nnz;
    magmaDoubleComplex zero = MAGMA_Z_MAKE(0.0, 0.0);
    magmaDoubleComplex s, sp;
    int il, iu, jl, ju;

    if (k < nnz)
    {     
        i = rowidx[k];
        j = colidx[k];

#if (__CUDA_ARCH__ >= 350) && (defined(PRECISION_d) || defined(PRECISION_s))
        s = __ldg( A_val+k );
#else
        s = A_val[k];
#endif

        il = rowptr[i];
        iu = rowptr[j];

        while (il < rowptr[i+1] && iu < rowptr[j+1])
        {
            sp = zero;
            jl = colidx[il];
            ju = colidx[iu];

            // avoid branching
            sp = ( jl == ju ) ? val[il] * val[iu] : sp;
            s = ( jl == ju ) ? s-sp : s;
            il = ( jl <= ju ) ? il+1 : il;
            iu = ( jl >= ju ) ? iu+1 : iu;
/*
            if (jl < ju)
                il++;
            else if (ju < jl)
                iu++;
            else
            {
                // we are going to modify this u entry
                sp = val[il] * val[iu];
                s -= sp;
                il++;
                iu++;
            }
*/
        }
        // undo the last operation (it must be the last)
        s += sp;
        __syncthreads();

        // modify entry
        if (i == j)
            val[k] = MAGMA_Z_MAKE(sqrt(abs(MAGMA_Z_REAL(s))), 0.0);
        else
            val[k] =  s / val[rowptr[j+1]-1];
    }

}// kernel 










/**
    Purpose
    -------
    
    This routine computes the IC approximation of a matrix iteratively. 
    The idea is according to Edmond Chow's presentation at SIAM 2014.
    The input format of the matrix is Magma_CSRCOO. 

    Arguments
    ---------

    @param
    A           magma_z_sparse_matrix
                input matrix A - initial guess (lower triangular)

    @param
    A_CSR       magma_z_sparse_matrix
                input/output matrix containing the IC approximation

    @ingroup magmasparse_zsygpuk
    ********************************************************************/

extern "C" magma_int_t
magma_zaic_csr_s( magma_z_sparse_matrix A,
                 magma_z_sparse_matrix A_CSR ){



    
    int blocksize1 = 256;
    int blocksize2 = 1;

    int dimgrid1 = ( A.nnz + blocksize1 -1 ) / blocksize1;
    int dimgrid2 = 1;
    int dimgrid3 = 1;

    dim3 grid( dimgrid1, dimgrid2, dimgrid3 );
    dim3 block( blocksize1, blocksize2, 1 );
    magma_zaic_csr_s_kernel<<< grid, block, 0, magma_stream >>>
            ( A.num_rows, A.nnz,  A.val, A_CSR.val, A_CSR.row, 
                                    A_CSR.rowidx,  A_CSR.col );

    return MAGMA_SUCCESS;
}



